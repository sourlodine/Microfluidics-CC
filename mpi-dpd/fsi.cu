#include "hip/hip_runtime.h"
/*
 *  rbc-interactions.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-02.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <../dpd-rng.h>

#include "common-kernels.h"
#include "fsi.h"

namespace KernelsFSI
{
    struct Params { float aij, gamma, sigmaf; };

    __constant__ Params params;
}

ComputeFSI::ComputeFSI(MPI_Comm comm)
{
    int myrank;
    MPI_CHECK( MPI_Comm_rank(comm, &myrank));

    local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);

    //TODO: use CUDA_CHECK(hipEventCreateWithFlags(&evuploaded, hipEventDisableTiming));

    KernelsFSI::Params params = {12.5 , gammadpd, sigmaf};

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsFSI::params), &params, sizeof(params)));

    CUDA_CHECK(hipPeekAtLastError());
}

namespace KernelsFSI
{
    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    bool firsttime = true;

    static const int NCELLS = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

    __global__  __launch_bounds__(128, 10)
	void interactions_3tpp(const float2 * const particles, const int np, const int nsolvent,
			       float * const acc, float * const accsolvent, const float seed)
    {
#if !defined(__CUDA_ARCH__)
#warning __CUDA_ARCH__ not defined! assuming 350
#define _ACCESS(x) __ldg(x)
#elif __CUDA_ARCH__ >= 350
#define _ACCESS(x) __ldg(x)
#else
#define _ACCESS(x) (*(x))
#endif

	assert(blockDim.x * gridDim.x >= np * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
       	const int pid = gid / 3;
	const int zplane = gid % 3;

	if (pid >= np)
	    return;

	const float2 dst0 = _ACCESS(particles + 3 * pid + 0);
	const float2 dst1 = _ACCESS(particles + 3 * pid + 1);
	const float2 dst2 = _ACCESS(particles + 3 * pid + 2);

	int scan1, scan2, ncandidates, spidbase;
	int deltaspid1, deltaspid2;

	{
	    enum
	    {
		XCELLS = XSIZE_SUBDOMAIN,
		YCELLS = YSIZE_SUBDOMAIN,
		ZCELLS = ZSIZE_SUBDOMAIN,
		XOFFSET = XCELLS / 2,
		YOFFSET = YCELLS / 2,
		ZOFFSET = ZCELLS / 2
	    };

	    const int xcenter = XOFFSET + (int)floorf(dst0.x);
	    const int xstart = max(0, xcenter - 1);
	    const int xcount = min(XCELLS, xcenter + 2) - xstart;

	    if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
		return;

	    assert(xcount >= 0);

	    const int ycenter = YOFFSET + (int)floorf(dst0.y);

	    const int zcenter = ZOFFSET + (int)floorf(dst1.x);
	    const int zmy = zcenter - 1 + zplane;
	    const bool zvalid = zmy >= 0 && zmy < ZCELLS;

	    int count0 = 0, count1 = 0, count2 = 0;

	    if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
	    {
		const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
		assert(cid0 >= 0 && cid0 + xcount <= NCELLS);
		spidbase = tex1Dfetch(texCellsStart, cid0);
		count0 = ((cid0 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid0 + xcount)) - spidbase;
	    }

	    if (zvalid && ycenter >= 0 && ycenter < YCELLS)
	    {
		const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
		assert(cid1 >= 0 && cid1 + xcount <= NCELLS);
		deltaspid1 = tex1Dfetch(texCellsStart, cid1);
		count1 = ((cid1 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid1 + xcount)) - deltaspid1;
	    }

	    if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
	    {
		const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
		deltaspid2 = tex1Dfetch(texCellsStart, cid2);
		assert(cid2 >= 0 && cid2 + xcount <= NCELLS);
		count2 = ((cid2 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid2 + xcount)) - deltaspid2;
	    }

	    scan1 = count0;
	    scan2 = count0 + count1;
	    ncandidates = scan2 + count2;

	    deltaspid1 -= scan1;
	    deltaspid2 -= scan2;
	}

	float xforce = 0, yforce = 0, zforce = 0;

#pragma unroll 3
	for(int i = 0; i < ncandidates; ++i)
	{
	    const int m1 = (int)(i >= scan1);
	    const int m2 = (int)(i >= scan2);
	    const int spid = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);

	    assert(spid >= 0 && spid < nsolvent);

	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry    );
	    const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
	    const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);

	    const float _xr = dst0.x - stmp0.x;
	    const float _yr = dst0.y - stmp0.y;
	    const float _zr = dst1.x - stmp1.x;

	    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

	    if (!(rij2 > 0))
		printf("oopsa rij2 %f : src = %f %f %f  dst= %f %f %f\n", rij2, stmp0.x, stmp0.y, stmp1.x, dst0.x, dst0.y, dst1.x);
	    assert(rij2 > 0);
	    
	    const float invrij = rsqrtf(rij2);

	    const float rij = rij2 * invrij;
	    
	    if (rij2 >= 1)
		continue;

	    const float argwr = 1.f - rij;
	    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

	    const float xr = _xr * invrij;
	    const float yr = _yr * invrij;
	    const float zr = _zr * invrij;

	    const float rdotv =
		xr * (dst1.y - stmp1.y) +
		yr * (dst2.x - stmp2.x) +
		zr * (dst2.y - stmp2.y);

	    const float myrandnr = Logistic::mean0var1(seed, pid, spid);

	    const float strength = params.aij * argwr + (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

	    const float xinteraction = strength * xr;
	    const float yinteraction = strength * yr;
	    const float zinteraction = strength * zr;

	    xforce += xinteraction;
	    yforce += yinteraction;
	    zforce += zinteraction;

	    assert(!isnan(xinteraction));
	    assert(!isnan(yinteraction));
	    assert(!isnan(zinteraction));
	    assert(fabs(xinteraction) < 1e4);
	    assert(fabs(yinteraction) < 1e4);
	    assert(fabs(zinteraction) < 1e4);

	    atomicAdd(accsolvent + sentry    , -xinteraction);
	    atomicAdd(accsolvent + sentry + 1, -yinteraction);
	    atomicAdd(accsolvent + sentry + 2, -zinteraction);
	}

	atomicAdd(acc + 3 * pid + 0, xforce);
	atomicAdd(acc + 3 * pid + 1, yforce);
	atomicAdd(acc + 3 * pid + 2, zforce);

	for(int c = 0; c < 3; ++c)
	    assert(!isnan(acc[3 * pid + c]));
    }

    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount)
    {
	if (firsttime)
	{
	    texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	    texCellsStart.filterMode = hipFilterModePoint;
	    texCellsStart.mipmapFilterMode = hipFilterModePoint;
	    texCellsStart.normalized = 0;

	    texCellsCount.channelDesc = hipCreateChannelDesc<int>();
	    texCellsCount.filterMode = hipFilterModePoint;
	    texCellsCount.mipmapFilterMode = hipFilterModePoint;
	    texCellsCount.normalized = 0;

	    texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
	    texSolventParticles.filterMode = hipFilterModePoint;
	    texSolventParticles.mipmapFilterMode = hipFilterModePoint;
	    texSolventParticles.normalized = 0;

	    CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(interactions_3tpp), hipFuncCachePreferL1));

	    firsttime = false;
	}

	size_t textureoffset = 0;

	if (npsolvent)
	{
	    CUDA_CHECK(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
				       sizeof(float) * 6 * npsolvent));
	    assert(textureoffset == 0);
	}

	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);

	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
    }
}

void ComputeFSI::bulk(std::vector<ParticlesWrap> wsolutes, hipStream_t stream)
{
    NVTX_RANGE("FSI/bulk", NVTX_C6);

    if (wsolutes.size() == 0)
	return;

    KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart, wsolvent.cellscount);

    CUDA_CHECK(hipPeekAtLastError());

    for(std::vector<ParticlesWrap>::iterator it = wsolutes.begin(); it != wsolutes.end(); ++it)
   	if (it->n)
	    KernelsFSI::interactions_3tpp<<< (3 * it->n + 127) / 128, 128, 0, stream >>>
		((float2 *)it->p, it->n, wsolvent.n, (float *)it->a, (float *)wsolvent.a, local_trunk.get_float());

    CUDA_CHECK(hipPeekAtLastError());
}
/*
namespace KernelsFSI
{
    __constant__ int packstarts_padded[27], packcount[26];
    __constant__ Particle * packstates[26];
    __constant__ Acceleration * packresults[26];

    __global__ 	void interactions_halo(const int nparticles_padded, const int nsolvent, float * const accsolvent, const float seed)
    {
	assert(blockDim.x * gridDim.x >= nparticles_padded);

	const int laneid = threadIdx.x & 0x1f;
	const int warpid = threadIdx.x >> 5;
	const int localbase = 32 * (warpid + 4 * blockIdx.x);
	const int pid = localbase + laneid;

	if (localbase >= nparticles_padded)
	    return;

	int nunpack;
	float2 dst0, dst1, dst2;
	float * dst = NULL;

	{
	    const uint key9 = 9 * (localbase >= packstarts_padded[9]) + 9 * (localbase >= packstarts_padded[18]);
	    const uint key3 = 3 * (localbase >= packstarts_padded[key9 + 3]) + 3 * (localbase >= packstarts_padded[key9 + 6]);
	    const uint key1 = (localbase >= packstarts_padded[key9 + key3 + 1]) + (localbase >= packstarts_padded[key9 + key3 + 2]);
	    const int code = key9 + key3 + key1;
	    assert(code >= 0 && code < 26);
	    assert(localbase >= packstarts_padded[code] && localbase < packstarts_padded[code + 1]);

	    const int unpackbase = localbase - packstarts_padded[code];
	    assert (unpackbase >= 0);
	    assert(unpackbase < packcount[code]);

	    nunpack = min(32, packcount[code] - unpackbase);

	    if (nunpack == 0)
		return;

	    read_AOS6f((float2 *)(packstates[code] + unpackbase), nunpack, dst0, dst1, dst2);

	    dst = (float*)(packresults[code] + unpackbase);
	}

	float xforce = 0, yforce = 0, zforce = 0;

	const int nzplanes = laneid < nunpack ? 3 : 0;

	for(int zplane = 0; zplane < nzplanes; ++zplane)
	{
	    int scan1, scan2, ncandidates, spidbase;
	    int deltaspid1, deltaspid2;

	    {
		enum
		{
		    XCELLS = XSIZE_SUBDOMAIN,
		    YCELLS = YSIZE_SUBDOMAIN,
		    ZCELLS = ZSIZE_SUBDOMAIN,
		    XOFFSET = XCELLS / 2,
		    YOFFSET = YCELLS / 2,
		    ZOFFSET = ZCELLS / 2
		};

		const int NCELLS = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;
		const int xcenter = XOFFSET + (int)floorf(dst0.x);
		const int xstart = max(0, xcenter - 1);
		const int xcount = min(XCELLS, xcenter + 2) - xstart;

		if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
		    continue;

		assert(xcount >= 0);

		const int ycenter = YOFFSET + (int)floorf(dst0.y);

		const int zcenter = ZOFFSET + (int)floorf(dst1.x);
		const int zmy = zcenter - 1 + zplane;
		const bool zvalid = zmy >= 0 && zmy < ZCELLS;

		int count0 = 0, count1 = 0, count2 = 0;

		if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
		{
		    const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
		    assert(cid0 >= 0 && cid0 + xcount <= NCELLS);
		    spidbase = tex1Dfetch(texCellsStart, cid0);
		    count0 = ((cid0 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid0 + xcount)) - spidbase;
		}

		if (zvalid && ycenter >= 0 && ycenter < YCELLS)
		{
		    const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
		    assert(cid1 >= 0 && cid1 + xcount <= NCELLS);
		    deltaspid1 = tex1Dfetch(texCellsStart, cid1);
		    count1 = ((cid1 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid1 + xcount)) - deltaspid1;
		}

		if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
		{
		    const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
		    deltaspid2 = tex1Dfetch(texCellsStart, cid2);
		    assert(cid2 >= 0 && cid2 + xcount <= NCELLS);
		    count2 = ((cid2 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid2 + xcount)) - deltaspid2;
		}

		scan1 = count0;
		scan2 = count0 + count1;
		ncandidates = scan2 + count2;

		deltaspid1 -= scan1;
		deltaspid2 -= scan2;
	    }

	    for(int i = 0; i < ncandidates; ++i)
	    {
		const int m1 = (int)(i >= scan1);
		const int m2 = (int)(i >= scan2);
		const int spid = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);

		assert(spid >= 0 && spid < nsolvent);

		const int sentry = 3 * spid;
		const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry    );
		const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
		const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);

		const float _xr = dst0.x - stmp0.x;
		const float _yr = dst0.y - stmp0.y;
		const float _zr = dst1.x - stmp1.x;

		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

		const float invrij = rsqrtf(rij2);

		const float rij = rij2 * invrij;

		if (rij2 >= 1)
		    continue;

		const float argwr = 1.f - rij;
		const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv =
		    xr * (dst1.y - stmp1.y) +
		    yr * (dst2.x - stmp2.x) +
		    zr * (dst2.y - stmp2.y);

		const float myrandnr = Logistic::mean0var1(seed, pid, spid);

		const float strength = params.aij * argwr + (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

		const float xinteraction = strength * xr;
		const float yinteraction = strength * yr;
		const float zinteraction = strength * zr;

		xforce += xinteraction;
		yforce += yinteraction;
		zforce += zinteraction;

		assert(!isnan(xinteraction));
		assert(!isnan(yinteraction));
		assert(!isnan(zinteraction));
		assert(fabs(xinteraction) < 1e4);
		assert(fabs(yinteraction) < 1e4);
		assert(fabs(zinteraction) < 1e4);

		atomicAdd(accsolvent + sentry    , -xinteraction);
		atomicAdd(accsolvent + sentry + 1, -yinteraction);
		atomicAdd(accsolvent + sentry + 2, -zinteraction);
	    }
	}

	write_AOS3f(dst, nunpack, xforce, yforce, zforce);
    }
    }*/

void ComputeFSI::halo(ParticlesWrap halowrap, hipStream_t stream)
{
    NVTX_RANGE("FSI/halo", NVTX_C7);

    KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart, wsolvent.cellscount);

    CUDA_CHECK(hipPeekAtLastError());
/*
    int nremote_padded = 0;

    {
	int recvpackcount[26], recvpackstarts_padded[27];

	for(int i = 0; i < 26; ++i)
	    recvpackcount[i] = halos[i].n;

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packcount), recvpackcount,
					   sizeof(recvpackcount), 0, hipMemcpyHostToDevice, stream));

	recvpackstarts_padded[0] = 0;
	for(int i = 0, s = 0; i < 26; ++i)
	    recvpackstarts_padded[i + 1] = (s += 32 * ((halos[i].n + 31) / 32));

	nremote_padded = recvpackstarts_padded[26];

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packstarts_padded), recvpackstarts_padded,
					   sizeof(recvpackstarts_padded), 0, hipMemcpyHostToDevice, stream));
    }

    {
	const Particle * recvpackstates[26];

	for(int i = 0; i < 26; ++i)
	    recvpackstates[i] = halos[i].p;

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packstates), recvpackstates,
					   sizeof(recvpackstates), 0, hipMemcpyHostToDevice, stream));
    }

    {
	Acceleration * packresults[26];

	for(int i = 0; i < 26; ++i)
	    packresults[i] = halos[i].a;

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packresults), packresults,
					   sizeof(packresults), 0, hipMemcpyHostToDevice, stream));
    }

    if(nremote_padded)
    	KernelsFSI::interactions_halo<<< (nremote_padded + 127) / 128, 128, 0, stream>>>
	    (nremote_padded, wsolvent.n, (float *)wsolvent.a, local_trunk.get_float());
*/
    //printf("before halo fsi\n");
    
    if (halowrap.n)
	KernelsFSI::interactions_3tpp<<< (3 * halowrap.n + 127) / 128, 128, 0, stream >>>
	    ((float2 *)halowrap.p, halowrap.n, wsolvent.n, (float *)halowrap.a, (float *)wsolvent.a, local_trunk.get_float());

    /*CUDA_CHECK(hipDeviceSynchronize());
      printf("after halo fsi\n");*/
    CUDA_CHECK(hipPeekAtLastError());
}
