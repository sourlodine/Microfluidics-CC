#include "hip/hip_runtime.h"
/*
 *  rbc-interactions.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-02.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <../dpd-rng.h>

#include "common-kernels.h"
#include "fsi.h"

namespace KernelsFSI
{
    struct Params { float aij, gamma, sigmaf; };

    __constant__ Params params;
}

ComputeFSI::ComputeFSI(MPI_Comm comm)
{
    int myrank;
    MPI_CHECK( MPI_Comm_rank(comm, &myrank));

    local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);

    //TODO: use CUDA_CHECK(hipEventCreateWithFlags(&evuploaded, hipEventDisableTiming));

    KernelsFSI::Params params = {0.0f, gammadpd, sigmaf};

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsFSI::params), &params, sizeof(params)));

    CUDA_CHECK(hipPeekAtLastError());
}

namespace KernelsFSI
{
    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    bool firsttime = true;

    static const int NCELLS = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

    __global__  __launch_bounds__(128, 10)
	void interactions_3tpp(const float2 * const particles, const int np, const int nsolvent,
			       float * const acc, float * const accsolvent, const float seed)
    {
#if !defined(__CUDA_ARCH__)
#warning __CUDA_ARCH__ not defined! assuming 350
#define _ACCESS(x) __ldg(x)
#elif __CUDA_ARCH__ >= 350
#define _ACCESS(x) __ldg(x)
#else
#define _ACCESS(x) (*(x))
#endif

	assert(blockDim.x * gridDim.x >= np * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
       	const int pid = gid / 3;
	const int zplane = gid % 3;

	if (pid >= np)
	    return;

	const float2 dst0 = _ACCESS(particles + 3 * pid + 0);
	const float2 dst1 = _ACCESS(particles + 3 * pid + 1);
	const float2 dst2 = _ACCESS(particles + 3 * pid + 2);

	int scan1, scan2, ncandidates, spidbase;
	int deltaspid1, deltaspid2;

	{
	    enum
	    {
		XCELLS = XSIZE_SUBDOMAIN,
		YCELLS = YSIZE_SUBDOMAIN,
		ZCELLS = ZSIZE_SUBDOMAIN,
		XOFFSET = XCELLS / 2,
		YOFFSET = YCELLS / 2,
		ZOFFSET = ZCELLS / 2
	    };

	    const int xcenter = XOFFSET + (int)floorf(dst0.x);
	    const int xstart = max(0, xcenter - 1);
	    const int xcount = min(XCELLS, xcenter + 2) - xstart;

	    if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
		return;

	    assert(xcount >= 0);

	    const int ycenter = YOFFSET + (int)floorf(dst0.y);

	    const int zcenter = ZOFFSET + (int)floorf(dst1.x);
	    const int zmy = zcenter - 1 + zplane;
	    const bool zvalid = zmy >= 0 && zmy < ZCELLS;

	    int count0 = 0, count1 = 0, count2 = 0;

	    if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
	    {
		const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
		assert(cid0 >= 0 && cid0 + xcount <= NCELLS);
		spidbase = tex1Dfetch(texCellsStart, cid0);
		count0 = ((cid0 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid0 + xcount)) - spidbase;
	    }

	    if (zvalid && ycenter >= 0 && ycenter < YCELLS)
	    {
		const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
		assert(cid1 >= 0 && cid1 + xcount <= NCELLS);
		deltaspid1 = tex1Dfetch(texCellsStart, cid1);
		count1 = ((cid1 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid1 + xcount)) - deltaspid1;
	    }

	    if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
	    {
		const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
		deltaspid2 = tex1Dfetch(texCellsStart, cid2);
		assert(cid2 >= 0 && cid2 + xcount <= NCELLS);
		count2 = ((cid2 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid2 + xcount)) - deltaspid2;
	    }

	    scan1 = count0;
	    scan2 = count0 + count1;
	    ncandidates = scan2 + count2;

	    deltaspid1 -= scan1;
	    deltaspid2 -= scan2;
	}

	float xforce = 0, yforce = 0, zforce = 0;

#pragma unroll 3
	for(int i = 0; i < ncandidates; ++i)
	{
	    const int m1 = (int)(i >= scan1);
	    const int m2 = (int)(i >= scan2);
	    const int spid = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);

	    assert(spid >= 0 && spid < nsolvent);

	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry    );
	    const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
	    const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);

	    const float _xr = dst0.x - stmp0.x;
	    const float _yr = dst0.y - stmp0.y;
	    const float _zr = dst1.x - stmp1.x;

	    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
	    assert(rij2 > 0);

	    const float invrij = rsqrtf(rij2);

	    const float rij = rij2 * invrij;

	    if (rij2 >= 1)
		continue;

	    const float argwr = 1.f - rij;
	    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

	    const float xr = _xr * invrij;
	    const float yr = _yr * invrij;
	    const float zr = _zr * invrij;

	    const float rdotv =
		xr * (dst1.y - stmp1.y) +
		yr * (dst2.x - stmp2.x) +
		zr * (dst2.y - stmp2.y);

	    const float myrandnr = Logistic::mean0var1(seed, pid, spid);

	    const float strength = params.aij * argwr + (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

	    const float xinteraction = strength * xr;
	    const float yinteraction = strength * yr;
	    const float zinteraction = strength * zr;

	    xforce += xinteraction;
	    yforce += yinteraction;
	    zforce += zinteraction;

	    assert(!isnan(xinteraction));
	    assert(!isnan(yinteraction));
	    assert(!isnan(zinteraction));
	    assert(fabs(xinteraction) < 1e4);
	    assert(fabs(yinteraction) < 1e4);
	    assert(fabs(zinteraction) < 1e4);

	    atomicAdd(accsolvent + sentry    , -xinteraction);
	    atomicAdd(accsolvent + sentry + 1, -yinteraction);
	    atomicAdd(accsolvent + sentry + 2, -zinteraction);
	}

	atomicAdd(acc + 3 * pid + 0, xforce);
	atomicAdd(acc + 3 * pid + 1, yforce);
	atomicAdd(acc + 3 * pid + 2, zforce);

	for(int c = 0; c < 3; ++c)
	    assert(!isnan(acc[3 * pid + c]));
    }

    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount)
    {
	if (firsttime)
	{
	    texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	    texCellsStart.filterMode = hipFilterModePoint;
	    texCellsStart.mipmapFilterMode = hipFilterModePoint;
	    texCellsStart.normalized = 0;

	    texCellsCount.channelDesc = hipCreateChannelDesc<int>();
	    texCellsCount.filterMode = hipFilterModePoint;
	    texCellsCount.mipmapFilterMode = hipFilterModePoint;
	    texCellsCount.normalized = 0;

	    texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
	    texSolventParticles.filterMode = hipFilterModePoint;
	    texSolventParticles.mipmapFilterMode = hipFilterModePoint;
	    texSolventParticles.normalized = 0;

	    CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(interactions_3tpp), hipFuncCachePreferL1));

	    firsttime = false;
	}

	size_t textureoffset = 0;

	if (npsolvent)
	{
	    CUDA_CHECK(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
				       sizeof(float) * 6 * npsolvent));
	    assert(textureoffset == 0);
	}

	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);

	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
    }
}

void ComputeFSI::bulk(std::vector<ParticlesWrap> wsolutes, hipStream_t stream)
{
    NVTX_RANGE("FSI/bulk", NVTX_C6);

    if (wsolutes.size() == 0)
	return;

    KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart, wsolvent.cellscount);

    CUDA_CHECK(hipPeekAtLastError());

    for(std::vector<ParticlesWrap>::iterator it = wsolutes.begin(); it != wsolutes.end(); ++it)
   	if (it->n)
	    KernelsFSI::interactions_3tpp<<< (3 * it->n + 127) / 128, 128, 0, stream >>>
		((float2 *)it->p, it->n, wsolvent.n, (float *)it->a, (float *)wsolvent.a, local_trunk.get_float());

    CUDA_CHECK(hipPeekAtLastError());
}

void ComputeFSI::halo(ParticlesWrap halowrap, hipStream_t stream)
{
    NVTX_RANGE("FSI/halo", NVTX_C7);

    KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart, wsolvent.cellscount);

    CUDA_CHECK(hipPeekAtLastError());

    if (halowrap.n)
	KernelsFSI::interactions_3tpp<<< (3 * halowrap.n + 127) / 128, 128, 0, stream >>>
	    ((float2 *)halowrap.p, halowrap.n, wsolvent.n, (float *)halowrap.a, (float *)wsolvent.a, local_trunk.get_float());

    CUDA_CHECK(hipPeekAtLastError());
}
