#include "hip/hip_runtime.h"
/*
 *  minmax-massimo.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Massimo Bernaschi on 2015-03-23.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include "minmax-massimo.h"

#define MAXTHREADS 1024
#define WARPSIZE     32
#define MAXV     100000000.
#define MINV    -100000000.

typedef struct 
{
    int g_block_id;
    int g_blockcnt;
    float3 minval;
    float3 maxval;
} sblockds_t;

__global__ void minmaxob(const Particle * const d_data, float3 *d_min, float3 *d_max, int size) {
    __shared__ float3 mintemp[32];
    __shared__ float3 maxtemp[32];
    __shared__ float shrtmp[3][MAXTHREADS];

    float3 mintemp1, maxtemp1;
    float3 mindef, maxdef;
    float temp2;
    if(blockDim.x>MAXTHREADS) {
        cuda_printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    mindef.x=MAXV;   mindef.y=MAXV;   mindef.z=MAXV;
    maxdef.x=MINV;   maxdef.y=MINV;   maxdef.z=MINV;
    __syncthreads();
    int tid = threadIdx.x;
    int xyz;
    for(int i=tid; i<3*blockDim.x; i+=blockDim.x) {
	xyz=i%3;
	//    if(xyz==0) {
	shrtmp[xyz][i/3] = (i/3<size)?d_data[i/3+blockIdx.x*size].x[xyz]:MINV;
	//    } else if(xyz==1) {
	//      shrtmp[xyz][i/3] = (i/3<size)?d_data[i/3+blockIdx.x*blockDim.x].y:MINV;
	//    } else {
	//      shrtmp[xyz][i/3] = (i/3<size)?d_data[i/3+blockIdx.x*blockDim.x].z:MINV;
	//    }
    }
    __syncthreads();
    mintemp1.x = (tid<size)?shrtmp[0][tid]:MAXV;
    mintemp1.y = (tid<size)?shrtmp[1][tid]:MAXV;
    mintemp1.z = (tid<size)?shrtmp[2][tid]:MAXV;
    maxtemp1.x = (tid<size)?shrtmp[0][tid]:MINV;
    maxtemp1.y = (tid<size)?shrtmp[1][tid]:MINV;
    maxtemp1.z = (tid<size)?shrtmp[2][tid]:MINV;
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(mintemp1.x,d);
	mintemp1.x=(mintemp1.x>temp2)?temp2:mintemp1.x;
	temp2 = __shfl_up(mintemp1.y,d);
	mintemp1.y=(mintemp1.y>temp2)?temp2:mintemp1.y;
	temp2 = __shfl_up(mintemp1.z,d);
	mintemp1.z=(mintemp1.z>temp2)?temp2:mintemp1.z;
	temp2 = __shfl_up(maxtemp1.x,d);
	maxtemp1.x=(maxtemp1.x<temp2)?temp2:maxtemp1.x;
	temp2 = __shfl_up(maxtemp1.y,d);
	maxtemp1.y=(maxtemp1.y<temp2)?temp2:maxtemp1.y;
	temp2 = __shfl_up(maxtemp1.z,d);
	maxtemp1.z=(maxtemp1.z<temp2)?temp2:maxtemp1.z;
    }
    if (tid%32 == 31) {
	mintemp[tid/32] = mintemp1;
	maxtemp[tid/32] = maxtemp1;
    }
    __syncthreads();
    if (threadIdx.x < 32) {
        mintemp1= (tid < blockDim.x/32)?mintemp[threadIdx.x]:mindef;
        maxtemp1= (tid < blockDim.x/32)?maxtemp[threadIdx.x]:maxdef;
        for (int d=1; d<32; d<<=1) {
	    temp2 = __shfl_up(mintemp1.x,d);
	    mintemp1.x=(mintemp1.x>temp2)?temp2:mintemp1.x;
	    temp2 = __shfl_up(mintemp1.y,d);
	    mintemp1.y=(mintemp1.y>temp2)?temp2:mintemp1.y;
	    temp2 = __shfl_up(mintemp1.z,d);
	    mintemp1.z=(mintemp1.z>temp2)?temp2:mintemp1.z;
	    temp2 = __shfl_up(maxtemp1.x,d);
	    maxtemp1.x=(maxtemp1.x<temp2)?temp2:maxtemp1.x;
	    temp2 = __shfl_up(maxtemp1.y,d);
	    maxtemp1.y=(maxtemp1.y<temp2)?temp2:maxtemp1.y;
	    temp2 = __shfl_up(maxtemp1.z,d);
	    maxtemp1.z=(maxtemp1.z<temp2)?temp2:maxtemp1.z;
        }
        if (tid < blockDim.x/32) {
	    mintemp[tid] = mintemp1;
	    maxtemp[tid] = maxtemp1;
        }
    }
    __syncthreads();
    if (threadIdx.x==blockDim.x-1) {
	d_min[blockIdx.x]=mintemp[blockDim.x/32-1];
	d_max[blockIdx.x]=maxtemp[blockDim.x/32-1];
    }

}


__global__ void minmaxmba(const Particle  *d_data, float3 *d_min, float3 *d_max,
			  int size, sblockds_t *ptoblockds) {

  __shared__ float3 mintemp[32];
  __shared__ float3 maxtemp[32];
  __shared__ float shrtmp[3][MAXTHREADS];

  __shared__ unsigned int my_blockId;
  const int which=blockIdx.x/((size+blockDim.x-1)/blockDim.x); /* which particle should manage */
  float3 mintemp1, maxtemp1;
  float3 mindef, maxdef;
  float temp2;
  if(blockDim.x > MAXTHREADS) {
        cuda_printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
  }
  if (threadIdx.x==0) {
    my_blockId = atomicAdd( &(ptoblockds[which].g_block_id), 1 );
  }
  mindef.x=MAXV;   mindef.y=MAXV;   mindef.z=MAXV;
  maxdef.x=MINV;   maxdef.y=MINV;   maxdef.z=MINV;
  __syncthreads();
  int tid = threadIdx.x;
  int xyz;
  for(int i=tid; i<3*blockDim.x; i+=blockDim.x) {
    xyz=i%3;
    //    if(xyz==0) {
    shrtmp[xyz][i/3] = (i/3+my_blockId*blockDim.x<size)?d_data[i/3+my_blockId*blockDim.x+which*size].x[xyz]:MINV;
    //    } else if(xyz==1) {
    //      shrtmp[xyz][i/3] = (i/3+my_blockId*blockDim.x<size)?d_data[i/3+my_blockId*blockDim.x+which*size].y:MINV;
    //    } else {
    //      shrtmp[xyz][i/3] = (i/3+my_blockId*blockDim.x<size)?d_data[i/3+my_blockId*blockDim.x+which*size].z:MINV;
     //    }
  }
  __syncthreads();
  mintemp1.x = (tid+my_blockId*blockDim.x<size)?shrtmp[0][tid]:MAXV;
  mintemp1.y = (tid+my_blockId*blockDim.x<size)?shrtmp[1][tid]:MAXV;
  mintemp1.z = (tid+my_blockId*blockDim.x<size)?shrtmp[2][tid]:MAXV;
  maxtemp1.x = (tid+my_blockId*blockDim.x<size)?shrtmp[0][tid]:MINV;
  maxtemp1.y = (tid+my_blockId*blockDim.x<size)?shrtmp[1][tid]:MINV;
  maxtemp1.z = (tid+my_blockId*blockDim.x<size)?shrtmp[2][tid]:MINV;
  for (int d=1; d<32; d<<=1) {
         temp2 = __shfl_up(mintemp1.x,d);
         mintemp1.x=(mintemp1.x>temp2)?temp2:mintemp1.x;
         temp2 = __shfl_up(mintemp1.y,d);
         mintemp1.y=(mintemp1.y>temp2)?temp2:mintemp1.y;
         temp2 = __shfl_up(mintemp1.z,d);
         mintemp1.z=(mintemp1.z>temp2)?temp2:mintemp1.z;
         temp2 = __shfl_up(maxtemp1.x,d);
         maxtemp1.x=(maxtemp1.x<temp2)?temp2:maxtemp1.x;
         temp2 = __shfl_up(maxtemp1.y,d);
         maxtemp1.y=(maxtemp1.y<temp2)?temp2:maxtemp1.y;
         temp2 = __shfl_up(maxtemp1.z,d);
         maxtemp1.z=(maxtemp1.z<temp2)?temp2:maxtemp1.z;
  }
  if (tid%32 == 31) {
    mintemp[tid/32] = mintemp1;
    maxtemp[tid/32] = maxtemp1;
  }
  __syncthreads();
  if (threadIdx.x < 32) {
        mintemp1= (tid < blockDim.x/32)?mintemp[threadIdx.x]:mindef;
        maxtemp1= (tid < blockDim.x/32)?maxtemp[threadIdx.x]:maxdef;
        for (int d=1; d<32; d<<=1) {
         temp2 = __shfl_up(mintemp1.x,d);
         mintemp1.x=(mintemp1.x>temp2)?temp2:mintemp1.x;
         temp2 = __shfl_up(mintemp1.y,d);
         mintemp1.y=(mintemp1.y>temp2)?temp2:mintemp1.y;
         temp2 = __shfl_up(mintemp1.z,d);
         mintemp1.z=(mintemp1.z>temp2)?temp2:mintemp1.z;
         temp2 = __shfl_up(maxtemp1.x,d);
         maxtemp1.x=(maxtemp1.x<temp2)?temp2:maxtemp1.x;
         temp2 = __shfl_up(maxtemp1.y,d);
         maxtemp1.y=(maxtemp1.y<temp2)?temp2:maxtemp1.y;
         temp2 = __shfl_up(maxtemp1.z,d);
         maxtemp1.z=(maxtemp1.z<temp2)?temp2:maxtemp1.z;
        }
        if (tid < blockDim.x/32) {
          mintemp[tid] = mintemp1;
          maxtemp[tid] = maxtemp1;
        }
  }
  __syncthreads();
  mintemp1=mintemp[blockDim.x/32-1];
  maxtemp1=maxtemp[blockDim.x/32-1];
  if (threadIdx.x==(blockDim.x-1)) {
        do {} while( atomicAdd(&(ptoblockds[which].g_blockcnt),0) < my_blockId );
        mintemp1.x=(ptoblockds[which].minval.x<mintemp1.x)?ptoblockds[which].minval.x:mintemp1.x;
        maxtemp1.x=(ptoblockds[which].maxval.x>maxtemp1.x)?ptoblockds[which].maxval.x:maxtemp1.x;
        mintemp1.y=(ptoblockds[which].minval.y<mintemp1.y)?ptoblockds[which].minval.y:mintemp1.y;
        maxtemp1.y=(ptoblockds[which].maxval.y>maxtemp1.y)?ptoblockds[which].maxval.y:maxtemp1.y;
        mintemp1.z=(ptoblockds[which].minval.z<mintemp1.z)?ptoblockds[which].minval.z:mintemp1.z;
        maxtemp1.z=(ptoblockds[which].maxval.z>maxtemp1.z)?ptoblockds[which].maxval.z:maxtemp1.z;
        if(my_blockId==(((size+blockDim.x-1)/blockDim.x))-1) { /* it is the last block; reset for next iteration */
                ptoblockds[which].minval=mindef;
                ptoblockds[which].maxval=maxdef;
                ptoblockds[which].g_blockcnt=0;
                ptoblockds[which].g_block_id=0;
                d_min[which]=mintemp1;
                d_max[which]=maxtemp1;
        } else {
                ptoblockds[which].minval=mintemp1;
                ptoblockds[which].maxval=maxtemp1;
                atomicAdd(&(ptoblockds[which].g_blockcnt),1);
        }
  }

}

void minmax_massimo(const Particle * const rbc, int size, int n, float3 *minrbc, float3 *maxrbc, hipStream_t stream) 
{
    const int size32 = ((size + 31) / 32) * 32;

    if (size32 < MAXTHREADS)
	minmaxob<<<n, size32, 0, stream>>>(rbc, minrbc, maxrbc, size);
    else
    {
	static int nctc = -1;

        static sblockds_t *ptoblockds = NULL;
	
        if( n > nctc) 
	{
	    sblockds_t * h_ptoblockds = new sblockds_t[n];
	    
	    for(int i=0; i < n; i++)  
	    {
		h_ptoblockds[i].g_block_id=0;
		h_ptoblockds[i].g_blockcnt=0;
		h_ptoblockds[i].minval.x=MAXV;
		h_ptoblockds[i].maxval.x=MINV;
		h_ptoblockds[i].minval.y=MAXV;
		h_ptoblockds[i].maxval.y=MINV;
		h_ptoblockds[i].minval.z=MAXV;
		h_ptoblockds[i].maxval.z=MINV;
	    }

	    if (ptoblockds != NULL) 
		CUDA_CHECK(hipFree(ptoblockds));

           CUDA_CHECK(hipMalloc((void **)&ptoblockds,sizeof(sblockds_t) * n));

           CUDA_CHECK(hipMemcpy(ptoblockds, h_ptoblockds, sizeof(sblockds_t) * n, hipMemcpyHostToDevice));

           delete [] h_ptoblockds;
        }

        int nblocks= n * ((size + MAXTHREADS - 1) / MAXTHREADS);

        minmaxmba<<<nblocks, MAXTHREADS, 0, stream>>>(rbc, minrbc, maxrbc, size, ptoblockds);
    }
}