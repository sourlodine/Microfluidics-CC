#include "hip/hip_runtime.h"
/*
 *  containers.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-05.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <sys/stat.h>

#include <rbc-cuda.h>

#include "containers.h"
#include "io.h"
	    	    
namespace ParticleKernels
{
    __global__ void update_stage1(Particle * p, Acceleration * a, int n, float dt,
				  const float driving_acceleration, const bool check = true)
    {
	assert(blockDim.x * gridDim.x >= n);
    
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;
    
	for(int c = 0; c < 3; ++c)
	{
	    assert(!isnan(p[pid].x[c]));
	    assert(!isnan(p[pid].u[c]));
	    assert(!isnan(a[pid].a[c]));
	}

	for(int c = 0; c < 3; ++c)
	    p[pid].u[c] += (a[pid].a[c] + (c == 0 ? driving_acceleration : 0)) * dt * 0.5;
    
	for(int c = 0; c < 3; ++c)
	    p[pid].x[c] += p[pid].u[c] * dt;

#ifndef NDEBUG
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	if (check)
	    for(int c = 0; c < 3; ++c)
	    {
		assert(p[pid].x[c] >= -L[c] -L[c]/2);
		assert(p[pid].x[c] <= +L[c] +L[c]/2);
	    }
#endif
    }

    __global__ void update_stage2_and_1(Particle * p, Acceleration * a, int n, float dt, const float driving_acceleration)
    {
	assert(blockDim.x * gridDim.x >= 3 * n);
	
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= 3 * n)
	    return;
	
	const int pid = gid / 3;
	const int c = gid % 3;

	const float mya = a[pid].a[c] + (c == 0 ? driving_acceleration : 0);
	
	float myu = p[pid].u[c];
	float myx = p[pid].x[c];
	
	myu += mya * dt;
	myx += myu * dt;

	assert(!isnan(myu) && !isnan(myx));
	
	p[pid].u[c] = myu;
	p[pid].x[c] = myx;
	
#ifndef NDEBUG
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };	
	
	if (!(myx >= -L[c] -L[c]/2) || !(myx <= +L[c] +L[c]/2))
	{
	    cuda_printf("Uau: pid %d c %d: x %f u %f and a %f\n",
		   pid, c, myx, myu, mya);
	    
	    assert(myx >= -L[c] -L[c]/2);
	    assert(myx <= +L[c] +L[c]/2);
	}
#endif
    }
    
    __global__ void clear_velocity(Particle * const p, const int n)
    {
	assert(blockDim.x * gridDim.x >= n);
    
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	for(int c = 0; c < 3; ++c)
	    p[pid].u[c] = 0;
    }
}

ParticleArray::ParticleArray(vector<Particle> ic)
{
    resize(ic.size());

    CUDA_CHECK(hipMemcpy(xyzuvw.data, (float*) &ic.front(), sizeof(Particle) * ic.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * ic.size()));

    void (*upkernel)(Particle * p, Acceleration * a, int n, float dt,
		     const float da) = ParticleKernels::update_stage2_and_1;
    
    CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*upkernel), hipFuncCachePreferL1));
}

void ParticleArray::update_stage1(const float driving_acceleration, hipStream_t stream)
{
    if (size)
	ParticleKernels::update_stage1<<<(xyzuvw.size + 127) / 128, 128, 0, stream>>>(
	    xyzuvw.data, axayaz.data, xyzuvw.size, dt, driving_acceleration , false);
}

void  ParticleArray::update_stage2_and_1(const float driving_acceleration, hipStream_t stream)
{
    if (size)
	ParticleKernels::update_stage2_and_1<<<(xyzuvw.size * 3 + 127) / 128, 128, 0, stream>>>
	    (xyzuvw.data, axayaz.data, xyzuvw.size, dt, driving_acceleration);
}

void ParticleArray::resize(int n)
{
    size = n;
    
    xyzuvw.resize(n);
    axayaz.resize(n);
    
    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * size));
}

void ParticleArray::preserve_resize(int n)
{
	int oldsize = size;
    size = n;

    xyzuvw.preserve_resize(n);
    axayaz.preserve_resize(n);

    if (size > oldsize)
    	CUDA_CHECK(hipMemset(axayaz.data + oldsize, 0, sizeof(Acceleration) * (size-oldsize)));
}
void ParticleArray::clear_velocity()
{
    if (size)
	ParticleKernels::clear_velocity<<<(xyzuvw.size + 127) / 128, 128 >>>(xyzuvw.data, xyzuvw.size);
}

void CollectionRBC::resize(const int count)
{
    nrbcs = count;

    ParticleArray::resize(count * nvertices);
}

void CollectionRBC::preserve_resize(const int count)
{
    nrbcs = count;

    ParticleArray::preserve_resize(count * nvertices);
}
    
struct TransformedExtent
{
    float com[3];
    float transform[4][4];
};

CollectionRBC::CollectionRBC(MPI_Comm cartcomm, const string path2ic): 
    cartcomm(cartcomm), nrbcs(0), path2xyz("rbcs.xyz"), format4ply("ply/rbcs-%04d.ply"), 
    path2ic("rbcs-ic.txt"), dumpcounter(0)
{
    MPI_CHECK(MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );
    
    CudaRBC::Extent extent;
    CudaRBC::setup(nvertices, extent);

    assert(extent.xmax - extent.xmin < XSIZE_SUBDOMAIN);
    assert(extent.ymax - extent.ymin < YSIZE_SUBDOMAIN);
    assert(extent.zmax - extent.zmin < ZSIZE_SUBDOMAIN);

    CudaRBC::get_triangle_indexing(indices, ntriangles);
}

void CollectionRBC::setup()
{
    vector<TransformedExtent> allrbcs;

    if (myrank == 0)
    {
	//read transformed extent from file
	FILE * f = fopen(path2ic.c_str(), "r");
	printf("READING FROM: <%s>\n", path2ic.c_str());
	bool isgood = true;
	
	while(isgood)
	{
	    float tmp[19];
	    for(int c = 0; c < 19; ++c)
	    {
		int retval = fscanf(f, "%f", tmp + c);
		
		isgood &= retval == 1;
	    }

	    if (isgood)
	    {
		TransformedExtent t;
		
		for(int c = 0; c < 3; ++c)
		    t.com[c] = tmp[c];

		int ctr = 3;
		for(int c = 0; c < 16; ++c, ++ctr)
		    t.transform[c / 4][c % 4] = tmp[ctr];

		allrbcs.push_back(t);
	    }
	}

	fclose(f);
    }

    if (myrank == 0)
	printf("Instantiating %d CELLs from...<%s>\n", (int)allrbcs.size(), path2ic.c_str());

    int allrbcs_count = allrbcs.size();
    MPI_CHECK(MPI_Bcast(&allrbcs_count, 1, MPI_INT, 0, cartcomm));

    allrbcs.resize(allrbcs_count);
    
    const int nfloats_per_entry = sizeof(TransformedExtent) / sizeof(float);
    assert( sizeof(TransformedExtent) % sizeof(float) == 0);

    MPI_CHECK(MPI_Bcast(&allrbcs.front(), nfloats_per_entry * allrbcs_count, MPI_FLOAT, 0, cartcomm));

    vector<TransformedExtent> good;

    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

    for(vector<TransformedExtent>::iterator it = allrbcs.begin(); it != allrbcs.end(); ++it)
    {
	bool inside = true;

	for(int c = 0; c < 3; ++c)
	    inside &= it->com[c] >= coords[c] * L[c] && it->com[c] < (coords[c] + 1) * L[c];

	if (inside)
	{
	    for(int c = 0; c < 3; ++c)
		it->transform[c][3] -= (coords[c] + 0.5) * L[c];

	    good.push_back(*it);
	}
    }
    
    resize(good.size());

    for(int i = 0; i < good.size(); ++i)
	_initialize((float *)(xyzuvw.data + nvertices * i), good[i].transform);
	//CudaRBC::initialize((float *)(xyzuvw.data + nvertices * i), good[i].transform);
}

void CollectionRBC::_initialize(float *device_xyzuvw, const float (*transform)[4])
{
    CudaRBC::initialize(device_xyzuvw, transform);
}

void CollectionRBC::remove(const int * const entries, const int nentries)
{
    std::vector<bool > marks(nrbcs, true);

    for(int i = 0; i < nentries; ++i)
	marks[entries[i]] = false;

    std::vector< int > survivors;
    for(int i = 0; i < nrbcs; ++i)
	if (marks[i])
	    survivors.push_back(i);

    const int nsurvived = survivors.size();

    SimpleDeviceBuffer<Particle> survived(nvertices * nsurvived);

    for(int i = 0; i < nsurvived; ++i)
	CUDA_CHECK(hipMemcpy(survived.data + nvertices * i, data() + nvertices * survivors[i], 
			      sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice));
	    
    resize(nsurvived);

    CUDA_CHECK(hipMemcpy(xyzuvw.data, survived.data, sizeof(Particle) * survived.size, hipMemcpyDeviceToDevice));
}

void CollectionRBC::dump(MPI_Comm comm, MPI_Comm cartcomm)
{
    int& ctr = dumpcounter;
    const bool firsttime = ctr == 0;
	    
    const int n = size;

    Particle * p = new Particle[n];
    Acceleration * a = new Acceleration[n];

    CUDA_CHECK(hipMemcpy(p, xyzuvw.data, sizeof(Particle) * n, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(a, axayaz.data, sizeof(Acceleration) * n, hipMemcpyDeviceToHost));
		   
    //we fused VV stages so we need to recover the state before stage 1
    for(int i = 0; i < n; ++i)
	for(int c = 0; c < 3; ++c)
	{
	    assert(!isnan(p[i].x[c]));
	    assert(!isnan(p[i].u[c]));
	    assert(!isnan(a[i].a[c]));
	    
	    p[i].x[c] -= dt * p[i].u[c];
	    p[i].u[c] -= 0.5 * dt * a[i].a[c];
	}

    if (xyz_dumps)
	xyz_dump(comm, cartcomm, path2xyz.c_str(), "cell-particles", p, n, !firsttime);

    char buf[200];
    sprintf(buf, format4ply.c_str(), ctr);

    if (ctr ==0)
    {
	int rank;
	MPI_CHECK(MPI_Comm_rank(comm, &rank));
		
	if(rank == 0)
	    mkdir("ply", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
    }
	    
    ply_dump(comm, cartcomm, buf, indices, nrbcs, ntriangles, p, nvertices, false);
		    
    delete [] p;
    delete [] a;

    ++ctr;
}
