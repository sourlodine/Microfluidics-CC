#include "hip/hip_runtime.h"
#include <sys/stat.h>

#include <rbc-cuda.h>

#include "containers.h"
	    	    
namespace ParticleKernels
{
    __global__ void update_stage1(Particle * p, Acceleration * a, int n, float dt,
				  const float dpdx, const float dpdy, const float dpdz, const bool check = true)
    {
	const float gradp[3] = {dpdx, dpdy, dpdz};
	
	assert(blockDim.x * gridDim.x >= n);
    
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;
    
	for(int c = 0; c < 3; ++c)
	{
	    assert(!isnan(p[pid].x[c]));
	    assert(!isnan(p[pid].u[c]));
	    assert(!isnan(a[pid].a[c]));
	}

	for(int c = 0; c < 3; ++c)
	    p[pid].u[c] += (a[pid].a[c] - gradp[c]) * dt * 0.5;
    
	for(int c = 0; c < 3; ++c)
	    p[pid].x[c] += p[pid].u[c] * dt;

	if (check)
	    for(int c = 0; c < 3; ++c)
	    {
		assert(p[pid].x[c] >= -L -L/2);
		assert(p[pid].x[c] <= +L +L/2);
	    }
    }

    __global__ void update_stage2_and_1(Particle * p, Acceleration * a, int n, float dt,
					const float dpdx, const float dpdy, const float dpdz, const bool check = true)
    {
	const float gradp[3] = {dpdx, dpdy, dpdz};
	
	assert(blockDim.x * gridDim.x >= n);
    
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	for(int c = 0; c < 3; ++c)
	    assert(!isnan(p[pid].u[c]));

	for(int c = 0; c < 3; ++c)
	    assert(!isnan(a[pid].a[c]));

	for(int c = 0; c < 3; ++c)
	{
	    const float mya = a[pid].a[c] - gradp[c];
	    float myu = p[pid].u[c];
	    float myx = p[pid].x[c];

	    myu += mya * dt;
	    myx += myu * dt;
	
	    p[pid].u[c] = myu; 
	    p[pid].x[c] = myx; 
	}

	if (check)
	    for(int c = 0; c < 3; ++c)
	    {
		if (!(p[pid].x[c] >= -L -L/2) || !(p[pid].x[c] <= +L +L/2))
		    printf("Uau: %f %f %f %f %f %f and acc %f %f %f\n", 
			   p[pid].x[0], p[pid].x[1], p[pid].x[2], 
			   p[pid].u[0], p[pid].u[1], p[pid].u[2],
			   a[pid].a[0], a[pid].a[1],a[pid].a[2]);
	    
		assert(p[pid].x[c] >= -L -L/2);
		assert(p[pid].x[c] <= +L +L/2);
	    }
    }
}

ParticleArray::ParticleArray(vector<Particle> ic)
{
    resize(ic.size());

    CUDA_CHECK(hipMemcpy(xyzuvw.data, (float*) &ic.front(), sizeof(Particle) * ic.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * ic.size()));
}

void ParticleArray::update_stage1(const float gradpressure[3])
{
    if (size)
	ParticleKernels::update_stage1<<<(xyzuvw.size + 127) / 128, 128 >>>(
	    xyzuvw.data, axayaz.data, xyzuvw.size, dt,
	    gradpressure[0], gradpressure[1], gradpressure[2] , false);
}

void  ParticleArray::update_stage2_and_1(const float gradpressure[3])
{
    if (size)
	ParticleKernels::update_stage2_and_1<<<(xyzuvw.size + 127) / 128, 128 >>>
	    (xyzuvw.data, axayaz.data, xyzuvw.size, dt,
	     gradpressure[0], gradpressure[1], gradpressure[2], false);
}

void ParticleArray::resize(int n)
{
    size = n;
    
    xyzuvw.resize(n);
    axayaz.resize(n);
    
    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * size));
}

struct TransformedExtent
{
    float transform[4][4];

    float xmin[3], xmax[3],local_xmin[3], local_xmax[3];
       
    void build_transform(CudaRBC::Extent extent, const int L)
	{
	    const float angles[3] = { 
		0.25 * (drand48() - 0.5) * 2 * M_PI, 
		M_PI * 0.5 + 0.25 * (drand48() * 2 - 1) * M_PI,
		0.25 * (drand48() - 0.5) * 2 * M_PI
	    };

	    for(int i = 0; i < 4; ++i)
		for(int j = 0; j < 4; ++j)
		    transform[i][j] = i == j;

	    for(int i = 0; i < 3; ++i)
		transform[i][3] = - 0.5 * (local_xmin[i] + local_xmax[i]);

	    for(int d = 0; d < 3; ++d)
	    {
		const float c = cos(angles[d]);
		const float s = sin(angles[d]);

		float tmp[4][4];

		for(int i = 0; i < 4; ++i)
		    for(int j = 0; j < 4; ++j)
			tmp[i][j] = i == j;

		if (d == 0)
		{
		    tmp[0][0] = tmp[1][1] = c;
		    tmp[0][1] = -(tmp[1][0] = s);
		} 
		else 
		    if (d == 1)
		    {
			tmp[0][0] = tmp[2][2] = c;
			tmp[0][2] = -(tmp[2][0] = s);
		    }
		    else
		    {  
			tmp[1][1] = tmp[2][2] = c;
			tmp[1][2] = -(tmp[2][1] = s);
		    }

		float res[4][4];
		for(int i = 0; i < 4; ++i)
		    for(int j = 0; j < 4; ++j)
		    {
			float s = 0;
			    
			for(int k = 0; k < 4; ++k)
			    s += transform[i][k] * tmp[k][j];

			res[i][j] = s;
		    }

		for(int i = 0; i < 4; ++i)
		    for(int j = 0; j < 4; ++j)
			transform[i][j] = res[i][j];
	    }

	    for(int i = 0; i < 3; ++i)
		transform[i][3] += (drand48() - 0.5) * (L - 4);
	}

    void apply(float x[3], float y[3])
	{
	    for(int i = 0; i < 3; ++i)
		y[i] = transform[i][0] * x[0] + transform[i][1] * x[1] + transform[i][2] * x[2] + transform[i][3];
	}

    TransformedExtent(CudaRBC::Extent extent, const int L)
	{
	    local_xmin[0] = extent.xmin;
	    local_xmin[1] = extent.ymin;
	    local_xmin[2] = extent.zmin;
		
	    local_xmax[0] = extent.xmax;
	    local_xmax[1] = extent.ymax;
	    local_xmax[2] = extent.zmax;
	
	    build_transform(extent, L);

	    for(int i = 0; i < 8; ++i)
	    {
		const int idx[3] = { i % 2, (i/2) % 2, (i/4) % 2 };

		float local[3];
		for(int c = 0; c < 3; ++c)
		    local[c] = idx[c] ? local_xmax[c] : local_xmin[c];

		float world[3];

		apply(local, world);

		if (i == 0)
		    for(int c = 0; c < 3; ++c)
			xmin[c] = xmax[c] = world[c];
		else
		    for(int c = 0; c < 3; ++c)
		    {
			xmin[c] = min(xmin[c], world[c]);
			xmax[c] = max(xmax[c], world[c]);
		    }
	    }
	}

    bool collides(const TransformedExtent a, const  float tol)
	{
	    int s[3], e[3];
	    for(int c = 0; c < 3; ++c)
	    {
		s[c] = max(xmin[c], a.xmin[c]);
		e[c] = min(xmax[c], a.xmax[c]);

		if (s[c] -e[c] >= tol)
		    return false;
	    }

	    return true;
	}
};


void CollectionRBC::resize(const int count)
{
    nrbcs = count;

    ParticleArray::resize(count * nvertices);
}

    
CollectionRBC::CollectionRBC(const int L): L(L), nrbcs(0)
{
    CudaRBC::Extent extent;
    CudaRBC::setup(nvertices, extent);

    /*
    printf("extent: %f %f %f %f %f %f\n",
	   extent.xmax , extent.xmin,
	   extent.ymax , extent.ymin,
	   extent.zmax , extent.zmin);	   
    */
    
    assert(extent.xmax - extent.xmin < L);
    assert(extent.ymax - extent.ymin < L);
    assert(extent.zmax - extent.zmin < L);

    vector<TransformedExtent*> good;
	    
    for(int attempt = 0; attempt < 1000; ++attempt)
    {
	TransformedExtent* t = new TransformedExtent(extent, L);
		
	bool noncolliding = true;
	for(int i = 0; i < good.size() && noncolliding; ++i)
	    noncolliding &= !t->collides(*good[i], 0.00);
		
	if (noncolliding)
	    good.push_back(t);
    }
	    
    resize(good.size());

    for(int i = 0; i < good.size(); ++i)
	CudaRBC::initialize((float *)(xyzuvw.data + nvertices * i), good[i]->transform);
}

void CollectionRBC::update_stage1()
{
    const float dpdx[3] = {0, 0, 0};
    ParticleArray::update_stage1(dpdx);
}

void CollectionRBC::update_stage2_and_1()
{
    const float dpdx[3] = {0, 0, 0};
    ParticleArray::update_stage2_and_1(dpdx);
}

void CollectionRBC::remove(const int * const entries, const int nentries)
{
    std::vector<bool > marks(nrbcs, true);

    for(int i = 0; i < nentries; ++i)
	marks[entries[i]] = false;

    std::vector< int > survivors;
    for(int i = 0; i < nrbcs; ++i)
	if (marks[i])
	    survivors.push_back(i);

    const int nsurvived = survivors.size();

    SimpleDeviceBuffer<Particle> survived(nvertices * nsurvived);

    for(int i = 0; i < nsurvived; ++i)
	CUDA_CHECK(hipMemcpy(survived.data + nvertices * i, data() + nvertices * survivors[i], 
			      sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice));
	    
    resize(nsurvived);

    CUDA_CHECK(hipMemcpy(xyzuvw.data, survived.data, sizeof(Particle) * survived.size, hipMemcpyDeviceToDevice));
}

void CollectionRBC::dump(MPI_Comm comm)
{
    static int ctr = 0;
    const bool firsttime = ctr == 0;
	    
    const int n = size;

    Particle * p = new Particle[n];
    Acceleration * a = new Acceleration[n];

    CUDA_CHECK(hipMemcpy(p, xyzuvw.data, sizeof(Particle) * n, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(a, axayaz.data, sizeof(Acceleration) * n, hipMemcpyDeviceToHost));
		   
    //we fused VV stages so we need to recover the state before stage 1
    for(int i = 0; i < n; ++i)
	for(int c = 0; c < 3; ++c)
	{
	    assert(!isnan(p[i].x[c]));
	    assert(!isnan(p[i].u[c]));
	    assert(!isnan(a[i].a[c]));
	    
	    p[i].x[c] -= dt * p[i].u[c];
	    p[i].u[c] -= 0.5 * dt * a[i].a[c];
	}

    if (xyz_dumps)
	xyz_dump(comm, "rbcs.xyz", "rbcparticles", p, n,  L, !firsttime);

    int (*indices)[3];
    int ntriangles;
    CudaRBC::get_triangle_indexing(indices, ntriangles);

    char buf[200];
    sprintf(buf, "ply/rbcs-%04d.ply", ctr);

    if (ctr ==0)
    {
	int rank;
	MPI_CHECK(MPI_Comm_rank(comm, &rank));
		
	if(rank == 0)
	    mkdir("ply", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
    }
	    
    ply_dump(comm, buf, indices, nrbcs, ntriangles, p, nvertices, L, false);
		    
    delete [] p;
    delete [] a;

    ++ctr;
}
