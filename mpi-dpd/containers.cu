#include "hip/hip_runtime.h"

/*
 *  containers.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-05.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <sys/stat.h>

#include <rbc-cuda.h>

#include "containers.h"
#include "io.h"
#include "ctc.h"

int (*CollectionRBC::indices)[3] = NULL, CollectionRBC::ntriangles = -1, CollectionRBC::nvertices = -1;

int (*CollectionCTC::indices)[3] = NULL, CollectionCTC::ntriangles = -1, CollectionCTC::nvertices = -1;

namespace ParticleKernels
{
    __global__ void update_stage1(Particle * p, Acceleration * a, int n, float dt,
				  const float driving_acceleration, const bool check = true)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	for(int c = 0; c < 3; ++c)
	{
	    assert(!isnan(p[pid].x[c]));
	    assert(!isnan(p[pid].u[c]));
	    assert(!isnan(a[pid].a[c]));
	}

	for(int c = 0; c < 3; ++c)
	    p[pid].u[c] += (a[pid].a[c] + (c == 0 ? driving_acceleration : 0)) * dt * 0.5;

	for(int c = 0; c < 3; ++c)
	    p[pid].x[c] += p[pid].u[c] * dt;

#ifndef NDEBUG
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	if (check)
	    for(int c = 0; c < 3; ++c)
	    {
		assert(p[pid].x[c] >= -L[c] -L[c]/2);
		assert(p[pid].x[c] <= +L[c] +L[c]/2);
	    }
#endif
    }

    __global__ void update_stage2_and_1(float2 * const pdata, const float * const adata,
					const int nparticles, const float dt, const float driving_acceleration)
    {
	
#if !defined(__CUDA_ARCH__)
#warning __CUDA_ARCH__ not defined! assuming 350
#define _ACCESS(x) __ldg(x)
#elif __CUDA_ARCH__ >= 350
#define _ACCESS(x) __ldg(x)
#else
#define _ACCESS(x) (*(x))
#endif

	enum { NWARPS = 4 };

	assert(blockDim.x * blockDim.y * gridDim.x >= nparticles && blockDim.x == 32 && blockDim.y == NWARPS);

	__shared__ volatile float shxv[NWARPS][32 * 6 + 1];
//	__shared__ volatile float sha[NWARPS][32 * 3];

	const int pidbase = 32 * (threadIdx.y + NWARPS * blockIdx.x);
	const int nlocalparticles = min(nparticles - pidbase, 32);
	const int nwords = nlocalparticles * 3;

	const int base = 3 * pidbase;
	const int wid = threadIdx.y;
	const int tid = threadIdx.x;

	float ax,ay,az;

	const bool valid = tid < nlocalparticles;
	if (valid)
	{
	    const int entry = 3 * (pidbase + tid);
	    ax = _ACCESS(adata + entry);
	    ay = _ACCESS(adata + entry + 1);
	    az = _ACCESS(adata + entry + 2);
	}


	float2 tmp2[3] = {0, 0, 0};
	//float tmp[3] = {0, 0, 0};

#pragma unroll 3
	for(int c = 0; c < 3; ++c)
	    if (tid + 32 * c < nwords)
		tmp2[c] = _ACCESS(pdata + base + tid + 32 * c);


	/*
#pragma unroll 3
	for(int c = 0; c < 3; ++c) 
	    if (tid + 32 * c < nwords)
		tmp[c] = adata[base + tid + 32 * c];
*/
#pragma unroll 3
	for(int c = 0; c < 3; ++c)
	{
	    shxv[wid][2 * (tid + 32 * c)] = tmp2[c].x;
	    shxv[wid][2 * (tid + 32 * c) + 1] = tmp2[c].y;
	}
	
//#pragma unroll 3
	/*for(int c = 0; c < 3; ++c)
	    sha[wid][tid + 32 *c] = tmp[c];
	*/
//	__syncthreads();



	if (valid)
	{
	    const int entry = 6 * tid;

	    float2 xy = make_float2(shxv[wid][entry], shxv[wid][entry + 1]);
	    float2 zu = make_float2(shxv[wid][entry + 2], shxv[wid][entry + 3]);
	    float2 vw = make_float2(shxv[wid][entry + 4], shxv[wid][entry + 5]);

	    /*  const float ax = sha[wid][entry];
	    const float ay = sha[wid][entry + 1];
	    const float az = sha[wid][entry + 2];
	    */
	    zu.y += (ax + driving_acceleration) * dt;
	    vw.x += ay * dt;
	    vw.y += az * dt;

	    xy.x += zu.y * dt;
	    xy.y += vw.x * dt;
	    zu.x += vw.y * dt;

#ifndef NDEBUG
	    {
		const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
		const float x[3] = {xy.x, xy.y, zu.x};
		const float u[3] = {zu.y, vw.x, vw.y};
		const float a[3] = {ax, ay, az};

		for(int c = 0; c < 3; ++c)
		    if (!(x[c] >= -L[c] -L[c]/2) || !(x[c] <= +L[c] +L[c]/2))
		    {
			cuda_printf("Uau: pid %d c %d: x %f u %f and a %f\n",
				    pid, c, x[c], u[c], a[c]);

			assert(x[c] >= -L[c] -L[c]/2);
			assert(x[c] <= +L[c] +L[c]/2);
		    }
	    }
#endif
	    shxv[wid][entry] = xy.x;
	    shxv[wid][entry + 1] = xy.y;
	    shxv[wid][entry + 2] = zu.x;
	    shxv[wid][entry + 3] = zu.y;
	    shxv[wid][entry + 4] = vw.x;
	    shxv[wid][entry + 5] = vw.y;
	}

//	__syncthreads();

#pragma unroll 3
	for(int c = 0; c < 3; ++c)
	{
	    tmp2[c].x = shxv[wid][2 * (tid + 32 * c)];
	    tmp2[c].y = shxv[wid][2 * (tid + 32 * c) + 1];
	}

#pragma unroll 3
	for(int c = 0; c < 3; ++c)
	    if (tid + 32 * c < nwords)
		pdata[base + tid + 32 * c] = tmp2[c];
    }

    __global__ void clear_velocity(Particle * const p, const int n)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	for(int c = 0; c < 3; ++c)
	    p[pid].u[c] = 0;
    }
}

ParticleArray::ParticleArray(vector<Particle> ic)
{
    resize(ic.size());

    CUDA_CHECK(hipMemcpy(xyzuvw.data, (float*) &ic.front(), sizeof(Particle) * ic.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * ic.size()));

//    CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*ParticleKernels::update_stage2_and_1), hipFuncCachePreferL1));
}

void ParticleArray::update_stage1(const float driving_acceleration, hipStream_t stream)
{
    if (size)
	ParticleKernels::update_stage1<<<(xyzuvw.size + 127) / 128, 128, 0, stream>>>(
	    xyzuvw.data, axayaz.data, xyzuvw.size, dt, driving_acceleration , false);
}

void  ParticleArray::update_stage2_and_1(const float driving_acceleration, hipStream_t stream)
{
    if (size)
	ParticleKernels::update_stage2_and_1<<<(xyzuvw.size + 127) / 128, dim3(32, 4), 0, stream>>>
	    ((float2 *)xyzuvw.data, (float *)axayaz.data, xyzuvw.size, dt, driving_acceleration);
}

void ParticleArray::resize(int n)
{
    size = n;

    // YTANG: need the array to be 32-padded for locally transposed storage of acceleration
    if ( n % 32 ) {
        xyzuvw.preserve_resize( n - n % 32 + 32 );
        axayaz.preserve_resize( n - n % 32 + 32 );
    }
    xyzuvw.resize(n);
    axayaz.resize(n);

    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * size));
}

void ParticleArray::preserve_resize(int n)
{
    int oldsize = size;
    size = n;

    xyzuvw.preserve_resize(n);
    axayaz.preserve_resize(n);

    if (size > oldsize)
    	CUDA_CHECK(hipMemset(axayaz.data + oldsize, 0, sizeof(Acceleration) * (size-oldsize)));
}

void ParticleArray::clear_velocity()
{
    if (size)
	ParticleKernels::clear_velocity<<<(xyzuvw.size + 127) / 128, 128 >>>(xyzuvw.data, xyzuvw.size);
}

void CollectionRBC::resize(const int count)
{
    ncells = count;

    ParticleArray::resize(count * get_nvertices());
}

void CollectionRBC::preserve_resize(const int count)
{
    ncells = count;

    ParticleArray::preserve_resize(count * get_nvertices());
}

struct TransformedExtent
{
    float com[3];
    float transform[4][4];
};

CollectionRBC::CollectionRBC(MPI_Comm cartcomm):
cartcomm(cartcomm), ncells(0)
{
    MPI_CHECK(MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    CudaRBC::get_triangle_indexing(indices, ntriangles);
    CudaRBC::Extent extent;
    CudaRBC::setup(nvertices, extent);

    assert(extent.xmax - extent.xmin < XSIZE_SUBDOMAIN);
    assert(extent.ymax - extent.ymin < YSIZE_SUBDOMAIN);
    assert(extent.zmax - extent.zmin < ZSIZE_SUBDOMAIN);
}

void CollectionRBC::setup(const char * const path2ic)
{
    vector<TransformedExtent> allrbcs;

    if (myrank == 0)
    {
	//read transformed extent from file
	FILE * f = fopen(path2ic, "r");
	printf("READING FROM: <%s>\n", path2ic);
	bool isgood = true;

	while(isgood)
	{
	    float tmp[19];
	    for(int c = 0; c < 19; ++c)
	    {
		int retval = fscanf(f, "%f", tmp + c);

		isgood &= retval == 1;
	    }

	    if (isgood)
	    {
		TransformedExtent t;

		for(int c = 0; c < 3; ++c)
		    t.com[c] = tmp[c];

		int ctr = 3;
		for(int c = 0; c < 16; ++c, ++ctr)
		    t.transform[c / 4][c % 4] = tmp[ctr];

		allrbcs.push_back(t);
	    }
	}

	fclose(f);
    }

    if (myrank == 0)
	printf("Instantiating %d CELLs from...<%s>\n", (int)allrbcs.size(), path2ic);

    int allrbcs_count = allrbcs.size();
    MPI_CHECK(MPI_Bcast(&allrbcs_count, 1, MPI_INT, 0, cartcomm));

    allrbcs.resize(allrbcs_count);

    const int nfloats_per_entry = sizeof(TransformedExtent) / sizeof(float);
    assert( sizeof(TransformedExtent) % sizeof(float) == 0);

    MPI_CHECK(MPI_Bcast(&allrbcs.front(), nfloats_per_entry * allrbcs_count, MPI_FLOAT, 0, cartcomm));

    vector<TransformedExtent> good;

    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

    for(vector<TransformedExtent>::iterator it = allrbcs.begin(); it != allrbcs.end(); ++it)
    {
	bool inside = true;

	for(int c = 0; c < 3; ++c)
	    inside &= it->com[c] >= coords[c] * L[c] && it->com[c] < (coords[c] + 1) * L[c];

	if (inside)
	{
	    for(int c = 0; c < 3; ++c)
		it->transform[c][3] -= (coords[c] + 0.5) * L[c];

	    good.push_back(*it);
	}
    }

    resize(good.size());

    for(int i = 0; i < good.size(); ++i)
	_initialize((float *)(xyzuvw.data + get_nvertices() * i), good[i].transform);
}

void CollectionRBC::_initialize(float *device_xyzuvw, const float (*transform)[4])
{
    CudaRBC::initialize(device_xyzuvw, transform);
}

void CollectionRBC::remove(const int * const entries, const int nentries)
{
    std::vector<bool > marks(ncells, true);

    for(int i = 0; i < nentries; ++i)
	marks[entries[i]] = false;

    std::vector< int > survivors;
    for(int i = 0; i < ncells; ++i)
	if (marks[i])
	    survivors.push_back(i);

    const int nsurvived = survivors.size();

    SimpleDeviceBuffer<Particle> survived(get_nvertices() * nsurvived);

    for(int i = 0; i < nsurvived; ++i)
	CUDA_CHECK(hipMemcpy(survived.data + get_nvertices() * i, data() + get_nvertices() * survivors[i],
			      sizeof(Particle) * get_nvertices(), hipMemcpyDeviceToDevice));

    resize(nsurvived);

    CUDA_CHECK(hipMemcpy(xyzuvw.data, survived.data, sizeof(Particle) * survived.size, hipMemcpyDeviceToDevice));
}

void CollectionRBC::_dump(const char * const path2xyz, const char * const format4ply,
			  MPI_Comm comm, MPI_Comm cartcomm, const int ntriangles, const int ncells, const int nvertices, int (* const indices)[3],
			  Particle * const p, const Acceleration * const a, const int n, const int iddatadump)
{
    int ctr = iddatadump;
    const bool firsttime = ctr == 0;

    //we fused VV stages so we need to recover the state before stage 1
    for(int i = 0; i < n; ++i)
	for(int c = 0; c < 3; ++c)
	{
	    assert(!isnan(p[i].x[c]));
	    assert(!isnan(p[i].u[c]));
	    assert(!isnan(a[i].a[c]));

	    p[i].x[c] -= dt * p[i].u[c];
	    p[i].u[c] -= 0.5 * dt * a[i].a[c];
	}

    if (xyz_dumps)
	xyz_dump(comm, cartcomm, path2xyz, "cell-particles", p, n, !firsttime);

    char buf[200];
    sprintf(buf, format4ply, ctr);

    if (ctr ==0)
    {
	int rank;
	MPI_CHECK(MPI_Comm_rank(comm, &rank));

	if(rank == 0)
	    mkdir("ply", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
    }

    ply_dump(comm, cartcomm, buf, indices, ncells, ntriangles, p, nvertices, false);
}
