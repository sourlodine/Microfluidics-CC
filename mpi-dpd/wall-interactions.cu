#include "hip/hip_runtime.h"
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/count.h>

#include <../saru.cuh>

#include "halo-exchanger.h"

#include "wall-interactions.h"

namespace SolidWallsKernel
{
    texture<float, 3, hipReadModeElementType> texSDF;

    __device__ float sdf(float x, float y, float z, const int L)
    {
	float p[3] = {x, y, z};
	
	float texcoord[3];
	for(int c = 0; c < 3; ++c)
	{
	    texcoord[c] = (p[c] - (-L/2 - 1)) / (L + 2);
	    assert(texcoord[c] >= 0 && texcoord[c] <= 1);
	}
	
	return tex3D(texSDF, texcoord[0], texcoord[1], texcoord[2]);
    }
    
    __global__ void fill_keys(const Particle * const particles, const int n, const int L, int * const key)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	const Particle p = particles[pid];

	key[pid] = (int)(sdf(p.x[0], p.x[1], p.x[2], L) > 0);
    }

    __global__ void zero_velocity(Particle * const dst, const int n)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	Particle p = dst[pid];

	for(int c = 0; c < 3; ++c)
	    p.u[c] = 0;

	dst[pid] = p;
    }

    __device__ bool handle_collision(float& x, float& y, float& z, float& u, float& v, float& w, /*float& dt,*/ const int L)
    {
	if (sdf(x, y, z, L) <= 0)
	    return false;

	const float xold = x - dt * u;
	const float yold = y - dt * v;
	const float zold = z - dt * w;

	float t = 0;

	for(int i = 1; i < 8; ++i)
	{
	    const float tcandidate = t + dt / (1 << i);
	    const float xcandidate = xold + tcandidate * u;
	    const float ycandidate = yold + tcandidate * v;
	    const float zcandidate = zold + tcandidate * w;

	    if (sdf(xcandidate, ycandidate, zcandidate, L) <= 0)
		t = tcandidate;
	}

	const float lambda = 2 * t - dt;

	x = xold + lambda * u;
	y = yold + lambda * v;
	z = zold + lambda * w;

	u  = -u;
	v  = -v;
	w  = -w;
	//dt = dt - t;

	return true;
    }

    __global__ void bounce(Particle * const particles, const int n, const int L) //, const float dt)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	Particle p = particles[pid];

	for(int c = 0; c < 3; ++c)
	    assert(abs(p.x[c]) <= L/2 + 1);

	if (handle_collision(p.x[0], p.x[1], p.x[2], p.u[0], p.u[1], p.u[2], L))
	    particles[pid] = p;
    }

    __global__ void interactions(const Particle * const particles, const int np, Acceleration * const acc,
				 const int * const starts, const int * const counts, const int L,
				 const Particle * const solid, const int nsolid, const int saru_tag1, const int saru_tag2,
				 const float aij, const float gamma, const float sigmaf)
    {
	assert(blockDim.x * gridDim.x >= np);

       	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= np)
	    return;

	Particle p = particles[pid];
	
	int base[3];
	for(int c = 0; c < 3; ++c)
	{
	    assert(p.x[c] >= -L/2 - 1);
	    base[c] = (int)(p.x[c] - (-L/2 -1));
	}

	const float xp = p.x[0], yp = p.x[1], zp = p.x[2];
	const float up = p.u[0], vp = p.u[1], wp = p.u[2];
	
	float xforce = 0, yforce = 0, zforce = 0;
	
	for(int code = 0; code < 27; ++code)
	{
	    const int xcid = base[0] + (code % 3) - 1;
	    const int ycid = base[1] + (code/3 % 3) - 1;
	    const int zcid = base[2] + (code/9 % 3) - 1;

	    if (xcid < 0 || xcid >= L + 2 ||
		ycid < 0 || ycid >= L + 2 ||
		zcid < 0 || zcid >= L + 2 )
		continue;
			    
	    const int cid = xcid + (L + 2) * (ycid + (L + 2) * zcid);
	    assert(cid >= 0 && cid < (L + 2) * (L + 2) * (L + 2));

	    const int start = starts[cid];
	    const int stop = start + counts[cid];

	    assert(start >= 0 && stop <= nsolid && start <= stop);

	    for(int s = start; s < stop; ++s)
	    {
		const float xq = solid[s].x[0];
		const float yq = solid[s].x[1];
		const float zq = solid[s].x[2];
		
	    	const float _xr = xp - xq;
		const float _yr = yp - yq;
		const float _zr = zp - zq;
		
		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		
		const float invrij = rsqrtf(rij2);
		 
		const float rij = rij2 * invrij;
		const float wr = max((float)0, 1 - rij);
		
		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv = 
		    xr * (up - 0) +
		    yr * (vp - 0) +
		    zr * (wp - 0);
		
		const float mysaru = saru(pid * nsolid + s, saru_tag1, saru_tag2);
	
		const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
		 
		const float strength = (aij - gamma * wr * rdotv + sigmaf * myrandnr) * wr;

		xforce += strength * xr;
		yforce += strength * yr;
		zforce += strength * zr;
	    }
	}

	acc[pid].a[0] += xforce;
	acc[pid].a[1] += yforce;
	acc[pid].a[2] += zforce;

	for(int c = 0; c < 3; ++c)
	    assert(!isnan(acc[pid].a[c]));
    }
}

float smoothstep(float edge0, float edge1, float x)
{
    // Scale, and clamp x to 0..1 range
    x = min(1.f, max(0.f, (x - edge0)/(edge1 - edge0)));
    // Evaluate polynomial
    return x*x*x*(x*(x*6 - 15) + 10);
}

ComputeInteractionsWall::ComputeInteractionsWall(MPI_Comm cartcomm, const int L, Particle* const p, 
						 const int n, int& nsurvived):
    cartcomm(cartcomm), L(L), arrSDF(NULL), solid(NULL), solid_size(0), cells(L+2)
{
    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );
    
    const int VPD = 64;

    float * field = new float[VPD * VPD * VPD];

    const double h = (L + 2) / (double)(VPD - 2);

#if 0

    //bifurcation
    const float r_cyl = 0.085 * L * dims[1];
    const float r2_cyl = r_cyl * r_cyl;
    
    static const int nh = 75 * 2;
    const float dh = 1./ (nh - 1);
    std::vector<float> xpts, ypts, zpts;
    
    for(int pass = 0; pass < 2; ++pass)
	for(int i = 0; i < nh; ++i)
	{
	    const float S0 = 0.15;
	    const float S1 = 0.5;
	    const float S2 = 0.85;
	    const float X0 = (S1 + S2) * 0.5 * dims[0] * L;
	    const float X1 = S2 * dims[0] * L;
	    const float Y0 = 0.5 * dims[1] * L;
	    const float Y1 = 0.8 * dims[1] * L;
	    const float Z0 = 0.5 * dims[2] * L;
	    const float Z1 = (0.5 + 1 * 0.35) * dims[2] * L;

	    const float lambda = i * dh;
	    
	    float x = lambda / S1 * X0;
	    if (lambda > S2)
		x = X0 + (lambda - S2) / (1 - S2) * (dims[0] * L - X0);
	    else
		if (lambda > S1)
		    x = X0 + sin( (lambda - S1) / (S2 - S1) * 2 * 3.1415) * (X1 - X0);
	   
	    const float y = Y0 + (2 * pass -1) * smoothstep(S0, S1, min(lambda, 1 - lambda)) * (Y1 - Y0);

	    float z = Z0;

	    if (lambda >= S1 && lambda < S2)
		z = 0.5 * (Z0 + Z1) - 0.5 * (Z1 - Z0) * cos( (lambda - S1) / (S2 - S1) * 2 * 3.1415);

	    xpts.push_back(x);
	    ypts.push_back(y);
	    zpts.push_back(z);
	}

    const int npts = xpts.size();
    
    for(int iz = 0; iz < VPD; ++iz)
	for(int iy = 0; iy < VPD; ++iy)
	    for(int ix = 0; ix < VPD; ++ix)
	    {
		const float x = coords[0] * L - 1 + (ix + 0.5) * h;
		const float y = coords[1] * L - 1 + (iy + 0.5) * h;
		const float z = coords[2] * L - 1 + (iz + 0.5) * h;

		float val = 1;
		for(int i = 0; i < npts && val == 1; ++i)
		{
		    const float dx = x - xpts[i];
		    const float dy = y - ypts[i];
		    const float dz = z - zpts[i];

		    if (dx * dx + dy * dy + dz * dz < r2_cyl)
			val = -1;
		}
				
		field[ix + VPD * (iy + VPD * iz)] = val;
	    }
#else

    const float y_cyl = 0.5 * L * dims[1];
    const float z_cyl = 0.5 * L * dims[2];
    const float r_cyl = 0.4 * L * dims[1];
    
    //cylinder / pipe
    for(int iz = 0; iz < VPD; ++iz)
	for(int iy = 0; iy < VPD; ++iy)
	    for(int ix = 0; ix < VPD; ++ix)
	    {
		//const float x = coords[0] * L - 1 + (ix + 0.5) * h;
		const float y = coords[1] * L - 1 + (iy + 0.5) * h;
		const float z = coords[2] * L - 1 + (iz + 0.5) * h;

		const float r = sqrt(pow(y - y_cyl, 2) + pow(z - z_cyl, 2));
		
		field[ix + VPD * (iy + VPD * iz)] = r - r_cyl;
	    }
#endif
    
    hipChannelFormatDesc fmt = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipMalloc3DArray (&arrSDF, &fmt, make_hipExtent(VPD, VPD, VPD)));

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void *)field, VPD * sizeof(float), VPD, VPD);
    copyParams.dstArray = arrSDF;
    copyParams.extent   = make_hipExtent(VPD, VPD, VPD);
    copyParams.kind     = hipMemcpyHostToDevice;
    CUDA_CHECK(hipMemcpy3D(&copyParams));

    for(int i = 0; i < 3; ++i)
	SolidWallsKernel::texSDF.addressMode[i] = hipAddressModeClamp;

    SolidWallsKernel::texSDF.normalized = true;
    SolidWallsKernel::texSDF.filterMode = hipFilterModeLinear;
    SolidWallsKernel::texSDF.addressMode[0] = hipAddressModeClamp;
    SolidWallsKernel::texSDF.addressMode[1] = hipAddressModeClamp;
    SolidWallsKernel::texSDF.addressMode[2] = hipAddressModeClamp;
		
    CUDA_CHECK(hipBindTextureToArray(SolidWallsKernel::texSDF, arrSDF, fmt));

    delete [] field;

    thrust::device_vector<int> keys(n);

    SolidWallsKernel::fill_keys<<< (n + 127) / 128, 128 >>>(p, n, L, thrust::raw_pointer_cast(&keys[0]));
    CUDA_CHECK(hipPeekAtLastError());
    
    thrust::sort_by_key(keys.begin(), keys.end(), thrust::device_ptr<Particle>(p));

    nsurvived = thrust::count(keys.begin(), keys.end(), 0);
    assert(nsurvived <= n);
    
    printf("rank %d nsurvived is %d -> %.2f%%\n", myrank, nsurvived, nsurvived * 100. /n);


    thrust::device_vector<Particle> solid_local(thrust::device_ptr<Particle>(p + nsurvived), thrust::device_ptr<Particle>(p + n));
  
    HaloExchanger halo(cartcomm, L);

    SimpleDeviceBuffer<Particle> solid_remote = halo.exchange(thrust::raw_pointer_cast(&solid_local[0]), solid_local.size());

    printf("receiving extra %d\n", solid_remote.size);
    
    solid_size = solid_local.size() + solid_remote.size;

    CUDA_CHECK(hipMalloc(&solid, sizeof(Particle) * solid_size));
    CUDA_CHECK(hipMemcpy(solid, thrust::raw_pointer_cast(&solid_local[0]), sizeof(Particle) * solid_local.size(), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(solid + solid_local.size(), solid_remote.data, sizeof(Particle) * solid_remote.size, hipMemcpyDeviceToDevice));
        
    SolidWallsKernel::zero_velocity<<< (solid_size + 127) / 128, 128>>>(solid, solid_size);

    if (solid_size > 0)
	cells.build(solid, solid_size);

    {
	const int n = solid_local.size();

	Particle * phost = new Particle[n];

	CUDA_CHECK(hipMemcpy(phost, thrust::raw_pointer_cast(&solid_local[0]), sizeof(Particle) * n, hipMemcpyDeviceToHost));

	H5PartDump solid_dump /*ha ha ha*/("solid-walls.h5part", cartcomm, L);
	solid_dump.dump(phost, n);

	delete [] phost;
    }
}

void ComputeInteractionsWall::bounce(Particle * const p, const int n)
{
    if (n > 0)
	SolidWallsKernel::bounce<<< (n + 127) / 128, 128>>>(p, n, L);
    
    CUDA_CHECK(hipPeekAtLastError());
}

void ComputeInteractionsWall::interactions(const Particle * const p, const int n, Acceleration * const acc,
			      const int * const cellsstart, const int * const cellscount, int& saru_tag)
{
    //cellsstart and cellscount IGNORED for now
    
    if (n > 0 && solid_size > 0)
	SolidWallsKernel::interactions<<< (n + 127) / 128, 128>>>(p, n, acc, cells.start, cells.count, L,
								  solid, solid_size, saru_tag, myrank, aij, gammadpd, sigmaf);

    CUDA_CHECK(hipPeekAtLastError());

    ++saru_tag;
}

ComputeInteractionsWall::~ComputeInteractionsWall()
{
    CUDA_CHECK(hipUnbindTexture(SolidWallsKernel::texSDF));
    CUDA_CHECK(hipFreeArray(arrSDF));
}
