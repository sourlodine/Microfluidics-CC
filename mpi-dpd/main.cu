#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <cassert>
#include <sys/stat.h>
#include <mpi.h>

#include <vector>

#include "common.h"
#include "containers.h"
#include "dpd-interactions.h"
#include "wall-interactions.h"
#include "redistribute-particles.h"
#include "redistribute-rbcs.h"
#include "rbc-interactions.h"

using namespace std;

int main(int argc, char ** argv)
{
    int ranks[3];
    
    if (argc != 4)
    {
	printf("usage: ./mpi-dpd <xranks> <yranks> <zranks>\n");
	exit(-1);
    }
    else
    	for(int i = 0; i < 3; ++i)
	    ranks[i] = atoi(argv[1 + i]);

    CUDA_CHECK(hipSetDevice(0));

    int nranks, rank;   
    
    {
	MPI_CHECK( MPI_Init(&argc, &argv) );
    
	{
	    MPI_CHECK( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	    MPI_CHECK( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );

	    srand48(rank);
	    
	    MPI_Comm cartcomm;
	    int periods[] = {1, 1, 1};	    
	    MPI_CHECK( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 1, &cartcomm) );
	
	    vector<Particle> ic(L * L * L * 3  );
	    
	    for(int i = 0; i < ic.size(); ++i)
		for(int c = 0; c < 3; ++c)
		{
		    ic[i].x[c] = -L * 0.5 + drand48() * L;
		    ic[i].u[c] = 0;
		}
	    	    	  
	    ParticleArray particles(ic);
	    CellLists cells(L);		  
	    CollectionRBC * rbcscoll = NULL;
	    
	    if (rbcs)
		rbcscoll = new CollectionRBC(L);
	    
	    RedistributeParticles redistribute(cartcomm, L);
	    RedistributeRBCs redistribute_rbcs(cartcomm, L);

	    ComputeInteractionsDPD dpd(cartcomm, L);
	    ComputeInteractionsRBC rbc_interactions(cartcomm, L);
	    ComputeInteractionsWall * wall = NULL;
	    
	    hipStream_t stream;
	    CUDA_CHECK(hipStreamCreate(&stream));
	    	    
	    redistribute_rbcs.stream = stream;

	    int saru_tag = rank;
	    
	    cells.build(particles.xyzuvw.data, particles.size);
	   
	    dpd.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count);
	    
	    if (rbcscoll != NULL)
		rbc_interactions.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
					  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc());

	    float dpdx[3] = {0, 0, 0};
		    
	    const size_t nsteps = (int)(tend / dt);

	    for(int it = 0; it < nsteps; ++it)
	    {
		if (it % steps_per_report == 0)
		{
		    report_host_memory_usage(cartcomm, stdout);

		    if (rank == 0)
		    {
			static double t0 = MPI_Wtime(), t1;

			t1 = MPI_Wtime();
		    
			if (it > 0)
			    printf("beginning of time step %d (%.3e s)\n", it, t1 - t0);

			t0 = t1;
		    }
		}
	    
		if (it == 0)
		{
		    particles.update_stage1(dpdx);
		    
		    if (rbcscoll)
			rbcscoll->update_stage1();
		}

		const int newnp = redistribute.stage1(particles.xyzuvw.data, particles.size);
		particles.resize(newnp);
		redistribute.stage2(particles.xyzuvw.data, particles.size);

		if (rbcscoll)
		{
		    const int nrbcs = redistribute_rbcs.stage1(rbcscoll->data(), rbcscoll->count());
		    rbcscoll->resize(nrbcs);
		    redistribute_rbcs.stage2(rbcscoll->data(), rbcscoll->count());
		}

		//create the wall when it is time
		if (walls && it > 500 && wall == NULL)
		{
		    int nsurvived = 0;
		    wall = new ComputeInteractionsWall(cartcomm, L, particles.xyzuvw.data, particles.size, nsurvived);
		    
		    particles.resize(nsurvived);
		    		    
		    if (rank == 0)
		    {
			if( access( "trajectories.xyz", F_OK ) != -1 )
			{
			    const int retval = rename ("trajectories.xyz", "trajectories-equilibration.xyz");
			    assert(retval != -1);
			}
		    
			if( access( "rbcscoll.xyz", F_OK ) != -1 )
			{
			    const int retval = rename ("rbcscoll.xyz", "rbcscoll-equilibration.xyz");
			    assert(retval != -1);
			}
		    }

		    //remove Rbcscoll touching the wall
		    if(rbcscoll)
		    {
			SimpleDeviceBuffer<int> marks(rbcscoll->pcount());
			
			SolidWallsKernel::fill_keys<<< (rbcscoll->pcount() + 127) / 128, 128 >>>(rbcscoll->data(), rbcscoll->pcount(), L, marks.data);
			
			vector<int> tmp(marks.size);
			CUDA_CHECK(hipMemcpy(tmp.data(), marks.data, sizeof(int) * marks.size, hipMemcpyDeviceToHost));
			
			const int nrbcs = rbcscoll->count();
			const int nvertices = rbcscoll->nvertices;

			std::vector<int> tokill;
			for(int i = 0; i < nrbcs; ++i)
			{
			    bool valid = true;

			    for(int j = 0; j < nvertices && valid; ++j)
				valid &= 0 == tmp[j + nvertices * i];
			    
			    if (!valid)
				tokill.push_back(i);
			}

			rbcscoll->remove(&tokill.front(), tokill.size());
		    }

		    if (pushtheflow)
			dpdx[0] = -0.1;
		}

		cells.build(particles.xyzuvw.data, particles.size);

		//THIS IS WHERE WE WANT TO ACHIEVE 70% OF THE PEAK
		//TODO: i need a coordinating class that performs all the local work while waiting for the communication
		{
		    dpd.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count);

		    if (rbcscoll)
			rbc_interactions.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data,
						  cells.start, cells.count, rbcscoll->data(), rbcscoll->count(), rbcscoll->acc());
		
		    if (wall)
		    {
			wall->interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, 
					   cells.start, cells.count, saru_tag);

			if (rbcscoll)
			    wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, saru_tag);
		    }
		}
		
		particles.update_stage2_and_1(dpdx);

		if (rbcscoll)
		    rbcscoll->update_stage2_and_1();

		if (wall)
		{
		    wall->bounce(particles.xyzuvw.data, particles.size);
		    
		    if (rbcscoll)
			wall->bounce(rbcscoll->data(), rbcscoll->pcount());
		}
	    
		if (it % steps_per_report == 0)
		{
		    const int n = particles.size;

		    Particle * p = new Particle[n];
		    Acceleration * a = new Acceleration[n];

		    CUDA_CHECK(hipMemcpy(p, particles.xyzuvw.data, sizeof(Particle) * n, hipMemcpyDeviceToHost));
		    CUDA_CHECK(hipMemcpy(a, particles.axayaz.data, sizeof(Acceleration) * n, hipMemcpyDeviceToHost));
		   
		    //we fused VV stages so we need to recover the state before stage 1
		    for(int i = 0; i < n; ++i)
			for(int c = 0; c < 3; ++c)
			{
			    assert(!isnan(p[i].x[c]));
			    assert(!isnan(p[i].u[c]));
			    assert(!isnan(a[i].a[c]));
	    
			    p[i].x[c] -= dt * p[i].u[c];
			    p[i].u[c] -= 0.5 * dt * a[i].a[c];
			}

		    diagnostics(cartcomm, p, n, dt, it, L, a);
		    
		    if (rbcscoll)
			rbcscoll->dump(cartcomm);
		   
		    delete [] p;
		    delete [] a;
		}
	    }

	    CUDA_CHECK(hipStreamDestroy(stream));
	
	    if (wall)
		delete wall;

	    if (rbcscoll)
		delete rbcscoll;

	    MPI_CHECK(MPI_Comm_free(&cartcomm));
	}
	
	MPI_CHECK( MPI_Finalize() );
	
	if (rank == 0)
	    printf("simulation is done. Ciao.\n");
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
	
