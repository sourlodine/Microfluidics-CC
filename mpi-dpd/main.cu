/*
 *  main.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-11-14.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdio>
#include <cassert>
#include <csignal>
#include <mpi.h>
#include <errno.h>
#if defined(REPORT_TOPOLOGY)
#include <pmi.h>
#endif

#include "argument-parser.h"
#include "simulation.h"

bool currently_profiling = false;
float tend;
bool walls, pushtheflow, doublepoiseuille, rbcs, ctcs, xyz_dumps, hdf5field_dumps, hdf5part_dumps, is_mps_enabled;
int steps_per_report, steps_per_dump, wall_creation_stepid, nvtxstart, nvtxstop;

namespace SignalHandling
{
    volatile sig_atomic_t graceful_exit = 0, graceful_signum = 0;

    void signal_handler(int signum)
    {
	graceful_exit = 1;
	graceful_signum = signum;
    }

    void setup()
    {
	struct sigaction action;
	memset(&action, 0, sizeof(struct sigaction));
	action.sa_handler = signal_handler;
	sigaction(SIGUSR1, &action, NULL);
    }

    bool check_termination_request()
    {
	return graceful_exit;
    }
}

int main(int argc, char ** argv)
{
    int ranks[3];

    //parsing of the positional arguments
    if (argc < 4)
    {
	printf("usage: ./mpi-dpd <xranks> <yranks> <zranks>\n");
	exit(-1);
    }
    else
    	for(int i = 0; i < 3; ++i)
	    ranks[i] = atoi(argv[1 + i]);

    ArgumentParser argp(vector<string>(argv + 4, argv + argc));

    tend = argp("-tend").asDouble(50);
    walls = argp("-walls").asBool(false);
    pushtheflow = argp("-pushtheflow").asBool(false);
    doublepoiseuille = argp("-doublepoiseuille").asBool(false);
    rbcs = argp("-rbcs").asBool(false);
    ctcs = argp("-ctcs").asBool(false);
    xyz_dumps = argp("-xyz_dumps").asBool(false);
    steps_per_report = argp("-steps_per_report").asInt(1000);
    steps_per_dump = argp("-steps_per_dump").asInt(1000);
    wall_creation_stepid = argp("-wall_creation_stepid").asInt(5000);
    nvtxstart = argp("-nvtxstart").asInt(10400);
    nvtxstop = argp("-nvtxstop").asInt(10500);
    
    SignalHandling::setup();

#ifdef _USE_NVTX_
    nvtxNameOsThread(pthread_self(), "MASTER_THREAD");
#endif

    CUDA_CHECK(hipSetDevice(0));

    CUDA_CHECK(hipDeviceReset());

    {
	is_mps_enabled = false;

	const char * mps_variables[] = {
	    "CRAY_CUDA_MPS",
	    "CUDA_MPS",
	    "CRAY_CUDA_PROXY",
	    "CUDA_PROXY"
	};

	for(int i = 0; i < 4; ++i)
	    is_mps_enabled |= getenv(mps_variables[i])!= NULL && atoi(getenv(mps_variables[i])) != 0;
    }

    int nranks, rank;

    {
	//needed for the asynchronous data dumps
	setenv("MPICH_MAX_THREAD_SAFETY", "multiple", 0);

	int provided_safety_level;
	MPI_CHECK( MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided_safety_level));

	MPI_CHECK( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_CHECK( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );

	if (provided_safety_level != MPI_THREAD_MULTIPLE)
	{
	    if (rank == 0)
		printf("ooooooooops MPI thread safety level is just %d. Aborting now.\n", provided_safety_level);

	    abort();
	}
	else
	    if (rank == 0)
		printf("I have set MPICH_MAX_THREAD_SAFETY=multiple\n");

	MPI_Comm activecomm = MPI_COMM_WORLD;

	bool reordering = true;

	const char * env_reorder = getenv("MPICH_RANK_REORDER_METHOD");

	//reordering of the ranks according to the computational domain and environment variables
	if (atoi(env_reorder ? env_reorder : "-1") == atoi("3"))
	{
	    reordering = false;

	    const bool usefulrank = rank < ranks[0] * ranks[1] * ranks[2];

	    MPI_CHECK(MPI_Comm_split(MPI_COMM_WORLD, usefulrank, rank, &activecomm)) ;

	    MPI_CHECK(MPI_Barrier(activecomm));

	    if (!usefulrank)
	    {
		printf("rank %d has been thrown away\n", rank);
		fflush(stdout);

		MPI_CHECK(MPI_Barrier(activecomm));

		MPI_Finalize();

		return 0;
	    }

	    MPI_CHECK(MPI_Barrier(activecomm));
	}

	MPI_Comm cartcomm;

	int periods[] = {1, 1, 1};

	MPI_CHECK( MPI_Cart_create(activecomm, 3, ranks, periods, (int)reordering, &cartcomm) );

	activecomm = cartcomm;

	//print the rank-to-node mapping
	{
	    char name[1024];
	    int len;
	    MPI_CHECK(MPI_Get_processor_name(name, &len));

	    int dims[3], periods[3], coords[3];
	    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

	    MPI_CHECK(MPI_Barrier(activecomm));
#if defined(REPORT_TOPOLOGY)
	    int nid;
	    int rc = PMI_Get_nid(rank, &nid);
	    pmi_mesh_coord_t xyz;
	    PMI_Get_meshcoord((uint16_t) nid, &xyz);
	    printf("RANK %d: (%d, %d, %d) -> %s (%d, %d, %d)\n", rank, coords[0], coords[1], coords[2], name, xyz.mesh_x, xyz.mesh_y, xyz.mesh_z);
#else
	    printf("RANK %d: (%d, %d, %d) -> %s\n", rank, coords[0], coords[1], coords[2], name);
#endif
	    fflush(stdout);

	    MPI_CHECK(MPI_Barrier(activecomm));
	}

	//RAII
	{
	    if (rank == 0)
		argp.print_arguments();

	    Simulation simulation(cartcomm, activecomm, SignalHandling::check_termination_request);

	    simulation.run();
	}

	if (activecomm != cartcomm)
	    MPI_CHECK(MPI_Comm_free(&activecomm));

	MPI_CHECK(MPI_Comm_free(&cartcomm));

	MPI_CHECK(MPI_Finalize());
    }

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipDeviceReset());

    return 0;
}
