/*
 *  main.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-11-14.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdio>
#include <cassert>
#include <csignal>
#include <mpi.h>
#include <errno.h>
#if defined(REPORT_TOPOLOGY)
#include <pmi.h>
#endif

#include "argument-parser.h"
#include "simulation.h"
#include "dumper.h"

bool currently_profiling = false;
float tend, couette;
bool walls, pushtheflow, doublepoiseuille, rbcs, ctcs, xyz_dumps, hdf5field_dumps,
hdf5part_dumps, is_mps_enabled, adjust_message_sizes, contactforces, stress;
int steps_per_report, steps_per_dump, wall_creation_stepid, nvtxstart, nvtxstop, nsubsteps;

LocalComm localcomm;

namespace SignalHandling
{
	volatile sig_atomic_t gracefulExit = 0;

	void signalHandler(int signum)
	{
		gracefulExit = 1;
	}

	void setup()
	{
		struct sigaction action;
		memset(&action, 0, sizeof(struct sigaction));
		action.sa_handler = signalHandler;
		sigaction(SIGUSR1, &action, NULL);
		sigaction(SIGTERM, &action, NULL);
	}

	bool checkTerminationRequest()
	{
		return gracefulExit;
	}
}

int main(int argc, char ** argv)
{
	int ranks[3];

	//parsing of the positional arguments
	if (argc < 4)
	{
		printf("usage: ./mpi-dpd <xranks> <yranks> <zranks>\n");
		exit(-1);
	}
	else
		for(int i = 0; i < 3; ++i)
			ranks[i] = atoi(argv[1 + i]);

	ArgumentParser argp(vector<string>(argv + 4, argv + argc));

	tend = argp("-tend").asDouble(50);
	walls = argp("-walls").asBool(false);
	pushtheflow = argp("-pushtheflow").asBool(false);
	doublepoiseuille = argp("-doublepoiseuille").asBool(false);
	rbcs = argp("-rbcs").asBool(false);
	ctcs = argp("-ctcs").asBool(false);
	xyz_dumps = argp("-xyz_dumps").asBool(false);
	hdf5field_dumps = argp("-hdf5field_dumps").asBool(false);
	steps_per_report = argp("-steps_per_report").asInt(1000);
	steps_per_dump = argp("-steps_per_dump").asInt(1000);
	wall_creation_stepid = argp("-wall_creation_stepid").asInt(5000);
	nvtxstart = argp("-nvtxstart").asInt(10400);
	nvtxstop = argp("-nvtxstop").asInt(10500);
	adjust_message_sizes = argp("-adjust_message_sizes").asBool(false);
	contactforces = argp("-contactforces").asBool(false);
	stress = argp("-stress").asBool(false);
	couette = argp("-couette").asDouble(0);
	nsubsteps = argp("-nsubsteps").asInt(0);

	SignalHandling::setup();

#ifdef _USE_NVTX_
	nvtxNameOsThread(pthread_self(), "MASTER_THREAD");
#endif

	CUDA_CHECK(hipSetDevice(0));

	CUDA_CHECK(hipDeviceReset());

	{
		is_mps_enabled = false;

		const char * mps_variables[] = {
				"CRAY_CUDA_MPS",
				"CUDA_MPS",
				"CRAY_CUDA_PROXY",
				"CUDA_PROXY"
		};

		for(int i = 0; i < 4; ++i)
			is_mps_enabled |= getenv(mps_variables[i])!= NULL && atoi(getenv(mps_variables[i])) != 0;
	}

	int nranks, rank;


	MPI_CHECK(MPI_Init(&argc, &argv));
	MPI_CHECK( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_CHECK( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );

	MPI_Comm  iocomm, activecomm, intercomm, splitcomm;

	assert(nranks & 0x1 == 0);
	int computeTask = (rank+1) % 2;
	MPI_CHECK( MPI_Comm_split(MPI_COMM_WORLD, computeTask, rank, &splitcomm) );
	if (computeTask)
		MPI_CHECK( MPI_Comm_dup(splitcomm, &activecomm) );
	else
		MPI_CHECK( MPI_Comm_dup(splitcomm, &iocomm) );

	if (computeTask)
		MPI_CHECK( MPI_Intercomm_create(activecomm, 0, MPI_COMM_WORLD, 1, 0, &intercomm) );
	else
		MPI_CHECK( MPI_Intercomm_create(iocomm,     0, MPI_COMM_WORLD, 0, 0, &intercomm) );


#if defined(CUSTOM_REORDERING)
    		activecomm = setup_reorder_comm(MPI_COMM_WORLD, rank, nranks);
#endif

    		bool reordering = true;

    		const char * env_reorder = getenv("MPICH_RANK_REORDER_METHOD");

    		//reordering of the ranks according to the computational domain and environment variables
    		if (computeTask && atoi(env_reorder ? env_reorder : "-1") == atoi("3"))
    		{
    			reordering = false;

    			const bool usefulrank = rank < ranks[0] * ranks[1] * ranks[2];

    			MPI_CHECK(MPI_Comm_split(MPI_COMM_WORLD, usefulrank, rank, &activecomm)) ;

    			MPI_CHECK(MPI_Barrier(activecomm));

    			if (!usefulrank)
    			{
    				printf("rank %d has been thrown away\n", rank);
    				fflush(stdout);

    				MPI_CHECK(MPI_Barrier(activecomm));

    				MPI_Finalize();

    				return 0;
    			}

    			MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
    		}

    		MPI_Comm cartcomm, iocartcomm;

    		int periods[] = {1, 1, 1};

    		if (computeTask)
    			MPI_CHECK( MPI_Cart_create(activecomm, 3, ranks, periods, (int)reordering, &cartcomm) );
    		else
    			MPI_CHECK( MPI_Cart_create(iocomm, 3, ranks, periods, (int)reordering, &iocartcomm) );

    		activecomm = cartcomm;

    		//print the rank-to-node mapping
    		if (computeTask)
    		{
    			char name[1024];
    			int len;
    			MPI_CHECK(MPI_Get_processor_name(name, &len));

    			int dims[3], periods[3], coords[3];
    			MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    			MPI_CHECK(MPI_Barrier(activecomm));
#if defined(REPORT_TOPOLOGY)
    			int nid;
    			int rc = PMI_Get_nid(rank, &nid);
    			pmi_mesh_coord_t xyz;
    			PMI_Get_meshcoord((uint16_t) nid, &xyz);
    			printf("RANK %d: (%d, %d, %d) -> %s (%d, %d, %d)\n", rank, coords[0], coords[1], coords[2], name, xyz.mesh_x, xyz.mesh_y, xyz.mesh_z);
#else
    			printf("RANK %d: (%d, %d, %d) -> %s\n", rank, coords[0], coords[1], coords[2], name);
#endif
    			fflush(stdout);

    			MPI_CHECK(MPI_Barrier(activecomm));
    		}

    		//RAII
    		{
    			if (computeTask)
    			{
    				MPI_CHECK(MPI_Barrier(activecomm));

    				if (rank == 0)
    				{
    					argp.print_arguments();
    					fflush(stdout);
    				}

    				localcomm.initialize(activecomm);

    				MPI_CHECK(MPI_Barrier(activecomm));

    				Simulation simulation(cartcomm, activecomm, intercomm, SignalHandling::checkTerminationRequest);
    				simulation.run();
    			}
    			else
    			{
    				Dumper dumper(iocomm, iocartcomm, intercomm);
    				dumper.do_dump();
    			}
    		}

    		if (computeTask)
    		{
    			if (activecomm != cartcomm)
    				MPI_CHECK(MPI_Comm_free(&activecomm));

    			MPI_CHECK(MPI_Comm_free(&cartcomm));
    			MPI_CHECK(MPI_Comm_free(&intercomm));
    		}
    		else
    		{
    			MPI_CHECK(MPI_Comm_free(&iocomm));
    			MPI_CHECK(MPI_Comm_free(&intercomm));
    		}

    		MPI_CHECK(MPI_Finalize());

    		if (computeTask)
    		{
    			CUDA_CHECK(hipDeviceSynchronize());

    			CUDA_CHECK(hipDeviceReset());
    		}

    		return 0;
}
