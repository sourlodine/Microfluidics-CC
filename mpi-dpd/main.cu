#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <cassert>
#include <sys/stat.h>
#include <mpi.h>

#include <vector>
#include <map>

#include "common.h"
#include "containers.h"
#include "dpd-interactions.h"
#include "wall-interactions.h"
#include "redistribute-particles.h"
#include "redistribute-rbcs.h"
#include "rbc-interactions.h"

using namespace std;

int main(int argc, char ** argv)
{
    int ranks[3];
    
    if (argc != 4)
    {
	printf("usage: ./mpi-dpd <xranks> <yranks> <zranks>\n");
	exit(-1);
    }
    else
    	for(int i = 0; i < 3; ++i)
	    ranks[i] = atoi(argv[1 + i]);

    CUDA_CHECK(hipSetDevice(0));

    int nranks, rank;   
    
    {
	MPI_CHECK( MPI_Init(&argc, &argv) );
    
	{
	    MPI_CHECK( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	    MPI_CHECK( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );

	    srand48(rank);
	    
	    MPI_Comm cartcomm;
	    int periods[] = {1, 1, 1};	    
	    MPI_CHECK( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 1, &cartcomm) );
	
	    vector<Particle> ic(L * L * L * 3  );
	    
	    for(int i = 0; i < ic.size(); ++i)
		for(int c = 0; c < 3; ++c)
		{
		    ic[i].x[c] = -L * 0.5 + drand48() * L;
		    ic[i].u[c] = 0;
		}
	    	    	  
	    ParticleArray particles(ic);
	    CellLists cells(L);		  
	    CollectionRBC * rbcscoll = NULL;
	    
	    if (rbcs)
		rbcscoll = new CollectionRBC(cartcomm, L);
	    
	    RedistributeParticles redistribute(cartcomm, L);
	    RedistributeRBCs redistribute_rbcs(cartcomm, L);

	    ComputeInteractionsDPD dpd(cartcomm, L);
	    ComputeInteractionsRBC rbc_interactions(cartcomm, L);
	    ComputeInteractionsWall * wall = NULL;
	    
	    hipStream_t stream;
	    CUDA_CHECK(hipStreamCreate(&stream));
	    	    
	    redistribute_rbcs.stream = stream;

	    int saru_tag = rank;
	    
	    CUDA_CHECK(hipPeekAtLastError());

	    cells.build(particles.xyzuvw.data, particles.size);
	    std::map<string, double> timings;
	    dpd.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count);
	    
	    if (rbcscoll)
		rbc_interactions.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
					  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc());

	    float dpdx[3] = {0, 0, 0};

	    if (!walls && pushtheflow)
		dpdx[0] = -0.01;		    

	    const size_t nsteps = (int)(tend / dt);
	    
	    for(int it = 0; it < nsteps; ++it)
	    {
		if (it % steps_per_report == 0)
		{
		    report_host_memory_usage(cartcomm, stdout);

		    if (rank == 0)
		    {
			static double t0 = MPI_Wtime(), t1;

			t1 = MPI_Wtime();
		    
			if (it > 0)
			{
			    printf("beginning of time step %d (%.3f ms)\n", it, (t1 - t0) * 1e3 / steps_per_report);
			    printf("in more details, per time step:\n");
			    double tt = 0;
			    for(std::map<string, double>::iterator it = timings.begin(); it != timings.end(); ++it)
			    {
				printf("%s: %.3f ms\n", it->first.c_str(), it->second * 1e3 / steps_per_report);
				tt += it->second;
				it->second = 0;
			    }
			    printf("discrepancy: %.3f ms\n", ((t1 - t0) - tt) * 1e3 / steps_per_report);
			}

			t0 = t1;
		    }
		}
	    
		double tstart;

		if (it == 0)
		{
		    particles.update_stage1(dpdx);
		    
		    if (rbcscoll)
			rbcscoll->update_stage1();
		}

		tstart = MPI_Wtime();
		
		const int newnp = redistribute.stage1(particles.xyzuvw.data, particles.size);
		particles.resize(newnp);
		redistribute.stage2(particles.xyzuvw.data, particles.size);
		
		timings["redistribute-particles"] += MPI_Wtime() - tstart;
		
		CUDA_CHECK(hipPeekAtLastError());

		if (rbcscoll)
		{	
		    tstart = MPI_Wtime();
		    const int nrbcs = redistribute_rbcs.stage1(rbcscoll->data(), rbcscoll->count());
		    rbcscoll->resize(nrbcs);
		    redistribute_rbcs.stage2(rbcscoll->data(), rbcscoll->count());
		    timings["redistribute-rbc"] += MPI_Wtime() - tstart;
		}

		CUDA_CHECK(hipPeekAtLastError());
		
		CUDA_CHECK(hipStreamSynchronize(redistribute.mystream));
		CUDA_CHECK(hipStreamSynchronize(redistribute_rbcs.stream));

		//create the wall when it is time
		if (walls && it > 5000 && wall == NULL)
		{
		    int nsurvived = 0;
		    wall = new ComputeInteractionsWall(cartcomm, L, particles.xyzuvw.data, particles.size, nsurvived);
		    
		    particles.resize(nsurvived);
		    particles.clear_velocity();
		    		    
		    if (rank == 0)
		    {
			if( access( "trajectories.xyz", F_OK ) != -1 )
			{
			    const int retval = rename ("trajectories.xyz", "trajectories-equilibration.xyz");
			    assert(retval != -1);
			}
		    
			if( access( "rbcs.xyz", F_OK ) != -1 )
			{
			    const int retval = rename ("rbcs.xyz", "rbcs-equilibration.xyz");
			    assert(retval != -1);
			}
		    }

		    //remove Rbcscoll touching the wall
		    if(rbcscoll)
		    {
			SimpleDeviceBuffer<int> marks(rbcscoll->pcount());
			
			SolidWallsKernel::fill_keys<<< (rbcscoll->pcount() + 127) / 128, 128 >>>(rbcscoll->data(), rbcscoll->pcount(), L, marks.data);
			
			vector<int> tmp(marks.size);
			CUDA_CHECK(hipMemcpy(tmp.data(), marks.data, sizeof(int) * marks.size, hipMemcpyDeviceToHost));
			
			const int nrbcs = rbcscoll->count();
			const int nvertices = rbcscoll->nvertices;

			std::vector<int> tokill;
			for(int i = 0; i < nrbcs; ++i)
			{
			    bool valid = true;

			    for(int j = 0; j < nvertices && valid; ++j)
				valid &= 0 == tmp[j + nvertices * i];
			    
			    if (!valid)
				tokill.push_back(i);
			}

			rbcscoll->remove(&tokill.front(), tokill.size());
			rbcscoll->clear_velocity();
		    }

		    if (pushtheflow)
			dpdx[0] = -0.01;
		}

		tstart = MPI_Wtime();
		cells.build(particles.xyzuvw.data, particles.size);
		timings["build-cells"] += MPI_Wtime() - tstart;
		
		CUDA_CHECK(hipPeekAtLastError());
		
		//THIS IS WHERE WE WANT TO ACHIEVE 70% OF THE PEAK
		//TODO: i need a coordinating class that performs all the local work while waiting for the communication
		{
		    tstart = MPI_Wtime();
		    dpd.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count);
		    timings["evaluate-dpd"] += MPI_Wtime() - tstart;
		    
		    CUDA_CHECK(hipPeekAtLastError());	
		    	
		    if (rbcscoll)
		    {
			tstart = MPI_Wtime();
			rbc_interactions.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data,
						  cells.start, cells.count, rbcscoll->data(), rbcscoll->count(), rbcscoll->acc());
			timings["evaluate-rbc"] += MPI_Wtime() - tstart;
		    }
		    
		    CUDA_CHECK(hipPeekAtLastError());

		    if (wall)
		    {
			tstart = MPI_Wtime();
			wall->interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, 
					   cells.start, cells.count, saru_tag);

			if (rbcscoll)
			    wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, saru_tag);

			timings["evaluate-walls"] += MPI_Wtime() - tstart;
		    }

		    CUDA_CHECK(hipDeviceSynchronize());
		}
		
		CUDA_CHECK(hipPeekAtLastError());

		particles.update_stage2_and_1(dpdx);

		CUDA_CHECK(hipPeekAtLastError());

		if (rbcscoll)
		    rbcscoll->update_stage2_and_1();

		if (wall)
		{
		    tstart = MPI_Wtime();
		    wall->bounce(particles.xyzuvw.data, particles.size);
		    
		    if (rbcscoll)
			wall->bounce(rbcscoll->data(), rbcscoll->pcount());
		    timings["bounce-walls"] += MPI_Wtime() - tstart;
		    
		    /*   CUDA_CHECK(hipDeviceSynchronize());
		    CUDA_CHECK(hipPeekAtLastError());

		    static int ctr = 0;
		    if (rank == 0)
			printf("ctr is %d\n", ctr);
		    //  if (ctr == 85)
		    //	break;

			++ctr;*/
		}

		CUDA_CHECK(hipPeekAtLastError());
	    
		if (it % steps_per_report == 0)
		{
		    int n = particles.size;

		    if (rbcscoll)
			n += rbcscoll->pcount();

		    Particle * p = new Particle[n];
		    Acceleration * a = new Acceleration[n];

		    CUDA_CHECK(hipMemcpy(p, particles.xyzuvw.data, sizeof(Particle) * particles.size, hipMemcpyDeviceToHost));
		    CUDA_CHECK(hipMemcpy(a, particles.axayaz.data, sizeof(Acceleration) * particles.size, hipMemcpyDeviceToHost));
		   
		    if (rbcscoll)
		    {
			const int start = particles.size;

			CUDA_CHECK(hipMemcpy(p + start, rbcscoll->xyzuvw.data, sizeof(Particle) * rbcscoll->pcount(), hipMemcpyDeviceToHost));
			CUDA_CHECK(hipMemcpy(a + start, rbcscoll->axayaz.data, sizeof(Acceleration) * rbcscoll->pcount(), hipMemcpyDeviceToHost));
		    }

		    //we fused VV stages so we need to recover the state before stage 1
		    for(int i = 0; i < n; ++i)
			for(int c = 0; c < 3; ++c)
			{
			    assert(!isnan(p[i].x[c]));
			    assert(!isnan(p[i].u[c]));
			    assert(!isnan(a[i].a[c]));
	    
			    p[i].x[c] -= dt * p[i].u[c];
			    p[i].u[c] -= 0.5 * dt * a[i].a[c];
			}

		    diagnostics(cartcomm, p, n, dt, it, L, a);
		    
		    if (rbcscoll && it % steps_per_dump == 0)
			rbcscoll->dump(cartcomm);
		   
		    delete [] p;
		    delete [] a;
		}
	    }

	    CUDA_CHECK(hipStreamDestroy(stream));
	
	    if (wall)
		delete wall;

	    if (rbcscoll)
		delete rbcscoll;

	    MPI_CHECK(MPI_Comm_free(&cartcomm));
	}
	
	MPI_CHECK( MPI_Finalize() );
	
	if (rank == 0)
	    printf("simulation is done. Ciao.\n");
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
	
