#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <cassert>

#include <mpi.h>

#include <vector>

//in common.h i define Particle and Acceleration data structures
//as well as the global parameters for the simulation
#include "common.h"

#include "dpd-interactions.h"
#include "wall-interactions.h"
#include "redistribute-particles.h"
#include "rbc-interactions.h"

#include <rbc-cuda.h>

#include "redistribute-rbcs.h"

using namespace std;

__constant__ float gradp[3];

//velocity verlet stages - first stage
__global__ void update_stage1(Particle * p, Acceleration * a, int n, float dt)
{
    assert(blockDim.x * gridDim.x >= n);
    
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;

    if (pid >= n)
	return;
    
    for(int c = 0; c < 3; ++c)
    {
	assert(!isnan(p[pid].x[c]));
	assert(!isnan(p[pid].u[c]));
	assert(!isnan(a[pid].a[c]));
    }

       for(int c = 0; c < 3; ++c)
	p[pid].u[c] += (a[pid].a[c] - gradp[c]) * dt * 0.5;
    
       for(int c = 0; c < 3; ++c)
	   p[pid].x[c] += p[pid].u[c] * dt;

    for(int c = 0; c < 3; ++c)
    {
	assert(p[pid].x[c] >= -L -L/2);
	assert(p[pid].x[c] <= +L +L/2);
    }
}

//fused velocity verlet stage 2 and 1 (for the next iteration)
__global__ void update_stage2_and_1(Particle * p, Acceleration * a, int n, float dt)
{
    assert(blockDim.x * gridDim.x >= n);
    
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;

    if (pid >= n)
	return;

    for(int c = 0; c < 3; ++c)
	assert(!isnan(p[pid].u[c]));

    for(int c = 0; c < 3; ++c)
	assert(!isnan(a[pid].a[c]));

    for(int c = 0; c < 3; ++c)
    {
	const float mya = a[pid].a[c] - gradp[c];
	float myu = p[pid].u[c];
	float myx = p[pid].x[c];

	myu += mya * dt;
	myx += myu * dt;
	
	p[pid].u[c] = myu; 
	p[pid].x[c] = myx; 
    }

    for(int c = 0; c < 3; ++c)
    {
	if (!(p[pid].x[c] >= -L -L/2) || !(p[pid].x[c] <= +L +L/2))
	    printf("Uau: %f %f %f %f %f %f and acc %f %f %f\n", 
		   p[pid].x[0], p[pid].x[1], p[pid].x[2], 
		   p[pid].u[0], p[pid].u[1], p[pid].u[2],
		   a[pid].a[0], a[pid].a[1],a[pid].a[2]);

	assert(p[pid].x[c] >= -L -L/2);
	assert(p[pid].x[c] <= +L +L/2);
    }
}

__global__ void fake_vel(Particle * p, const int n)
{
    const int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if (gid < n)
	for(int c = 0; c < 3; ++c)
	    p[gid].u[c] = 3 * (c == 0);
}

//container for the gpu particles during the simulation
struct ParticleArray
{
    int size;

    SimpleDeviceBuffer<Particle> xyzuvw;
    SimpleDeviceBuffer<Acceleration> axayaz;

    ParticleArray() {}
    
    ParticleArray(vector<Particle> ic)
	{
	    resize(ic.size());

	    CUDA_CHECK(hipMemcpy(xyzuvw.data, (float*) &ic.front(), sizeof(Particle) * ic.size(), hipMemcpyHostToDevice));
	    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * ic.size()));
	}

    void resize(int n)
	{
	    size = n;

	    xyzuvw.resize(n);
	    axayaz.resize(n);

	    CUDA_CHECK(hipMemset(axayaz.data, 0, sizeof(Acceleration) * size));
	}
};

class CollectionRBC : ParticleArray
{
    int nrbcs, nvertices, L;
    
    CudaRBC::Extent extent;

public:
    CollectionRBC(const int L): L(L), nrbcs(0)
	{
	    printf("hellouus\n");
	    
	    CudaRBC::setup(nvertices, extent);
	    
	    printf("extent: %f %f %f %f %f %f\n",
		   extent.xmax , extent.xmin,
		   extent.ymax , extent.ymin,
		   extent.zmax , extent.zmin);
		   
	}

    void createone()
	{	    
	    assert(extent.xmax - extent.xmin < L);
	    assert(extent.ymax - extent.ymin < L);
	    assert(extent.zmax - extent.zmin < L);

	    const int n = (int)(0.75 * L / (extent.zmax - extent.zmin));
	    
	    const float dx = L / (float)n;
	    
	    resize(n);

	    for(int i = 0; i < n; ++i)
	    {
		float target[3] = { (i + 0.5) * dx,  0.35 * L * (drand48() - 0.5), 0.35 * L * (drand48() - 0.5)};

		float transform[4][4] = { {0, 0, -1, -target[0] + 0.5 * (extent.zmin + extent.zmax) }, 
					  {0, 1, 0, target[1] -0.5 * (extent.ymin + extent.ymax)}, 
					  {1, 0, 0, target[2] -0.5 * (extent.xmin + extent.xmax)}, 
					  {0, 0, 0, 1} };

		//CUDA_CHECK(hipMemset(xyzuvw.data, 0, sizeof(Particle) * nvertices));
		CudaRBC::initialize((float *)(xyzuvw.data + nvertices * i), transform);
	    }
	}

    Particle * data() { return xyzuvw.data; }
    Acceleration * acc() { return axayaz.data; }
    int count() { return nrbcs; }
    int pcount() { return nrbcs * nvertices; }
    
    void resize(const int count)
	{
	    nrbcs = count;

	    ParticleArray::resize(count * nvertices);
	}

    void update(const int it)
	{
	    if (nrbcs == 0)
		return;
	    
	    if (it < 0)
		update_stage1<<<(xyzuvw.size + 127) / 128, 128 >>>(
		    xyzuvw.data, axayaz.data, xyzuvw.size, dt);
	    else
		update_stage2_and_1<<<(xyzuvw.size + 127) / 128, 128 >>>
		    (xyzuvw.data, axayaz.data, xyzuvw.size, dt);

	}

    void dump(MPI_Comm comm)
	{
	    static bool firsttime = true;
	    
	    const int n = size;

	    Particle * p = new Particle[n];
	    Acceleration * a = new Acceleration[n];

	    CUDA_CHECK(hipMemcpy(p, xyzuvw.data, sizeof(Particle) * n, hipMemcpyDeviceToHost));
	    CUDA_CHECK(hipMemcpy(a, axayaz.data, sizeof(Acceleration) * n, hipMemcpyDeviceToHost));
		   
	    //we fused VV stages so we need to recover the state before stage 1
	    for(int i = 0; i < n; ++i)
		for(int c = 0; c < 3; ++c)
		{
		    assert(!isnan(p[i].x[c]));
		    assert(!isnan(p[i].u[c]));
		    assert(!isnan(a[i].a[c]));
	    
		    p[i].x[c] -= dt * p[i].u[c];
		    p[i].u[c] -= 0.5 * dt * a[i].a[c];
		}
	    
	    xyz_dump(comm, "rbcs.xyz", "rbcparticles", p, n,  L, !firsttime);

	    //std::vector<int> indices(3 * ntriangles)
	    int (*indices)[3];
	    int ntriangles;
	    CudaRBC::get_triangle_indexing(indices, ntriangles);
	    
	    ply_dump(comm, "prova.ply", indices, nrbcs, ntriangles, p, nvertices, L, false);
		    
	    delete [] p;
	    delete [] a;

	    firsttime = false;
	}
};

int main(int argc, char ** argv)
{
    int ranks[3];
    
    if (argc != 4)
    {
	printf("usage: ./mpi-dpd <xranks> <yranks> <zranks>\n");
	exit(-1);
    }
    else
    {
	for(int i = 0; i < 3; ++i)
	    ranks[i] = atoi(argv[1 + i]);
    }

    CUDA_CHECK(hipSetDevice(0));

    int nranks, rank;   
 
    {
	MPI_CHECK( MPI_Init(&argc, &argv) );
    
	{

	    MPI_CHECK( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	    MPI_CHECK( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	
	    MPI_Comm cartcomm;
	
	    int periods[] = {1,1,1};
	    MPI_CHECK( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 1, &cartcomm) );
	
	    vector<Particle> ic(L * L * L * 3  );
	    srand48(rank);

	    for(int i = 0; i < ic.size(); ++i)
		for(int c = 0; c < 3; ++c)
		{
		    ic[i].x[c] = -L * 0.5 + drand48() * L;
		    ic[i].u[c] = 0;// (drand48()*2  - 1);
		}

	    float dpdx[3] = {-0.01, 0, 0};
	    
	    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gradp), dpdx, sizeof(float) * 3));
	    
	    //the methods of these classes are not expected to call hipDeviceSynchronize unless really necessary
	    //(be aware of that)
	    H5PartDump dump("trajectories.h5part", cartcomm, L);
	    ParticleArray particles(ic);
	    CellLists cells(L);		  
	    RedistributeParticles redistribute(cartcomm, L);
	    ComputeInteractionsDPD dpd(cartcomm, L);
	    ComputeInteractionsWall * wall = NULL;
	    CollectionRBC rbcs(L);
	    
	    //if (rank == 0)
		rbcs.createone();
	    //exit(0);
	    hipStream_t stream;
	    CUDA_CHECK(hipStreamCreate(&stream));
	    
	    RedistributeRBCs redistribute_rbcs(cartcomm, L);
	    redistribute_rbcs.stream = stream;
	    int saru_tag = rank;

	    ComputeInteractionsRBC rbc_interactions(cartcomm, L);

	    rbcs.update(-1);
	    
	    cells.build(particles.xyzuvw.data, particles.size);
	   
	    dpd.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count);
	    rbc_interactions.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				      rbcs.data(), rbcs.count(), rbcs.acc());

	    const size_t nsteps = (int)(tend / dt);

	    for(int it = 0; it < nsteps; ++it)
	    {
		if (rank == 0 && it % 50 == 0)
		    printf("beginning of time step %d\n", it);
	    
		if (it == 0)
		    update_stage1<<<(particles.size + 127) / 128, 128 >>>(
			particles.xyzuvw.data, particles.axayaz.data, particles.size, dt);

		int newnp = redistribute.stage1(particles.xyzuvw.data, particles.size);
		
		particles.resize(newnp);
	    
		redistribute.stage2(particles.xyzuvw.data, particles.size);

		int nrbcs = redistribute_rbcs.stage1(rbcs.data(), rbcs.count());

		rbcs.resize(nrbcs);

		redistribute_rbcs.stage2(rbcs.data(), rbcs.count());

		if (walls && it > 500 && wall == NULL)
		{
		    int nsurvived = 0;
		    wall = new ComputeInteractionsWall(cartcomm, L, particles.xyzuvw.data, particles.size, nsurvived);
		    
		    particles.resize(nsurvived);
		    
		    dpdx[0] = -0.1;
		    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gradp), dpdx, sizeof(float) * 3));

		    if (rank == 0)
			if( access( "trajectories.xyz", F_OK ) != -1 )
			{
			    const int retval = rename ("trajectories.xyz", "trajectories-equilibration.xyz");
			    assert(retval != -1);
			}
		}

		cells.build(particles.xyzuvw.data, particles.size);

		dpd.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count);
		rbc_interactions.evaluate(saru_tag, particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
					  rbcs.data(), rbcs.count(), rbcs.acc());

		//I NEED A REGISTRATION MECHANISM FOR DPD-INTERACTION AND RBC-INTERACTION
		if (wall != NULL)
		{
		    wall->interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, 
				       cells.start, cells.count, saru_tag);
		    
		    wall->interactions(rbcs.data(), rbcs.pcount(), rbcs.acc(), 
				       NULL, NULL, saru_tag);
		}

		if (particles.size > 0)
		    update_stage2_and_1<<<(particles.size + 127) / 128, 128 >>>
			(particles.xyzuvw.data, particles.axayaz.data, particles.size, dt);

		rbcs.update(it);

		if (wall != NULL)
		{
		    wall->bounce(particles.xyzuvw.data, particles.size);
		    wall->bounce(rbcs.data(), rbcs.pcount());
		}
	    
		if (it % 50 == 0)
		{
		    const int n = particles.size;

		    Particle * p = new Particle[n];
		    Acceleration * a = new Acceleration[n];

		    CUDA_CHECK(hipMemcpy(p, particles.xyzuvw.data, sizeof(Particle) * n, hipMemcpyDeviceToHost));
		    CUDA_CHECK(hipMemcpy(a, particles.axayaz.data, sizeof(Acceleration) * n, hipMemcpyDeviceToHost));
		   
		    //we fused VV stages so we need to recover the state before stage 1
		    for(int i = 0; i < n; ++i)
			for(int c = 0; c < 3; ++c)
			{
			    assert(!isnan(p[i].x[c]));
			    assert(!isnan(p[i].u[c]));
			    assert(!isnan(a[i].a[c]));
	    
			    p[i].x[c] -= dt * p[i].u[c];
			    p[i].u[c] -= 0.5 * dt * a[i].a[c];
			}

		    diagnostics(cartcomm, p, n, dt, it, L, a, true);
		    rbcs.dump(cartcomm);
		    
		    if (it > 100)
			dump.dump(p, n);

		    delete [] p;
		    delete [] a;
		}
	    }

	    CUDA_CHECK(hipStreamDestroy(stream));
	
	    if (wall != NULL)
		delete wall;

	    MPI_CHECK(MPI_Comm_free(&cartcomm));
	}
	
	MPI_CHECK( MPI_Finalize() );
	
	if (rank == 0)
	    printf("simulation is done. Ciao.\n");
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
	
