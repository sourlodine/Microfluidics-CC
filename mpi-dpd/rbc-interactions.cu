#include "hip/hip_runtime.h"
#include <../saru.cuh>

#include "rbc-interactions.h"

namespace KernelsRBC
{
    struct ParamsFSI
    {
	float aij, gamma, sigmaf;
    };

    __constant__ ParamsFSI params;
    
    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    static bool firsttime = true;
    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount, const int L)
    {
	if (firsttime)
	{
	    texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	    texCellsStart.filterMode = hipFilterModePoint;
	    texCellsStart.mipmapFilterMode = hipFilterModePoint;
	    texCellsStart.normalized = 0;
    
	    texCellsCount.channelDesc = hipCreateChannelDesc<int>();
	    texCellsCount.filterMode = hipFilterModePoint;
	    texCellsCount.mipmapFilterMode = hipFilterModePoint;
	    texCellsCount.normalized = 0;

	    texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
	    texSolventParticles.filterMode = hipFilterModePoint;
	    texSolventParticles.mipmapFilterMode = hipFilterModePoint;
	    texSolventParticles.normalized = 0;
	    firsttime = false;
	}
	
	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
				   sizeof(float) * 6 * npsolvent));

	const int ncells = L * L * L;
	
	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
    }
    
    __global__ void shift_send_particles(const Particle * const src, const int n, const int L, const int code, Particle * const dst)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	
	if (gid < n)
	{
	    Particle p = src[gid];
	    
	    for(int c = 0; c < 3; ++c)
		p.x[c] -= d[c] * L;

	    dst[gid] = p;
	}
    }

    __device__ bool fsi_interaction(const int saru_tag,
				      const int dpid, const float3 xp, const float3 up, const int spid,
				      float& xforce, float& yforce, float& zforce)
    {
	xforce = yforce = zforce = 0;
	
	const int sentry = 3 * spid;
	
	const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry);
	const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
	const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);
	
	const float _xr = xp.x - stmp0.x;
	const float _yr = xp.y - stmp0.y;
	const float _zr = xp.z - stmp1.x;

	const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
	
	if (rij2 > 1)
	    return false;
	
	const float invrij = rsqrtf(rij2);
	
	const float rij = rij2 * invrij;
	const float wr = max((float)0, 1 - rij);

//	if (fabs(wr) > 10)
//	    printf("WWWWWHAT: %f\n", wr);
	
	const float xr = _xr * invrij;
	const float yr = _yr * invrij;
	const float zr = _zr * invrij;
	
	const float rdotv = 
	    xr * (up.x - stmp1.y) +
	    yr * (up.y - stmp2.x) +
	    zr * (up.z - stmp2.y);
	
	const float mysaru = saru(saru_tag, dpid, spid);
	const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
	
	const float strength = (params.aij - params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

	/*	if ( strength * xr > 1e2)
		    printf("XXXXXXXXXXXXXXXXXXXXXXXXXxx %f %f %f \n", strength , xr, invrij);
		
		if ( strength * yr > 1e2)
		    printf("YYYYYYYYYYYYYYYYYYYYYYYYYYY %f %f %f\n", strength , yr, invrij);
		
		if ( strength * zr > 1e2)
		    printf("ZZZZZZZZZZZZZZZZZZZZZZZZZZZ %f %f %f %f  -> %f gamma = %f, %f %f \n", strength , zr, invrij, wr,
			   params.aij , params.gamma,  wr * rdotv , params.sigmaf * myrandnr	);
	*/
		
	xforce = strength * xr;
	yforce = strength * yr;
	zforce = strength * zr;

	return true;
    }

    __global__ void fsi_forces(const int saru_tag,
			       Acceleration * accsolvent, const int npsolvent,
			       const Particle * const particle, const int nparticles, Acceleration * accrbc, const int L)
    {
	//if (threadIdx.x == 0)
	//  printf("params are %f %f %f\n", params.aij, params.gamma, params.sigmaf);
	
	const int dpid = threadIdx.x + blockDim.x * blockIdx.x;

	if (dpid >= nparticles)
	    return;

	const Particle p = particle[dpid];

	//for(int c = 0; c < 3; ++c)
	//  assert(p.x[c] >= -L/2 && p.x[c] < L/2);
	
	const float3 xp = make_float3(p.x[0], p.x[1], p.x[2]);
	const float3 up = make_float3(p.u[0], p.u[1], p.u[2]);
		
	int mycid[3];
	for(int c = 0; c < 3; ++c)
	    mycid[c] = (int)floor(p.x[c] + L/2);

	float fsum[3] = {0, 0, 0};
	
	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		(code/3 % 3) - 1,
		(code/9 % 3) - 1
	    };
	    
	    int vcid[3];
	    for(int c = 0; c < 3; ++c)
		vcid[c] = mycid[c] + d[c];

	    bool validcid = true;
	    for(int c = 0; c < 3; ++c)
		validcid &= vcid[c] >= 0 && vcid[c] < L;

	    // if ( !validcid)
	    //	printf("NOT VALID vcid %d %d %d!\n", vcid[0], vcid[1], vcid[2]);
	    
	    if (!validcid)
		continue;
	    
	    const int cid = vcid[0] + L * (vcid[1] + L * vcid[2]);
	    const int mystart = tex1Dfetch(texCellsStart, cid);
	    const int myend = mystart + tex1Dfetch(texCellsCount, cid);
	    
	    //printf("mystart end %d %d out of %d particles\n", mystart, myend - mystart, npsolvent);
	    assert(mystart >= 0 && mystart <= myend);
	    assert(myend <= npsolvent);
	    
	    for(int s = mystart; s < myend; ++s)
	    {
		float f[3];
		const bool nonzero = fsi_interaction(saru_tag, dpid, xp, up, s, f[0], f[1], f[2]);

		if (nonzero)
		{
		    //printf("interaction: %f %f %f\n", f[0], f[1], f[2]);
		    for(int c = 0; c < 3; ++c)
			fsum[c] += f[c];
		    
		    for(int c = 0; c < 3; ++c)
		    	atomicAdd(c + (float *)(accsolvent + s), -f[c]);
		}
	    }
	}

	/*
	for(int c = 0; c< 3; ++c)
	    if (fabs(fsum[c]) > 100)
		printf("UAU: %f\n", fsum[c]);
	*/
	
	for(int c = 0; c < 3; ++c)
	    accrbc[dpid].a[c] = fsum[c];
    }

    __global__ void merge_accelerations(const Acceleration * const src, const int n, Acceleration * const dst)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid < n)
	    for(int c = 0; c < 3; ++c)
	    {
		dst[gid].a[c] += src[gid].a[c];

		//	printf("now: %f\n", dst[gid].a[c]);
	    }
    }
}

ComputeInteractionsRBC::ComputeInteractionsRBC(MPI_Comm _cartcomm, int L):  L(L), nvertices(CudaRBC::get_nvertices()), stream(0)
{
    assert(L % 2 == 0);
    assert(L >= 2);

    MPI_CHECK( MPI_Comm_dup(_cartcomm, &cartcomm));

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));

    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );

	const int nhalocells = pow(L, 3 - fabs(d[0]) - fabs(d[1]) - fabs(d[2]));
	const int estimate = 8 * nvertices;

	remote[i].setup(estimate);
	local[i].setup(estimate);

	/*halosize[i].x = d[0] != 0 ? 1 : L;
	  halosize[i].y = d[1] != 0 ? 1 : L;
	  halosize[i].z = d[2] != 0 ? 1 : L;
	  assert(nhalocells == halosize[i].x * halosize[i].y * halosize[i].z);

	  recvhalos[i].expected = estimate;
	  recvhalos[i].buf.resize(estimate);
	  recvhalos[i].secondary.resize(estimate);
	  recvhalos[i].cellstarts.resize(nhalocells + 1);

	  sendhalos[i].expected = estimate;
	  sendhalos[i].buf.resize(estimate);
	  sendhalos[i].secondary.resize(estimate);
	  sendhalos[i].scattered_entries.resize(estimate);
	  sendhalos[i].cellstarts.resize(nhalocells + 1);
	  sendhalos[i].tmpcount.resize(nhalocells + 1);
	  sendhalos[i].tmpstart.resize(nhalocells + 1);*/
    }

    /*CUDA_CHECK(hipHostAlloc((void **)&required_send_bag_size_host, sizeof(int) * 26, hipHostMallocMapped));
      CUDA_CHECK(hipHostGetDevicePointer(&required_send_bag_size, required_send_bag_size_host, 0));

      for(int i = 0; i < 7; ++i)
      CUDA_CHECK(hipStreamCreate(streams + i));

      for(int i = 0, ctr = 1; i < 26; ++i)
      {
      int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

      const bool isface = abs(d[0]) + abs(d[1]) + abs(d[2]) == 1;

      code2stream[i] = 0;

      if (isface)
      {
      code2stream[i] = ctr;
      ctr++;
      }
      }*/

    KernelsRBC::ParamsFSI params = {aij, gammadpd, sigmaf};
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsRBC::params), &params, sizeof(KernelsRBC::ParamsFSI)));
}


void ComputeInteractionsRBC::pack_and_post(const Particle * const rbcs, const int nrbcs)
{
    extents.resize(nrbcs);
 
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(rbcs + nvertices * i), extents.devptr + i);

    CUDA_CHECK(hipStreamSynchronize(stream));

    for(int i = 0; i < 26; ++i)
	haloreplica[i].clear();

    for(int i = 0; i < nrbcs; ++i)
    {
	const CudaRBC::Extent ext = extents.data[i];
	
	float pmin[3] = {ext.xmin, ext.ymin, ext.zmin};
	float pmax[3] = {ext.xmax, ext.ymax, ext.zmax};

	for(int code = 0; code < 26; ++code)
	{
	    int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	    bool interacting = true;
	    
	    for(int c = 0; c < 3; ++c)
	    {
		const float range_start = max((float)(d[c] * L - L/2 - 1), pmin[c]);
		const float range_end = min((float)(d[c] * L + L/2 + 1), pmax[c]);

		interacting &= range_end > range_start;
	    }

	    if (interacting)
		haloreplica[code].push_back(i);

	    //if (interacting)
	    //	printf("RBC INTERACTING WITH RANK %d %d %d\n", coords[0] + d[0], coords[1] + d[1], coords[2] + d[2]);
	}
    }

    MPI_Request reqrecvcounts[26];
    for(int i = 0; i <26; ++i)
	MPI_CHECK(MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, dstranks[i], recv_tags[i] + 2077, cartcomm, reqrecvcounts + i));

    MPI_Request reqsendcounts[26];
    for(int i = 0; i < 26; ++i)
    {
	send_counts[i] = haloreplica[i].size();
	MPI_CHECK(MPI_Isend(send_counts + i, 1, MPI_INTEGER, dstranks[i], i + 2077, cartcomm, reqsendcounts + i));
    }

    {
	MPI_Status statuses[26];
	MPI_CHECK(MPI_Waitall(26, reqrecvcounts, statuses));
	MPI_CHECK(MPI_Waitall(26, reqsendcounts, statuses));
    }

    for(int i = 0; i < 26; ++i)
	local[i].setup(send_counts[i] * nvertices);

    for(int i = 0; i < 26; ++i)
    {
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::shift_send_particles<<< (nvertices + 127) / 128, 128, 0, stream>>>
		(rbcs + nvertices * haloreplica[i][j], nvertices, L, i, local[i].state.data + nvertices * j);
	 
	CUDA_CHECK(hipPeekAtLastError());
    }
     
    CUDA_CHECK(hipStreamSynchronize(stream));

    for(int i = 0; i < 26; ++i)
	remote[i].setup(recv_counts[i] * nvertices);

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;
	    
	    MPI_CHECK(MPI_Irecv(remote[i].state.data, recv_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				recv_tags[i] + 2011, cartcomm, &request));

	    reqrecvp.push_back(request);
	}

    for(int i = 0; i < 26; ++i)
	if (send_counts[i] > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(local[i].result.data, send_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				recv_tags[i] + 2285, cartcomm, &request));

	    reqrecvacc.push_back(request);
	    
	    MPI_CHECK(MPI_Isend(local[i].state.data, send_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				i + 2011, cartcomm, &request));

	    reqsendp.push_back(request);
	}

//    printf("done done done\n");
}

void ComputeInteractionsRBC::evaluate(int& saru_tag,
				      const Particle * const solvent, const int nparticles, Acceleration * accsolvent,
				      const int * const cellsstart_solvent, const int * const cellscount_solvent,
				      const Particle * const rbcs, const int nrbcs, Acceleration * accrbc)
{
    /*CUDA_CHECK(hipMemset(accrbc, 0, sizeof(Acceleration) * nvertices * nrbcs));
	
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::forces_nohost(0, (float *)(rbcs + nvertices * i), (float *)(accrbc + nvertices * i));
    */ 
	
    KernelsRBC::setup(solvent, nparticles, cellsstart_solvent, cellscount_solvent, L);

    pack_and_post(rbcs, nrbcs);

    if (nrbcs > 0 && nparticles > 0)
    {
	KernelsRBC::fsi_forces<<< (nrbcs * nvertices + 127) / 128, 128, 0, stream >>>
	    (saru_tag + myrank, accsolvent, nparticles, rbcs, nrbcs * nvertices, accrbc, L);
	
	//CUDA_CHECK(hipMemset(accrbc, 0, sizeof(Acceleration) * nvertices * nrbcs));
	
	for(int i = 0; i < nrbcs; ++i)
	    CudaRBC::forces_nohost(stream, (float *)(rbcs + nvertices * i), (float *)(accrbc + nvertices * i));

	saru_tag += nranks;
    }
    
    _wait(reqrecvp);
    _wait(reqsendp);
    
    for(int i = 0; i < 26; ++i)
    {
	const int count = remote[i].state.size;

	if (count > 0)
	    KernelsRBC::fsi_forces<<< (count + 127) / 128, 128, 0, stream >>>
	    	(saru_tag + 26 * myrank + i, accsolvent, nparticles, remote[i].state.data, count, remote[i].result.data, L);
    }

    saru_tag += 26 * nranks;

    CUDA_CHECK(hipStreamSynchronize(stream));

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;
	    
	    MPI_CHECK(MPI_Isend(remote[i].result.data, recv_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				i + 2285, cartcomm, &request));

	    reqsendacc.push_back(request);
	}

    _wait(reqrecvacc);
    _wait(reqsendacc);

    for(int i = 0; i < 26; ++i)
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::merge_accelerations<<< (nvertices + 127) / 128, 128 >>>(local[i].result.data + nvertices * j, nvertices,
										accrbc + nvertices * haloreplica[i][j]);
}

ComputeInteractionsRBC::~ComputeInteractionsRBC()
{
    MPI_CHECK(MPI_Comm_free(&cartcomm));
}

