#include "hip/hip_runtime.h"
#include <../dpd-rng.h>

#include "rbc-interactions.h"

namespace KernelsRBC
{
    struct ParamsFSI
    {
	float aij, gamma, sigmaf;
    };

    __constant__ ParamsFSI params;
    
    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    static bool firsttime = true;
    
    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount)
    {
	if (firsttime)
	{
	    texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	    texCellsStart.filterMode = hipFilterModePoint;
	    texCellsStart.mipmapFilterMode = hipFilterModePoint;
	    texCellsStart.normalized = 0;
    
	    texCellsCount.channelDesc = hipCreateChannelDesc<int>();
	    texCellsCount.filterMode = hipFilterModePoint;
	    texCellsCount.mipmapFilterMode = hipFilterModePoint;
	    texCellsCount.normalized = 0;

	    texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
	    texSolventParticles.filterMode = hipFilterModePoint;
	    texSolventParticles.mipmapFilterMode = hipFilterModePoint;
	    texSolventParticles.normalized = 0;
	    firsttime = false;
	}
	
	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
				   sizeof(float) * 6 * npsolvent));

	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;
	
	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
    }
    
    __global__ void shift_send_particles(const Particle * const src, const int n, const int code, Particle * const dst)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	
	if (gid < n)
	{
	    Particle p = src[gid];
	    
	    for(int c = 0; c < 3; ++c)
		p.x[c] -= d[c] * L[c];

	    dst[gid] = p;
	}
    }

    __device__ bool fsi_kernel(const float seed,
			       const int dpid, const float3 xp, const float3 up, const int spid,
			       float& xforce, float& yforce, float& zforce)
    {
	xforce = yforce = zforce = 0;
	
	const int sentry = 3 * spid;
	
	const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry);
	const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
	const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);
	
	const float _xr = xp.x - stmp0.x;
	const float _yr = xp.y - stmp0.y;
	const float _zr = xp.z - stmp1.x;

	const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
	
	if (rij2 > 1)
	    return false;
	
	const float invrij = rsqrtf(rij2);
	
	const float rij = rij2 * invrij;
	const float argwr = max((float)0, 1 - rij);
	const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));
	
	const float xr = _xr * invrij;
	const float yr = _yr * invrij;
	const float zr = _zr * invrij;
	
	const float rdotv = 
	    xr * (up.x - stmp1.y) +
	    yr * (up.y - stmp2.x) +
	    zr * (up.z - stmp2.y);
	
	//const float mysaru = saru(saru_tag, dpid, spid);
	//const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
	const float myrandnr = Logistic::mean0var1(seed, dpid, spid);
	
	const float strength = params.aij * argwr +  (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;
	
	xforce = strength * xr;
	yforce = strength * yr;
	zforce = strength * zr;

	return true;
    }

    __global__ void fsi_forces(const float seed,
			       Acceleration * accsolvent, const int npsolvent,
			       const Particle * const particle, const int nparticles, Acceleration * accrbc)
    {
	const int dpid = threadIdx.x + blockDim.x * blockIdx.x;

	if (dpid >= nparticles)
	    return;

	const Particle p = particle[dpid];

	const float3 xp = make_float3(p.x[0], p.x[1], p.x[2]);
	const float3 up = make_float3(p.u[0], p.u[1], p.u[2]);
		
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	int mycid[3];
	for(int c = 0; c < 3; ++c)
	    mycid[c] = (int)floor(p.x[c] + L[c]/2);

	float fsum[3] = {0, 0, 0};
	
	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		(code/3 % 3) - 1,
		(code/9 % 3) - 1
	    };
	    
	    int vcid[3];
	    for(int c = 0; c < 3; ++c)
		vcid[c] = mycid[c] + d[c];

	    bool validcid = true;
	    for(int c = 0; c < 3; ++c)
		validcid &= vcid[c] >= 0 && vcid[c] < L[c];

	    if (!validcid)
		continue;
	    
	    const int cid = vcid[0] + XSIZE_SUBDOMAIN * (vcid[1] + YSIZE_SUBDOMAIN * vcid[2]);
	    const int mystart = tex1Dfetch(texCellsStart, cid);
	    const int myend = mystart + tex1Dfetch(texCellsCount, cid);
	    
	    assert(mystart >= 0 && mystart <= myend);
	    assert(myend <= npsolvent);
	    
	    for(int s = mystart; s < myend; ++s)
	    {
		float f[3];
		const bool nonzero = fsi_kernel(seed, dpid, xp, up, s, f[0], f[1], f[2]);

		if (nonzero)
		{
		    for(int c = 0; c < 3; ++c)
			fsum[c] += f[c];
		    
		    for(int c = 0; c < 3; ++c)
		    	atomicAdd(c + (float *)(accsolvent + s), -f[c]);
		}
	    }
	}
	
	for(int c = 0; c < 3; ++c)
	    accrbc[dpid].a[c] = fsum[c];
    }

    __global__ void merge_accelerations(const Acceleration * const src, const int n, Acceleration * const dst)
    {	
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid < n)
	    for(int c = 0; c < 3; ++c)
		dst[gid].a[c] += src[gid].a[c];
    }
}

ComputeInteractionsRBC::ComputeInteractionsRBC(MPI_Comm _cartcomm): nvertices(CudaRBC::get_nvertices())
{
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);

    MPI_CHECK( MPI_Comm_dup(_cartcomm, &cartcomm));

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);
    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));

    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );
    }

    KernelsRBC::ParamsFSI params = {aij, gammadpd, sigmaf};
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsRBC::params), &params, sizeof(KernelsRBC::ParamsFSI)));

    CUDA_CHECK(hipEventCreate(&evextents, hipEventDisableTiming));
    CUDA_CHECK(hipEventCreate(&evfsi, hipEventDisableTiming));
}

void ComputeInteractionsRBC::_compute_extents(const Particle * const rbcs, const int nrbcs, hipStream_t stream)
{
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(rbcs + nvertices * i), extents.devptr + i);
}

void ComputeInteractionsRBC::pack_and_post(const Particle * const rbcs, const int nrbcs, hipStream_t stream)
{
    extents.resize(nrbcs);
 
    _compute_extents(rbcs, nrbcs, stream);

    CUDA_CHECK(hipEventRecord(evextents));
    CUDA_CHECK(hipEventSynchronize(evextents));

    for(int i = 0; i < 26; ++i)
	haloreplica[i].clear();

    for(int i = 0; i < nrbcs; ++i)
    {
	const CudaRBC::Extent ext = extents.data[i];
	
	float pmin[3] = {ext.xmin, ext.ymin, ext.zmin};
	float pmax[3] = {ext.xmax, ext.ymax, ext.zmax};

	for(int code = 0; code < 26; ++code)
	{
	    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	    const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	    bool interacting = true;
	    
	    for(int c = 0; c < 3; ++c)
	    {
		const float range_start = max((float)(d[c] * L[c] - L[c]/2 - 1), pmin[c]);
		const float range_end = min((float)(d[c] * L[c] + L[c]/2 + 1), pmax[c]);

		interacting &= range_end > range_start;
	    }

	    if (interacting)
		haloreplica[code].push_back(i);
	}
    }

    MPI_Request reqrecvcounts[26];
    for(int i = 0; i <26; ++i)
	MPI_CHECK(MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, dstranks[i], recv_tags[i] + 2077, cartcomm, reqrecvcounts + i));

    MPI_Request reqsendcounts[26];
    for(int i = 0; i < 26; ++i)
    {
	send_counts[i] = haloreplica[i].size();
	MPI_CHECK(MPI_Isend(send_counts + i, 1, MPI_INTEGER, dstranks[i], i + 2077, cartcomm, reqsendcounts + i));
    }

    {
	MPI_Status statuses[26];
	MPI_CHECK(MPI_Waitall(26, reqrecvcounts, statuses));
	MPI_CHECK(MPI_Waitall(26, reqsendcounts, statuses));
    }

    for(int i = 0; i < 26; ++i)
	local[i].setup(send_counts[i] * nvertices);

    for(int i = 0; i < 26; ++i)
    {
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::shift_send_particles<<< (nvertices + 127) / 128, 128, 0, stream>>>
		(rbcs + nvertices * haloreplica[i][j], nvertices, i, local[i].state.devptr + nvertices * j);
	 
	CUDA_CHECK(hipPeekAtLastError());
    }
     
    CUDA_CHECK(hipEventRecord(evfsi));
    CUDA_CHECK(hipEventSynchronize(evfsi));
  
    for(int i = 0; i < 26; ++i)
	remote[i].setup(recv_counts[i] * nvertices);

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;
	    
	    MPI_CHECK(MPI_Irecv(remote[i].state.data, recv_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				recv_tags[i] + 2011, cartcomm, &request));

	    reqrecvp.push_back(request);
	}

    for(int i = 0; i < 26; ++i)
	if (send_counts[i] > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(local[i].result.data, send_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				recv_tags[i] + 2285, cartcomm, &request));

	    reqrecvacc.push_back(request);
	    
	    MPI_CHECK(MPI_Isend(local[i].state.data, send_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				i + 2011, cartcomm, &request));

	    reqsendp.push_back(request);
	}
}

void ComputeInteractionsRBC::_internal_forces(const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::forces_nohost(stream, (float *)(rbcs + nvertices * i), (float *)(accrbc + nvertices * i));
}

void ComputeInteractionsRBC::evaluate(const Particle * const solvent, const int nparticles, Acceleration * accsolvent,
				      const int * const cellsstart_solvent, const int * const cellscount_solvent,
				      const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{	
    KernelsRBC::setup(solvent, nparticles, cellsstart_solvent, cellscount_solvent);

    pack_and_post(rbcs, nrbcs, stream);

    if (nrbcs > 0 && nparticles > 0)
    {
	KernelsRBC::fsi_forces<<< (nrbcs * nvertices + 127) / 128, 128, 0, stream >>>
	    (local_trunk.get_float(), accsolvent, nparticles, rbcs, nrbcs * nvertices, accrbc);
		
	_internal_forces(rbcs, nrbcs, accrbc, stream);
    }
    
    _wait(reqrecvp);
    _wait(reqsendp);
    
    for(int i = 0; i < 26; ++i)
    {
	const int count = remote[i].state.size;

	if (count > 0)
	    KernelsRBC::fsi_forces<<< (count + 127) / 128, 128, 0, stream >>>
	    	(local_trunk.get_float(), accsolvent, nparticles, remote[i].state.devptr, count, remote[i].result.devptr);
    }

    CUDA_CHECK(hipEventRecord(evfsi));
    CUDA_CHECK(hipEventSynchronize(evfsi));

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;
	    
	    MPI_CHECK(MPI_Isend(remote[i].result.data, recv_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				i + 2285, cartcomm, &request));

	    reqsendacc.push_back(request);
	}

    _wait(reqrecvacc);
    _wait(reqsendacc);

    for(int i = 0; i < 26; ++i)
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::merge_accelerations<<< (nvertices + 127) / 128, 128, 0, stream>>>(local[i].result.devptr + nvertices * j, nvertices,
										accrbc + nvertices * haloreplica[i][j]);
}

ComputeInteractionsRBC::~ComputeInteractionsRBC()
{
    MPI_CHECK(MPI_Comm_free(&cartcomm));
}

