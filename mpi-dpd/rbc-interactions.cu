#include "hip/hip_runtime.h"
/*
 *  rbc-interactions.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-02.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <set>
#include <../dpd-rng.h>

#include "rbc-interactions.h"
#include "minmax-massimo.h"

namespace KernelsRBC
{
    struct ParamsFSI
    {
	float aij, gamma, sigmaf;
    };

    __constant__ ParamsFSI params;

    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    static bool firsttime = true;

    __global__ void fsi_forces(const float seed,
			       Acceleration * accsolvent, const int npsolvent,
			       const Particle * const particle, const int nparticles, Acceleration * accrbc);

    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount)
    {
	if (firsttime)
	{
	    texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	    texCellsStart.filterMode = hipFilterModePoint;
	    texCellsStart.mipmapFilterMode = hipFilterModePoint;
	    texCellsStart.normalized = 0;

	    texCellsCount.channelDesc = hipCreateChannelDesc<int>();
	    texCellsCount.filterMode = hipFilterModePoint;
	    texCellsCount.mipmapFilterMode = hipFilterModePoint;
	    texCellsCount.normalized = 0;

	    texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
	    texSolventParticles.filterMode = hipFilterModePoint;
	    texSolventParticles.mipmapFilterMode = hipFilterModePoint;
	    texSolventParticles.normalized = 0;
	    firsttime = false;
	}

	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
				   sizeof(float) * 6 * npsolvent));

	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);

	CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fsi_forces), hipFuncCachePreferL1));
    }

    __global__ void shift_send_particles_kernel(const Particle * const src, const int n, const int code, Particle * const dst)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	if (gid < n)
	{
	    Particle p = src[gid];

	    for(int c = 0; c < 3; ++c)
		p.x[c] -= d[c] * L[c];

	    dst[gid] = p;
	}
    }

    static const int cmaxnrbcs = 64;
    __constant__ float * csources[cmaxnrbcs], * cdestinations[cmaxnrbcs];
    __constant__ int ccodes[cmaxnrbcs];

    template <bool from_cmem>
    __global__ void shift_all_send_particles(const int nrbcs, const int nvertices,
					     const float ** const dsources, const int * dcodes, float ** const ddestinations)
    {
	const int nfloats_per_rbc = 6 * nvertices;

	assert(nfloats_per_rbc * nrbcs <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= nfloats_per_rbc * nrbcs)
	    return;

	const int idrbc = gid / nfloats_per_rbc;
	assert(idrbc < nrbcs);

	const int offset = gid % nfloats_per_rbc;

	float val;
	if (from_cmem)
	    val = csources[idrbc][offset];
	else
	    val = dsources[idrbc][offset];

	int code;
	if (from_cmem)
	    code = ccodes[idrbc];
	else
	    code = dcodes[idrbc];

	const int c = gid % 6;

	val -=
	    (c == 0) * ((code     + 2) % 3 - 1) * XSIZE_SUBDOMAIN +
	    (c == 1) * ((code / 3 + 2) % 3 - 1) * YSIZE_SUBDOMAIN +
	    (c == 2) * ((code / 9 + 2) % 3 - 1) * ZSIZE_SUBDOMAIN ;

	if (from_cmem)
	    cdestinations[idrbc][offset] = val;
	else
	    ddestinations[idrbc][offset] = val;
    }

    SimpleDeviceBuffer<float *> _ddestinations;
    SimpleDeviceBuffer<const float *> _dsources;
    SimpleDeviceBuffer<int> _dcodes;

    void dispose()
    {
	_ddestinations.dispose();
	_dsources.dispose();
	_dcodes.dispose();
    }

    void shift_send_particles(hipStream_t stream, const int nrbcs, const int nvertices,
			      const float ** const sources, const int * codes, float ** const destinations)
    {
	if (nrbcs == 0)
	    return;

	const int nthreads = nrbcs * nvertices * 6;

	if (nrbcs < cmaxnrbcs)
	{
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(ccodes), codes, sizeof(int) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdestinations), destinations, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csources), sources, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));

	    shift_all_send_particles<true><<<(nthreads + 127) / 128, 128, 0, stream>>>
		(nrbcs, nvertices, NULL, NULL, NULL);

	    CUDA_CHECK(hipPeekAtLastError());
	}
	else
	{
	    _dcodes.resize(nrbcs);
	    _ddestinations.resize(nrbcs);
	    _dsources.resize(nrbcs);

	    CUDA_CHECK(hipMemcpyAsync(_dcodes.data, codes, sizeof(int) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_ddestinations.data, destinations, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_dsources.data, sources, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));

	    shift_all_send_particles<false><<<(nthreads + 127) / 128, 128, 0, stream>>>
		(nrbcs, nvertices, _dsources.data, _dcodes.data, _ddestinations.data);
	}
    }

    template <bool from_cmem>
    __global__ void merge_all_acc(const int nrbcs, const int nvertices,
				  const float ** const dsources, float ** const ddestinations)
    {
	if (nrbcs == 0)
	    return;

	const int nfloats_per_rbc = 3 * nvertices;

	assert(nfloats_per_rbc * nrbcs <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= nfloats_per_rbc * nrbcs)
	    return;

	const int idrbc = gid / nfloats_per_rbc;
	assert(idrbc < nrbcs);

	const int offset = gid % nfloats_per_rbc;

	float val;
	if (from_cmem)
	    val = csources[idrbc][offset];
	else
	    val = dsources[idrbc][offset];

	if (from_cmem)
	    atomicAdd(cdestinations[idrbc] + offset, val);
	else
	    atomicAdd(ddestinations[idrbc] + offset, val);
    }

    void merge_all_accel(hipStream_t stream, const int nrbcs, const int nvertices,
			 const float ** const sources, float ** const destinations)
    {
	if (nrbcs == 0)
	    return;

	const int nthreads = nrbcs * nvertices * 3;

	CUDA_CHECK(hipPeekAtLastError());

	if (nrbcs < cmaxnrbcs)
	{
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdestinations), destinations, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csources), sources, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));

	    merge_all_acc<true><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, NULL, NULL);

	    CUDA_CHECK(hipPeekAtLastError());
	}
	else
	{
	    _ddestinations.resize(nrbcs);
	    _dsources.resize(nrbcs);

	    CUDA_CHECK(hipMemcpyAsync(_ddestinations.data, destinations, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_dsources.data, sources, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));

	    merge_all_acc<false><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, _dsources.data, _ddestinations.data);
	}
    }

    __device__ bool fsi_kernel(const float seed,
			       const int dpid, const float3 xp, const float3 up, const int spid,
			       float& xforce, float& yforce, float& zforce)
    {
	xforce = yforce = zforce = 0;

	const int sentry = 3 * spid;

	const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry);
	const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
	const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);

	const float _xr = xp.x - stmp0.x;
	const float _yr = xp.y - stmp0.y;
	const float _zr = xp.z - stmp1.x;

	const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

	if (rij2 > 1)
	    return false;

	const float invrij = rsqrtf(rij2);

	const float rij = rij2 * invrij;
	const float argwr = max((float)0, 1 - rij);
	const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));

	const float xr = _xr * invrij;
	const float yr = _yr * invrij;
	const float zr = _zr * invrij;

	const float rdotv =
	    xr * (up.x - stmp1.y) +
	    yr * (up.y - stmp2.x) +
	    zr * (up.z - stmp2.y);

	//const float mysaru = saru(saru_tag, dpid, spid);
	//const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
	const float myrandnr = Logistic::mean0var1(seed, dpid, spid);

	const float strength = params.aij * argwr +  (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

	xforce = strength * xr;
	yforce = strength * yr;
	zforce = strength * zr;

	return true;
    }

    __device__ float3 fsi_interaction(const float seed,
				      const int dpid, const float3 up, const int spid,
				      const float2 stmp1, const float2 stmp2,
				      const float _xr, const float _yr, const float _zr, const float rij2)
    {
	const float invrij = rsqrtf(rij2);

	const float rij = rij2 * invrij;
	const float argwr = 1 - rij;
	const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

	const float xr = _xr * invrij;
	const float yr = _yr * invrij;
	const float zr = _zr * invrij;

	const float rdotv =
	    xr * (up.x - stmp1.y) +
	    yr * (up.y - stmp2.x) +
	    zr * (up.z - stmp2.y);

	const float myrandnr = Logistic::mean0var1(seed, dpid, spid);

	const float strength = params.aij * argwr +  (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

	return make_float3(strength * xr, strength * yr, strength * zr);
    }


    template<int XCPB, int YCPB, int ZCPB, int COLS, int ROWS>
    __global__ void fsi_forces(const float seed,
			       hipTextureObject_t texSoluteCellStart,
			       hipTextureObject_t texSoluteParticles,
			       float * const accsolute, const int nsolute,
			       float * const accsolvent, const int nsolvent)
    {
	enum { CPB = XCPB * YCPB * ZCPB };

	assert(warpSize == COLS * ROWS);
	assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
	assert(ROWS * 3 <= warpSize);

	const int tid = threadIdx.x;
	const int wid = threadIdx.y;

	const int subtid = tid % COLS;
	const int slot = tid / COLS;

	__shared__ int volatile starts[CPB][32], scan[CPB][32];

	const int xmycid = blockIdx.x * XCPB + ((wid) % XCPB);
	const int ymycid = blockIdx.y * YCPB + ((wid / XCPB) % YCPB);
	const int zmycid = blockIdx.z * ZCPB + ((wid / (XCPB * YCPB)) % ZCPB);
	const int mycid = xmycid + XSIZE_SUBDOMAIN * (ymycid + YSIZE_SUBDOMAIN * zmycid);

	int mycount = 0, myscan = 0;

	if (tid < 27)
	{
	    const int dx = tid % 3;
	    const int dy = (tid / 3) % 3;
	    const int dz = (tid / 9) % 3;

	    int xcid = xmycid + dx - 1;
	    int ycid = ymycid + dy - 1;
	    int zcid = zmycid + dz - 1;

	    const bool valid_cid =
		xcid >= 0 && xcid < XSIZE_SUBDOMAIN &&
		ycid >= 0 && ycid < YSIZE_SUBDOMAIN &&
		zcid >= 0 && zcid < ZSIZE_SUBDOMAIN ;

	    xcid = min(XSIZE_SUBDOMAIN - 1, max(0, xcid));
	    ycid = min(YSIZE_SUBDOMAIN - 1, max(0, ycid));
	    zcid = min(ZSIZE_SUBDOMAIN - 1, max(0, zcid));

	    const int cid = max(0, xcid + XSIZE_SUBDOMAIN * (ycid + YSIZE_SUBDOMAIN * zcid));

	    starts[wid][tid] = tex1Dfetch(texCellsStart, cid);

	    myscan = mycount = valid_cid * tex1Dfetch(texCellsCount, cid);
	}

	for(int L = 1; L < 32; L <<= 1)
	    myscan += (tid >= L) * __shfl_up(myscan, L);

	if (tid < 28)
	    scan[wid][tid] = myscan - mycount;

	const int nsrc = scan[wid][27];

	const int dststart = tex1Dfetch<int>(texSoluteCellStart, mycid);
	const int lastdst = tex1Dfetch<int>(texSoluteCellStart, mycid + 1);

	for(int pid = subtid; pid < nsrc; pid += COLS)
	{
	    const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
	    const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
	    const int key = key9 + key3;

	    const int spid = pid - scan[wid][key] + starts[wid][key];
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry);
	    const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);

	    for(int dpid = dststart + slot; dpid < lastdst; dpid += ROWS)
	    {
		float3 xdest, udest;

		float2 dtmp0 = tex1Dfetch<float2>(texSoluteParticles, 3 * dpid);
		xdest.x = dtmp0.x;
		xdest.y = dtmp0.y;

		dtmp0 = tex1Dfetch<float2>(texSoluteParticles, 3 * dpid + 1);
		xdest.z = dtmp0.x;
		udest.x = dtmp0.y;

		dtmp0 = tex1Dfetch<float2>(texSoluteParticles, 3 * dpid + 2);
		udest.y = dtmp0.x;
		udest.z = dtmp0.y;

		const float xr = xdest.x - stmp0.x;
		const float yr = xdest.y - stmp0.y;
		const float zr = xdest.z - stmp1.x;
		const float rij2 = xr * xr + yr * yr + zr * zr;

		if (rij2 < 1.0f)
		{
		    const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);
		    const float3 f = fsi_interaction(seed, dpid, udest, spid, stmp1, stmp2, xr, yr, zr, rij2);

		    atomicAdd(accsolute + 3 * dpid    , f.x);
		    atomicAdd(accsolute + 3 * dpid + 1, f.y);
		    atomicAdd(accsolute + 3 * dpid + 2, f.z);

		    atomicAdd(accsolvent + 3 * spid    , -f.x);
		    atomicAdd(accsolvent + 3 * spid + 1, -f.y);
		    atomicAdd(accsolvent + 3 * spid + 2, -f.z);
		}
	    }
	}
    }

    __global__ void fsi_forces(const float seed,
			       Acceleration * accsolvent, const int npsolvent,
			       const Particle * const particle, const int nparticles, Acceleration * accrbc)
    {
	const int dpid = threadIdx.x + blockDim.x * blockIdx.x;

	if (dpid >= nparticles)
	    return;

	const Particle p = particle[dpid];

	const float3 xp = make_float3(p.x[0], p.x[1], p.x[2]);
	const float3 up = make_float3(p.u[0], p.u[1], p.u[2]);

	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	int mycid[3];
	for(int c = 0; c < 3; ++c)
	    mycid[c] = L[c]/2 + (int)floor(p.x[c]);

	for(int c = 0; c < 3; ++c)
	    if (mycid[c] < -1 || mycid[c] >= L[c] + 1)
	    {
		for(int c = 0; c < 3; ++c)
		    accrbc[dpid].a[c] = 0;

		return;
	    }

	float fsum[3] = {0, 0, 0};

	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		(code/3 % 3) - 1,
		(code/9 % 3) - 1
	    };

	    int vcid[3];
	    for(int c = 0; c < 3; ++c)
		vcid[c] = mycid[c] + d[c];

	    bool validcid = true;
	    for(int c = 0; c < 3; ++c)
		validcid &= vcid[c] >= 0 && vcid[c] < L[c];

	    if (!validcid)
		continue;

	    const int cid = vcid[0] + XSIZE_SUBDOMAIN * (vcid[1] + YSIZE_SUBDOMAIN * vcid[2]);
	    const int mystart = tex1Dfetch(texCellsStart, cid);
	    const int myend = mystart + tex1Dfetch(texCellsCount, cid);

	    assert(mystart >= 0 && mystart <= myend);
	    assert(myend <= npsolvent);

#pragma unroll 4
	    for(int s = mystart; s < myend; ++s)
	    {
		float f[3];
		const bool nonzero = fsi_kernel(seed, dpid, xp, up, s, f[0], f[1], f[2]);

		if (nonzero)
		{
		    for(int c = 0; c < 3; ++c)
			fsum[c] += f[c];

		    for(int c = 0; c < 3; ++c)
			atomicAdd(c + (float *)(accsolvent + s), -f[c]);
		}
	    }
	}

	for(int c = 0; c < 3; ++c)
	    accrbc[dpid].a[c] = fsum[c];
    }

    __constant__ int packstarts[27];
    __constant__ Particle * packstates[26];
    __constant__ Acceleration * packresults[26];

    template<int BLOCKSIZE> __global__  __launch_bounds__(32 * 4, 16)
	void fsi_forces_all(const float seed, Acceleration * accsolvent, const int npsolvent, const int nremote)
    {
	assert(blockDim.x == BLOCKSIZE);
	assert(blockDim.x * gridDim.x >= nremote);

	__shared__ float tmp[BLOCKSIZE * 3];

	const int tid = threadIdx.x;
	const int gidstart =  BLOCKSIZE * blockIdx.x;

	const int nlocal = min(BLOCKSIZE, nremote - gidstart);

	float3 xp, up;

#ifndef NDEBUG
	xp = make_float3(-313.313f, -313.313f, -313.313f); //che e' poi l'auto di paperino
	up = make_float3(-313.313f, -313.313f, -313.313f);
#endif

	{
	    const int n = nlocal * 6;
	    const int h = nlocal * 3;

	    for(int base = 0; base < n; base += h)
	    {
#pragma unroll 3
		for(int x = tid; x < h; x += BLOCKSIZE)
		{
		    const int l = base + x;
		    const int gid = gidstart + l / 6;

		    const int key9 = 9 * ((gid >= packstarts[9]) + (gid >= packstarts[18]));
		    const int key3 = 3 * ((gid >= packstarts[key9 + 3]) + (gid >= packstarts[key9 + 6]));
		    const int key1 = (gid >= packstarts[key9 + key3 + 1]) + (gid >= packstarts[key9 + key3 + 2]);

		    const int code = key9 + key3 + key1;
		    const int lpid = gid - packstarts[code];

		    assert(x < BLOCKSIZE * 3);
		    tmp[x] = *((l % 6) + (float *)&packstates[code][lpid]);
		}

		__syncthreads();

		const int xstart = tid * 6 - base;

		if (0 <= xstart && xstart + 3 <= h)
		{
		    xp.x = tmp[0 + xstart];
		    xp.y = tmp[1 + xstart];
		    xp.z = tmp[2 + xstart];

		    assert(0 + 6 * tid - base >= 0);
		    assert(2 + 6 * tid - base < 3 * BLOCKSIZE);
		}

		const int ustart = 3 + 6 * tid - base;

		if (0 <= ustart && ustart + 3 <= h)
		{
		    up.x = tmp[0 + ustart];
		    up.y = tmp[1 + ustart];
		    up.z = tmp[2 + ustart];

		    assert(3 + 6 * tid - base >= 0);
		    assert(5 + 6 * tid - base < 3 * BLOCKSIZE);
		}
	    }
	}

#ifndef NDEBUG
	assert(xp.x != -313.313f || gidstart + tid >= nremote);
	assert(xp.y != -313.313f || gidstart + tid >= nremote);
	assert(xp.z != -313.313f || gidstart + tid >= nremote);
	assert(up.x != -313.313f || gidstart + tid >= nremote);
	assert(up.y != -313.313f || gidstart + tid >= nremote);
	assert(up.z != -313.313f || gidstart + tid >= nremote);
#endif

	assert(!isnan(xp.x) && !isnan(xp.y) && !isnan(xp.z));
	assert(!isnan(up.x) && !isnan(up.y) && !isnan(up.z));

	__syncthreads();

	if (tid + gidstart < nremote)
	{
	    float fsum[3] = {0, 0, 0};

	    const int xcid = XSIZE_SUBDOMAIN / 2 + (int)floor(xp.x);
	    const int ycid = YSIZE_SUBDOMAIN / 2 + (int)floor(xp.y);
	    const int zcid = ZSIZE_SUBDOMAIN / 2 + (int)floor(xp.z);

	    const bool invalid =
		xcid < -1 || xcid >= XSIZE_SUBDOMAIN + 1 ||
		ycid < -1 || ycid >= YSIZE_SUBDOMAIN + 1 ||
		zcid < -1 || zcid >= ZSIZE_SUBDOMAIN + 1 ;

	    if (!invalid)
		for(int code = 0; code < 27; ++code)
		{
		    const int xsrccid = xcid + (code % 3) - 1;
		    const int ysrccid = ycid + (code/3 % 3) - 1;
		    const int zsrccid = zcid + (code/9 % 3) - 1;

		    const bool invalidsrccid =
			xsrccid < 0 || xsrccid >= XSIZE_SUBDOMAIN ||
			ysrccid < 0 || ysrccid >= YSIZE_SUBDOMAIN ||
			zsrccid < 0 || zsrccid >= ZSIZE_SUBDOMAIN ;

		    if (invalidsrccid)
			continue;

		    const int srccid = xsrccid + XSIZE_SUBDOMAIN * (ysrccid + YSIZE_SUBDOMAIN * zsrccid);

		    const int mystart = tex1Dfetch(texCellsStart, srccid);
		    const int myend = mystart + tex1Dfetch(texCellsCount, srccid);

		    assert(mystart >= 0 && mystart <= myend);
		    assert(myend <= npsolvent);

#pragma unroll 4
		    for(int s = mystart; s < myend; ++s)
		    {
			float f[3];
			const bool nonzero = fsi_kernel(seed, tid + gidstart, xp, up, s, f[0], f[1], f[2]);

			if (nonzero)
			{
			    for(int c = 0; c < 3; ++c)
				fsum[c] += f[c];

			    for(int c = 0; c < 3; ++c)
				atomicAdd(c + (float *)(accsolvent + s), -f[c]);
			}
		    }
		}

	    for(int c = 0; c < 3;  ++c)
		assert(!isnan(fsum[c]));

	    tmp[0 + 3 * tid] = fsum[0];
	    tmp[1 + 3 * tid] = fsum[1];
	    tmp[2 + 3 * tid] = fsum[2];
	}

	__syncthreads();

	{
	    const int n = nlocal * 3;

#pragma unroll 3
	    for(int l = tid; l < n; l += BLOCKSIZE)
	    {
		const int gid = gidstart + l / 3;

		const int key9 = 9 * ((gid >= packstarts[9]) + (gid >= packstarts[18]));
		const int key3 = 3 * ((gid >= packstarts[key9 + 3]) + (gid >= packstarts[key9 + 6]));
		const int key1 = (gid >= packstarts[key9 + key3 + 1]) + (gid >= packstarts[key9 + key3 + 2]);

		const int code = key9 + key3 + key1;
		const int lpid = gid - packstarts[code];

		packresults[code][lpid].a[l % 3] = tmp[l];
	    }
	}
    }

    __global__ void merge_accelerations(const Acceleration * const src, const int n, Acceleration * const dst)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid < n)
	    for(int c = 0; c < 3; ++c)
		dst[gid].a[c] += src[gid].a[c];
    }

    __global__ void merge_accelerations_float(const Acceleration * const src, const int n, Acceleration * const dst)
    {
	assert(blockDim.x * gridDim.x >= n * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int pid = gid / 3;
	const int c = gid % 3;

	if (pid < n)
	    dst[pid].a[c] += src[pid].a[c];
    }

    template<bool accumulation>
    __global__ void merge_accelerations_scattered_float(const int * const reordering, const Acceleration * const src,
							const int n, Acceleration * const dst)
    {
	assert(blockDim.x * gridDim.x >= n * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int pid = gid / 3;
	const int c = gid % 3;

	if (pid < n)
	{
	    const int actualpid = reordering[pid];

	    if (accumulation)
		dst[actualpid].a[c] += src[pid].a[c];
	    else
		dst[actualpid].a[c] = src[pid].a[c];
	}
    }
}

ComputeInteractionsRBC::ComputeInteractionsRBC(MPI_Comm _cartcomm):
nvertices(0), dualcells(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN)
{
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);

    if (rbcs)
    {
	CudaRBC::Extent host_extent;
	CudaRBC::setup(nvertices, host_extent);
    }

    MPI_CHECK( MPI_Comm_dup(_cartcomm, &cartcomm));

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));

    local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);

    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));

    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );
    }

    KernelsRBC::ParamsFSI params = {12.5 , gammadpd, sigmaf};

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsRBC::params), &params, sizeof(KernelsRBC::ParamsFSI)));

    CUDA_CHECK(hipEventCreate(&evextents, hipEventDisableTiming));
    CUDA_CHECK(hipEventCreate(&evfsi, hipEventDisableTiming));
}

void ComputeInteractionsRBC::_compute_extents(const Particle * const rbcs, const int nrbcs, hipStream_t stream)
{
#if 1
    if (nrbcs)
	minmax_massimo(rbcs, nvertices, nrbcs, minextents.devptr, maxextents.devptr, stream);
#else
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(rbcs + nvertices * i), extents.devptr + i);
#endif
}

void ComputeInteractionsRBC::extent(const Particle * const rbcs, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RBC/extent", NVTX_C2);

    minextents.resize(nrbcs);
    maxextents.resize(nrbcs);

    _compute_extents(rbcs, nrbcs, stream);

    CUDA_CHECK(hipEventRecord(evextents, stream));
}

void ComputeInteractionsRBC::count(const int nrbcs)
{
    NVTX_RANGE("RBC/count", NVTX_C3);

    CUDA_CHECK(hipEventSynchronize(evextents));

    for(int i = 0; i < 26; ++i)
	haloreplica[i].clear();

    for(int i = 0; i < nrbcs; ++i)
    {
	float pmin[3] = { minextents.data[i].x, minextents.data[i].y, minextents.data[i].z };
	float pmax[3] = { maxextents.data[i].x, maxextents.data[i].y, maxextents.data[i].z };

	for(int code = 0; code < 26; ++code)
	{
	    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	    const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	    bool interacting = true;

	    for(int c = 0; c < 3; ++c)
	    {
		const float range_start = max((float)(d[c] * L[c] - L[c]/2 - 1), pmin[c]);
		const float range_end = min((float)(d[c] * L[c] + L[c]/2 + 1), pmax[c]);

		interacting &= range_end > range_start;
	    }

	    if (interacting)
		haloreplica[code].push_back(i);
	}
    }

    for(int i = 0; i <26; ++i)
	MPI_CHECK(MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, dstranks[i], recv_tags[i] + 2077, cartcomm, reqrecvcounts + i));


    for(int i = 0; i < 26; ++i)
    {
	send_counts[i] = haloreplica[i].size();
	MPI_CHECK(MPI_Isend(send_counts + i, 1, MPI_INTEGER, dstranks[i], i + 2077, cartcomm, reqsendcounts + i));
    }

    for(int i = 0; i < 26; ++i)
	local[i].setup(send_counts[i] * nvertices);
}

void ComputeInteractionsRBC::exchange_count()
{
    NVTX_RANGE("RBC/exchange-count", NVTX_C4);

    MPI_Status statuses[26];
    MPI_CHECK(MPI_Waitall(26, reqrecvcounts, statuses));
    MPI_CHECK(MPI_Waitall(26, reqsendcounts, statuses));

    for(int i = 0; i < 26; ++i)
	remote[i].setup(recv_counts[i] * nvertices);
}

void ComputeInteractionsRBC::pack_p(const Particle * const rbcs, hipStream_t stream)
{
    NVTX_RANGE("RBC/pack", NVTX_C4);

#if 1
    {
	std::vector<int> codes;
	std::vector<const float *> src;
	std::vector<float *> dst;

	for(int i = 0; i < 26; ++i)
	    for(int j = 0; j < haloreplica[i].size(); ++j)
	    {
		codes.push_back(i);
		src.push_back((float *)(rbcs + nvertices * haloreplica[i][j]));
		dst.push_back((float *)(local[i].state.devptr + nvertices * j));
	    }

	KernelsRBC::shift_send_particles(stream, src.size(), nvertices, &src.front(), &codes.front(), &dst.front());

	CUDA_CHECK(hipPeekAtLastError());
    }
#else
    for(int i = 0; i < 26; ++i)
    {
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::shift_send_particles<<< (nvertices + 127) / 128, 128, 0, stream>>>
		(rbcs + nvertices * haloreplica[i][j], nvertices, i, local[i].state.devptr + nvertices * j);

	CUDA_CHECK(hipPeekAtLastError());
    }
#endif

    CUDA_CHECK(hipEventRecord(evfsi, stream));
}

void ComputeInteractionsRBC::post_p()
{
    NVTX_RANGE("RBC/post-p", NVTX_C5);

    CUDA_CHECK(hipEventSynchronize(evfsi));

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(remote[i].state.data, recv_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				recv_tags[i] + 2011, cartcomm, &request));

	    reqrecvp.push_back(request);
	}

    for(int i = 0; i < 26; ++i)
	if (send_counts[i] > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(local[i].result.data, send_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				recv_tags[i] + 2285, cartcomm, &request));

	    reqrecvacc.push_back(request);

	    MPI_CHECK(MPI_Isend(local[i].state.data, send_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				i + 2011, cartcomm, &request));

	    reqsendp.push_back(request);
	}
}

void ComputeInteractionsRBC::internal_forces(const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{
    CudaRBC::forces_nohost(stream, nrbcs, (float *)rbcs, (float *)accrbc);
}

void ComputeInteractionsRBC::fsi_bulk(const Particle * const solvent, const int nparticles, Acceleration * accsolvent,
				      const int * const cellsstart_solvent, const int * const cellscount_solvent,
				      const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{
    NVTX_RANGE("RBC/fsi-bulk", NVTX_C6);

    KernelsRBC::setup(solvent, nparticles, cellsstart_solvent, cellscount_solvent);

    if (nrbcs > 0 && nparticles > 0)
    {
	const float seed = local_trunk.get_float();

#if 1
	const int nsolvent = nparticles;
	const int nsolute = nrbcs * nvertices;
	const int3 vcells = make_int3(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN);
	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

	reordered_solute.resize(nsolute);
	CUDA_CHECK(hipMemcpyAsync(reordered_solute.data, rbcs, sizeof(Particle) * nrbcs * nvertices, hipMemcpyDeviceToDevice, stream));

	reordering.resize(nsolute);
	dualcells.build(reordered_solute.data, nrbcs * nvertices, stream, reordering.data);

	texSoluteStart.acquire(const_cast<int *>(dualcells.start), ncells + 1);
	texSolute.acquire((float2 *)const_cast<Particle *>(reordered_solute.data), reordered_solute.capacity);

	lacc_solute.resize(nsolute);
	CUDA_CHECK(hipMemsetAsync(lacc_solute.data, 0, sizeof(float) * 3 * lacc_solute.size, stream));

	KernelsRBC::fsi_forces<2, 2, 1, 32, 1><<<
	    dim3(vcells.x / 2, vcells.y / 2, vcells.z), dim3(32, 4), 0, stream>>>
	    (seed, texSoluteStart.texObj, texSolute.texObj, (float *)lacc_solute.data, lacc_solute.size, (float *)accsolvent, nsolvent);

        KernelsRBC::merge_accelerations_scattered_float<false><<< (nrbcs * nvertices * 3 + 127) / 128, 128, 0, stream >>>(
	    reordering.data, lacc_solute.data, nrbcs * nvertices, accrbc);

#else
	KernelsRBC::fsi_forces<<< (nrbcs * nvertices + 127) / 128, 128, 0, stream >>>
	    (seed, accsolvent, nparticles, rbcs, nrbcs * nvertices, accrbc);
#endif
    }
}

void ComputeInteractionsRBC::fsi_halo(const Particle * const solvent, const int nparticles, Acceleration * accsolvent,
				      const int * const cellsstart_solvent, const int * const cellscount_solvent,
				      const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{
    NVTX_RANGE("RBC/fsi-halo", NVTX_C7);

    _wait(reqrecvp);
    _wait(reqsendp);

#if 1
    {
	int nremote = 0;

	{
	    int packstarts[27];

	    packstarts[0] = 0;
	    for(int i = 0, s = 0; i < 26; ++i)
		packstarts[i + 1] = (s += remote[i].state.size);

	    nremote = packstarts[26];

	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsRBC::packstarts), packstarts,
					       sizeof(packstarts), 0, hipMemcpyHostToDevice, stream));
	}

	{
	    Particle * packstates[26];

	    for(int i = 0; i < 26; ++i)
		packstates[i] = remote[i].state.devptr;

	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsRBC::packstates), packstates,
					       sizeof(packstates), 0, hipMemcpyHostToDevice, stream));
	}

	{
	    Acceleration * packresults[26];

	    for(int i = 0; i < 26; ++i)
		packresults[i] = remote[i].result.devptr;

	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsRBC::packresults), packresults,
					       sizeof(packresults), 0, hipMemcpyHostToDevice, stream));
	}

	if(nremote)
	    KernelsRBC::fsi_forces_all<128><<< (nremote + 127) / 128, 128, 0, stream>>>(local_trunk.get_float(), accsolvent, nparticles, nremote);

    }
#else
    for(int i = 0; i < 26; ++i)
    {
	const int count = remote[i].state.size;

	if (count > 0)
	    KernelsRBC::fsi_forces<<< (count + 127) / 128, 128, 0, stream >>>
		(local_trunk.get_float(), accsolvent, nparticles, remote[i].state.devptr, count, remote[i].result.devptr);
    }
#endif

    CUDA_CHECK(hipEventRecord(evfsi));

    CUDA_CHECK(hipPeekAtLastError());
}

void ComputeInteractionsRBC::post_a()
{
    NVTX_RANGE("RBC/send-results", NVTX_C1);

    CUDA_CHECK(hipEventSynchronize(evfsi));

    _wait(reqsendacc);

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Isend(remote[i].result.data, recv_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				i + 2285, cartcomm, &request));

	    reqsendacc.push_back(request);
	}
}

void ComputeInteractionsRBC::merge_a(Acceleration * accrbc, hipStream_t stream)
{
    NVTX_RANGE("RBC/merge", NVTX_C2);

    _wait(reqrecvacc);

#if 1
    {
	std::vector<const float *> src;
	std::vector<float *> dst;

	for(int i = 0; i < 26; ++i)
	    for(int j = 0; j < haloreplica[i].size(); ++j)
	    {
		src.push_back((float *)(local[i].result.devptr + nvertices * j));
		dst.push_back((float *)(accrbc + nvertices * haloreplica[i][j]));
	    }

	KernelsRBC::merge_all_accel(stream, src.size(), nvertices, &src.front(), &dst.front());

	CUDA_CHECK(hipPeekAtLastError());
    }
#else
    for(int i = 0; i < 26; ++i)
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::merge_accelerations<<< (nvertices + 127) / 128, 128, 0, stream>>>(local[i].result.devptr + nvertices * j, nvertices,
											  accrbc + nvertices * haloreplica[i][j]);
#endif
}

ComputeInteractionsRBC::~ComputeInteractionsRBC()
{
    MPI_CHECK(MPI_Comm_free(&cartcomm));

    CUDA_CHECK(hipEventDestroy(evextents));
    CUDA_CHECK(hipEventDestroy(evfsi));

    KernelsRBC::dispose();
}

