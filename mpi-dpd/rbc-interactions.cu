#include "hip/hip_runtime.h"
/*
 *  rbc-interactions.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-02.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <set>
#include <../dpd-rng.h>

#include "rbc-interactions.h"
#include "minmax-massimo.h"

namespace KernelsRBC
{
    struct ParamsFSI
    {
	float aij, gamma, sigmaf;
    };

    __constant__ ParamsFSI params;
    
    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    static bool firsttime = true;
    
    __global__ void fsi_forces(const float seed,
			       Acceleration * accsolvent, const int npsolvent,
			       const Particle * const particle, const int nparticles, Acceleration * accrbc);
    
    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount)
    {
	if (firsttime)
	{
	    texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	    texCellsStart.filterMode = hipFilterModePoint;
	    texCellsStart.mipmapFilterMode = hipFilterModePoint;
	    texCellsStart.normalized = 0;
    
	    texCellsCount.channelDesc = hipCreateChannelDesc<int>();
	    texCellsCount.filterMode = hipFilterModePoint;
	    texCellsCount.mipmapFilterMode = hipFilterModePoint;
	    texCellsCount.normalized = 0;

	    texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
	    texSolventParticles.filterMode = hipFilterModePoint;
	    texSolventParticles.mipmapFilterMode = hipFilterModePoint;
	    texSolventParticles.normalized = 0;
	    firsttime = false;
	}
	
	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
				   sizeof(float) * 6 * npsolvent));

	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;
	
	assert(textureoffset == 0);
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0); 
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
	assert(textureoffset == 0);

	CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fsi_forces), hipFuncCachePreferL1));
    }
    
    __global__ void shift_send_particles_kernel(const Particle * const src, const int n, const int code, Particle * const dst)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	if (gid < n)
	{
	    Particle p = src[gid];
	    
	    for(int c = 0; c < 3; ++c)
		p.x[c] -= d[c] * L[c];

	    dst[gid] = p;
	}
    }

    static const int cmaxnrbcs = 64;
    __constant__ float * csources[cmaxnrbcs], * cdestinations[cmaxnrbcs];
    __constant__ int ccodes[cmaxnrbcs];

    template <bool from_cmem>
    __global__ void shift_all_send_particles(const int nrbcs, const int nvertices, 
					     const float ** const dsources, const int * dcodes, float ** const ddestinations)
    {
	const int nfloats_per_rbc = 6 * nvertices;

	assert(nfloats_per_rbc * nrbcs <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (gid >= nfloats_per_rbc * nrbcs) 
	    return;

	const int idrbc = gid / nfloats_per_rbc;
	assert(idrbc < nrbcs);

	const int offset = gid % nfloats_per_rbc;
	
	float val;
	if (from_cmem)
	    val = csources[idrbc][offset];
	else
	    val = dsources[idrbc][offset];
	
	int code;
	if (from_cmem)
	    code = ccodes[idrbc];
	else
	    code = dcodes[idrbc];

	const int c = gid % 6;

	val -= 
	    (c == 0) * ((code     + 2) % 3 - 1) * XSIZE_SUBDOMAIN + 
	    (c == 1) * ((code / 3 + 2) % 3 - 1) * YSIZE_SUBDOMAIN + 
	    (c == 2) * ((code / 9 + 2) % 3 - 1) * ZSIZE_SUBDOMAIN ;

	if (from_cmem)
	    cdestinations[idrbc][offset] = val;
	else
	    ddestinations[idrbc][offset] = val;
    }

    SimpleDeviceBuffer<float *> _ddestinations;
    SimpleDeviceBuffer<const float *> _dsources;
    SimpleDeviceBuffer<int> _dcodes;

    void dispose()
    {
	_ddestinations.dispose();
	_dsources.dispose();
	_dcodes.dispose();
    }
    
    void shift_send_particles(hipStream_t stream, const int nrbcs, const int nvertices,
			      const float ** const sources, const int * codes, float ** const destinations)
    {
	if (nrbcs == 0)
	    return;

	const int nthreads = nrbcs * nvertices * 6;

	if (nrbcs < cmaxnrbcs)
	{
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(ccodes), codes, sizeof(int) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdestinations), destinations, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csources), sources, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    
	    shift_all_send_particles<true><<<(nthreads + 127) / 128, 128, 0, stream>>>
		(nrbcs, nvertices, NULL, NULL, NULL);

	    CUDA_CHECK(hipPeekAtLastError());
	}
	else
	{
	    _dcodes.resize(nrbcs);
	    _ddestinations.resize(nrbcs);
	    _dsources.resize(nrbcs);

	    CUDA_CHECK(hipMemcpyAsync(_dcodes.data, codes, sizeof(int) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_ddestinations.data, destinations, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_dsources.data, sources, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));

	    shift_all_send_particles<false><<<(nthreads + 127) / 128, 128, 0, stream>>>
		(nrbcs, nvertices, _dsources.data, _dcodes.data, _ddestinations.data);
	}
    }

  template <bool from_cmem>
    __global__ void merge_all_acc(const int nrbcs, const int nvertices, 
				  const float ** const dsources, float ** const ddestinations)
    {
	if (nrbcs == 0)
	    return;

	const int nfloats_per_rbc = 3 * nvertices;

	assert(nfloats_per_rbc * nrbcs <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (gid >= nfloats_per_rbc * nrbcs) 
	    return;

	const int idrbc = gid / nfloats_per_rbc;
	assert(idrbc < nrbcs);

	const int offset = gid % nfloats_per_rbc;
	
	float val;
	if (from_cmem)
	    val = csources[idrbc][offset];
	else
	    val = dsources[idrbc][offset];
	
	if (from_cmem)
	    atomicAdd(cdestinations[idrbc] + offset, val);
	else
	    atomicAdd(ddestinations[idrbc] + offset, val);
    }

    void merge_all_accel(hipStream_t stream, const int nrbcs, const int nvertices,
			 const float ** const sources, float ** const destinations)
    {
	if (nrbcs == 0)
	    return;

	const int nthreads = nrbcs * nvertices * 3;

	CUDA_CHECK(hipPeekAtLastError());

	if (nrbcs < cmaxnrbcs)
	{
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdestinations), destinations, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csources), sources, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    
	    merge_all_acc<true><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, NULL, NULL);

	    CUDA_CHECK(hipPeekAtLastError());
	}
	else
	{
	    _ddestinations.resize(nrbcs);
	    _dsources.resize(nrbcs);

	    CUDA_CHECK(hipMemcpyAsync(_ddestinations.data, destinations, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_dsources.data, sources, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));

	    merge_all_acc<false><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, _dsources.data, _ddestinations.data);
	}
    }


    __device__ bool fsi_kernel(const float seed,
			       const int dpid, const float3 xp, const float3 up, const int spid,
			       float& xforce, float& yforce, float& zforce)
    {
	xforce = yforce = zforce = 0;
	
	const int sentry = 3 * spid;
	
	const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry);
	const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
	const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);
	
	const float _xr = xp.x - stmp0.x;
	const float _yr = xp.y - stmp0.y;
	const float _zr = xp.z - stmp1.x;

	const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
	
	if (rij2 > 1)
	    return false;
	
	const float invrij = rsqrtf(rij2);
	
	const float rij = rij2 * invrij;
	const float argwr = max((float)0, 1 - rij);
	const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));
	
	const float xr = _xr * invrij;
	const float yr = _yr * invrij;
	const float zr = _zr * invrij;
	
	const float rdotv = 
	    xr * (up.x - stmp1.y) +
	    yr * (up.y - stmp2.x) +
	    zr * (up.z - stmp2.y);
	
	//const float mysaru = saru(saru_tag, dpid, spid);
	//const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
	const float myrandnr = Logistic::mean0var1(seed, dpid, spid);
	
	const float strength = params.aij * argwr +  (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;
	
	xforce = strength * xr;
	yforce = strength * yr;
	zforce = strength * zr; 

	return true;
    }
    
    __global__ void fsi_forces(const float seed,
			       Acceleration * accsolvent, const int npsolvent,
			       const Particle * const particle, const int nparticles, Acceleration * accrbc)
    {
	const int dpid = threadIdx.x + blockDim.x * blockIdx.x;

	if (dpid >= nparticles)
	    return;

	const Particle p = particle[dpid];

	const float3 xp = make_float3(p.x[0], p.x[1], p.x[2]);
	const float3 up = make_float3(p.u[0], p.u[1], p.u[2]);
		
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	int mycid[3];
	for(int c = 0; c < 3; ++c)
	    mycid[c] = L[c]/2 + (int)floor(p.x[c]);

	for(int c = 0; c < 3; ++c)
	    if (mycid[c] < -1 || mycid[c] >= L[c] + 1)
	    {
		for(int c = 0; c < 3; ++c)
		    accrbc[dpid].a[c] = 0;

		return;
	    }

	float fsum[3] = {0, 0, 0};
	
	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		(code/3 % 3) - 1,
		(code/9 % 3) - 1
	    };
	    
	    int vcid[3];
	    for(int c = 0; c < 3; ++c)
		vcid[c] = mycid[c] + d[c];

	    bool validcid = true;
	    for(int c = 0; c < 3; ++c)
		validcid &= vcid[c] >= 0 && vcid[c] < L[c];

	    if (!validcid)
		continue;
	    
	    const int cid = vcid[0] + XSIZE_SUBDOMAIN * (vcid[1] + YSIZE_SUBDOMAIN * vcid[2]);
	    const int mystart = tex1Dfetch(texCellsStart, cid);
	    const int myend = mystart + tex1Dfetch(texCellsCount, cid);
	    
	    assert(mystart >= 0 && mystart <= myend);
	    assert(myend <= npsolvent);

	    #pragma unroll 4
	    for(int s = mystart; s < myend; ++s)
	    {
		float f[3];
		const bool nonzero = fsi_kernel(seed, dpid, xp, up, s, f[0], f[1], f[2]);

		if (nonzero)
		{
		    for(int c = 0; c < 3; ++c)
			fsum[c] += f[c];
		     
		    for(int c = 0; c < 3; ++c)
		    	   atomicAdd(c + (float *)(accsolvent + s), -f[c]);
		}
	    }
	}
	
	for(int c = 0; c < 3; ++c)
	    accrbc[dpid].a[c] = fsum[c];
    }

    __constant__ int packstarts[27];
    __constant__ Particle * packstates[26];
    __constant__ Acceleration * packresults[26];

    __global__ void fsi_forces_all(const float seed,
				   Acceleration * accsolvent, const int npsolvent, const int nremote)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= nremote)
	    return;

	const int key9 = 9 * ((gid >= packstarts[9]) + (gid >= packstarts[18]));
	const int key3 = 3 * ((gid >= packstarts[key9 + 3]) + (gid >= packstarts[key9 + 6]));
	const int key1 = (gid >= packstarts[key9 + key3 + 1]) + (gid >= packstarts[key9 + key3 + 2]);
	const int code = key9 + key3 + key1;

	assert(code >= 0 && code < 26);
	assert(gid >= packstarts[code] && gid < packstarts[code + 1]);

	const int lpid = gid - packstarts[code];
	const Particle p = packstates[code][lpid];

	const float3 xp = make_float3(p.x[0], p.x[1], p.x[2]);
	const float3 up = make_float3(p.u[0], p.u[1], p.u[2]);
		
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	int mycid[3];
	for(int c = 0; c < 3; ++c)
	    mycid[c] = L[c]/2 + (int)floor(p.x[c]);

	for(int c = 0; c < 3; ++c)
	    if (mycid[c] < -1 || mycid[c] >= L[c] + 1)
	    {
		for(int c = 0; c < 3; ++c)
		    packresults[code][lpid].a[c] = 0;

		return;
	    }

	float fsum[3] = {0, 0, 0};
	
	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		(code/3 % 3) - 1,
		(code/9 % 3) - 1
	    };
	    
	    int vcid[3];
	    for(int c = 0; c < 3; ++c)
		vcid[c] = mycid[c] + d[c];

	    bool validcid = true;
	    for(int c = 0; c < 3; ++c)
		validcid &= vcid[c] >= 0 && vcid[c] < L[c];

	    if (!validcid)
		continue;
	    
	    const int cid = vcid[0] + XSIZE_SUBDOMAIN * (vcid[1] + YSIZE_SUBDOMAIN * vcid[2]);
	    const int mystart = tex1Dfetch(texCellsStart, cid);
	    const int myend = mystart + tex1Dfetch(texCellsCount, cid);
	    
	    assert(mystart >= 0 && mystart <= myend);
	    assert(myend <= npsolvent);

	    #pragma unroll 4
	    for(int s = mystart; s < myend; ++s)
	    {
		float f[3];
		const bool nonzero = fsi_kernel(seed, gid, xp, up, s, f[0], f[1], f[2]);

		if (nonzero)
		{
		    for(int c = 0; c < 3; ++c)
			fsum[c] += f[c];
		     
		    for(int c = 0; c < 3; ++c)
		    	   atomicAdd(c + (float *)(accsolvent + s), -f[c]);
		}
	    }
	}
	
	for(int c = 0; c < 3; ++c)
	    packresults[code][lpid].a[c] = fsum[c];
    }

    __global__ void merge_accelerations(const Acceleration * const src, const int n, Acceleration * const dst)
    {	
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid < n)
	    for(int c = 0; c < 3; ++c)
		dst[gid].a[c] += src[gid].a[c];
    }

    __global__ void merge_accelerations_float(const Acceleration * const src, const int n, Acceleration * const dst)
    {	
	assert(blockDim.x * gridDim.x >= n * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int pid = gid / 3;
	const int c = gid % 3;

	if (pid < n)
	    dst[pid].a[c] += src[pid].a[c];
    }

    template<bool accumulation>
    __global__ void merge_accelerations_scattered_float(const int * const reordering, const Acceleration * const src, 
							const int n, Acceleration * const dst)
    {
	assert(blockDim.x * gridDim.x >= n * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int pid = gid / 3;
	const int c = gid % 3;

	if (pid < n)
	{
	    const int actualpid = reordering[pid];

	    if (accumulation)
		dst[actualpid].a[c] += src[pid].a[c];
	    else
		dst[actualpid].a[c] = src[pid].a[c];
	}
    }
}

ComputeInteractionsRBC::ComputeInteractionsRBC(MPI_Comm _cartcomm): 
nvertices(0), dualcells(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN)
{ 
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);
    
    if (rbcs)
    {
	CudaRBC::Extent host_extent;
	CudaRBC::setup(nvertices, host_extent);
    }
    
    MPI_CHECK( MPI_Comm_dup(_cartcomm, &cartcomm));
    
    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    
    local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);
    
    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));

    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );
    }

    KernelsRBC::ParamsFSI params = {12.5 , gammadpd, sigmaf};
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsRBC::params), &params, sizeof(KernelsRBC::ParamsFSI)));
    
    CUDA_CHECK(hipEventCreate(&evextents, hipEventDisableTiming));
    CUDA_CHECK(hipEventCreate(&evfsi, hipEventDisableTiming));
}

void ComputeInteractionsRBC::_compute_extents(const Particle * const rbcs, const int nrbcs, hipStream_t stream)
{
#if 1
    if (nrbcs)
	minmax_massimo(rbcs, nvertices, nrbcs, minextents.devptr, maxextents.devptr, stream);
#else
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(rbcs + nvertices * i), extents.devptr + i);
#endif
}

void ComputeInteractionsRBC::extent(const Particle * const rbcs, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RBC/extent", NVTX_C2);

    minextents.resize(nrbcs);
    maxextents.resize(nrbcs);

    _compute_extents(rbcs, nrbcs, stream);

    CUDA_CHECK(hipEventRecord(evextents, stream));
}

void ComputeInteractionsRBC::count(const int nrbcs)
{
    NVTX_RANGE("RBC/count", NVTX_C3);

    CUDA_CHECK(hipEventSynchronize(evextents));

    for(int i = 0; i < 26; ++i)
	haloreplica[i].clear();

    for(int i = 0; i < nrbcs; ++i)
    {
	float pmin[3] = { minextents.data[i].x, minextents.data[i].y, minextents.data[i].z };
	float pmax[3] = { maxextents.data[i].x, maxextents.data[i].y, maxextents.data[i].z };

	for(int code = 0; code < 26; ++code)
	{
	    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	    const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	    bool interacting = true;
	    
	    for(int c = 0; c < 3; ++c)
	    {
		const float range_start = max((float)(d[c] * L[c] - L[c]/2 - 1), pmin[c]);
		const float range_end = min((float)(d[c] * L[c] + L[c]/2 + 1), pmax[c]);

		interacting &= range_end > range_start;
	    }

	    if (interacting)
		haloreplica[code].push_back(i);
	}
    }

    for(int i = 0; i <26; ++i)
	MPI_CHECK(MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, dstranks[i], recv_tags[i] + 2077, cartcomm, reqrecvcounts + i));

    
    for(int i = 0; i < 26; ++i)
    {
	send_counts[i] = haloreplica[i].size();
	MPI_CHECK(MPI_Isend(send_counts + i, 1, MPI_INTEGER, dstranks[i], i + 2077, cartcomm, reqsendcounts + i));
    }

    for(int i = 0; i < 26; ++i)
	local[i].setup(send_counts[i] * nvertices);
}

void ComputeInteractionsRBC::exchange_count()
{
    NVTX_RANGE("RBC/exchange-count", NVTX_C4);

    MPI_Status statuses[26];
    MPI_CHECK(MPI_Waitall(26, reqrecvcounts, statuses));
    MPI_CHECK(MPI_Waitall(26, reqsendcounts, statuses));
        
    for(int i = 0; i < 26; ++i)
	remote[i].setup(recv_counts[i] * nvertices);
}

void ComputeInteractionsRBC::pack_p(const Particle * const rbcs, hipStream_t stream)
{
    NVTX_RANGE("RBC/pack", NVTX_C4);

#if 1
    {
	std::vector<int> codes;
	std::vector<const float *> src;
	std::vector<float *> dst;
	
	for(int i = 0; i < 26; ++i)
	    for(int j = 0; j < haloreplica[i].size(); ++j)
	    {
		codes.push_back(i);
		src.push_back((float *)(rbcs + nvertices * haloreplica[i][j]));
		dst.push_back((float *)(local[i].state.devptr + nvertices * j));
	    }

	KernelsRBC::shift_send_particles(stream, src.size(), nvertices, &src.front(), &codes.front(), &dst.front());
	
	CUDA_CHECK(hipPeekAtLastError());
    }
#else
    for(int i = 0; i < 26; ++i)
    {
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::shift_send_particles<<< (nvertices + 127) / 128, 128, 0, stream>>>
		(rbcs + nvertices * haloreplica[i][j], nvertices, i, local[i].state.devptr + nvertices * j);
	 
	CUDA_CHECK(hipPeekAtLastError());
    }
#endif
     
    CUDA_CHECK(hipEventRecord(evfsi, stream));
}

void ComputeInteractionsRBC::post_p()
{
    NVTX_RANGE("RBC/post-p", NVTX_C5);

    CUDA_CHECK(hipEventSynchronize(evfsi));

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;
	    
	    MPI_CHECK(MPI_Irecv(remote[i].state.data, recv_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				recv_tags[i] + 2011, cartcomm, &request));

	    reqrecvp.push_back(request);
	}

    for(int i = 0; i < 26; ++i)
	if (send_counts[i] > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(local[i].result.data, send_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				recv_tags[i] + 2285, cartcomm, &request));

	    reqrecvacc.push_back(request);
	    
	    MPI_CHECK(MPI_Isend(local[i].state.data, send_counts[i] * nvertices, Particle::datatype(), dstranks[i],
				i + 2011, cartcomm, &request));

	    reqsendp.push_back(request);
	}
}

void ComputeInteractionsRBC::internal_forces(const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{
    CudaRBC::forces_nohost(stream, nrbcs, (float *)rbcs, (float *)accrbc);
}

void ComputeInteractionsRBC::fsi_bulk(const Particle * const solvent, const int nparticles, Acceleration * accsolvent,
				      const int * const cellsstart_solvent, const int * const cellscount_solvent,
				      const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{	
    NVTX_RANGE("RBC/fsi-bulk", NVTX_C6);

    KernelsRBC::setup(solvent, nparticles, cellsstart_solvent, cellscount_solvent);

    if (nrbcs > 0 && nparticles > 0)
    {
	const float seed = local_trunk.get_float();

#if 0
	const int nsolvent = nparticles;
	const int nsolute = nrbcs * nvertices;
	const int3 vcells = make_int3(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN);
	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

	reordered_solute.resize(nsolute);	
	CUDA_CHECK(hipMemcpyAsync(reordered_solute.data, rbcs, sizeof(Particle) * nrbcs * nvertices, hipMemcpyDeviceToDevice, stream));

	reordering.resize(nsolute);
	dualcells.build(reordered_solute.data, nrbcs * nvertices, stream, reordering.data);

	texSolventStart.acquire(const_cast<int *>(cellsstart_solvent), ncells + 1);
	texSolvent.acquire((float2 *)const_cast<Particle *>(solvent), nsolvent * 3);
	texSoluteStart.acquire(const_cast<int *>(dualcells.start), ncells + 1);
	texSolute.acquire((float2 *)const_cast<Particle *>(reordered_solute.data), reordered_solute.capacity);

	//solute to solvent
	lacc_solvent.resize(nsolvent);
	forces_dpd_cuda_bipartite_nohost(stream, (float2 *)solvent, nsolvent, texSolventStart.texObj, texSoluteStart.texObj, texSolute.texObj,
					 nsolute, vcells, 12.5, gammadpd, sigma / sqrt(dt), seed, 0, (float *)lacc_solvent.data);

	//solvent to solute
	lacc_solute.resize(nsolute);
	forces_dpd_cuda_bipartite_nohost(stream, (float2 *)reordered_solute.data, nsolute, texSoluteStart.texObj, texSolventStart.texObj, texSolvent.texObj, 
					 nsolvent, vcells, 12.5, gammadpd, sigma / sqrt(dt), seed, 1, (float *)lacc_solute.data);
	
	KernelsRBC::merge_accelerations_float<<< (nparticles * 3 + 127) / 128, 128, 0, stream >>>(lacc_solvent.data, nparticles, accsolvent);
	
        KernelsRBC::merge_accelerations_scattered_float<false><<< (nrbcs * nvertices * 3 + 127) / 128, 128, 0, stream >>>(
	    reordering.data, lacc_solute.data, nrbcs * nvertices, accrbc);

#else
	KernelsRBC::fsi_forces<<< (nrbcs * nvertices + 127) / 128, 128, 0, stream >>>
	    (seed, accsolvent, nparticles, rbcs, nrbcs * nvertices, accrbc);
#endif
    }
}

void ComputeInteractionsRBC::fsi_halo(const Particle * const solvent, const int nparticles, Acceleration * accsolvent,
				      const int * const cellsstart_solvent, const int * const cellscount_solvent,
				      const Particle * const rbcs, const int nrbcs, Acceleration * accrbc, hipStream_t stream)
{
    NVTX_RANGE("RBC/fsi-halo", NVTX_C7);

    _wait(reqrecvp);
    _wait(reqsendp);

#if 1
    {
	int nremote = 0;

	{
	    int packstarts[27];
	    
	    packstarts[0] = 0;
	    for(int i = 0, s = 0; i < 26; ++i)
		packstarts[i + 1] = (s += remote[i].state.size);
		
	    nremote = packstarts[26];
		
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsRBC::packstarts), packstarts,
					       sizeof(packstarts), 0, hipMemcpyHostToDevice, stream));
	}
	    
	{
	    Particle * packstates[26];
		
	    for(int i = 0; i < 26; ++i)
		packstates[i] = remote[i].state.devptr;

	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsRBC::packstates), packstates,
					       sizeof(packstates), 0, hipMemcpyHostToDevice, stream));
	}

	{
	    Acceleration * packresults[26];
		
	    for(int i = 0; i < 26; ++i)
		packresults[i] = remote[i].result.devptr;

	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsRBC::packresults), packresults,
					       sizeof(packresults), 0, hipMemcpyHostToDevice, stream));
	}
	    
	if(nremote)
	    KernelsRBC::fsi_forces_all<<< (nremote + 127) / 128, 128, 0, stream>>>(local_trunk.get_float(), accsolvent, nparticles, nremote);

    }
#else
    for(int i = 0; i < 26; ++i)
    {
	const int count = remote[i].state.size;

	if (count > 0)
	    KernelsRBC::fsi_forces<<< (count + 127) / 128, 128, 0, stream >>>
		(local_trunk.get_float(), accsolvent, nparticles, remote[i].state.devptr, count, remote[i].result.devptr);
    }
#endif
	
    CUDA_CHECK(hipEventRecord(evfsi));
    
    CUDA_CHECK(hipPeekAtLastError());
}

void ComputeInteractionsRBC::post_a()
{
    NVTX_RANGE("RBC/send-results", NVTX_C1);

    CUDA_CHECK(hipEventSynchronize(evfsi));

    _wait(reqsendacc);

    for(int i = 0; i < 26; ++i)
	if (recv_counts[i] > 0)
	{
	    MPI_Request request;
		
	    MPI_CHECK(MPI_Isend(remote[i].result.data, recv_counts[i] * nvertices, Acceleration::datatype(), dstranks[i],
				i + 2285, cartcomm, &request));
		
	    reqsendacc.push_back(request);
	}
}

void ComputeInteractionsRBC::merge_a(Acceleration * accrbc, hipStream_t stream)
{
    NVTX_RANGE("RBC/merge", NVTX_C2);

    _wait(reqrecvacc);
	
#if 1
    {
	std::vector<const float *> src;
	std::vector<float *> dst;
	    
	for(int i = 0; i < 26; ++i)
	    for(int j = 0; j < haloreplica[i].size(); ++j)
	    {
		src.push_back((float *)(local[i].result.devptr + nvertices * j));
		dst.push_back((float *)(accrbc + nvertices * haloreplica[i][j]));
	    }
	    
	KernelsRBC::merge_all_accel(stream, src.size(), nvertices, &src.front(), &dst.front());
	    
	CUDA_CHECK(hipPeekAtLastError());
    }
#else
    for(int i = 0; i < 26; ++i)
	for(int j = 0; j < haloreplica[i].size(); ++j)
	    KernelsRBC::merge_accelerations<<< (nvertices + 127) / 128, 128, 0, stream>>>(local[i].result.devptr + nvertices * j, nvertices,
											  accrbc + nvertices * haloreplica[i][j]);
#endif
}

ComputeInteractionsRBC::~ComputeInteractionsRBC()
{
    MPI_CHECK(MPI_Comm_free(&cartcomm));

    CUDA_CHECK(hipEventDestroy(evextents));
    CUDA_CHECK(hipEventDestroy(evfsi));

    KernelsRBC::dispose();
}

