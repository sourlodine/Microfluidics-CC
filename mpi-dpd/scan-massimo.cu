#include "hip/hip_runtime.h"
/*
 *  scan-massimo.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Massimo Bernaschi on 2015-03-09.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdio>

typedef struct {
    int g_block_id;
    int g_blockcnt;
    int sum;
} sblockds_t;

#define MAXTHREADS 1024
#define WARPSIZE     32

__global__ void exclscn2e(int *d_data0, int *d_output0,
			  int *d_data1, int *d_output1,
			  int *d_data2, int *d_output2,
			  int *d_data3, int *d_output3,
			  int *d_data4, int *d_output4,
			  int *d_data5, int *d_output5,
			  int *d_data6, int *d_output6,
			  int *d_data7, int *d_output7) {
    const int twid=threadIdx.x%32;
    int wid=threadIdx.x/32;
    switch(wid) {
    case 0:
	if(twid<2) {
	    d_output0[twid]=d_data0[0]*twid;
	}
	return;
    case 1:
	if(twid<2) {
	    d_output1[twid]=d_data1[0]*twid;
	}
	return;
    case 2:
	if(twid<2) {
	    d_output2[twid]=d_data2[0]*twid;
	}
	return;
    case 3:
	if(twid<2) {
	    d_output3[twid]=d_data3[0]*twid;
	}
	return;
    case 4:
	if(twid<2) {
	    d_output4[twid]=d_data4[0]*twid;
	}
	return;
    case 5:
	if(twid<2) {
	    d_output5[twid]=d_data5[0]*twid;
	}
	return;
    case 6:
	if(twid<2) {
	    d_output6[twid]=d_data6[0]*twid;
	}
	return;
    case 7:
	if(twid<2) {
	    d_output7[twid]=d_data7[0]*twid;
	}
	return;
    }
}

__global__ void exclscnmb2e(int *d_data0, int *d_output0,
			    int *d_data1, int *d_output1,
			    int *d_data2, int *d_output2,
			    int *d_data3, int *d_output3,
			    int *d_data4, int *d_output4,
			    int *d_data5, int *d_output5,
			    int *d_data6, int *d_output6,
			    int *d_data7, int *d_output7) {
    const int twid=threadIdx.x;
    switch(blockIdx.x) {
    case 0:
	if(twid<2) {
	    d_output0[twid]=d_data0[0]*twid;
	}
	return;
    case 1:
	if(twid<2) {
	    d_output1[twid]=d_data1[0]*twid;
	}
	return;
    case 2:
	if(twid<2) {
	    d_output2[twid]=d_data2[0]*twid;
	}
	return;
    case 3:
	if(twid<2) {
	    d_output3[twid]=d_data3[0]*twid;
	}
	return;
    case 4:
	if(twid<2) {
	    d_output4[twid]=d_data4[0]*twid;
	}
	return;
    case 5:
	if(twid<2) {
	    d_output5[twid]=d_data5[0]*twid;
	}
	return;
    case 6:
	if(twid<2) {
	    d_output6[twid]=d_data6[0]*twid;
	}
	return;
    case 7:
	if(twid<2) {
	    d_output7[twid]=d_data7[0]*twid;
	}
	return;
    }
}

__global__ void exclscn2w(int *d_data, int *d_output, int size) {
    __shared__ int temp[32];
    int temp1, temp2, temp4;
    if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    const int tid = threadIdx.x;
    temp4 = temp1 = (tid+blockIdx.x*blockDim.x<size)?d_data[tid+blockIdx.x*blockDim.x]:0;
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
        if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (tid >= 32) { temp1 += temp[0]; }
    if(tid+blockIdx.x*blockDim.x<size) {
        d_output[tid+blockIdx.x*blockDim.x]=temp1-temp4;
    }
}

__global__ void exclscnmb2w(int *d_data0, int *d_output0,
			    int *d_data1, int *d_output1,
			    int *d_data2, int *d_output2,
			    int *d_data3, int *d_output3,
			    int *d_data4, int *d_output4,
			    int *d_data5, int *d_output5,
			    int *d_data6, int *d_output6,
			    int *d_data7, int *d_output7,
			    int *d_data8, int *d_output8,
			    int *d_data9, int *d_output9,
			    int *d_data10, int *d_output10,
			    int *d_data11, int *d_output11,
			    int size) {
    __shared__ int temp[32];
    int temp1, temp2, temp4;
    if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    const int tid = threadIdx.x;
    switch(blockIdx.x) {
    case 0:
	temp4 = temp1 = (tid<size)?d_data0[tid]:0;
	break;
    case 1:
	temp4 = temp1 = (tid<size)?d_data1[tid]:0;
	break;
    case 2:
	temp4 = temp1 = (tid<size)?d_data2[tid]:0;
	break;
    case 3:
	temp4 = temp1 = (tid<size)?d_data3[tid]:0;
	break;
    case 4:
	temp4 = temp1 = (tid<size)?d_data4[tid]:0;
	break;
    case 5:
	temp4 = temp1 = (tid<size)?d_data5[tid]:0;
	break;
    case 6:
	temp4 = temp1 = (tid<size)?d_data6[tid]:0;
	break;
    case 7:
	temp4 = temp1 = (tid<size)?d_data7[tid]:0;
	break;
    case 8:
	temp4 = temp1 = (tid<size)?d_data8[tid]:0;
	break;
    case 9:
	temp4 = temp1 = (tid<size)?d_data9[tid]:0;
	break;
    case 10:
	temp4 = temp1 = (tid<size)?d_data10[tid]:0;
	break;
    case 11:
	temp4 = temp1 = (tid<size)?d_data11[tid]:0;
	break;
    }
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
	if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (tid >= 32) { temp1 += temp[0]; }
    if(tid<size) {
	switch(blockIdx.x) {
	case 0:
	    d_output0[tid]=temp1-temp4;
	    break;
	case 1:
	    d_output1[tid]=temp1-temp4;
	    break;
	case 2:
	    d_output2[tid]=temp1-temp4;
	    break;
	case 3:
	    d_output3[tid]=temp1-temp4;
	    break;
	case 4:
	    d_output4[tid]=temp1-temp4;
	    break;
	case 5:
	    d_output5[tid]=temp1-temp4;
	    break;
	case 6:
	    d_output6[tid]=temp1-temp4;
	    break;
	case 7:
	    d_output7[tid]=temp1-temp4;
	    break;
	case 8:
	    d_output8[tid]=temp1-temp4;
	    break;
	case 9:
	    d_output9[tid]=temp1-temp4;
	    break;
	case 10:
	    d_output10[tid]=temp1-temp4;
	    break;
	case 11:
	    d_output11[tid]=temp1-temp4;
	    break;
	}
    }
}

__global__ void exclscnmb2ew(int *d_data0, int *d_output0,
			     int *d_data1, int *d_output1,
			     int *d_data2, int *d_output2,
			     int *d_data3, int *d_output3,
			     int *d_data4, int *d_output4,
			     int *d_data5, int *d_output5,
			     int *d_data6, int *d_output6,
			     int *d_data7, int *d_output7,
			     int *d_data8, int *d_output8,
			     int *d_data9, int *d_output9,
			     int *d_data10, int *d_output10,
			     int *d_data11, int *d_output11,
			     int *d_data20, int *d_output20,
			     int *d_data21, int *d_output21,
			     int *d_data22, int *d_output22,
			     int *d_data23, int *d_output23,
			     int *d_data24, int *d_output24,
			     int *d_data25, int *d_output25,
			     int *d_data26, int *d_output26,
			     int *d_data27, int *d_output27,
			     int size) {
    __shared__ int temp[32];
    int temp1, temp2, temp4;
    if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    const int tid = threadIdx.x;
    switch(blockIdx.x) {
    case 0:
	temp4 = temp1 = (tid<size)?d_data0[tid]:0;
	break;
    case 1:
	temp4 = temp1 = (tid<size)?d_data1[tid]:0;
	break;
    case 2:
	temp4 = temp1 = (tid<size)?d_data2[tid]:0;
	break;
    case 3:
	temp4 = temp1 = (tid<size)?d_data3[tid]:0;
	break;
    case 4:
	temp4 = temp1 = (tid<size)?d_data4[tid]:0;
	break;
    case 5:
	temp4 = temp1 = (tid<size)?d_data5[tid]:0;
	break;
    case 6:
	temp4 = temp1 = (tid<size)?d_data6[tid]:0;
	break;
    case 7:
	temp4 = temp1 = (tid<size)?d_data7[tid]:0;
	break;
    case 8:
	temp4 = temp1 = (tid<size)?d_data8[tid]:0;
	break;
    case 9:
	temp4 = temp1 = (tid<size)?d_data9[tid]:0;
	break;
    case 10:
	temp4 = temp1 = (tid<size)?d_data10[tid]:0;
	break;
    case 11:
	temp4 = temp1 = (tid<size)?d_data11[tid]:0;
	break;
    }
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
	if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (tid >= 32) { temp1 += temp[0]; }
    if(tid<size) {
	switch(blockIdx.x) {
	case 0:
	    d_output0[tid]=temp1-temp4;
	    if(tid<2) {
		d_output20[tid]=d_data20[0]*tid;
	    }
	    break;
	case 1:
	    d_output1[tid]=temp1-temp4;
	    if(tid<2) {
		d_output21[tid]=d_data21[0]*tid;
	    }
	    break;
	case 2:
	    d_output2[tid]=temp1-temp4;
	    if(tid<2) {
		d_output22[tid]=d_data22[0]*tid;
	    }
	    break;
	case 3:
	    d_output3[tid]=temp1-temp4;
	    if(tid<2) {
		d_output23[tid]=d_data23[0]*tid;
	    }
	    break;
	case 4:
	    d_output4[tid]=temp1-temp4;
	    if(tid<2) {
		d_output24[tid]=d_data24[0]*tid;
	    }
	    break;
	case 5:
	    d_output5[tid]=temp1-temp4;
	    if(tid<2) {
		d_output25[tid]=d_data25[0]*tid;
	    }
	    break;
	case 6:
	    d_output6[tid]=temp1-temp4;
	    if(tid<2) {
		d_output26[tid]=d_data26[0]*tid;
	    }
	    break;
	case 7:
	    d_output7[tid]=temp1-temp4;
	    if(tid<2) {
		d_output27[tid]=d_data27[0]*tid;
	    }
	    break;
	case 8:
	    d_output8[tid]=temp1-temp4;
	    break;
	case 9:
	    d_output9[tid]=temp1-temp4;
	    break;
	case 10:
	    d_output10[tid]=temp1-temp4;
	    break;
	case 11:
	    d_output11[tid]=temp1-temp4;
	    break;
	}
    }
}


__global__ void exclscnmb(int *d_data, int *d_output, int size) {
    __shared__ int temp[32];
    int temp1, temp2, temp3, temp4;
    if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    int tid = threadIdx.x;
    temp4 = temp1 = (tid+blockIdx.x*blockDim.x<size)?d_data[tid+blockIdx.x*blockDim.x]:0;
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
        if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (threadIdx.x < 32) {
        temp2 = 0;
        if (tid < blockDim.x/32) {
	    temp2 = temp[threadIdx.x];
        }
        for (int d=1; d<32; d<<=1) {
	    temp3 = __shfl_up(temp2,d);
	    if (tid%32 >= d) {temp2 += temp3;}
        }
        if (tid < blockDim.x/32) { temp[tid] = temp2; }
    }
    __syncthreads();
    if (tid >= 32) { temp1 += temp[tid/32 - 1]; }
    __syncthreads();
    if(tid+blockIdx.x*blockDim.x<size) {
        d_output[tid+blockIdx.x*blockDim.x]=temp1-temp4;
    }
}

__global__ void exclscan(int *d_data, int *d_output, int size, sblockds_t *ptoblockds) {
    __shared__ int temp[32];
    __shared__ unsigned int my_blockId;
    int temp1, temp2, temp3, temp4;
    if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    if (threadIdx.x==0) {
	my_blockId = atomicAdd( &(ptoblockds->g_block_id), 1 );
    }
    __syncthreads();
    int tid = threadIdx.x;
    temp4 = temp1 = (tid+my_blockId*blockDim.x<size)?d_data[tid+my_blockId*blockDim.x]:0;
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
        if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (threadIdx.x < 32) {
        temp2 = 0;
        if (tid < blockDim.x/32) {
	    temp2 = temp[threadIdx.x];
        }
        for (int d=1; d<32; d<<=1) {
	    temp3 = __shfl_up(temp2,d);
	    if (tid%32 >= d) {temp2 += temp3;}
        }
        if (tid < blockDim.x/32) { temp[tid] = temp2; }
    }
    __syncthreads();
    if (tid >= 32) { temp1 += temp[tid/32 - 1]; }
    __syncthreads();
    if (threadIdx.x==(blockDim.x-1)) {
        do {} while( atomicAdd(&(ptoblockds->g_blockcnt),0) < my_blockId );
        temp[0]=ptoblockds->sum;
        if(my_blockId==(gridDim.x-1)) { /* it is the last block; reset for next iteration */
	    ptoblockds->sum=0;
	    ptoblockds->g_blockcnt=0;
	    ptoblockds->g_block_id=0;
        } else {
	    ptoblockds->sum=temp[0]+temp1;
	    atomicAdd(&(ptoblockds->g_blockcnt),1);
        }
        __threadfence();  // wait for write completion
    }
    __syncthreads();
    temp1+=temp[0];
    if(tid+my_blockId*blockDim.x<size) {
        d_output[tid+my_blockId*blockDim.x]=temp1-temp4;
    }
}

__global__ void excl26scan(const int *d_data30,  int *d_output30,
			   const int *d_data31,  int *d_output31,
			   const int *d_data32,  int *d_output32,
			   const int *d_data33,  int *d_output33,
			   const int *d_data34,  int *d_output34,
			   const int *d_data35,  int *d_output35,
			   const int *d_data0, int *d_output0,
			   const int *d_data1, int *d_output1,
			   const int *d_data2, int *d_output2,
			   const int *d_data3, int *d_output3,
			   const int *d_data4, int *d_output4,
			   const int *d_data5, int *d_output5,
			   const int *d_data6, int *d_output6,
			   const int *d_data7, int *d_output7,
			   const int *d_data8, int *d_output8,
			   const int *d_data9, int *d_output9,
			   const int *d_data10, int *d_output10,
			   const int *d_data11, int *d_output11,
			   const int *d_data20, int *d_output20,
			   const int *d_data21, int *d_output21,
			   const int *d_data22, int *d_output22,
			   const int *d_data23, int *d_output23,
			   const int *d_data24, int *d_output24,
			   const int *d_data25, int *d_output25,
			   const int *d_data26, int *d_output26,
			   const int *d_data27, int *d_output27,
			   const int *size, int maxsize, sblockds_t *ptoblockds)  {
    __shared__ int temp[32];
    __shared__ unsigned int my_blockId;
    int temp1, temp2, temp3, temp4;
    if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
    }
    temp3=6*((maxsize+(blockDim.x-1))/blockDim.x);
    int tid = threadIdx.x;
    const int which=blockIdx.x/((maxsize+(blockDim.x-1))/blockDim.x);
    if(blockIdx.x>=temp3) {
	goto smallscan;
    }
    if (threadIdx.x==0) {
	my_blockId = atomicAdd( &(ptoblockds[which].g_block_id), 1 );
    }
    __syncthreads();
    if(my_blockId*blockDim.x>=size[which]) {
	return;
    }
    switch(which) {
    case 0:
	temp4 = temp1 = (tid+my_blockId*blockDim.x<size[0])?
	    d_data30[tid+my_blockId*blockDim.x]:0;
	break;
    case 1:
	temp4 = temp1 = (tid+my_blockId*blockDim.x<size[1])?
	    d_data31[tid+my_blockId*blockDim.x]:0;
	break;
    case 2:
	temp4 = temp1 = (tid+my_blockId*blockDim.x<size[2])?
	    d_data32[tid+my_blockId*blockDim.x]:0;
	break;
    case 3:
	temp4 = temp1 = (tid+my_blockId*blockDim.x<size[3])?
	    d_data33[tid+my_blockId*blockDim.x]:0;
	break;
    case 4:
	temp4 = temp1 = (tid+my_blockId*blockDim.x<size[4])?
	    d_data34[tid+my_blockId*blockDim.x]:0;
	break;
    case 5:
	temp4 = temp1 = (tid+my_blockId*blockDim.x<size[5])?
	    d_data35[tid+my_blockId*blockDim.x]:0;
	break;
    }

    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
        if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (threadIdx.x < 32) {
        temp2 = 0;
        if (tid < blockDim.x/32) {
	    temp2 = temp[threadIdx.x];
        }
        for (int d=1; d<32; d<<=1) {
	    temp3 = __shfl_up(temp2,d);
	    if (tid%32 >= d) {temp2 += temp3;}
        }
        if (tid < blockDim.x/32) { temp[tid] = temp2; }
    }
    __syncthreads();
    if (tid >= 32) { temp1 += temp[tid/32 - 1]; }
    __syncthreads();
    if (threadIdx.x==(blockDim.x-1)) {
        do {} while( atomicAdd(&(ptoblockds[which].g_blockcnt),0) < my_blockId );
        temp[0]=ptoblockds[which].sum;
        if(my_blockId==(((size[which]+(blockDim.x-1))/blockDim.x)-1)) { /* it is the last block; reset for next iteration */
	    ptoblockds[which].sum=0;
	    ptoblockds[which].g_blockcnt=0;
	    ptoblockds[which].g_block_id=0;
        } else {
	    ptoblockds[which].sum=temp[0]+temp1;
	    atomicAdd(&(ptoblockds[which].g_blockcnt),1);
        }
        __threadfence();  // wait for write completion
    }
    __syncthreads();
    temp1+=temp[0];
    if(tid+my_blockId*blockDim.x<size[which]) {
	switch(which) {
        case 0:
	    d_output30[tid+my_blockId*blockDim.x]=temp1-temp4;
	    break;
        case 1:
	    d_output31[tid+my_blockId*blockDim.x]=temp1-temp4;
	    break;
        case 2:
	    d_output32[tid+my_blockId*blockDim.x]=temp1-temp4;
	    break;
        case 3:
	    d_output33[tid+my_blockId*blockDim.x]=temp1-temp4;
	    break;
        case 4:
	    d_output34[tid+my_blockId*blockDim.x]=temp1-temp4;
	    break;
        case 5:
	    d_output35[tid+my_blockId*blockDim.x]=temp1-temp4;
	    break;
	}
    }
    return;
smallscan:
    if(tid>=(((size[6+blockIdx.x-temp3]+WARPSIZE-1)/WARPSIZE)*WARPSIZE)) {
	return;
    }
    switch(blockIdx.x-temp3) {
    case 0:
	temp4 = temp1 = (tid<size[6])?d_data0[tid]:0;
	break;
    case 1:
	temp4 = temp1 = (tid<size[7])?d_data1[tid]:0;
	break;
    case 2:
	temp4 = temp1 = (tid<size[8])?d_data2[tid]:0;
	break;
    case 3:
	temp4 = temp1 = (tid<size[9])?d_data3[tid]:0;
	break;
    case 4:
	temp4 = temp1 = (tid<size[10])?d_data4[tid]:0;
	break;
    case 5:
	temp4 = temp1 = (tid<size[11])?d_data5[tid]:0;
	break;
    case 6:
	temp4 = temp1 = (tid<size[12])?d_data6[tid]:0;
	break;
    case 7:
	temp4 = temp1 = (tid<size[13])?d_data7[tid]:0;
	break;
    case 8:
	temp4 = temp1 = (tid<size[14])?d_data8[tid]:0;
	break;
    case 9:
	temp4 = temp1 = (tid<size[15])?d_data9[tid]:0;
	break;
    case 10:
	temp4 = temp1 = (tid<size[16])?d_data10[tid]:0;
	break;
    case 11:
	temp4 = temp1 = (tid<size[17])?d_data11[tid]:0;
	break;
    default:
	return;
    }
    for (int d=1; d<32; d<<=1) {
	temp2 = __shfl_up(temp1,d);
	if (tid%32 >= d) temp1 += temp2;
    }
    if (tid%32 == 31) temp[tid/32] = temp1;
    __syncthreads();
    if (tid >= 32) { 
	temp1 += temp[0];
	if(tid >= 64) {
	    temp1 += temp[1];
	    if(tid>=96) {
		temp1 += temp[2];
		if(tid>=128) {
		    temp1 += temp[3];
		    if(tid>=160) {
			temp1 += temp[4];
			if(tid>=192) {
			    temp1 += temp[5];
			    if(tid>=224) {
				temp1 += temp[6];
				if(tid>=256) {
				    temp1 += temp[7];
				}
			    }
			}
		    }
		}
	    }
	}
    }
    switch(blockIdx.x-temp3) {
    case 0:
	if(tid<size[6]) {
	    d_output0[tid]=temp1-temp4;
	}	
	if(tid<2) {
	    d_output20[tid]=d_data20[0]*tid;
	}
	break;
    case 1:
	if(tid<size[7]) {
	    d_output1[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output21[tid]=d_data21[0]*tid;
	}
	break;
    case 2:
	if(tid<size[8]) {
	    d_output2[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output22[tid]=d_data22[0]*tid;
	}
	break;
    case 3:
	if(tid<size[9]) {
	    d_output3[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output23[tid]=d_data23[0]*tid;
	}
	break;
    case 4:
	if(tid<size[10]) {
	    d_output4[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output24[tid]=d_data24[0]*tid;
	}
	break;
    case 5:
	if(tid<size[11]) {
	    d_output5[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output25[tid]=d_data25[0]*tid;
	}
	break;
    case 6:
	if(tid<size[12]) {
	    d_output6[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output26[tid]=d_data26[0]*tid;
	}
	break;
    case 7:
	if(tid<size[13]) {
	    d_output7[tid]=temp1-temp4;
	}
	if(tid<2) {
	    d_output27[tid]=d_data27[0]*tid;
	}
	break;
    case 8:
	if(tid<size[14]) {
	    d_output8[tid]=temp1-temp4;
	}
	break;
    case 9:
	if(tid<size[15]) {
	    d_output9[tid]=temp1-temp4;
	}
	break;
    case 10:
	if(tid<size[16]) {
	    d_output10[tid]=temp1-temp4;
	}
	break;
    case 11:
	if(tid<size[17]) {
	    d_output11[tid]=temp1-temp4;
	}
	break;
    }
  
}

__global__ void excl26scanaopob(int **d_data,  int **d_output,
                              int *size)  {
  __shared__ int temp[3][WARPSIZE];
  __shared__ int sum;
  int temp1, temp2, temp3, temp4;
  if(blockDim.x>MAXTHREADS) {
        printf("Invalid number of threads per block: %d, must be <=%d\n",blockDim.x,MAXTHREADS);
  }
  const int tid = threadIdx.x;
  int which=blockIdx.x;
  const int iiw = tid%32;
  const int *linput=d_data[which];
  int *loutput=d_output[which];
  const int lsize=size[which];
  if(blockIdx.x>=6) {
	goto smallscan;
  }
  if(tid==0) {
	sum=0;
  }
  for(int i=tid; i<(((lsize+WARPSIZE-1)/WARPSIZE)*WARPSIZE); i+=blockDim.x) {
  temp4 = temp1 = (i<lsize)?linput[i]:0;
  for (int d=1; d<32; d<<=1) {
         temp2 = __shfl_up(temp1,d);
         temp1 += temp2*(iiw>=d);
  }
  if (iiw == 31) temp[i/blockDim.x][tid/32] = temp1;
  __syncthreads();
  if (tid < 32) {
        temp2 = 0;
        if (tid < blockDim.x/32) {
                temp2 = temp[i/blockDim.x][tid];
        }
        for (int d=1; d<32; d<<=1) {
         temp3 = __shfl_up(temp2,d);
         temp2 += temp3*(iiw>=d);
        }
        if (tid < blockDim.x/32) { temp[i/blockDim.x][tid] = temp2; }
  }
  temp3=sum;
  __syncthreads();
  if(tid>=32) temp1+=temp[i/blockDim.x][tid/32-1];
  if(i<lsize) {
    loutput[i]=temp3+temp1-temp4;
  }
  if (tid==(blockDim.x-1)) {
	sum+=temp1;
  }
  }
  return;
smallscan:
  if(tid>=(((lsize+WARPSIZE-1)/WARPSIZE)*WARPSIZE)) {
	return;
  }
  which=blockIdx.x;
  temp4 = temp1 = (tid<lsize)?linput[tid]:0;
  for (int d=1; d<32; d<<=1) {
         temp2 = __shfl_up(temp1,d);
         if (tid%32 >= d) temp1 += temp2;
  }
  if (tid%32 == 31) temp[0][tid/32] = temp1;
  __syncthreads();
  if (tid >= 32) { 
	temp1 += temp[0][0];
	if(tid >= 64) {
		temp1 += temp[0][1];
		if(tid>=96) {
			temp1 += temp[0][2];
			if(tid>=128) {
				temp1 += temp[0][3];
				if(tid>=160) {
					temp1 += temp[0][4];
					if(tid>=192) {
						temp1 += temp[0][5];
						if(tid>=224) {
							temp1 += temp[0][6];
							if(tid>=256) {
							   temp1 += temp[0][7];
						        }
						}
					}
				}
			}
		}
	}
  }
  if(tid<lsize) {
      	loutput[tid]=temp1-temp4;
  }
  if(which<14) {
	if(tid<2) {
        	d_output[which+12][tid]=d_data[which+12][0]*tid;
	}
  } 
}

#undef MAXTHREADS
#undef WARPSIZE

#include "common.h"

 
using namespace std;
void scan_massimo(const int * const count[26], int * const result[26], const int sizes[26], hipStream_t stream)
{
    CUDA_CHECK(hipPeekAtLastError());
#if defined(_TIME_PROFILE_)
    static  int lit=0;
    if(lit==0) {
        CUDA_CHECK(hipEventCreate(&evstart));
        CUDA_CHECK(hipEventCreate(&evstop));
    }
#endif
    static int newscani=1;
//    static  sblockds_t *ptoblockds;

    int mb[6], mw[12], maxscan=0;
    static int **d_aopd, **d_aopr;
    static int *d_sizescan;
    int h_sizescan[18];
    const int *h_aopd[26], *h_aopr[26];
    if(newscani) {
//        CUDA_CHECK(hipMalloc((void **)&ptoblockds,6*sizeof(sblockds_t)));
//        CUDA_CHECK(hipMemset(ptoblockds,0,6*sizeof(sblockds_t)));
        CUDA_CHECK(hipMalloc((void **)&d_sizescan,18*sizeof(int)));
        CUDA_CHECK(hipMalloc((void ***)&d_aopd,sizeof(int *)*26));
        CUDA_CHECK(hipMalloc((void ***)&d_aopr,sizeof(int *)*26));
        mb[0]=8;
        mb[1]=17;
        mb[2]=20;
        mb[3]=23;
        mb[4]=24;
        mb[5]=25;
        mw[0]=2;
        mw[1]=5;
        mw[2]=6;
        mw[3]=7;
        mw[4]=11;
        mw[5]=14;
        mw[6]=15;
        mw[7]=16;
        mw[8]=18;
        mw[9]=19;
        mw[10]=21;
        mw[11]=22;
        for(int i = 0; i < 6; ++i) {
                h_sizescan[i]=sizes[mb[i]];
                h_aopd[i]=count[mb[i]];
                h_aopr[i]=result[mb[i]];
                maxscan=(maxscan<h_sizescan[i])?h_sizescan[i]:maxscan;
        }
        for(int i = 0; i < 12; ++i) {
                h_sizescan[6+i]=sizes[mw[i]];
                h_aopd[6+i]=count[mw[i]];
                h_aopr[6+i]=result[mw[i]];
        }
        h_aopd[18]=count[0];
        h_aopr[18]=result[0];
        h_aopd[19]=count[1];
        h_aopr[19]=result[1];
        h_aopd[20]=count[3];
        h_aopr[20]=result[3];
        h_aopd[21]=count[4];
        h_aopr[21]=result[4];
        h_aopd[22]=count[9];
        h_aopr[22]=result[9];
        h_aopd[23]=count[10];
        h_aopr[23]=result[10];
        h_aopd[24]=count[12];
        h_aopr[24]=result[12];
        h_aopd[25]=count[13];
        h_aopr[25]=result[13];
        CUDA_CHECK(hipMemcpy(d_sizescan,h_sizescan,18*sizeof(int),
                              hipMemcpyHostToDevice));
        CUDA_CHECK( hipMemcpy( d_aopd, h_aopd, 26*sizeof(int *),
                                hipMemcpyHostToDevice ) );
        CUDA_CHECK( hipMemcpy( d_aopr, h_aopr, 26*sizeof(int *),
                                hipMemcpyHostToDevice ) );

        newscani=0;
    }


#if defined(_TIME_PROFILE_)
   if (lit % 500 == 0)
        CUDA_CHECK(hipEventRecord(evstart));
#endif
#define NTHREADS 1024
#if 0
        excl26scan<<<12+(6*((maxscan+NTHREADS-1)/NTHREADS)),NTHREADS,0,stream>>>(
          count[mb[0]],result[mb[0]],
          count[mb[1]],result[mb[1]],
          count[mb[2]],result[mb[2]],
          count[mb[3]],result[mb[3]],
          count[mb[4]],result[mb[4]],
          count[mb[5]],result[mb[5]],
          count[mw[0]],result[mw[0]],
          count[mw[1]],result[mw[1]],
          count[mw[2]],result[mw[2]],
          count[mw[3]],result[mw[3]],
          count[mw[4]],result[mw[4]],
          count[mw[5]],result[mw[5]],
          count[mw[6]],result[mw[6]],
          count[mw[7]],result[mw[7]],
          count[mw[8]],result[mw[8]],
          count[mw[9]],result[mw[9]],
          count[mw[10]],result[mw[10]],
          count[mw[11]],result[mw[11]],
          count[0],result[0],
          count[1],result[1],
          count[3],result[3],
          count[4],result[4],
          count[9],result[9],
          count[10],result[10],
          count[12],result[12],
          count[13],result[13],
          d_sizescan, maxscan, ptoblockds);
#else
	excl26scanaopob<<<18,NTHREADS, 0, stream>>>(d_aopd, d_aopr, d_sizescan);
#endif
#if defined(_TIME_PROFILE_)
    if (lit % 500 == 0)
    {
        CUDA_CHECK(hipEventRecord(evstop));
        CUDA_CHECK(hipEventSynchronize(evstop));

        float tms;
        CUDA_CHECK(hipEventElapsedTime(&tms, evstart, evstop));
        if(cntlwtimer<maxcntimer) {
                lwtimer[cntlwtimer++]=tms;
        }
    }
    lit++;
#endif
}
