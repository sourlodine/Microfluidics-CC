#include "hip/hip_runtime.h"
/*
 *  simulation.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-03-24.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include "simulation.h"

std::vector<Particle> Simulation::_ic()
{
    srand48(rank);

    std::vector<Particle> ic(XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN * numberdensity);

    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

    for(int iz = 0; iz < L[2]; iz++)
	for(int iy = 0; iy < L[1]; iy++)
	    for(int ix = 0; ix < L[0]; ix++)
		for(int l = 0; l < numberdensity; ++l)
		{
		    const int p = l + numberdensity * (ix + L[0] * (iy + L[1] * iz));

		    ic[p].x[0] = -L[0]/2 + ix + drand48();
		    ic[p].x[1] = -L[1]/2 + iy + drand48();
		    ic[p].x[2] = -L[2]/2 + iz + drand48();
		    ic[p].u[0] = 0;
		    ic[p].u[1] = 0;
		    ic[p].u[2] = 0;
		}

    /* use this to check robustness
       for(int i = 0; i < ic.size(); ++i)
       for(int c = 0; c < 3; ++c)
       {
       ic[i].x[c] = -L[c] * 0.5 + drand48() * L[c];
       ic[i].u[c] = 0;
       }
    */

    return ic;
}

void Simulation::_redistribute()
{
    double tstart = MPI_Wtime();

    redistribute.pack(particles.xyzuvw.data, particles.size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    redistribute.send();

    if (rbcscoll)
	redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);

    redistribute.bulk(particles.size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);

    int nrbcs;
    if (rbcscoll)
	nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
	nctcs = redistribute_ctcs.post();

    if (rbcscoll)
	rbcscoll->resize(nrbcs);

    if (ctcscoll)
	ctcscoll->resize(nctcs);

    unordered_particles.resize(newnp);

    redistribute.recv_unpack(unordered_particles.data, newnp, mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    particles.resize(newnp);

    cells.build(particles.xyzuvw.data, particles.size, mainstream, NULL, unordered_particles.data);

    if (rbcscoll)
	redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    localcomm.barrier();

    timings["redistribute"] += MPI_Wtime() - tstart;
}

void Simulation::_report(const bool verbose, const int idtimestep)
{
    report_host_memory_usage(activecomm, stdout);

    {
	static double t0 = MPI_Wtime(), t1;

	t1 = MPI_Wtime();

	float host_busy_time = (MPI_Wtime() - t0) - host_idle_time;

	host_busy_time *= 1e3 / steps_per_report;

	float sumval, maxval, minval;
	MPI_CHECK(MPI_Reduce(&host_busy_time, &sumval, 1, MPI_FLOAT, MPI_SUM, 0, activecomm));
	MPI_CHECK(MPI_Reduce(&host_busy_time, &maxval, 1, MPI_FLOAT, MPI_MAX, 0, activecomm));
	MPI_CHECK(MPI_Reduce(&host_busy_time, &minval, 1, MPI_FLOAT, MPI_MIN, 0, activecomm));

	int commsize;
	MPI_CHECK(MPI_Comm_size(activecomm, &commsize));

	const double imbalance = 100 * (maxval / sumval * commsize - 1);

	if (verbose && imbalance >= 0)
	    printf("\x1b[93moverall imbalance: %.f%%, host workload min/avg/max: %.2f/%.2f/%.2f ms\x1b[0m\n",
		   imbalance , minval, sumval / commsize, maxval);

	host_idle_time = 0;
	t0 = t1;
    }

    {
	static double t0 = MPI_Wtime(), t1;

	t1 = MPI_Wtime();

	if (verbose)
	{
	    printf("\x1b[92mbeginning of time step %d (%.3f ms)\x1b[0m\n", idtimestep, (t1 - t0) * 1e3 / steps_per_report);
	    printf("in more details, per time step:\n");
	    double tt = 0;
	    for(std::map<string, double>::iterator it = timings.begin(); it != timings.end(); ++it)
	    {
		printf("%s: %.3f ms\n", it->first.c_str(), it->second * 1e3 / steps_per_report);
		tt += it->second;
		it->second = 0;
	    }
	    printf("discrepancy: %.3f ms\n", ((t1 - t0) - tt) * 1e3 / steps_per_report);
	}

	t0 = t1;
    }
}

void Simulation::_remove_bodies_from_wall(CollectionRBC * coll)
{
    if(!coll || !coll->count())
	return;

    SimpleDeviceBuffer<int> marks(coll->pcount());

    SolidWallsKernel::fill_keys<<< (coll->pcount() + 127) / 128, 128 >>>(coll->data(), coll->pcount(), marks.data);

    vector<int> tmp(marks.size);
    CUDA_CHECK(hipMemcpy(tmp.data(), marks.data, sizeof(int) * marks.size, hipMemcpyDeviceToHost));

    const int nbodies = coll->count();
    const int nvertices = coll->nvertices;

    std::vector<int> tokill;
    for(int i = 0; i < nbodies; ++i)
    {
	bool valid = true;

	for(int j = 0; j < nvertices && valid; ++j)
	    valid &= 0 == tmp[j + nvertices * i];

	if (!valid)
	    tokill.push_back(i);
    }

    coll->remove(&tokill.front(), tokill.size());
    coll->clear_velocity();

    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_create_walls(const bool verbose, bool & termination_request)
{
    if (verbose)
	printf("creation of the walls...\n");

    int nsurvived = 0;
    ExpectedMessageSizes new_sizes;
    wall = new ComputeInteractionsWall(cartcomm, particles.xyzuvw.data, particles.size, nsurvived, new_sizes, verbose);

    //adjust the message sizes if we're pushing the flow in x
    {
	const double xvelavg = getenv("XVELAVG") ? atof(getenv("XVELAVG")) : pushtheflow;
	const double yvelavg = getenv("YVELAVG") ? atof(getenv("YVELAVG")) : 0;
	const double zvelavg = getenv("ZVELAVG") ? atof(getenv("ZVELAVG")) : 0;

	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		((code / 3) % 3) - 1,
		((code / 9) % 3) - 1
	    };

	    const double IudotnI =
		fabs(d[0] * xvelavg) +
		fabs(d[1] * yvelavg) +
		fabs(d[2] * zvelavg) ;

	    const float factor = 1 + IudotnI * dt * 10 * numberdensity;

	    //printf("RANK %d: direction %d %d %d -> IudotnI is %f and final factor is %f\n",
	    //rank, d[0], d[1], d[2], IudotnI, 1 + IudotnI * dt * numberdensity);

	    new_sizes.msgsizes[code] *= factor;
	}
    }

    MPI_CHECK(MPI_Barrier(activecomm));
    redistribute.adjust_message_sizes(new_sizes);
    dpd.adjust_message_sizes(new_sizes);
    MPI_CHECK(MPI_Barrier(activecomm));

    //there is no support for killing zero-workload ranks for rbcs and ctcs just yet
    /* this is unnecessarily complex for now
       if (!rbcs && !ctcs)
    {
	const bool local_work = new_sizes.msgsizes[1 + 3 + 9] > 0;

	MPI_CHECK(MPI_Comm_split(cartcomm, local_work, rank, &activecomm)) ;

	MPI_CHECK(MPI_Comm_rank(activecomm, &rank));

	if (!local_work )
	{
	    if (rank == 0)
	    {
		int nkilled;
		MPI_CHECK(MPI_Comm_size(activecomm, &nkilled));

		printf("THERE ARE %d RANKS WITH ZERO WORKLOAD THAT WILL MPI-FINALIZE NOW.\n", nkilled);
	    }

	    termination_request = true;
	    return;
	}
    }
    */

    particles.resize(nsurvived);
    particles.clear_velocity();

    CUDA_CHECK(hipPeekAtLastError());

    //remove cells touching the wall
    _remove_bodies_from_wall(rbcscoll);
    _remove_bodies_from_wall(ctcscoll);

    {
	H5PartDump sd("survived-particles.h5part", activecomm, cartcomm);
	Particle * p = new Particle[particles.size];

	CUDA_CHECK(hipMemcpy(p, particles.xyzuvw.data, sizeof(Particle) * particles.size, hipMemcpyDeviceToHost));

	sd.dump(p, particles.size);

	delete [] p;
    }

    if (rank == 0)
    {
	if( access( "particles.xyz", F_OK ) != -1 )
	{
	    const int retval = rename ("particles.xyz", "particles-equilibration.xyz");
	    assert(retval != -1);
	}

	if( access( "rbcs.xyz", F_OK ) != -1 )
	{
	    const int retval = rename ("rbcs.xyz", "rbcs-equilibration.xyz");
	    assert(retval != -1);
	}
    }
}

void Simulation::_forces()
{
    double tstart = MPI_Wtime();

    particles.clear_acc(mainstream);

    if (rbcscoll)
	rbcscoll->clear_acc(mainstream);

    if (ctcscoll)
    	ctcscoll->clear_acc(mainstream);

    if (rbcscoll)
	rbc_interactions.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	ctc_interactions.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    if (rbcscoll)
	rbc_interactions.count(rbcscoll->count());

    if (ctcscoll)
	ctc_interactions.count(ctcscoll->count());

    dpd.pack(particles.xyzuvw.data, particles.size, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions. pack_p(rbcscoll->data(), mainstream);

    if (ctcscoll)
	ctc_interactions.pack_p(ctcscoll->data(), mainstream);

    dpd.consolidate_and_post(particles.xyzuvw.data, particles.size, mainstream);

    dpd.local_interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions.exchange_count();

    if (ctcscoll)
	ctc_interactions.exchange_count();

    if (rbcscoll)
	rbc_interactions.post_p();

    if (ctcscoll)
	ctc_interactions.post_p();

    if (rbcscoll)
	rbc_interactions.fsi_bulk(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_bulk(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll && wall)
	wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);

    if (ctcscoll && wall)
	wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);

    if (rbcscoll)
	rbc_interactions.fsi_halo(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_halo(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbc_interactions.internal_forces(rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.internal_forces(ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (wall)
	wall->interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data,
			   cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions.post_a();

    if (ctcscoll)
	ctc_interactions.post_a();

    dpd.wait_for_messages(mainstream);
    dpd.remote_interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, mainstream);

    if (rbcscoll)
	rbc_interactions.merge_a(rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.merge_a(ctcscoll->acc(), mainstream);

    timings["interactions"] += MPI_Wtime() - tstart;

    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_datadump(const int idtimestep)
{
    double tstart = MPI_Wtime();

    pthread_mutex_lock(&mutex_datadump);

    while (datadump_pending)
	pthread_cond_wait(&done_datadump, &mutex_datadump);

    int n = particles.size;

    if (rbcscoll)
	n += rbcscoll->pcount();

    if (ctcscoll)
	n += ctcscoll->pcount();

    particles_datadump.resize(n);
    accelerations_datadump.resize(n);

    CUDA_CHECK(hipMemcpyAsync(particles_datadump.data, particles.xyzuvw.data, sizeof(Particle) * particles.size, hipMemcpyDeviceToHost,0));
    CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data, particles.axayaz.data, sizeof(Acceleration) * particles.size, hipMemcpyDeviceToHost,0));

    int start = particles.size;

    if (rbcscoll)
    {
	CUDA_CHECK(hipMemcpyAsync(particles_datadump.data + start, rbcscoll->xyzuvw.data, sizeof(Particle) * rbcscoll->pcount(), hipMemcpyDeviceToHost, 0));
	CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data + start, rbcscoll->axayaz.data, sizeof(Acceleration) * rbcscoll->pcount(), hipMemcpyDeviceToHost, 0));

	start += rbcscoll->pcount();
    }

    if (ctcscoll)
    {
	CUDA_CHECK(hipMemcpyAsync(particles_datadump.data + start, ctcscoll->xyzuvw.data, sizeof(Particle) * ctcscoll->pcount(), hipMemcpyDeviceToHost, 0));
	CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data + start, ctcscoll->axayaz.data, sizeof(Acceleration) * ctcscoll->pcount(), hipMemcpyDeviceToHost, 0));

	start += ctcscoll->pcount();
    }

    assert(start == n);

    CUDA_CHECK(hipEventRecord(evdownloaded, 0));

    datadump_idtimestep = idtimestep;
    datadump_nsolvent = particles.size;
    datadump_nrbcs = rbcscoll ? rbcscoll->pcount() : 0;
    datadump_nctcs = ctcscoll ? ctcscoll->pcount() : 0;
    datadump_pending = true;

    pthread_cond_signal(&request_datadump);

    pthread_mutex_unlock(&mutex_datadump);

    timings["data-dump"] += MPI_Wtime() - tstart;
}

void Simulation::_datadump_async()
{
#ifdef _USE_NVTX_
    nvtxNameOsThread(pthread_self(), "DATADUMP_THREAD");
#endif

    int iddatadump = 0;
    int curr_idtimestep = -1;

    MPI_Comm myactivecomm, mycartcomm;

    MPI_CHECK(MPI_Comm_dup(activecomm, &myactivecomm) );
    MPI_CHECK(MPI_Comm_dup(cartcomm, &mycartcomm) );

    H5PartDump dump_part("allparticles.h5part", activecomm, cartcomm), *dump_part_solvent = NULL;
    H5FieldDump dump_field(cartcomm);

    while (true)
    {
	pthread_mutex_lock(&mutex_datadump);

	while (!datadump_pending)
	    pthread_cond_wait(&request_datadump, &mutex_datadump);

	pthread_mutex_unlock(&mutex_datadump);

	if (curr_idtimestep == datadump_idtimestep)
	    if (simulation_is_done)
		break;

	CUDA_CHECK(hipEventSynchronize(evdownloaded));

	const int n = particles_datadump.size;
	Particle * p = particles_datadump.data;
	Acceleration * a = accelerations_datadump.data;

	{
	    NVTX_RANGE("diagnostics", NVTX_C1);
	    diagnostics(myactivecomm, mycartcomm, p, n, dt, datadump_idtimestep, a);
	}

	if (xyz_dumps)	
	{
	    NVTX_RANGE("xyz dump", NVTX_C2);
	    xyz_dump(myactivecomm, mycartcomm, "particles.xyz", "all-particles", p, n, datadump_idtimestep > 0);
	}

	if (hdf5part_dumps)
	{	
	    NVTX_RANGE("h5part dump", NVTX_C3);

	    if (!dump_part_solvent && walls && datadump_idtimestep >= wall_creation_stepid)
	    {
		dump_part.close();

		dump_part_solvent = new H5PartDump("solvent-particles.h5part", activecomm, cartcomm);
	    }

	    if (dump_part_solvent)
		dump_part_solvent->dump(p, n);
	    else
		dump_part.dump(p, n);
	}

	if (hdf5field_dumps)
	{
	    NVTX_RANGE("hdf5 field dump", NVTX_C4);

	    dump_field.dump(activecomm, p, particles.size, datadump_idtimestep);
	}

	{
	    NVTX_RANGE("ply dump", NVTX_C5);

	    if (rbcscoll)
		rbcscoll->dump(myactivecomm, mycartcomm, p + datadump_nsolvent, a + datadump_nsolvent, datadump_nrbcs, iddatadump);

	    if (ctcscoll)
		ctcscoll->dump(myactivecomm, mycartcomm, p + datadump_nsolvent + datadump_nrbcs, a + datadump_nsolvent + datadump_nrbcs, datadump_nctcs, iddatadump);
	}

	curr_idtimestep = datadump_idtimestep;

	pthread_mutex_lock(&mutex_datadump);

	datadump_pending = false;

	pthread_cond_signal(&request_datadump);

	pthread_mutex_unlock(&mutex_datadump);

	++iddatadump;
    }

    if (dump_part_solvent)
	delete dump_part_solvent;

    CUDA_CHECK(hipEventDestroy(evdownloaded));
}

void Simulation::_update_and_bounce()
{
    double tstart = MPI_Wtime();
    particles.update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (ctcscoll)
	ctcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    timings["update"] += MPI_Wtime() - tstart;

    if (wall)
    {
	tstart = MPI_Wtime();
	wall->bounce(particles.xyzuvw.data, particles.size, mainstream);

	if (rbcscoll)
	    wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);

	if (ctcscoll)
	    wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);

	timings["bounce-walls"] += MPI_Wtime() - tstart;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

Simulation::Simulation(MPI_Comm cartcomm, MPI_Comm activecomm, bool (*check_termination)()) :
    cartcomm(cartcomm), activecomm(activecomm),
    particles(_ic()), cells(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN),
    rbcscoll(NULL), ctcscoll(NULL), wall(NULL),
    redistribute(cartcomm),  redistribute_rbcs(cartcomm),  redistribute_ctcs(cartcomm),
    dpd(cartcomm), rbc_interactions(cartcomm), ctc_interactions(cartcomm),
    check_termination(check_termination),
    driving_acceleration(0), host_idle_time(0), nsteps((int)(tend / dt)),
    datadump_pending(false), simulation_is_done(false)
{
    localcomm.initialize(activecomm);

    MPI_CHECK( MPI_Comm_size(activecomm, &nranks) );
    MPI_CHECK( MPI_Comm_rank(activecomm, &rank) );

    CUDA_CHECK(hipStreamCreate(&mainstream));

    if (rbcs)
    {
	rbcscoll = new CollectionRBC(cartcomm);
	rbcscoll->setup();
    }

    if (ctcs)
    {
	ctcscoll = new CollectionCTC(cartcomm);
	ctcscoll->setup();
    }

    //setting up the asynchronous data dumps
    {
	CUDA_CHECK(hipEventCreate(&evdownloaded, hipEventDisableTiming | hipEventBlockingSync));

	particles_datadump.resize(particles.size * 1.5);
	accelerations_datadump.resize(particles.size * 1.5);

	int rc = pthread_mutex_init(&mutex_datadump, NULL);
	rc |= pthread_cond_init(&done_datadump, NULL);
	rc |= pthread_cond_init(&request_datadump, NULL);
	rc |= pthread_create(&thread_datadump, NULL, datadump_trampoline, this);

	if (rc)
	{
	    printf("ERROR; return code from pthread_create() is %d\n", rc);
	    exit(-1);
	}
    }
}

void Simulation::_lockstep()
{
    double tstart = MPI_Wtime();

    particles.clear_acc(mainstream);

    if (rbcscoll)
	rbcscoll->clear_acc(mainstream);

    if (ctcscoll)
    	ctcscoll->clear_acc(mainstream);

    if (rbcscoll)
	rbc_interactions.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	ctc_interactions.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    dpd.pack(particles.xyzuvw.data, particles.size, cells.start, cells.count, mainstream);

    if (rbcscoll)
	rbc_interactions.count(rbcscoll->count());

    if (ctcscoll)
	ctc_interactions.count(ctcscoll->count());

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions.pack_p(rbcscoll->data(), mainstream);

    if (ctcscoll)
	ctc_interactions.pack_p(ctcscoll->data(), mainstream);

    dpd.consolidate_and_post(particles.xyzuvw.data, particles.size, mainstream);

    dpd.local_interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    localcomm.barrier(); // peh: 1

    if (rbcscoll)
	rbc_interactions.exchange_count();

    if (ctcscoll)
	ctc_interactions.exchange_count();

    if (rbcscoll)
	rbc_interactions.post_p();

    if (ctcscoll)
	ctc_interactions.post_p();

    if (rbcscoll)
	rbc_interactions.fsi_bulk(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_bulk(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbc_interactions.fsi_halo(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_halo(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbc_interactions.post_a();

    if (ctcscoll)
	ctc_interactions.post_a();

    if (wall)
	wall->interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data,
			   cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    dpd.wait_for_messages(mainstream);

    dpd.remote_interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, mainstream);

    particles.update_stage2_and_1(driving_acceleration, mainstream);

    if (wall)
	wall->bounce(particles.xyzuvw.data, particles.size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    redistribute.pack(particles.xyzuvw.data, particles.size, mainstream);

    redistribute.send();

    redistribute.bulk(particles.size, mainstream);

    if (rbcscoll)
	rbc_interactions.internal_forces(rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.internal_forces(ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll && wall)
	wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);

    if (ctcscoll && wall)
	wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);

    if (rbcscoll)
	rbc_interactions.merge_a(rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.merge_a(ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (ctcscoll)
	ctcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    if (wall && rbcscoll)
	wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);

    if (wall && ctcscoll)
	wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    if (rbcscoll)
	redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);

    unordered_particles.resize(newnp);

    redistribute.recv_unpack(unordered_particles.data, newnp, mainstream, host_idle_time);

    localcomm.barrier();	// peh: +2

    particles.resize(newnp);

    cells.build(particles.xyzuvw.data, particles.size, mainstream, NULL, unordered_particles.data);

    int nrbcs;
    if (rbcscoll)
	nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
	nctcs = redistribute_ctcs.post();

    if (rbcscoll)
	rbcscoll->resize(nrbcs);

    if (ctcscoll)
	ctcscoll->resize(nctcs);

    CUDA_CHECK(hipPeekAtLastError());


    if (rbcscoll)
	redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

//  localcomm.barrier();  // peh: +3

    timings["lockstep"] += MPI_Wtime() - tstart;
}


void Simulation::run()
{
    if (rank == 0 && !walls)
	printf("the simulation begins now and it consists of %.3e steps\n", (double)nsteps);

    double time_simulation_start = MPI_Wtime();

    _redistribute();
    _forces();

    if (!walls && pushtheflow)
	driving_acceleration = hydrostatic_a;

    particles.update_stage1(driving_acceleration, mainstream);

    if (rbcscoll)
	rbcscoll->update_stage1(driving_acceleration, mainstream);

    if (ctcscoll)
	ctcscoll->update_stage1(driving_acceleration, mainstream);

    int it;
    enum { nvtxstart = 7651, nvtxstop = 8051 } ;

    for(it = 0; it < nsteps; ++it)
    {
	const bool verbose = it > 0 && rank == 0;

#ifdef _USE_NVTX_
	if (it == nvtxstart)
	{
	    NvtxTracer::currently_profiling = true;
	    CUDA_CHECK(hipProfilerStart());
	}
	else if (it == nvtxstop)
	{
	    CUDA_CHECK(hipProfilerStop());
	    NvtxTracer::currently_profiling = false;
	    CUDA_CHECK(hipDeviceSynchronize());

	    if (rank == 0)
		printf("profiling session ended. terminating the simulation now...\n");

	    break;
	}
#endif

	if (it % steps_per_report == 0)
	{
	    CUDA_CHECK(hipStreamSynchronize(mainstream));

	    if (simulation_is_done = check_termination())
		break;

	    _report(verbose, it);
	}

	_redistribute();

#if 1
    lockstep_check:

	const bool lockstep_OK =
	    !(walls && it >= wall_creation_stepid && wall == NULL) &&
	    !(it % steps_per_dump == 0) &&
	    !(it + 1 == nvtxstart) &&
	    !(it + 1 == nvtxstop) &&
	    !((it + 1) % steps_per_report == 0) &&
	    !(it + 1 == nsteps);

	if (lockstep_OK)
	{
	    _lockstep();

	    ++it;

	    goto lockstep_check;
	}
#endif

	if (walls && it >= wall_creation_stepid && wall == NULL)
	{
	    CUDA_CHECK(hipDeviceSynchronize());

	    bool termination_request = false;

	    _create_walls(verbose, termination_request);

	    _redistribute();

	    if (termination_request)
		break;

	    time_simulation_start = MPI_Wtime();

	    if (pushtheflow)
		driving_acceleration = hydrostatic_a;

	    if (rank == 0)
		printf("the simulation begins now and it consists of %.3e steps\n", (double)(nsteps - it));
	}

	_forces();

	if (it % steps_per_dump == 0)
	    _datadump(it);

	_update_and_bounce();
    }

    const double time_simulation_stop = MPI_Wtime();
    const double telapsed = time_simulation_stop - time_simulation_start;

    simulation_is_done = true;

    if (rank == 0)
	if (it == nsteps)
	    printf("simulation is done after %.2lf s (%dm%ds). Ciao.\n",
		   telapsed, (int)(telapsed / 60), (int)(telapsed) % 60);
	else
	    if (it != wall_creation_stepid)
		printf("external termination request (signal) after %.3e s. Bye.\n", telapsed);

    fflush(stdout);
}

Simulation::~Simulation()
{
    pthread_mutex_lock(&mutex_datadump);

    datadump_pending = true;
    pthread_cond_signal(&request_datadump);

    pthread_mutex_unlock(&mutex_datadump);

    pthread_join(thread_datadump, NULL);

    CUDA_CHECK(hipStreamDestroy(mainstream));

    if (wall)
	delete wall;

    if (rbcscoll)
	delete rbcscoll;

    if (ctcscoll)
	delete ctcscoll;
}
