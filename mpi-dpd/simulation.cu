#include "hip/hip_runtime.h"
/*
 *  simulation.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-03-24.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include "simulation.h"

std::vector<Particle> Simulation::_ic()
{
    srand48(rank);

    std::vector<Particle> ic(XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN * numberdensity);
    
    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
    
    for(int iz = 0; iz < L[2]; iz++)
	for(int iy = 0; iy < L[1]; iy++)
	    for(int ix = 0; ix < L[0]; ix++)
		for(int l = 0; l < numberdensity; ++l)
		{
		    const int p = l + numberdensity * (ix + L[0] * (iy + L[1] * iz));
		    
		    ic[p].x[0] = -L[0]/2 + ix + drand48();
		    ic[p].x[1] = -L[1]/2 + iy + drand48();
		    ic[p].x[2] = -L[2]/2 + iz + drand48();
		    ic[p].u[0] = 0;
		    ic[p].u[1] = 0;
		    ic[p].u[2] = 0;
		}

    /* use this to check robustness 
    for(int i = 0; i < ic.size(); ++i)
	for(int c = 0; c < 3; ++c)
	    {
		ic[i].x[c] = -L[c] * 0.5 + drand48() * L[c];
		ic[i].u[c] = 0;
	    }
    */
    
    return ic;
}

void Simulation::_redistribute()
{
    double tstart = MPI_Wtime();
    
    redistribute.pack(particles.xyzuvw.data, particles.size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll) 
	redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);
    
    if (ctcscoll)
	redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);
    
    redistribute.send();

    if (rbcscoll) 
	redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);
    
    if (ctcscoll)
	redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);
    
    redistribute.bulk(particles.size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);
    
    int nrbcs;
    if (rbcscoll) 
	nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
	nctcs = redistribute_ctcs.post();

    if (rbcscoll) 
	rbcscoll->resize(nrbcs);

    if (ctcscoll)
	ctcscoll->resize(nctcs);

    unordered_particles.resize(newnp);

    redistribute.recv_unpack(unordered_particles.data, newnp, mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    particles.resize(newnp);

    cells.build(particles.xyzuvw.data, particles.size, mainstream, NULL, unordered_particles.data);
    
    if (rbcscoll)
	redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);
    
    CUDA_CHECK(hipPeekAtLastError());

    timings["redistribute"] += MPI_Wtime() - tstart;
}

void Simulation::_report(const bool verbose, const int idtimestep)
{ 
    report_host_memory_usage(activecomm, stdout);
    
    {
	static double t0 = MPI_Wtime(), t1;
	
	t1 = MPI_Wtime();
	
	float host_busy_time = (MPI_Wtime() - t0) - host_idle_time;
	
	host_busy_time *= 1e3 / steps_per_report;
	
	float sumval, maxval, minval;
	MPI_CHECK(MPI_Reduce(&host_busy_time, &sumval, 1, MPI_FLOAT, MPI_SUM, 0, activecomm));
	MPI_CHECK(MPI_Reduce(&host_busy_time, &maxval, 1, MPI_FLOAT, MPI_MAX, 0, activecomm));
	MPI_CHECK(MPI_Reduce(&host_busy_time, &minval, 1, MPI_FLOAT, MPI_MIN, 0, activecomm));
	
	int commsize;
	MPI_CHECK(MPI_Comm_size(activecomm, &commsize));
	
	const double imbalance = 100 * (maxval / sumval * commsize - 1);
	
	if (verbose && imbalance > 5)
	    printf("\x1b[93moverall imbalance: %.f%%, host workload min/avg/max: %.2f/%.2f/%.2f ms\x1b[0m\n", 
		   imbalance , minval, sumval / commsize, maxval);
	
	host_idle_time = 0;
	t0 = t1;
    }
    
    {
	static double t0 = MPI_Wtime(), t1;
	
	t1 = MPI_Wtime();
	
	if (verbose)
	{
	    printf("\x1b[92mbeginning of time step %d (%.3f ms)\x1b[0m\n", idtimestep, (t1 - t0) * 1e3 / steps_per_report);
	    printf("in more details, per time step:\n");
	    double tt = 0;
	    for(std::map<string, double>::iterator it = timings.begin(); it != timings.end(); ++it)
	    {
		printf("%s: %.3f ms\n", it->first.c_str(), it->second * 1e3 / steps_per_report);
		tt += it->second;
		it->second = 0;
	    }
	    printf("discrepancy: %.3f ms\n", ((t1 - t0) - tt) * 1e3 / steps_per_report);
	}
	
	t0 = t1;
    }
}

void Simulation::_remove_bodies_from_wall(CollectionRBC * coll)
{
    if(!coll || !coll->count())
	return;
    
    SimpleDeviceBuffer<int> marks(coll->pcount());
    
    SolidWallsKernel::fill_keys<<< (coll->pcount() + 127) / 128, 128 >>>(coll->data(), coll->pcount(), marks.data);
    
    vector<int> tmp(marks.size);
    CUDA_CHECK(hipMemcpy(tmp.data(), marks.data, sizeof(int) * marks.size, hipMemcpyDeviceToHost));
    
    const int nbodies = coll->count();
    const int nvertices = coll->nvertices;
    
    std::vector<int> tokill;
    for(int i = 0; i < nbodies; ++i)
    {
	bool valid = true;
	
	for(int j = 0; j < nvertices && valid; ++j)
	    valid &= 0 == tmp[j + nvertices * i];
	
	if (!valid)
	    tokill.push_back(i);
    }
    
    coll->remove(&tokill.front(), tokill.size());
    coll->clear_velocity();
    
    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_create_walls(const bool verbose, bool & termination_request)
{
    if (verbose)
	printf("creation of the walls...\n");
    
    int nsurvived = 0;
    ExpectedMessageSizes new_sizes;
    wall = new ComputeInteractionsWall(cartcomm, particles.xyzuvw.data, particles.size, nsurvived, new_sizes);
    
    //adjust the message sizes if we're pushing the flow in x
    {
	const double xvelavg = getenv("XVELAVG") ? atof(getenv("XVELAVG")) : pushtheflow;
	const double yvelavg = getenv("YVELAVG") ? atof(getenv("YVELAVG")) : 0;
	const double zvelavg = getenv("ZVELAVG") ? atof(getenv("ZVELAVG")) : 0;
	
	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		((code / 3) % 3) - 1,
		((code / 9) % 3) - 1
	    };
	    
	    const double IudotnI = 
		fabs(d[0] * xvelavg) + 
		fabs(d[1] * yvelavg) + 
		fabs(d[2] * zvelavg) ;
		
	    const float factor = 1 + IudotnI * dt * 10 * numberdensity;
	    
	    //printf("RANK %d: direction %d %d %d -> IudotnI is %f and final factor is %f\n",
	    //rank, d[0], d[1], d[2], IudotnI, 1 + IudotnI * dt * numberdensity);
	    
	    new_sizes.msgsizes[code] *= factor;
	}
    }
    
    MPI_CHECK(MPI_Barrier(activecomm));
    redistribute.adjust_message_sizes(new_sizes);
    dpd.adjust_message_sizes(new_sizes);
    MPI_CHECK(MPI_Barrier(activecomm));
    
    if (hdf5part_dumps)
	dump_part.close();
    
    //there is no support for killing zero-workload ranks for rbcs and ctcs just yet
    if (!rbcs && !ctcs)
    {
	const bool local_work = new_sizes.msgsizes[1 + 3 + 9] > 0;
	
	MPI_CHECK(MPI_Comm_split(cartcomm, local_work, rank, &activecomm)) ;
	
	MPI_CHECK(MPI_Comm_rank(activecomm, &rank));
	
	if (!local_work )
	{
	    if (rank == 0)
	    {
		int nkilled;
		MPI_CHECK(MPI_Comm_size(activecomm, &nkilled));
		
		printf("THERE ARE %d RANKS WITH ZERO WORKLOAD THAT WILL MPI-FINALIZE NOW.\n", nkilled);
	    } 
	    
	    termination_request = true;
	    return;
	}
    }
    
    if (hdf5part_dumps)
	dump_part_solvent = new H5PartDump("solvent-particles.h5part", activecomm, cartcomm);
    
    particles.resize(nsurvived);
    particles.clear_velocity();

    CUDA_CHECK(hipPeekAtLastError());    
    
    //remove cells touching the wall
    _remove_bodies_from_wall(rbcscoll);
    _remove_bodies_from_wall(ctcscoll);

    {
	H5PartDump sd("survived-particles.h5part", activecomm, cartcomm);
	Particle * p = new Particle[particles.size];
	
	CUDA_CHECK(hipMemcpy(p, particles.xyzuvw.data, sizeof(Particle) * particles.size, hipMemcpyDeviceToHost));
	
	sd.dump(p, particles.size);
	
	delete [] p;
    }
    
    if (rank == 0)
    {
	if( access( "particles.xyz", F_OK ) != -1 )
	{
	    const int retval = rename ("particles.xyz", "particles-equilibration.xyz");
	    assert(retval != -1);
	}
	
	if( access( "rbcs.xyz", F_OK ) != -1 )  
	{
	    const int retval = rename ("rbcs.xyz", "rbcs-equilibration.xyz");
	    assert(retval != -1);
	}
    }
}

void Simulation::_forces()
{
    //THIS IS WHERE WE WANT TO ACHIEVE 70% OF THE PEAK
    //TODO: i need a coordinating class that performs all the local work while waiting for the communication
    {
	double tstart = MPI_Wtime();
	
	dpd.pack(particles.xyzuvw.data, particles.size, cells.start, cells.count, mainstream);
	dpd.local_interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, cells.start, cells.count, mainstream);
	
	if (wall)
	    wall->interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, 
			       cells.start, cells.count, mainstream);
	
	dpd.consolidate_and_post(particles.xyzuvw.data, particles.size, mainstream);
	dpd.wait_for_messages(mainstream);
	dpd.remote_interactions(particles.xyzuvw.data, particles.size, particles.axayaz.data, mainstream);
	
	timings["evaluate-interactions"] += MPI_Wtime() - tstart; 
	
	CUDA_CHECK(hipPeekAtLastError());	
	
	if (rbcscoll)
	{
	    tstart = MPI_Wtime();
	    rbc_interactions.evaluate(particles.xyzuvw.data, particles.size, particles.axayaz.data,
				      cells.start, cells.count, rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);
	    timings["evaluate-rbc"] += MPI_Wtime() - tstart;
	}
	
	CUDA_CHECK(hipPeekAtLastError());
	
	if (ctcscoll)
	{
	    tstart = MPI_Wtime();
	    ctc_interactions.evaluate(particles.xyzuvw.data, particles.size, particles.axayaz.data,
				      cells.start, cells.count, ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);
	    timings["evaluate-ctc"] += MPI_Wtime() - tstart;
	}
	
	CUDA_CHECK(hipPeekAtLastError());
	
	if (wall)
	{
	    tstart = MPI_Wtime();
	    
	    if (rbcscoll)
		wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);
	    
	    if (ctcscoll)
		wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);
	    
	    timings["body-walls interactions"] += MPI_Wtime() - tstart;
	}
    }
    
    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_data_dump(const int idtimestep)
{
    NVTX_RANGE("data-dump");
    
    double tstart = MPI_Wtime();
    
    int n = particles.size;
    
    if (rbcscoll)
	n += rbcscoll->pcount();
    
    if (ctcscoll)
	n += ctcscoll->pcount();
    
    Particle * p = new Particle[n];
    Acceleration * a = new Acceleration[n];
    
    CUDA_CHECK(hipMemcpy(p, particles.xyzuvw.data, sizeof(Particle) * particles.size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(a, particles.axayaz.data, sizeof(Acceleration) * particles.size, hipMemcpyDeviceToHost));
    
    int start = particles.size;
    
    if (rbcscoll)
    {
	CUDA_CHECK(hipMemcpy(p + start, rbcscoll->xyzuvw.data, sizeof(Particle) * rbcscoll->pcount(), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(a + start, rbcscoll->axayaz.data, sizeof(Acceleration) * rbcscoll->pcount(), hipMemcpyDeviceToHost));
	
	start += rbcscoll->pcount();
    }
    
    if (ctcscoll)
    {
	CUDA_CHECK(hipMemcpy(p + start, ctcscoll->xyzuvw.data, sizeof(Particle) * ctcscoll->pcount(), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(a + start, ctcscoll->axayaz.data, sizeof(Acceleration) * ctcscoll->pcount(), hipMemcpyDeviceToHost));
	
	start += ctcscoll->pcount();
    }
    
    assert(start == n);
    
    diagnostics(activecomm, cartcomm, p, n, dt, idtimestep, a);
	
    if (xyz_dumps)
	xyz_dump(activecomm, cartcomm, "particles.xyz", "all-particles", p, n, idtimestep > 0);
    
    if (hdf5part_dumps)
	if (dump_part_solvent)
	    dump_part_solvent->dump(p, n);
	else
	    dump_part.dump(p, n);
    
    if (hdf5field_dumps)
	dump_field.dump(activecomm, p, particles.size, idtimestep);
    
    if (rbcscoll)
	rbcscoll->dump(activecomm, cartcomm);
    
    if (ctcscoll)
	ctcscoll->dump(activecomm, cartcomm);
    
    delete [] p;
    delete [] a;
    
    timings["data-dump"] += MPI_Wtime() - tstart;
}

void Simulation::_update_and_bounce()
{
    double tstart = MPI_Wtime();
    particles.update_stage2_and_1(driving_acceleration, mainstream);
    
    CUDA_CHECK(hipPeekAtLastError());
    
    if (rbcscoll)
	rbcscoll->update_stage2_and_1(driving_acceleration, mainstream);
    
    CUDA_CHECK(hipPeekAtLastError());
    
    if (ctcscoll)
	ctcscoll->update_stage2_and_1(driving_acceleration, mainstream);
    timings["update"] += MPI_Wtime() - tstart;
    
    if (wall)
    {
	tstart = MPI_Wtime();
	wall->bounce(particles.xyzuvw.data, particles.size, mainstream);
	
	if (rbcscoll)
	    wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);
	
	if (ctcscoll)
	    wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);
	
	timings["bounce-walls"] += MPI_Wtime() - tstart;
    }
    
    CUDA_CHECK(hipPeekAtLastError());
}

Simulation::Simulation(MPI_Comm cartcomm, MPI_Comm activecomm, bool (*check_termination)()) :  
    cartcomm(cartcomm), activecomm(activecomm),
    particles(_ic()), cells(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN), 
    rbcscoll(NULL), ctcscoll(NULL), wall(NULL),
    redistribute(cartcomm),  redistribute_rbcs(cartcomm),  redistribute_ctcs(cartcomm),
    dpd(cartcomm), rbc_interactions(cartcomm), ctc_interactions(cartcomm),
    dump_part("allparticles.h5part", activecomm, cartcomm),  dump_field(cartcomm),  dump_part_solvent(NULL), 
    check_termination(check_termination),
    driving_acceleration(0), host_idle_time(0), nsteps((int)(tend / dt))
{
    //Side not of Yu-Hang:
    //in production runs replace the numbers with 4 unique ones that are same across ranks
    //KISS rng_trunk( 0x26F94D92, 0x383E7D1E, 0x46144B48, 0x6DDD73CB );
    
    MPI_CHECK( MPI_Comm_size(activecomm, &nranks) );
    MPI_CHECK( MPI_Comm_rank(activecomm, &rank) );
    
    CUDA_CHECK(hipStreamCreate(&mainstream));
	
    if (rbcs)
    {
	rbcscoll = new CollectionRBC(cartcomm);
	rbcscoll->setup();
    }
    
    if (ctcs) 
    {
	ctcscoll = new CollectionCTC(cartcomm);
	ctcscoll->setup();
    }
}

void Simulation::run()
{
    if (rank == 0 && !walls)
	printf("the simulation begins now and it consists of %.3e steps\n", (double)nsteps);	  
    
    double time_simulation_start = MPI_Wtime();
    
    _redistribute();
    _forces();
    
    if (!walls && pushtheflow)
	driving_acceleration = hydrostatic_a;		    
    
    particles.update_stage1(driving_acceleration, mainstream);
    
    if (rbcscoll)
	rbcscoll->update_stage1(driving_acceleration, mainstream);
    
    if (ctcscoll)
	ctcscoll->update_stage1(driving_acceleration, mainstream);
    
    int it;
    
    for(it = 0; it < nsteps; ++it)
    {
	const bool verbose = it > 0 && rank == 0;
	
#ifdef _USE_NVTX_
	if (it == 7001)
	{
	    NvtxTracer::currently_profiling = true;
	    CUDA_CHECK(hipProfilerStart());
	}
	else if (it == 7051)
	{
	    CUDA_CHECK(hipProfilerStop());
	    NvtxTracer::currently_profiling = false;
	    CUDA_CHECK(hipDeviceSynchronize());
	    
	    if (rank == 0)
		printf("profiling session ended. terminating the simulation now...\n");

	    break;
	}
#endif
	
	if (it % steps_per_report == 0)
	{
	    CUDA_CHECK(hipStreamSynchronize(mainstream));
	    
	    if (check_termination())
		break;
	    
	    _report(verbose, it);
	}
	
	_redistribute();
	
	if (walls && it >= wall_creation_stepid && wall == NULL)
	{
	    CUDA_CHECK(hipDeviceSynchronize());
	    
	    bool termination_request = false;

	    _create_walls(verbose, termination_request);

	    _redistribute();
	    
	    if (termination_request)
		break;
	    
	    time_simulation_start = MPI_Wtime();
	    
	    if (pushtheflow)
		driving_acceleration = hydrostatic_a;
	    
	    if (rank == 0)
		printf("the simulation begins now and it consists of %.3e steps\n", (double)(nsteps - it));
	}
	
	_forces();
	
	if (it % steps_per_dump == 0)
	{
	    CUDA_CHECK(hipStreamSynchronize(mainstream));
	    
	    _data_dump(it);
	}
	
	_update_and_bounce();
    }
    
    const double time_simulation_stop = MPI_Wtime();
    const double telapsed = time_simulation_stop - time_simulation_start;
    
    if (rank == 0)
	if (it == nsteps)
	    printf("simulation is done after %.3e s (%dm%ds). Ciao.\n", 
		   telapsed, (int)(telapsed / 60), (int)(telapsed) % 60);
	else
	    if (it != wall_creation_stepid)
		printf("external termination request (signal) after %.3e s. Bye.\n", telapsed);
    
    fflush(stdout);
}

Simulation::~Simulation()
{
    CUDA_CHECK(hipStreamDestroy(mainstream));
    
    if (wall)
	delete wall;
    
    if (rbcscoll)
	delete rbcscoll;
    
    if (ctcscoll)
	delete ctcscoll;
    
    if (dump_part_solvent)
	delete dump_part_solvent;
}
