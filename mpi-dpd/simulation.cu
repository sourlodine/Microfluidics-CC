#include "hip/hip_runtime.h"
/*
 *  simulation.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-03-24.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <sys/stat.h>

#include "simulation.h"

__global__ void make_texture( float4 * __restrict xyzouvwo, ushort4 * __restrict xyzo_half, const float * __restrict xyzuvw, const uint n )
{
    extern __shared__ volatile float  smem[];
    const uint warpid = threadIdx.x / 32;
    const uint lane = threadIdx.x % 32;
    //for( uint i = ( blockIdx.x * blockDim.x + threadIdx.x ) & 0xFFFFFFE0U ; i < n ; i += blockDim.x * gridDim.x ) {
    const uint i =  (blockIdx.x * blockDim.x + threadIdx.x ) & 0xFFFFFFE0U;

    const float2 * base = ( float2* )( xyzuvw +  i * 6 );
#pragma unroll 3
    for( uint j = lane; j < 96; j += 32 ) {
	float2 u = base[j];
	// NVCC bug: no operator = between volatile float2 and float2
	asm volatile( "st.volatile.shared.v2.f32 [%0], {%1, %2};" : : "r"( ( warpid * 96 + j )*8 ), "f"( u.x ), "f"( u.y ) : "memory" );
    }
    // SMEM: XYZUVW XYZUVW ...
    uint pid = lane / 2;
    const uint x_or_v = ( lane % 2 ) * 3;
    xyzouvwo[ i * 2 + lane ] = make_float4( smem[ warpid * 192 + pid * 6 + x_or_v + 0 ],
					    smem[ warpid * 192 + pid * 6 + x_or_v + 1 ],
					    smem[ warpid * 192 + pid * 6 + x_or_v + 2 ], 0 );
    pid += 16;
    xyzouvwo[ i * 2 + lane + 32] = make_float4( smem[ warpid * 192 + pid * 6 + x_or_v + 0 ],
						smem[ warpid * 192 + pid * 6 + x_or_v + 1 ],
						smem[ warpid * 192 + pid * 6 + x_or_v + 2 ], 0 );

    xyzo_half[i + lane] = make_ushort4( __float2half_rn( smem[ warpid * 192 + lane * 6 + 0 ] ),
					__float2half_rn( smem[ warpid * 192 + lane * 6 + 1 ] ),
					__float2half_rn( smem[ warpid * 192 + lane * 6 + 2 ] ), 0 );
// }
}

void Simulation::_update_helper_arrays()
{
    CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>( make_texture), hipFuncCachePreferShared ) );

    const int np = particles->size;

    xyzouvwo.resize(2 * np);
    xyzo_half.resize(np);

    make_texture <<< (np + 1023) / 1024, 1024, 1024 * 6 * sizeof( float )>>>(xyzouvwo.data, xyzo_half.data, (float *)particles->xyzuvw.data, np );

    CUDA_CHECK(hipPeekAtLastError());
}

std::vector<Particle> Simulation::_ic()
{
    srand48(rank);

    std::vector<Particle> ic(XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN * numberdensity);

    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

    for(int iz = 0; iz < L[2]; iz++)
	for(int iy = 0; iy < L[1]; iy++)
	    for(int ix = 0; ix < L[0]; ix++)
		for(int l = 0; l < numberdensity; ++l)
		{
		    const int p = l + numberdensity * (ix + L[0] * (iy + L[1] * iz));

		    ic[p].x[0] = -L[0]/2 + ix + 0.99 * drand48();
		    ic[p].x[1] = -L[1]/2 + iy + 0.99 * drand48();
		    ic[p].x[2] = -L[2]/2 + iz + 0.99 * drand48();
		    ic[p].u[0] = 0;
		    ic[p].u[1] = 0;
		    ic[p].u[2] = 0;
		}

    /* use this to check robustness
       for(int i = 0; i < ic.size(); ++i)
       for(int c = 0; c < 3; ++c)
       {
       ic[i].x[c] = -L[c] * 0.5 + drand48() * L[c];
       ic[i].u[c] = 0;
       }
    */

    return ic;
}

void Simulation::_redistribute()
{
    double tstart = MPI_Wtime();

    redistribute.pack(particles->xyzuvw.data, particles->size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    redistribute.send();

    if (rbcscoll)
	redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);

    redistribute.bulk(particles->size, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);

    int nrbcs;
    if (rbcscoll)
	nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
	nctcs = redistribute_ctcs.post();

    if (rbcscoll)
	rbcscoll->resize(nrbcs);

    if (ctcscoll)
	ctcscoll->resize(nctcs);

    newparticles->resize(newnp);
    xyzouvwo.resize(newnp * 2);
    xyzo_half.resize(newnp);

    redistribute.recv_unpack(newparticles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, newnp, cells.start, cells.count, mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    swap(particles, newparticles);

    if (rbcscoll)
	redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    localcomm.barrier();

    timings["redistribute"] += MPI_Wtime() - tstart;
}

void Simulation::_report(const bool verbose, const int idtimestep)
{
    report_host_memory_usage(activecomm, stdout);

    {
	static double t0 = MPI_Wtime(), t1;

	t1 = MPI_Wtime();

	float host_busy_time = (MPI_Wtime() - t0) - host_idle_time;

	host_busy_time *= 1e3 / steps_per_report;

	float sumval, maxval, minval;
	MPI_CHECK(MPI_Reduce(&host_busy_time, &sumval, 1, MPI_FLOAT, MPI_SUM, 0, activecomm));
	MPI_CHECK(MPI_Reduce(&host_busy_time, &maxval, 1, MPI_FLOAT, MPI_MAX, 0, activecomm));
	MPI_CHECK(MPI_Reduce(&host_busy_time, &minval, 1, MPI_FLOAT, MPI_MIN, 0, activecomm));

	int commsize;
	MPI_CHECK(MPI_Comm_size(activecomm, &commsize));

	const double imbalance = 100 * (maxval / sumval * commsize - 1);

	if (verbose && imbalance >= 0)
	    printf("\x1b[93moverall imbalance: %.f%%, host workload min/avg/max: %.2f/%.2f/%.2f ms\x1b[0m\n",
		   imbalance , minval, sumval / commsize, maxval);

	host_idle_time = 0;
	t0 = t1;
    }

    {
	static double t0 = MPI_Wtime(), t1;

	t1 = MPI_Wtime();

	if (verbose)
	{
	    printf("\x1b[92mbeginning of time step %d (%.3f ms)\x1b[0m\n", idtimestep, (t1 - t0) * 1e3 / steps_per_report);
	    printf("in more details, per time step:\n");
	    double tt = 0;
	    for(std::map<string, double>::iterator it = timings.begin(); it != timings.end(); ++it)
	    {
		printf("%s: %.3f ms\n", it->first.c_str(), it->second * 1e3 / steps_per_report);
		tt += it->second;
		it->second = 0;
	    }
	    printf("discrepancy: %.3f ms\n", ((t1 - t0) - tt) * 1e3 / steps_per_report);
	}

	t0 = t1;
    }
}

void Simulation::_remove_bodies_from_wall(CollectionRBC * coll)
{
    if(!coll || !coll->count())
	return;

    SimpleDeviceBuffer<int> marks(coll->pcount());

    SolidWallsKernel::fill_keys<<< (coll->pcount() + 127) / 128, 128 >>>(coll->data(), coll->pcount(), marks.data);

    vector<int> tmp(marks.size);
    CUDA_CHECK(hipMemcpy(tmp.data(), marks.data, sizeof(int) * marks.size, hipMemcpyDeviceToHost));

    const int nbodies = coll->count();
    const int nvertices = coll->get_nvertices();

    std::vector<int> tokill;
    for(int i = 0; i < nbodies; ++i)
    {
	bool valid = true;

	for(int j = 0; j < nvertices && valid; ++j)
	    valid &= 0 == tmp[j + nvertices * i];

	if (!valid)
	    tokill.push_back(i);
    }

    coll->remove(&tokill.front(), tokill.size());
    coll->clear_velocity();

    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_create_walls(const bool verbose, bool & termination_request)
{
    if (verbose)
	printf("creation of the walls...\n");

    int nsurvived = 0;
    ExpectedMessageSizes new_sizes;
    wall = new ComputeInteractionsWall(cartcomm, particles->xyzuvw.data, particles->size, nsurvived, new_sizes, verbose);

    //adjust the message sizes if we're pushing the flow in x
    {
	const double xvelavg = getenv("XVELAVG") ? atof(getenv("XVELAVG")) : pushtheflow;
	const double yvelavg = getenv("YVELAVG") ? atof(getenv("YVELAVG")) : 0;
	const double zvelavg = getenv("ZVELAVG") ? atof(getenv("ZVELAVG")) : 0;

	for(int code = 0; code < 27; ++code)
	{
	    const int d[3] = {
		(code % 3) - 1,
		((code / 3) % 3) - 1,
		((code / 9) % 3) - 1
	    };

	    const double IudotnI =
		fabs(d[0] * xvelavg) +
		fabs(d[1] * yvelavg) +
		fabs(d[2] * zvelavg) ;

	    const float factor = 1 + IudotnI * dt * 10 * numberdensity;

	    //printf("RANK %d: direction %d %d %d -> IudotnI is %f and final factor is %f\n",
	    //rank, d[0], d[1], d[2], IudotnI, 1 + IudotnI * dt * numberdensity);

	    new_sizes.msgsizes[code] *= factor;
	}
    }

    MPI_CHECK(MPI_Barrier(activecomm));
    redistribute.adjust_message_sizes(new_sizes);
    dpd.adjust_message_sizes(new_sizes);
    MPI_CHECK(MPI_Barrier(activecomm));

    //there is no support for killing zero-workload ranks for rbcs and ctcs just yet
    /* this is unnecessarily complex for now
       if (!rbcs && !ctcs)
       {
       const bool local_work = new_sizes.msgsizes[1 + 3 + 9] > 0;

       MPI_CHECK(MPI_Comm_split(cartcomm, local_work, rank, &activecomm)) ;

       MPI_CHECK(MPI_Comm_rank(activecomm, &rank));

       if (!local_work )
       {
       if (rank == 0)
       {
       int nkilled;
       MPI_CHECK(MPI_Comm_size(activecomm, &nkilled));

       printf("THERE ARE %d RANKS WITH ZERO WORKLOAD THAT WILL MPI-FINALIZE NOW.\n", nkilled);
       }

       termination_request = true;
       return;
       }
       }
    */

    particles->resize(nsurvived);
    particles->clear_velocity();
    cells.build(particles->xyzuvw.data, particles->size, 0, NULL, NULL);

    _update_helper_arrays();

    CUDA_CHECK(hipPeekAtLastError());

    //remove cells touching the wall
    _remove_bodies_from_wall(rbcscoll);
    _remove_bodies_from_wall(ctcscoll);

    {
	H5PartDump sd("survived-particles->h5part", activecomm, cartcomm);
	Particle * p = new Particle[particles->size];

	CUDA_CHECK(hipMemcpy(p, particles->xyzuvw.data, sizeof(Particle) * particles->size, hipMemcpyDeviceToHost));

	sd.dump(p, particles->size);

	delete [] p;
    }
}

void Simulation::_forces()
{
    double tstart = MPI_Wtime();

    particles->clear_acc(mainstream);

    if (rbcscoll)
	rbcscoll->clear_acc(mainstream);

    if (ctcscoll)
    	ctcscoll->clear_acc(mainstream);

    if (rbcscoll)
	rbc_interactions.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	ctc_interactions.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    if (rbcscoll)
	rbc_interactions.count(rbcscoll->count());

    if (ctcscoll)
	ctc_interactions.count(ctcscoll->count());

    dpd.pack(particles->xyzuvw.data, particles->size, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions. pack_p(rbcscoll->data(), mainstream);

    if (ctcscoll)
	ctc_interactions.pack_p(ctcscoll->data(), mainstream);

    dpd.local_interactions(particles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, particles->size, particles->axayaz.data, cells.start, cells.count, mainstream);

    dpd.consolidate_and_post(particles->xyzuvw.data, particles->size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions.exchange_count();

    if (ctcscoll)
	ctc_interactions.exchange_count();

    if (rbcscoll)
	rbc_interactions.post_p();

    if (ctcscoll)
	ctc_interactions.post_p();

    if (rbcscoll)
	rbc_interactions.fsi_bulk(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_bulk(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll && wall)
	wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);

    if (ctcscoll && wall)
	wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);

    if (rbcscoll)
	rbc_interactions.fsi_halo(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_halo(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbc_interactions.internal_forces(rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.internal_forces(ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (wall)
	wall->interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data,
			   cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions.post_a();

    if (ctcscoll)
	ctc_interactions.post_a();

    dpd.wait_for_messages(mainstream);
    dpd.remote_interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data, mainstream);

    if (rbcscoll)
	rbc_interactions.merge_a(rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.merge_a(ctcscoll->acc(), mainstream);

    timings["interactions"] += MPI_Wtime() - tstart;

    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_datadump(const int idtimestep)
{
    double tstart = MPI_Wtime();

    pthread_mutex_lock(&mutex_datadump);

    while (datadump_pending)
	pthread_cond_wait(&done_datadump, &mutex_datadump);

    int n = particles->size;

    if (rbcscoll)
	n += rbcscoll->pcount();

    if (ctcscoll)
	n += ctcscoll->pcount();

    particles_datadump.resize(n);
    accelerations_datadump.resize(n);

    CUDA_CHECK(hipMemcpyAsync(particles_datadump.data, particles->xyzuvw.data, sizeof(Particle) * particles->size, hipMemcpyDeviceToHost,0));
    CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data, particles->axayaz.data, sizeof(Acceleration) * particles->size, hipMemcpyDeviceToHost,0));

    int start = particles->size;

    if (rbcscoll)
    {
	CUDA_CHECK(hipMemcpyAsync(particles_datadump.data + start, rbcscoll->xyzuvw.data, sizeof(Particle) * rbcscoll->pcount(), hipMemcpyDeviceToHost, 0));
	CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data + start, rbcscoll->axayaz.data, sizeof(Acceleration) * rbcscoll->pcount(), hipMemcpyDeviceToHost, 0));

	start += rbcscoll->pcount();
    }

    if (ctcscoll)
    {
	CUDA_CHECK(hipMemcpyAsync(particles_datadump.data + start, ctcscoll->xyzuvw.data, sizeof(Particle) * ctcscoll->pcount(), hipMemcpyDeviceToHost, 0));
	CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data + start, ctcscoll->axayaz.data, sizeof(Acceleration) * ctcscoll->pcount(), hipMemcpyDeviceToHost, 0));

	start += ctcscoll->pcount();
    }

    assert(start == n);

    CUDA_CHECK(hipEventRecord(evdownloaded, 0));

    datadump_idtimestep = idtimestep;
    datadump_nsolvent = particles->size;
    datadump_nrbcs = rbcscoll ? rbcscoll->pcount() : 0;
    datadump_nctcs = ctcscoll ? ctcscoll->pcount() : 0;
    datadump_pending = true;

    pthread_cond_signal(&request_datadump);

    pthread_mutex_unlock(&mutex_datadump);

    timings["data-dump"] += MPI_Wtime() - tstart;
}

void Simulation::_datadump_async()
{
#ifdef _USE_NVTX_
    nvtxNameOsThread(pthread_self(), "DATADUMP_THREAD");
#endif

    int iddatadump = 0, rank;
    int curr_idtimestep = -1;
    bool wallcreated = false;

    MPI_Comm myactivecomm, mycartcomm;

    MPI_CHECK(MPI_Comm_dup(activecomm, &myactivecomm) );
    MPI_CHECK(MPI_Comm_dup(cartcomm, &mycartcomm) );

    H5PartDump dump_part("allparticles->h5part", activecomm, cartcomm), *dump_part_solvent = NULL;
    H5FieldDump dump_field(cartcomm);

    MPI_CHECK(MPI_Comm_rank(myactivecomm, &rank));

    if (rank == 0)
	mkdir("xyz", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);

    MPI_CHECK(MPI_Barrier(myactivecomm));

    while (true)
    {
	pthread_mutex_lock(&mutex_datadump);
	async_thread_initialized = 1;

	while (!datadump_pending)
	    pthread_cond_wait(&request_datadump, &mutex_datadump);

	pthread_mutex_unlock(&mutex_datadump);

	if (curr_idtimestep == datadump_idtimestep)
	    if (simulation_is_done)
		break;

	CUDA_CHECK(hipEventSynchronize(evdownloaded));

	const int n = particles_datadump.size;
	Particle * p = particles_datadump.data;
	Acceleration * a = accelerations_datadump.data;

	{
	    NVTX_RANGE("diagnostics", NVTX_C1);
	    diagnostics(myactivecomm, mycartcomm, p, n, dt, datadump_idtimestep, a);
	}

	if (xyz_dumps)
	{
	    NVTX_RANGE("xyz dump", NVTX_C2);

	    if (walls && datadump_idtimestep >= wall_creation_stepid && !wallcreated)
	    {
		if (rank == 0)
		{
		    if( access("xyz/particles-equilibration.xyz", F_OK ) == -1 )
			rename ("xyz/particles->xyz", "xyz/particles-equilibration.xyz");

		    if( access( "xyz/rbcs-equilibration.xyz", F_OK ) == -1 )
			rename ("xyz/rbcs.xyz", "xyz/rbcs-equilibration.xyz");

		    if( access( "xyz/ctcs-equilibration.xyz", F_OK ) == -1 )
			rename ("xyz/ctcs.xyz", "xyz/ctcs-equilibration.xyz");
		}

		MPI_CHECK(MPI_Barrier(myactivecomm));

		wallcreated = true;
	    }

	    xyz_dump(myactivecomm, mycartcomm, "xyz/particles->xyz", "all-particles", p, n, datadump_idtimestep > 0);
	}

	if (hdf5part_dumps)
	{
	    NVTX_RANGE("h5part dump", NVTX_C3);

	    if (!dump_part_solvent && walls && datadump_idtimestep >= wall_creation_stepid)
	    {
		dump_part.close();

		dump_part_solvent = new H5PartDump("solvent-particles->h5part", activecomm, cartcomm);
	    }

	    if (dump_part_solvent)
		dump_part_solvent->dump(p, n);
	    else
		dump_part.dump(p, n);
	}

	if (hdf5field_dumps)
	{
	    NVTX_RANGE("hdf5 field dump", NVTX_C4);

	    dump_field.dump(activecomm, p, particles->size, datadump_idtimestep);
	}

	{
	    NVTX_RANGE("ply dump", NVTX_C5);

	    if (rbcscoll)
		CollectionRBC::dump(myactivecomm, mycartcomm, p + datadump_nsolvent, a + datadump_nsolvent, datadump_nrbcs, iddatadump);

	    if (ctcscoll)
		CollectionCTC::dump(myactivecomm, mycartcomm, p + datadump_nsolvent + datadump_nrbcs,
				    a + datadump_nsolvent + datadump_nrbcs, datadump_nctcs, iddatadump);
	}

	curr_idtimestep = datadump_idtimestep;

	pthread_mutex_lock(&mutex_datadump);
	
	if (simulation_is_done)
	{
	    pthread_mutex_unlock(&mutex_datadump);
	    break;
	}

	datadump_pending = false;

	pthread_cond_signal(&done_datadump);

	pthread_mutex_unlock(&mutex_datadump);

	++iddatadump;
    }

    if (dump_part_solvent)
	delete dump_part_solvent;

    CUDA_CHECK(hipEventDestroy(evdownloaded));
}

void Simulation::_update_and_bounce()
{
    double tstart = MPI_Wtime();
    particles->update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (ctcscoll)
	ctcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    timings["update"] += MPI_Wtime() - tstart;

    if (wall)
    {
	tstart = MPI_Wtime();
	wall->bounce(particles->xyzuvw.data, particles->size, mainstream);

	if (rbcscoll)
	    wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);

	if (ctcscoll)
	    wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);

	timings["bounce-walls"] += MPI_Wtime() - tstart;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

Simulation::Simulation(MPI_Comm cartcomm, MPI_Comm activecomm, bool (*check_termination)()) :
    cartcomm(cartcomm), activecomm(activecomm),
    /*particles(_ic()),*/ cells(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN),
    rbcscoll(NULL), ctcscoll(NULL), wall(NULL),
    redistribute(cartcomm),  redistribute_rbcs(cartcomm),  redistribute_ctcs(cartcomm),
    dpd(cartcomm), rbc_interactions(cartcomm), ctc_interactions(cartcomm),
    check_termination(check_termination),
    driving_acceleration(0), host_idle_time(0), nsteps((int)(tend / dt)),
    datadump_pending(false), simulation_is_done(false)
{
    localcomm.initialize(activecomm);

    MPI_CHECK( MPI_Comm_size(activecomm, &nranks) );
    MPI_CHECK( MPI_Comm_rank(activecomm, &rank) );

    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    {
	particles = &particles_pingpong[0];
	newparticles = &particles_pingpong[1];

	vector<Particle> ic = _ic();

	for(int c = 0; c < 2; ++c)
	{
	    particles_pingpong[c].resize(ic.size());

	    particles_pingpong[c].origin = make_float3((0.5 + coords[0]) * XSIZE_SUBDOMAIN,
						       (0.5 + coords[1]) * YSIZE_SUBDOMAIN,
						       (0.5 + coords[2]) * ZSIZE_SUBDOMAIN);

	    particles_pingpong[c].globalextent = make_float3(dims[0] * XSIZE_SUBDOMAIN,
							     dims[1] * YSIZE_SUBDOMAIN,
							     dims[2] * ZSIZE_SUBDOMAIN);
	}

	CUDA_CHECK(hipMemcpy(particles->xyzuvw.data, &ic.front(), sizeof(Particle) * ic.size(), hipMemcpyHostToDevice));

	cells.build(particles->xyzuvw.data, particles->size, 0, NULL, NULL);

	_update_helper_arrays();
    }

    CUDA_CHECK(hipStreamCreate(&mainstream));

    if (rbcs)
    {
	rbcscoll = new CollectionRBC(cartcomm);
	rbcscoll->setup("rbcs-ic.txt");
    }

    if (ctcs)
    {
	ctcscoll = new CollectionCTC(cartcomm);
	ctcscoll->setup("ctcs-ic.txt");
    }

#ifndef _NO_DUMPS_
    //setting up the asynchronous data dumps
    {
	CUDA_CHECK(hipEventCreate(&evdownloaded, hipEventDisableTiming | hipEventBlockingSync));

	particles_datadump.resize(particles->size * 1.5);
	accelerations_datadump.resize(particles->size * 1.5);

	int rc = pthread_mutex_init(&mutex_datadump, NULL);
	rc |= pthread_cond_init(&done_datadump, NULL);
	rc |= pthread_cond_init(&request_datadump, NULL);
	async_thread_initialized = 0;
	rc |= pthread_create(&thread_datadump, NULL, datadump_trampoline, this);

	while (1) 
	{
	    pthread_mutex_lock(&mutex_datadump);
	    int done = async_thread_initialized;
	    pthread_mutex_unlock(&mutex_datadump);
	
	    if (done) 
		break;
	}

	if (rc)
	{
	    printf("ERROR; return code from pthread_create() is %d\n", rc);
	    exit(-1);
	}
    }
#endif
}

void Simulation::_lockstep()
{
    double tstart = MPI_Wtime();

    particles->clear_acc(mainstream);

    if (rbcscoll)
	rbcscoll->clear_acc(mainstream);

    if (ctcscoll)
    	ctcscoll->clear_acc(mainstream);

    if (rbcscoll)
	rbc_interactions.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	ctc_interactions.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    dpd.pack(particles->xyzuvw.data, particles->size, cells.start, cells.count, mainstream);

    if (rbcscoll)
	rbc_interactions.count(rbcscoll->count());

    if (ctcscoll)
	ctc_interactions.count(ctcscoll->count());

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	rbc_interactions.pack_p(rbcscoll->data(), mainstream);

    if (ctcscoll)
	ctc_interactions.pack_p(ctcscoll->data(), mainstream);

    dpd.local_interactions(particles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, particles->size, particles->axayaz.data, cells.start, cells.count, mainstream);

    dpd.consolidate_and_post(particles->xyzuvw.data, particles->size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    localcomm.barrier(); // peh: 1

    if (rbcscoll)
	rbc_interactions.exchange_count();

    if (ctcscoll)
	ctc_interactions.exchange_count();

    if (rbcscoll)
	rbc_interactions.post_p();

    if (ctcscoll)
	ctc_interactions.post_p();

    if (rbcscoll)
	rbc_interactions.fsi_bulk(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_bulk(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbc_interactions.fsi_halo(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.fsi_halo(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count,
				  ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbc_interactions.post_a();

    if (ctcscoll)
	ctc_interactions.post_a();

    if (wall)
	wall->interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data,
			   cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    dpd.wait_for_messages(mainstream);

    dpd.remote_interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data, mainstream);

    particles->update_stage2_and_1(driving_acceleration, mainstream);

    if (wall)
	wall->bounce(particles->xyzuvw.data, particles->size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    redistribute.pack(particles->xyzuvw.data, particles->size, mainstream);

    redistribute.send();

    redistribute.bulk(particles->size, cells.start, cells.count, mainstream);

    if (rbcscoll)
	rbc_interactions.internal_forces(rbcscoll->data(), rbcscoll->count(), rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.internal_forces(ctcscoll->data(), ctcscoll->count(), ctcscoll->acc(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll && wall)
	wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);

    if (ctcscoll && wall)
	wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);

    if (rbcscoll)
	rbc_interactions.merge_a(rbcscoll->acc(), mainstream);

    if (ctcscoll)
	ctc_interactions.merge_a(ctcscoll->acc(), mainstream);

    if (rbcscoll)
	rbcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (ctcscoll)
	ctcscoll->update_stage2_and_1(driving_acceleration, mainstream);

    if (wall && rbcscoll)
	wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);

    if (wall && ctcscoll)
	wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
	redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    if (rbcscoll)
	redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);

    newparticles->resize(newnp);
    xyzouvwo.resize(newnp * 2);
    xyzo_half.resize(newnp);

    redistribute.recv_unpack(newparticles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, newnp, cells.start, cells.count, mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    swap(particles, newparticles);

    localcomm.barrier();	// peh: +2

    int nrbcs;
    if (rbcscoll)
	nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
	nctcs = redistribute_ctcs.post();

    if (rbcscoll)
	rbcscoll->resize(nrbcs);

    if (ctcscoll)
	ctcscoll->resize(nctcs);

    CUDA_CHECK(hipPeekAtLastError());


    if (rbcscoll)
	redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
	redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

//  localcomm.barrier();  // peh: +3

    timings["lockstep"] += MPI_Wtime() - tstart;
}


void Simulation::run()
{
    if (rank == 0 && !walls)
	printf("the simulation begins now and it consists of %.3e steps\n", (double)nsteps);

    double time_simulation_start = MPI_Wtime();

    _redistribute();
    _forces();

    if (!walls && pushtheflow)
	driving_acceleration = hydrostatic_a;

    particles->update_stage1(driving_acceleration, mainstream);

    if (rbcscoll)
	rbcscoll->update_stage1(driving_acceleration, mainstream);

    if (ctcscoll)
	ctcscoll->update_stage1(driving_acceleration, mainstream);

    int it;
    

    for(it = 0; it < nsteps; ++it)
    {
	const bool verbose = it > 0 && rank == 0;

#ifdef _USE_NVTX_
	if (it == nvtxstart)
	{
	    NvtxTracer::currently_profiling = true;
	    CUDA_CHECK(hipProfilerStart());
	}
	else if (it == nvtxstop)
	{
	    CUDA_CHECK(hipProfilerStop());
	    NvtxTracer::currently_profiling = false;
	    CUDA_CHECK(hipDeviceSynchronize());

	    if (rank == 0)
		printf("profiling session ended. terminating the simulation now...\n");

	    break;
	}
#endif

	if (it % steps_per_report == 0)
	{
	    CUDA_CHECK(hipStreamSynchronize(mainstream));

	    if (simulation_is_done = check_termination())
		break;

	    _report(verbose, it);
	}

	_redistribute();

#if 1
    lockstep_check:

	const bool lockstep_OK =
	    !(walls && it >= wall_creation_stepid && wall == NULL) &&
	    !(it % steps_per_dump == 0) &&
	    !(it + 1 == nvtxstart) &&
	    !(it + 1 == nvtxstop) &&
	    !((it + 1) % steps_per_report == 0) &&
	    !(it + 1 == nsteps);

	if (lockstep_OK)
	{
	    _lockstep();

	    ++it;

	    goto lockstep_check;
	}
#endif

	if (walls && it >= wall_creation_stepid && wall == NULL)
	{
	    CUDA_CHECK(hipDeviceSynchronize());

	    bool termination_request = false;

	    _create_walls(verbose, termination_request);

	    _redistribute();

	    if (termination_request)
		break;

	    time_simulation_start = MPI_Wtime();

	    if (pushtheflow)
		driving_acceleration = hydrostatic_a;

	    if (rank == 0)
		printf("the simulation begins now and it consists of %.3e steps\n", (double)(nsteps - it));
	}

	_forces();

#ifndef _NO_DUMPS_
	if (it % steps_per_dump == 0)
	    _datadump(it);
#endif
	_update_and_bounce();
    }

    const double time_simulation_stop = MPI_Wtime();
    const double telapsed = time_simulation_stop - time_simulation_start;

    simulation_is_done = true;

    if (rank == 0)
	if (it == nsteps)
	    printf("simulation is done after %.2lf s (%dm%ds). Ciao.\n",
		   telapsed, (int)(telapsed / 60), (int)(telapsed) % 60);
	else
	    if (it != wall_creation_stepid)
		printf("external termination request (signal) after %.3e s. Bye.\n", telapsed);

    fflush(stdout);
}

Simulation::~Simulation()
{
#ifndef _NO_DUMPS_
    pthread_mutex_lock(&mutex_datadump);

    datadump_pending = true;
    pthread_cond_signal(&request_datadump);

    pthread_mutex_unlock(&mutex_datadump);

    pthread_join(thread_datadump, NULL);
#endif
    CUDA_CHECK(hipStreamDestroy(mainstream));

    if (wall)
	delete wall;

    if (rbcscoll)
	delete rbcscoll;

    if (ctcscoll)
	delete ctcscoll;
}
