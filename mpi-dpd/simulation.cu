#include "hip/hip_runtime.h"
/*
 *  simulation.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-03-24.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <sys/stat.h>

#include "simulation.h"

__global__ void make_texture( float4 * __restrict xyzouvwo, ushort4 * __restrict xyzo_half, const float * __restrict xyzuvw, const uint n )
{
    extern __shared__ volatile float  smem[];
    const uint warpid = threadIdx.x / 32;
    const uint lane = threadIdx.x % 32;

    const uint i =  (blockIdx.x * blockDim.x + threadIdx.x ) & 0xFFFFFFE0U;

    const float2 * base = ( float2* )( xyzuvw +  i * 6 );
#pragma unroll 3
    for( uint j = lane; j < 96; j += 32 ) {
        float2 u = base[j];
        // NVCC bug: no operator = between volatile float2 and float2
        asm volatile( "st.volatile.shared.v2.f32 [%0], {%1, %2};" : : "r"( ( warpid * 96 + j )*8 ), "f"( u.x ), "f"( u.y ) : "memory" );
    }
    // SMEM: XYZUVW XYZUVW ...
    uint pid = lane / 2;
    const uint x_or_v = ( lane % 2 ) * 3;
    xyzouvwo[ i * 2 + lane ] = make_float4( smem[ warpid * 192 + pid * 6 + x_or_v + 0 ],
            smem[ warpid * 192 + pid * 6 + x_or_v + 1 ],
            smem[ warpid * 192 + pid * 6 + x_or_v + 2 ], 0 );
    pid += 16;
    xyzouvwo[ i * 2 + lane + 32] = make_float4( smem[ warpid * 192 + pid * 6 + x_or_v + 0 ],
            smem[ warpid * 192 + pid * 6 + x_or_v + 1 ],
            smem[ warpid * 192 + pid * 6 + x_or_v + 2 ], 0 );

    xyzo_half[i + lane] = make_ushort4( __float2half_rn( smem[ warpid * 192 + lane * 6 + 0 ] ),
            __float2half_rn( smem[ warpid * 192 + lane * 6 + 1 ] ),
            __float2half_rn( smem[ warpid * 192 + lane * 6 + 2 ] ), 0 );
    // }
}

void Simulation::_update_helper_arrays()
{
    CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>( make_texture), hipFuncCachePreferShared ) );

    const int np = particles->size;

    xyzouvwo.resize(2 * np);
    xyzo_half.resize(np);

    if (np)
    make_texture <<< (np + 1023) / 1024, 1024, 1024 * 6 * sizeof( float )>>>(xyzouvwo.data, xyzo_half.data, (float *)particles->xyzuvw.data, np );

    CUDA_CHECK(hipPeekAtLastError());
}

std::vector<Particle> Simulation::_ic()
{
    srand48(rank);

    std::vector<Particle> ic(XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN * numberdensity);

    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

    for(int iz = 0; iz < L[2]; iz++)
        for(int iy = 0; iy < L[1]; iy++)
            for(int ix = 0; ix < L[0]; ix++)
                for(int l = 0; l < numberdensity; ++l)
                {
                    const int p = l + numberdensity * (ix + L[0] * (iy + L[1] * iz));

                    ic[p].x[0] = -L[0]/2 + ix + 0.99 * drand48();
                    ic[p].x[1] = -L[1]/2 + iy + 0.99 * drand48();
                    ic[p].x[2] = -L[2]/2 + iz + 0.99 * drand48();
                    ic[p].u[0] = 0;
                    ic[p].u[1] = 0;
                    ic[p].u[2] = 0;
                }

    /* use this to check robustness
       for(int i = 0; i < ic.size(); ++i)
       for(int c = 0; c < 3; ++c)
       {
       ic[i].x[c] = -L[c] * 0.5 + drand48() * L[c];
       ic[i].u[c] = 0;
       }
     */

    return ic;
}

void Simulation::_redistribute()
{
    double tstart = MPI_Wtime();

    redistribute.pack(particles->xyzuvw.data, particles->size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
        redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
        redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    redistribute.send();

    if (rbcscoll)
        redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
        redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);

    redistribute.bulk(particles->size, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);

    int nrbcs;
    if (rbcscoll)
        nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
        nctcs = redistribute_ctcs.post();

    if (rbcscoll)
        rbcscoll->resize(nrbcs);

    if (ctcscoll)
        ctcscoll->resize(nctcs);

    newparticles->resize(newnp);
    xyzouvwo.resize(newnp * 2);
    xyzo_half.resize(newnp);

    redistribute.recv_unpack(newparticles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, newnp, cells.start, cells.count, mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    swap(particles, newparticles);

    if (rbcscoll)
        redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
        redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    //localcomm.barrier();

    timings["redistribute"] += MPI_Wtime() - tstart;
}

void Simulation::_report(const bool verbose, const int idtimestep)
{
    report_host_memory_usage(activecomm, stdout);

    {
        static double t0 = MPI_Wtime(), t1;

        t1 = MPI_Wtime();

        float host_busy_time = (MPI_Wtime() - t0) - host_idle_time;

        host_busy_time *= 1e3 / steps_per_report;

        float sumval, maxval, minval;
        MPI_CHECK(MPI_Reduce(&host_busy_time, &sumval, 1, MPI_FLOAT, MPI_SUM, 0, activecomm));
        MPI_CHECK(MPI_Reduce(&host_busy_time, &maxval, 1, MPI_FLOAT, MPI_MAX, 0, activecomm));
        MPI_CHECK(MPI_Reduce(&host_busy_time, &minval, 1, MPI_FLOAT, MPI_MIN, 0, activecomm));

        int commsize;
        MPI_CHECK(MPI_Comm_size(activecomm, &commsize));

        const double imbalance = 100 * (maxval / sumval * commsize - 1);

        if (verbose && imbalance >= 0)
            printf("\x1b[93moverall imbalance: %.f%%, host workload min/avg/max: %.2f/%.2f/%.2f ms\x1b[0m\n",
                    imbalance , minval, sumval / commsize, maxval);

        localcomm.print_particles(particles->size);

        host_idle_time = 0;
        t0 = t1;
    }

    {
        static double t0 = MPI_Wtime(), t1;

        t1 = MPI_Wtime();

        if (verbose)
        {
            printf("\x1b[92mbeginning of time step %d (%.3f ms)\x1b[0m\n", idtimestep, (t1 - t0) * 1e3 / steps_per_report);
            printf("in more details, per time step:\n");
            double tt = 0;
            for(std::map<string, double>::iterator it = timings.begin(); it != timings.end(); ++it)
            {
                printf("%s: %.3f ms\n", it->first.c_str(), it->second * 1e3 / steps_per_report);
                tt += it->second;
                it->second = 0;
            }
            printf("discrepancy: %.3f ms\n", ((t1 - t0) - tt) * 1e3 / steps_per_report);
        }

        t0 = t1;
    }
}

void Simulation::_remove_bodies_from_wall(CollectionRBC * coll)
{
    if (!coll || !coll->count())
        return;

    SimpleDeviceBuffer<int> marks(coll->pcount());

    SolidWallsKernel::fill_keys<<< (coll->pcount() + 127) / 128, 128 >>>(coll->data(), coll->pcount(), marks.data);

    vector<int> tmp(marks.size);
    CUDA_CHECK(hipMemcpy(tmp.data(), marks.data, sizeof(int) * marks.size, hipMemcpyDeviceToHost));

    const int nbodies = coll->count();
    const int nvertices = coll->get_nvertices();

    std::vector<int> tokill;
    for(int i = 0; i < nbodies; ++i)
    {
        bool valid = true;

        for(int j = 0; j < nvertices && valid; ++j)
            valid &= 0 == tmp[j + nvertices * i];

        if (!valid)
            tokill.push_back(i);
    }

    coll->remove(&tokill.front(), tokill.size());
    coll->clear_velocity();

    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_create_walls(const bool verbose, bool & termination_request)
{
    if (verbose)
        printf("creation of the walls...\n");

    int nsurvived = 0;
    ExpectedMessageSizes new_sizes;
    wall = new ComputeWall(cartcomm, particles->xyzuvw.data, particles->size, nsurvived, new_sizes, verbose);

    //adjust the message sizes if we're pushing the flow in x
    {
        const double xvelavg = getenv("XVELAVG") ? atof(getenv("XVELAVG")) : pushtheflow;
        const double yvelavg = getenv("YVELAVG") ? atof(getenv("YVELAVG")) : 0;
        const double zvelavg = getenv("ZVELAVG") ? atof(getenv("ZVELAVG")) : 0;

        for(int code = 0; code < 27; ++code)
        {
            const int d[3] = {
                    (code % 3) - 1,
                    ((code / 3) % 3) - 1,
                    ((code / 9) % 3) - 1
            };

            const double IudotnI =
                    fabs(d[0] * xvelavg) +
                    fabs(d[1] * yvelavg) +
                    fabs(d[2] * zvelavg) ;

            const float factor = 1 + IudotnI * dt * 10 * numberdensity;

            //printf("RANK %d: direction %d %d %d -> IudotnI is %f and final factor is %f\n",
            //rank, d[0], d[1], d[2], IudotnI, 1 + IudotnI * dt * numberdensity);

            new_sizes.msgsizes[code] *= factor;
        }
    }

    //MPI_CHECK(MPI_Barrier(activecomm));
    //redistribute.adjust_message_sizes(new_sizes);
    //dpd.adjust_message_sizes(new_sizes);
    //MPI_CHECK(MPI_Barrier(activecomm));

    //there is no support for killing zero-workload ranks for rbcs and ctcs just yet
    /* this is unnecessarily complex for now
       if (!rbcs && !ctcs)
       {
       const bool local_work = new_sizes.msgsizes[1 + 3 + 9] > 0;

       MPI_CHECK(MPI_Comm_split(cartcomm, local_work, rank, &activecomm)) ;

       MPI_CHECK(MPI_Comm_rank(activecomm, &rank));

       if (!local_work )
       {
       if (rank == 0)
       {
       int nkilled;
       MPI_CHECK(MPI_Comm_size(activecomm, &nkilled));

       printf("THERE ARE %d RANKS WITH ZERO WORKLOAD THAT WILL MPI-FINALIZE NOW.\n", nkilled);
       }

       termination_request = true;
       return;
       }
       }
     */

    particles->resize(nsurvived);
    particles->clear_velocity();
    cells.build(particles->xyzuvw.data, particles->size, 0, NULL, NULL);

    _update_helper_arrays();

    CUDA_CHECK(hipPeekAtLastError());

    //remove cells touching the wall
    _remove_bodies_from_wall(rbcscoll);
    _remove_bodies_from_wall(ctcscoll);

    {
        H5PartDump sd("survived-particles->h5part", activecomm, cartcomm);
        Particle * p = new Particle[particles->size];

        CUDA_CHECK(hipMemcpy(p, particles->xyzuvw.data, sizeof(Particle) * particles->size, hipMemcpyDeviceToHost));

        sd.dump(p, particles->size);

        delete [] p;
    }
}

void Simulation::_forces(bool firsttime)
{
    double tstart = MPI_Wtime();

    SolventWrap wsolvent(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count);

    std::vector<ParticlesWrap> wsolutes;

    if (rbcscoll)
        wsolutes.push_back(ParticlesWrap(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc()));

    if (ctcscoll)
        wsolutes.push_back(ParticlesWrap(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc()));

    fsi.bind_solvent(wsolvent);

    solutex.bind_solutes(wsolutes);

    particles->clear_acc(mainstream);

    if (rbcscoll)
        rbcscoll->clear_acc(mainstream);

    if (ctcscoll)
        ctcscoll->clear_acc(mainstream);

    dpd.pack(particles->xyzuvw.data, particles->size, cells.start, cells.count, mainstream);

    solutex.pack_p(mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (contactforces)
        contact.build_cells(wsolutes, mainstream);

    dpd.local_interactions(particles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, particles->size, particles->axayaz.data,
            cells.start, cells.count, mainstream);

    dpd.post(particles->xyzuvw.data, particles->size, mainstream, downloadstream);

    solutex.post_p(mainstream, downloadstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll && wall)
        wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);

    if (ctcscoll && wall)
        wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);

    if (wall)
        wall->interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data,
                cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    dpd.recv(mainstream, uploadstream);

    solutex.recv_p(uploadstream);

    solutex.halo(uploadstream, mainstream);

    dpd.remote_interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data, mainstream, uploadstream);

    fsi.bulk(wsolutes, mainstream);

    if (contactforces)
        contact.bulk(wsolutes, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (nsubsteps == 0)
    {
        if (rbcscoll)
            CudaRBC::forces_nohost(mainstream, rbcscoll->count(), (float *)rbcscoll->data(), (float *)rbcscoll->acc());

        if (ctcscoll)
            CudaCTC::forces_nohost(mainstream, ctcscoll->count(), (float *)ctcscoll->data(), (float *)ctcscoll->acc());
    }

    CUDA_CHECK(hipPeekAtLastError());

    solutex.post_a();

    solutex.recv_a(mainstream);

    if (nsubsteps)
    { // TSS
        if (rbcscoll)
            CUDA_CHECK( hipMemcpyAsync(rbcscoll->fsiacc(), rbcscoll->acc(), 3*rbcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );

        if (ctcscoll)
            CUDA_CHECK( hipMemcpyAsync(ctcscoll->fsiacc(), ctcscoll->acc(), 3*ctcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );

        for (int sstep = 0; sstep < nsubsteps; sstep++)
        {
            // Start with acc induced by solvent
            if (rbcscoll)
            {
                if (sstep > 0)
                    CUDA_CHECK( hipMemcpyAsync(rbcscoll->acc(), rbcscoll->fsiacc(), 3*rbcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );
                CudaRBC::forces_nohost(mainstream, rbcscoll->count(), (float *)rbcscoll->data(), (float *)rbcscoll->acc());

                if (firsttime)
                    rbcscoll->update_stage1(0.0, mainstream, (dt / nsubsteps));
                else
                    rbcscoll->update_stage2_and_1(0.0, mainstream, (dt / nsubsteps));

                if (wall)
                    wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream, dt / (nsubsteps));
            }

            if (ctcscoll)
            {
                if (sstep > 0)
                    CUDA_CHECK( hipMemcpyAsync(ctcscoll->acc(), ctcscoll->fsiacc(), 3*ctcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );
                CudaCTC::forces_nohost(mainstream, ctcscoll->count(), (float *)ctcscoll->data(), (float *)ctcscoll->acc());

                if (firsttime)
                    ctcscoll->update_stage2_and_1(0.0, mainstream, dt / (nsubsteps));
                else
                    ctcscoll->update_stage1(0.0, mainstream, dt / (nsubsteps));

                if (wall)
                    wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream, dt / (nsubsteps));
            }
        }
    }

    timings["interactions"] += MPI_Wtime() - tstart;

    CUDA_CHECK(hipPeekAtLastError());
}

void Simulation::_datadump(const int idtimestep)
{
    double tstart = MPI_Wtime();

    int n = particles->size;

    if (rbcscoll)
        n += rbcscoll->pcount();

    if (ctcscoll)
        n += ctcscoll->pcount();

    particles_datadump.resize(n);
    accelerations_datadump.resize(n);

    CUDA_CHECK(hipMemcpyAsync(particles_datadump.data, particles->xyzuvw.data, sizeof(Particle) * particles->size, hipMemcpyDeviceToHost,0));
    CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data, particles->axayaz.data, sizeof(Acceleration) * particles->size, hipMemcpyDeviceToHost,0));
    if (nsubsteps > 0)
    {
        CUDA_CHECK( hipStreamSynchronize(0) );
        for (int i=0; i<particles->size; i++)
            for (int c=0; c<3; c++)
                particles_datadump.data[i].u[c] += dt * accelerations_datadump.data[i].a[c];
    }

    int start = particles->size;

    if (rbcscoll)
    {
        CUDA_CHECK(hipMemcpyAsync(particles_datadump.data + start, rbcscoll->xyzuvw.data, sizeof(Particle) * rbcscoll->pcount(), hipMemcpyDeviceToHost, 0));
        CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data + start, rbcscoll->axayaz.data, sizeof(Acceleration) * rbcscoll->pcount(), hipMemcpyDeviceToHost, 0));

        start += rbcscoll->pcount();
    }

    if (ctcscoll)
    {
        CUDA_CHECK(hipMemcpyAsync(particles_datadump.data + start, ctcscoll->xyzuvw.data, sizeof(Particle) * ctcscoll->pcount(), hipMemcpyDeviceToHost, 0));
        CUDA_CHECK(hipMemcpyAsync(accelerations_datadump.data + start, ctcscoll->axayaz.data, sizeof(Acceleration) * ctcscoll->pcount(), hipMemcpyDeviceToHost, 0));

        start += ctcscoll->pcount();
    }

    assert(start == n);
    CUDA_CHECK(hipEventRecord(evdownloaded, 0));

    datadump_nsolvent = particles->size;
    datadump_nrbcs = rbcscoll ? rbcscoll->pcount() : 0;
    datadump_nctcs = ctcscoll ? ctcscoll->pcount() : 0;

    MPI_CHECK( MPI_Send(&datadump_nsolvent, 1, MPI_INT, rank, 0, intercomm) );
    MPI_CHECK( MPI_Send(&datadump_nrbcs,    1, MPI_INT, rank, 0, intercomm) );
    MPI_CHECK( MPI_Send(&datadump_nctcs,    1, MPI_INT, rank, 0, intercomm) );

    CUDA_CHECK( hipEventSynchronize(evdownloaded) );

    MPI_CHECK( MPI_Send(particles_datadump.data,     n, Particle::datatype(),     rank, 0, intercomm) );
    MPI_CHECK( MPI_Send(accelerations_datadump.data, n, Acceleration::datatype(), rank, 0, intercomm) );

    timings["data-dump"] += MPI_Wtime() - tstart;
}

void Simulation::_update_and_bounce()
{
    double tstart = MPI_Wtime();
    particles->update_stage2_and_1(driving_acceleration, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (nsubsteps == 0)
    {
        if (rbcscoll)
            rbcscoll->update_stage2_and_1(0.0f, mainstream);

        CUDA_CHECK(hipPeekAtLastError());

        if (ctcscoll)
            ctcscoll->update_stage2_and_1(0.0f, mainstream);
    }

    timings["update"] += MPI_Wtime() - tstart;

    if (wall)
    {
        tstart = MPI_Wtime();
        wall->bounce(particles->xyzuvw.data, particles->size, mainstream);

        if (nsubsteps == 0)
        {
            if (rbcscoll)
                wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);

            if (ctcscoll)
                wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);
        }

        timings["bounce-walls"] += MPI_Wtime() - tstart;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

Simulation::Simulation(MPI_Comm cartcomm, MPI_Comm activecomm, MPI_Comm intercomm, bool (*check_termination)()) :
            cartcomm(cartcomm), activecomm(activecomm), intercomm(intercomm),
            /*particles(_ic()),*/ cells(XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN),
            rbcscoll(NULL), ctcscoll(NULL), wall(NULL),
            redistribute(cartcomm),  redistribute_rbcs(cartcomm),  redistribute_ctcs(cartcomm),
            dpd(cartcomm), fsi(cartcomm), contact(cartcomm), solutex(cartcomm),
            check_termination(check_termination),
            driving_acceleration(0), host_idle_time(0), nsteps((int)(tend / dt)),
            datadump_pending(false), simulation_is_done(false),
            qoiid(0)
{
    MPI_CHECK( MPI_Comm_size(activecomm, &nranks) );
    MPI_CHECK( MPI_Comm_rank(activecomm, &rank) );

    solutex.attach_halocomputation(fsi);

    if (contactforces)
        solutex.attach_halocomputation(contact);
    //localcomm.initialize(activecomm);

    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    {
        particles = &particles_pingpong[0];
        newparticles = &particles_pingpong[1];

        vector<Particle> ic = _ic();

        for(int c = 0; c < 2; ++c)
        {
            particles_pingpong[c].resize(ic.size());

            particles_pingpong[c].origin = make_float3((0.5 + coords[0]) * XSIZE_SUBDOMAIN,
                    (0.5 + coords[1]) * YSIZE_SUBDOMAIN,
                    (0.5 + coords[2]) * ZSIZE_SUBDOMAIN);

            particles_pingpong[c].globalextent = make_float3(dims[0] * XSIZE_SUBDOMAIN,
                    dims[1] * YSIZE_SUBDOMAIN,
                    dims[2] * ZSIZE_SUBDOMAIN);
        }

        CUDA_CHECK(hipMemcpy(particles->xyzuvw.data, &ic.front(), sizeof(Particle) * ic.size(), hipMemcpyHostToDevice));

        cells.build(particles->xyzuvw.data, particles->size, 0, NULL, NULL);

        _update_helper_arrays();
    }

    CUDA_CHECK(hipStreamCreate(&mainstream));
    CUDA_CHECK(hipStreamCreate(&uploadstream));
    CUDA_CHECK(hipStreamCreate(&downloadstream));

    if (rbcs)
    {
        rbcscoll = new CollectionRBC(cartcomm);
        rbcscoll->setup("rbcs-ic.txt");
    }

    if (ctcs)
    {
        ctcscoll = new CollectionCTC(cartcomm);
        ctcscoll->setup("ctcs-ic.txt");
    }

    CUDA_CHECK(hipEventCreate(&evdownloaded, hipEventDisableTiming | hipEventBlockingSync));
    particles_datadump.resize(particles->size * 1.5);
    accelerations_datadump.resize(particles->size * 1.5);
}

void Simulation::_lockstep()
{
    double tstart = MPI_Wtime();

    SolventWrap wsolvent(particles->xyzuvw.data, particles->size, particles->axayaz.data, cells.start, cells.count);

    std::vector<ParticlesWrap> wsolutes;

    if (rbcscoll)
        wsolutes.push_back(ParticlesWrap(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc()));

    if (ctcscoll)
        wsolutes.push_back(ParticlesWrap(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc()));

    fsi.bind_solvent(wsolvent);

    solutex.bind_solutes(wsolutes);

    particles->clear_acc(mainstream);

    if (rbcscoll)
        rbcscoll->clear_acc(mainstream);

    if (ctcscoll)
        ctcscoll->clear_acc(mainstream);

    solutex.pack_p(mainstream);

    dpd.pack(particles->xyzuvw.data, particles->size, cells.start, cells.count, mainstream);

    dpd.local_interactions(particles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, particles->size, particles->axayaz.data,
            cells.start, cells.count, mainstream);

    if (contactforces)
        contact.build_cells(wsolutes, mainstream);

    solutex.post_p(mainstream, downloadstream);

    dpd.post(particles->xyzuvw.data, particles->size, mainstream, downloadstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (wall)
        wall->interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data,
                cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    dpd.recv(mainstream, uploadstream);

    solutex.recv_p(uploadstream);

    solutex.halo(uploadstream, mainstream);

    dpd.remote_interactions(particles->xyzuvw.data, particles->size, particles->axayaz.data, mainstream, uploadstream);

    fsi.bulk(wsolutes, mainstream);

    if (contactforces)
        contact.bulk(wsolutes, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (nsubsteps == 0)
    {
        if (rbcscoll)
            CudaRBC::forces_nohost(mainstream, rbcscoll->count(), (float *)rbcscoll->data(), (float *)rbcscoll->acc());

        if (ctcscoll)
            CudaCTC::forces_nohost(mainstream, ctcscoll->count(), (float *)ctcscoll->data(), (float *)ctcscoll->acc());
    }
    CUDA_CHECK(hipPeekAtLastError());

    solutex.post_a();

    particles->update_stage2_and_1(driving_acceleration, mainstream);

    if (wall)
        wall->bounce(particles->xyzuvw.data, particles->size, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    redistribute.pack(particles->xyzuvw.data, particles->size, mainstream);

    redistribute.send();

    redistribute.bulk(particles->size, cells.start, cells.count, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll && wall)
        wall->interactions(rbcscoll->data(), rbcscoll->pcount(), rbcscoll->acc(), NULL, NULL, mainstream);

    if (ctcscoll && wall)
        wall->interactions(ctcscoll->data(), ctcscoll->pcount(), ctcscoll->acc(), NULL, NULL, mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    solutex.recv_a(mainstream);

    if (nsubsteps == 0)
    {
        if (rbcscoll)
            rbcscoll->update_stage2_and_1(0.0f, mainstream);

        if (ctcscoll)
            ctcscoll->update_stage2_and_1(0.0f, mainstream);

        if (wall && rbcscoll)
            wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream);

        if (wall && ctcscoll)
            wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream);
    }
    else
    { // TSS
        if (rbcscoll)
            CUDA_CHECK( hipMemcpyAsync(rbcscoll->fsiacc(), rbcscoll->acc(), 3*rbcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );

        if (ctcscoll)
            CUDA_CHECK( hipMemcpyAsync(ctcscoll->fsiacc(), ctcscoll->acc(), 3*ctcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );

        for (int sstep = 0; sstep < nsubsteps; sstep++)
        {
            // Start with acc induced by solvent
            if (rbcscoll)
            {
                if (sstep > 0)
                    CUDA_CHECK( hipMemcpyAsync(rbcscoll->acc(), rbcscoll->fsiacc(), 3*rbcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );
                CudaRBC::forces_nohost(mainstream, rbcscoll->count(), (float *)rbcscoll->data(), (float *)rbcscoll->acc());

                rbcscoll->update_stage2_and_1(0.0, mainstream, (dt / nsubsteps));

                if (wall)
                    wall->bounce(rbcscoll->data(), rbcscoll->pcount(), mainstream, dt / (nsubsteps));
            }

            if (ctcscoll)
            {
                if (sstep > 0)
                    CUDA_CHECK( hipMemcpyAsync(ctcscoll->acc(), ctcscoll->fsiacc(), 3*ctcscoll->pcount()*sizeof(float), hipMemcpyDeviceToDevice, mainstream) );
                CudaCTC::forces_nohost(mainstream, ctcscoll->count(), (float *)ctcscoll->data(), (float *)ctcscoll->acc());

                ctcscoll->update_stage1(0.0, mainstream, dt / (nsubsteps));

                if (wall)
                    wall->bounce(ctcscoll->data(), ctcscoll->pcount(), mainstream, dt / (nsubsteps));
            }
        }
    }

    const int newnp = redistribute.recv_count(mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
        redistribute_rbcs.extent(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
        redistribute_ctcs.extent(ctcscoll->data(), ctcscoll->count(), mainstream);

    if (rbcscoll)
        redistribute_rbcs.pack_sendcount(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
        redistribute_ctcs.pack_sendcount(ctcscoll->data(), ctcscoll->count(), mainstream);

    newparticles->resize(newnp);
    xyzouvwo.resize(newnp * 2);
    xyzo_half.resize(newnp);

    redistribute.recv_unpack(newparticles->xyzuvw.data, xyzouvwo.data, xyzo_half.data, newnp, cells.start, cells.count, mainstream, host_idle_time);

    CUDA_CHECK(hipPeekAtLastError());

    swap(particles, newparticles);

    int nrbcs;
    if (rbcscoll)
        nrbcs = redistribute_rbcs.post();

    int nctcs;
    if (ctcscoll)
        nctcs = redistribute_ctcs.post();

    if (rbcscoll)
        rbcscoll->resize(nrbcs);

    if (ctcscoll)
        ctcscoll->resize(nctcs);

    CUDA_CHECK(hipPeekAtLastError());

    if (rbcscoll)
        redistribute_rbcs.unpack(rbcscoll->data(), rbcscoll->count(), mainstream);

    if (ctcscoll)
        redistribute_ctcs.unpack(ctcscoll->data(), ctcscoll->count(), mainstream);

    CUDA_CHECK(hipPeekAtLastError());

    timings["lockstep"] += MPI_Wtime() - tstart;
}


void Simulation::run()
{
    if (rank == 0 && !walls)
        printf("the simulation begins now and it consists of %.3e steps\n", (double)nsteps);

    double time_simulation_start = MPI_Wtime();

    _redistribute();
    _forces(nsubsteps > 0);

    if (!walls && pushtheflow)
        driving_acceleration = hydrostatic_a;

    particles->update_stage1(driving_acceleration, mainstream);

    if (nsubsteps == 0)
    {
        if (rbcscoll)
            rbcscoll->update_stage1(0.0f, mainstream);

        if (ctcscoll)
            ctcscoll->update_stage1(0.0f, mainstream);
    }

    int it;


    for(it = 0; it < nsteps; ++it)
    {
        const bool verbose = it > 0 && rank == 0;

#ifdef _USE_NVTX_
        if (it == nvtxstart)
        {
            NvtxTracer::currently_profiling = true;
            CUDA_CHECK(hipProfilerStart());
        }
        else if (it == nvtxstop)
        {
            CUDA_CHECK(hipProfilerStop());
            NvtxTracer::currently_profiling = false;
            CUDA_CHECK(hipDeviceSynchronize());

            if (rank == 0)
                printf("profiling session ended. terminating the simulation now...\n");

            break;
        }
#endif

        if (it % steps_per_report == 0)
        {
            CUDA_CHECK(hipStreamSynchronize(mainstream));

            if (simulation_is_done = check_termination())
                break;

            _report(verbose, it);
        }

        _redistribute();

#if 1
        lockstep_check:

        const bool lockstep_OK =
                !(walls && it >= wall_creation_stepid && wall == NULL) &&
                !(it % steps_per_dump == 0) &&
                !(it + 1 == nvtxstart) &&
                !(it + 1 == nvtxstop) &&
                !((it + 1) % steps_per_report == 0) &&
                !(it + 1 == nsteps);

        if (lockstep_OK)
        {
            _lockstep();

            ++it;

            goto lockstep_check;
        }
#endif

        if (walls && it >= wall_creation_stepid && wall == NULL)
        {
            CUDA_CHECK(hipDeviceSynchronize());

            bool termination_request = false;

            _create_walls(verbose, termination_request);

            _redistribute();

            if (termination_request)
                break;

            time_simulation_start = MPI_Wtime();

            if (pushtheflow)
                driving_acceleration = hydrostatic_a;

            if (rank == 0)
                printf("the simulation begins now and it consists of %.3e steps\n", (double)(nsteps - it));
        }

        _forces();

#ifndef _NO_DUMPS_
        if (it % steps_per_dump == 0)
            _datadump(it);
#endif
        _update_and_bounce();
    }

    const double time_simulation_stop = MPI_Wtime();
    const double telapsed = time_simulation_stop - time_simulation_start;

    simulation_is_done = true;

    datadump_nsolvent = datadump_nrbcs = datadump_nctcs = -1;
    MPI_CHECK( MPI_Send(&datadump_nsolvent, 1, MPI_INT, rank, 0, intercomm) );
    MPI_CHECK( MPI_Send(&datadump_nrbcs,    1, MPI_INT, rank, 0, intercomm) );
    MPI_CHECK( MPI_Send(&datadump_nctcs,    1, MPI_INT, rank, 0, intercomm) );

    if (rank == 0)
        if (it == nsteps)
            printf("simulation is done after %.2lf s (%dm%ds). Ciao.\n",
                    telapsed, (int)(telapsed / 60), (int)(telapsed) % 60);
        else
            if (it != wall_creation_stepid)
                printf("external termination request (signal) after %.3e s. Bye.\n", telapsed);

    fflush(stdout);
}

Simulation::~Simulation()
{
    CUDA_CHECK(hipStreamDestroy(mainstream));
    CUDA_CHECK(hipStreamDestroy(uploadstream));
    CUDA_CHECK(hipStreamDestroy(downloadstream));

    if (wall)
        delete wall;

    if (rbcscoll)
        delete rbcscoll;

    if (ctcscoll)
        delete ctcscoll;
}
