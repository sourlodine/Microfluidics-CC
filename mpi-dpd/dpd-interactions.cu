#include "hip/hip_runtime.h"
#include <cassert>

#include <algorithm>

#include <cuda-dpd.h>

#include "dpd-interactions.h"

using namespace std;

ComputeInteractionsDPD::ComputeInteractionsDPD(MPI_Comm cartcomm, int L):
    HaloExchanger(cartcomm, L)
{
    for(int i = 0; i < 7; ++i)
	CUDA_CHECK(hipStreamCreate(streams + i));

    for(int i = 0, ctr = 1; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	const bool isface = abs(d[0]) + abs(d[1]) + abs(d[2]) == 1;

	code2stream[i] = 0;

	if (isface)
	{
	    code2stream[i] = ctr;
	    ctr++;
	}
    }
}

void ComputeInteractionsDPD::evaluate(int& saru_tag, const Particle * const p, const int n, Acceleration * const a,
				      const int * const cellsstart, const int * const cellscount)
{
    dpd_remote_interactions_stage1(p, n, cellsstart, cellscount);

    if (n > 0)
	forces_dpd_cuda_nohost((float *)p, (float *)a, n, 
			       cellsstart, cellscount,
			       1, L, L, L, aij, gammadpd, sigma, 1. / sqrt(dt), saru_tag);

    saru_tag += nranks - myrank;

    dpd_remote_interactions_stage2(p, n, saru_tag, a);

    saru_tag += 1 + myrank;  
}


__global__ void not_nan(float * p, const int n)
{
    assert(gridDim.x * blockDim.x >= n);

    const int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if (gid < n)
	assert(!isnan(p[gid]));
}

__global__ void fill_random(float * p, const int n)
{
    assert(gridDim.x * blockDim.x >= n);

    const int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if (gid < n)
	p[gid] = 2 * (gid % 100) * 0.01 - 1;
}

namespace RemoteDPD
{
    __global__ void merge_accelerations(const Acceleration * const aremote, const int nremote,
					Acceleration * const alocal, const int nlocal,
					const Particle * premote, const Particle * plocal,
					const int * const scattered_entries, int rank)
    {
	assert(blockDim.x * gridDim.x >= nremote);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= nremote)
	    return;

	int pid = scattered_entries[ gid ];
	assert(pid >= 0 && pid < nlocal);

	Acceleration a = aremote[gid];

#ifndef NDEBUG
	Particle p1 = plocal[pid];
	Particle p2 = premote[gid];

	for(int c = 0; c < 3; ++c)
	{
	    assert(p1.x[c] == p2.x[c]);
	    assert(p1.x[c] == p2.x[c]);
	}

	for(int c = 0; c < 3; ++c)
	{
	    if (isnan(a.a[c]))
		printf("rank %d) oouch gid %d %f out of %d remote entries going to pid %d of %d particles\n", rank, gid, a.a[c], nremote, pid, nlocal);

	    assert(!isnan(a.a[c]));
	}
#endif
	for(int c = 0; c < 3; ++c)
	{
	    const float val = alocal[pid].a[c];
	    
	    alocal[pid].a[c] = val + a.a[c];
	    
	    assert(!isnan(val));
	}
    }
}

void ComputeInteractionsDPD::dpd_remote_interactions_stage1(const Particle * const p, const int n, const int * const cellsstart, const int * const cellscount)
{
    HaloExchanger::pack_and_post(p, n, cellsstart, cellscount);
}

void ComputeInteractionsDPD::dpd_remote_interactions_stage2(const Particle * const p, const int n, const int saru_tag1, Acceleration * const a)
{
    wait_for_messages();

    int saru_tag2[26];
    bool saru_mask[26];
    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = (coords[c] + d[c] + dims[c]) % dims[c];

	int indx[3];
	for(int c = 0; c < 3; ++c)
	    indx[c] = min(coords[c], coordsneighbor[c]) * dims[c] + max(coords[c], coordsneighbor[c]);

	saru_tag2[i] = indx[0] + dims[0] * dims[0] * (indx[1] + dims[1] * dims[1] * indx[2]);

	int dstrank;
	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, &dstrank) );

	if (dstrank != myrank)
	    saru_mask[i] = min(dstrank, myrank) == myrank;
	else
	{
	    int alter_ego = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));
	    saru_mask[i] = min(i, alter_ego) == i;
	}
    }

    for(int i = 0; i < 26; ++i)
    {
	const int nd = sendhalos[i].buf.size;//send_counts[i];
	const int ns = recvbufs[i].size;//recv_counts[i];

	acc_remote[i].resize(nd);

	if (nd == 0)
	    continue;
	
#ifndef NDEBUG
	//fill acc entries with nan
	CUDA_CHECK(hipMemset(acc_remote[i].data, 0xff, sizeof(Acceleration) * acc_remote[i].size));
#endif
	
	if (ns == 0)
	{
	    CUDA_CHECK(hipMemset((float *)acc_remote[i].data, 0, nd * sizeof(Acceleration)));
	    continue;
	}
	CUDA_CHECK(hipDeviceSynchronize());

	hipStream_t mystream = streams[code2stream[i]];

	directforces_dpd_cuda_bipartite_nohost(
	    (float *)sendhalos[i].buf.data, (float *)acc_remote[i].data, nd,
	    (float *)recvbufs[i].data, ns,
	    aij, gammadpd, sigma, 1. / sqrt(dt), saru_tag1, saru_tag2[i], saru_mask[i], mystream);
	CUDA_CHECK(hipPeekAtLastError());
	CUDA_CHECK(hipDeviceSynchronize());
    }

    CUDA_CHECK(hipDeviceSynchronize());

    for(int i = 0; i < 26; ++i)
    {
	const int nd = acc_remote[i].size;
	
	if (nd > 0)
	    RemoteDPD::merge_accelerations<<<(nd + 127) / 128, 128>>>(acc_remote[i].data, nd, a, n,
								      sendhalos[i].buf.data, p, sendhalos[i].scattered_entries.data, myrank);
	CUDA_CHECK(hipPeekAtLastError());
	CUDA_CHECK(hipDeviceSynchronize());
    }
   
    CUDA_CHECK(hipPeekAtLastError());
}

ComputeInteractionsDPD::~ComputeInteractionsDPD()
{
    for(int i = 0; i < 7; ++i)
	CUDA_CHECK(hipStreamDestroy(streams[i]));
}
