#include "hip/hip_runtime.h"
/*
 *  redistribute-particles.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-02-09.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cassert>
#include <vector>
#include <algorithm>

#include "common-kernels.h"
#include "scan.h"
#include "redistribute-particles.h"

#ifndef WARPSIZE
#define WARPSIZE 32
#endif

using namespace std;

namespace RedistributeParticlesKernels
{
    __constant__ RedistributeParticles::PackBuffer pack_buffers[27];

    __constant__ RedistributeParticles::UnpackBuffer unpack_buffers[27];

    __device__ int pack_count[27], pack_start_padded[28];

    __constant__ int unpack_start[28], unpack_start_padded[28];

    __device__ bool failed;

    int ntexparticles = 0;
    float2 * texparticledata;
    texture<float, hipTextureType1D> texAllParticles;
    texture<float2, hipTextureType1D> texAllParticlesFloat2;

#if !defined(__CUDA_ARCH__)
#warning __CUDA_ARCH__ not defined! assuming 350
#define _ACCESS(x) __ldg(x)
#elif __CUDA_ARCH__ >= 350
#define _ACCESS(x) __ldg(x)
#else
#define _ACCESS(x) (*(x))
#endif

    __global__ void setup()
    {
	if (threadIdx.x == 0)
	    failed = false;

	if (threadIdx.x < 27)
	    pack_count[threadIdx.x] = 0;
    }

    __global__ void scatter_halo_indices_pack(const int np)
    {
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid < np)
	{
	    float xp[3];
	    for(int c = 0; c < 3; ++c)
		xp[c] = tex1Dfetch(texAllParticles, 6 * pid + c);

	    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	    int vcode[3];
	    for(int c = 0; c < 3; ++c)
		vcode[c] = (2 + (xp[c] >= -L[c]/2) + (xp[c] >= L[c]/2)) % 3;

	    const int code = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);
	    assert(code >= 0 && code < 27);

	    if (code > 0)
	    {
		const int entry = atomicAdd(pack_count + code, 1);

		if (entry < pack_buffers[code].capacity)
		    pack_buffers[code].scattered_indices[entry] = pid;
	    }
	}
    }

    __global__ void tiny_scan(const int nparticles, const int bulkcapacity, int * const packsizes, bool * const failureflag)
    {
	assert(blockDim.x > 27 && gridDim.x == 1);

	const int tid = threadIdx.x;

	int myval = 0, mycount = 0;

	if (tid < 27)
	{
	    myval = mycount = pack_count[threadIdx.x];
	    if (tid > 0)
		packsizes[tid] = mycount;

	    if (mycount > pack_buffers[tid].capacity)
	    {
		failed = true;
		*failureflag = true;
	    }
	}

	//myval = 32 * ((myval + 31) / 32);

	for(int L = 1; L < 32; L <<= 1)
	    myval += (tid >= L) * __shfl_up(myval, L) ;

	if (tid < 28)
	    pack_start_padded[tid] = myval - mycount;

	if (tid == 26)
	{
	    pack_start_padded[tid + 1] = myval;

	    const int nbulk = nparticles - myval;
	    packsizes[0] = nbulk;

	    if (nbulk > bulkcapacity)
	    {
		failed = true;
		*failureflag = true;
	    }
	}
    }

#ifndef NDEBUG
    __global__ void check_scan()
    {
	assert(blockDim.x == 1 && gridDim.x == 1);

	for(int i = 1; i < 28; ++i)
	    assert(pack_start_padded[i - 1] <= pack_start_padded[i]);
    }
#endif

    __global__ void pack(const int nparticles, const int nfloat2s)
    {

	assert(blockDim.x * gridDim.x >= nfloat2s);

	if (failed)
	    return;

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	const int slot = gid / 3;

	const int tid = threadIdx.x;

	__shared__ int start[28]; //, count[27];

	if (tid < 28)
	    start[tid] = pack_start_padded[tid];

	//if (tid < 27)
	//   count[tid] = pack_count[tid];

	__syncthreads();

	const int key9 = 9 * (slot >= start[9]) + 9 * (slot >= start[18]);
	const int key3 = 3 * (slot >= start[key9 + 3]) + 3 * (slot >= start[key9 + 6]);
	const int key1 = (slot >= start[key9 + key3 + 1]) + (slot >= start[key9 + key3 + 2]);

	const int idpack = key9 + key3 + key1;

	if (slot >= start[27])
	    return;

	const int offset = slot - start[idpack];

	//if (offset >= count[idpack])
	//    return;

	assert (offset >= 0 && offset < pack_buffers[idpack].capacity);

	const int pid = _ACCESS(pack_buffers[idpack].scattered_indices + offset);
	assert(pid < nparticles && pid >= 0);

	const int c = gid % 3;
	const int d = c + 3 * offset;
	assert (d < pack_buffers[idpack].capacity * 3);

	pack_buffers[idpack].buffer[d] = tex1Dfetch(texAllParticlesFloat2, c + 3 * pid);
    }

    __global__ void subindex_remote(const uint nparticles_padded,
				    const uint nparticles, int * const partials, float2 * const dstbuf, uchar4 * const subindices)
    {
	assert(blockDim.x * gridDim.x >= nparticles_padded && blockDim.x == 128);

	const uint warpid = threadIdx.x >> 5;

	const uint localbase = 32 * (warpid + 4 * blockIdx.x);

	if (localbase >= nparticles_padded)
	    return;

	const uint key9 = 9 * (localbase >= unpack_start_padded[9]) + 9 * (localbase >= unpack_start_padded[18]);
	const uint key3 = 3 * (localbase >= unpack_start_padded[key9 + 3]) + 3 * (localbase >= unpack_start_padded[key9 + 6]);
	const uint key1 = (localbase >= unpack_start_padded[key9 + key3 + 1]) + (localbase >= unpack_start_padded[key9 + key3 + 2]);
	const int code = key9 + key3 + key1;
	assert(code >= 1 && code < 28);
	assert(localbase >= unpack_start_padded[code] && localbase < unpack_start_padded[code + 1]);

	const int unpackbase = localbase - unpack_start_padded[code];
	assert (unpackbase >= 0);
	assert(unpackbase < unpack_buffers[code].capacity);

	const uint nunpack = min(32, unpack_start[code + 1] - unpack_start[code] - unpackbase);

	if (nunpack == 0)
	    return;

	float2 data0, data1, data2;

	read_AOS6f(unpack_buffers[code].buffer + 3 * unpackbase, nunpack, data0, data1, data2);

	const uint laneid = threadIdx.x & 0x1f;

	int xcid, ycid, zcid, subindex;

	if (laneid < nunpack)
	{
	    data0.x += XSIZE_SUBDOMAIN * ((code + 1) % 3 - 1);
	    data0.y += YSIZE_SUBDOMAIN * ((code / 3 + 1) % 3 - 1);
	    data1.x += ZSIZE_SUBDOMAIN * ((code / 9 + 1) % 3 - 1);

	    xcid = (int)floor((double)data0.x + XSIZE_SUBDOMAIN / 2);
	    ycid = (int)floor((double)data0.y + YSIZE_SUBDOMAIN / 2);
	    zcid = (int)floor((double)data1.x + ZSIZE_SUBDOMAIN / 2);

	    assert(xcid >= 0 && xcid < XSIZE_SUBDOMAIN &&
		   ycid >= 0 && ycid < YSIZE_SUBDOMAIN &&
		   zcid >= 0 && zcid < ZSIZE_SUBDOMAIN );

	    const int cid = xcid + XSIZE_SUBDOMAIN * (ycid + YSIZE_SUBDOMAIN * zcid);

	    subindex = atomicAdd(partials + cid, 1);

	    assert(subindex < 255);
	}

	const uint dstbase = unpack_start[code] + unpackbase;

	write_AOS6f(dstbuf + 3 * dstbase, nunpack, data0, data1, data2);

	if (laneid < nunpack)
	    subindices[dstbase + laneid] = make_uchar4(xcid, ycid, zcid, subindex);
    }

    __global__ void scatter_indices(const bool remote, const uchar4 * const subindices, const int nparticles,
				    const int * const starts, uint * const scattered_indices, const int nscattered)
    {
	assert(blockDim.x * gridDim.x >= nparticles);

	uint pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= nparticles)
	    return;

	const uchar4 entry = subindices[pid];

	const int subindex = entry.w;

	if (subindex != 255)
	{
	    const int cid = entry.x + XSIZE_SUBDOMAIN * (entry.y + YSIZE_SUBDOMAIN * entry.z);
	    const int base = _ACCESS(starts + cid);

	    pid |= remote << 31;

	    assert(base + subindex < nscattered);

	    //if (pid == 0)
	    //	printf("pid %d: base: %d subindex %d cid %d\n", pid, base, subindex, cid);

	    scattered_indices[base + subindex] = pid;
	}
    }

    __forceinline__ __device__ void xchg_aos2f(const int srclane0, const int srclane1, const int start, float& s0, float& s1)
    {
	const float t0 = __shfl(s0, srclane0);
	const float t1 = __shfl(s1, srclane1);

	s0 = start == 0 ? t0 : t1;
	s1 = start == 0 ? t1 : t0;

	s1 = __shfl_xor(s1, 1);
    }

    __forceinline__ __device__ void xchg_aos4f(const int srclane0, const int srclane1, const int start, float3& s0, float3& s1)
    {
	xchg_aos2f(srclane0, srclane1, start, s0.x, s1.x);
	xchg_aos2f(srclane0, srclane1, start, s0.y, s1.y);
	xchg_aos2f(srclane0, srclane1, start, s0.z, s1.z);
    }

    __global__ void gather_particles(const uint * const scattered_indices,
				     const float2 * const remoteparticles, const int nremoteparticles,
				     const int noldparticles,
				     const int nparticles,
				     float2 * const dstbuf,
				     float4 * const xyzouvwo,
				     ushort4 * const xyzo_half)
    {
	assert(blockDim.x == 128);

	const int warpid = threadIdx.x >> 5;
	const int tid = threadIdx.x & 0x1f;

	const int base = 32 * (warpid + 4 * blockIdx.x);
	const int pid = base + tid;

	const bool valid = (pid < nparticles);

	uint spid;

	if (valid)
	    spid = scattered_indices[pid];

	float2 data0, data1, data2;

	if (valid)
	{
	    const bool remote = (spid >> 31) & 1;

	    spid &= ~(1 << 31);

	    if (remote)
	    {
		assert(spid < nremoteparticles);
		data0 = _ACCESS(remoteparticles + 0 + 3 * spid);
		data1 = _ACCESS(remoteparticles + 1 + 3 * spid);
		data2 = _ACCESS(remoteparticles + 2 + 3 * spid);
	    }
	    else
	    {
		if (spid >= noldparticles)
		    cuda_printf("ooops pid %d spid %d noldp%d\n", pid, spid, noldparticles);

		assert(spid < noldparticles);
		data0 = tex1Dfetch(texAllParticlesFloat2, 0 + 3 * spid);
		data1 = tex1Dfetch(texAllParticlesFloat2, 1 + 3 * spid);
		data2 = tex1Dfetch(texAllParticlesFloat2, 2 + 3 * spid);
	    }
	}

	const int nsrc = min(32, nparticles - base);


	{
	    //if (tid < nsrc) {xyzouvwo[2 * (base + tid) + 0] = make_float4(data0.x, data0.y, data1.x, 0);
	    //	xyzouvwo[2 * (base + tid) + 1] = make_float4(data1.y, data2.x, data2.y, 0);}


	    const int srclane0 = (32 * ((tid) & 0x1) + tid) >> 1;
	    const int srclane1 = (32 * ((tid + 1) & 0x1) + tid) >> 1;
	    const int start = tid % 2;
	    const int destbase = 2 * base;

	    float3 s0 = make_float3(data0.x, data0.y, data1.x);
	    float3 s1 = make_float3(data1.y, data2.x, data2.y);

	    xchg_aos4f(srclane0, srclane1, start, s0, s1);

	    if (tid < 2 * nsrc)
		xyzouvwo[destbase + tid] = make_float4(s0.x, s0.y, s0.z, 0);

	    if (tid + 32 < 2 * nsrc)
		xyzouvwo[destbase + tid + 32] = make_float4(s1.x, s1.y, s1.z, 0);
	}

	if (tid < nsrc)
	{
	    xyzo_half[base + tid] = make_ushort4(
		__float2half_rn(data0.x),
		__float2half_rn(data0.y),
		__float2half_rn(data1.x), 0);
	}

	write_AOS6f(dstbuf + 3 * base, nsrc, data0, data1, data2);
    }

#ifndef NDEBUG
    __global__ void check(const int * const starts, const int * const counts, const Particle * const p, const int np)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN* ZSIZE_SUBDOMAIN)
	    return;

	const int count = counts[gid];
	const int start = starts[gid];


	const int xcid = gid % XSIZE_SUBDOMAIN;
	const int ycid = (gid / XSIZE_SUBDOMAIN) % YSIZE_SUBDOMAIN;
	const int zcid = gid / XSIZE_SUBDOMAIN / YSIZE_SUBDOMAIN ;

	const float xmin[3] = { xcid - XSIZE_SUBDOMAIN / 2,
				ycid - YSIZE_SUBDOMAIN / 2,
				zcid - ZSIZE_SUBDOMAIN / 2 };

	for(int i = 0; i < count; ++i)
	{
	    const int pid = start + i;

	    assert(pid < np && pid >= 0);

	    for(int c = 0; c < 3; ++c)
	    {
		assert(!isnan(p[pid].x[c]));

		if (!(p[pid].x[c] >= xmin[c] && p[pid].x[c] < xmin[c] + 1))
		{
		    printf("oooops pid %d c %d is %f of cell %d with count %d at entry %d not win [%f, %f[\n", pid, c, p[pid].x[c], gid, count, i,
			   xmin[c], xmin[c] + 1);
		}

		assert(p[pid].x[c] >= xmin[c] && p[pid].x[c] < xmin[c] + 1);
	    }
	}
    }
#endif

#undef _ACCESS
}

RedistributeParticles::RedistributeParticles(MPI_Comm _cartcomm):
failure(1), packsizes(27), nactiveneighbors(26), firstcall(true),
compressed_cellcounts(XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN),
subindices(1.5 * numberdensity * XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN),
subindices_remote(1.5 * numberdensity * (XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN -
					 (XSIZE_SUBDOMAIN - 2) * (YSIZE_SUBDOMAIN - 2) * (ZSIZE_SUBDOMAIN - 2)))
{
    safety_factor = getenv("RDP_COMM_FACTOR") ? atof(getenv("RDP_COMM_FACTOR")) : 1.2;

    MPI_CHECK(MPI_Comm_dup(_cartcomm, &cartcomm) );

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank) );
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 27; ++i)
    {
	const int d[3] = { (i + 1) % 3 - 1, (i / 3 + 1) % 3 - 1, (i / 9 + 1) % 3 - 1 };

	recv_tags[i] = (3 - d[0]) % 3 + 3 * ((3 - d[1]) % 3 + 3 * ((3 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, neighbor_ranks + i) );

	const int nhalodir[3] =  {
	    d[0] != 0 ? 1 : XSIZE_SUBDOMAIN,
	    d[1] != 0 ? 1 : YSIZE_SUBDOMAIN,
	    d[2] != 0 ? 1 : ZSIZE_SUBDOMAIN
	};

	const int nhalocells = nhalodir[0] * nhalodir[1] * nhalodir[2];

	const int estimate = numberdensity * safety_factor * nhalocells;

	CUDA_CHECK(hipMalloc(&packbuffers[i].scattered_indices, sizeof(int) * estimate));

	if (i && estimate)
	{
	    CUDA_CHECK(hipHostAlloc(&pinnedhost_sendbufs[i], sizeof(float) * 6 * estimate, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&packbuffers[i].buffer, pinnedhost_sendbufs[i], 0));

	    CUDA_CHECK(hipHostAlloc(&pinnedhost_recvbufs[i], sizeof(float) * 6 * estimate, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&unpackbuffers[i].buffer, pinnedhost_recvbufs[i], 0));
	}
	else
	{
      	    CUDA_CHECK(hipMalloc(&packbuffers[i].buffer, sizeof(float) * 6 * estimate));
	    unpackbuffers[i].buffer = packbuffers[i].buffer;

	    pinnedhost_sendbufs[i] = NULL;
	    pinnedhost_recvbufs[i] = NULL;
	}

	packbuffers[i].capacity = estimate;
	unpackbuffers[i].capacity = estimate;
	default_message_sizes[i] = estimate;
    }

    RedistributeParticlesKernels::texAllParticles.channelDesc = hipCreateChannelDesc<float>();
    RedistributeParticlesKernels::texAllParticles.filterMode = hipFilterModePoint;
    RedistributeParticlesKernels::texAllParticles.mipmapFilterMode = hipFilterModePoint;
    RedistributeParticlesKernels::texAllParticles.normalized = 0;

    RedistributeParticlesKernels::texAllParticlesFloat2.channelDesc = hipCreateChannelDesc<float2>();
    RedistributeParticlesKernels::texAllParticlesFloat2.filterMode = hipFilterModePoint;
    RedistributeParticlesKernels::texAllParticlesFloat2.mipmapFilterMode = hipFilterModePoint;
    RedistributeParticlesKernels::texAllParticlesFloat2.normalized = 0;

    CUDA_CHECK(hipEventCreate(&evpacking, hipEventDisableTiming));
    CUDA_CHECK(hipEventCreate(&evsizes, hipEventDisableTiming));
    //CUDA_CHECK(hipEventCreate(&evcompaction, hipEventDisableTiming));

CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>( RedistributeParticlesKernels::gather_particles), hipFuncCachePreferL1 ) );
}

void RedistributeParticles::_post_recv()
{
    for(int i = 1, c = 0; i < 27; ++i)
    	if (default_message_sizes[i])
	    MPI_CHECK( MPI_Irecv(recv_sizes + i, 1, MPI_INTEGER, neighbor_ranks[i], basetag + recv_tags[i], cartcomm, recvcountreq + c++) );
	else
	    recv_sizes[i] = 0;

    for(int i = 1, c = 0; i < 27; ++i)
	if (default_message_sizes[i])
	    MPI_CHECK( MPI_Irecv(pinnedhost_recvbufs[i], default_message_sizes[i] * 6, MPI_FLOAT,
				 neighbor_ranks[i], basetag + recv_tags[i] + 333, cartcomm, recvmsgreq + c++) );
}

void RedistributeParticles::_adjust_send_buffers(const int requested_capacities[27])
{
    for(int i = 0; i < 27; ++i)
    {
	if (requested_capacities[i] <= packbuffers[i].capacity)
	    continue;

	const int capacity = requested_capacities[i];

	CUDA_CHECK(hipFree(packbuffers[i].scattered_indices));
	CUDA_CHECK(hipMalloc(&packbuffers[i].scattered_indices, sizeof(int) * capacity));

	if (i)
	{
	    CUDA_CHECK(hipHostFree(pinnedhost_sendbufs[i]));

	    CUDA_CHECK(hipHostAlloc(&pinnedhost_sendbufs[i], sizeof(float) * 6 * capacity, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&packbuffers[i].buffer, pinnedhost_sendbufs[i], 0));

	    packbuffers[i].capacity = capacity;
	}
	else
	{
	    CUDA_CHECK(hipFree(packbuffers[i].buffer));

	    CUDA_CHECK(hipMalloc(&packbuffers[i].buffer, sizeof(float) * 6 * capacity));
	    unpackbuffers[i].buffer = packbuffers[i].buffer;

	    assert(pinnedhost_sendbufs[i] == NULL);

	    packbuffers[i].capacity = capacity;
	    unpackbuffers[i].capacity = capacity;
	}
    }
}

bool RedistributeParticles::_adjust_recv_buffers(const int requested_capacities[27])
{
    bool haschanged = false;

    for(int i = 0; i < 27; ++i)
    {
	if (requested_capacities[i] <= unpackbuffers[i].capacity)
	    continue;

	haschanged = true;

	const int capacity = requested_capacities[i];

	if (i)
	{
	    //preserve-resize policy
	    float * const old = pinnedhost_recvbufs[i];

	    CUDA_CHECK(hipHostAlloc(&pinnedhost_recvbufs[i], sizeof(float) * 6 * capacity, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&unpackbuffers[i].buffer, pinnedhost_recvbufs[i], 0));

	    CUDA_CHECK(hipMemcpy(pinnedhost_recvbufs[i], old, sizeof(float) * 6 * unpackbuffers[i].capacity,
				  hipMemcpyHostToHost));

	    CUDA_CHECK(hipHostFree(old));
	}
	else
	{
	    printf("RedistributeParticles::_adjust_recv_buffers i==0 ooooooooooooooops %d , req %d!!\n", unpackbuffers[i].capacity, capacity);
	    abort();
	    //CUDA_CHECK(hipFree(unpackbuffers[i].buffer));
	    //CUDA_CHECK(hipMalloc(&unpackbuffers[i].buffer, sizeof(float) * 6 * capacity));
	    //assert(pinnedhost_recvbufs[i] == NULL);
	}

	unpackbuffers[i].capacity = capacity;
    }

    return haschanged;
}

void RedistributeParticles::pack(const Particle * const particles, const int nparticles, hipStream_t mystream)
{
    NVTX_RANGE("RDP/pack");

    bool secondchance = false;

    if (firstcall)
	_post_recv();

    size_t textureoffset;
    if (nparticles)
    CUDA_CHECK(hipBindTexture(&textureoffset, &RedistributeParticlesKernels::texAllParticles, particles,
			       &RedistributeParticlesKernels::texAllParticles.channelDesc,
			       sizeof(float) * 6 * nparticles));

    if (nparticles)
    CUDA_CHECK(hipBindTexture(&textureoffset, &RedistributeParticlesKernels::texAllParticlesFloat2, particles,
			       &RedistributeParticlesKernels::texAllParticlesFloat2.channelDesc,
			       sizeof(float) * 6 * nparticles));

    RedistributeParticlesKernels::ntexparticles = nparticles;
    RedistributeParticlesKernels::texparticledata = (float2 *)particles;

pack_attempt:

    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::pack_buffers), packbuffers,
					   sizeof(PackBuffer) * 27, 0, hipMemcpyHostToDevice, mystream));

    *failure.data = false;
    RedistributeParticlesKernels::setup<<<1, 32, 0, mystream>>>();

    if (nparticles)
	RedistributeParticlesKernels::scatter_halo_indices_pack<<< (nparticles + 127) / 128, 128, 0, mystream>>>(nparticles);

    RedistributeParticlesKernels::tiny_scan<<<1, 32, 0, mystream>>>(nparticles, packbuffers[0].capacity, packsizes.devptr, failure.devptr);

    CUDA_CHECK(hipEventRecord(evsizes, mystream));

#ifndef NDEBUG
    RedistributeParticlesKernels::check_scan<<<1, 1, 0, mystream>>>();
#endif

    if (nparticles)
	RedistributeParticlesKernels::pack<<< (3 * nparticles + 127) / 128, 128, 0, mystream>>> (nparticles, nparticles * 3);

    CUDA_CHECK(hipEventRecord(evpacking, mystream));

    CUDA_CHECK(hipEventSynchronize(evsizes));

    if (*failure.data)
    {
	//wait for packing to finish
	CUDA_CHECK(hipEventSynchronize(evpacking));

	printf("RedistributeParticles::pack RANK %d ...FAILED! Recovering now...\n", myrank);

	_adjust_send_buffers(packsizes.data);

	if (myrank == 0)
	    for(int i = 0; i < 27; ++i)
		printf("ASD: %d\n", packsizes.data[i]);

	if (secondchance)
	{
	    printf("...non siamo qui a far la ceretta allo yeti.\n");
	    abort();
	}

	if (!secondchance)
	    secondchance = true;

	goto pack_attempt;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

void RedistributeParticles::send()
{
    NVTX_RANGE("RDP/send", NVTX_C2);

    if (!firstcall)
	_waitall(sendcountreq, nactiveneighbors);

    for(int i = 0; i < 27; ++i)
	send_sizes[i] = packsizes.data[i];

    nbulk = recv_sizes[0] = send_sizes[0];

    {
	int c = 0;
	for(int i = 1; i < 27; ++i)
	    if (default_message_sizes[i])
		MPI_CHECK( MPI_Isend(send_sizes + i, 1, MPI_INTEGER, neighbor_ranks[i], basetag + i, cartcomm, sendcountreq + c++) );

	assert(c == nactiveneighbors);
    }

    CUDA_CHECK(hipEventSynchronize(evpacking));

    if (!firstcall)
	_waitall(sendmsgreq, nsendmsgreq);

    nsendmsgreq = 0;
    for(int i = 1; i < 27; ++i)
	if (default_message_sizes[i])
	{
	    MPI_CHECK( MPI_Isend(pinnedhost_sendbufs[i], default_message_sizes[i] * 6, MPI_FLOAT, neighbor_ranks[i], basetag + i + 333,
				 cartcomm, sendmsgreq + nsendmsgreq) );

	    ++nsendmsgreq;
	}

    for(int i = 1; i < 27; ++i)
	if (default_message_sizes[i] && send_sizes[i] > default_message_sizes[i])
	{
	    const int count = send_sizes[i] - default_message_sizes[i];

	    MPI_CHECK( MPI_Isend(pinnedhost_sendbufs[i] + default_message_sizes[i] * 6, count * 6, MPI_FLOAT,
				 neighbor_ranks[i], basetag + i + 666, cartcomm, sendmsgreq + nsendmsgreq) );
	    ++nsendmsgreq;
	}

    assert(nactiveneighbors <= nsendmsgreq && nsendmsgreq <= 2 * nactiveneighbors);
}

void RedistributeParticles::bulk(const int nparticles, int * const cellstarts, int * const cellcounts, hipStream_t mystream)
{
    CUDA_CHECK(hipMemsetAsync(cellcounts, 0, sizeof(int) * XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN, mystream));
/*    CUDA_CHECK(hipPeekAtLastError());
    dim3 bs(8, 8, 8);

    dim3 gs((XSIZE_SUBDOMAIN + bs.x - 1) / bs.x,
	    (YSIZE_SUBDOMAIN + bs.y - 1) / bs.y,
	    (ZSIZE_SUBDOMAIN + bs.z - 1) / bs.z);

    subindices.resize(nparticles);
*/
    subindices.resize(nparticles);

    if (nparticles)
    subindex_local<false><<< (nparticles + 127) / 128, 128, 0, mystream>>>
	(nparticles, RedistributeParticlesKernels::texparticledata, cellcounts, subindices.data);
/*
#ifndef NDEBUG
    CUDA_CHECK(hipDeviceSynchronize());

    {
	const int n =  XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;
	int * c = new int[n];
	hipMemcpy(c, cellcounts, sizeof(int) * n, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; ++i)
	    assert(c[i] == 4);
	delete [] c;

	int * w = new unit4[n];
	hipMemcpy(c, cellcounts, sizeof(int) * n, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; ++i)
	    assert(c[i] == 4);
	delete [] c;

    }
    #endif*/
    //RedistributeParticlesKernels::subindex_local<0><<<gs, bs, 0, mystream>>>(nparticles, cellstarts, cellcounts, subindices.data);
    //RedistributeParticlesKernels::subindex_local<1><<<gs, bs, 0, mystream>>>(nparticles, cellstarts, cellcounts, subindices.data);

    CUDA_CHECK(hipPeekAtLastError());
}

int RedistributeParticles::recv_count(hipStream_t mystream, float& host_idle_time)
{
    CUDA_CHECK(hipPeekAtLastError());

    NVTX_RANGE("RDP/recv-count", NVTX_C3);

    host_idle_time += _waitall(recvcountreq, nactiveneighbors);

    {
	static int usize[27], ustart[28], ustart_padded[28];

	usize[0] = 0;
	for(int i = 1; i < 27; ++i)
	    usize[i] = recv_sizes[i] * (default_message_sizes[i] > 0);

	ustart[0] = 0;
	for(int i = 1; i < 28; ++i)
	    ustart[i] = ustart[i - 1] + usize[i - 1];

	nexpected = nbulk + ustart[27];
	nhalo = ustart[27];

	ustart_padded[0] = 0;
	for(int i = 1; i < 28; ++i)
	    ustart_padded[i] = ustart_padded[i - 1] + 32 * ((usize[i - 1] + 31) / 32);

	nhalo_padded = ustart_padded[27];

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::unpack_start), ustart,
					   sizeof(int) * 28, 0, hipMemcpyHostToDevice, mystream));

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::unpack_start_padded), ustart_padded,
					   sizeof(int) * 28, 0, hipMemcpyHostToDevice, mystream));
    }

    {
	remote_particles.resize(nhalo);
	subindices_remote.resize(nhalo);
	scattered_indices.resize(nexpected);
    }

    firstcall = false;

    return nexpected;
}

void RedistributeParticles::recv_unpack(Particle * const particles, float4 * const xyzouvwo, ushort4 * const xyzo_half, const int nparticles,
					int * const cellstarts, int * const cellcounts, hipStream_t mystream, float& host_idling_time)
{
    NVTX_RANGE("RDP/recv-unpack", NVTX_C4);

    assert(nparticles == nexpected);

    host_idling_time += _waitall(recvmsgreq, nactiveneighbors);

    const bool haschanged = true;
    _adjust_recv_buffers(recv_sizes);

    if (haschanged)
	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::unpack_buffers), unpackbuffers,
					       sizeof(UnpackBuffer) * 27, 0, hipMemcpyHostToDevice, mystream));

    for(int i = 1; i < 27; ++i)
	if (default_message_sizes[i] && recv_sizes[i] > default_message_sizes[i])
	{
	    const int count = recv_sizes[i] - default_message_sizes[i];

	    MPI_Status status;
	    MPI_CHECK( MPI_Recv(pinnedhost_recvbufs[i] + default_message_sizes[i] * 6, count * 6, MPI_FLOAT,
				neighbor_ranks[i], basetag + recv_tags[i] + 666, cartcomm, &status) );
	}

    CUDA_CHECK(hipPeekAtLastError());

#ifndef NDEBUG
    CUDA_CHECK(hipMemset(remote_particles.data, 0xff, sizeof(Particle) * remote_particles.size));
#endif

    if (nhalo)
	RedistributeParticlesKernels::subindex_remote<<< (nhalo_padded + 127) / 128, 128, 0, mystream >>>
	    (nhalo_padded, nhalo, cellcounts, (float2 *)remote_particles.data, subindices_remote.data);

    if (compressed_cellcounts.size)
    compress_counts<<< (compressed_cellcounts.size + 127) / 128, 128, 0, mystream >>>
	(compressed_cellcounts.size, (int4 *)cellcounts, (uchar4 *)compressed_cellcounts.data);

    scan(compressed_cellcounts.data, compressed_cellcounts.size, mystream, (uint *)cellstarts);

#ifndef NDEBUG
    CUDA_CHECK(hipMemset(scattered_indices.data, 0xff, sizeof(int) * scattered_indices.size));
#endif

    if (subindices.size)
    RedistributeParticlesKernels::scatter_indices<<< (subindices.size + 127) / 128, 128, 0, mystream>>>
	(false, subindices.data, subindices.size, cellstarts, scattered_indices.data, scattered_indices.size);

    if (nhalo)
	RedistributeParticlesKernels::scatter_indices<<< (nhalo + 127) / 128, 128, 0, mystream>>>
	    (true, subindices_remote.data, nhalo, cellstarts, scattered_indices.data, scattered_indices.size);

    assert(scattered_indices.size == nparticles);

    if (nparticles)
    RedistributeParticlesKernels::gather_particles<<< (nparticles + 127) / 128, 128, 0, mystream>>>
	(scattered_indices.data, (float2 *)remote_particles.data, nhalo,
	 RedistributeParticlesKernels::ntexparticles, nparticles, (float2 *)particles, xyzouvwo, xyzo_half);

    CUDA_CHECK(hipPeekAtLastError());

#ifndef NDEBUG
    RedistributeParticlesKernels::check<<<(XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN + 127) / 128, 128, 0, mystream>>>(cellstarts, cellcounts, particles, nparticles);
#endif

    _post_recv();

    CUDA_CHECK(hipPeekAtLastError());
}

void RedistributeParticles::_cancel_recv()
{
    if (!firstcall)
    {
	_waitall(sendcountreq, nactiveneighbors);
	_waitall(sendmsgreq, nsendmsgreq);

	for(int i = 0; i < nactiveneighbors; ++i)
	    MPI_CHECK( MPI_Cancel(recvcountreq + i) );

	for(int i = 0; i < nactiveneighbors; ++i)
	    MPI_CHECK( MPI_Cancel(recvmsgreq + i) );

	firstcall = true;
    }
}

void RedistributeParticles::adjust_message_sizes(ExpectedMessageSizes sizes)
{
    _cancel_recv();

    nactiveneighbors = 0;
    for(int i = 1; i < 27; ++i)
    {
	const int d[3] = { (i + 1) % 3, (i / 3 + 1) % 3, (i / 9 + 1) % 3 };
       	const int entry = d[0] + 3 * (d[1] + 3 * d[2]);

	int estimate = (int)ceil(safety_factor * sizes.msgsizes[entry]);
	estimate = 32 * ((estimate + 31) / 32);

	default_message_sizes[i] = estimate;
	nactiveneighbors += (estimate > 0);
    }

    _adjust_send_buffers(default_message_sizes);
    _adjust_recv_buffers(default_message_sizes);
}

RedistributeParticles::~RedistributeParticles()
{
    CUDA_CHECK(hipEventDestroy(evpacking));
    CUDA_CHECK(hipEventDestroy(evsizes));

    _cancel_recv();

    for(int i = 0; i < 27; ++i)
    {
	CUDA_CHECK(hipFree(packbuffers[i].scattered_indices));

	if (i)
	    CUDA_CHECK(hipHostFree(packbuffers[i].buffer));
	else
	    CUDA_CHECK(hipFree(packbuffers[i].buffer));
    }
}
