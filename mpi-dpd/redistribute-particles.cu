#include "hip/hip_runtime.h"
/*
 *  redistribute-particles.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-02-09.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cassert>
#include <vector>
#include <algorithm>

#include "redistribute-particles.h"

#ifndef WARPSIZE
#define WARPSIZE 32
#endif

using namespace std;

namespace RedistributeParticlesKernels
{
    __constant__ RedistributeParticles::PackBuffer pack_buffers[27];
    
    __constant__ RedistributeParticles::UnpackBuffer unpack_buffers[27];
    
    __device__ int pack_count[27], pack_start[28];

    __constant__ int unpack_start[28];

    __device__ bool failed;
    
    texture<float, hipTextureType1D> texAllParticles;
 
    __global__ void setup()
    {
	if (threadIdx.x == 0)
	    failed = false;
	
	if (threadIdx.x < 27)
	    pack_count[threadIdx.x] = 0;
    }
    
    __global__ void scatter_halo_indices(const int np)
    {
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid < np)
	{
	    float xp[3];
	    for(int c = 0; c < 3; ++c)
		xp[c] = tex1Dfetch(texAllParticles, 6 * pid + c);

	    const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	    int vcode[3];
	    for(int c = 0; c < 3; ++c)
		vcode[c] = (2 + (xp[c] >= -L[c]/2) + (xp[c] >= L[c]/2)) % 3;
	
	    const int code = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);
	    assert(code >= 0 && code < 27);

	    if (code > 0)
	    {
		const int entry = atomicAdd(pack_count + code, 1);
		
		if (entry < pack_buffers[code].capacity)
		    pack_buffers[code].scattered_indices[entry] = pid;
	    }
	}
    }

    __global__ void tiny_scan(const int nparticles, const int bulkcapacity, int * const packsizes, bool * const failureflag)
    {
	assert(blockDim.x > 27 && gridDim.x == 1);
	
	const int tid = threadIdx.x;

	int myval = 0, mycount = 0;
	
	if (tid < 27)
	{
	    myval = mycount = pack_count[threadIdx.x];
	    if (tid > 0)
		packsizes[tid] = mycount;
	   
	    if (mycount > pack_buffers[tid].capacity)
	    {
		failed = true;
		*failureflag = true;
	    }
	}

	for(int L = 1; L < 32; L <<= 1)
	    myval += (tid >= L) * __shfl_up(myval, L) ;

	pack_start[tid] = myval - mycount;

	if (tid == 26)
	{
	    pack_start[tid + 1] = myval;
	    
	    const int nbulk = nparticles - myval;
	    packsizes[0] = nbulk;

	    if (nbulk > bulkcapacity)
	    {
		failed = true;
		*failureflag = true;
	    }
	}
    }
    
#ifndef NDEBUG
    __global__ void check_scan()
    {
	assert(blockDim.x == 1 && gridDim.x == 1);

	for(int i = 1; i < 28; ++i)
	    assert(pack_start[i - 1] <= pack_start[i]);
    }
#endif

    __global__ void pack(const int nparticles, const int nfloats)
    {
	assert(blockDim.x * gridDim.x >= nfloats);

	if (failed)
	    return;
	
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	const int slot = gid / 6;

	const int tid = threadIdx.x;
	
	__shared__ int start[28];

	if (tid < 28)
	    start[tid] = pack_start[tid];

	__syncthreads();

	const int key9 = 9 * (slot >= start[9]) + 9 * (slot >= start[18]);
	const int key3 = 3 * (slot >= start[key9 + 3]) + 3 * (slot >= start[key9 + 6]);
	const int key1 = (slot >= start[key9 + key3 + 1]) + (slot >= start[key9 + key3 + 2]);

	const int idpack = key9 + key3 + key1;

	if (slot >= pack_start[27])
	    return;

	const int offset = slot - pack_start[idpack];
	assert (offset >= 0 && offset < pack_buffers[idpack].capacity);
	
	const int pid = pack_buffers[idpack].scattered_indices[offset];
	assert(pid < nparticles && pid >= 0);

	const int c = gid % 6;
	const int d = c + 6 * offset;
	assert (d < pack_buffers[idpack].capacity * 6);	
	   
	pack_buffers[idpack].buffer[d] = tex1Dfetch(texAllParticles, c + 6 * pid);
    }

    __device__ void bitonic_warp(int& key, int& val)
    {
	const int lane = threadIdx.x & (WARPSIZE - 1);

#pragma unroll
	for(int D = 1; D <= 16; D <<= 1)
#pragma unroll
	    for(int L = D; L >= 1; L >>= 1)
	    { 
		const int mask = L == D ? 2 * D - 1 : L;
		
		const int otherkey = __shfl_xor(key, mask);
		const int otherval = __shfl_xor(val, mask);
		
		const bool exchange =  (2 * (int)(lane < (lane ^ mask)) - 1) * (key - otherkey) > 0;
		
		if (exchange)
		{
		    key = otherkey;
		    val = otherval;
		}
	    }
    }

    __device__ int count_warp(int p)
    {
	for(int L = WARPSIZE / 2; L > 0; L >>=1)
	    p += __shfl_xor(p, L);

	return p;
    }
    
    template<int STRIPESIZE, int ILP>
    __global__ void recompact_bulk(const int np)
    {
	assert(STRIPESIZE == blockDim.x);
	assert(WARPSIZE == warpSize);
	assert(STRIPESIZE % WARPSIZE == 0);

	const int tid = threadIdx.x;
	const int lid = threadIdx.x & (WARPSIZE - 1);
	const int gid = threadIdx.x + STRIPESIZE * blockIdx.x;
     
	int tagged = gid >= np;

	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	if (gid < np)
	    for(int c = 0; c < 3; ++c)
	    {
		const float val = tex1Dfetch(texAllParticles, c + 6 * gid);
		tagged += (int)(val < -L[c] / 2 || val >= L[c] / 2);
	    }
			
	__shared__ int global_offset, local_offset, values[STRIPESIZE];

	if (tid == 0)
	    local_offset = 0;	   

	const int ntags = __syncthreads_count(tagged > 0);
	const int nvalid = STRIPESIZE - ntags;

	if (tid == 0)
	    global_offset = atomicAdd(&pack_count[0], nvalid);
	
	if (ntags)
	{
	    int pid = gid;
	    bitonic_warp(tagged, pid);
	    const int ngoodones = count_warp(!tagged);
	    
	    int warp_offset;

	    if (lid == 0)
		warp_offset = atomicAdd(&local_offset, ngoodones);

	    warp_offset = __shfl(warp_offset, 0);
		
	    if (!tagged)
		values[warp_offset + lid] = pid;

	    assert(warp_offset + lid < nvalid && warp_offset + lid >= 0 || tagged);

	    __syncthreads();

	    const int start = 6 * global_offset;
	    const int stop = start + 6 * nvalid;
	    	    
	    for(int dbase = start + tid; dbase < stop; dbase += STRIPESIZE * ILP)
	    {
		float data[ILP];
#pragma unroll
		for(int i = 0; i < ILP; ++i)
		{
		    const int d = dbase + i * STRIPESIZE;
		    const int c = d % 6;
		    const int s =  (d - start) / 6;
		    assert(s >= 0);
		    assert(s < STRIPESIZE || d >= stop);
		    data[i] = d < stop ? tex1Dfetch(texAllParticles, c + 6 * values[s]) : 0;
		}
#pragma unroll
		for(int i = 0; i < ILP; ++i)
		{
		    const int d = dbase + i * STRIPESIZE;
		   
		    assert(d < pack_buffers[0].capacity * 6 || d >= stop);
		    assert(d >= 0);
		    
		    if (d < stop)
			pack_buffers[0].buffer[d] = data[i];
		}
	    }
	}
	else
	{
	    __syncthreads();
	    
	    const int start = 6 * global_offset;
	    const int stop = start + 6 * nvalid;
	    const int srcbase = -start + 6 * STRIPESIZE * blockIdx.x;
	    
	    for(int d = start + tid; d < stop; d += STRIPESIZE * ILP)
	    {
		float data[ILP];

#pragma unroll
		for(int i = 0; i < ILP; ++i)
		{
		    const int s = d + i * STRIPESIZE;
		    data[i] = s < stop ? tex1Dfetch(texAllParticles, srcbase + s) : 0;
		}

#pragma unroll
		for(int i = 0; i < ILP; ++i)
		{
		    const int dest = d + i * STRIPESIZE;
		    if (dest < stop)
		    {
			assert(dest < pack_buffers[0].capacity * 6);
			assert(dest >= 0);
			pack_buffers[0].buffer[dest] = data[i];
		    }
		}
	    }
	}
    }

    __global__ void unpack(float * dstbuf, const int nfloats, const int nparticles, const int base)
    {
	assert(blockDim.x * gridDim.x >= nfloats);
	
	const int gid = threadIdx.x + blockDim.x * blockIdx.x + base;

	if (gid >= nfloats + base)
	    return;
	
	const int slot = gid / 6;
	
	const int key9 = 9 * (slot >= unpack_start[9]) + 9 * (slot >= unpack_start[18]);
	const int key3 = 3 * (slot >= unpack_start[key9 + 3]) + 3 * (slot >= unpack_start[key9 + 6]);
	const int key1 = (slot >= unpack_start[key9 + key3 + 1]) + (slot >= unpack_start[key9 + key3 + 2]);
	const int code = key9 + key3 + key1;
	
	assert(slot >= unpack_start[code] && slot < unpack_start[code + 1]);
	
	const int offset = slot - unpack_start[code];
	assert (offset >= 0 && offset < unpack_buffers[code].capacity);
	
	const int c = gid % 6;
	assert(c >= 0 && c < 6);

	const int s = c + 6 * offset;
	assert (s < unpack_buffers[code].capacity * 6);
	const float value = unpack_buffers[code].buffer[s];
	
	const int shift =
	    XSIZE_SUBDOMAIN * (c == 0) * ((code + 1) % 3 - 1) +
	    YSIZE_SUBDOMAIN * (c == 1) * ((code / 3 + 1) % 3 - 1) +
	    ZSIZE_SUBDOMAIN * (c == 2) * ((code / 9 + 1) % 3 - 1);

	dstbuf[gid] = value + shift;

#ifndef NDEBUG
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	if (!(c >= 3 || fabs(dstbuf[gid]) <= L[c] /2))
		printf("ooooooooops REDISTRIBUTE: code: %d c: %d gid: %d dst value: %f, value : %f  shift : %f\n", 
				code, c, gid, dstbuf[gid], value, (float)shift);
	
	assert(c >= 3 || fabs(dstbuf[gid]) <= L[c] /2);
#endif
    }

#ifndef NDEBUG
    __global__ void check(const Particle * const p, const int np)
    {
	assert(blockDim.x * gridDim.x >= np);

	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };	

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid < np)
	    for(int c = 0; c < 3; ++c)
	    {
		if (!(p[pid].x[c] >= -L[c]/2 && p[pid].x[c] < L[c]/2))
		{
		     printf("oooops pid %d component %d is %f\n", pid, c, p[pid].x[c]);
		}
		
		assert(p[pid].x[c] >= -L[c]/2 && p[pid].x[c] < L[c]/2);
	    }
    }
#endif
}

RedistributeParticles::RedistributeParticles(MPI_Comm _cartcomm): 
failure(1), packsizes(27), nactiveneighbors(26), firstcall(true)
{
    safety_factor = getenv("RDP_COMM_FACTOR") ? atof(getenv("RDP_COMM_FACTOR")) : 1.2;

    MPI_CHECK(MPI_Comm_dup(_cartcomm, &cartcomm) );

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank) );
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 27; ++i)
    {
	const int d[3] = { (i + 1) % 3 - 1, (i / 3 + 1) % 3 - 1, (i / 9 + 1) % 3 - 1 };

	recv_tags[i] = (3 - d[0]) % 3 + 3 * ((3 - d[1]) % 3 + 3 * ((3 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];
		
	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, neighbor_ranks + i) );
	
	const int nhalodir[3] =  { 
		d[0] != 0 ? 1 : XSIZE_SUBDOMAIN, 
		d[1] != 0 ? 1 : YSIZE_SUBDOMAIN, 
		d[2] != 0 ? 1 : ZSIZE_SUBDOMAIN 
	    };

	const int nhalocells = nhalodir[0] * nhalodir[1] * nhalodir[2];
	
	const int estimate = numberdensity * safety_factor * nhalocells;
	
	CUDA_CHECK(hipMalloc(&packbuffers[i].scattered_indices, sizeof(int) * estimate));
	
	if (i)
	{
	    CUDA_CHECK(hipHostAlloc(&pinnedhost_sendbufs[i], sizeof(float) * 6 * estimate, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&packbuffers[i].buffer, pinnedhost_sendbufs[i], 0));

	    CUDA_CHECK(hipHostAlloc(&pinnedhost_recvbufs[i], sizeof(float) * 6 * estimate, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&unpackbuffers[i].buffer, pinnedhost_recvbufs[i], 0));
	}
	else
	{
      	    CUDA_CHECK(hipMalloc(&packbuffers[i].buffer, sizeof(float) * 6 * estimate));
	    unpackbuffers[i].buffer = packbuffers[i].buffer;

	    pinnedhost_sendbufs[i] = NULL;
	    pinnedhost_recvbufs[i] = NULL;
	}
	
	packbuffers[i].capacity = estimate;
	unpackbuffers[i].capacity = estimate;
	default_message_sizes[i] = estimate;
    }
    
    RedistributeParticlesKernels::texAllParticles.channelDesc = hipCreateChannelDesc<float>();
    RedistributeParticlesKernels::texAllParticles.filterMode = hipFilterModePoint;
    RedistributeParticlesKernels::texAllParticles.mipmapFilterMode = hipFilterModePoint;
    RedistributeParticlesKernels::texAllParticles.normalized = 0;

    CUDA_CHECK(hipEventCreate(&evpacking, hipEventDisableTiming));
    CUDA_CHECK(hipEventCreate(&evsizes, hipEventDisableTiming));
    //CUDA_CHECK(hipEventCreate(&evcompaction, hipEventDisableTiming));
}

void RedistributeParticles::_post_recv()
{
    for(int i = 1, c = 0; i < 27; ++i)
    	if (default_message_sizes[i])
	    MPI_CHECK( MPI_Irecv(recv_sizes + i, 1, MPI_INTEGER, neighbor_ranks[i], basetag + recv_tags[i], cartcomm, recvcountreq + c++) );
	else
	    recv_sizes[i] = 0;
 
    for(int i = 1, c = 0; i < 27; ++i)
	if (default_message_sizes[i])
	    MPI_CHECK( MPI_Irecv(pinnedhost_recvbufs[i], default_message_sizes[i] * 6, MPI_FLOAT, 
				 neighbor_ranks[i], basetag + recv_tags[i] + 333, cartcomm, recvmsgreq + c++) );
}

void RedistributeParticles::_adjust_send_buffers(const int requested_capacities[27])
{
    for(int i = 0; i < 27; ++i)
    {
	if (requested_capacities[i] <= packbuffers[i].capacity)
	    continue;

	const int capacity = requested_capacities[i];
	
	CUDA_CHECK(hipFree(packbuffers[i].scattered_indices));
	CUDA_CHECK(hipMalloc(&packbuffers[i].scattered_indices, sizeof(int) * capacity));
	
	if (i)
	{
	    CUDA_CHECK(hipHostFree(pinnedhost_sendbufs[i]));
	   	    
	    CUDA_CHECK(hipHostAlloc(&pinnedhost_sendbufs[i], sizeof(float) * 6 * capacity, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&packbuffers[i].buffer, pinnedhost_sendbufs[i], 0));

	    packbuffers[i].capacity = capacity;
	}
	else
	{
	    CUDA_CHECK(hipFree(packbuffers[i].buffer));
	    
	    CUDA_CHECK(hipMalloc(&packbuffers[i].buffer, sizeof(float) * 6 * capacity));
	    unpackbuffers[i].buffer = packbuffers[i].buffer;
	    
	    assert(pinnedhost_sendbufs[i] == NULL);

	    packbuffers[i].capacity = capacity;
	    unpackbuffers[i].capacity = capacity;
	}
    }
}

void RedistributeParticles::_adjust_recv_buffers(const int requested_capacities[27])
{
    for(int i = 0; i < 27; ++i)
    {
	if (requested_capacities[i] <= unpackbuffers[i].capacity)
	    continue;

	const int capacity = requested_capacities[i];
	
	if (i)
	{
	    //preserve-resize policy
	    float * const old = pinnedhost_recvbufs[i];
	    
	    CUDA_CHECK(hipHostAlloc(&pinnedhost_recvbufs[i], sizeof(float) * 6 * capacity, hipHostMallocMapped));
	    CUDA_CHECK(hipHostGetDevicePointer(&unpackbuffers[i].buffer, pinnedhost_recvbufs[i], 0));

	    CUDA_CHECK(hipMemcpy(pinnedhost_recvbufs[i], old, sizeof(float) * 6 * unpackbuffers[i].capacity,
				  hipMemcpyHostToHost));
	    
	    CUDA_CHECK(hipHostFree(old));
	}
	else
	{
	    printf("RedistributeParticles::_adjust_recv_buffers i==0 ooooooooooooooops %d , req %d!!\n", unpackbuffers[i].capacity, capacity);
	    abort();
	    //CUDA_CHECK(hipFree(unpackbuffers[i].buffer));
	    //CUDA_CHECK(hipMalloc(&unpackbuffers[i].buffer, sizeof(float) * 6 * capacity));
	    //assert(pinnedhost_recvbufs[i] == NULL);
	}
	
	unpackbuffers[i].capacity = capacity;
    }
}

void RedistributeParticles::pack(const Particle * const particles, const int nparticles, hipStream_t mystream)
{
    NVTX_RANGE("RDP/pack");

    bool secondchance = false;
    
    if (firstcall)
	_post_recv();
	
    size_t textureoffset;
    CUDA_CHECK(hipBindTexture(&textureoffset, &RedistributeParticlesKernels::texAllParticles, particles, 
			       &RedistributeParticlesKernels::texAllParticles.channelDesc,
			       sizeof(float) * 6 * nparticles));
pack_attempt:
	
    if (!is_mps_enabled)
	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::pack_buffers), packbuffers,
				       sizeof(PackBuffer) * 27, 0, hipMemcpyHostToDevice, mystream));
    else
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(RedistributeParticlesKernels::pack_buffers), packbuffers,
				       sizeof(PackBuffer) * 27, 0, hipMemcpyHostToDevice));

    *failure.data = false;
    RedistributeParticlesKernels::setup<<<1, 32, 0, mystream>>>();
	
    if (nparticles)
	RedistributeParticlesKernels::scatter_halo_indices<<< (nparticles + 127) / 128, 128, 0, mystream>>>(nparticles);
	
    RedistributeParticlesKernels::tiny_scan<<<1, 32, 0, mystream>>>(nparticles, packbuffers[0].capacity, packsizes.devptr, failure.devptr);

    CUDA_CHECK(hipEventRecord(evsizes, mystream));
	
#ifndef NDEBUG
    RedistributeParticlesKernels::check_scan<<<1, 1, 0, mystream>>>();
#endif 
	
    if (nparticles)
	RedistributeParticlesKernels::pack<<< (6 * nparticles + 127) / 128, 128, 0, mystream>>> (nparticles, nparticles * 6);
	
    CUDA_CHECK(hipEventRecord(evpacking, mystream));

    CUDA_CHECK(hipEventSynchronize(evsizes));

    if (*failure.data)
    {
	//wait for packing to finish
	CUDA_CHECK(hipEventSynchronize(evpacking));
	    
	printf("RedistributeParticles::pack RANK %d ...FAILED! Recovering now...\n", myrank);

	_adjust_send_buffers(packsizes.data);
	    	    
	if (myrank == 0)
	for(int i = 0; i < 27; ++i)
	    printf("ASD: %d\n", packsizes.data[i]);

	if (secondchance)
	{
	    printf("CAZZ...cioe' va bene tutto ragassi ma non siamo qui a far la ceretta allo yeti.\n");
	    abort();
	}

	if (!secondchance)
	    secondchance = true;

	goto pack_attempt;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

void RedistributeParticles::bulk(const int nparticles, hipStream_t mystream)
{
    NVTX_RANGE("RDP/bulk", NVTX_C2);

    enum { BS = 128, ILP = 2 };
	
    if (nparticles)
	RedistributeParticlesKernels::recompact_bulk<BS, ILP><<< (nparticles + BS - 1) / BS, BS, 0, mystream>>>(nparticles);
}

void RedistributeParticles::send()
{
    NVTX_RANGE("RDP/send", NVTX_C2);

    if (!firstcall)
	_waitall(sendcountreq, nactiveneighbors);
	
    for(int i = 0; i < 27; ++i)
	send_sizes[i] = packsizes.data[i];

    nbulk = recv_sizes[0] = send_sizes[0];
        
    {
	int c = 0;
	for(int i = 1; i < 27; ++i)
	    if (default_message_sizes[i])
		MPI_CHECK( MPI_Isend(send_sizes + i, 1, MPI_INTEGER, neighbor_ranks[i], basetag + i, cartcomm, sendcountreq + c++) );
		
	assert(c == nactiveneighbors);
    }

    CUDA_CHECK(hipEventSynchronize(evpacking));
    
    if (!firstcall)
	_waitall(sendmsgreq, nsendmsgreq);
    
    nsendmsgreq = 0;
    for(int i = 1; i < 27; ++i)
	if (default_message_sizes[i])
	{
	    MPI_CHECK( MPI_Isend(pinnedhost_sendbufs[i], default_message_sizes[i] * 6, MPI_FLOAT, neighbor_ranks[i], basetag + i + 333,
				 cartcomm, sendmsgreq + nsendmsgreq) );
		
	    ++nsendmsgreq;
	}
	
    for(int i = 1; i < 27; ++i)
	if (default_message_sizes[i] && send_sizes[i] > default_message_sizes[i])
	{
	    const int count = send_sizes[i] - default_message_sizes[i];
		
	    MPI_CHECK( MPI_Isend(pinnedhost_sendbufs[i] + default_message_sizes[i] * 6, count * 6, MPI_FLOAT,
				 neighbor_ranks[i], basetag + i + 666, cartcomm, sendmsgreq + nsendmsgreq) );
	    ++nsendmsgreq;
	}
	
    assert(nactiveneighbors <= nsendmsgreq && nsendmsgreq <= 2 * nactiveneighbors);
}

int RedistributeParticles::recv_count(hipStream_t mystream, float& host_idle_time)
{
    CUDA_CHECK(hipPeekAtLastError());  

    NVTX_RANGE("RDP/recv-count", NVTX_C3);

    host_idle_time += _waitall(recvcountreq, nactiveneighbors);

    {
	static int ustart[28];
	    
	ustart[0] = 0;	
	for(int i = 1; i < 28; ++i)
	    ustart[i] = ustart[i - 1] + recv_sizes[i - 1] * (default_message_sizes[i - 1] > 0);
	    
	if (!is_mps_enabled)
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::unpack_start), ustart,
					   sizeof(int) * 28, 0, hipMemcpyHostToDevice, mystream));
	else
	    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(RedistributeParticlesKernels::unpack_start), ustart,
					   sizeof(int) * 28, 0, hipMemcpyHostToDevice));
    }

    nexpected = 0;
    for(int i = 0; i < 27; ++i)
	nexpected += recv_sizes[i];
    
    nhalo = nexpected - nbulk;
    
//CUDA_CHECK(hipEventSynchronize(evcompaction));
    
    firstcall = false;
    
    return nexpected;
}
    
void RedistributeParticles::recv_unpack(Particle * const particles, const int nparticles, hipStream_t mystream, float& host_idle_time)
{
    NVTX_RANGE("RDP/recv-unpack", NVTX_C4);
    
    assert(nparticles == nexpected);
    
    host_idle_time += _waitall(recvmsgreq, nactiveneighbors);
    
    _adjust_recv_buffers(recv_sizes);

    if (!is_mps_enabled)
	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(RedistributeParticlesKernels::unpack_buffers), unpackbuffers,
				       sizeof(UnpackBuffer) * 27, 0, hipMemcpyHostToDevice, mystream));
    else
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(RedistributeParticlesKernels::unpack_buffers), unpackbuffers,
				       sizeof(UnpackBuffer) * 27, 0, hipMemcpyHostToDevice));
    
    for(int i = 1; i < 27; ++i)
	if (default_message_sizes[i] && recv_sizes[i] > default_message_sizes[i])
	{
	    const int count = recv_sizes[i] - default_message_sizes[i];
	    
	    MPI_Status status;
	    MPI_CHECK( MPI_Recv(pinnedhost_recvbufs[i] + default_message_sizes[i] * 6, count * 6, MPI_FLOAT,
				neighbor_ranks[i], basetag + recv_tags[i] + 666, cartcomm, &status) );
	}

    CUDA_CHECK(hipMemcpyAsync(particles, packbuffers[0].buffer, sizeof(Particle) * nbulk, hipMemcpyDeviceToDevice, mystream));

    if (nhalo)
	RedistributeParticlesKernels::unpack<<<(nhalo * 6 + 127) / 128, 128, 0, mystream>>>((float *)particles, nhalo * 6,
										 nhalo, nbulk * 6);	
    
#ifndef NDEBUG
    RedistributeParticlesKernels::check<<<(nparticles + 127) / 128, 128, 0, mystream>>>(particles, nparticles);
#endif
    
    _post_recv();
    
    CUDA_CHECK(hipPeekAtLastError());
}

void RedistributeParticles::_cancel_recv()
{
    if (!firstcall)
    {
	_waitall(sendcountreq, nactiveneighbors);
	_waitall(sendmsgreq, nsendmsgreq);

	for(int i = 0; i < nactiveneighbors; ++i)
	    MPI_CHECK( MPI_Cancel(recvcountreq + i) );
    
	for(int i = 0; i < nactiveneighbors; ++i)
	    MPI_CHECK( MPI_Cancel(recvmsgreq + i) );

	firstcall = true;
    }
}

void RedistributeParticles::adjust_message_sizes(ExpectedMessageSizes sizes)
{
    _cancel_recv();

    nactiveneighbors = 0;
    for(int i = 1; i < 27; ++i)
    {
	const int d[3] = { (i + 1) % 3, (i / 3 + 1) % 3, (i / 9 + 1) % 3 };
       	const int entry = d[0] + 3 * (d[1] + 3 * d[2]);

	int estimate = (int)ceil(safety_factor * sizes.msgsizes[entry]);
	estimate = 32 * ((estimate + 31) / 32);

	default_message_sizes[i] = estimate;
	nactiveneighbors += (estimate > 0);
    }

    _adjust_send_buffers(default_message_sizes);
    _adjust_recv_buffers(default_message_sizes);
}

RedistributeParticles::~RedistributeParticles()
{
    CUDA_CHECK(hipEventDestroy(evpacking));
    CUDA_CHECK(hipEventDestroy(evsizes));
      
    _cancel_recv();
    
    for(int i = 0; i < 27; ++i)
    {
	CUDA_CHECK(hipFree(packbuffers[i].scattered_indices));

	if (i)
	    CUDA_CHECK(hipHostFree(packbuffers[i].buffer));
	else
	    CUDA_CHECK(hipFree(packbuffers[i].buffer));
    }
}

