#include "hip/hip_runtime.h"
/*
 * contact.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-02.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

static const int maxsolutes = 32;
static const float ljsigma = 0.5;
static const float ljsigma2 = ljsigma * ljsigma;

#include <../dpd-rng.h>

#include "common-kernels.h"
#include "scan.h"
#include "contact.h"

namespace KernelsContact
{
    enum
    {
	XCELLS = XSIZE_SUBDOMAIN,
	YCELLS = YSIZE_SUBDOMAIN,
	ZCELLS = ZSIZE_SUBDOMAIN,
	XOFFSET = XCELLS / 2,
	YOFFSET = YCELLS / 2,
	ZOFFSET = ZCELLS / 2
    };

    static const int NCELLS = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

    union CellEntry { int pid; uchar4 code; };

    struct Params { float gamma, sigmaf, rc2; };

    __constant__ Params params;

    texture<int, hipTextureType1D> texCellsStart, texCellEntries;

    void setup()
    {
	texCellsStart.channelDesc = hipCreateChannelDesc<int>();
	texCellsStart.filterMode = hipFilterModePoint;
	texCellsStart.mipmapFilterMode = hipFilterModePoint;
	texCellsStart.normalized = 0;

	texCellEntries.channelDesc = hipCreateChannelDesc<int>();
	texCellEntries.filterMode = hipFilterModePoint;
	texCellEntries.mipmapFilterMode = hipFilterModePoint;
	texCellEntries.normalized = 0;
    }
}

ComputeContact::ComputeContact(MPI_Comm comm):
cellsstart(KernelsContact::NCELLS + 16), cellscount(KernelsContact::NCELLS + 16), compressed_cellscount(KernelsContact::NCELLS + 16)
{
    int myrank;
    MPI_CHECK( MPI_Comm_rank(comm, &myrank));

    local_trunk = Logistic::KISS(7119 - myrank, 187 + myrank, 18278, 15674);

    KernelsContact::Params params = { gammadpd, sigmaf, 1};

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelsContact::params), &params, sizeof(params)));

    CUDA_CHECK(hipPeekAtLastError());
}

namespace KernelsContact
{
     __global__ void populate(const uchar4 * const subindices, const int * const cellstart,
			     const int nparticles, const int soluteid, const int ntotalparticles,
			     CellEntry * const entrycells)
    {
#if !defined(__CUDA_ARCH__)
#warning __CUDA_ARCH__ not defined! assuming 350
#define _ACCESS(x) __ldg(x)
#elif __CUDA_ARCH__ >= 350
#define _ACCESS(x) __ldg(x)
#else
#define _ACCESS(x) (*(x))
#endif

	assert(blockDim.x == 128);

	const int warpid = threadIdx.x >> 5;
	const int tid = threadIdx.x & 0x1f;

	const int base = 32 * (warpid + 4 * blockIdx.x);
	const int pid = base + tid;

	if (pid >= nparticles)
	    return;

	const uchar4 subindex = subindices[pid];

	if (subindex.x == 0xff && subindex.y == 0xff && subindex.z == 0xff)
	    return;

	assert(subindex.x < XCELLS && subindex.y < YCELLS && subindex.z < ZCELLS);

	const int cellid = subindex.x + XCELLS * (subindex.y + YCELLS * subindex.z);
	const int mystart = _ACCESS(cellstart + cellid);
	const int slot = mystart + subindex.w;
	assert(slot < ntotalparticles);

	CellEntry myentrycell;
	myentrycell.pid = pid;
	myentrycell.code.w = soluteid;

	entrycells[slot] = myentrycell;
    }

    __constant__ int cnsolutes[maxsolutes];
    __constant__ const float2 * csolutes[maxsolutes];
    __constant__ float * csolutesacc[maxsolutes];

    void bind(const int * const cellsstart, const int * const cellentries, const int ncellentries,
	      std::vector<ParticlesWrap> wsolutes, hipStream_t stream, const int * const cellscount)
    {
	size_t textureoffset = 0;

	if (ncellentries)
	    CUDA_CHECK(hipBindTexture(&textureoffset, &texCellEntries, cellentries, &texCellEntries.channelDesc,
				       sizeof(int) * ncellentries));

	assert(textureoffset == 0);

	const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

	CUDA_CHECK(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * (1 + ncells)));
	assert(textureoffset == 0);

	const int n = wsolutes.size();

	int ns[n];
	float2 * ps[n];
	float * as[n];

	for(int i = 0; i < n; ++i)
	{
	    ns[i] = wsolutes[i].n;
	    ps[i] = (float2 *)wsolutes[i].p;
	    as[i] = (float * )wsolutes[i].a;
	}

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cnsolutes), ns, sizeof(int) * n, 0, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csolutes), ps, sizeof(float2 *) * n, 0, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csolutesacc), as, sizeof(float *) * n, 0, hipMemcpyHostToDevice, stream));
    }

    __global__ void bulk_3tpp(const int nsolutes, const float seed)
    {
	const int np = tex1Dfetch(texCellsStart, XCELLS * YCELLS * ZCELLS);

	assert(blockDim.x * gridDim.x >= np * 3);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
       	const int myslot = gid / 3;
	const int zplane = gid % 3;

	if (myslot >= np)
	    return;

	float2 dst0, dst1, dst2;
	int soluteid, actualpid;

	{
	    CellEntry ce;
	    ce.pid = tex1Dfetch(texCellEntries, myslot);

	    soluteid = ce.code.w;

	    ce.code.w = 0;
	    actualpid = ce.pid;

	    assert(soluteid < nsolutes);
	    assert(actualpid >= 0 && actualpid < cnsolutes[soluteid]);

	    dst0 = _ACCESS(csolutes[soluteid] + 3 * actualpid + 0);
	    dst1 = _ACCESS(csolutes[soluteid] + 3 * actualpid + 1);
	    dst2 = _ACCESS(csolutes[soluteid] + 3 * actualpid + 2);

	    assert(dst0.x >= -XOFFSET && dst0.x < XOFFSET);
	    assert(dst0.y >= -YOFFSET && dst0.y < YOFFSET);
	    assert(dst1.x >= -ZOFFSET && dst1.x < ZOFFSET);
	}

	int scan1, scan2, ncandidates, spidbase;
	int deltaspid1, deltaspid2;

	{
	    const int xcenter = min(XCELLS - 1, max(0, XOFFSET + (int)floorf(dst0.x)));
	    const int xstart = max(0, xcenter - 1);
	    const int xcount = min(XCELLS, xcenter + 2) - xstart;

	    if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
		return;

	    assert(xcount >= 0);

	    const int ycenter = min(YCELLS - 1, max(0, YOFFSET + (int)floorf(dst0.y)));

	    const int zcenter = min(ZCELLS - 1, max(0, ZOFFSET + (int)floorf(dst1.x)));
	    const int zmy = zcenter - 1 + zplane;
	    const bool zvalid = zmy >= 0 && zmy < ZCELLS;

	    int count0 = 0, count1 = 0, count2 = 0;

	    if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
	    {
		const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
		assert(cid0 >= 0 && cid0 + xcount <= NCELLS);
		spidbase = tex1Dfetch(texCellsStart, cid0);
		count0 = tex1Dfetch(texCellsStart, cid0 + xcount) - spidbase;
	    }

	    if (zvalid && ycenter >= 0 && ycenter < YCELLS)
	    {
		const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
		assert(cid1 >= 0 && cid1 + xcount <= NCELLS);
		deltaspid1 = tex1Dfetch(texCellsStart, cid1);
		count1 = tex1Dfetch(texCellsStart, cid1 + xcount) - deltaspid1;
	    }

	    if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
	    {
		const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
		deltaspid2 = tex1Dfetch(texCellsStart, cid2);
		assert(cid2 >= 0 && cid2 + xcount <= NCELLS);
		count2 = tex1Dfetch(texCellsStart, cid2 + xcount) - deltaspid2;
	    }

	    scan1 = count0;
	    scan2 = count0 + count1;
	    ncandidates = scan2 + count2;

	    deltaspid1 -= scan1;
	    deltaspid2 -= scan2;
	}

	float xforce = 0, yforce = 0, zforce = 0;

	for(int i = 0; i < ncandidates; ++i)
	{
	    const int m1 = (int)(i >= scan1);
	    const int m2 = (int)(i >= scan2);
	    const int slot = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);
	    assert(slot >= 0 && slot < np);

	    if (slot >= myslot)
		continue;

	    CellEntry ce;
	    ce.pid = tex1Dfetch(texCellEntries, slot);
	    const int soluteid = ce.code.w;

	    assert(soluteid >= 0 && soluteid < nsolutes);
	    ce.code.w = 0;

	    const int spid = ce.pid;
	    assert(spid >= 0 && spid < cnsolutes[soluteid]);

	    const int sentry = 3 * spid;
	    const float2 stmp0 = _ACCESS(csolutes[soluteid] +  sentry    );
	    const float2 stmp1 = _ACCESS(csolutes[soluteid] +  sentry + 1);
	    const float2 stmp2 = _ACCESS(csolutes[soluteid] +  sentry + 2);

	    const float _xr = dst0.x - stmp0.x;
	    const float _yr = dst0.y - stmp0.y;
	    const float _zr = dst1.x - stmp1.x;

	    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
	    assert(rij2 > 0);

	    const float invrij = rsqrtf(rij2);

	    const float rij = rij2 * invrij;

	    if (rij2 >= params.rc2)
		continue;

	    const float invr2 = invrij * invrij;
	    const float t2 = ljsigma2 * invr2;
	    const float t4 = t2 * t2;
	    const float t6 = t4 * t2;
	    const float lj = min(1e4f, max(0.f, 24.f * invrij * t6 * (2.f * t6 - 1.f)));

	    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(1.f - rij);

	    const float xr = _xr * invrij;
	    const float yr = _yr * invrij;
	    const float zr = _zr * invrij;

	    const float rdotv =
		xr * (dst1.y - stmp1.y) +
		yr * (dst2.x - stmp2.x) +
		zr * (dst2.y - stmp2.y);

	    const float myrandnr = Logistic::mean0var1(seed, myslot, slot);

	    const float strength = lj + (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

	    const float xinteraction = strength * xr;
	    const float yinteraction = strength * yr;
	    const float zinteraction = strength * zr;

	    xforce += xinteraction;
	    yforce += yinteraction;
	    zforce += zinteraction;

	    assert(!isnan(xinteraction));
	    assert(!isnan(yinteraction));
	    assert(!isnan(zinteraction));

	    assert(fabs(xinteraction) < 1e4);
	    assert(fabs(yinteraction) < 1e4);
	    assert(fabs(zinteraction) < 1e4);

	    atomicAdd(csolutesacc[soluteid] + sentry    , -xinteraction);
	    atomicAdd(csolutesacc[soluteid] + sentry + 1, -yinteraction);
	    atomicAdd(csolutesacc[soluteid] + sentry + 2, -zinteraction);
	}

	const float xacc = atomicAdd(csolutesacc[soluteid] + 3 * actualpid + 0, xforce);
	const float yacc = atomicAdd(csolutesacc[soluteid] + 3 * actualpid + 1, yforce);
	const float zacc = atomicAdd(csolutesacc[soluteid] + 3 * actualpid + 2, zforce);

	assert(!isnan(xacc));
	assert(!isnan(yacc));
	assert(!isnan(zacc));
    }

    __global__ void halo(const float2 * halo, const int nhalo, const int nsolutes, const float seed, float * const acc)
    {
	const int nbulk = tex1Dfetch(texCellsStart, XCELLS * YCELLS * ZCELLS);

	assert(blockDim.x * gridDim.x >= nhalo);

	const int laneid = threadIdx.x & 0x1f;
	const int warpid = threadIdx.x >> 5;
	const int unpackbase = 32 * (warpid + 4 * blockIdx.x);
	const int nunpack =  min(32, nhalo - unpackbase);

	float2 dst0, dst1, dst2;
	read_AOS6f((float2 *)(halo + 3 * unpackbase), nunpack, dst0, dst1, dst2);

	float xforce, yforce, zforce;
	read_AOS3f(acc + 3 * unpackbase, nunpack, xforce, yforce, zforce);

	const bool outside_plus =
	    dst0.x >= XOFFSET ||
	    dst0.x >= -XOFFSET && dst0.y >= YOFFSET ||
	    dst0.x >= -XOFFSET && dst0.y >= -YOFFSET && dst1.x >= ZOFFSET;

	const bool inside_outerhalo =
	    dst0.x < XOFFSET + 1 &&
	    dst0.y < YOFFSET + 1 &&
	    dst1.x < ZOFFSET + 1 ;

	const bool valid = laneid < nunpack && outside_plus && inside_outerhalo;

	if (!valid)
	    return;

	for(int zplane = 0; zplane < 3; ++zplane)
	{
	    int scan1, scan2, ncandidates, spidbase;
	    int deltaspid1, deltaspid2;

	    {
       		const int xcenter = XOFFSET + (int)floorf(dst0.x);
		const int xstart = max(0, xcenter - 1);
		const int xcount = min(XCELLS, xcenter + 2) - xstart;

		if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
		    continue;

		assert(xcount >= 0);

		const int ycenter = YOFFSET + (int)floorf(dst0.y);
		const int zcenter = ZOFFSET + (int)floorf(dst1.x);

		const int zmy = zcenter - 1 + zplane;
		const bool zvalid = zmy >= 0 && zmy < ZCELLS;

		int count0 = 0, count1 = 0, count2 = 0;

		if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
		{
		    const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
		    assert(cid0 >= 0 && cid0 + xcount <= NCELLS);
		    spidbase = tex1Dfetch(texCellsStart, cid0);
		    count0 = tex1Dfetch(texCellsStart, cid0 + xcount) - spidbase;
		}

		if (zvalid && ycenter >= 0 && ycenter < YCELLS)
		{
		    const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
		    assert(cid1 >= 0 && cid1 + xcount <= NCELLS);
		    deltaspid1 = tex1Dfetch(texCellsStart, cid1);
		    count1 = tex1Dfetch(texCellsStart, cid1 + xcount) - deltaspid1;
		}

		if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
		{
		    const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
		    deltaspid2 = tex1Dfetch(texCellsStart, cid2);
		    assert(cid2 >= 0 && cid2 + xcount <= NCELLS);
		    count2 = tex1Dfetch(texCellsStart, cid2 + xcount) - deltaspid2;
		}

		scan1 = count0;
		scan2 = count0 + count1;
		ncandidates = scan2 + count2;

		deltaspid1 -= scan1;
		deltaspid2 -= scan2;
	    }

	    for(int i = 0; i < ncandidates; ++i)
	    {
		const int m1 = (int)(i >= scan1);
		const int m2 = (int)(i >= scan2);
		const int slot = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);

		assert(slot >= 0 && slot < nbulk);
		CellEntry ce;
		ce.pid = tex1Dfetch(texCellEntries, slot);
		const int soluteid = ce.code.w;
		assert(soluteid >= 0 && soluteid < nsolutes);
		ce.code.w = 0;

		const int spid = ce.pid;
		assert(spid >= 0 && spid < cnsolutes[soluteid]);

		const int sentry = 3 * spid;
		const float2 stmp0 = _ACCESS(csolutes[soluteid] + sentry    );
		const float2 stmp1 = _ACCESS(csolutes[soluteid] + sentry + 1);
		const float2 stmp2 = _ACCESS(csolutes[soluteid] + sentry + 2);

		const float _xr = dst0.x - stmp0.x;
		const float _yr = dst0.y - stmp0.y;
		const float _zr = dst1.x - stmp1.x;

		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		assert(rij2 > 0);

		const float invrij = rsqrtf(rij2);

		const float rij = rij2 * invrij;

		if (rij2 >= params.rc2)
		    continue;

		const float invr2 = invrij * invrij;
		const float t2 = ljsigma2 * invr2;
		const float t4 = t2 * t2;
		const float t6 = t4 * t2;
		const float lj = min(1e4f, max(0.f, 24.f * invrij * t6 * (2.f * t6 - 1.f)));

		const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(1.f - rij);

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv =
		    xr * (dst1.y - stmp1.y) +
		    yr * (dst2.x - stmp2.x) +
		    zr * (dst2.y - stmp2.y);

		const float myrandnr = Logistic::mean0var1(seed, unpackbase + laneid, spid);

		const float strength = lj + (- params.gamma * wr * rdotv + params.sigmaf * myrandnr) * wr;

		const float xinteraction = strength * xr;
		const float yinteraction = strength * yr;
		const float zinteraction = strength * zr;

		xforce += xinteraction;
		yforce += yinteraction;
		zforce += zinteraction;

		assert(!isnan(xinteraction));
		assert(!isnan(yinteraction));
		assert(!isnan(zinteraction));

		assert(fabs(xinteraction) < 1e4);
		assert(fabs(yinteraction) < 1e4);
		assert(fabs(zinteraction) < 1e4);

		atomicAdd(csolutesacc[soluteid] + sentry    , -xinteraction);
		atomicAdd(csolutesacc[soluteid] + sentry + 1, -yinteraction);
		atomicAdd(csolutesacc[soluteid] + sentry + 2, -zinteraction);
	    }
	}

	//write_AOS3f(acc + unpackbase, nunpack, xforce, yforce, zforce);
	//if (valid)
	{
	    assert(valid);
	    acc[3 * (unpackbase + laneid) + 0] = xforce;
	    acc[3 * (unpackbase + laneid) + 1] = yforce;
	    acc[3 * (unpackbase + laneid) + 2] = zforce;
	}
    }
}

void ComputeContact::halo(ParticlesWrap halos[26], hipStream_t stream)
{
    NVTX_RANGE("Contact/halo", NVTX_C7);

    //collate halos
    {
	int c = 0;
	for(int i = 0; i < 26; ++i)
	    c += halos[i].n;

	allhalos.resize(c);
	allhalosacc.resize(c);

	c = 0;
	for(int i = 0; i < 26; ++i)
	{
	    CUDA_CHECK(hipMemcpyAsync(allhalos.data + c, halos[i].p, sizeof(Particle) * halos[i].n, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(allhalosacc.data + c, halos[i].a, sizeof(Acceleration) * halos[i].n, hipMemcpyHostToDevice, stream));

	    c += halos[i].n;
	}
    }

    CUDA_CHECK(hipPeekAtLastError());

    ParticlesWrap halowrap(allhalos.data, allhalos.size, allhalosacc.data);

    wsolutes.push_back(halowrap);

    int ntotal = 0;

    for(int i = 0; i < wsolutes.size(); ++i)
	ntotal += wsolutes[i].n;

    subindices.resize(ntotal);
    cellsentries.resize(ntotal);

    CUDA_CHECK(hipMemsetAsync(cellscount.data, 0, sizeof(int) * cellscount.size, stream));

#ifndef NDEBUG
    CUDA_CHECK(hipMemsetAsync(cellsentries.data, 0xff, sizeof(int) * cellsentries.capacity, stream));
    CUDA_CHECK(hipMemsetAsync(subindices.data, 0xff, sizeof(int) * subindices.capacity, stream));
    CUDA_CHECK(hipMemsetAsync(compressed_cellscount.data, 0xff, sizeof(unsigned char) * compressed_cellscount.capacity, stream));
    CUDA_CHECK(hipMemsetAsync(cellsstart.data, 0xff, sizeof(int) * cellsstart.capacity, stream));
#endif

    CUDA_CHECK(hipPeekAtLastError());

    int ctr = 0;
    for(int i = 0; i < wsolutes.size(); ++i)
    {
	const ParticlesWrap it = wsolutes[i];

	if (it.n)
	    subindex_local<<< (it.n + 127) / 128, 128, 0, stream >>>
		(it.n, (float2 *)it.p, cellscount.data, subindices.data + ctr);

	ctr += it.n;
    }

    compress_counts<<< (compressed_cellscount.size + 127) / 128, 128, 0, stream >>>
	(compressed_cellscount.size, (int4 *)cellscount.data, (uchar4 *)compressed_cellscount.data);

    scan(compressed_cellscount.data, compressed_cellscount.size, stream, (uint *)cellsstart.data);

    ctr = 0;
    for(int i = 0; i < wsolutes.size(); ++i)
    {
	const ParticlesWrap it = wsolutes[i];

	if (it.n)
	    KernelsContact::populate<<< (it.n + 127) / 128, 128, 0, stream >>>
		(subindices.data + ctr, cellsstart.data, it.n, i, ntotal, (KernelsContact::CellEntry *)cellsentries.data);

	ctr += it.n;
    }

    CUDA_CHECK(hipPeekAtLastError());

    KernelsContact::bind(cellsstart.data, cellsentries.data, ntotal, wsolutes, stream, cellscount.data);

    KernelsContact::bulk_3tpp<<< (3 * cellsentries.size + 127) / 128, 128, 0, stream >>>
     	(wsolutes.size(), local_trunk.get_float());

    ctr = 0;
    for(int i = 0; i < wsolutes.size(); ++i)
    {
	const ParticlesWrap it = wsolutes[i];

	if (it.n)
	    KernelsContact::halo<<< (it.n + 127) / 128, 128, 0, stream>>>
		((float2 *)it.p, it.n, wsolutes.size(), local_trunk.get_float(), (float *)it.a);


	ctr += it.n;
    }

    //split back halos
    {
    	int c = 0;
	for(int i = 0; i < 26; ++i)
	{
	    CUDA_CHECK(hipMemcpyAsync(halos[i].a, allhalosacc.data + c, sizeof(Acceleration) * halos[i].n, hipMemcpyDeviceToHost, stream));
	    c += halos[i].n;
	}
    }

    CUDA_CHECK(hipPeekAtLastError());
}
