#include "hip/hip_runtime.h"
#include "halo-exchanger.h"

#include "dpd-interactions.h"

using namespace std;

HaloExchanger::HaloExchanger(MPI_Comm cartcomm, int L):
    cartcomm(cartcomm), L(L), pending_send(false), recv_bag(NULL), send_bag(NULL), recv_bag_size(0), scattered_entries(NULL)
{
    assert(L % 2 == 0);
    assert(L >= 2);
	
    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));
	
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );
	
    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = tagbase_dpd_remote_interactions + (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));
	    
	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];
	    
	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );
    }

    CUDA_CHECK(hipHostAlloc((void **)&sendpacks_start, sizeof(int) * 27, hipHostMallocMapped));
    CUDA_CHECK(hipHostGetDevicePointer(&sendpacks_start_host, sendpacks_start, 0));

    CUDA_CHECK(hipHostAlloc((void **)&send_bag_size_required, sizeof(int), hipHostMallocMapped));
    CUDA_CHECK(hipHostGetDevicePointer(&send_bag_size_required_host, send_bag_size_required, 0));

    send_bag_size = L * L * 3 * 27;
    CUDA_CHECK(hipMalloc(&send_bag, send_bag_size * sizeof(Particle)));
    CUDA_CHECK(hipMalloc(&scattered_entries, send_bag_size * sizeof(int)));
    
    recv_bag_size = L * L * 3 * 27;
    CUDA_CHECK(hipMalloc(&recv_bag, recv_bag_size * sizeof(Particle)));
}

namespace PackingHalo
{
    __device__ int blockcount, global_histo[27], requiredsize;

    __global__ void setup(bool firsttime)
    {
	blockcount = 0;

	for(int i = 0; i < 27; ++i)
	    global_histo[i] = 0;
    }

    template< int work >
    __global__ void count(int * const packs_start, const Particle * const p, const int np, const int L, int * bag_size_required)
    {
	assert(blockDim.x * gridDim.x * work >= np);
	assert(blockDim.x >= 26);
    
	__shared__ int histo[26];

	const int tid = threadIdx.x; 

	if (tid < 26)
	    histo[tid] = 0;

	__syncthreads();

	for(int t = 0; t < work; ++t)
	{
	    const int pid = tid + blockDim.x * (blockIdx.x + gridDim.x * t);

	    if (pid < np)
		for(int i = 0; i < 26; ++i)
		{
		    int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

		    bool halo = true;			
	
		    for(int c = 0; c < 3; ++c)
		    {
			const float halo_start = max(d[c] * L - L/2 - 1, -L/2);
			const float halo_end = min(d[c] * L + L/2 + 1, L/2);
		
			const float x = p[pid].x[c];
		
			halo &= (x >= halo_start && x < halo_end);
		    }

		    if (halo)
			atomicAdd(histo + i, 1);
		}
	}

	__syncthreads();
    
	if (tid < 26 && histo[tid] > 0)
	    atomicAdd(global_histo + tid, histo[tid]);

	if (tid == 0)
	{
	    const int timestamp = atomicAdd(&blockcount, 1);

	    if (timestamp == gridDim.x - 1)
	    {
		blockcount = 0;

		int s = 0, curr;

		for(int i = 0; i < 26; ++i)
		{
		    curr = global_histo[i];
		    global_histo[i] = packs_start[i] = s;
		    s += curr;
		}

		global_histo[26] = packs_start[26] = s;
		requiredsize = s;		
		*bag_size_required = s;
	    }
	}
    }

    __global__ void pack(const Particle * const particles, int np, const int L, Particle * const bag, const int bagsize,
			 int * const scattered_entries)
    {
	if (bagsize < requiredsize)
	    return;
	    
	assert(blockDim.x * gridDim.x >= np);
	assert(blockDim.x >= 26);
    
	__shared__ int histo[26];
	__shared__ int base[26];

	const int tid = threadIdx.x; 

	if (tid < 26)
	    histo[tid] = 0;

	__syncthreads();

	int offset[26];
	for(int i = 0; i < 26; ++i)
	    offset[i] = -1;

	Particle p;
    
	const int pid = tid + blockDim.x * blockIdx.x;

	if (pid < np)
	{
	    p = particles[pid];

	    for(int c = 0; c < 3; ++c)
		assert(p.x[c] >= -L / 2 && p.x[c] < L / 2);
	
	    for(int i = 0; i < 26; ++i)
	    {
		int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

		bool halo = true;			
	
		for(int c = 0; c < 3; ++c)
		{
		    const float halo_start = max(d[c] * L - L/2 - 1, -L/2);
		    const float halo_end = min(d[c] * L + L/2 + 1, L/2);
		
		    const float x = p.x[c];
		
		    halo &= (x >= halo_start && x < halo_end);
		}
   
		if (halo)
		    offset[i] = atomicAdd(histo + i, 1);
	    }
	}
	__syncthreads();
    
	if (tid < 26 && histo[tid] > 0)
	    base[tid] = atomicAdd(global_histo + tid, histo[tid]);

	__syncthreads();

	for(int i = 0; i < 26; ++i)
	    if (offset[i] != -1)
	    {
		const int entry = base[i] + offset[i];
		assert(entry >= 0 && entry < global_histo[26]); 
		
		bag[ entry ] = p; 
		scattered_entries[ entry ] = pid;
	    }
    }

    __global__ void shift_recv_particles(Particle * p, int n, int L, int code)
    {
	assert(blockDim.x * gridDim.x >= n);
	
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	for(int c = 0; c < 3; ++c)
	    assert(p[pid].x[c] >= -L / 2 && p[pid].x[c] < L / 2);

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	
	for(int c = 0; c < 3; ++c)
	    p[pid].x[c] += d[c] * L;

#ifndef NDEBUG

	assert(p[pid].x[0] <= -L / 2 || p[pid].x[0] >= L / 2 ||
	       p[pid].x[1] <= -L / 2 || p[pid].x[1] >= L / 2 || 
	       p[pid].x[2] <= -L / 2 || p[pid].x[2] >= L / 2);

	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L - L/2, -L/2 - 1);
	    const float halo_end = min(d[c] * L + L/2, L/2 + 1);

	    assert(p[pid].x[c] >= halo_start && p[pid].x[c] <= halo_end);
	}
	
#endif
    }

    __global__ void check_send_particles(Particle * p, int n, int L, int code)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	assert(p[pid].x[0] >= -L / 2 || p[pid].x[0] < L / 2 ||
	       p[pid].x[1] >= -L / 2 || p[pid].x[1] < L / 2 || 
	       p[pid].x[2] >= -L / 2 || p[pid].x[2] < L / 2);

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	
	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L - L/2 - 1, -L/2);
	    const float halo_end = min(d[c] * L + L/2 + 1, L/2);

	    assert(p[pid].x[c] >= halo_start && p[pid].x[c] < halo_end);
	}
    }
}

void HaloExchanger::pack_and_post(const Particle * const p, const int n)
{
    nlocal = n;
    
    MPI_Status statuses[26];
    
    if (pending_send)
	MPI_CHECK( MPI_Waitall(26, sendreq, statuses) );

    PackingHalo::setup<<<1, 1>>>(false);

    if (n > 0)
	PackingHalo::count<1> <<<(n + 127) / 128, 128>>> (sendpacks_start, p, n, L, send_bag_size_required);    
    else
	for(int i = 0; i < 27; ++i)
	    sendpacks_start_host[i] = 0;
    
stage2:
    if (n > 0)
	PackingHalo::pack <<<(n + 127) / 128, 128>>>(p, n, L, send_bag, send_bag_size, scattered_entries);

    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    if (send_bag_size < *send_bag_size_required_host)
    {
	printf("Ooops SIZE: %d REQUIRED: %d\n", send_bag_size, *send_bag_size_required_host);
	
	CUDA_CHECK(hipFree(send_bag));
	CUDA_CHECK(hipFree(scattered_entries));

	send_bag_size = *send_bag_size_required_host;
	
	CUDA_CHECK(hipMalloc(&send_bag, sizeof(Particle) * send_bag_size));
	CUDA_CHECK(hipMalloc(&scattered_entries, sizeof(int) * send_bag_size));
	
	goto stage2;
    }

    for(int i = 0; i < 27; ++i)
	send_offsets[i] = sendpacks_start_host[i];

    assert(send_offsets[26] == *send_bag_size_required_host);
    
    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Isend(send_bag + send_offsets[i], send_offsets[i + 1] - send_offsets[i],
			     Particle::datatype(), dstranks[i], tagbase_dpd_remote_interactions + i, cartcomm, sendreq + i) );

    pending_send = true;
    
    {
	int sum = 0;
	
	for(int i = 0; i < 26; ++i)
	{
	    MPI_Status status;
	    MPI_CHECK( MPI_Probe(MPI_ANY_SOURCE, recv_tags[i], cartcomm, &status) );

	    int count;
	    MPI_CHECK( MPI_Get_count(&status, Particle::datatype(), &count) );

	    recv_offsets[i] = sum;
	    sum += count;
	}

	recv_offsets[26] = sum;
		
	if (recv_bag_size < sum)
	{
	    if (recv_bag_size > 0)
		CUDA_CHECK(hipFree(recv_bag));
	    
	    CUDA_CHECK(hipMalloc(&recv_bag, sizeof(Particle) * sum));
	    
	    recv_bag_size = sum;
	}
    }

    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Irecv(recv_bag + recv_offsets[i], recv_offsets[i + 1] - recv_offsets[i],
			     Particle::datatype(), MPI_ANY_SOURCE, recv_tags[i], cartcomm, recvreq + i) );
}
    
void HaloExchanger::wait_for_messages()
{
    MPI_Status statuses[26];
    MPI_CHECK( MPI_Waitall(26, recvreq, statuses) );
    
    const int nremote = send_offsets[26];

    if (nremote !=*send_bag_size_required_host)
	printf("about to abort: %d %d\n", nremote, *send_bag_size_required_host);
    
    assert(nremote ==*send_bag_size_required_host);
    assert(nremote <= send_bag_size);

    for(int i = 0; i < 26; ++i)
    {
	const int ns = recv_offsets[i + 1] - recv_offsets[i];

	if (ns > 0)
	    PackingHalo::shift_recv_particles<<<(ns + 127) / 128, 128>>>(recv_bag + recv_offsets[i], ns, L, i);
	
#ifndef NDEBUG
	const int nd = send_offsets[i + 1] - send_offsets[i];
	
	if (nd > 0)
	    PackingHalo::check_send_particles <<<(nd + 127) / 128, 128>>>(send_bag + send_offsets[i], nd, L, i);
#endif	
    }
}

int HaloExchanger::nof_sent_particles()
{
    const int nsend = send_offsets[26];

    if (nsend !=*send_bag_size_required_host)
	printf("about to abort: %d %d\n", nsend, *send_bag_size_required_host);
    
    assert(nsend ==*send_bag_size_required_host);
    assert(nsend <= send_bag_size);

    for(int i = 0; i < 26; ++i)
	assert(send_offsets[i + 1] - send_offsets[i] <= nlocal);

    return nsend;
}

HaloExchanger::~HaloExchanger()
{
    CUDA_CHECK(hipFree(send_bag));
    CUDA_CHECK(hipFree(recv_bag));
    CUDA_CHECK(hipHostFree(sendpacks_start));
    CUDA_CHECK(hipHostFree(send_bag_size_required));
}