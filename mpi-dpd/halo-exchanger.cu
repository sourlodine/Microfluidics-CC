#include "hip/hip_runtime.h"
#include <cstring>
#include <algorithm>

#include "halo-exchanger.h"
 
using namespace std;

HaloExchanger::HaloExchanger(MPI_Comm _cartcomm, const int basetag):  basetag(basetag), firstpost(true)
{
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);

    MPI_CHECK( MPI_Comm_dup(_cartcomm, &cartcomm));

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));

    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );

	halosize[i].x = d[0] != 0 ? 1 : XSIZE_SUBDOMAIN;
	halosize[i].y = d[1] != 0 ? 1 : YSIZE_SUBDOMAIN;
	halosize[i].z = d[2] != 0 ? 1 : ZSIZE_SUBDOMAIN; 
	
	const int nhalocells = halosize[i].x * halosize[i].y * halosize[i].z;

	int estimate = 6 * nhalocells;
	estimate = 32 * ((estimate + 31) / 32);

	recvhalos[i].setup(estimate, nhalocells);
	sendhalos[i].setup(estimate, nhalocells);
    }

    CUDA_CHECK(hipHostAlloc((void **)&required_send_bag_size_host, sizeof(int) * 26, hipHostMallocMapped));
    CUDA_CHECK(hipHostGetDevicePointer(&required_send_bag_size, required_send_bag_size_host, 0));

    for(int i = 0; i < 7; ++i)
	CUDA_CHECK(hipStreamCreate(streams + i));

    for(int i = 0, ctr = 1; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	const bool isface = abs(d[0]) + abs(d[1]) + abs(d[2]) == 1;

	code2stream[i] = 0;

	if (isface)
	{
	    code2stream[i] = ctr;
	    ctr++;
	}
    }
}

namespace PackingHalo
{
    __global__ void count(const int * const cellsstart, const int * const cellscount,
			  const int3 halo_offset, const int3 halo_size,
			  int * const output_start, int * const output_count)
    {
	assert(halo_size.x * halo_size.y * halo_size.z <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int3 tmp = make_int3(gid % halo_size.x, (gid / halo_size.x) % halo_size.y, (gid / (halo_size.x * halo_size.y)));
	 
	const int3 dst = make_int3(halo_offset.x + tmp.x,
				   halo_offset.y + tmp.y,
				   halo_offset.z + tmp.z);

	const int nsize = halo_size.x * halo_size.y * halo_size.z;
	 
	if (gid < nsize)
	{
	    assert(dst.x >= 0 && dst.x < XSIZE_SUBDOMAIN);
	    assert(dst.y >= 0 && dst.y < YSIZE_SUBDOMAIN);
	    assert(dst.z >= 0 && dst.z < ZSIZE_SUBDOMAIN);
	    
	    const int srcentry = dst.x + XSIZE_SUBDOMAIN * (dst.y + YSIZE_SUBDOMAIN * dst.z);

	    assert(srcentry < XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN);

	    output_start[gid] = cellsstart[srcentry];
	    output_count[gid] = cellscount[srcentry];
	}
	else
	    if (gid == nsize)
		output_start[gid] = output_count[gid] = 0;
    }
    
#ifndef NDEBUG
    __device__ void halo_particle_check(const Particle p, const int pid, const int code)
     {
	 const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
	 const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	 for(int c = 0; c < 3; ++c)
	 {
	     const float halo_start = max(d[c] * L[c] - L[c]/2 - 1, -L[c]/2);
	     const float halo_end = min(d[c] * L[c] + L[c]/2 + 1, L[c]/2);
	     const float eps = 1e-5;
	     if (!(p.x[c] >= halo_start - eps && p.x[c] < halo_end + eps))
	     {
		 printf("fill particles (pack) oooops particle %d: %e %e %e component %d not within %f , %f eps %e\n", pid, 
			p.x[0], p.x[1], p.x[2], c, halo_start, halo_end, eps);
		 
	     }
	     
	     assert(p.x[c] >= halo_start - eps && p.x[c] < halo_end + eps);
	 }
     }
#endif

    __global__ void fill(const Particle * const particles, const int np, const int ncells,
			 const int * const start_src, const int * const count_src,
			 const int * const start_dst, 
			 Particle * const dbag, Particle * const hbag, const int bagsize, int * const scattered_entries, 
			 int * const required_bag_size, const int code)
    {
	assert(sizeof(Particle) == 6 * sizeof(float));
	assert(blockDim.x == warpSize);

	const int cellid = (threadIdx.x >> 4) + 2 * blockIdx.x;

	if (cellid > ncells)
	    return;
	
	const int tid = threadIdx.x & 0xf;
	
	const int base_src = start_src[cellid];
	const int base_dst = start_dst[cellid];
	const int nsrc = min(count_src[cellid], bagsize - base_dst);
	
	const int nfloats = nsrc * 6;
	for(int i = 2 * tid; i < nfloats; i += warpSize)
	{
	    const int lpid = i / 6;
	    const int dpid = base_dst + lpid;
	    const int spid = base_src + lpid;
	    assert(spid < np && spid >= 0);

	    const int c = i % 6;
	    
	    float2 word = *(float2 *)&particles[spid].x[c];
	    *(float2 *)&dbag[dpid].x[c] = word;
	    *(float2 *)&hbag[dpid].x[c] = word;

#ifndef NDEBUG
	    halo_particle_check(particles[spid], spid, code)   ;
#endif
	}

	for(int lpid = tid; lpid < nsrc; lpid += warpSize / 2)
	{
	    const int dpid = base_dst + lpid;
	    const int spid = base_src + lpid;

	    scattered_entries[dpid] = spid;
	}
	
	if (cellid == ncells)
	    *required_bag_size = base_dst;
    }
   
    __constant__ Particle * srcpacks[26], * dstpacks[26];
    __constant__ int packstarts[27];

    __global__ void shift_recv_particles_float(const int np)
    {
	assert(sizeof(Particle) == 6 * sizeof(float));
	assert(blockDim.x * gridDim.x >= np * 6);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	const int pid = gid / 6;
	const int c = gid % 6;

	const int key9 = 9 * (pid >= packstarts[8]) + 9 * (pid >= packstarts[17]);
	const int key3 = 3 * (pid >= packstarts[key9 + 2]) + 3 * (pid >= packstarts[key9 + 5]);
	const int key1 = (pid >= packstarts[key9 + key3]) + (pid >= packstarts[key9 + key3 + 1]);
	const int code = key9 + key3 + key1 - 1;

	assert(code >= 0 && code < 26);

	const int base = packstarts[code];
	const int offset = pid - base;

	const float val = *(c + (float *)&srcpacks[code][offset].x[0]);

	const int dx = (code + 2) % 3 - 1;
	const int dy = (code / 3 + 2) % 3 - 1;
	const int dz = (code / 9 + 2) % 3 - 1;

	*(c + (float *)&dstpacks[code][offset].x[0]) =  val + 
	    XSIZE_SUBDOMAIN * dx * (c == 0) + 
	    YSIZE_SUBDOMAIN * dy * (c == 1) + 
	    ZSIZE_SUBDOMAIN * dz * (c == 2);
    }

#ifndef NDEBUG
    __global__ void check_recv_particles(Particle *const particles, const int n,
					 const int code, const int rank)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;
	
	Particle myp = particles[pid];

	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	assert(myp.x[0] <= -L[0] / 2 || myp.x[0] >= L[0] / 2 ||
	       myp.x[1] <= -L[1] / 2 || myp.x[1] >= L[1] / 2 || 
	       myp.x[2] <= -L[2] / 2 || myp.x[2] >= L[2] / 2);

	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L[c] - L[c]/2, -L[c]/2 - 1);
	    const float halo_end = min(d[c] * L[c] + L[c]/2, L[c]/2 + 1);
	    const float eps = 1e-5;
	    if (!(myp.x[c] >= halo_start - eps && myp.x[c] <= halo_end + eps))
		printf("ooops RANK %d: shift_recv_particle: pid %d \npos %f %f %f vel: %f %f %f halo_start-end: %f %f\neps: %f, code %d c: %d direction %d %d %d\n",
		       rank, pid, myp.x[0], myp.x[1], myp.x[2]
		       ,myp.u[0], myp.u[1], myp.u[2], halo_start, halo_end, eps, code, c,
		       d[0], d[1], d[2]);

	    assert(myp.x[c] >= halo_start - eps && myp.x[c] <= halo_end + eps);
	}
    }
#endif
    
#ifndef NDEBUG
    __global__ void check_send_particles(Particle * p, int n, int code)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	assert(p[pid].x[0] >= -L[0] / 2 || p[pid].x[0] < L[0] / 2 ||
	       p[pid].x[1] >= -L[1] / 2 || p[pid].x[1] < L[1] / 2 || 
	       p[pid].x[2] >= -L[2] / 2 || p[pid].x[2] < L[2] / 2);

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L[c] - L[c]/2 - 1, -L[c]/2);
	    const float halo_end = min(d[c] * L[c] + L[c]/2 + 1, L[c]/2);
	    const float eps = 1e-5;
	    if (!(p[pid].x[c] >= halo_start - eps && p[pid].x[c] < halo_end + eps))
		printf("oooops particle %d: %e %e %e component %d not within %f , %f eps %f\n", 
		       pid, p[pid].x[0], p[pid].x[1], p[pid].x[2],
		       c, halo_start, halo_end, eps);
	    
	    assert(p[pid].x[c] >= halo_start - eps && p[pid].x[c] < halo_end + eps);
	}
    }
#endif
}

void HaloExchanger::pack_and_post(const Particle * const p, const int n, const int * const cellsstart, const int * const cellscount)
{
    CUDA_CHECK(hipPeekAtLastError());

    nlocal = n;
    
    for(int i = 0; i < 26; ++i)
    {
	const int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };
	
	int halo_start[3], halo_size[3];
	for(int c = 0; c < 3; ++c)
	{
	    halo_start[c] = max(d[c] * L[c] - L[c]/2 - 1, -L[c]/2);
	    halo_size[c] = min(d[c] * L[c] + L[c]/2 + 1, L[c]/2) - halo_start[c];
	}

	const int nentries = sendhalos[i].dcellstarts.size;
	
	PackingHalo::count<<< (nentries + 127) / 128, 128, 0, streams[code2stream[i]] >>>
	    (cellsstart, cellscount,  
	     make_int3(halo_start[0] + XSIZE_SUBDOMAIN / 2 , 
		       halo_start[1] + YSIZE_SUBDOMAIN / 2, 
		       halo_start[2] + ZSIZE_SUBDOMAIN / 2),
	     make_int3(halo_size[0], halo_size[1], halo_size[2]), 
	     sendhalos[i].tmpstart.data, sendhalos[i].tmpcount.data);
    }
    
    for(int i = 0; i < 26; ++i)
	scan.exclusive(streams[code2stream[i]], (uint*)sendhalos[i].dcellstarts.data, (uint*)sendhalos[i].tmpcount.data,
		       sendhalos[i].tmpcount.size);
    
    if (firstpost)
	post_expected_recv();
    else
    {
	MPI_Status statuses[26 * 2];

	MPI_CHECK( MPI_Waitall(nsendreq, sendreq, statuses) );
	MPI_CHECK( MPI_Waitall(26, sendcellsreq, statuses) );
	MPI_CHECK( MPI_Waitall(26, sendcountreq, statuses) );
    }
      
    for(int i = 0; i < 26; ++i)
	CUDA_CHECK(hipMemcpyAsync(sendhalos[i].hcellstarts.devptr, sendhalos[i].dcellstarts.data, 
				   sizeof(int) * sendhalos[i].dcellstarts.size, 
				   hipMemcpyDeviceToDevice, streams[code2stream[i]]));
 
    for(int pass = 0; pass < 2; ++pass)
    {
	bool needsync = pass == 0;

	for(int i = 0; i < 26; ++i)
	{
	    bool fail = false;
	    int nrequired;

	    if (pass == 1)
	    {
		nrequired = required_send_bag_size_host[i];
		fail = sendhalos[i].dbuf.capacity < nrequired;
	    }

	    if (pass == 0 || fail)
	    {
		if (fail)
		{
		    printf("------------------- rank %d - code %d : oops now: %d, expected: %d required: %d, current capacity: %d\n", 
			   myrank, i, sendhalos[i].dbuf.size,
			   sendhalos[i].expected, nrequired, sendhalos[i].dbuf.capacity);
		    sendhalos[i].dbuf.resize(nrequired);
		    sendhalos[i].hbuf.resize(nrequired);
		    sendhalos[i].scattered_entries.resize(nrequired);
		    needsync = true;
		}
		
		const int nentries = sendhalos[i].dcellstarts.size;

		PackingHalo::fill<<<nentries, 32, 0, streams[code2stream[i]] >>>
		    (p, n, nentries - 1, sendhalos[i].tmpstart.data, sendhalos[i].tmpcount.data, sendhalos[i].dcellstarts.data,
		     sendhalos[i].dbuf.data, sendhalos[i].hbuf.data, sendhalos[i].dbuf.capacity, sendhalos[i].scattered_entries.data, required_send_bag_size + i, i);	
	    }

	    if (pass == 1)
	    {
		sendhalos[i].dbuf.size = nrequired;
		sendhalos[i].hbuf.size = nrequired;
		sendhalos[i].scattered_entries.size = nrequired;
	    }
	} 

	CUDA_CHECK(hipPeekAtLastError());

	if (needsync)
	    for(int i = 0; i < 7; ++i)
		CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }
	
#ifndef NDEBUG
    for(int i = 0; i < 26; ++i)
    {
    	const int nd = sendhalos[i].dbuf.size;
	
	if (nd > 0)
	    PackingHalo::check_send_particles<<<(nd + 127)/ 128, 128>>>(sendhalos[i].dbuf.data, nd, i);
    }

    //CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipPeekAtLastError());
#endif

    spawn_local_work();
   
    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Isend(sendhalos[i].hcellstarts.data, sendhalos[i].hcellstarts.size, MPI_INTEGER, dstranks[i],
			     basetag + i + 350, cartcomm,  sendcellsreq + i) );

    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Isend(&sendhalos[i].hbuf.size, 1, MPI_INTEGER, dstranks[i], basetag +  i + 150, cartcomm, sendcountreq + i) );

    nsendreq = 0;
    
    for(int i = 0; i < 26; ++i)
    {
	const int count = sendhalos[i].hbuf.size;
	const int expected = sendhalos[i].expected;
	
	MPI_CHECK( MPI_Isend(sendhalos[i].hbuf.data, expected, Particle::datatype(), dstranks[i], 
			    basetag +  i, cartcomm, sendreq + nsendreq) );

	++nsendreq;
	
	if (count > expected)
	{
	    const int difference = count - expected;
	    printf("extra message from rank %d to rank %d! difference %d\n", myrank, dstranks[i], difference);
	    
	    MPI_CHECK( MPI_Isend(sendhalos[i].hbuf.data + expected, difference, Particle::datatype(), dstranks[i], 
				 basetag + i + 555, cartcomm, sendreq + nsendreq) );

	    ++nsendreq;
	}
    }

    firstpost = false;
}

void HaloExchanger::post_expected_recv()
{
    for(int i = 0; i < 26; ++i)
    {
	assert(recvhalos[i].hbuf.capacity >= recvhalos[i].expected);
	
	MPI_CHECK( MPI_Irecv(recvhalos[i].hbuf.data, recvhalos[i].expected, Particle::datatype(), dstranks[i], 
			     basetag + recv_tags[i], cartcomm, recvreq + i) );
    }

    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Irecv(recvhalos[i].hcellstarts.data, recvhalos[i].hcellstarts.size, MPI_INTEGER, dstranks[i],
			     basetag + recv_tags[i] + 350, cartcomm,  recvcellsreq + i) );
    
    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, dstranks[i],
			     basetag + recv_tags[i] + 150, cartcomm, recvcountreq + i) );
}

void HaloExchanger::wait_for_messages()
{
    CUDA_CHECK(hipPeekAtLastError());
    
    {
	MPI_Status statuses[26];

	MPI_CHECK( MPI_Waitall(26, recvreq, statuses) );    
	MPI_CHECK( MPI_Waitall(26, recvcellsreq, statuses) );
	MPI_CHECK( MPI_Waitall(26, recvcountreq, statuses) );
    }

    for(int i = 0; i < 26; ++i)
    {
	const int count = recv_counts[i];
	const int expected = recvhalos[i].expected;
	const int difference = count - expected;
	
	if (count <= expected)
	{
	    recvhalos[i].hbuf.resize(count);
	    recvhalos[i].dbuf.resize(count);
	}
	else
	{
	    printf("RANK %d waiting for RECV-extra message: count %d expected %d (difference %d) from rank %d\n", 
		   myrank, count, expected, difference, dstranks[i]);
	    
	    recvhalos[i].hbuf.preserve_resize(count);
	    recvhalos[i].dbuf.resize(count);

	    MPI_Status status;

	    MPI_Recv(recvhalos[i].hbuf.data + expected, difference, Particle::datatype(), dstranks[i], 
		     basetag + recv_tags[i] + 555, cartcomm, &status);
	}
    }

    for(int code = 0; code < 26; ++code)
	CUDA_CHECK(hipMemcpyAsync(recvhalos[code].dcellstarts.data, recvhalos[code].hcellstarts.devptr,
				   sizeof(int) * recvhalos[code].hcellstarts.size, hipMemcpyDeviceToDevice, streams[code2stream[code]]));

    //shift the received particles
    {
	int packstarts[27];
	
	packstarts[0] = 0;
	for(int code = 0, s = 0; code < 26; ++code)
	    packstarts[code + 1] = (s += recv_counts[code]);
	
	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(PackingHalo::packstarts), packstarts, sizeof(packstarts), 0, hipMemcpyHostToDevice));

	Particle * srcpacks[26];
	for(int i = 0; i < 26; ++i)
	    srcpacks[i] = recvhalos[i].hbuf.devptr;

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(PackingHalo::srcpacks), srcpacks, sizeof(srcpacks), 0, hipMemcpyHostToDevice));

	Particle * dstpacks[26];
	for(int i = 0; i < 26; ++i)
	    dstpacks[i] = recvhalos[i].dbuf.data;

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(PackingHalo::dstpacks), dstpacks, sizeof(dstpacks), 0, hipMemcpyHostToDevice));

	const int np = packstarts[26];

	PackingHalo::shift_recv_particles_float<<<(np * 6 + 127) / 128, 128>>>(np);
    }

    CUDA_CHECK(hipPeekAtLastError());

#ifndef NDEBUG
    for(int code = 0; code < 26; ++code)
    {
	const int count = recv_counts[code];
	
	if (count > 0)
	    PackingHalo::check_recv_particles<<<(count + 127) / 128, 128, 0>>>(
		recvhalos[code].dbuf.data, count, code, myrank);	
    }

    CUDA_CHECK(hipPeekAtLastError());
#endif

    post_expected_recv();
}

int HaloExchanger::nof_sent_particles()
{
    int s = 0;
    for(int i = 0; i < 26; ++i)
	s += sendhalos[i].hbuf.size;

    return s;
}

HaloExchanger::~HaloExchanger()
{
    for(int i = 0; i < 7; ++i)
	CUDA_CHECK(hipStreamDestroy(streams[i]));
    
    CUDA_CHECK(hipHostFree(required_send_bag_size));

    MPI_CHECK(MPI_Comm_free(&cartcomm));

    if (!firstpost)
    {
	for(int i = 0; i < 26; ++i)
	    MPI_CHECK( MPI_Cancel(recvreq + i) );
	
	for(int i = 0; i < 26; ++i)
	    MPI_CHECK( MPI_Cancel(recvcellsreq + i) );
	
	for(int i = 0; i < 26; ++i)
	    MPI_CHECK( MPI_Cancel(recvcountreq + i) );
    }
}
