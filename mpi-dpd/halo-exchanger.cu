#include "hip/hip_runtime.h"
#include <cstring>
#include <algorithm>

#include "halo-exchanger.h"

using namespace std;

HaloExchanger::HaloExchanger(MPI_Comm _cartcomm, int L, const int basetag):  L(L), basetag(basetag)
{
    assert(L % 2 == 0);
    assert(L >= 2);

    MPI_CHECK( MPI_Comm_dup(_cartcomm, &cartcomm));

    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
    MPI_CHECK( MPI_Comm_size(cartcomm, &nranks));

    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	recv_tags[i] = (2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i) );

	const int nhalocells = pow(L, 3 - abs(d[0]) - abs(d[1]) - abs(d[2]));

	int estimate = 6 * nhalocells; //1;//max(128, (int)( 3 * nhalocells));
	estimate = 32 * ((estimate + 31) / 32);

	halosize[i].x = d[0] != 0 ? 1 : L;
	halosize[i].y = d[1] != 0 ? 1 : L;
	halosize[i].z = d[2] != 0 ? 1 : L;
	assert(nhalocells == halosize[i].x * halosize[i].y * halosize[i].z);

	recvhalos[i].expected = estimate;
	recvhalos[i].buf.resize(estimate);
	//recvhalos[i].secondary.resize(estimate);
	recvhalos[i].cellstarts.resize(nhalocells + 1);

	sendhalos[i].expected = estimate;
	sendhalos[i].buf.resize(estimate);
	//sendhalos[i].secondary.resize(estimate);
	sendhalos[i].scattered_entries.resize(estimate);
	sendhalos[i].cellstarts.resize(nhalocells + 1);
	sendhalos[i].tmpcount.resize(nhalocells + 1);
	sendhalos[i].tmpstart.resize(nhalocells + 1);
    }

    CUDA_CHECK(hipHostAlloc((void **)&required_send_bag_size_host, sizeof(int) * 26, hipHostMallocMapped));
    CUDA_CHECK(hipHostGetDevicePointer(&required_send_bag_size, required_send_bag_size_host, 0));

    for(int i = 0; i < 7; ++i)
	CUDA_CHECK(hipStreamCreate(streams + i));

    for(int i = 0, ctr = 1; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };

	const bool isface = abs(d[0]) + abs(d[1]) + abs(d[2]) == 1;

	code2stream[i] = 0;

	if (isface)
	{
	    code2stream[i] = ctr;
	    ctr++;
	}
    }
}

namespace PackingHalo
{
    __global__ void count(const int * const cellsstart, const int * const cellscount,
			  const int3 halo_offset, const int3 halo_size, const int L,
			  int * const output_start, int * const output_count)
    {
	assert(halo_size.x * halo_size.y * halo_size.z <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	const int3 tmp = make_int3(gid % halo_size.x, (gid / halo_size.x) % halo_size.y, (gid / (halo_size.x * halo_size.y)));
	 
	const int3 dst = make_int3(halo_offset.x + tmp.x,
				   halo_offset.y + tmp.y,
				   halo_offset.z + tmp.z);

	const int nsize = halo_size.x * halo_size.y * halo_size.z;
	 
	if (gid < nsize)
	{
	    assert(dst.x >= 0 && dst.x < L);
	    assert(dst.y >= 0 && dst.y < L);
	    assert(dst.z >= 0 && dst.z < L);
	    
	    const int srcentry = dst.x + L * (dst.y + L * dst.z);

	    assert(srcentry < L * L * L);

	    output_start[gid] = cellsstart[srcentry];
	    output_count[gid] = cellscount[srcentry];
	}
	else
	    if (gid == nsize)
		output_start[gid] = output_count[gid] = 0;
    }
    
    __global__ void fill(const Particle * const particles, const int np, const int ncells,
			 const int * const start_src, const int * const count_src,
			 const int * const start_dst, 
			 Particle * const bag, const int bagsize, int * const scattered_entries, int * const required_bag_size, const int code)
    {
	assert(sizeof(Particle) == 6 * sizeof(float));
	assert(blockDim.x == warpSize);

	const int cellid = (threadIdx.x >> 4) + 2 * blockIdx.x;

	if (cellid > ncells)
	    return;
	
	const int tid = threadIdx.x & 0xf;
	
	const int base_src = start_src[cellid];
	const int base_dst = start_dst[cellid];
	const int nsrc = min(count_src[cellid], bagsize - base_dst);
	
	const int nfloats = nsrc * 6;
	for(int i = 2 * tid; i < nfloats; i += warpSize)
	{
	    const int lpid = i / 6;
	    const int dpid = base_dst + lpid;
	    const int spid = base_src + lpid;
	    const int c = i % 6;

	    *(float2 *)&bag[dpid].x[c] = *(float2 *)&particles[spid].x[c];
	}

	for(int lpid = tid; lpid < nsrc; lpid += warpSize / 2)
	{
	    const int dpid = base_dst + lpid;
	    const int spid = base_src + lpid;

	    scattered_entries[dpid] = spid;
	}
	
	if (cellid == ncells)
	    *required_bag_size = base_dst;
    }
    
    __global__ void fill_v1(const Particle * const particles, const int np,
			 const int * const start_src, const int * const count_src, const int L,
			 const int * const start_dst, 
			 Particle * const bag, const int bagsize, int * const scattered_entries, int * const required_bag_size, const int code)
    {
	assert(blockDim.x == warpSize);

	const int cellid = blockIdx.x;
	const int tid = threadIdx.x;

	const int base_src = start_src[cellid];
	const int base_dst = start_dst[cellid];

	const int nsrc = min(count_src[cellid], bagsize - base_dst);
	
	for(int i = tid; i < nsrc; i += warpSize)
	{
	    const int pid = base_src + i;

	    if (!(pid < np && pid >= 0))
	    {
		printf("ooooooooooops: pid %d, but np %d and nsrc is %d, cell id %d\n", pid, np, nsrc, cellid);
	    }
	    assert(pid < np && pid >= 0);
	    
	    bag[base_dst + i] = particles[pid];
	    scattered_entries[base_dst + i] = pid;

#ifndef NDEBUG
	    {
		int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };
		
		for(int c = 0; c < 3; ++c)
		{
		    const float halo_start = max(d[c] * L - L/2 - 1, -L/2);
		    const float halo_end = min(d[c] * L + L/2 + 1, L/2);
		    const float eps = 1e-5;
		    if (!(particles[pid].x[c] >= halo_start - eps && particles[pid].x[c] < halo_end + eps))
		    {
			printf("fill particles (pack) oooops particle %d: %e %e %e component %d not within %f , %f eps %e\n", pid, 
			       particles[pid].x[0], particles[pid].x[1], particles[pid].x[2], c, halo_start, halo_end, eps);
		
		    }
		   
		    assert(particles[pid].x[c] >= halo_start - eps && particles[pid].x[c] < halo_end + eps);
		}
	    }
#endif
	}

	if (cellid == gridDim.x - 1)
	    *required_bag_size = base_dst;
    }

      __global__ void shift_recv_particles(Particle *const particles, const int n, const int L,
					 const int code, const int dx, const int dy, const int dz, const int rank)
      {
	  assert(sizeof(Particle) == 6 * sizeof(float));
	  assert(blockDim.x * gridDim.x >= n);

	  const int base = threadIdx.x + 6 * blockDim.x * blockIdx.x;
	  const int stop = min(base + 6 * blockDim.x, 6 * n);
	  const int d[3] = {dx, dy, dz};
	  
	  for(int e = base ; e < stop; e += blockDim.x)
	  {
	      const int pid = e / 6;
	      const int c = e % 6;

	      if (c < 3)
		  particles[pid].x[c] += d[c] * L;
	  }    	  
      }

    __global__ void shift_recv_particles4(float4 * const particles4, const int n4, const int np, const int L,
					 const int code, const int dx, const int dy, const int dz, const int rank)
      {
	  assert(sizeof(Particle) == 6 * sizeof(float));
	  assert(blockDim.x * gridDim.x >= n4);
	  assert(blockDim.x * gridDim.x >= (np * 6) / 4);

	  const int gid4 = threadIdx.x + blockDim.x * blockIdx.x;

	  if (gid4 >= n4)
	      return;

	  float4 t = particles4[gid4];

	  float data[4] = {t.x, t.y, t.z, t.w};

#pragma unroll
	  for(int i = 0; i < 4; ++i)
	  {
	      const int e = i + 4 * gid4;
	      const int c = e % 6;

	      data[i] += L * (dx * (c == 0) + dy * (c == 1) + dz * (c == 2));
	  }

	  if (gid4 * 4 + 4 <= np * 6)
	      particles4[gid4] = make_float4(data[0], data[1], data[2], data[3]);
	  else
	  {
	      particles4[gid4].x = data[0];
	      particles4[gid4].y = data[1];
	  }
      }

#ifndef NDEBUG
    __global__ void check_recv_particles(Particle *const particles, const int n, const int L,
					 const int code, const int rank)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;
	
	Particle myp = particles[pid];

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	assert(myp.x[0] <= -L / 2 || myp.x[0] >= L / 2 ||
	       myp.x[1] <= -L / 2 || myp.x[1] >= L / 2 || 
	       myp.x[2] <= -L / 2 || myp.x[2] >= L / 2);

	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L - L/2, -L/2 - 1);
	    const float halo_end = min(d[c] * L + L/2, L/2 + 1);
	    const float eps = 1e-5;
	    if (!(myp.x[c] >= halo_start - eps && myp.x[c] <= halo_end + eps))
		printf("ooops RANK %d: shift_recv_particle: pid %d \npos %f %f %f vel: %f %f %f halo_start-end: %f %f\neps: %f, code %d c: %d direction %d %d %d\n",
		       rank, pid, myp.x[0], myp.x[1], myp.x[2]
		       ,myp.u[0], myp.u[1], myp.u[2], halo_start, halo_end, eps, code, c,
		       d[0], d[1], d[2]);

	    assert(myp.x[c] >= halo_start - eps && myp.x[c] <= halo_end + eps);
	}
    }
#endif
    
    __global__ void shift_recv_particles_old(Particle *const particles, const int n, const int L,
					 const int code, const int dx, const int dy, const int dz, const int rank)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;
	
	Particle myp = particles[pid];
	const Particle oldp = myp;

	for(int c = 0; c < 3; ++c)
	    assert(myp.x[c] >= -L / 2 && myp.x[c] < L / 2);

	const int d[3] = {dx, dy, dz}; //{ (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };

	for(int c = 0; c < 3; ++c)
	    myp.x[c] += d[c] * L;

	particles[pid] = myp;

#ifndef NDEBUG

	assert(myp.x[0] <= -L / 2 || myp.x[0] >= L / 2 ||
	       myp.x[1] <= -L / 2 || myp.x[1] >= L / 2 || 
	       myp.x[2] <= -L / 2 || myp.x[2] >= L / 2);

	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L - L/2, -L/2 - 1);
	    const float halo_end = min(d[c] * L + L/2, L/2 + 1);
	    const float eps = 1e-5;
	    if (!(myp.x[c] >= halo_start - eps && myp.x[c] <= halo_end + eps))
		printf("ooops RANK %d: shift_recv_particle: pid %d \npos %f %f %f vel: %f %f %f halo_start-end: %f %f\neps: %f, code %d c: %d,\n original: %f %f %f %f %f %f, direction %d %d %d\n",
		       rank, pid, myp.x[0], myp.x[1], myp.x[2]
		       ,myp.u[0], myp.u[1], myp.u[2], halo_start, halo_end, eps, code, c,
		       oldp.x[0], oldp.x[1], oldp.x[2] ,oldp.u[0], oldp.u[1], oldp.u[2], d[0], d[1], d[2]);

	    assert(myp.x[c] >= halo_start - eps && myp.x[c] <= halo_end + eps);
	}

#endif
    }

#ifndef NDEBUG
    __global__ void check_send_particles(Particle * p, int n, int L, int code)
    {
	assert(blockDim.x * gridDim.x >= n);

	const int pid = threadIdx.x + blockDim.x * blockIdx.x;

	if (pid >= n)
	    return;

	assert(p[pid].x[0] >= -L / 2 || p[pid].x[0] < L / 2 ||
	       p[pid].x[1] >= -L / 2 || p[pid].x[1] < L / 2 || 
	       p[pid].x[2] >= -L / 2 || p[pid].x[2] < L / 2);

	const int d[3] = { (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1 };


	for(int c = 0; c < 3; ++c)
	{
	    const float halo_start = max(d[c] * L - L/2 - 1, -L/2);
	    const float halo_end = min(d[c] * L + L/2 + 1, L/2);
	    const float eps = 1e-5;
	    if (!(p[pid].x[c] >= halo_start - eps && p[pid].x[c] < halo_end + eps))
	    {
		printf("oooops particle %d: %e %e %e component %d not within %f , %f eps %f\n", pid, p[pid].x[0], p[pid].x[1], p[pid].x[2],
		       c, halo_start, halo_end, eps);
		
	    }
	    
	    assert(p[pid].x[c] >= halo_start - eps && p[pid].x[c] < halo_end + eps);
	}
    }
#endif
}

void HaloExchanger::pack_and_post(const Particle * const p, const int n, const int * const cellsstart, const int * const cellscount)
{
    CUDA_CHECK(hipPeekAtLastError());

    nlocal = n;
    
    for(int i = 0; i < 26; ++i)
    {
	int d[3] = { (i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1 };
	 
	int halo_start[3], halo_size[3];
	for(int c = 0; c < 3; ++c)
	{
	    halo_start[c] = max(d[c] * L - L/2 - 1, -L/2);
	    halo_size[c] = min(d[c] * L + L/2 + 1, L/2) - halo_start[c];
	}

	const int nentries = sendhalos[i].cellstarts.size;
	
	PackingHalo::count<<< (nentries + 127) / 128, 128, 0, streams[code2stream[i]] >>>
	    (cellsstart, cellscount,  make_int3(halo_start[0] + L/2 , halo_start[1] + L/2, halo_start[2] + L/2),
	     make_int3(halo_size[0], halo_size[1], halo_size[2]), L, sendhalos[i].tmpstart.data, sendhalos[i].tmpcount.data);
    }
    
    for(int i = 0; i < 26; ++i)
	scan.exclusive(streams[code2stream[i]], (uint*)sendhalos[i].cellstarts.devptr, (uint*)sendhalos[i].tmpcount.data,
		       sendhalos[i].tmpcount.size);
 
    for(int pass = 0; pass < 2; ++pass)
    {
	bool needsync = pass == 0;

	for(int i = 0; i < 26; ++i)
	{
	    bool fail = false;
	    int nrequired;

	    if (pass == 1)
	    {
		nrequired = required_send_bag_size_host[i];
		fail = sendhalos[i].buf.capacity < nrequired;
	    }

	    if (pass == 0 || fail)
	    {
		if (fail)
		{
		    printf("------------------- rank %d - code %d : oops now: %d, expected: %d required: %d, current capacity: %d\n", 
			   myrank, i, sendhalos[i].buf.size,
			   sendhalos[i].expected, nrequired, sendhalos[i].buf.capacity);
		    sendhalos[i].buf.resize(nrequired);
		    sendhalos[i].scattered_entries.resize(nrequired);
		    needsync = true;
		}
		
		const int nentries = sendhalos[i].cellstarts.size;
#if 1
		PackingHalo::fill<<<nentries, 32, 0, streams[code2stream[i]] >>>
		    (p, n, nentries - 1, sendhalos[i].tmpstart.data, sendhalos[i].tmpcount.data, sendhalos[i].cellstarts.devptr,
		     sendhalos[i].buf.devptr, sendhalos[i].buf.capacity, sendhalos[i].scattered_entries.data, required_send_bag_size + i, i);
#else
		PackingHalo::fill_v1<<<nentries, 32, 0, streams[code2stream[i]] >>>
		    (p, n, sendhalos[i].tmpstart.data, sendhalos[i].tmpcount.data, L, sendhalos[i].cellstarts.devptr,
		     sendhalos[i].buf.devptr, sendhalos[i].buf.capacity, sendhalos[i].scattered_entries.data, required_send_bag_size + i, i);
#endif
	    }

	    if (pass == 1)
	    {
		sendhalos[i].buf.size = nrequired;
		sendhalos[i].scattered_entries.size = nrequired;
	    }
	}

	CUDA_CHECK(hipPeekAtLastError());

	if (needsync)
	    for(int i = 0; i < 7; ++i)
		CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }
	
#ifndef NDEBUG
    for(int i = 0; i < 26; ++i)
    {
    	const int nd = sendhalos[i].buf.size;
	
	if (nd > 0)
	    PackingHalo::check_send_particles<<<(nd + 127)/ 128, 128>>>(sendhalos[i].buf.devptr, nd, L, i);

	printf("send all good\n");
    }

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipPeekAtLastError());
#endif

    for(int i = 0; i < 26; ++i)
    {
	assert(recvhalos[i].buf.capacity >= recvhalos[i].expected);
	
	MPI_CHECK( MPI_Irecv(recvhalos[i].buf.data, recvhalos[i].expected, Particle::datatype(), dstranks[i], 
			     basetag + recv_tags[i], cartcomm, recvreq + i) );
    }

    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Irecv(recvhalos[i].cellstarts.data, recvhalos[i].cellstarts.size, MPI_INTEGER, dstranks[i],
			     basetag + recv_tags[i] + 350, cartcomm,  recvcellsreq + i) );

    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, dstranks[i],
			     basetag + recv_tags[i] + 150, cartcomm, recvcountreq + i) );
     
    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Isend(sendhalos[i].cellstarts.data, sendhalos[i].cellstarts.size, MPI_INTEGER, dstranks[i],
			     basetag + i + 350, cartcomm,  sendcellsreq + i) );

    for(int i = 0; i < 26; ++i)
	MPI_CHECK( MPI_Isend(&sendhalos[i].buf.size, 1, MPI_INTEGER, dstranks[i], basetag +  i + 150, cartcomm, sendcountreq + i) );

    nsendreq = 0;
    
    for(int i = 0; i < 26; ++i)
    {
	const int count = sendhalos[i].buf.size;
	const int expected = sendhalos[i].expected;
	
	MPI_CHECK( MPI_Isend(sendhalos[i].buf.data, expected, Particle::datatype(), dstranks[i], 
			    basetag +  i, cartcomm, sendreq + nsendreq) );

	++nsendreq;
	
	if (count > expected)
	{
	    const int difference = count - expected;
	    printf("extra message from rank %d to rank %d! difference %d\n", myrank, dstranks[i], difference);
	    
	    MPI_CHECK( MPI_Isend(sendhalos[i].buf.data + expected, difference, Particle::datatype(), dstranks[i], 
				 basetag + i + 555, cartcomm, sendreq + nsendreq) );

	    ++nsendreq;
	}
    }
}

void HaloExchanger::wait_for_messages()
{
    CUDA_CHECK(hipPeekAtLastError());
    
    {
	MPI_Status statuses[26];

	MPI_CHECK( MPI_Waitall(26, recvreq, statuses) );    
	MPI_CHECK( MPI_Waitall(26, recvcellsreq, statuses) );
	MPI_CHECK( MPI_Waitall(26, recvcountreq, statuses) );
	MPI_CHECK( MPI_Waitall(26, sendcellsreq, statuses) );
	MPI_CHECK( MPI_Waitall(26, sendcountreq, statuses) );
    }

    for(int i = 0; i < 26; ++i)
    {
	const int count = recv_counts[i];
	const int expected = recvhalos[i].expected;
	const int difference = count - expected;
	
	if (count <= expected)
	    recvhalos[i].buf.resize(count);
	else
	{
	    printf("RANK %d waiting for RECV-extra message: count %d expected %d (difference %d) from rank %d\n", 
		   myrank, count, expected, difference, dstranks[i]);
	    
	    recvhalos[i].buf.preserve_resize(count);

	    MPI_Status status;

	    MPI_Recv(recvhalos[i].buf.data + expected, difference, Particle::datatype(), dstranks[i], 
		     basetag + recv_tags[i] + 555, cartcomm, &status);

	}
    }

    {
	MPI_Status statuses[26];
	
	MPI_CHECK( MPI_Waitall(nsendreq, sendreq, statuses) );
    }
    
    for(int code = 0; code < 26; ++code)
    {
	const int count = recv_counts[code];
#if 1
	if (count > 0)
	{
	    const int n4 = (count * 6 + 3) / 4;
	    PackingHalo::shift_recv_particles4<<<(n4 + 127) / 128, 128, 0, streams[code2stream[code]]>>>(
		(float4 *)recvhalos[code].buf.devptr, n4, count, L, code, (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1, myrank);
	}
#else
	if (count > 0)
	    PackingHalo::shift_recv_particles<<<(count + 127) / 128, 128, 0, streams[code2stream[code]]>>>(
		recvhalos[code].buf.devptr, count, L, code, (code + 2) % 3 - 1, (code / 3 + 2) % 3 - 1, (code / 9 + 2) % 3 - 1, myrank);
#endif
    }

    CUDA_CHECK(hipPeekAtLastError());

#ifndef NDEBUG
    for(int code = 0; code < 26; ++code)
    {
	const int count = recv_counts[code];
	
	if (count > 0)
	    PackingHalo::check_recv_particles<<<(count + 127) / 128, 128, 0, streams[code2stream[code]]>>>(
		recvhalos[code].buf.devptr, count, L, code, myrank);	
    }

    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    printf("all good\n");
#endif
    
    
}

int HaloExchanger::nof_sent_particles()
{
    int s = 0;
    for(int i = 0; i < 26; ++i)
	s += sendhalos[i].buf.size;

    return s;
}

void HaloExchanger::exchange(Particle * const plocal, int nlocal, SimpleDeviceBuffer<Particle>& retval)
{
    CellLists cells(L);	
    cells.build(plocal, nlocal);
   
    pack_and_post(plocal, nlocal, cells.start, cells.count);
    wait_for_messages();

    int s = 0;
    for(int i = 0; i < 26; ++i)
	s += recvhalos[i].buf.size;
    
    retval.resize(s);

    s = 0;
    for(int i = 0; i < 26; ++i)
    {
	CUDA_CHECK(hipMemcpy(retval.data + s, recvhalos[i].buf.devptr, recvhalos[i].buf.size * sizeof(Particle), hipMemcpyDeviceToDevice));
	s += recvhalos[i].buf.size;
    }
}

HaloExchanger::~HaloExchanger()
{
    for(int i = 0; i < 7; ++i)
	CUDA_CHECK(hipStreamDestroy(streams[i]));
    
    CUDA_CHECK(hipHostFree(required_send_bag_size));

    MPI_CHECK(MPI_Comm_free(&cartcomm));
}
