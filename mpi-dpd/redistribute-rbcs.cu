#include "hip/hip_runtime.h"
/*
 *  redistribute-rbcs.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-01.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <vector>

#include "redistribute-particles.h"
#include "redistribute-rbcs.h"
#include "minmax-massimo.h"

RedistributeRBCs::RedistributeRBCs(MPI_Comm _cartcomm): nvertices(CudaRBC::get_nvertices())
{
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);
    
    if (rbcs)
    {
	CudaRBC::Extent host_extent;
	CudaRBC::setup(nvertices, host_extent);
    }
    
    MPI_CHECK(MPI_Comm_dup(_cartcomm, &cartcomm));
	    
    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
	    
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );
	    
    rankneighbors[0] = myrank;
    for(int i = 1; i < 27; ++i)
    {
	int d[3] = { (i + 1) % 3 - 1, (i / 3 + 1) % 3 - 1, (i / 9 + 1) % 3 - 1 };
	
	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];
		
	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, rankneighbors + i) );

	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] - d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, anti_rankneighbors + i) );

	//recvbufs[i].resize(nvertices * 10);
	//sendbufs[i].resize(nvertices * 10);
    }

    CUDA_CHECK(hipEventCreate(&evextents, hipEventDisableTiming));

    _post_recvcount();
}

void RedistributeRBCs::_compute_extents(const Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/extent", NVTX_C7);

#if 1
    if (nrbcs)
	minmax_massimo(xyzuvw, nvertices, nrbcs, minextents.devptr, maxextents.devptr, stream);
#else
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(xyzuvw + nvertices * i), extents.devptr + i);
#endif
}

namespace ReorderingRBC
{
    static const int cmaxnrbcs = 64 * 4;
    __constant__ float * csources[cmaxnrbcs], * cdestinations[cmaxnrbcs];

    template <bool from_cmem>
    __global__ void pack_all_kernel(const int nrbcs, const int nvertices, 
				    const float ** const dsources, float ** const ddestinations)
    {
	if (nrbcs == 0)
	    return;

	const int nfloats_per_rbc = 6 * nvertices;

	assert(nfloats_per_rbc * nrbcs <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (gid >= nfloats_per_rbc * nrbcs) 
	    return;

	const int idrbc = gid / nfloats_per_rbc;
	assert(idrbc < nrbcs);

	const int offset = gid % nfloats_per_rbc;
	
	float val;
	if (from_cmem)
	    val = csources[idrbc][offset];
	else
	    val = dsources[idrbc][offset];
	
	if (from_cmem)
	    cdestinations[idrbc][offset] = val;
	else
	    ddestinations[idrbc][offset] = val;
    }

    SimpleDeviceBuffer<float *> _ddestinations;
    SimpleDeviceBuffer<const float *> _dsources;

    void pack_all(hipStream_t stream, const int nrbcs, const int nvertices, const float ** const sources, float ** const destinations)
    {
	if (nrbcs == 0)
	    return;

	const int nthreads = nrbcs * nvertices * 6;

	if (nrbcs < cmaxnrbcs)
	{
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdestinations), destinations, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csources), sources, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    
	    pack_all_kernel<true><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, NULL, NULL);
	}
	else
	{
	    _ddestinations.resize(nrbcs);
	    _dsources.resize(nrbcs);

	    CUDA_CHECK(hipMemcpyAsync(_ddestinations.data, destinations, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_dsources.data, sources, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));

	    pack_all_kernel<false><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, _dsources.data, _ddestinations.data);
	}

	CUDA_CHECK(hipPeekAtLastError());
    }
}

void RedistributeRBCs::extent(const Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/extent", NVTX_C2);

    minextents.resize(nrbcs);
    maxextents.resize(nrbcs);

    CUDA_CHECK(hipPeekAtLastError());

    _compute_extents(xyzuvw, nrbcs, stream);

    CUDA_CHECK(hipPeekAtLastError());

    CUDA_CHECK(hipEventRecord(evextents, stream));
}
    
void RedistributeRBCs::pack_sendcount(const Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/pack-sendcount", NVTX_C3);

    CUDA_CHECK(hipEventSynchronize(evextents));

    std::vector<int> reordering_indices[27];

    for(int i = 0; i < nrbcs; ++i)
    {
	const float3 minext = minextents.data[i];
	const float3 maxext = maxextents.data[i];

	float p[3] = {
	    0.5 * (minext.x + maxext.x),
	    0.5 * (minext.y + maxext.y),
	    0.5 * (minext.z + maxext.z)
	};
	
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	int vcode[3];
	for(int c = 0; c < 3; ++c)
	    vcode[c] = (2 + (p[c] >= -L[c]/2) + (p[c] >= L[c]/2)) % 3;
	
	const int code = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);

	reordering_indices[code].push_back(i);
    }

    bulk.resize(reordering_indices[0].size() * nvertices);

    for(int i = 1; i < 27; ++i)
	halo_sendbufs[i].resize(reordering_indices[i].size() * nvertices);

#if 1
    {
	std::vector<const float *> src;
	std::vector<float *> dst;

	for(int i = 0; i < 27; ++i)
	    for(int j = 0; j < reordering_indices[i].size(); ++j)
	    {
		src.push_back((float *)(xyzuvw + nvertices * reordering_indices[i][j]));
		
		if (i)
		    dst.push_back((float *)(halo_sendbufs[i].devptr + nvertices * j));
		else
		    dst.push_back((float *)(bulk.data + nvertices * j));
	    }
	
	ReorderingRBC::pack_all(stream, src.size(), nvertices, &src.front(), &dst.front());
	
	CUDA_CHECK(hipPeekAtLastError());
    }
#else
    for(int j = 0; j < reordering_indices[0].size(); ++j)
	CUDA_CHECK(hipMemcpyAsync(bulk.data + nvertices * j, xyzuvw + nvertices * reordering_indices[0][j],
				   sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice, stream));

    for(int i = 1; i < 27; ++i)
	for(int j = 0; j < reordering_indices[i].size(); ++j)
	    CUDA_CHECK(hipMemcpyAsync(halo_sendbufs[i].devptr + nvertices * j, xyzuvw + nvertices * reordering_indices[i][j],
				       sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice, stream));
#endif

    CUDA_CHECK(hipStreamSynchronize(stream));
    
    for(int i = 1; i < 27; ++i)
	MPI_CHECK( MPI_Isend(&halo_sendbufs[i].size, 1, MPI_INTEGER, rankneighbors[i], i + 1024, cartcomm, &sendcountreq[i-1]) );
}

void RedistributeRBCs::_post_recvcount()
{
    recv_counts[0] = 0;

    for(int i = 1; i < 27; ++i)
    {
	MPI_Request req;

	MPI_CHECK( MPI_Irecv(recv_counts + i, 1, MPI_INTEGER, anti_rankneighbors[i], i + 1024, cartcomm, &req) );
	
	recvcountreq.push_back(req);
    }
}

int RedistributeRBCs::post()
{
    NVTX_RANGE("RDC/post", NVTX_C3);

    {
	MPI_Status statuses[recvcountreq.size()];
	MPI_CHECK( MPI_Waitall(recvcountreq.size(), &recvcountreq.front(), statuses) );
	recvcountreq.clear();
    }

    arriving = 0;
    for(int i = 1; i < 27; ++i)
    {
	const int count = recv_counts[i];
	
	arriving += count;
	halo_recvbufs[i].resize(count);
    }
    
    arriving /= nvertices;
    notleaving = bulk.size / nvertices;
  
    MPI_Status statuses[26];	    
    MPI_CHECK( MPI_Waitall(26, sendcountreq, statuses) );

    for(int i = 1; i < 27; ++i)
	if (halo_recvbufs[i].size > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(halo_recvbufs[i].data, halo_recvbufs[i].size, Particle::datatype(),
				anti_rankneighbors[i], i + 1155, cartcomm, &request));

	    recvreq.push_back(request);
	}

    for(int i = 1; i < 27; ++i)
	if (halo_sendbufs[i].size > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Isend(halo_sendbufs[i].data, halo_sendbufs[i].size, Particle::datatype(),
				rankneighbors[i], i + 1155, cartcomm, &request));

	    sendreq.push_back(request);
	}

    return notleaving + arriving;
}

namespace ParticleReorderingRBC
{
    __global__ void shift(const Particle * const psrc, const int np, const int code, const int rank, 
			  const bool check, Particle * const pdst)
    {
	assert(blockDim.x * gridDim.x >= np);
	
	int pid = threadIdx.x + blockDim.x * blockIdx.x;
	
	int d[3] = { (code + 1) % 3 - 1, (code / 3 + 1) % 3 - 1, (code / 9 + 1) % 3 - 1 };
	
	if (pid >= np)
	    return;
	
#ifndef NDEBUG
	Particle old = psrc[pid];
#endif
	Particle pnew = psrc[pid];

	const int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};

	for(int c = 0; c < 3; ++c)
	    pnew.x[c] -= d[c] * L[c];

	pdst[pid] = pnew;

#ifndef NDEBUG
	if (check)
	{
	    int vcode[3];
	    for(int c = 0; c < 3; ++c)
		vcode[c] = (2 + (pnew.x[c] >= -L[c]/2) + (pnew.x[c] >= L[c]/2)) % 3;
		
	    int newcode = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);

	    if(newcode != 0)
		printf("rank %d) particle %d: ouch: new code is %d %d %d arriving from code %d -> %d %d %d \np: %f %f %f (before: %f %f %f)\n", 
		       rank,  pid, vcode[0], vcode[1], vcode[2], code,
		       d[0], d[1], d[2], pnew.x[0], pnew.x[1], pnew.x[2],
		       old.x[0], old.x[1], old.x[2]);
	    
	    assert(newcode == 0);
	}
#endif
    }
}

void RedistributeRBCs::unpack(Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/recv-unpack", NVTX_C7);

    assert(notleaving + arriving == nrbcs);

    MPI_Status statuses[26];
    MPI_CHECK(MPI_Waitall(recvreq.size(), &recvreq.front(), statuses) );
    MPI_CHECK(MPI_Waitall(sendreq.size(), &sendreq.front(), statuses) );
    
    recvreq.clear();
    sendreq.clear();
   
    CUDA_CHECK(hipMemcpyAsync(xyzuvw, bulk.data, notleaving * nvertices * sizeof(Particle), 
			       hipMemcpyDeviceToDevice, stream));
    
    for(int i = 1, s = notleaving * nvertices; i < 27; ++i)
    {
	const int count =  halo_recvbufs[i].size;

	if (count > 0)
	    ParticleReorderingRBC::shift<<< (count + 127) / 128, 128, 0, stream >>>
		(halo_recvbufs[i].devptr, count, i, myrank, false, xyzuvw + s);

	assert(s <= nrbcs * nvertices);

	s += halo_recvbufs[i].size;
    }

    CUDA_CHECK(hipPeekAtLastError());

    _post_recvcount();
}

RedistributeRBCs::~RedistributeRBCs()
{    
    MPI_CHECK(MPI_Comm_free(&cartcomm));
}