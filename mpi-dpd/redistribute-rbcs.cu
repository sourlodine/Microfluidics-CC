#include "hip/hip_runtime.h"
/*
 *  redistribute-rbcs.cu
 *  Part of CTC/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-12-01.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <vector>

#include "redistribute-particles.h"
#include "redistribute-rbcs.h"
#include "minmax-massimo.h"

RedistributeRBCs::RedistributeRBCs(MPI_Comm _cartcomm): nvertices(CudaRBC::get_nvertices())
{
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);
    
    if (rbcs)
    {
	CudaRBC::Extent host_extent;
	CudaRBC::setup(nvertices, host_extent);
    }
    
    MPI_CHECK(MPI_Comm_dup(_cartcomm, &cartcomm));
	    
    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
	    
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );
	    
    rankneighbors[0] = myrank;
    for(int i = 1; i < 27; ++i)
    {
	int d[3] = { (i + 1) % 3 - 1, (i / 3 + 1) % 3 - 1, (i / 9 + 1) % 3 - 1 };
	
	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];
		
	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, rankneighbors + i) );

	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] - d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, anti_rankneighbors + i) );

	//recvbufs[i].resize(nvertices * 10);
	//sendbufs[i].resize(nvertices * 10);
    }

    CUDA_CHECK(hipEventCreate(&evextents, hipEventDisableTiming));
}

void RedistributeRBCs::_compute_extents(const Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/extent", NVTX_C7);

#if 1
    if (nrbcs)
	minmax_massimo(xyzuvw, nvertices, nrbcs, minextents.devptr, maxextents.devptr, stream);
#else
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(xyzuvw + nvertices * i), extents.devptr + i);
#endif
}

namespace ReorderingRBC
{
    static const int cmaxnrbcs = 64 * 4;
    __constant__ float * csources[cmaxnrbcs], * cdestinations[cmaxnrbcs];

    template <bool from_cmem>
    __global__ void pack_all_kernel(const int nrbcs, const int nvertices, 
				    const float ** const dsources, float ** const ddestinations)
    {
	if (nrbcs == 0)
	    return;

	const int nfloats_per_rbc = 6 * nvertices;

	assert(nfloats_per_rbc * nrbcs <= blockDim.x * gridDim.x);

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (gid >= nfloats_per_rbc * nrbcs) 
	    return;

	const int idrbc = gid / nfloats_per_rbc;
	assert(idrbc < nrbcs);

	const int offset = gid % nfloats_per_rbc;
	
	float val;
	if (from_cmem)
	    val = csources[idrbc][offset];
	else
	    val = dsources[idrbc][offset];
	
	if (from_cmem)
	    cdestinations[idrbc][offset] = val;
	else
	    ddestinations[idrbc][offset] = val;
    }

    SimpleDeviceBuffer<float *> _ddestinations;
    SimpleDeviceBuffer<const float *> _dsources;

    void pack_all(hipStream_t stream, const int nrbcs, const int nvertices, const float ** const sources, float ** const destinations)
    {
	const int nthreads = nrbcs * nvertices * 6;

	if (nrbcs < cmaxnrbcs)
	{
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdestinations), destinations, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(csources), sources, sizeof(float *) * nrbcs, 0, hipMemcpyHostToDevice, stream));
	    
	    pack_all_kernel<true><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, NULL, NULL);

	    CUDA_CHECK(hipPeekAtLastError());
	}
	else
	{
	    _ddestinations.resize(nrbcs);
	    _dsources.resize(nrbcs);

	    CUDA_CHECK(hipMemcpyAsync(_ddestinations.data, destinations, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));
	    CUDA_CHECK(hipMemcpyAsync(_dsources.data, sources, sizeof(float *) * nrbcs, hipMemcpyHostToDevice, stream));

	    pack_all_kernel<false><<<(nthreads + 127) / 128, 128, 0, stream>>>(nrbcs, nvertices, _dsources.data, _ddestinations.data);
	}
    }
}

int RedistributeRBCs::stage1(const Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/stage1", NVTX_C3);

    //extents.resize(nrbcs);
    minextents.resize(nrbcs);
    maxextents.resize(nrbcs);

    _compute_extents(xyzuvw, nrbcs, stream);

    CUDA_CHECK(hipEventRecord(evextents, stream));
    CUDA_CHECK(hipEventSynchronize(evextents));

    std::vector<int> reordering_indices[27];

    for(int i = 0; i < nrbcs; ++i)
    {
	const float3 minext = minextents.data[i];
	const float3 maxext = maxextents.data[i];

	float p[3] = {
	    0.5 * (minext.x + maxext.x),
	    0.5 * (minext.y + maxext.y),
	    0.5 * (minext.z + maxext.z)
	};
	
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	int vcode[3];
	for(int c = 0; c < 3; ++c)
	    vcode[c] = (2 + (p[c] >= -L[c]/2) + (p[c] >= L[c]/2)) % 3;
	
	const int code = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);

	reordering_indices[code].push_back(i);
    }

    bulk.resize(reordering_indices[0].size() * nvertices);

    for(int i = 1; i < 27; ++i)
	halo_sendbufs[i].resize(reordering_indices[i].size() * nvertices);

#if 1
    {
	std::vector<const float *> src;
	std::vector<float *> dst;

	for(int i = 0; i < 27; ++i)
	    for(int j = 0; j < reordering_indices[i].size(); ++j)
	    {
		src.push_back((float *)(xyzuvw + nvertices * reordering_indices[i][j]));
		
		if (i)
		    dst.push_back((float *)(halo_sendbufs[i].devptr + nvertices * j));
		else
		    dst.push_back((float *)(bulk.data + nvertices * j));
	    }
	
	ReorderingRBC::pack_all(stream, src.size(), nvertices, &src.front(), &dst.front());
	
	CUDA_CHECK(hipPeekAtLastError());
    }
#else
    for(int j = 0; j < reordering_indices[0].size(); ++j)
	CUDA_CHECK(hipMemcpyAsync(bulk.data + nvertices * j, xyzuvw + nvertices * reordering_indices[0][j],
				   sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice, stream));

    for(int i = 1; i < 27; ++i)
	for(int j = 0; j < reordering_indices[i].size(); ++j)
	    CUDA_CHECK(hipMemcpyAsync(halo_sendbufs[i].devptr + nvertices * j, xyzuvw + nvertices * reordering_indices[i][j],
				       sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice, stream));
#endif

    CUDA_CHECK(hipStreamSynchronize(stream));

    //I need to post receive first
    MPI_Request sendcountreq[26];
    for(int i = 1; i < 27; ++i)
	MPI_CHECK( MPI_Isend(&halo_sendbufs[i].size, 1, MPI_INTEGER, rankneighbors[i], i + 1024, cartcomm, &sendcountreq[i-1]) );

    arriving = 0;
    for(int i = 1; i < 27; ++i)
    {
	int count;
	
	MPI_Status status;
	MPI_CHECK( MPI_Recv(&count, 1, MPI_INTEGER, anti_rankneighbors[i], i + 1024, cartcomm, &status) );

	arriving += count;
	halo_recvbufs[i].resize(count);
    }
    
    arriving /= nvertices;
    notleaving = bulk.size / nvertices;

    //if (arriving)
    //printf("YEE something is arriving to rank %d (arriving %d)\n", myrank, arriving);
  
    MPI_Status statuses[26];	    
    MPI_CHECK( MPI_Waitall(26, sendcountreq, statuses) );

    for(int i = 1; i < 27; ++i)
	if (halo_recvbufs[i].size > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(halo_recvbufs[i].data, halo_recvbufs[i].size, Particle::datatype(),
				anti_rankneighbors[i], i + 1155, cartcomm, &request));

	    recvreq.push_back(request);
	}

    for(int i = 1; i < 27; ++i)
	if (halo_sendbufs[i].size > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Isend(halo_sendbufs[i].data, halo_sendbufs[i].size, Particle::datatype(),
				rankneighbors[i], i + 1155, cartcomm, &request));

	    sendreq.push_back(request);
	}

    return notleaving + arriving;
}

namespace ParticleReorderingRBC
{
    __global__ void shift(const Particle * const psrc, const int np, const int code, const int rank, 
			  const bool check, Particle * const pdst)
    {
	assert(blockDim.x * gridDim.x >= np);
	
	int pid = threadIdx.x + blockDim.x * blockIdx.x;
	
	int d[3] = { (code + 1) % 3 - 1, (code / 3 + 1) % 3 - 1, (code / 9 + 1) % 3 - 1 };
	
	if (pid >= np)
	    return;
	
#ifndef NDEBUG
	Particle old = psrc[pid];
#endif
	Particle pnew = psrc[pid];

	const int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};

	for(int c = 0; c < 3; ++c)
	    pnew.x[c] -= d[c] * L[c];

	pdst[pid] = pnew;

#ifndef NDEBUG
	if (check)
	{
	    int vcode[3];
	    for(int c = 0; c < 3; ++c)
		vcode[c] = (2 + (pnew.x[c] >= -L[c]/2) + (pnew.x[c] >= L[c]/2)) % 3;
		
	    int newcode = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);

	    if(newcode != 0)
		printf("rank %d) particle %d: ouch: new code is %d %d %d arriving from code %d -> %d %d %d \np: %f %f %f (before: %f %f %f)\n", 
		       rank,  pid, vcode[0], vcode[1], vcode[2], code,
		       d[0], d[1], d[2], pnew.x[0], pnew.x[1], pnew.x[2],
		       old.x[0], old.x[1], old.x[2]);
	    
	    assert(newcode == 0);
	}
#endif
    }
}

void RedistributeRBCs::stage2(Particle * const xyzuvw, const int nrbcs, hipStream_t stream)
{
    NVTX_RANGE("RDC/stage2", NVTX_C7);

    assert(notleaving + arriving == nrbcs);

    MPI_Status statuses[26];
    MPI_CHECK(MPI_Waitall(recvreq.size(), &recvreq.front(), statuses) );
    MPI_CHECK(MPI_Waitall(sendreq.size(), &sendreq.front(), statuses) );
    
    recvreq.clear();
    sendreq.clear();
   
    CUDA_CHECK(hipMemcpyAsync(xyzuvw, bulk.data, notleaving * nvertices * sizeof(Particle), 
			       hipMemcpyDeviceToDevice, stream));
    
    for(int i = 1, s = notleaving * nvertices; i < 27; ++i)
    {
	const int count =  halo_recvbufs[i].size;

	if (count > 0)
	    ParticleReorderingRBC::shift<<< (count + 127) / 128, 128, 0, stream >>>
		(halo_recvbufs[i].devptr, count, i, myrank, false, xyzuvw + s);

	assert(s <= nrbcs * nvertices);

	s += halo_recvbufs[i].size;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

RedistributeRBCs::~RedistributeRBCs()
{    
    MPI_CHECK(MPI_Comm_free(&cartcomm));
}