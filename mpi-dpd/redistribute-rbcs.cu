#include "hip/hip_runtime.h"
#include <vector>

#include "redistribute-particles.h"
#include "redistribute-rbcs.h"

RedistributeRBCs::RedistributeRBCs(MPI_Comm _cartcomm): nvertices(CudaRBC::get_nvertices()), stream(0)
{
    assert(XSIZE_SUBDOMAIN % 2 == 0 && YSIZE_SUBDOMAIN % 2 == 0 && ZSIZE_SUBDOMAIN % 2 == 0);
    assert(XSIZE_SUBDOMAIN >= 2 && YSIZE_SUBDOMAIN >= 2 && ZSIZE_SUBDOMAIN >= 2);

    MPI_CHECK(MPI_Comm_dup(_cartcomm, &cartcomm));
	    
    MPI_CHECK( MPI_Comm_rank(cartcomm, &myrank));
	    
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );
	    
    rankneighbors[0] = myrank;
    for(int i = 1; i < 27; ++i)
    {
	int d[3] = { (i + 1) % 3 - 1, (i / 3 + 1) % 3 - 1, (i / 9 + 1) % 3 - 1 };
	
	int coordsneighbor[3];
	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] + d[c];
		
	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, rankneighbors + i) );

	for(int c = 0; c < 3; ++c)
	    coordsneighbor[c] = coords[c] - d[c];

	MPI_CHECK( MPI_Cart_rank(cartcomm, coordsneighbor, anti_rankneighbors + i) );

	//recvbufs[i].resize(nvertices * 10);
	//sendbufs[i].resize(nvertices * 10);
    }
}

void RedistributeRBCs::_compute_extents(const Particle * const xyzuvw, const int nrbcs)
{
    for(int i = 0; i < nrbcs; ++i)
	CudaRBC::extent_nohost(stream, (float *)(xyzuvw + nvertices * i), extents.devptr + i);
}

int RedistributeRBCs::stage1(const Particle * const xyzuvw, const int nrbcs)
{
    extents.resize(nrbcs);
 
    _compute_extents(xyzuvw, nrbcs);

    CUDA_CHECK(hipStreamSynchronize(stream));
   
    std::vector<int> reordering_indices[27];

    for(int i = 0; i < nrbcs; ++i)
    {
	const CudaRBC::Extent ext = extents.data[i];
	
	float p[3] = {
	    0.5 * (ext.xmin + ext.xmax),
	    0.5 * (ext.ymin + ext.ymax),
	    0.5 * (ext.zmin + ext.zmax)
	};
	
	const int L[3] = { XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN };

	int vcode[3];
	for(int c = 0; c < 3; ++c)
	    vcode[c] = (2 + (p[c] >= -L[c]/2) + (p[c] >= L[c]/2)) % 3;
	
	const int code = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);

	reordering_indices[code].push_back(i);
    }

    for(int i = 0; i < 27; ++i)
	sendbufs[i].resize(reordering_indices[i].size() * nvertices);

    for(int i = 0; i < 27; ++i)
	for(int j = 0; j < reordering_indices[i].size(); ++j)
	    CUDA_CHECK(hipMemcpyAsync(sendbufs[i].devptr + nvertices * j, xyzuvw + nvertices * reordering_indices[i][j],
				       sizeof(Particle) * nvertices, hipMemcpyDeviceToDevice, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    MPI_Request sendcountreq[26];
    for(int i = 1; i < 27; ++i)
	MPI_CHECK( MPI_Isend(&sendbufs[i].size, 1, MPI_INTEGER, rankneighbors[i], i + 1024, cartcomm, &sendcountreq[i-1]) );

    arriving = 0;
    for(int i = 1; i < 27; ++i)
    {
	int count;
	
	MPI_Status status;
	MPI_CHECK( MPI_Recv(&count, 1, MPI_INTEGER, anti_rankneighbors[i], i + 1024, cartcomm, &status) );

	arriving += count;
	recvbufs[i].resize(count);
    }
    
    arriving /= nvertices;
    notleaving = sendbufs[0].size / nvertices;

    if (arriving)
	printf("YEE something is arriving to rank %d (arriving %d)\n", myrank, arriving);
  
    MPI_Status statuses[26];	    
    MPI_CHECK( MPI_Waitall(26, sendcountreq, statuses) );


    for(int i = 1; i < 27; ++i)
	if (recvbufs[i].size > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Irecv(recvbufs[i].data, recvbufs[i].size, Particle::datatype(),
				anti_rankneighbors[i], i + 1155, cartcomm, &request));

	    recvreq.push_back(request);
	}

    for(int i = 1; i < 27; ++i)
	if (sendbufs[i].size > 0)
	{
	    MPI_Request request;

	    MPI_CHECK(MPI_Isend(sendbufs[i].data, sendbufs[i].size, Particle::datatype(),
				rankneighbors[i], i + 1155, cartcomm, &request));

	    sendreq.push_back(request);
	}

    return notleaving + arriving;
}

namespace ParticleReorderingRBC
{
    __global__ void shift(const Particle * const psrc, const int np, const int code, const int rank, 
			  const bool check, Particle * const pdst)
    {
	assert(blockDim.x * gridDim.x >= np);
	
	int pid = threadIdx.x + blockDim.x * blockIdx.x;
	
	int d[3] = { (code + 1) % 3 - 1, (code / 3 + 1) % 3 - 1, (code / 9 + 1) % 3 - 1 };
	
	if (pid >= np)
	    return;
	
#ifndef NDEBUG
	Particle old = psrc[pid];
#endif
	Particle pnew = psrc[pid];

	const int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};

	for(int c = 0; c < 3; ++c)
	    pnew.x[c] -= d[c] * L[c];

	pdst[pid] = pnew;

#ifndef NDEBUG
	if (check)
	{
	    int vcode[3];
	    for(int c = 0; c < 3; ++c)
		vcode[c] = (2 + (pnew.x[c] >= -L[c]/2) + (pnew.x[c] >= L[c]/2)) % 3;
		
	    int newcode = vcode[0] + 3 * (vcode[1] + 3 * vcode[2]);

	    if(newcode != 0)
		printf("rank %d) particle %d: ouch: new code is %d %d %d arriving from code %d -> %d %d %d \np: %f %f %f (before: %f %f %f)\n", 
		       rank,  pid, vcode[0], vcode[1], vcode[2], code,
		       d[0], d[1], d[2], pnew.x[0], pnew.x[1], pnew.x[2],
		       old.x[0], old.x[1], old.x[2]);
	    
	    assert(newcode == 0);
	}
#endif
    }
}

void RedistributeRBCs::stage2(Particle * const xyzuvw, const int nrbcs)
{
    assert(notleaving + arriving == nrbcs);

    MPI_Status statuses[26];
    MPI_CHECK(MPI_Waitall(recvreq.size(), &recvreq.front(), statuses) );
    MPI_CHECK(MPI_Waitall(sendreq.size(), &sendreq.front(), statuses) );
    
    recvreq.clear();
    sendreq.clear();
   
    CUDA_CHECK(hipMemcpyAsync(xyzuvw, sendbufs[0].devptr, notleaving * nvertices * sizeof(Particle), 
			       hipMemcpyDeviceToDevice, stream));
    
    for(int i = 1, s = notleaving * nvertices; i < 27; ++i)
    {
	const int count =  recvbufs[i].size;

	if (count > 0)
	    ParticleReorderingRBC::shift<<< (count + 127) / 128, 128, 0, stream >>>
		(recvbufs[i].devptr, count, i, myrank, false, xyzuvw + s);

	assert(s <= nrbcs * nvertices);

	s += recvbufs[i].size;
    }

    CUDA_CHECK(hipPeekAtLastError());
}

RedistributeRBCs::~RedistributeRBCs()
{    
    MPI_CHECK(MPI_Comm_free(&cartcomm));
}