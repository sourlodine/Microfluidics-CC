#include "hip/hip_runtime.h"
#include <string>
#include <sstream>
#include <vector>


#include <sys/time.h>
#include <sys/resource.h>

#include <cuda-dpd.h>
#ifndef NO_H5PART
#include <H5Part.h>
#endif

#include "common.h"

using namespace std;

bool Particle::initialized = false;

MPI_Datatype Particle::mytype;

bool Acceleration::initialized = false;

MPI_Datatype Acceleration::mytype;

void CellLists::build(Particle * const p, const int n)
{
    if (n > 0)
	build_clists((float * )p, n, 1, L, L, L, -L/2, -L/2, -L/2, NULL, start, count,  NULL, 0);
    else
    {
	CUDA_CHECK(hipMemset(start, 0, sizeof(int) * ncells));
	CUDA_CHECK(hipMemset(count, 0, sizeof(int) * ncells));
    }
}

H5PartDump::H5PartDump(const string fname, MPI_Comm cartcomm, const int L):
    cartcomm(cartcomm), fname(fname), tstamp(0)
{
#ifndef NO_H5PART
    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    for(int c = 0; c < 3; ++c)
	origin[c] = L / 2 + coords[c] * L;

    H5PartFile * f = H5PartOpenFileParallel(fname.c_str(), H5PART_WRITE, cartcomm);

    assert(f != NULL);

    handler = f;
#endif
}

void H5PartDump::dump(Particle * host_particles, int n)
{
#ifndef NO_H5PART
    H5PartFile * f = (H5PartFile *)handler;

    H5PartSetStep(f, tstamp);

    H5PartSetNumParticles(f, n);

    string labels[] = {"x", "y", "z"};

    for(int c = 0; c < 3; ++c)
    {
	vector<float> data(n);

	for(int i = 0; i < n; ++i)
	    data[i] = host_particles[i].x[c] + origin[c];

	H5PartWriteDataFloat32(f, labels[c].c_str(), &data.front());
    }

    tstamp++;
#endif
}
    
H5PartDump::~H5PartDump()
{
#ifndef NO_H5PART
    H5PartFile * f = (H5PartFile *)handler;

    H5PartCloseFile(f);
#endif
}

void _write_bytes(const void * const ptr, const int nbytes, MPI_File f, MPI_Comm comm)
{
    MPI_Offset base;
    MPI_CHECK( MPI_File_get_position(f, &base));
    
    int offset = 0;
    MPI_CHECK( MPI_Exscan(&nbytes, &offset, 1, MPI_INTEGER, MPI_SUM, comm)); 
	
    MPI_Status status;
	
    MPI_CHECK( MPI_File_write_at_all(f, base + offset, ptr, nbytes, MPI_CHAR, &status));

    int ntotal = 0;
    MPI_CHECK( MPI_Allreduce(&nbytes, &ntotal, 1, MPI_INT, MPI_SUM, comm) );
    
    MPI_CHECK( MPI_File_seek(f, ntotal, MPI_SEEK_CUR));
}

void ply_dump(MPI_Comm comm, const char * filename,
	      int (*mesh_indices)[3], const int ninstances, const int ntriangles_per_instance,
	      Particle * _particles, int nvertices_per_instance, int L, bool append)
{
    std::vector<Particle> particles(_particles, _particles + ninstances * nvertices_per_instance);
    
    int rank;
    MPI_CHECK( MPI_Comm_rank(comm, &rank) );
    
    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(comm, 3, dims, periods, coords) );

    int NPOINTS = 0;
    const int n = particles.size();
    MPI_CHECK( MPI_Reduce(&n, &NPOINTS, 1, MPI_INT, MPI_SUM, 0, comm) );

    const int ntriangles = ntriangles_per_instance * ninstances;
    int NTRIANGLES = 0;
    MPI_CHECK( MPI_Reduce(&ntriangles, &NTRIANGLES, 1, MPI_INT, MPI_SUM, 0, comm) );
    
    MPI_File f;
    MPI_CHECK( MPI_File_open(comm, filename , MPI_MODE_WRONLY | (append ? MPI_MODE_APPEND : MPI_MODE_CREATE), MPI_INFO_NULL, &f) );

    if (!append)
	MPI_CHECK( MPI_File_set_size (f, 0));
	
    std::stringstream ss;

    if (rank == 0)
    {
	ss <<  "ply\n";
	ss <<  "format binary_little_endian 1.0\n";
	ss <<  "element vertex " << NPOINTS << "\n";
	ss <<  "property float x\nproperty float y\nproperty float z\n";
	ss <<  "property float u\nproperty float v\nproperty float w\n"; 
	//ss <<  "property float xnormal\nproperty float ynormal\nproperty float znormal\n";
	ss <<  "element face " << NTRIANGLES << "\n";
	ss <<  "property list int int vertex_index\n";
	ss <<  "end_header\n";
    } 
    
    string content = ss.str();
    
    _write_bytes(content.c_str(), content.size(), f, comm);
    
    for(int i = 0; i < n; ++i)
	for(int c = 0; c < 3; ++c)
	    particles[i].x[c] += L / 2 + coords[c] * L;

    _write_bytes(&particles.front(), sizeof(Particle) * n, f, comm);

    int poffset = 0;
    
    MPI_CHECK( MPI_Exscan(&n, &poffset, 1, MPI_INTEGER, MPI_SUM, comm));

    std::vector<int> buf;

    for(int j = 0; j < ninstances; ++j)
	for(int i = 0; i < ntriangles_per_instance; ++i)
	{
	    int primitive[4] = { 3,
				 poffset + nvertices_per_instance * j + mesh_indices[i][0],
				 poffset + nvertices_per_instance * j + mesh_indices[i][1],
				 poffset + nvertices_per_instance * j + mesh_indices[i][2] };
	    
	    buf.insert(buf.end(), primitive, primitive + 4);
	}

    _write_bytes(&buf.front(), sizeof(int) * buf.size(), f, comm);
    
    MPI_CHECK( MPI_File_close(&f));
}

void xyz_dump(MPI_Comm comm, const char * filename, const char * particlename, Particle * particles, int n, int L, bool append)
{
    int rank;
    MPI_CHECK( MPI_Comm_rank(comm, &rank) );
    
    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(comm, 3, dims, periods, coords) );

    const int nlocal = n;
    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &n, &n, 1, MPI_INT, MPI_SUM, 0, comm) );
    
    bool filenotthere;
    if (rank == 0)
	filenotthere = access(filename, F_OK ) == -1;

    MPI_CHECK( MPI_Bcast(&filenotthere, 1, MPI_INT, 0, comm) );

    append &= !filenotthere;

    MPI_File f;
    MPI_CHECK( MPI_File_open(comm, filename , MPI_MODE_WRONLY | (append ? MPI_MODE_APPEND : MPI_MODE_CREATE), MPI_INFO_NULL, &f) );

    if (!append)
	MPI_CHECK( MPI_File_set_size (f, 0));
	
    MPI_Offset base;
    MPI_CHECK( MPI_File_get_position(f, &base));
	
    std::stringstream ss;

    if (rank == 0)
    {
	ss <<  n << "\n";
	ss << particlename << "\n";

	printf("total number of particles: %d\n", n);
    }
    
    for(int i = 0; i < nlocal; ++i)
	ss << rank << " " 
	   << (particles[i].x[0] + L / 2 + coords[0] * L) << " "
	   << (particles[i].x[1] + L / 2 + coords[1] * L) << " "
	   << (particles[i].x[2] + L / 2 + coords[2] * L) << "\n";

    string content = ss.str();
	
    int len = content.size();
    int offset = 0;
    MPI_CHECK( MPI_Exscan(&len, &offset, 1, MPI_INTEGER, MPI_SUM, comm)); 
	
    MPI_Status status;
	
    MPI_CHECK( MPI_File_write_at_all(f, base + offset, const_cast<char *>(content.c_str()), len, MPI_CHAR, &status));
	
    MPI_CHECK( MPI_File_close(&f));
}

void diagnostics(MPI_Comm comm, Particle * particles, int n, float dt, int idstep, int L, Acceleration * acc)
{
    const int nlocal = n;
    
    double p[] = {0, 0, 0};
    for(int i = 0; i < n; ++i)
	for(int c = 0; c < 3; ++c)
	    p[c] += particles[i].u[c];

    int rank;
    MPI_CHECK( MPI_Comm_rank(comm, &rank) );

    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(comm, 3, dims, periods, coords) );
    
    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &p, rank == 0 ? &p : NULL, 3, MPI_DOUBLE, MPI_SUM, 0, comm) );
    
    if (rank == 0)
	printf("momentum: %f %f %f\n", p[0], p[1], p[2]);

    double ke = 0;
    for(int i = 0; i < n; ++i)
	ke += pow(particles[i].u[0], 2) + pow(particles[i].u[1], 2) + pow(particles[i].u[2], 2);

    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &ke, &ke, 1, MPI_DOUBLE, MPI_SUM, 0, comm) );
    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &n, &n, 1, MPI_INT, MPI_SUM, 0, comm) );
    
    double kbt = 0.5 * ke / (n * 3. / 2);

    if (rank == 0)
    {
	static bool firsttime = true;
	FILE * f = fopen("diag.txt", firsttime ? "w" : "a");
	firsttime = false;
	
	if (idstep == 0)
	    fprintf(f, "TSTEP\tKBT\tPX\tPY\tPZ\n");
	
	fprintf(f, "%e\t%.10e\t%.10e\t%.10e\t%.10e\n", idstep * dt, kbt, p[0], p[1], p[2]);
	
	fclose(f);
    }

    if (xyz_dumps)
    {
	bool filenotthere;
	if (rank == 0)
	    filenotthere = access( "trajectories.xyz", F_OK ) == -1;

	MPI_CHECK( MPI_Bcast(&filenotthere, 1, MPI_INT, 0, comm) );
	
	static bool firsttime = true;

	firsttime |= filenotthere;

	xyz_dump(comm, "trajectories.xyz", "dpd-particles", particles, nlocal, L, !firsttime);

	firsttime = false;	
    }
}

void report_host_memory_usage(MPI_Comm comm, FILE * foutput)
{
    struct rusage rusage;
    long peak_rss;

    getrusage(RUSAGE_SELF, &rusage);
    peak_rss = rusage.ru_maxrss*1024;
    
    long rss = 0;
    FILE* fp = NULL;
    if ( (fp = fopen( "/proc/self/statm", "r" )) == NULL ) 
    {
	return;
    }

    if ( fscanf( fp, "%*s%ld", &rss ) != 1 )
    {
	fclose( fp );
	return;
    }
    fclose( fp );

    long current_rss;

    current_rss = rss * sysconf( _SC_PAGESIZE);
    
    long max_peak_rss, sum_peak_rss;
    long max_current_rss, sum_current_rss;

    MPI_Reduce(&peak_rss, &max_peak_rss, 1, MPI_LONG, MPI_MAX, 0, comm);
    MPI_Reduce(&peak_rss, &sum_peak_rss, 1, MPI_LONG, MPI_SUM, 0, comm);
    MPI_Reduce(&current_rss, &max_current_rss, 1, MPI_LONG, MPI_MAX, 0, comm);
    MPI_Reduce(&current_rss, &sum_current_rss, 1, MPI_LONG, MPI_SUM, 0, comm);

    int rank;
    MPI_Comm_rank(comm, &rank);

    if (rank == 0)
    {
	fprintf(foutput, "> peak resident set size: max = %.2lf Mbytes sum = %.2lf Mbytes\n",
		max_peak_rss/(1024.0*1024.0), sum_peak_rss/(1024.0*1024.0));
	fprintf(foutput, "> current resident set size: max = %.2lf Mbytes sum = %.2lf Mbytes\n",
		max_current_rss/(1024.0*1024.0), sum_current_rss/(1024.0*1024.0));
    }
}


