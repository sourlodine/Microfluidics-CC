#include "hip/hip_runtime.h"
#include <string>

#include <core/simulation.h>
#include "freeze_particles.h"
#include <core/interactions/sampler.h>
#include <core/initial_conditions/dummy.h>
#include <core/initial_conditions/uniform.h>
#include <core/walls/sdf_wall.h>

#include <core/argument_parser.h>

Logger logger;

void writeXYZ(MPI_Comm comm, std::string fname, ParticleVector* pv)
{
	int rank;
	MPI_Check( MPI_Comm_rank(comm, &rank) );

	int dims[3], periods[3], coords[3];
	MPI_Check( MPI_Cart_get(comm, 3, dims, periods, coords) );

	const int nlocal = pv->local()->size();
	int n = nlocal;
	MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &n, &n, 1, MPI_INT, MPI_SUM, 0, comm) );

	MPI_File f;
	MPI_Check( MPI_File_open(comm, fname.c_str(), MPI_MODE_CREATE|MPI_MODE_DELETE_ON_CLOSE|MPI_MODE_WRONLY, MPI_INFO_NULL, &f) );
	MPI_Check( MPI_File_close(&f) );
	MPI_Check( MPI_File_open(comm, fname.c_str(), MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &f) );

	std::stringstream ss;
	ss.setf(std::ios::fixed, std::ios::floatfield);
	ss.precision(5);

	if (rank == 0)
	{
		ss <<  n << "\n";
		ss << pv->name << "\n";

		info("xyz dump of %s: total number of particles: %d", pv->name.c_str(), n);
	}

	pv->local()->coosvels.downloadFromDevice(0);
	for(int i = 0; i < nlocal; ++i)
	{
		Particle p = pv->local()->coosvels[i];
		p.r = pv->local2global(p.r);

		ss << rank << " "
				<< std::setw(10) << p.r.x << " "
				<< std::setw(10) << p.r.y << " "
				<< std::setw(10) << p.r.z << "\n";
	}

	string content = ss.str();

	MPI_Offset len = content.size();
	MPI_Offset offset = 0;
	MPI_Check( MPI_Exscan(&len, &offset, 1, MPI_OFFSET, MPI_SUM, comm));

	MPI_Status status;
	MPI_Check( MPI_File_write_at_all(f, offset, content.c_str(), len, MPI_CHAR, &status) );
	MPI_Check( MPI_File_close(&f));
}

int main(int argc, char** argv)
{
	srand48(0);

	int rank;
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	std::string xmlname, wname;
	int nepochs;
	bool needXYZ;

	{
		using namespace ArgumentParser;

		std::vector<OptionStruct> opts
		({
			{'i', "input",  STRING, "Input script",                &xmlname,   std::string("script.xml")},
			{'n', "name",   STRING, "Name of the wall to process", &wname,     std::string("wall")},
			{'e', "epochs", INT,    "Number of sampling epochs",   &nepochs,   50},
			{'x', "xyz",    BOOL,   "Also dump .xyz files",        &needXYZ,   false}
		});

		Parser parser(opts, rank == 0);
		parser.parse(argc, argv);
	}

	logger.init(MPI_COMM_WORLD, "genwall.log", 9);


	pugi::xml_document config;
	pugi::xml_parse_result result = config.load_file(xmlname.c_str());
	if (!result)
		die("Couldn't open script file, parser says: \"%s\"", result.description());

	pugi::xml_node wallNode, wallGenNode;
	for (auto node : config.child("simulation").children("wall"))
	{
		if ( std::string(node.attribute("name").as_string()) == wname )
		{
			wallNode = node;
			wallGenNode = node.child("generate_frozen");
		}
	}

	if (wallNode.type() == pugi::node_null)
		die("Wall %s was not found in the script", wname.c_str());
	if (wallGenNode.type() == pugi::node_null)
		die("Wall %s has no generation instructions", wname.c_str());


	float3 globalDomainSize = config.child("simulation").child("domain").attribute("size").as_float3({32, 32, 32});
	int3 nranks3D = config.child("simulation").attribute("mpi_ranks").as_int3({1, 1, 1});

	Simulation* sim = new Simulation(nranks3D, globalDomainSize, MPI_COMM_WORLD, MPI_COMM_NULL);

	ParticleVector *startingPV = new ParticleVector("starting", 1.0);
	ParticleVector *wallPV     = new ParticleVector("wall", 1.0);
	ParticleVector *final      = new ParticleVector(wallGenNode.attribute("name").as_string("final"), 1.0);
	InitialConditions* ic      = new UniformIC(wallGenNode.attribute("density").as_float(4));
	InitialConditions* dummyIC = new DummyIC();

	SDFWall* wall = new SDFWall(
			wallNode.attribute("name").as_string(),
			wallNode.attribute("sdf_filename").as_string(),
			wallNode.attribute("sdf_h").as_float3({0.25, 0.25, 0.25}));

	// Generate pv, but don't register it
	ic->exec(sim->getCartComm(), startingPV, sim->globalDomainStart, sim->localDomainSize, 0);

	// Register and create sdf
	sim->registerWall((Wall*)wall);
	// Produce new pv out of particles inside the wall
	freezeParticlesInWall(wall, startingPV, wallPV, -3, 4);
	sim->registerParticleVector(wallPV, dummyIC);

	auto rc    = wallGenNode.attribute("rc")   .as_float(1.0f);
	auto a     = wallGenNode.attribute("a")    .as_float(50);
	auto kbT   = wallGenNode.attribute("kbt")  .as_float(1.0);
	auto power = wallGenNode.attribute("power").as_float(1.0f);

	Interaction* sampler = new MCMCSampler("sampler",rc, a, kbT, power, wall, -3, 4);
	sim->registerInteraction(sampler);
	sim->setInteraction("sampler", "wall", "wall");

	sim->init();
	sim->run(nepochs);

	freezeParticlesInWall(wall, wallPV, final, 0, 1.2);

	if (needXYZ)
	{
		writeXYZ(sim->getCartComm(), "wall.xyz", wallPV);
		writeXYZ(sim->getCartComm(), final->name+".xyz", final);
	}

	final->checkpoint(sim->getCartComm(), wallGenNode.attribute("path").as_string("./"));

	sim->finalize();
}
