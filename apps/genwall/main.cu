#include "hip/hip_runtime.h"
#include <string>
#include <utility>

#include <core/simulation.h>
#include "freeze_particles.h"
#include <core/interactions/sampler.h>
#include <core/initial_conditions/uniform_ic.h>

#include <core/argument_parser.h>
#include <core/pvs/particle_vector.h>
#include <core/utils/kernel_launch.h>
#include <core/parser/walls_factory.h>

__global__ void zeroVels(PVview view)
{
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;
	if (pid >= view.size) return;

	view.particles[2*pid+1] = make_float4(0);
}

Logger logger;

void writeXYZ(MPI_Comm comm, std::string fname, ParticleVector* pv)
{
	int rank;
	MPI_Check( MPI_Comm_rank(comm, &rank) );

	int dims[3], periods[3], coords[3];
	MPI_Check( MPI_Cart_get(comm, 3, dims, periods, coords) );

	const int nlocal = pv->local()->size();
	int n = nlocal;
	MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &n, &n, 1, MPI_INT, MPI_SUM, 0, comm) );

	MPI_File f;
	MPI_Check( MPI_File_open(comm, fname.c_str(), MPI_MODE_CREATE|MPI_MODE_DELETE_ON_CLOSE|MPI_MODE_WRONLY, MPI_INFO_NULL, &f) );
	MPI_Check( MPI_File_close(&f) );
	MPI_Check( MPI_File_open(comm, fname.c_str(), MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &f) );

	std::stringstream ss;
	ss.setf(std::ios::fixed, std::ios::floatfield);
	ss.precision(5);

	if (rank == 0)
	{
		ss <<  n << "\n";
		ss << pv->name << "\n";

		info("xyz dump of %s: total number of particles: %d", pv->name.c_str(), n);
	}

	pv->local()->coosvels.downloadFromDevice(0);
	for(int i = 0; i < nlocal; ++i)
	{
		Particle p = pv->local()->coosvels[i];
		p.r = pv->domain.local2global(p.r);

		ss << rank << " "
				<< std::setw(10) << p.r.x << " "
				<< std::setw(10) << p.r.y << " "
				<< std::setw(10) << p.r.z << "\n";
	}

	string content = ss.str();

	MPI_Offset len = content.size();
	MPI_Offset offset = 0;
	MPI_Check( MPI_Exscan(&len, &offset, 1, MPI_OFFSET, MPI_SUM, comm));

	MPI_Status status;
	MPI_Check( MPI_File_write_at_all(f, offset, content.c_str(), len, MPI_CHAR, &status) );
	MPI_Check( MPI_File_close(&f));
}


template<class InsideWallChecker>
static Interaction* createMCMCSampler(pugi::xml_node node, const InsideWallChecker& insideWallChecker)
{
	auto name = node.attribute("name").as_string("");
	auto rc   = node.attribute("rc").as_float(1.0f);

	auto a     = node.attribute("a")    .as_float(50);
	auto kbT   = node.attribute("kbt")  .as_float(1.0);
	auto power = node.attribute("power").as_float(1.0f);

	float minVal = -3;
	float maxVal = 4;

	return (Interaction*) new MCMCSampler<InsideWallChecker>(
			name, rc, a, kbT, power, minVal, maxVal, insideWallChecker );
}

static Interaction* createMCMCSamplerWrapper(pugi::xml_node node, Wall* wall)
{
	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Cylinder>* >(wall);
		if (w != nullptr)
			return createMCMCSampler<StationaryWall_Cylinder> (node, w->getChecker());
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Sphere>* >(wall);
		if (w != nullptr)
			return createMCMCSampler<StationaryWall_Sphere> (node, w->getChecker());
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_SDF>* >(wall);
		if (w != nullptr)
			return createMCMCSampler<StationaryWall_SDF> (node, w->getChecker());
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Plane>* >(wall);
		if (w != nullptr)
			return createMCMCSampler<StationaryWall_Plane> (node, w->getChecker());
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Box>* >(wall);
		if (w != nullptr)
			return createMCMCSampler<StationaryWall_Box> (node, w->getChecker());
	}

	return nullptr;
}


int main(int argc, char** argv)
{
	srand48(0);

	int rank;
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	std::string xmlname, wname;
	int nepochs;
	bool needXYZ;

	{
		using namespace ArgumentParser;

		std::vector<OptionStruct> opts
		({
			{'i', "input",  STRING, "Input script",                &xmlname,   std::string("script.xml")},
			{'n', "name",   STRING, "Name of the wall to process", &wname,     std::string("wall")},
			{'e', "epochs", INT,    "Number of sampling epochs",   &nepochs,   50},
			{'x', "xyz",    BOOL,   "Also dump .xyz files",        &needXYZ,   false}
		});

		ArgumentParser::Parser parser(opts, rank == 0);
		parser.parse(argc, argv);
	}

	logger.init(MPI_COMM_WORLD, "genwall.log", 6);

	pugi::xml_document config;
	pugi::xml_parse_result result = config.load_file(xmlname.c_str());
	if (!result)
		die("Couldn't open script file, parser says: \"%s\"", result.description());

	float3 globalDomainSize = config.child("simulation").child("domain").attribute("size").as_float3({32, 32, 32});
	int3 nranks3D = config.child("simulation").attribute("mpi_ranks").as_int3({1, 1, 1});

	auto genOne = [=] (pugi::xml_node wallNode, pugi::xml_node wallGenNode) {

		if (wallGenNode.type() == pugi::node_null)
			die("Wall %s has no generation instructions", wallNode.attribute("name").as_string());

		info("Generating wall %s", wallNode.attribute("name").as_string());


		auto sim = std::make_unique<Simulation>(nranks3D, globalDomainSize, MPI_COMM_WORLD, MPI_COMM_NULL);

		auto startingPV = std::make_unique<ParticleVector>("starting", 1.0);
		auto wallPV     = std::make_unique<ParticleVector>("wall", 1.0);
		auto final      = std::make_unique<ParticleVector>(wallGenNode.attribute("name").as_string("final"), 1.0);
		auto ic         = std::make_unique<UniformIC>     (wallGenNode.attribute("density").as_float(4));


		// Generate pv, but don't register it
		ic->exec(sim->getCartComm(), startingPV.get(), sim->domain, 0);

		// Create and setup wall
		auto wall = std::unique_ptr<Wall>( WallFactory::create(wallNode) );
		sim->registerWall(wall.get());

		// Produce new pv out of particles inside the wall
		freezeParticlesWrapper(wall.get(), startingPV.get(), wallPV.get(), -3, 4);
		sim->registerParticleVector(wallPV.get(), nullptr);

		// Sampler
		auto sampler = std::unique_ptr<Interaction>( createMCMCSamplerWrapper(wallGenNode, wall.get()) );
		sim->registerInteraction(sampler.get());
		sim->setInteraction(sampler->name, "wall", "wall");

		sim->init();
		sim->run(nepochs);

		freezeParticlesWrapper(wall.get(), wallPV.get(), final.get(), 0, 1.2);

		if (needXYZ)
		{
			writeXYZ(sim->getCartComm(), "wall.xyz", wallPV.get());
			writeXYZ(sim->getCartComm(), final->name+".xyz", final.get());
		}

		std::string path = wallGenNode.attribute("path").as_string("./");
		std::string command = "mkdir -p " + path;
		if (rank == 0)
		{
			if ( system(command.c_str()) != 0 )
				die("Could not create folders by given path %s", path.c_str());
		}

		PVview view(final.get(), final->local());
		const int nthreads = 128;
		SAFE_KERNEL_LAUNCH( zeroVels,
				getNblocks(view.size, nthreads), nthreads, 0, 0,
				view);

		final->checkpoint(sim->getCartComm(), path);

		sim->finalize();
	};

	for (auto node : config.child("simulation").children("wall"))
	{
		genOne(node, node.child("generate_frozen"));
	}
}
