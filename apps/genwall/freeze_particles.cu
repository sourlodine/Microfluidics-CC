#include "hip/hip_runtime.h"
#include "freeze_particles.h"

#include <core/logger.h>
#include <core/pvs/particle_vector.h>
#include <core/wall.h>
#include <core/cuda_common.h>
#include <core/sdf_kernels.h>


__global__ void countFrozen(const float4* pv, const int np, Wall::SdfInfo sdfInfo, float minSdf, float maxSdf, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coo = pv[2*pid];
	const float sdf = evalSdf(coo, sdfInfo);

	if (sdf > minSdf && sdf < maxSdf)
		atomicAggInc(nFrozen);
}

__global__ void collectFrozen(const float4* input, const int np, Wall::SdfInfo sdfInfo, float minSdf, float maxSdf,
		float4* frozen, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coo = input[2*pid];
	const float4 vel = input[2*pid+1];

	const float sdf = evalSdf(coo, sdfInfo);

	if (sdf > minSdf && sdf < maxSdf)
	{
		const int ind = atomicAggInc(nFrozen);
		frozen[2*ind] = coo;
		frozen[2*ind + 1] = make_float4(0.0f, 0.0f, 0.0f, vel.w);
	}
}

void freezeParticlesInWall(Wall* wall, ParticleVector* pv, ParticleVector* frozen, float minSdf, float maxSdf)
{
	CUDA_Check( hipDeviceSynchronize() );

	PinnedBuffer<int> nFrozen(1);

	nFrozen.clear(0);
	countFrozen<<< (pv->local()->size() + 127) / 128, 128, 0, 0 >>>(
			(float4*)pv->local()->coosvels.devPtr(), pv->local()->size(), wall->sdfInfo, minSdf, maxSdf, nFrozen.devPtr());
	nFrozen.downloadFromDevice(0);

	frozen->local()->resize(nFrozen[0], 0);
	frozen->mass = pv->mass;
	frozen->globalDomainStart = pv->globalDomainStart;
	frozen->localDomainSize = pv->localDomainSize;

	debug("Freezing %d particles", nFrozen[0]);

	nFrozen.clear(0);
	collectFrozen<<< (pv->local()->size() + 127) / 128, 128, 0, 0 >>>(
			(float4*)pv->local()->coosvels.devPtr(), pv->local()->size(), wall->sdfInfo, minSdf, maxSdf,
			(float4*)frozen->local()->coosvels.devPtr(), nFrozen.devPtr());
	nFrozen.downloadFromDevice(0);

	CUDA_Check( hipDeviceSynchronize() );
}
