#include "hip/hip_runtime.h"
#include "freeze_particles.h"

#include <core/logger.h>
#include <core/pvs/particle_vector.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

#include <core/walls/simple_stationary_wall.h>

#include <core/walls/stationary_walls/cylinder.h>
#include <core/walls/stationary_walls/sphere.h>
#include <core/walls/stationary_walls/plane.h>
#include <core/walls/stationary_walls/sdf.h>


template<bool QUERY, typename InsideWallChecker>
__global__ void collectFrozen(PVview view, float minVal, float maxVal, float4* frozen, int* nFrozen, InsideWallChecker checker)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= view.size) return;

	Particle p(view.particles, pid);
	p.u = make_float3(0);

	const float val = checker(p.r);

	if (val > minVal && val < maxVal)
	{
		const int ind = atomicAggInc(nFrozen);

		if (!QUERY)
			p.write2Float4(frozen, ind);
	}
}

template<typename InsideWallChecker>
void freezeParticlesInWall(const InsideWallChecker& checker, ParticleVector* pv, ParticleVector* frozen, float minVal, float maxVal)
{
	CUDA_Check( hipDeviceSynchronize() );

	PinnedBuffer<int> nFrozen(1);

	PVview view(pv, pv->local());
	const int nthreads = 128;
	const int nblocks = getNblocks(view.size, nthreads);

	nFrozen.clear(0);
	SAFE_KERNEL_LAUNCH(collectFrozen<true>,
				nblocks, nthreads, 0, 0,
				view, minVal, maxVal,
				(float4*)frozen->local()->coosvels.devPtr(), nFrozen.devPtr(), checker.handler());

	nFrozen.downloadFromDevice(0);

	frozen->local()->resize(nFrozen[0], 0);
	frozen->mass = pv->mass;
	frozen->domain = pv->domain;

	debug("Freezing %d particles", nFrozen[0]);

	nFrozen.clear(0);
	SAFE_KERNEL_LAUNCH(collectFrozen<false>,
			nblocks, nthreads, 0, 0,
			view, minVal, maxVal,
			(float4*)frozen->local()->coosvels.devPtr(), nFrozen.devPtr(), checker.handler());
	nFrozen.downloadFromDevice(0);

	CUDA_Check( hipDeviceSynchronize() );
}

void freezeParticlesWrapper(Wall* wall, ParticleVector* pv, ParticleVector* frozen, float minVal, float maxVal)
{
	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Cylinder>* >(wall);
		if (w != nullptr)
			freezeParticlesInWall<StationaryWall_Cylinder> (w->getChecker(), pv, frozen, minVal, maxVal);
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Sphere>* >(wall);
		if (w != nullptr)
			freezeParticlesInWall<StationaryWall_Sphere> (w->getChecker(), pv, frozen, minVal, maxVal);
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_SDF>* >(wall);
		if (w != nullptr)
			freezeParticlesInWall<StationaryWall_SDF> (w->getChecker(), pv, frozen, minVal, maxVal);
	}

	{
		auto w = dynamic_cast< SimpleStationaryWall<StationaryWall_Plane>* >(wall);
		if (w != nullptr)
			freezeParticlesInWall<StationaryWall_Plane> (w->getChecker(), pv, frozen, minVal, maxVal);
	}
}

