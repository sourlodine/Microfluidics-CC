#include "hip/hip_runtime.h"
#include <core/simulation.h>
#include <plugins/plugin.h>
#include <plugins/stats.h>
#include <plugins/dumpavg.h>
#include <core/xml/pugixml.hpp>
#include <core/wall.h>

Logger logger;

int main(int argc, char** argv)
{
	pugi::xml_document config;
	pugi::xml_parse_result result = config.load_file("poiseuille.xml");

	float3 globalDomainSize = config.child("simulation").child("domain").attribute("size").as_float3({32, 32, 32});
	int3 nranks3D{1, 2, 1};
	uDeviceX udevice(argc, argv, nranks3D, globalDomainSize, logger, "poiseuille.log", 9, false);

	SimulationPlugin  *simStat,  *simAvg;
	PostprocessPlugin *postStat, *postAvg;
	if (udevice.isComputeTask())
	{
		Integrator  constDP = createIntegrator(config.child("simulation").child("integrator"));
		Interaction dpdInt = createInteraction(config.child("simulation").child("interaction"));
		InitialConditions dpdIc = createIC(config.child("simulation").child("particle_vector"));
		Wall wall = createWall(config.child("simulation").child("wall"));

		ParticleVector* dpd = new ParticleVector(config.child("simulation").child("particle_vector").attribute("name").as_string());

		udevice.sim->registerParticleVector(dpd, &dpdIc);

		udevice.sim->registerIntegrator(&constDP);
		udevice.sim->registerInteraction(&dpdInt);
		udevice.sim->registerWall(&wall);

		udevice.sim->setIntegrator("dpd", "const_dp");
		udevice.sim->setInteraction("dpd", "dpd", "dpd_int");

		simStat = new SimulationStats("stats", 500);
		simAvg  = new Avg3DPlugin("averaging", "dpd", 10, 500, {24, 12, 24}, true, true, true);
	}
	else
	{
		postStat = new PostprocessStats("stats");
		postAvg = new Avg3DDumper("averaging", "xdmf/avgfields", nranks3D);
	}

	udevice.registerJointPlugins(simStat, postStat);
	udevice.registerJointPlugins(simAvg,  postAvg);
	udevice.run();

	return 0;
}
