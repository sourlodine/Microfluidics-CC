#include "hip/hip_runtime.h"
/*
 *  main.cu
 *  Part of uDeviceX/cuda-dpd-sem/sem/
 *
 *  Created and authored by Diego Rossinelli on 2014-07-29.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdlib>
#include <cmath>
#include <cstdio>
#include <cassert>

#include <algorithm>

#include <thrust/device_vector.h>

#include "cuda-sem.h"
#include "CellFactory.h"
#include "../hacks.h"

//__global__ void _update_pos(float * const xyzuvw, const float f, const int n, const float L)
//{
//    const int tid = threadIdx.x + blockDim.x * blockIdx.x;
//
//    if (tid < n)
//    {
//	for(int c = 0; c < 3; ++c)
//	{
//	    const float xold = xyzuvw[c + 6 * tid];
//
//	    float xnew = xold + f * xyzuvw[3 + c + 6 * tid];
//	    xnew -= L * floor((xnew + 0.5 * L) / L);
//
//	    xyzuvw[c + 6 * tid] = xnew;
//	}
//    }
//}
//
//__global__ void _update_vel(float * const xyzuvw, const float * const axayaz, const float f, const int n)
//{
//    const int tid = threadIdx.x + blockDim.x * blockIdx.x;
//
//    if (tid < n)
//    {
//	for(int c = 0; c < 3; ++c)
//	{
//	    const float vold = xyzuvw[3 + c + 6 * tid];
//
//	    float vnew = vold + f * axayaz[c + 3 * tid];
//
//	    xyzuvw[3 + c + 6 * tid] = vnew;
//	}
//    }
//}
//
//__global__ void _diag_kbt(const float * const xyzuvw, float * const diag, const int n)
//{
//    const int tid = threadIdx.x + blockDim.x * blockIdx.x;
//
//    if (tid < n)
//	diag[tid] =
//	    pow(xyzuvw[3 + 6 * tid], 2) +
//	    pow(xyzuvw[4 + 6 * tid], 2) +
//	    pow(xyzuvw[5 + 6 * tid], 2);
//}
//
//__global__ void _diag_p(const float * const xyzuvw, float * const diag, const int n, const int c)
//{
//    const int tid = threadIdx.x + blockDim.x * blockIdx.x;
//
//    if (tid < n)
//	diag[tid] = xyzuvw[3 + c + 6 * tid];
//}
//
//using namespace thrust;
//
//void vmd_xyz(const char * path, device_vector<float>& _xyzuvw, const int n, bool append)
//{
//    host_vector<float> xyzuvw(_xyzuvw);
//
//    FILE * f = fopen(path, append ? "a" : "w");
//
//    if (f == NULL)
//    {
//	printf("I could not open the file <%s>\n", path);
//	printf("Aborting now.\n");
//	abort();
//    }
//
//    fprintf(f, "%d\n", n);
//    fprintf(f, "mymolecule\n");
//
//    for(int i = 0; i < n; ++i)
//	fprintf(f, "1 %f %f %f\n",
//		(float)xyzuvw[0 + 6 * i],
//		(float)xyzuvw[1 + 6 * i],
//		(float)xyzuvw[2 + 6 * i]);
//
//    fclose(f);
//
//    printf("vmd_xyz: wrote to <%s>\n", path);
//}

void vmd_xyz_3comp(const char * path, float* xyz, const int n, bool append)
{
    FILE * f = fopen(path, append ? "a" : "w");

    if (f == NULL)
    {
	printf("I could not open the file <%s>\n", path);
	printf("Aborting now.\n");
	abort();
    }

    fprintf(f, "%d\n", n);
    fprintf(f, "mymolecule\n");

    for(int i = 0; i < n; ++i)
	fprintf(f, "1 %f %f %f\n",
		(float)xyz[0 + 3 * i],
		(float)xyz[1 + 3 * i],
		(float)xyz[2 + 3 * i]);

    fclose(f);

    printf("vmd_xyz: wrote to <%s>\n", path);
}

//class SimSEM
//{
//    const int n;
//    const float L;
//    device_vector<float> xyzuvw, axayaz, diag;
//
//public:
//
//    SimSEM(const int n, const int npd, const float L, const float h): n(n), L(L), xyzuvw(6 * n), axayaz(3 * n), diag(n)
//	{
//	    srand48(6516L);
//
//	    for(int i = 0; i < npd; ++i)
//		    for(int j = 0; j < npd; ++j)
//			    for(int k = 0; k < npd; ++k)
//			    {
//			    	int id = (i*npd +j) * npd + k;
//			    	if (id >= n) break;
//					xyzuvw[0 + 6 *id] = -h * npd * 0.5 + i*h;
//					xyzuvw[1 + 6 *id] = -h * npd * 0.5 + j*h;
//					xyzuvw[2 + 6 *id] = -h * npd * 0.5 + k*h;
//				}
//	}
//
//    void _diag(FILE ** fs, const int nfs, float t)
//	{
//	    _diag_kbt<<< (n + 127) / 128, 128 >>>(_ptr(xyzuvw), _ptr(diag), n);
//	    const float sv2 = reduce(diag.begin(), diag.end());
//	    float T = 0.5 * sv2 / (n * 3. / 2);
//
//	    float p[3];
//	    for(int c = 0; c < 3; ++c)
//	    {
//		_diag_p<<< (n + 127) / 128, 128 >>>(_ptr(xyzuvw), _ptr(diag), n, 0);
//		p[c] = reduce(diag.begin(), diag.end());
//	    }
//
//	    for(int i = 0; i < nfs; ++i)
//	    {
//		FILE * f = fs[i];
//
//		if (ftell(f) == 0)
//		    fprintf(f, "TIME\tkBT\tX-MOMENTUM\tY-MOMENTUM\tZ-MOMENTUM\n");
//
//		fprintf(f, "%s %+e\t%+e\t%+e\t%+e\t%+e\n", (f == stdout ? "DIAG:" : ""), t, T, p[0], p[1], p[2]);
//	    }
//	}
//
//     void _f(const float dt)
//	{
//	    //np,  rc,  LX, LY, LZ,  gamma, temp, dt,   u0,    rho,  req, D
//	    //1e3, 1.0, 10, 10, 10,  80,    0.1,  0.01, 0.001, 1.5,  0.85, 0.0001
//	    const float rcutoff = 2.5, gamma = 20, temp = 0.1, u0 = 0.018, rho = 1.5, req = 0.85, D = .01, rc = 1;
//
//	    forces_sem_cuda_direct_nohost(_ptr(xyzuvw), _ptr(axayaz),
//		    n, rcutoff, L, L, L, gamma, temp, dt, u0, rho, req, D, rc);
//	};
//
//    void run(const double tend, const double dt)
//	{
//	    vmd_xyz("ic.xyz", xyzuvw, n, false);
//
//	    FILE * fdiags[2] = {stdout, fopen("diag.txt", "w") };
//
//	    const size_t nt = (int)(tend / dt);
//
//	    _f(dt);
//
//	    for(int it = 0; it < nt; ++it)
//	    {
//		if (it % 200 == 0)
//		{
//		    float t = it * dt;
//		    _diag(fdiags, 2, t);
//		}
//
//		_update_vel<<<(n + 127) / 128, 128>>>(_ptr(xyzuvw), _ptr(axayaz), dt * 0.5, n);
//
//		_update_pos<<<(n + 127) / 128, 128>>>(_ptr(xyzuvw), dt, n, L);
//
//		_f(dt);
//
//		_update_vel<<<(n + 127) / 128, 128>>>(_ptr(xyzuvw), _ptr(axayaz), dt * 0.5, n);
//
//		if (it % 200 == 0)
//		    vmd_xyz("evolution.xyz", xyzuvw, n, it > 0);
//	    }
//
//	    fclose(fdiags[1]);
//	}
//};

int main()
{
    printf("hello gpu only test\n");
    
    float L = 15; //  /Volumes/Phenix/CTC/cuda-dpd-sem/sem/evolution.xyz

    const float Nm = 0.25;
    const int npd = 10;
    const int n = npd * npd * npd;

    //SimSEM sim(n, npd, L, 1.0);
       
    //sim.run(250 * 4, 0.01);

    float* xyz = new float[3*n];

    CellParams params;
    produceCell(n, xyz, params);

    printf("Params:  cutoff: %f,  gamma: %f,  u0: %f,  rho: %f,  req: %f,  D: %f,  rc: %f\n",
    		params.rcutoff, params.gamma, params.u0, params.rho, params.req, params.D, params.rc);

    vmd_xyz_3comp("final.xyz", xyz, n, false);
    
    return 0;
}
