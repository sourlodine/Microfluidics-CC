#include "hip/hip_runtime.h"
/*
 *  cuda-dpd-bipartite.cu
 *  Part of uDeviceX/cuda-dpd-sem/dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-07-28.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cassert>

#include "../dpd-rng.h"
#include "cuda-dpd.h"

struct BipartiteInfoDPD
{
	int3 ncells;
	float3 domainsize, invdomainsize, domainstart;
	float invrc, aij, gamma, sigmaf;
};

__constant__ BipartiteInfoDPD bipart_info;

#ifndef NDEBUG
//#define _CHECK_
#endif

#define COLS 8
#define ROWS (32 / COLS)
#define CPB 4

#include "../hacks.h"

__global__
void _bipartite_dpd_directforces(float * const axayaz, const int np, const int np_src,
		const float seed, const int mask, const float * xyzuvw, const float * xyzuvw_src,
		const float invrc, const float aij, const float gamma, const float sigmaf)
{
	assert(blockDim.x % warpSize == 0);
	assert(blockDim.x * gridDim.x >= np);

	const int tid = threadIdx.x % warpSize;
	const int pid = threadIdx.x + blockDim.x * blockIdx.x;
	const bool valid = pid < np;

	float xp, yp, zp, up, vp, wp;

	if (valid)
	{
		xp = xyzuvw[0 + pid * 6];
		yp = xyzuvw[1 + pid * 6];
		zp = xyzuvw[2 + pid * 6];
		up = xyzuvw[3 + pid * 6];
		vp = xyzuvw[4 + pid * 6];
		wp = xyzuvw[5 + pid * 6];
	}

	float xforce = 0, yforce = 0, zforce = 0;

	for(int s = 0; s < np_src; s += warpSize)
	{
		float my_xq, my_yq, my_zq, my_uq, my_vq, my_wq;

		const int batchsize = min(warpSize, np_src - s);

		if (tid < batchsize)
		{
			my_xq = xyzuvw_src[0 + (tid + s) * 6];
			my_yq = xyzuvw_src[1 + (tid + s) * 6];
			my_zq = xyzuvw_src[2 + (tid + s) * 6];
			my_uq = xyzuvw_src[3 + (tid + s) * 6];
			my_vq = xyzuvw_src[4 + (tid + s) * 6];
			my_wq = xyzuvw_src[5 + (tid + s) * 6];
		}

		for(int l = 0; l < batchsize; ++l)
		{
			const float xq = __shfl(my_xq, l);
			const float yq = __shfl(my_yq, l);
			const float zq = __shfl(my_zq, l);
			const float uq = __shfl(my_uq, l);
			const float vq = __shfl(my_vq, l);
			const float wq = __shfl(my_wq, l);

			//necessary to force the execution shuffles here below
			//__syncthreads();

			//if (valid)
			{
				const float _xr = xp - xq;
				const float _yr = yp - yq;
				const float _zr = zp - zq;

				const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

				const float invrij = rsqrtf(rij2);

				const float rij = rij2 * invrij;
				const float argwr = max((float)0, 1 - rij * invrc);
				const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

				const float xr = _xr * invrij;
				const float yr = _yr * invrij;
				const float zr = _zr * invrij;

				const float rdotv =
						xr * (up - uq) +
						yr * (vp - vq) +
						zr * (wp - wq);

				const int spid = s + l;
				const int dpid = pid;

				const int arg1 = mask * dpid + (1 - mask) * spid;
				const int arg2 = mask * spid + (1 - mask) * dpid;
				const float myrandnr = Logistic::mean0var1(seed, arg1, arg2);

				const float strength = aij * argwr + (- gamma * wr * rdotv + sigmaf * myrandnr) * wr;
				//if (valid && spid < np_src)
				{
					xforce += strength * xr;
					yforce += strength * yr;
					zforce += strength * zr;
				}
			}
		}
	}

	if (valid)
	{
		assert(!isnan(xforce));
		assert(!isnan(yforce));
		assert(!isnan(zforce));

		axayaz[0 + 3 * pid] = xforce;
		axayaz[1 + 3 * pid] = yforce;
		axayaz[2 + 3 * pid] = zforce;
	}
}

void directforces_dpd_cuda_bipartite_nohost(
		const float * const xyzuvw, float * const axayaz, const int np,
		const float * const xyzuvw_src, const int np_src,
		const float aij, const float gamma, const float sigma, const float invsqrtdt,
		const float seed, const int mask, hipStream_t stream)
{
	if (np == 0 || np_src == 0)
	{
		printf("warning: directforces_dpd_cuda_bipartite_nohost called with ZERO!\n");
		return;
	}

	_bipartite_dpd_directforces<<<(np + 127) / 128, 128, 0, stream>>>(axayaz, np, np_src, seed, mask,
			xyzuvw, xyzuvw_src, 1, aij, gamma, sigma * invsqrtdt);

	CUDA_CHECK(hipPeekAtLastError());
}

__global__ __launch_bounds__(32 * CPB, 16) 
void _dpd_bipforces(const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
		hipTextureObject_t texSrcStart,  hipTextureObject_t texSrcParticles, const int np_src, const int3 halo_ncells,
		const float aij, const float gamma, const float sigmaf,
		const float seed, const int mask, float * const axayaz)
{
	assert(warpSize == COLS * ROWS);
	assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
	assert(ROWS * 3 <= warpSize);

	const int tid = threadIdx.x;
	const int subtid = tid % COLS;
	const int slot = tid / COLS;
	const int wid = threadIdx.y;

	__shared__ int volatile starts[CPB][32], scan[CPB][32];

	const int mycid = blockIdx.x * CPB + threadIdx.y;

	if (mycid >= halo_ncells.x * halo_ncells.y * halo_ncells.z)
		return;

	int mycount = 0, myscan = 0;
	if (tid < 27)
	{
		const int dx = (tid) % 3;
		const int dy = ((tid / 3)) % 3;
		const int dz = ((tid / 9)) % 3;

		int xcid = (mycid % halo_ncells.x) + dx - 1;
		int ycid = ((mycid / halo_ncells.x) % halo_ncells.y) + dy - 1;
		int zcid = ((mycid / halo_ncells.x / halo_ncells.y) % halo_ncells.z) + dz - 1;

		const bool valid_cid =
				xcid >= 0 && xcid < halo_ncells.x &&
				ycid >= 0 && ycid < halo_ncells.y &&
				zcid >= 0 && zcid < halo_ncells.z ;

		xcid = min(halo_ncells.x - 1, max(0, xcid));
		ycid = min(halo_ncells.y - 1, max(0, ycid));
		zcid = min(halo_ncells.z - 1, max(0, zcid));

		const int cid = max(0, xcid + halo_ncells.x * (ycid + halo_ncells.y * zcid));

		starts[wid][tid] = tex1Dfetch<int>(texSrcStart, cid);

		myscan = mycount = valid_cid * (tex1Dfetch<int>(texSrcStart, cid + 1) - tex1Dfetch<int>(texSrcStart, cid));
	}

	for(int L = 1; L < 32; L <<= 1)
		myscan += (tid >= L) * __shfl_up(myscan, L) ;

	if (tid < 28)
		scan[wid][tid] = myscan - mycount;

	const int dststart = tex1Dfetch<int>(texDstStart, mycid);
	const int nsrc = scan[wid][27], ndst = tex1Dfetch<int>(texDstStart, mycid + 1) - tex1Dfetch<int>(texDstStart, mycid);

	for(int d = 0; d < ndst; d += ROWS)
	{
		const int np1 = min(ndst - d, ROWS);

		const int dpid = dststart + d + slot;

		const int entry = 3 * dpid;
		float2 dtmp0 = xyzuvw[entry];
		float2 dtmp1 = xyzuvw[entry + 1];
		float2 dtmp2 = xyzuvw[entry + 2];

		float f[3] = {0, 0, 0};

		for(int s = 0; s < nsrc; s += COLS)
		{
			const int np2 = min(nsrc - s, COLS);
			const int pid = s + subtid;
			const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
			const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
			const int key = key9 + key3;
			const int spid = pid - scan[wid][key] + starts[wid][key];
			const int sentry = 3 * spid;

			const float2 stmp0 = tex1Dfetch<float2>(texSrcParticles, sentry);
			const float2 stmp1 = tex1Dfetch<float2>(texSrcParticles, sentry + 1);
			const float2 stmp2 = tex1Dfetch<float2>(texSrcParticles, sentry + 2);

			{
				const float xforce = f[0];
				const float yforce = f[1];
				const float zforce = f[2];

				const float _xr = dtmp0.x - stmp0.x;
				const float _yr = dtmp0.y - stmp0.y;
				const float _zr = dtmp1.x - stmp1.x;

				const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
				const float invrij = rsqrtf(rij2);
				const float rij = rij2 * invrij;
				const float argwr = max((float)0, 1 - rij);
				const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

				const float xr = _xr * invrij;
				const float yr = _yr * invrij;
				const float zr = _zr * invrij;

				const float rdotv =
						xr * (dtmp1.y - stmp1.y) +
						yr * (dtmp2.x - stmp2.x) +
						zr * (dtmp2.y - stmp2.y);

				const int arg1 = mask * dpid + (1 - mask) * spid;
				const int arg2 = mask * spid + (1 - mask) * dpid;
				const float myrandnr = Logistic::mean0var1(seed, arg1, arg2);

				const float strength = aij * argwr + (- gamma * wr * rdotv + sigmaf * myrandnr) * wr;
				const bool valid = (slot < np1) && (subtid < np2);

				assert( (dpid >= 0 && dpid < np && spid >= 0 && spid < np_src) || ! valid);

				if (valid)
				{
					f[0] = xforce + strength * xr;
					f[1] = yforce + strength * yr;
					f[2] = zforce + strength * zr;
				}
			}
		}

		for(int L = COLS / 2; L > 0; L >>=1)
			for(int c = 0; c < 3; ++c)
				f[c] += __shfl_xor(f[c], L);

		const float fcontrib = f[subtid % 3];
		const int dstpid = dststart + d + slot;
		const int c = (subtid % 3);

		if (slot < np1)
			axayaz[c + 3 * dstpid] = fcontrib;
	}
}

void forces_dpd_cuda_bipartite_nohost(hipStream_t stream, const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
		hipTextureObject_t texSrcStart, hipTextureObject_t texSrcParticles, const int np_src,
		const int3 halo_ncells,
		const float aij, const float gamma, const float sigmaf,
		const float seed, const int mask, float * const axayaz)
{ 
	const int ncells = halo_ncells.x * halo_ncells.y * halo_ncells.z;

	static bool fbip_init = false;

	if (!fbip_init)
	{
		CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(_dpd_bipforces), hipFuncCachePreferL1));

		fbip_init = true;
	}

	_dpd_bipforces<<<(ncells + CPB - 1) / CPB, dim3(32, CPB), 0, stream>>>(
			xyzuvw, np, texDstStart, texSrcStart, texSrcParticles, np_src,
			halo_ncells, aij, gamma, sigmaf, seed, mask,
			axayaz);
}
