#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_vector_types.h>
#include <hip/hip_fp16.h>
#include <cassert>
#include "../tiny-float.h"

#include "cuda-dpd.h"
#include "../dpd-rng.h"

__device__ __forceinline__ float3 readCoosFromAll4(const float4* xyzouvwo, int pid)
{
	const float4 tmp = xyzouvwo[2*pid];

	return make_float3(tmp.x, tmp.y, tmp.z);
}

__device__ __forceinline__ void readAll4(const float4* xyzouvwo, int pid, float3& coo, float3& vel)
{
	const float4 tmp1 = xyzouvwo[pid*2];
	const float4 tmp2 = xyzouvwo[pid*2+1];

	coo = make_float3(tmp1.x, tmp1.y, tmp1.z);
	vel = make_float3(tmp2.x, tmp2.y, tmp2.z);
}

__device__ __forceinline__ int getCellId(const float x, const float start, const float invrc, const int ncells)
{
	const float v = invrc * (x - start);
	const float robustV = min(min(floor(v), floor(v - 1.0e-6f)), floor(v + 1.0e-6f));
	return min(ncells - 1, max(0, (int)robustV));
}

__device__ __forceinline__ float sqr(float x)
{
	return x*x;
}

template<typename Ta, typename Tb>
__device__ __forceinline__ float distance2(const Ta a, const Tb b)
{
	return sqr(a.x - b.x) + sqr(a.y - b.y) + sqr(a.z - b.z);
}

//__launch_bounds__(128, 16)
template<typename Interaction>
__global__ void computeSelfInteractions(const float4 * __restrict__ xyzouvwo, float* axayaz, const int * __restrict__ cellsstart,
		int3 ncells, float3 domainStart, int ncells_1, int np, Interaction interaction)
{
	const int dstId = blockIdx.x*blockDim.x + threadIdx.x;
	if (dstId >= np) return;

	float3 dstCoo, dstVel;
	float3 dstAcc = make_float3(0.0f);
	readAll4(xyzouvwo, dstId, dstCoo, dstVel);

	const int cellX0 = getCellId(dstCoo.x, domainStart.x, 1.0f, ncells.x);
	const int cellY0 = getCellId(dstCoo.y, domainStart.y, 1.0f, ncells.y);
	const int cellZ0 = getCellId(dstCoo.z, domainStart.z, 1.0f, ncells.z);

#pragma unroll
	for (int cellY = cellY0-1; cellY <= cellY0; cellY++)
		for (int cellZ = cellZ0-1; cellZ <= cellZ0+1; cellZ++)
		{
			if ( !(cellY >= 0 && cellY < ncells.y && cellZ >= 0 && cellZ < ncells.z) ) continue;
			if (cellY == cellY0 && cellZ > cellZ0) continue;

			const int midCellId = (cellZ*ncells.y + cellY)*ncells.x + cellX0;
			int rowStart  = max(midCellId-1, 0);
			int rowEnd    = min(midCellId+2, ncells_1);
			if ( cellY == cellY0 && cellZ == cellZ0 ) rowEnd = midCellId + 1; // this row is already partly covered

			const int pstart = cellsstart[rowStart];
			const int pend   = cellsstart[rowEnd];

			for (int srcId = pstart; srcId < pend; srcId ++)
			{
				const float3 srcCoo = readCoosFromAll4(xyzouvwo, srcId);

				bool interacting = distance2(srcCoo, dstCoo) < 1.00f;
				if (dstId <= srcId && cellY == cellY0 && cellZ == cellZ0) interacting = false;

				if (interacting)
				{
					float3 srcCoo, srcVel;
					readAll4(xyzouvwo, srcId, srcCoo, srcVel);

					float3 frc = interaction(dstCoo, dstVel, dstId, srcCoo, srcVel, srcId);

					dstAcc += frc;

					float* dest = axayaz + srcId*3;
					atomicAdd(dest,     -frc.x);
					atomicAdd(dest + 1, -frc.y);
					atomicAdd(dest + 2, -frc.z);
				}
			}
		}

	float* dest = axayaz + dstId*3;
	atomicAdd(dest,     dstAcc.x);
	atomicAdd(dest + 1, dstAcc.y);
	atomicAdd(dest + 2, dstAcc.z);
}


__device__ __forceinline__ float3 dpd_interaction(
		const float3 dstCoo, const float3 dstVel, const int dstId,
		const float3 srcCoo, const float3 srcVel, const int srcId,
		float adpd, float gammadpd, float sigmadpd, float seed)
{
	const float _xr = dstCoo.x - srcCoo.x;
	const float _yr = dstCoo.y - srcCoo.y;
	const float _zr = dstCoo.z - srcCoo.z;
	const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
	if (rij2 > 1.0f) return make_float3(0.0f);

	const float invrij = rsqrtf(rij2);
	const float rij = rij2 * invrij;
	const float argwr = 1.0f - rij;
	const float wr = viscosity_function<0>(argwr);

	const float xr = _xr * invrij;
	const float yr = _yr * invrij;
	const float zr = _zr * invrij;

	const float rdotv =
			xr * (dstVel.x - srcVel.x) +
			yr * (dstVel.y - srcVel.y) +
			zr * (dstVel.z - srcVel.z);

	const float myrandnr = 0*Logistic::mean0var1(seed, min(srcId, dstId), max(srcId, dstId));

	const float strength = adpd * argwr - (gammadpd * wr * rdotv + sigmadpd * myrandnr) * wr;

	return make_float3(strength * xr, strength * yr, strength * zr);
}


template<typename T>
struct SimpleDeviceBuffer
{
	int capacity, size;

	T * data;

	SimpleDeviceBuffer(int n = 0): capacity(0), size(0), data(NULL) { resize(n);}

	~SimpleDeviceBuffer()
	{
		if (data != NULL)
			hipFree(data);

		data = NULL;
	}

	void dispose()
	{
		if (data != NULL)
			hipFree(data);

		data = NULL;
	}

	void resize(const int n)
	{
		assert(n >= 0);

		size = n;

		if (capacity >= n)
			return;

		if (data != NULL)
			hipFree(data);

		const int conservative_estimate = (int)ceil(1.1 * n);
		capacity = 128 * ((conservative_estimate + 129) / 128);

		hipMalloc(&data, sizeof(T) * capacity);

#ifndef NDEBUG
		hipMemset(data, 0, sizeof(T) * capacity);
#endif
	}
};



void forces_dpd_cuda_nohost( const float * const xyzuvw, const float4 * const xyzouvwo, const ushort4 * const xyzo_half, float * const axayaz,  const int np,
		const int * const cellsstart, const int * const cellscount,
		const float rc,
		const float XL, const float YL, const float ZL,
		const float adpd,
		const float gammadpd,
		const float sigmadpd,
		const float invsqrtdt,
		const float seed, hipStream_t stream )
{
	const int nx = round(XL / rc);
	const int ny = round(YL / rc);
	const int nz = round(ZL / rc);

	auto dpdInt = [=] __device__ ( const float3 dstCoo, const float3 dstVel, const int dstId,
					   const float3 srcCoo, const float3 srcVel, const int srcId) {
		return dpd_interaction(dstCoo, dstVel, dstId, srcCoo, srcVel, srcId,
			adpd, gammadpd, sigmadpd*invsqrtdt, seed);
	};

	hipFuncSetCacheConfig( computeSelfInteractions<decltype(dpdInt)>, hipFuncCachePreferL1 );

	hipMemsetAsync( axayaz, 0, sizeof( float )* np * 3, stream );
	const int nth = 128;
	computeSelfInteractions<<< (np + nth - 1) / nth, nth, 0, stream >>>(xyzouvwo, axayaz, cellsstart,
			make_int3(nx, ny, nz), make_float3(-nx/2, -ny/2, -nz/2), nx*ny*nz+1, np, dpdInt);
}




