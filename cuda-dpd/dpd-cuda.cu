#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

struct InfoDPD
{
    int nx, ny, nz, np, nsamples, rsamples_start;
    float XL, YL, ZL;
    float xstart, ystart, zstart, rc, invrc, aij, gamma, sigma, invsqrtdt, sigmaf;
    float * xp, *yp, *zp, *xv, *yv, *zv, *xa, *ya, *za, *rsamples;
    int * starts;
};

__constant__ InfoDPD info;

__global__ void pid2code(int * codes, int * pids)
{
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;

    if (pid >= info.np)
	return;

    const float x = (info.xp[pid] - info.xstart) / info.rc;
    const float y = (info.yp[pid] - info.ystart) / info.rc;
    const float z = (info.zp[pid] - info.zstart) / info.rc;
    
    int ix = (int)floor(x);
    int iy = (int)floor(y);
    int iz = (int)floor(z);
    
    if( !(ix >= 0 && ix < info.nx) ||
	!(iy >= 0 && iy < info.ny) ||
	!(iz >= 0 && iz < info.nz))
	printf("pid %d: oops %f %f %f -> %d %d %d\n", pid, x, y, z, ix, iy, iz);
#if 0 
    assert(ix >= 0 && ix < info.nx);
    assert(iy >= 0 && iy < info.ny);
    assert(iz >= 0 && iz < info.nz);
#else
    ix = max(0, min(info.nx - 1, ix));
    iy = max(0, min(info.ny - 1, iy));
    iz = max(0, min(info.nz - 1, iz));
#endif
    
    codes[pid] = ix + info.nx * (iy + info.nx * iz);
    pids[pid] = pid;
};

__global__ void _gather(const float * input, const int * indices, float * output, const int n)
{
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n)
	output[tid] = input[indices[tid]];
}

const int xbs = 16;
const int ybs = 4;
const int xts = xbs;
const int yts = 8;

__device__ void _ftable(
    float p1[3][yts], float p2[3][xts], float v1[3][yts], float v2[3][xts],
    const int np1, const int np2, const int nonzero_start, const int rsamples_start,
    float a1[3][yts], float a2[3][xts])
{
    assert(np2 <= xts);
    assert(np1 <= yts);
    assert(np1 <= xbs * ybs);
    assert(blockDim.x == xbs && xbs == xts);
    assert(blockDim.y == ybs);

    if (threadIdx.x == 0 && threadIdx.y == 0)\
	printf("calling ftable for %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
    
    __shared__ float forces[3][yts][xts];

    const int lx = threadIdx.x;

    if (lx < np2)
	for(int ly = threadIdx.y; ly < np1; ly += blockDim.y)
	{
	    assert(lx < np2 && ly < np1);
	
	    forces[0][ly][lx] = forces[1][ly][lx] = forces[2][ly][lx] = 0;
	
	    if (lx > ly + nonzero_start)
	    {
		float xr = p1[0][ly] - p2[0][lx];
		float yr = p1[1][ly] - p2[1][lx];
		float zr = p1[2][ly] - p2[2][lx];
				
		xr -= info.XL * floorf(0.5f + xr / info.XL);
		yr -= info.YL * floorf(0.5f + yr / info.YL);
		zr -= info.ZL * floorf(0.5f + zr / info.ZL);

		const float rij2 = xr * xr + yr * yr + zr * zr;
		const float invrij = rsqrtf(rij2);
		const float rij = rij2 * invrij;
		const float wr = max((float)0, 1 - rij * info.invrc);
	
		xr *= invrij;
		yr *= invrij;
		zr *= invrij;

		const float rdotv = xr * (v1[0][ly] - v2[0][lx]) + yr * (v1[1][ly] - v2[1][lx]) + zr * (v1[2][ly] - v2[2][lx]);

		int entry = lx + np2 * ly;
		const float myrandnr = info.rsamples[(info.rsamples_start + rsamples_start + entry) % info.nsamples];
#if 1
		assert(myrandnr != -313);
		info.rsamples[(info.rsamples_start + rsamples_start + entry) % info.nsamples] = -313;
#endif

		const float strength = (info.aij - info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;
#if 1
		forces[0][ly][lx] = rij2 < 1;
		forces[1][ly][lx] = 0;
		forces[2][ly][lx] = 0;
#else
		forces[0][ly][lx] = strength * xr;
		forces[1][ly][lx] = strength * yr;
		forces[2][ly][lx] = strength * zr;
#endif
	    }
	}

    __syncthreads();

    if (threadIdx.y == 0 && lx < np2)
    {
	a2[0][lx] = a2[1][lx] = a2[2][lx] = 0;
	
	for(int iy = 0; iy < np1; ++iy)
	{
	    assert(lx < np2 && iy < np1);
	    a2[0][lx] += forces[0][iy][lx];
	    a2[1][lx] += forces[1][iy][lx];
	    a2[2][lx] += forces[2][iy][lx];
	}
    }

    if (lx == 0)
	for(int ly = threadIdx.y; ly < np1; ly += blockDim.y)
	{
	    for(int ix = 0; ix < np2; ++ix)
	    {
#if 1
		assert(ix < np2 && ly < np1);
		a1[0][ly] += forces[0][ly][ix];
		a1[1][ly] += forces[1][ly][ix];
		a1[2][ly] += forces[2][ly][ix];
#else
		a1[0][ly] -= forces[0][ly][ix];
		a1[1][ly] -= forces[1][ly][ix];
		a1[2][ly] -= forces[2][ly][ix];
#endif
	    }
	}
}

__device__ void _cellcells(const int p1start, const int p1count, const int p2start[4], const int p2counts[4],
				 const bool self, int rsamples_start,
				 float * const xa, float * const ya, float * const za)
{ 
    __shared__ float
	p1[3][yts], p2[3][xts],
	v1[3][yts], v2[3][xts],
	a1[3][yts], a2[3][xts];

    const int lx = threadIdx.x;
    const int ly = threadIdx.y;

    const bool master = lx + ly == 0;

    __shared__ int scan[5];

    if (master)
    {
	scan[0] = 0;
	for(int i = 1; i < 5; ++i)
	    scan[i] = scan[i - 1] + p2counts[i - 1];
    }

    __syncthreads();

    const int p2count = scan[4];
    
    for(int ty = 0; ty < p1count; ty += yts)
    {
	const int np1 = min(yts, p1count - ty);
	
	if (master)
	    for(int s = ty, d = 0; d < np1; ++s, ++d)
	    {
		assert(d < yts);
		p1[0][d] = info.xp[p1start + s];
		p1[1][d] = info.yp[p1start + s];
		p1[2][d] = info.zp[p1start + s];

		v1[0][d] = info.xv[p1start + s];
		v1[1][d] = info.yv[p1start + s];
		v1[2][d] = info.zv[p1start + s];

		a1[0][d] = a1[1][d] = a1[2][d] = 0;
	    }
	
	for(int tx = 0; tx < p2count; tx += xts)
	{
	    const int np2 = min(xts, p2count - tx);
	    
	    if (self && !(tx + xts - 1 > ty))
	    	continue;
	    
	    if (master)
		for(int s = tx, d = 0; d < np2; ++s, ++d)
		{
		    assert(d < xts);
		    const int entry = (s >= scan[1]) + (s >= scan[2]) + (s >= scan[3]);
		    assert(scan[entry + 1] > s && s >= scan[entry]);

		    const int pid = s - scan[entry] + p2start[entry];

		    p2[0][d] = info.xp[pid];
		    p2[1][d] = info.yp[pid];
		    p2[2][d] = info.zp[pid];
		      		
		    v2[0][d] = info.xv[pid];
		    v2[1][d] = info.yv[pid];
		    v2[2][d] = info.zv[pid];
		}

	    __syncthreads();

	    _ftable(p1, p2, v1, v2, np1, np2, self ? ty - tx : - p1count, rsamples_start, a1, a2);

	    __syncthreads();
	    
	    if (master)
		rsamples_start += np1 * np2;

	    if (master)
		for(int s = 0, d = tx; s < np2; ++s, ++d)
		{
		    assert(s < xts);
		    const int entry = (d >= scan[1]) + (d >= scan[2]) + (d >= scan[3]);
		    assert(scan[entry + 1] > d && d >= scan[entry]);

		    const int pid = d - scan[entry] + p2start[entry];

		    xa[pid] += a2[0][s];
		    ya[pid] += a2[1][s]; 
		    za[pid] += a2[2][s]; 
		}
	}

	if (master)
	    for(int s = 0, d = ty; s < np1; ++s, ++d)
	    {
		assert(s < yts);
		xa[p1start + d] += a1[0][s];
		ya[p1start + d] += a1[1][s];
		za[p1start + d] += a1[2][s];
	    }
    }
}

__device__ int _cid(int shiftcode)
{
    int3 indx = make_int3(blockIdx.x + (shiftcode & 1),
			  blockIdx.y + ((shiftcode >> 1) & 1),
			  blockIdx.z + ((shiftcode >> 2) & 1));

    indx.x = (indx.x + info.nx) % info.nx;
    indx.y = (indx.y + info.ny) % info.ny;
    indx.z = (indx.z + info.nz) % info.nz;

    return indx.x + info.nx * (indx.y + info.ny * indx.z);
}

__constant__ int edgeslutcount[4] = {4, 4, 3, 3};
__constant__ int edgeslutstart[4] = {0, 4, 8, 11};
__constant__ int edgeslut[14] = {0, 1, 2, 7, 2, 4, 6, 7, 4, 5, 7, 4, 0, 7};

__global__ void _dpd_forces(float * tmp, int * consumed)
{
    const int idbuf = (blockIdx.x & 1) | ((blockIdx.y & 1) << 1) | ((blockIdx.z & 1) << 2);

    float * const xa = tmp + info.np * (idbuf + 8 * 0);
    float * const ya = tmp + info.np * (idbuf + 8 * 1);
    float * const za = tmp + info.np * (idbuf + 8 * 2);
    
    const bool master = threadIdx.x + threadIdx.y == 0;
   
    __shared__ int offsetrsamples, rconsumption;
    __shared__ int p2starts[4], p2counts[4];
    
    for(int i = 0; i < 4; ++i)
    {
	const int cid1 = _cid(i);
	const int s1 = info.starts[cid1];
	const int e1 = info.starts[cid1 + 1];
	
	const int nentries = edgeslutcount[i];
	const int entrystart = edgeslutstart[i];

	if (master)
	{
	    rconsumption = 0;
	    for(int j = 0; j < nentries; ++j)
	    {
		const int cid2 = _cid(edgeslut[j + entrystart]);
		assert(!(cid1 == cid2) || i == 0 && j == 0);

		const int s2 = info.starts[cid2];
		const int e2 = info.starts[cid2 + 1];
	     		
		p2starts[j] = s2;
		p2counts[j] = e2 - s2;

		rconsumption += (e1 - s1) * (e2 - s2); 
	    }

	    for(int j = nentries; j < 4; ++j)
		p2starts[j] = p2counts[j] = 0;
	    
	    offsetrsamples = atomicAdd(consumed, rconsumption);
	}

	__syncthreads();

	if (offsetrsamples + rconsumption >= info.nsamples)
	//running out of samples. this is bad.
	    return;

	_cellcells(s1, e1 - s1, p2starts, p2counts, i == 0, offsetrsamples, xa, ya, za);
    }
}

__global__ void _reduce(float * tmp)
{
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;

    if (pid < info.np)
    {
	float xa = 0;
	for(int idbuf = 0; idbuf < 8; ++idbuf)
	    xa += tmp[pid + info.np * (idbuf + 8 * 0)];

	float ya = 0;
	for(int idbuf = 0; idbuf < 8; ++idbuf)
	    ya += tmp[pid + info.np * (idbuf + 8 * 1)];
	
	float za = 0;	
    	for(int idbuf = 0; idbuf < 8; ++idbuf)
	    za += tmp[pid + info.np * (idbuf + 8 * 2)];

	info.xa[pid] = xa;
	info.ya[pid] = ya;
	info.za[pid] = za;
    }
}

#include <cmath>
#include <unistd.h>

#include <hiprand.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>

using namespace thrust;

#define CUDA_CHECK(ans) do { cudaAssert((ans), __FILE__, __LINE__); } while(0)
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
	fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	sleep(5);
	if (abort) exit(code);
    }
}

template<typename T> T * _ptr(device_vector<T>& v) { return raw_pointer_cast(v.data()); }

void _reorder(device_vector<float>& v, device_vector<int>& indx)
{
    device_vector<float> tmp(v.begin(), v.end());
   
    _gather<<<(v.size() + 127) / 128, 128>>>(_ptr(tmp), _ptr(indx), _ptr(v), v.size());

    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

class RRingBuffer
{
    const int n;
    int s, c, olds;
    float * drsamples;
    hiprandGenerator_t prng;

protected:

    void _refill(int s, int e)
	{
	    assert(e > s && e <= n);
	    
	    const int multiple = 2;

	    s = s - (s % multiple);
	    e = e + (multiple - (e % multiple));
	    e = min(e, n);
	    
	    hiprandStatus_t res;
	    res = hiprandGenerateNormal(prng, drsamples + s, e - s, 0, 1);
	    assert(res == HIPRAND_STATUS_SUCCESS);
	}
    
public:

    RRingBuffer(const int n): n(n), s(0), olds(0), c(0)
	{
	    hiprandStatus_t res;
	    res = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	    //we could try HIPRAND_RNG_PSEUDO_MTGP32 or HIPRAND_RNG_PSEUDO_MT19937
	    
	    assert(res == HIPRAND_STATUS_SUCCESS);
	    res = hiprandSetPseudoRandomGeneratorSeed(prng, 1234ULL);
	    assert(res == HIPRAND_STATUS_SUCCESS);
	    
	    CUDA_CHECK(hipMalloc(&drsamples, sizeof(float) * n));

	    update(n);
	    assert(s == 0);
	}

    ~RRingBuffer()
	{
	    CUDA_CHECK(hipFree(drsamples));
	    hiprandStatus_t res = hiprandDestroyGenerator(prng);
	    assert(res == HIPRAND_STATUS_SUCCESS);
	}
    
    void update(const int consumed)
	{
	    assert(consumed >= 0 && consumed <= n);

	    c += consumed;
	    assert(c >= 0 && c <= n);
	    
	    if (c > 0.45 * n)
	    {
		const int c1 = min(olds + c, n) - olds;
	    
		if (c1 > 0)
		    _refill(olds, olds + c1);

		const int c2 = c - c1;

		if (c2 > 0)
		    _refill(0, c2);
	    
		olds = (olds + c) % n;
		s = olds;
		c = 0;
	    }
	    else
		s = (olds + c) % n;
	}

    int start() const { return s; }
    float * buffer() const { return drsamples; }
    int nsamples() const { return n; }
};

void forces_dpd_cuda(float * const _xp, float * const _yp, float * const _zp,
		     float * const _xv, float * const _yv, float * const _zv,
		     float * const _xa, float * const _ya, float * const _za,
		     int * const order, const int np,
		     const float rc,
		     const float XL, const float YL, const float ZL,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
		     float * const _rsamples, int nsamples)
{
    static bool initialized = false;

    if (!initialized)
    {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	if (!prop.canMapHostMemory)
	{
	    printf("Capability zero-copy not there! Aborting now.\n");
	    abort();
	}
	else
	    hipSetDeviceFlags(hipDeviceMapHost);

	initialized = true;
    }

    static RRingBuffer * rrbuf = NULL;

    if (rrbuf == NULL)
	rrbuf = new RRingBuffer(50 * np * 3);
     
    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;
    
    device_vector<int> starts(ncells + 1);
    
    device_vector<float> xp(_xp, _xp + np), yp(_yp, _yp + np), zp(_zp, _zp + np),
	xv(_xv, _xv + np), yv(_yv, _yv + np), zv(_zv, _zv + np);	

    device_vector<float> xa(np), ya(np), za(np);
    fill(xa.begin(), xa.end(), 0);
    fill(ya.begin(), ya.end(), 0);
    fill(za.begin(), za.end(), 0);
    
    InfoDPD c;
    c.nx = nx;
    c.ny = ny;
    c.nz = nz;
    c.np = np;
    c.XL = XL;
    c.YL = YL;
    c.ZL = ZL;
    c.xstart = -XL * 0.5; 
    c.ystart = -YL * 0.5; 
    c.zstart = -ZL * 0.5; 
    c.rc = rc;
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigma = sigma;
    c.invsqrtdt = invsqrtdt;
    c.sigmaf = sigma * invsqrtdt;
    c.xp = _ptr(xp);
    c.yp = _ptr(yp);
    c.zp = _ptr(zp);
    c.xv = _ptr(xv);
    c.yv = _ptr(yv);
    c.zv = _ptr(zv);
    c.xa = _ptr(xa);
    c.ya = _ptr(ya);
    c.za = _ptr(za);
    c.nsamples = rrbuf->nsamples();
    c.rsamples = rrbuf->buffer();
    c.rsamples_start = rrbuf->start();

    device_vector<float> rsamples;
    if (_rsamples != NULL)
    {
	rsamples.resize(nsamples);
	copy(_rsamples, _rsamples + nsamples, rsamples.begin());

	c.nsamples = nsamples;
	c.rsamples = _ptr(rsamples);
	c.rsamples_start = 0;
    }
    else
	nsamples = rrbuf->nsamples();
    
    c.starts = _ptr(starts);
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(info), &c, sizeof(c)));

    device_vector<int> codes(np), pids(np);
    pid2code<<<(np + 127) / 128, 128>>>(_ptr(codes), _ptr(pids));

    sort_by_key(codes.begin(), codes.end(), pids.begin());

    _reorder(xp, pids);
    _reorder(yp, pids);
    _reorder(zp, pids);
    
    _reorder(xv, pids);
    _reorder(yv, pids);
    _reorder(zv, pids);
    
    device_vector<int> cids(ncells + 1);
    sequence(cids.begin(), cids.end());

    lower_bound(codes.begin(), codes.end(), cids.begin(), cids.end(), starts.begin());

    int * consumed = NULL;
    hipHostAlloc((void **)&consumed, sizeof(int), hipHostMallocMapped);
    assert(consumed != NULL);
    *consumed = 0;
    
    {
	float * tmp;

	CUDA_CHECK(hipMalloc(&tmp, sizeof(float) * np * 24));
	CUDA_CHECK(hipMemset(tmp, 0, sizeof(float) * np * 24));
	
	int * dconsumed = NULL;
	hipHostGetDevicePointer(&dconsumed, consumed, 0);
    
	_dpd_forces<<<dim3(c.nx, c.ny, c.nz), dim3(xbs, ybs, 1)>>>(tmp, dconsumed);
	
	CUDA_CHECK(hipPeekAtLastError());

	_reduce<<<(np + 127) / 128, 128>>>(tmp);
	
	CUDA_CHECK(hipPeekAtLastError());
	
	CUDA_CHECK(hipFree(tmp));
	
	if (*consumed >= nsamples)
	{
	    printf("done with code %d: consumed: %d\n", 7, *consumed);
	    printf("not a nice situation.\n");
	    abort();
	}

	//printf("consumed: %d\n", *consumed);
    }

#if 1
    CUDA_CHECK(hipDeviceSynchronize());
    for(int i = 0; i < np; ++i)
	;//assert((float)xa[i] > 0);

    //printf("positivity test passed\n");
    
    for(int i = 0; i < np; ++i)
    {
	printf("pid %d -> %f %f %f\n", i, (float)xa[i], (float)ya[i], (float)za[i]);

	int cnt = 0;
	const int pid = pids[i];

	printf("devi coords are %f %f %f\n", (float)xp[i], (float)yp[i], (float)zp[i]);
	printf("host coords are %f %f %f\n", (float)_xp[pid], (float)_yp[pid], (float)_zp[pid]);
	
	
	for(int j = 0; j < np; ++j)
	{
	    if (pid == j)
		continue;
 
	    float xr = _xp[pid] - _xp[j];
	    float yr = _yp[pid] - _yp[j];
	    float zr = _zp[pid] - _zp[j];

	    xr -= c.XL *  ::floor(0.5f + xr / c.XL);
	    yr -= c.YL *  ::floor(0.5f + yr / c.YL);
	    zr -= c.ZL *  ::floor(0.5f + zr / c.ZL);

	    const float rij2 = xr * xr + yr * yr + zr * zr;
	    

	    cnt += rij2 < 1;
	}
	printf("i found %d host interactions and with cuda i found %d\n", cnt, (int)xa[i]);
	assert(cnt == (float)xa[i]);

	//sleep(3);
    }
    printf("test done.\n");
    sleep(1);
    exit(0);
#endif

	
    if (_rsamples == NULL)
	rrbuf->update(*consumed);
    
    hipHostFree(consumed);
    
    copy(xp.begin(), xp.end(), _xp);
    copy(yp.begin(), yp.end(), _yp);
    copy(zp.begin(), zp.end(), _zp);
	
    copy(xv.begin(), xv.end(), _xv);
    copy(yv.begin(), yv.end(), _yv);
    copy(zv.begin(), zv.end(), _zv);

    copy(xa.begin(), xa.end(), _xa);
    copy(ya.begin(), ya.end(), _ya);
    copy(za.begin(), za.end(), _za);

    if (order != NULL)
	copy(pids.begin(), pids.end(), order);
}