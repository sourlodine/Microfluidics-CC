#include "hip/hip_runtime.h"
#include <cassert>

#include "rring-buffer.h"

void RRingBuffer::_refill(int s, int e)
{
    assert(e > s && e <= n);
	    
    const int multiple = 2;

    s = s - (s % multiple);
    e = e + (multiple - (e % multiple));
    e = min(e, n);
	    
    hiprandStatus_t res;
    res = hiprandGenerateNormal(prng, drsamples + s, e - s, 0, 1);
    assert(res == HIPRAND_STATUS_SUCCESS);
}
   
RRingBuffer::RRingBuffer(const int n): n(n), s(0), olds(0), c(0), drsamples(NULL)
{
    hiprandStatus_t res;
    res = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    //we could try HIPRAND_RNG_PSEUDO_MTGP32 or HIPRAND_RNG_PSEUDO_MT19937
	    
    assert(res == HIPRAND_STATUS_SUCCESS);
    res = hiprandSetPseudoRandomGeneratorSeed(prng, 1234ULL);
    assert(res == HIPRAND_STATUS_SUCCESS);
	    
    hipMalloc(&drsamples, sizeof(float) * n);
    assert(drsamples != NULL);
    
    update(n);
    assert(s == 0);
}

RRingBuffer::~RRingBuffer()
{
    hipFree(drsamples);
    hiprandStatus_t res = hiprandDestroyGenerator(prng);
    assert(res == HIPRAND_STATUS_SUCCESS);
}
    
void RRingBuffer::update(const int consumed)
{
    assert(consumed >= 0 && consumed <= n);

    c += consumed;
    assert(c >= 0 && c <= n);
	    
    if (c > 0.45 * n)
    {
	const int c1 = min(olds + c, n) - olds;
	    
	if (c1 > 0)
	    _refill(olds, olds + c1);

	const int c2 = c - c1;

	if (c2 > 0)
	    _refill(0, c2);
	    
	olds = (olds + c) % n;
	s = olds;
	c = 0;
    }
    else
	s = (olds + c) % n;
}
