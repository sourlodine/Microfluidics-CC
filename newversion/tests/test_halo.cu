#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public

#include "../core/containers.h"
#include "../core/celllist.h"
#include "../core/dpd.h"
#include "../core/halo_exchanger.h"
#include "../core/logger.h"

Logger logger;

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	MPI_Init(&argc, &argv);
	logger.init(MPI_COMM_WORLD, "halo.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	// Initial cells

	int3 ncells = {64, 64, 64};
	float3 domainStart = {-ncells.x / 2.0f, -ncells.y / 2.0f, -ncells.z / 2.0f};
	float3 length{(float)ncells.x, (float)ncells.y, (float)ncells.z};

	const int ndens = 8;

	ParticleVector dpds(ncells, domainStart, length);

	dpds.resize(dpds.totcells*ndens);

	srand48(0);

	printf("initializing...\n");

	int c = 0;
	for (int i=0; i<ncells.x; i++)
		for (int j=0; j<ncells.y; j++)
			for (int k=0; k<ncells.z; k++)
				for (int p=0; p<ndens; p++)
				{
					dpds.coosvels[c].x[0] = i + drand48() + domainStart.x;
					dpds.coosvels[c].x[1] = j + drand48() + domainStart.y;
					dpds.coosvels[c].x[2] = k + drand48() + domainStart.z;
					dpds.coosvels[c].i1 = c;

					dpds.coosvels[c].u[0] = drand48() - 0.5;
					dpds.coosvels[c].u[1] = drand48() - 0.5;
					dpds.coosvels[c].u[2] = drand48() - 0.5;
					c++;
				}

	dpds.resize(c);
	dpds.coosvels.synchronize(synchronizeDevice);

	hipStream_t defStream = 0;

	HaloExchanger halo(cartComm);
	halo.attach(&dpds, 7);

	buildCellList((float4*)dpds.coosvels.devdata, dpds.np, dpds.domainStart, dpds.ncells, dpds.totcells, 1.0f, (float4*)dpds.pingPongBuf.devdata, dpds.cellsSize.devdata, dpds.cellsStart.devdata, defStream);
	swap(dpds.coosvels, dpds.pingPongBuf, defStream);
	CUDA_Check( hipStreamSynchronize(defStream) );

	for (int i=0; i<100; i++)
	{
		halo.exchangeInit();
		halo.exchangeFinalize();
	}

	std::vector<Particle> bufs[27];
	dpds.coosvels.synchronize(synchronizeHost);
	for (int i=0; i<dpds.np; i++)
	{
		Particle& p = dpds.coosvels[i];
		float3 coo{p.x[0], p.x[1], p.x[2]};

		int cx = getCellIdAlongAxis(coo.x, domainStart.x, ncells.x, 1.0f);
		int cy = getCellIdAlongAxis(coo.y, domainStart.y, ncells.y, 1.0f);
		int cz = getCellIdAlongAxis(coo.z, domainStart.z, ncells.z, 1.0f);

		// 6
		if (cx == 0)          bufs[ (1*3 + 1)*3 + 0 ].push_back(p);
		if (cx == ncells.x-1) bufs[ (1*3 + 1)*3 + 2 ].push_back(p);
		if (cy == 0)          bufs[ (1*3 + 0)*3 + 1 ].push_back(p);
		if (cy == ncells.y-1) bufs[ (1*3 + 2)*3 + 1 ].push_back(p);
		if (cz == 0)          bufs[ (0*3 + 1)*3 + 1 ].push_back(p);
		if (cz == ncells.z-1) bufs[ (2*3 + 1)*3 + 1 ].push_back(p);

		// 12
		if (cx == 0          && cy == 0)          bufs[ (1*3 + 0)*3 + 0 ].push_back(p);
		if (cx == ncells.x-1 && cy == 0)          bufs[ (1*3 + 0)*3 + 2 ].push_back(p);
		if (cx == 0          && cy == ncells.y-1) bufs[ (1*3 + 2)*3 + 0 ].push_back(p);
		if (cx == ncells.x-1 && cy == ncells.y-1) bufs[ (1*3 + 2)*3 + 2 ].push_back(p);

		if (cy == 0          && cz == 0)          bufs[ (0*3 + 0)*3 + 1 ].push_back(p);
		if (cy == ncells.y-1 && cz == 0)          bufs[ (0*3 + 2)*3 + 1 ].push_back(p);
		if (cy == 0          && cz == ncells.z-1) bufs[ (2*3 + 0)*3 + 1 ].push_back(p);
		if (cy == ncells.y-1 && cz == ncells.z-1) bufs[ (2*3 + 2)*3 + 1 ].push_back(p);


		if (cz == 0          && cx == 0)          bufs[ (0*3 + 1)*3 + 0 ].push_back(p);
		if (cz == ncells.z-1 && cx == 0)          bufs[ (2*3 + 1)*3 + 0 ].push_back(p);
		if (cz == 0          && cx == ncells.x-1) bufs[ (0*3 + 1)*3 + 2 ].push_back(p);
		if (cz == ncells.z-1 && cx == ncells.x-1) bufs[ (2*3 + 1)*3 + 2 ].push_back(p);

		// 8
		if (cx == 0          && cy == 0          && cz == 0)          bufs[ (0*3 + 0)*3 + 0 ].push_back(p);
		if (cx == 0          && cy == 0          && cz == ncells.z-1) bufs[ (2*3 + 0)*3 + 0 ].push_back(p);
		if (cx == 0          && cy == ncells.y-1 && cz == 0)          bufs[ (0*3 + 2)*3 + 0 ].push_back(p);
		if (cx == 0          && cy == ncells.y-1 && cz == ncells.z-1) bufs[ (2*3 + 2)*3 + 0 ].push_back(p);
		if (cx == ncells.x-1 && cy == 0          && cz == 0)          bufs[ (0*3 + 0)*3 + 2 ].push_back(p);
		if (cx == ncells.x-1 && cy == 0          && cz == ncells.z-1) bufs[ (2*3 + 0)*3 + 2 ].push_back(p);
		if (cx == ncells.x-1 && cy == ncells.y-1 && cz == 0)          bufs[ (0*3 + 2)*3 + 2 ].push_back(p);
		if (cx == ncells.x-1 && cy == ncells.y-1 && cz == ncells.z-1) bufs[ (2*3 + 2)*3 + 2 ].push_back(p);
	}

	for (int i = 0; i<27; i++)
	{
		std::sort(bufs[i].begin(), bufs[i].end(), [] (Particle& a, Particle& b) { return a.i1 < b.i1; });

		std::sort(halo.helpers[0].sendBufs[i].hostdata, halo.helpers[0].sendBufs[i].hostdata + halo.helpers[0].counts[i],
				[] (Particle& a, Particle& b) { return a.i1 < b.i1; });

		if (bufs[i].size() != halo.helpers[0].counts[i])
			printf("%2d-th halo differs in size: %5d, expected %5d\n", i, halo.helpers[0].counts[i], (int)bufs[i].size());
		else
			for (int pid = 0; pid < halo.helpers[0].counts[i]; pid++)
			{
				const float diff = std::max({
					fabs(halo.helpers[0].sendBufs[i][pid].x[0] - bufs[i][pid].x[0]),
					fabs(halo.helpers[0].sendBufs[i][pid].x[1] - bufs[i][pid].x[1]),
					fabs(halo.helpers[0].sendBufs[i][pid].x[2] - bufs[i][pid].x[2]) });

				if (bufs[i][pid].i1 != halo.helpers[0].sendBufs[i][pid].i1 || diff > 1e-5)
					printf("Halo %2d:  %5d [%10.3e %10.3e %10.3e], expected %5d [%10.3e %10.3e %10.3e]\n",
							i, halo.helpers[0].sendBufs[i][pid].i1, halo.helpers[0].sendBufs[i][pid].x[0],
							halo.helpers[0].sendBufs[i][pid].x[1], halo.helpers[0].sendBufs[i][pid].x[2],
							bufs[i][pid].i1, bufs[i][pid].x[0], bufs[i][pid].x[1], bufs[i][pid].x[2]);
			}
	}

	//for (int i=0; i<dpds.halo.size; i++)
	//	printf("%d  %f %f %f\n", i, dpds.halo[i].x[0], dpds.halo[i].x[1], dpds.halo[i].x[2]);


	// Forces
	//   || Halo
	// Integrate
	// Redistribute
	// Cell list

	return 0;
}
