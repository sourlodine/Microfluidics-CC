#include "hip/hip_runtime.h"
#include "../core/containers.h"
#include "../core/celllist.h"
#include "../core/dpd.h"
#include "../core/halo_exchanger.h"
#include "../core/common.h"

int main(int argc, char ** argv)
{

	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	logger.MPI_Check( MPI_Init(&argc, &argv) );
	logger.MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	logger.MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	logger.MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	// Initial cells

	int3 ncells = {64, 64, 64};
	float3 domainStart = {-ncells.x / 2.0f, -ncells.y / 2.0f, -ncells.z / 2.0f};
	ParticleVector dpds(ncells, domainStart);

	const int ndens = 12;
	dpds.resize(dpds.totcells*ndens);

	srand48(0);

	printf("initializing...\n");

	int c = 0;
	for (int i=0; i<ncells.x; i++)
		for (int j=0; j<ncells.y; j++)
			for (int k=0; k<ncells.z; k++)
				for (int p=0; p<ndens * drand48(); p++)
				{
					dpds.coosvels[c].x[0] = i + drand48() + domainStart.x;
					dpds.coosvels[c].x[1] = j + drand48() + domainStart.y;
					dpds.coosvels[c].x[2] = k + drand48() + domainStart.z;
					dpds.coosvels[c].i1 = c;

					dpds.coosvels[c].u[0] = drand48() - 0.5;
					dpds.coosvels[c].u[1] = drand48() - 0.5;
					dpds.coosvels[c].u[2] = drand48() - 0.5;
					c++;
				}

	dpds.resize(c);
	dpds.coosvels.synchronize(synchronizeDevice);

	hipStream_t defStream = 0;

	HaloExchanger halo(cartComm);
	halo.attach(&dpds);

	buildCellList((float4*)dpds.coosvels.devdata, dpds.np, dpds.domainStart, dpds.ncells, 1.0f, (float4*)dpds.accs.devdata, dpds.cellsSize.devdata, dpds.cellsStart.devdata, defStream);
	CUDA_CHECK( hipStreamSynchronize(defStream) );
	computeInternalDPD(dpds, defStream);
	halo.exchangeInit();
	halo.exchangeFinalize();

	// Forces
	//   || Halo
	// Integrate
	// Redistribute
	// Cell list

	return 0;
}
