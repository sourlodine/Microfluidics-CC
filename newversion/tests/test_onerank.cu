#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public

#include "../core/containers.h"
#include "../core/celllist.h"
#include "../core/dpd.h"
#include "../core/halo_exchanger.h"
#include "../core/logger.h"
#include "../core/integrate.h"

#include <unistd.h>

Logger logger;

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
	    printf("ERROR: The MPI library does not have full thread support\n");
	    MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, "onerank.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	// Initial cells

	int3 ncells = {64, 64, 64};
	float3 domainStart = {-ncells.x / 2.0f, -ncells.y / 2.0f, -ncells.z / 2.0f};
	float3 length{(float)ncells.x, (float)ncells.y, (float)ncells.z};
	ParticleVector dpds(ncells, domainStart, length);

	const int ndens = 8;
	dpds.resize(ncells.x*ncells.y*ncells.z * ndens);

	srand48(0);

	printf("initializing...\n");

	int c = 0;
	for (int i=0; i<ncells.x; i++)
		for (int j=0; j<ncells.y; j++)
			for (int k=0; k<ncells.z; k++)
				for (int p=0; p<ndens; p++)
				{
					dpds.coosvels[c].x[0] = i + drand48() + domainStart.x;
					dpds.coosvels[c].x[1] = j + drand48() + domainStart.y;
					dpds.coosvels[c].x[2] = k + drand48() + domainStart.z;
					dpds.coosvels[c].i1 = c;

					dpds.coosvels[c].u[0] = drand48() - 0.5;
					dpds.coosvels[c].u[1] = drand48() - 0.5;
					dpds.coosvels[c].u[2] = drand48() - 0.5;
					c++;
				}

	dpds.resize(c);
	dpds.coosvels.synchronize(synchronizeDevice);

	hipStream_t defStream;
	CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );

	HaloExchanger halo(cartComm);
	halo.attach(&dpds, 7);

	buildCellList((float4*)dpds.coosvels.devdata, dpds.np, dpds.domainStart, dpds.ncells, dpds.totcells, 1.0f, (float4*)dpds.pingPongBuf.devdata, dpds.cellsSize.devdata, dpds.cellsStart.devdata, defStream);
	swap(dpds.coosvels, dpds.pingPongBuf, defStream);
	CUDA_Check( hipStreamSynchronize(defStream) );

	for (int i=0; i<100; i++)
	{
		buildCellList((float4*)dpds.coosvels.devdata, dpds.np, dpds.domainStart, dpds.ncells, dpds.totcells, 1.0f, (float4*)dpds.pingPongBuf.devdata, dpds.cellsSize.devdata, dpds.cellsStart.devdata, defStream);
		swap(dpds.coosvels, dpds.pingPongBuf, defStream);
		hipStreamSynchronize(defStream);

		computeInternalDPD(dpds, defStream);

		halo.exchangeInit();
		halo.exchangeFinalize();

		computeHaloDPD(dpds, defStream);
		integrate(dpds, 1e-15f, defStream);


		//hipDeviceSynchronize();



		//hipDeviceSynchronize();

		hipStreamSynchronize(defStream);
	}




	return 0;
}
