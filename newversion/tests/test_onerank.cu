#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public

#include "../core/containers.h"
#include "../core/celllist.h"
#include "../core/dpd.h"
#include "../core/halo_exchanger.h"
#include "../core/redistributor.h"
#include "../core/logger.h"
#include "../core/integrate.h"

#include <unistd.h>

Logger logger;

void makeCells(const Particle* __restrict__ coos, int* __restrict__ cellsStart, int* __restrict__ cellsSize,
		int3 ncells, float3 domainStart, float rc)
{

}

void forces(const Particle* __restrict__ coos, Acceleration* __restrict__ accs, const int* __restrict__ cellsStart, const int* __restrict__ cellsSize,
		int3 ncells, int totcells, float3 domainStart, float3 length)
{

	const float dt = 0.0025;
	const float kBT = 1.0;
	const float gammadpd = 20;
	const float sigma = sqrt(2 * gammadpd * kBT);
	const float sigmaf = sigma / sqrt(dt);
	const float aij = 50;

	auto addForce = [=] (int dstId, int srcId, Acceleration& a)
	{
		float _xr = coos[dstId].x[0] - coos[srcId].x[0];
		float _yr = coos[dstId].x[1] - coos[srcId].x[1];
		float _zr = coos[dstId].x[2] - coos[srcId].x[2];

		_xr = std::min({_xr, _xr - length.x, _xr + length.x});
		_yr = std::min({_yr, _yr - length.y, _yr + length.y});
		_zr = std::min({_zr, _zr - length.z, _zr + length.z});

		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

		if (rij2 > 1.0f) return;
		//assert(rij2 < 1);

		const float invrij = 1.0f / sqrt(rij2);
		const float rij = rij2 * invrij;
		const float argwr = 1.0f - rij;
		const float wr = argwr;

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv =
				xr * (coos[dstId].u[0] - coos[srcId].u[0]) +
				yr * (coos[dstId].u[1] - coos[srcId].u[1]) +
				zr * (coos[dstId].u[2] - coos[srcId].u[2]);

		const float myrandnr = 0;//Logistic::mean0var1(1, min(srcId, dstId), max(srcId, dstId));

		const float strength = aij * argwr - (gammadpd * wr * rdotv + sigmaf * myrandnr) * wr;

		a.a[0] += strength * xr;
		a.a[1] += strength * yr;
		a.a[2] += strength * zr;
	};

#pragma omp parallel for collapse(3)
	for (int cx = 0; cx < ncells.x; cx++)
		for (int cy = 0; cy < ncells.y; cy++)
			for (int cz = 0; cz < ncells.z; cz++)
			{
				const int cid = encode(cx, cy, cz, ncells);

				for (int dstId = cellsStart[cid]; dstId < cellsStart[cid] + cellsSize[cid]; dstId++)
				{
					Acceleration a {0,0,0,0};

					for (int dx = -1; dx <= 1; dx++)
						for (int dy = -1; dy <= 1; dy++)
							for (int dz = -1; dz <= 1; dz++)
							{
								int ncx, ncy, ncz;
								ncx = (cx+dx + ncells.x) % ncells.x;
								ncy = (cy+dy + ncells.y) % ncells.y;
								ncz = (cz+dz + ncells.z) % ncells.z;

								const int srcCid = encode(ncx, ncy, ncz, ncells);
								if (srcCid >= totcells || srcCid < 0) continue;

								for (int srcId = cellsStart[srcCid]; srcId < cellsStart[srcCid] + cellsSize[srcCid]; srcId++)
								{
									if (dstId != srcId)
										addForce(dstId, srcId, a);
								}
							}

					accs[dstId].a[0] = a.a[0];
					accs[dstId].a[1] = a.a[1];
					accs[dstId].a[2] = a.a[2];
				}
			}
}

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
	    printf("ERROR: The MPI library does not have full thread support\n");
	    MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, "onerank.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	// Initial cells

	int3 ncells = {64, 64, 64};
	float3 domainStart = {-ncells.x / 2.0f, -ncells.y / 2.0f, -ncells.z / 2.0f};
	float3 length{(float)ncells.x, (float)ncells.y, (float)ncells.z};
	ParticleVector dpds(ncells, domainStart, length);

	const int ndens = 8;
	dpds.resize(ncells.x*ncells.y*ncells.z * ndens);

	srand48(0);

	printf("initializing...\n");

	int c = 0;
	for (int i=0; i<ncells.x; i++)
		for (int j=0; j<ncells.y; j++)
			for (int k=0; k<ncells.z; k++)
				for (int p=0; p<ndens; p++)
				{
					dpds.coosvels[c].x[0] = i + drand48() + domainStart.x;
					dpds.coosvels[c].x[1] = j + drand48() + domainStart.y;
					dpds.coosvels[c].x[2] = k + drand48() + domainStart.z;
					dpds.coosvels[c].i1 = c;

					dpds.coosvels[c].u[0] = drand48() - 0.5;
					dpds.coosvels[c].u[1] = drand48() - 0.5;
					dpds.coosvels[c].u[2] = drand48() - 0.5;
					c++;
				}


	dpds.resize(c);
	dpds.coosvels.synchronize(synchronizeDevice);
	dpds.accs.clear();

	HostBuffer<Particle> particles(dpds.np);
	for (int i=0; i<dpds.np; i++)
		particles[i] = dpds.coosvels[i];

	hipStream_t defStream;
	CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );

	HaloExchanger halo(cartComm);
	halo.attach(&dpds, ndens);
	Redistributor redist(cartComm);
	redist.attach(&dpds, ndens);

	buildCellList(dpds,defStream);
	CUDA_Check( hipStreamSynchronize(defStream) );

	const float dt = 0.001;
	const int niters = 100;

	for (int i=0; i<niters; i++)
	{
		dpds.accs.clear(defStream);
		computeInternalDPD(dpds, defStream);

		halo.exchangeInit();
		halo.exchangeFinalize();

		computeHaloDPD(dpds, defStream);
		CUDA_Check( hipStreamSynchronize(defStream) );

		redist.redistribute(dt);

		buildCellListAndIntegrate(dpds, dt, defStream);
		CUDA_Check( hipStreamSynchronize(defStream) );

		hipDeviceSynchronize();
	}


	HostBuffer<Acceleration> accs(particles.size);








	return 0;
}
