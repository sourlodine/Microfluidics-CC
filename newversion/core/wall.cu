#include "hip/hip_runtime.h"
#include "wall.h"
#include "flows.h"
#include "celllist.h"
#include "interactions.h"
#include "interaction_engine.h"

#include <fstream>
#include <cmath>
#include <hip/hip_vector_types.h>
#include <texture_types.h>

// This should be in hip/hip_vector_types.h, but not there for some reason
//***************************************************************
inline __host__ __device__ int3 operator%(int3 a, int3 b)
{
    return make_int3(a.x % b.x, a.y % b.y, a.z % b.z);
}

inline __host__ __device__ int3 operator/(int3 a, int b)
{
    return make_int3(a.x / b, a.y / b, a.z / b);
}

inline __host__ __device__ float3 ceilf(float3 v)
{
    return make_float3(ceilf(v.x), ceilf(v.y), ceilf(v.z));
}
//***************************************************************


__device__ __forceinline__ float cubicInterpolate1D(float y[4], float mu)
{
   const float a0 = y[3] - y[2] - y[0] + y[1];
   const float a1 = y[0] - y[1] - a0;
   const float a2 = y[2] - y[0];
   const float a3 = y[1];

   return ((a0*mu + a1)*mu + a2)*mu + a3;
}


__global__ void cubicInterpolate3D(const float* in, int3 inDims, float3 inH, float* out, int3 outDims, float3 outH, float3 offset)
{
	// Inspired by http://paulbourke.net/miscellaneous/interpolation/
	// Center of the output domain is in offset
	// Center of the input domain is in (0,0,0)

	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	const int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= outDims.x || iy >= outDims.y || iz >= outDims.z) return;

	float interp2D[4][4];
	float interp1D[4];

	// Coordinates where to interpolate
	float3 tmp = make_float3(ix, iy, iz);
	float3 coos = tmp*outH + offset;


	// Reference point of the original grid
	int3 closestInPoint = min( make_int3( fmaxf( floorf(coos / inH), make_float3(0.0f)) ),  inDims-1);

	// Interpolate along x
	for (int dz = -1; dz <= 2; dz++)
		for (int dy = -1; dy <= 2; dy++)
		{
			float vals[4];

			for (int dx = -1; dx <= 2; dx++)
			{
				int3 delta{dx, dy, dz};
				const int3 curCoos = (closestInPoint+delta + inDims) % inDims;

				vals[dx+1] = in[ (curCoos.z*inDims.y + curCoos.y) * inDims.x + curCoos.x ];
			}

			interp2D[dz+1][dy+1] = cubicInterpolate1D(vals, (coos.x - closestInPoint.x*inH.x) / inH.x);
		}

	// Interpolate along y
	for (int dz = 0; dz <= 3; dz++)
		interp1D[dz] = cubicInterpolate1D(interp2D[dz], (coos.y - closestInPoint.y*inH.y) / inH.y);

	// Interpolate along z
	out[ (iz*outDims.y + iy) * outDims.x + ix ] = cubicInterpolate1D(interp1D, (coos.z - closestInPoint.z*inH.z) / inH.z);
}

__forceinline__ __device__ float gRouyTourin(float a, float b, float c, float d, float e, float f)
{
	// Rouy-Tourin scheme
	// http://epubs.siam.org/doi/pdf/10.1137/0729053

	return sqrt(
			max( sqr(max(a, 0.0f)), sqr(min(b, 0.0f)) ) +
			max( sqr(max(c, 0.0f)), sqr(min(d, 0.0f)) ) +
			max( sqr(max(e, 0.0f)), sqr(min(f, 0.0f)) )
			);
}

__global__ void redistance(const float* in, int3 dims, float3 h, float dt, float* out)
{
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	const int iz = blockIdx.z * blockDim.z + threadIdx.z;

	auto sqr  = [](float x) { return x*x; };

	auto encode = [=](int i, int j, int k) {
		i = (i+dims.x) % dims.x;
		j = (j+dims.y) % dims.y;
		k = (k+dims.z) % dims.z;

		return (k*dims.y + j) * dims.x + i;
	};

	const int id0 = encode(ix, iy, iz);

	const float u        = in[id0];

	if (fabs(u) < max(h.x, max(h.y, h.z)))
	{
		out[id0] = in[id0];
		return;
	}

	const float ux_minus = in[encode(ix-1, iy,   iz  )];
	const float ux_plus  = in[encode(ix+1, iy,   iz  )];
	const float uy_minus = in[encode(ix,   iy-1, iz  )];
	const float uy_plus  = in[encode(ix,   iy+1, iz  )];
	const float uz_minus = in[encode(ix,   iy,   iz-1)];
	const float uz_plus  = in[encode(ix,   iy,   iz+1)];

	const float dx_minus = (u - ux_minus) / h.x;
	const float dx_plus  = (ux_plus  - u) / h.x;
	const float dy_minus = (u - uy_minus) / h.y;
	const float dy_plus  = (uy_plus  - u) / h.y;
	const float dz_minus = (u - uz_minus) / h.z;
	const float dz_plus  = (uz_plus  - u) / h.z;

	const float grad = gRouyTourin(dx_minus, dx_plus, dy_minus, dy_plus, dz_minus, dz_plus);

//	if (grad < 0.6f)
//		printf("%f,  x %f %f %f,  y %f %f %f,  z %f %f %f,  %d %d %d\n", grad,
//				ux_minus, u, ux_plus,  uy_minus, u, uy_plus,  uz_minus, u, uz_plus, ix, iy, iz);

	const float S = fabs(u) / sqrt( u*u + sqr(grad * h.x) );
	out[id0] = u + S * dt * (1-grad);
}

template<typename T>
__device__ __forceinline__ float evalSdf(hipTextureObject_t sdfTex, T x, float3 length, float3 h, float3 invH)
{
	float3 x3{x.x, x.y, x.z};
	float3 texcoord = floorf((x3 + length*0.5f) * invH);
	float3 lambda = (x3 - (texcoord * h - length*0.5f)) * invH;

	const float s000 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 0, texcoord.z + 0);
	const float s001 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 0, texcoord.z + 0);
	const float s010 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 1, texcoord.z + 0);
	const float s011 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 1, texcoord.z + 0);
	const float s100 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 0, texcoord.z + 1);
	const float s101 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 0, texcoord.z + 1);
	const float s110 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 1, texcoord.z + 1);
	const float s111 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 1, texcoord.z + 1);

	const float s00x = s000 * (1 - lambda.x) + lambda.x * s001;
	const float s01x = s010 * (1 - lambda.x) + lambda.x * s011;
	const float s10x = s100 * (1 - lambda.x) + lambda.x * s101;
	const float s11x = s110 * (1 - lambda.x) + lambda.x * s111;

	const float s0yx = s00x * (1 - lambda.y) + lambda.y * s01x;
	const float s1yx = s10x * (1 - lambda.y) + lambda.y * s11x;

	const float szyx = s0yx * (1 - lambda.z) + lambda.z * s1yx;

//	printf("[%f %f %f]  [%f %f %f]  [%f %f %f]  = %f  vs  %f\n", x.x, x.y, x.z,  texcoord.x, texcoord.y, texcoord.z,
//			lambda.x, lambda.y, lambda.z, szyx, sqrt(x.x*x.x + x.y*x.y + x.z*x.z) - 5);

	return szyx;
}

// warp-aggregated atomic increment
// https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
__device__ __forceinline__ int atomicAggInc(int *ctr)
{
	int lane_id = (threadIdx.x % 32);

	int mask = __ballot(1);
	// select the leader
	int leader = __ffs(mask) - 1;
	// leader does the update
	int res;
	if(lane_id == leader)
	res = atomicAdd(ctr, __popc(mask));
	// broadcast result
	res = __shfl(res, leader);
	// each thread computes its own value
	return res + __popc(mask & ((1 << lane_id) - 1));
}


namespace FreezingActions
{
	const int Freeze = 1;
	const int Keep   = 2;
	const int Remove = 3;
}

__global__ void countFrozen(float4* particles, hipTextureObject_t sdfTex, float3 length, float3 h, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;

	const float4 coos = particles[2*pid];
	float4 vels = particles[2*pid+1];

	const float sdf = evalSdf(sdfTex, coos, length, h, 1.0f / h);

	if (sdf < 0.0f)			vels.w = __int_as_float(FreezingActions::Keep);  // keep
	else if (sdf > 1.5f)	vels.w = __int_as_float(FreezingActions::Remove);  // remove
	else
	{
		vels.w = __int_as_float(FreezingActions::Freeze);  // freeze
		atomicAggInc(nFrozen);
	}

	particles[2*pid+1].w = vels.w;
}

__global__ void collectFrozen(hipTextureObject_t sdfTex, const float4* input, float4* output, float4* frozen, int* nRemaining, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;

	const float4 coos = input[2*pid];
	const float4 vels = input[2*pid+1];

	const int key = __float_as_int(vels.w);

	if (key == FreezingActions::Keep)
	{
		const int ind = atomicAggInc(nRemaining);
		output[2*ind] = coos;
		output[2*ind + 1] = vels;
	}

	if (key == FreezingActions::Freeze)
	{
		const int ind = atomicAggInc(nFrozen);
		frozen[2*ind] = coos;
		frozen[2*ind + 1] = vels;
	}
}

__global__ void countBoundaryCells(const int3 ncells, const float3 domainStart, const float rc, hipTextureObject_t sdfTex,
		const float3 length, const float3 h, int* nBoundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix, iy, iz;

	decode(cid, ix, iy, iz, ncells);

	const float3 invH = 1.0f / h;

	const float cx = domainStart.x + ix*rc - 1e-6f;
	const float cy = domainStart.y + iy*rc - 1e-6f;
	const float cz = domainStart.z + iz*rc - 1e-6f;

	const float l = rc+2e-6f;
	const float s000 = evalSdf(sdfTex, make_float3(cx,   cy,   cz),   length, h, invH);
	const float s001 = evalSdf(sdfTex, make_float3(cx,   cy,   cz+l), length, h, invH);
	const float s010 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz),   length, h, invH);
	const float s011 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz+l), length, h, invH);
	const float s100 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz),   length, h, invH);
	const float s101 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz+l), length, h, invH);
	const float s110 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz),   length, h, invH);
	const float s111 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz+l), length, h, invH);

	if ( (1e-6f > s000 && s000 > -1.000001f) || (1e-6f > s001 && s001 > -1.000001f) ||
		 (1e-6f > s010 && s010 > -1.000001f) || (1e-6f > s011 && s011 > -1.000001f) ||
		 (1e-6f > s100 && s100 > -1.000001f) || (1e-6f > s101 && s101 > -1.000001f) ||
		 (1e-6f > s110 && s110 > -1.000001f) || (1e-6f > s111 && s111 > -1.000001f) )
	{
		atomicAggInc(nBoundaryCells);
	}
}

__global__ void getBoundaryCells(const int3 ncells, const float3 domainStart, const float rc,
		hipTextureObject_t sdfTex, const float3 length, const float3 h, int* nBoundaryCells, int* boundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix, iy, iz;

	decode(cid, ix, iy, iz, ncells);

	const float3 invH = 1.0f / h;

	const float cx = domainStart.x + ix*rc - 1e-6f;
	const float cy = domainStart.y + iy*rc - 1e-6f;
	const float cz = domainStart.z + iz*rc - 1e-6f;

	const float l = rc+2e-6f;
	const float s000 = evalSdf(sdfTex, make_float3(cx,   cy,   cz),   length, h, invH);
	const float s001 = evalSdf(sdfTex, make_float3(cx,   cy,   cz+l), length, h, invH);
	const float s010 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz),   length, h, invH);
	const float s011 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz+l), length, h, invH);
	const float s100 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz),   length, h, invH);
	const float s101 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz+l), length, h, invH);
	const float s110 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz),   length, h, invH);
	const float s111 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz+l), length, h, invH);

	if ( (1e-6f > s000 && s000 > -1.000001f) || (1e-6f > s001 && s001 > -1.000001f) ||
		 (1e-6f > s010 && s010 > -1.000001f) || (1e-6f > s011 && s011 > -1.000001f) ||
		 (1e-6f > s100 && s100 > -1.000001f) || (1e-6f > s101 && s101 > -1.000001f) ||
		 (1e-6f > s110 && s110 > -1.000001f) || (1e-6f > s111 && s111 > -1.000001f) )
	{
		int id = atomicAggInc(nBoundaryCells);
		boundaryCells[id] = cid;
	}
}


template<typename Transform>
__global__ void bounceBeforeIntegration(const int* wallCells, const int* __restrict__ cellsStart, const float4* accs,
		hipTextureObject_t sdfTex, const float3 length, const float3 h, const float3 invH, float4* xyzouvwo, Transform transform)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int cid = wallCells[tid];

	const int2 startSize = decodeStartSize(cellsStart[cid]);

	for (int pid = startSize.x; pid < startSize.x + startSize.y; pid++)
	{
		float va, vb;

		float4 coo = xyzouvwo[2*pid];
		float4 vel = xyzouvwo[2*pid + 1];
		float4 acc = accs[pid];

		float4 oldCoo = coo;
		transform(coo, vel, acc, pid);

		va = evalSdf(sdfTex, coo, length, h, invH);
		if (va > 0.0f) continue;

		vb = evalSdf(sdfTex, oldCoo, length, h, invH);
		assert( vb >= 0.0f ); // Accuracy issues here!

		// Determine where we cross
		// Interpolation search

		float3 a{oldCoo.x, oldCoo.y, oldCoo.z};
		float3 b{coo.x, coo.y, coo.z};
		float vmid = 1.0f;
		float lambda;

		while (fabs(vmid) > 1e-6f)
		{
			lambda = vb / (vb - va);  // va*l + (1-l)*vb = 0
			const float3 mid = a*lambda + b*(1.0f - lambda);
			vmid = evalSdf(sdfTex, mid, length, h, invH);

			if (va * vmid < 0.0f)
				vb = vmid;
			else
				va = vmid;
		}

		// In the corners long bounce may place the particle into another wall
		// Need to find a safe step in that case
		float beta = 1-2*lambda;
		float4 candidate = oldCoo - beta * (coo - oldCoo);

		while (evalSdf(sdfTex, candidate, length, h, invH) > -1e-6f)
		{
			beta *= 0.5;
			candidate = oldCoo - beta * (coo - oldCoo);
		}

		xyzouvwo[2*pid] = candidate;
		xyzouvwo[2*pid + 1] = -vel;
	}
}


Wall::Wall(MPI_Comm& comm, IniParser& config): config(config)
{
	dt = config.getFloat("Common", "dt");

	std::string sdfname = config.getString("Wall", "SdfFileName");
	std::string velname = config.getString("Wall", "VelocityFileName", "");

	length = config.getFloat3("Common", "SubdomainSize");
	resolution = config.getInt3("Wall", "Resolution", make_int3(256));
	float floatMargin = config.getFloat("Wall", "margin");

	MPI_Check( MPI_Comm_dup(comm, &wallComm) );

	int nranks, rank;
	int ranks[3], periods[3], coords[3];
	MPI_Check( MPI_Comm_size(wallComm, &nranks) );
	MPI_Check( MPI_Comm_rank(wallComm, &rank) );
	MPI_Check( MPI_Cart_get (wallComm, 3, ranks, periods, coords) );

	int3 fullSdfResolution;
	float3 fullSdfExtent;
	int fullSdfSize;  // TODO int64_t
	int endHeader;

	// Read header
	if (rank == 0)
	{
		printf("'%s'\n", sdfname.c_str());
		std::ifstream file(sdfname);
		if (!file.good())
			die("File not found or not accessible");

		auto fstart = file.tellg();

		file >> fullSdfExtent.x >> fullSdfExtent.y >> fullSdfExtent.z >>
			fullSdfResolution.x >> fullSdfResolution.y >> fullSdfResolution.z;
		fullSdfSize = fullSdfResolution.x * fullSdfResolution.y * fullSdfResolution.z;

		info("Using wall file '%s' of size %.2fx%f.2x%f.2 and resolution %dx%dx%d", sdfname.c_str(),
				fullSdfExtent.x, fullSdfExtent.y, fullSdfExtent.z,
				fullSdfResolution.x, fullSdfResolution.y, fullSdfResolution.z);

		file.seekg( 0, std::ios::end );
		auto fend = file.tellg();

		endHeader = (int)(fend - fstart) - fullSdfSize * sizeof(float);

		file.close();
	}

	MPI_Check( MPI_Bcast(&fullSdfExtent,     3, MPI_FLOAT, 0, wallComm) );
	MPI_Check( MPI_Bcast(&fullSdfResolution, 3, MPI_INT,   0, wallComm) );
	MPI_Check( MPI_Bcast(&endHeader,         1, MPI_INT,   0, wallComm) );

	// Read part and allgather
	// TODO int64_t
	const int nPerProc = (fullSdfSize + nranks - 1) / nranks;
	std::vector<float> readBuffer(nPerProc);

	// Limits in bytes
	const int start = sizeof(float) * nPerProc * rank + endHeader;
	const int end   = std::min( start + sizeof(float) * nPerProc, sizeof(float) * fullSdfSize + endHeader);

	MPI_File fh;
	MPI_Status status;
	MPI_Check( MPI_File_open(wallComm, sdfname.c_str(), MPI_MODE_RDONLY, MPI_INFO_NULL, &fh) );
	MPI_Check( MPI_File_read_at_all(fh, start, &readBuffer[0], end - start, MPI_BYTE, &status) );

	std::vector<float> fullSdfData(nPerProc * nranks);  // May be bigger than fullSdfSize, to make gather easier
	MPI_Check( MPI_Allgather(&readBuffer[0], nPerProc, MPI_FLOAT, &fullSdfData[0], nPerProc, MPI_FLOAT, wallComm) );

	// Find your relevant chunk of data

	const float3 scale3 = length / fullSdfExtent;

	if ( fabs(scale3.x - scale3.y) > 1e-5 || fabs(scale3.x - scale3.z) > 1e-5 )
		die("Sdf size and domain size mismatch");

	const float scale = (scale3.x + scale3.y + scale3.z) / 3;

	float3 sdfH = scale * fullSdfExtent / make_float3(fullSdfResolution-1);
	float3 domainStart{length.x*coords[0], length.x*coords[1], length.x*coords[2]};  // TODO get it from settings

	int margin = 3; // +2 from cubic interpolation, +1 from possible round-off errors
	const int3 inputResolution = fullSdfResolution + make_int3(2*margin);

	int3 inputStart = make_int3( floorf(domainStart / sdfH) );

	PinnedBuffer<float> inputSdfData ( inputResolution.x * inputResolution.y * inputResolution.z );

	for (int k = 0; k < inputResolution.z; k++)
		for (int j = 0; j < inputResolution.y; j++)
			for (int i = 0; i < inputResolution.x; i++)
			{
				const int origIx = (i+inputStart.x-margin + fullSdfResolution.x) % fullSdfResolution.x;
				const int origIy = (j+inputStart.y-margin + fullSdfResolution.y) % fullSdfResolution.y;
				const int origIz = (k+inputStart.z-margin + fullSdfResolution.z) % fullSdfResolution.z;

				inputSdfData[ (k*inputResolution.y + j)*inputResolution.x + i ] =
						fullSdfData[ (origIz*fullSdfResolution.y + origIy)*fullSdfResolution.x + origIx ];
			}
	inputSdfData.synchronize(synchronizeDevice);

	// Compute offset
	float3 offset = margin*sdfH;

	// Interpolate
	sdfRawData.resize(resolution.x * resolution.y * resolution.z);

	h = length / make_float3(resolution-1);
	dim3 threads(8, 8, 8);
	dim3 blocks((resolution.x+threads.x-1) / threads.x, (resolution.y+threads.y-1) / threads.y, (resolution.z+threads.z-1) / threads.z);

	cubicInterpolate3D<<< blocks, threads >>>(inputSdfData.devdata, inputResolution, sdfH, sdfRawData.devdata, resolution, h, offset);

	// Redistance
	// Need 2 arrays for redistancing

//	DeviceBuffer<float> tmp(sdfData.size);
//	const float redistDt = 0.1;
//	for (float t = 0; t < 200; t+=redistDt)
//	{
//		redistance<<< blocks, threads >>>(sdfData.devdata, resolution, h, redistDt, tmp.devdata);
//		swap(sdfData, tmp);
//	}

	// Prepare array to be transformed into texture
	auto chDesc = hipCreateChannelDesc<float>();
	CUDA_Check( hipMalloc3DArray(&sdfArray, &chDesc, make_hipExtent(resolution.x, resolution.y, resolution.z)) );

	hipMemcpy3DParms copyParams = {};
	copyParams.srcPtr = make_hipPitchedPtr(sdfRawData.devdata, resolution.x*sizeof(float), resolution.y, resolution.z);
	copyParams.dstArray = sdfArray;
	copyParams.extent = make_hipExtent(resolution.x, resolution.y, resolution.z);
	copyParams.kind = hipMemcpyDeviceToDevice;

	CUDA_Check( hipMemcpy3D(&copyParams) );

	// Create texture
	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = sdfArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.addressMode[2]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	CUDA_Check( hipCreateTextureObject(&sdfTex, &resDesc, &texDesc, nullptr) );
}

void Wall::attach(ParticleVector* pv)
{
	particleVectors.push_back(pv);
}

void Wall::create(ParticleVector& dpds)
{
	PinnedBuffer<int> nFrozen(1), nRemaining(1), nBoundaryCells(1);

	nFrozen.clear();
	countFrozen<<< (dpds.np + 127) / 128, 128 >>>((float4*)dpds.coosvels.devdata, sdfTex, length, h, nFrozen.devdata);

	nFrozen.synchronize(synchronizeHost);
	frozen.resize(nFrozen[0]);

	nFrozen.   clear();
	nRemaining.clear();
	collectFrozen<<< (dpds.np + 127) / 128, 128 >>>(sdfTex, (float4*)dpds.coosvels.devdata, (float4*)dpds.pingPongBuf.devdata,
			(float4*)frozen.devdata, nRemaining.devdata, nFrozen.devdata);

	nRemaining.synchronize(synchronizeHost);
	dpds.resize(nRemaining[0]);
	swap(dpds.coosvels, dpds.pingPongBuf);


	nBoundaryCells.clear();
	countBoundaryCells<<< (dpds.totcells + 127) / 128, 128 >>> (dpds.ncells, dpds.domainStart, rc, sdfTex, length, h, nBoundaryCells.devdata);

	nBoundaryCells.synchronize(synchronizeHost);
	boundaryCells.resize(nBoundaryCells[0]);
	nBoundaryCells.clear();
	getBoundaryCells<<< (dpds.totcells + 127) / 128, 128 >>> (dpds.ncells, dpds.domainStart, rc, sdfTex, length, h, nBoundaryCells.devdata, boundaryCells.devdata);
}

void Wall::bounce(hipStream_t stream)
{
	const float dt = this->dt;

	for (auto pv : particleVectors)
	{
		flowMacroWrapper( (bounceBeforeIntegration<<< (boundaryCells.size + 127) / 128, 128, 0, stream >>>(
				boundaryCells.devdata, pv->cellsStart.devdata, (float4*)pv->accs.devdata, sdfTex, length, h, 1.0 / h, (float4*)pv->coosvels.devdata, integrate)) );
	}
}

void Wall::computeInteractions(hipStream_t stream)
{
	const float kBT = config.getFloat("Common", "kbt");
	const float gammadpd = config.getFloat("Common", "gamma");
	const float sigmadpd = sqrt(2 * gammadpd * kBT);
	const float adpd = config.getFloat("Common", "a");
	const float seed = 1.0f;

	const float sigma_dt = sigmadpd / sqrt(dt);
	auto dpdInt = [=] __device__ ( const float3 dstCoo, const float3 dstVel, const int dstId,
					   const float3 srcCoo, const float3 srcVel, const int srcId) {
		return dpd_interaction(dstCoo, dstVel, dstId, srcCoo, srcVel, srcId, adpd, gammadpd, sigma_dt, seed);
	};

	const int nth = 128;
	int i = 0;
	for (auto pv : particleVectors)
	{
		debug("Computing wall forces for %d-th particle vector", i++);
		computeExternalInteractions<false, true> <<< (frozen.size + nth - 1) / nth, nth, 0, stream >>>(
				(float4*)frozen.devdata, nullptr, (float4*)pv->coosvels.devdata, (float*)pv->accs.devdata, pv->cellsStart.devdata,
					pv->ncells, pv->domainStart, pv->totcells+1, frozen.size, dpdInt);
	}
}

