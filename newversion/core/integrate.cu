#include "hip/hip_runtime.h"
#include "integrate.h"
#include "non_cached_rw.h"


template<typename Transform>
__global__ void integrationKernel(float4* coosvels, const float4* forces, const int n, const float dt, Transform transform)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int pid = gid / 2;
	const int sh  = gid % 2;  // sh = 0 loads coordinate, sh = 1 -- velocity
	if (pid >= n) return;

	// instead of:
	// const float4 val = in_xyzouvwo[gid];
	//
	// this is to allow more cache for atomics
	// loads / stores here need no cache
	float4 val = coosvels[gid]; //readNoCache(coosvels+gid);
	float4 frc = forces[pid];


	// Send velocity to adjacent thread that has the coordinate
	float4 othval;
	othval.x = __shfl_down(val.x, 1);
	othval.y = __shfl_down(val.y, 1);
	othval.z = __shfl_down(val.z, 1);
	othval.w = __shfl_down(val.w, 1);

	// val is coordinate, othval is corresponding velocity
	if (sh == 0)
		transform(val, othval, frc, dt, pid);

	// val is velocity, othval is rubbish
	if (sh == 1)
		transform(othval, val, frc, dt, pid);

	coosvels[gid] = val; //writeNoCache(coosvels + gid, val);
}



void integrateNoFlow(ParticleVector& pv, const float dt, const float mass, hipStream_t stream)
{
	const float invm = 1.0 / mass;
	auto noflow = [invm] __device__ (float4& x, float4& v, const float4 f, const float dt, const int pid) {
		v.x += f.x*invm*dt;
		v.y += f.y*invm*dt;
		v.z += f.z*invm*dt;

		x.x += v.x*dt;
		x.y += v.y*dt;
		x.z += v.z*dt;
	};

	integrationKernel<<< (2*pv.np + 127)/128, 128, 0, stream >>>((float4*)pv.coosvels.devdata, (float4*)pv.forces.devdata, pv.np, dt, noflow);
	CUDA_Check( hipPeekAtLastError() );
}

void integrateConstDP(ParticleVector& pv, const float dt, const float mass, const float3 extraForce, hipStream_t stream)
{
	const float invm = 1.0 / mass;
	auto constDP = [invm, extraForce] __device__ (float4& x, float4& v, const float4 f, const float dt, const int pid) {
		v.x += (f.x+extraForce.x) * invm*dt;
		v.y += (f.y+extraForce.y) * invm*dt;
		v.z += (f.z+extraForce.z) * invm*dt;

		x.x += v.x*dt;
		x.y += v.y*dt;
		x.z += v.z*dt;
	};

	integrationKernel<<< (2*pv.np + 127)/128, 128, 0, stream >>>((float4*)pv.coosvels.devdata, (float4*)pv.forces.devdata, pv.np, dt, constDP);
	CUDA_Check( hipPeekAtLastError() );
}
