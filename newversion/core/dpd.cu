#include "hip/hip_runtime.h"
#include "dpd.h"
#include "dpd-rng.h"
#include "interaction_engine.h"
#include "interactions.h"
#include "logger.h"

void computeInternalDPD(ParticleVector& pv, hipStream_t stream)
{
	const float dt = 0.0025;
	const float kBT = 1.0;
	const float gammadpd = 20;
	const float sigmadpd = sqrt(2 * gammadpd * kBT);
	const float adpd = 50;
	const float seed = 1.0f;

	const float sigma_dt = sigmadpd / sqrt(dt);
	auto dpdInt = [=] __device__ ( const float3 dstCoo, const float3 dstVel, const int dstId,
					   const float3 srcCoo, const float3 srcVel, const int srcId) {
		return dpd_interaction(dstCoo, dstVel, dstId, srcCoo, srcVel, srcId,
			adpd, gammadpd, sigma_dt, seed);
	};

	hipFuncSetCacheConfig( computeSelfInteractions<decltype(dpdInt)>, hipFuncCachePreferL1 );

	const int nth = 32 * 4;

	debug("Computing internal forces for %d paricles", pv.np);
	computeSelfInteractions<<< (pv.np + nth - 1) / nth, nth, 0, stream >>>(
			(float4*)pv.coosvels.devdata, (float*)pv.accs.devdata, pv.cellsStart.devdata, pv.cellsSize.devdata,
			pv.ncells, pv.domainStart, pv.ncells.x*pv.ncells.y*pv.ncells.z+1, pv.np, dpdInt);
}

void computeHaloDPD(ParticleVector& pv, hipStream_t stream)
{
	const float dt = 0.0025;
	const float kBT = 1.0;
	const float gammadpd = 20;
	const float sigmadpd = sqrt(2 * gammadpd * kBT);
	const float adpd = 50;
	const float seed = 1.0f;

	const float sigma_dt = sigmadpd / sqrt(dt);
	auto dpdInt = [=] __device__ ( const float3 dstCoo, const float3 dstVel, const int dstId,
					   const float3 srcCoo, const float3 srcVel, const int srcId) {
		return dpd_interaction(dstCoo, dstVel, dstId, srcCoo, srcVel, srcId,
			adpd, gammadpd, sigma_dt, seed);
	};

	const int nth = 128;
	debug("Computing halo forces for %d ext paricles", pv.halo.size);
	computeHaloInteractions<false, true> <<< (pv.halo.size + nth - 1) / nth, nth, 0, stream >>>(
			(float4*)pv.halo.devdata, nullptr, (float4*)pv.coosvels.devdata, (float*)pv.accs.devdata, pv.cellsStart.devdata,
				pv.ncells, pv.domainStart, pv.totcells+1, pv.halo.size, dpdInt);
}
