#include "hip/hip_runtime.h"
#include "udevicex.h"

#include <mpi.h>
#include <core/logger.h>
#include <core/simulation.h>
#include <core/postproc.h>
#include <plugins/interface.h>

uDeviceX::uDeviceX(int3 nranks3D, float3 globalDomainSize,
		Logger& logger, std::string logFileName, int verbosity, bool noPostprocess) : noPostprocess(noPostprocess)
{
	int nranks, rank;

	if (logFileName == "stdout")
		logger.init(MPI_COMM_WORLD, stdout, verbosity);
	else if (logFileName == "stderr")
		logger.init(MPI_COMM_WORLD, stderr, verbosity);
	else
		logger.init(MPI_COMM_WORLD, logFileName+".log", verbosity);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );

	if (rank == 0)
		sayHello();

	MPI_Comm ioComm, compComm, interComm, splitComm;

	if (noPostprocess)
	{
		warn("No postprocess will be started now, use this mode for debugging. All the joint plugins will be turned off too.");

		sim = new Simulation(nranks3D, globalDomainSize, MPI_COMM_WORLD, MPI_COMM_NULL);
		computeTask = 0;
		return;
	}

	if (nranks % 2 != 0)
		die("Number of MPI ranks should be even");

	info("Program started, splitting communicator");

	computeTask = (rank) % 2;
	MPI_Check( MPI_Comm_split(MPI_COMM_WORLD, computeTask, rank, &splitComm) );

	if (isComputeTask())
	{
		MPI_Check( MPI_Comm_dup(splitComm, &compComm) );
		MPI_Check( MPI_Intercomm_create(compComm, 0, MPI_COMM_WORLD, 1, 0, &interComm) );

		MPI_Check( MPI_Comm_rank(compComm, &rank) );

		sim = new Simulation(nranks3D, globalDomainSize, compComm, interComm);
	}
	else
	{
		MPI_Check( MPI_Comm_dup(splitComm, &ioComm) );
		MPI_Check( MPI_Intercomm_create(ioComm,   0, MPI_COMM_WORLD, 0, 0, &interComm) );

		MPI_Check( MPI_Comm_rank(ioComm, &rank) );

		post = new Postprocess(ioComm, interComm);
	}
}

void uDeviceX::sayHello()
{
	printf("\n");
	printf("************************************************\n");
	printf("*                   uDeviceX                   *\n");
	printf("*     compiled: on %s at %s     *\n", __DATE__, __TIME__);
	printf("************************************************\n");
	printf("\n");
}

bool uDeviceX::isComputeTask()
{
	return computeTask == 0;
}

void uDeviceX::registerJointPlugins(SimulationPlugin* simPl, PostprocessPlugin* postPl)
{
	if (noPostprocess) return;

	const int id = pluginId++;

	if (isComputeTask())
	{
		simPl->setId(id);
		sim->registerPlugin(simPl);
	}
	else
	{
		postPl->setId(id);
		post->registerPlugin(postPl);
	}
}

void uDeviceX::run(int nsteps)
{
	if (isComputeTask())
	{
		sim->init();
		sim->run(nsteps);
		sim->finalize();

		CUDA_Check( hipDeviceSynchronize() );
	}
	else
		post->run();

	MPI_Finalize();
}

