#include <queue>
#include <unistd.h>

#include <core/task_scheduler.h>
#include <core/logger.h>


TaskScheduler::Node* TaskScheduler::findTaskOrDie(const std::string& label)
{
	auto node = findTask(label);
	if (node == nullptr)
		die("Task group with label %s not found", label.c_str());

	return node;
}

TaskScheduler::Node* TaskScheduler::findTask(const std::string& label)
{
	Node* node = nullptr;
	for (auto n : nodes)
		if (n->label == label) node = n;

	return node;
}




TaskScheduler::TaskScheduler()
{
	CUDA_Check( hipDeviceGetStreamPriorityRange(&cudaPriorityLow, &cudaPriorityHigh) );
}

void TaskScheduler::addTask(std::string label, std::function<void(hipStream_t)> task, int every)
{
	Node* node = findTask(label);

	if (node == nullptr)
	{
		node = new Node();
		node->label = label;
		node->priority = cudaPriorityLow;
		nodes.push_back(node);
	}

	if (every <= 0)
		die("What the fuck is this value %d???", every);

	node->funcs.push_back({task, every});
}


void TaskScheduler::addDependency(std::string label, std::vector<std::string> before, std::vector<std::string> after)
{
	Node* node = findTask(label);
	if (node == nullptr)
	{
		warn("Skipping dependencies for non-existent task '%s'", label.c_str());
		return;
	}

	node->before.insert(node->before.end(), before.begin(), before.end());
	node->after .insert(node->after .end(), after .begin(), after .end());
}

void TaskScheduler::setHighPriority(std::string label)
{
	Node* node = findTaskOrDie(label);

	node->priority = cudaPriorityHigh;
}

void TaskScheduler::forceExec(std::string label)
{
	Node* node = findTaskOrDie(label);

	info("Forced execution of group %s", node->label.c_str());

	for (auto& func_every : node->funcs)
		func_every.first(0);
}

void TaskScheduler::compile()
{
	for (auto& n : nodes)
	{
		// Set streams member according to priority
		if      (n->priority == cudaPriorityLow)
			n->streams = &streamsLo;
		else if (n->priority == cudaPriorityHigh)
			n->streams = &streamsHi;
		else
			n->streams = &streamsLo;

		// Set dependencies
		for (auto& dep : n->before)
		{
			Node* depPtr = nullptr;
			for (auto ndep : nodes)
				if (ndep->label == dep)
				{
					depPtr = ndep;
					break;
				}

			if (depPtr == nullptr)
				die("Could not resolve dependency %s  -->  %s", n->label.c_str(), dep.c_str());

			n->to.push_back(depPtr);
			depPtr->from_backup.push_back(n);
		}

		for (auto& dep : n->after)
		{
			Node* depPtr = nullptr;
			for (auto ndep : nodes)
				if (ndep->label == dep)
				{
					depPtr = ndep;
					break;
				}

			if (depPtr == nullptr)
				die("Could not resolve dependency %s  -->  %s", dep.c_str(),  n->label.c_str());

			n->from_backup.push_back(depPtr);
			depPtr->to.push_back(n);
		}
	}
}


void TaskScheduler::run()
{
	// Kahn's algorithm
	// https://en.wikipedia.org/wiki/Topological_sorting

	auto compareNodes = [] (Node* a, Node* b) {
		// lower number means higher priority
		return a->priority < b->priority;
	};
	std::priority_queue<Node*, std::vector<Node*>, decltype(compareNodes)> S(compareNodes);
	std::vector<std::pair<hipStream_t, Node*>> workMap;

	for (auto n : nodes)
	{
		n->from = n->from_backup;

		if (n->from.empty())
			S.push(n);
	}

	int completed = 0;
	const int total = nodes.size();

	while (true)
	{
		// Check the status of all running kernels
		while (completed < total && S.empty())
		{
			for (auto streamNode_it = workMap.begin(); streamNode_it != workMap.end(); )
			{
				auto result = hipStreamQuery(streamNode_it->first);
				if ( result == hipSuccess )
				{
					auto node = streamNode_it->second;

					info("Completed group %s ", node->label.c_str());

					// Return freed stream back to the corresponding queue
					node->streams->push(streamNode_it->first);

					// Remove resolved dependencies
					for (auto dep : node->to)
					{
						if (!dep->from.empty())
						{
							dep->from.remove(node);
							if (dep->from.empty())
								S.push(dep);
						}
					}

					// Remove task from the list of currently in progress
					completed++;
					streamNode_it = workMap.erase(streamNode_it);
				}
				else if (result == hipErrorNotReady)
				{
					streamNode_it++;
				}
				else CUDA_Check( result );
			}
		}

		if (completed == total)
			break;

		Node* node = S.top();
		S.pop();

		hipStream_t stream;
		if (node->streams->empty())
			CUDA_Check( hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, node->priority) );
		else
		{
			stream = node->streams->front();
			node->streams->pop();
		}

		info("Executing group %s on stream %lld with priority %d", node->label.c_str(), (int64_t)stream, node->priority);
		workMap.push_back({stream, node});

		for (auto& func_every : node->funcs)
			if (nExecutions % func_every.second == 0)
				func_every.first(stream);
	}

	nExecutions++;
	CUDA_Check( hipDeviceSynchronize() );
}





