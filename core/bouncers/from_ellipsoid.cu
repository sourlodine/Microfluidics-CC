/*
 * bounce.cu
 *
 *  Created on: Jul 20, 2017
 *      Author: alexeedm
 */

#include "from_ellipsoid.h"

#include <core/utils/kernel_launch.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>

#include <core/rigid_kernels/bounce.h>
#include <core/rigid_kernels/integration.h>


void BounceFromRigidEllipsoid::exec(ParticleVector* pv, CellList* cl, float dt, hipStream_t stream, bool local)
{
	auto reov = dynamic_cast<RigidEllipsoidObjectVector*>(ov);
	if (reov == nullptr)
		die("Analytic ellispoid bounce only works with RigidObjectVector");

	debug("Bouncing %s particles from %s object vector", pv->name.c_str(), reov->name.c_str());

	REOVview_withOldMotion ovView(reov, local ? reov->local() : reov->halo());
	PVview_withOldParticles pvView(pv, pv->local());

	int nthreads = 256;
	if (!local)
	{
		SAFE_KERNEL_LAUNCH(
				clearRigidForces,
				getNblocks(ovView.nObjects, 64), 64, 0, stream,
				ovView );
	}

	SAFE_KERNEL_LAUNCH(
			bounceEllipsoid,
			ovView.nObjects, nthreads, 2*nthreads*sizeof(int), stream,
			ovView, pvView, cl->cellInfo(), dt );
}



