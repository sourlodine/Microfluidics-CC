/*
 * bounce.cu
 *
 *  Created on: Jul 20, 2017
 *      Author: alexeedm
 */

#include "from_ellipsoid.h"

#include <core/utils/kernel_launch.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>

#include <core/rigid_kernels/bounce.h>

void BounceFromRigidEllipsoid::exec(ParticleVector* pv, CellList* cl, float dt, hipStream_t stream, bool local)
{
	auto reov = dynamic_cast<RigidEllipsoidObjectVector*>(ov);
	if (reov == nullptr)
		die("Analytic ellispoid bounce only works with RigidObjectVector");

	debug("Bouncing %s particles from %s object vector", pv->name.c_str(), reov->name.c_str());

	REOVview ovView(reov, local ? reov->local() : reov->halo());
	PVview pvView(pv, pv->local());

	int nthreads = 512;
	SAFE_KERNEL_LAUNCH(
			bounceEllipsoid,
			ovView.nObjects, nthreads, 2*nthreads*sizeof(int), stream,
			ovView, pvView, cl->cellInfo(), dt );
}



