/*
 * bounce.cu
 *
 *  Created on: Jul 20, 2017
 *      Author: alexeedm
 */

#include "from_ellipsoid.h"

#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>

#include <core/rigid_kernels/bounce.h>

void BounceFromRigidEllipsoid::exec(ObjectVector* ov, ParticleVector* pv, CellList* cl, float dt, hipStream_t stream, bool local)
{
	auto reov = dynamic_cast<RigidEllipsoidObjectVector*>(ov);
	if (reov == nullptr)
		die("Analytic ellispoid bounce only works with RigidObjectVector");

	debug("Bouncing %s particles from %s object vector", pv->name.c_str(), reov->name.c_str());

	auto ovView = create_REOVview(reov, local ? reov->local() : reov->halo());
	auto pvView = create_PVview(pv, pv->local());

	int nthreads = 512;
	bounceEllipsoid<<< ovView.nObjects, nthreads, 2*nthreads*sizeof(int), stream >>> (ovView, pvView, cl->cellsStartSize.devPtr(), cl->cellInfo(), dt);
}



