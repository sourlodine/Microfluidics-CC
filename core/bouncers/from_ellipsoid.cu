/*
 * bounce.cu
 *
 *  Created on: Jul 20, 2017
 *      Author: alexeedm
 */

#include "from_ellipsoid.h"

#include <core/utils/kernel_launch.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>

#include <core/rigid_kernels/bounce.h>
#include <core/rigid_kernels/integration.h>


void BounceFromRigidEllipsoid::exec(ParticleVector* pv, CellList* cl, float dt, hipStream_t stream, bool local)
{
	auto reov = dynamic_cast<RigidEllipsoidObjectVector*>(ov);
	if (reov == nullptr)
		die("Analytic ellispoid bounce only works with RigidObjectVector");

	debug("Bouncing %d %s particles from %d %s objects (%s)",
			pv->local()->size(), pv->name.c_str(),
			local ? reov->local()->size() : reov->halo()->size(), reov->name.c_str(),
			local ? "local objs" : "halo objs");

	ov->findExtentAndCOM(stream, local);

	REOVview_withOldMotion ovView(reov, local ? reov->local() : reov->halo());
	PVview_withOldParticles pvView(pv, pv->local());

	int nthreads = 256;
	if (!local)
	{
		SAFE_KERNEL_LAUNCH(
				clearRigidForces,
				getNblocks(ovView.nObjects, nthreads), nthreads, 0, stream,
				ovView );
	}

	SAFE_KERNEL_LAUNCH(
			bounceEllipsoid,
			ovView.nObjects, nthreads, 2*nthreads*sizeof(int), stream,
			ovView, pvView, cl->cellInfo(), dt );
}



