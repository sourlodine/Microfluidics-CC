#include "hip/hip_runtime.h"
#include <core/datatypes.h>
#include <core/scan.h>
#include <core/celllist.h>
#include <core/non_cached_rw.h>
#include <core/hip/hip_vector_types.h>

__global__ void blendStartSize(const uchar4* cellsSize, int4* cellsStart, const CellListInfo cinfo)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (4*gid >= cinfo.totcells) return;

	uchar4 sizes  = cellsSize [gid];

	cellsStart[gid] += make_int4(sizes.x << 24, sizes.y << 24, sizes.z << 24, sizes.w << 24);
}

__global__ void computeCellSizes(const float4* xyzouvwo, const int n, const int nMovable,
		const CellListInfo cinfo, uint* cellsSize)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= n) return;

	float4 coo = readNoCache(xyzouvwo + pid*2);//xyzouvwo[gid*2];

	int cid = cinfo.getCellId(coo);

	// No atomic for chars
	// Workaround: pad zeros around char in proper position and add as int
	// Care: BIG endian!

	// XXX: relying here only on redistribution
	if (coo.x > -900.0f)
	{
		const int addr = cid / 4;
		const int slot = cid % 4;
		const int increment = 1 << (slot*8);

		atomicAdd(cellsSize + addr, increment);
	}
}

__global__ void rearrangeParticles(const float4* in_xyzouvwo, const int n, const int nMovable,
		const CellListInfo cinfo, uint* cellsSize, const int* cellsStart, float4* out_xyzouvwo)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int pid = gid / 2;
	const int sh  = gid % 2;  // sh = 0 copies coordinates, sh = 1 -- velocity
	if (pid >= n) return;

	int dstId;
	// instead of:
	// const float4 val = in_xyzouvwo[gid];
	//
	// this is to allow more cache for atomics
	// loads / stores here need no cache
	float4 val = readNoCache(in_xyzouvwo+gid);

	int cid;
	if (sh == 0)
	{
		cid = cinfo.getCellId(val);

		//  XXX: relying here only on redistribution
		if (val.x > -900.0f)
		{
			// See above
			const int addr = cid / 4;
			const int slot = cid % 4;
			const int increment = 1 << (slot*8);

			const int rawOffset = atomicAdd(cellsSize + addr, -increment);
			const int offset = ((rawOffset >> (slot*8)) & 255) - 1;

			int2 start_size = cinfo.decodeStartSize(cellsStart[cid]);
			dstId = start_size.x + offset;  // mask blended Start
		}
		else
		{
			dstId = -1;
		}
	}

	int otherDst = __shfl_up(dstId, 1);
	if (sh == 1)
		dstId = otherDst;

	if (dstId >= 0) writeNoCache(out_xyzouvwo + 2*dstId+sh, val);
}


CellListInfo::CellListInfo(float rc, float3 domainStart, float3 length) :
		rc(rc), h(make_float3(rc)), invh(make_float3(1.0/rc)), domainStart(domainStart), length(length)
{
	ncells = make_int3( ceilf(length / rc - 1e-6) );
	totcells = ncells.x * ncells.y * ncells.z;
}

CellListInfo::CellListInfo(float3 h, float3 domainStart, float3 length) :
		domainStart(domainStart), length(length), h(h), invh(1.0f/h)
{
	rc = std::min( {h.x, h.y, h.z} );
	ncells = make_int3( ceilf(length / h - 1e-6f) );
	totcells = ncells.x * ncells.y * ncells.z;
}


CellList::CellList(ParticleVector* pv, float rc, float3 domainStart, float3 length) :
		CellListInfo(rc, domainStart, length), pv(pv)
{
	cellsStart.resize(totcells + 1);
	cellsSize.resize(totcells + 1);
}

CellList::CellList(ParticleVector* pv, int3 resolution, float3 domainStart, float3 length) :
		CellListInfo(length / make_float3(resolution), domainStart, length), pv(pv)
{
	cellsStart.resize(totcells + 1);
	cellsSize.resize(totcells + 1);
}

void CellList::build(hipStream_t stream)
{
	// Containers setup
	pv->pushStreamWOhalo(stream);

	// Compute cell sizes
	debug2("Computing cell sizes for %d particles with %d newcomers", pv->np, pv->received);
	CUDA_Check( hipMemsetAsync(cellsSize.devPtr(), 0, (totcells + 1)*sizeof(uint8_t), stream) );  // +1 to have correct cellsStart[totcells]

	auto cinfo = cellInfo();

	computeCellSizes<<< (pv->np+127)/128, 128, 0, stream >>> (
						(float4*)pv->coosvels.devPtr(), pv->np, pv->np - pv->received, cinfo, (uint*)cellsSize.devPtr());

	// Scan to get cell starts
	scan(cellsSize.devPtr(), totcells+1, cellsStart.devPtr(), stream);

	// Blend size and start together
	blendStartSize<<< ((totcells+3)/4 + 127) / 128, 128, 0, stream >>>((uchar4*)cellsSize.devPtr(), (int4*)cellsStart.devPtr(), cinfo);

	// Rearrange the data
	debug2("Rearranging %d particles", pv->np);

	rearrangeParticles<<< (2*pv->np+127)/128, 128, 0, stream >>> (
						(float4*)pv->coosvels.devPtr(), pv->np, pv->np - pv->received, cinfo,
						(uint*)cellsSize.devPtr(), cellsStart.devPtr(), (float4*)pv->pingPongBuf.devPtr());


	// Now we need the new size of particles array.
	int newSize;
	CUDA_Check( hipMemcpyAsync(&newSize, cellsStart.devPtr() + totcells, sizeof(int), hipMemcpyDeviceToHost, stream) );
	CUDA_Check( hipStreamSynchronize(stream) );
	debug2("Rearranging completed, new size of particle vector is %d", newSize);

	pv->resize(newSize, resizePreserve);
	CUDA_Check( hipStreamSynchronize(stream) );
	containerSwap(pv->coosvels, pv->pingPongBuf);

	// Containers setup
	pv->popStreamWOhalo();

	// TODO: is this fine? need something for not the first celllist
	pv->received = 0;//pv->np;
}
