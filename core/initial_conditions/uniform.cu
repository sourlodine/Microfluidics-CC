#include "hip/hip_runtime.h"
#include "uniform.h"

#include <random>

#include <core/pvs/particle_vector.h>
#include <core/logger.h>

void UniformIC::exec(const MPI_Comm& comm, ParticleVector* pv, float3 globalDomainStart, float3 localDomainSize, hipStream_t stream)
{
	int3 ncells = make_int3( ceilf(localDomainSize) );
	float3 h = localDomainSize / make_float3(ncells);

	float volume = h.x*h.y*h.z;
	float avg = volume * density;
	int predicted = round(avg * ncells.x*ncells.y*ncells.z * 1.05);
	pv->local()->resize(predicted, stream, ResizeKind::resizeAnew);

	int rank;
	MPI_Check( MPI_Comm_rank(comm, &rank) );

	std::hash<std::string> nameHash;
	const int seed = rank + nameHash(pv->name);
	std::mt19937 gen(seed);
	std::poisson_distribution<> particleDistribution(avg);
	std::uniform_real_distribution<float> udistr(0, 1);

	int mycount = 0;
	auto cooPtr = pv->local()->coosvels.hostPtr();
	for (int i=0; i<ncells.x; i++)
		for (int j=0; j<ncells.y; j++)
			for (int k=0; k<ncells.z; k++)
			{
				int nparts = particleDistribution(gen);
				for (int p=0; p<nparts; p++)
				{
					pv->local()->resize(mycount+1, stream, ResizeKind::resizePreserve);
					cooPtr[mycount].r.x = i*h.x - 0.5*localDomainSize.x + udistr(gen);
					cooPtr[mycount].r.y = j*h.y - 0.5*localDomainSize.y + udistr(gen);
					cooPtr[mycount].r.z = k*h.z - 0.5*localDomainSize.z + udistr(gen);
					cooPtr[mycount].i1 = mycount;

					cooPtr[mycount].u.x = 0*udistr(gen);
					cooPtr[mycount].u.y = 0*udistr(gen);
					cooPtr[mycount].u.z = 0*udistr(gen);

					cooPtr[mycount].i1 = mycount;
					mycount++;
				}
			}

	pv->globalDomainStart = globalDomainStart;
	pv->localDomainSize = localDomainSize;

	int totalCount=0; // TODO: int64!
	MPI_Check( MPI_Exscan(&mycount, &totalCount, 1, MPI_INT, MPI_SUM, comm) );
	for (int i=0; i < pv->local()->size(); i++)
		cooPtr[i].i1 += totalCount;

	pv->local()->coosvels.uploadToDevice(stream);

	debug2("Generated %d %s particles", pv->local()->size(), pv->name.c_str());
}
