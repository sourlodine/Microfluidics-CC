#include "hip/hip_runtime.h"
#include "mesh_belonging.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>
#include <core/celllist.h>

#include <core/rigid_kernels/quaternion.h>
#include <core/rigid_kernels/rigid_motion.h>

const float tolerance = 1e-6f;

__device__ __forceinline__ float whichTriangSide(float3 r, float3 a, float3 b, float3 c)
{
	return dot(r-a, cross(b-a, c-a));
}

// Mesh normals look INSIDE
__device__ __forceinline__ int particleInsideTetrahedron(float3 r, float3 v0, float3 v1, float3 v2, float3 v3)
{
	float s0 = whichTriangSide(r,  v2, v1, v3);
	float s1 = whichTriangSide(r,  v0, v1, v2);
	float s2 = whichTriangSide(r,  v0, v2, v3);
	float s3 = whichTriangSide(r,  v0, v3, v1);

	if (s0 < 0 || s1 < 0 || s2 < 0 || s3 < 0)
		return 0;

//	if (fabs(s0) < tolerance) return 1;
//	if (fabs(s1) < tolerance) return 1;
//	if (fabs(s2) < tolerance) return 1;
//	if (fabs(s3) < tolerance) return 1;

	return 2;
}


/**
 * One warp works on one particle
 */
__device__ BelongingTags oneParticleInsideMesh(int pid, float3 r, int objId, const float3 com, const MeshView mesh)
{
	int counter = 0;

	// Work in obj reference frame for simplicity
	r = r - com;

	float3 tot = make_float3(0);

	for (int i = __laneid(); i < mesh.ntriangles; i += warpSize)
	{
		float3 v0 = make_float3(0.0f);

		int3 trid = mesh.triangles[i];

		float3 v1 = Particle(mesh.vertices, objId*mesh.nvertices + trid.x).r - com;
		float3 v2 = Particle(mesh.vertices, objId*mesh.nvertices + trid.y).r - com;
		float3 v3 = Particle(mesh.vertices, objId*mesh.nvertices + trid.z).r - com;

		// If the particle is very close to the boundary
		// return immediately
		if ( fabs( dot(r-v1, normalize(cross(v2-v1, v3-v1))) ) < 10*tolerance )
			return BelongingTags::Boundary;

		// += 2 if inside
		// += 1 if exactly on a side
		counter += particleInsideTetrahedron(r, v0, v1, v2, v3);
	}

	counter = warpReduce(counter, [] (int a, int b) { return a+b; });

	// Incorrect result. Disregard the guy just in case
	if (counter % 2 != 0) return BelongingTags::Boundary;


	// Inside even number of tetrahedra => outside of object
	if ( (counter/2) % 2 == 0 ) return BelongingTags::Outside;

	// Inside odd number of tetrahedra => inside object
	if ( (counter/2) % 2 != 0 ) return BelongingTags::Inside;

	// Shut up compiler warning
	return BelongingTags::Boundary;
}

/**
 * OVview view is only used to provide # of objects and extent information
 * Actual data is in mesh.vertices
 * cinfo is the cell-list sync'd with the target ParticleVector data
 */
template<int WARPS_PER_OBJ>
__global__ void insideMesh(const OVview view, const MeshView mesh, CellListInfo cinfo, BelongingTags* tags)
{
	const int gid = blockIdx.x*blockDim.x + threadIdx.x;
	const int wid = gid / warpSize;
	const int objId = wid / WARPS_PER_OBJ;

	const int locWid = wid % WARPS_PER_OBJ;

	if (objId >= view.nObjects) return;

	const int3 cidLow  = cinfo.getCellIdAlongAxes(view.comAndExtents[objId].low  - 0.5f);
	const int3 cidHigh = cinfo.getCellIdAlongAxes(view.comAndExtents[objId].high + 0.5f);

	const int3 span = cidHigh - cidLow + make_int3(1,1,1);
	const int totCells = span.x * span.y * span.z;

	for (int i=locWid; i<totCells; i+=WARPS_PER_OBJ)
	{
		const int3 cid3 = make_int3( i % span.x, (i/span.x) % span.y, i / (span.x*span.y) ) + cidLow;
		const int  cid = cinfo.encode(cid3);
		if (cid < 0 || cid >= cinfo.totcells) continue;

		int pstart = cinfo.cellStarts[cid];
		int pend   = cinfo.cellStarts[cid+1];

		for (int pid = pstart; pid < pend; pid++)
		{
			const Particle p(cinfo.particles, pid);

			auto tag = oneParticleInsideMesh(pid, p.r, objId, view.comAndExtents[objId].com, mesh);

			// Only tag particles inside, default is outside anyways
			if (__laneid() == 0 && tag != BelongingTags::Outside)
				tags[pid] = tag;
		}
	}
}


void MeshBelongingChecker::tagInner(ParticleVector* pv, CellList* cl, hipStream_t stream)
{
	int nthreads = 128;

	tags.resize_anew(pv->local()->size());
	tags.clearDevice(stream);

	const int warpsPerObject = 32;

	ov->findExtentAndCOM(stream, true);
	ov->findExtentAndCOM(stream, false);

	// Local
	auto lov = ov->local();
	auto view = OVview(ov, lov);
	auto vertices = lov->getMeshVertices(stream);
	auto meshView = MeshView(ov->mesh, lov->getMeshVertices(stream));

	debug("Computing inside/outside tags (against mesh) for %d local objects '%s' and %d '%s' particles",
			view.nObjects, ov->name.c_str(), pv->local()->size(), pv->name.c_str());

	SAFE_KERNEL_LAUNCH(
			insideMesh<warpsPerObject>,
			getNblocks(warpsPerObject*32*view.nObjects, nthreads), nthreads, 0, stream,
			view, meshView, cl->cellInfo(), tags.devPtr());

	// Halo
	lov = ov->halo();       // Note ->halo() here
	view = OVview(ov, lov);
	vertices = lov->getMeshVertices(stream);
	meshView = MeshView(ov->mesh, lov->getMeshVertices(stream));

	debug("Computing inside/outside tags (against mesh) for %d halo objects '%s' and %d '%s' particles",
			view.nObjects, ov->name.c_str(), pv->local()->size(), pv->name.c_str());

	SAFE_KERNEL_LAUNCH(
			insideMesh<warpsPerObject>,
			getNblocks(warpsPerObject*32*view.nObjects, nthreads), nthreads, 0, stream,
			view, meshView, cl->cellInfo(), tags.devPtr());
}



