#include "hip/hip_runtime.h"
#include <core/object_vector.h>
#include <core/hip/hip_vector_types.h>

template<typename Operation>
__inline__ __device__ float3 warpReduce(float3 val, Operation op)
{
#pragma unroll
	for (int offset = warpSize/2; offset > 0; offset /= 2)
	{
		val.x = op(val.x, __shfl_down(val.x, offset));
		val.y = op(val.y, __shfl_down(val.y, offset));
		val.z = op(val.z, __shfl_down(val.z, offset));
	}
	return val;
}

__global__ void min_max_com(const float4 * coosvels, ObjectVector::Properties* props, const int nObj, const int objSize)
{
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	const int objId = gid >> 5;
	const int tid = gid & 0x1f;
	if (objId >= nObj) return;

	float3 mymin = make_float3( 1e+10f);
	float3 mymax = make_float3(-1e+10f);
	float3 mycom = make_float3(0);

#pragma unroll 3
	for (int i = tid; i < objSize; i += warpSize)
	{
		const int offset = (objId * objSize + i) * 2;

		const float3 coo = make_float3(coosvels[offset]);

		mymin = fminf(mymin, coo);
		mymax = fmaxf(mymax, coo);
		mycom += coo;
	}

	mycom = warpReduce( mycom, [] (float a, float b) { return a+b; } );
	mymin = warpReduce( mymin, [] (float a, float b) { return fmin(a, b); } );
	mymax = warpReduce( mymax, [] (float a, float b) { return fmax(a, b); } );

	if (tid == 0)
		props[objId] = {mycom / objSize, mymin, mymax};
}

void ObjectVector::findExtentAndCOM(hipStream_t stream)
{
	const int nthreads = 128;
	min_max_com<<< (nObjects*32 + nthreads-1)/nthreads, nthreads, 0, stream >>> ((float4*)coosvels.devPtr(), properties.devPtr(), nObjects, objSize);
}
