#include "hip/hip_runtime.h"
#include "simple_stationary_wall.h"

#include <fstream>
#include <cmath>
#include <texture_types.h>
#include <cassert>

#include <core/logger.h>
#include <core/utils/kernel_launch.h>
#include <core/utils/cuda_common.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/bounce_solver.h>

#include <core/utils/cuda_rng.h>

#include "stationary_walls/cylinder.h"
#include "stationary_walls/sdf.h"
#include "stationary_walls/sphere.h"

//===============================================================================================
// Removing kernels
//===============================================================================================

template<typename InsideWallChecker>
__global__ void collectRemaining(PVview view, float4* remaining, int* nRemaining, InsideWallChecker checker)
{
	const float tolerance = 1e-6f;

	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= view.size) return;

	Particle p(view.particles, pid);

	const float val = checker(p.r);

	if (val <= -tolerance)
	{
		const int ind = atomicAggInc(nRemaining);
		p.write2Float4(remaining, ind);
	}
}

template<typename InsideWallChecker>
__global__ void packRemainingObjects(OVviewWithExtraData view, char* output, int* nRemaining, InsideWallChecker checker)
{
	const float tolerance = 1e-6f;

	// One warp per object
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int objId = gid / warpSize;
	const int tid = gid % warpSize;

	if (objId >= view.nObjects) return;

	bool isRemaining = true;
	for (int i=tid; i < view.objSize; i++)
	{
		Particle p(view.particles, objId * view.objSize + i);
		if (checker(p.r) <= -tolerance)
		{
			isRemaining = false;
			break;
		}
	}

	if (!isRemaining) return;

	int dstObjId;
	if (tid == 0)
		dstObjId = atomicAggInc(nRemaining);
	dstObjId = __shfl(dstObjId, 0);


	Particle* dstAddr = (Particle*)(output + dstObjId * view.packedObjSize_byte);
	for (int i=tid; i < view.objSize; i+=warpSize)
		dstAddr[i] = Particle(view.particles, objId * view.objSize + i);

	view.packExtraData(objId, (char*)(dstAddr+view.objSize));
}

__global__ void unpackRemainingObjects(OVviewWithExtraData view, const char* input)
{
	// One warp per object
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int objId = gid / warpSize;
	const int tid = gid % warpSize;

	if (objId >= view.nObjects) return;

	Particle* srcAddr = (Particle*)(input + objId * view.packedObjSize_byte);
	for (int i=tid; i < view.objSize; i+=warpSize)
		((Particle*)view.particles)[objId * view.objSize + i] = srcAddr[i];

	view.unpackExtraData(objId, (char*)(srcAddr+view.objSize));
}
//===============================================================================================
// Boundary cells kernels
//===============================================================================================

template<typename InsideWallChecker>
__device__ __forceinline__ bool isCellOnBoundary(PVview view, float3 cornerCoo, float3 len, InsideWallChecker checker)
{
	// About maximum distance a particle can cover in one step
	const float tol = 0.25f;
	int pos = 0, neg = 0;

	for (int i=0; i<2; i++)
		for (int j=0; j<2; j++)
			for (int k=0; k<2; k++)
			{
				// Value in the cell corner
				const float3 shift = make_float3(i ? len.x : 0.0f, j ? len.y : 0.0f, k ? len.z : 0.0f);
				const float s = checker(cornerCoo + shift);

				if (s >  tol) pos++;
				if (s < -tol) neg++;
			}

	return (pos != 8 && neg != 8);
}

template<bool QUERY, typename InsideWallChecker>
__global__ void getBoundaryCells(PVview view, CellListInfo cinfo, int* nBoundaryCells, int* boundaryCells, InsideWallChecker checker)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;

	int3 ind;
	cinfo.decode(cid, ind.x, ind.y, ind.z);
	float3 cornerCoo = -0.5f*cinfo.localDomainSize + make_float3(ind)*cinfo.h;

	if (isCellOnBoundary(view, cornerCoo, cinfo.h, checker))
	{
		int id = atomicAggInc(nBoundaryCells);
		if (!QUERY) boundaryCells[id] = cid;
	}
}

//===============================================================================================
// SDF bouncing kernel
//===============================================================================================

template<typename InsideWallChecker>
__global__ void bounceKernel(PVview view, const int* wallCells, const int nWallCells, CellListInfo cinfo, const float dt, InsideWallChecker checker)
{
	const int maxIters = 50;
	const float corrStep = (1.0f / (float)maxIters) * dt;

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nWallCells) return;
	const int cid = wallCells[tid];
	const int pstart = cinfo.cellStarts[cid];
	const int pend   = cinfo.cellStarts[cid+1];

	for (int pid = pstart; pid < pend; pid++)
	{
		Particle p(cinfo.particles, pid);
		if (checker(p.r) <= 0.0f) continue;

		float3 oldCoo = p.r - p.u*dt;

		for (int i=0; i<maxIters; i++)
		{
			if (checker(oldCoo) < 0.0f) break;
			oldCoo -= p.u*corrStep;
		}

		const float alpha = solveLinSearch([=] (float lambda) {
			return checker(oldCoo + (p.r-oldCoo)*lambda);
		});
		float3 candidate = (alpha >= 0.0f) ? oldCoo + alpha * (p.r - oldCoo) : oldCoo;

		if (checker(candidate) >= 0.0f)
		for (int i=0; i<maxIters; i++)
		{
			if (checker(candidate) < 0.0f) break;

			float3 rndShift;
				rndShift.x = Saru::mean0var1(p.r.x - floorf(p.r.x), p.i1+i, p.i1*p.i1);
				rndShift.y = Saru::mean0var1(rndShift.x,            p.i1+i, p.i1*p.i1);
				rndShift.z = Saru::mean0var1(rndShift.y,            p.i1+i, p.i1*p.i1);

				if (checker(candidate + 5.0f*rndShift*dt) < 0.0f)
				{
					candidate += 5.0f*rndShift*dt;
					break;
				}
		}

		p.r = candidate;
		p.u = -p.u;

		p.write2Float4(cinfo.particles, pid);
	}
}

//===============================================================================================
// Checking kernel
//===============================================================================================

template<typename InsideWallChecker>
__global__ void checkInside(PVview view, int* nInside, InsideWallChecker checker)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= view.size) return;

	Float3_int coo(view.particles[2*pid]);

	float v = checker(coo.v);

	if (v > 0) atomicAggInc(nInside);
}

//===============================================================================================
// Member functions
//===============================================================================================

template<class InsideWallChecker>
void SimpleStationaryWall<InsideWallChecker>::setup(MPI_Comm& comm, DomainInfo domain)
{
	info("Setting up wall %s", name.c_str());

	CUDA_Check( hipDeviceSynchronize() );
	MPI_Check( MPI_Comm_dup(comm, &wallComm) );

	insideWallChecker.setup(wallComm, domain);

	CUDA_Check( hipDeviceSynchronize() );
}


template<class InsideWallChecker>
void SimpleStationaryWall<InsideWallChecker>::attach(ParticleVector* pv, CellList* cl)
{
	if (dynamic_cast<PrimaryCellList*>(cl) == nullptr)
		die("PVs should only be attached to walls with the primary cell-lists! "
				"Invalid combination: wall %s, pv %s", name.c_str(), pv->name.c_str());

	CUDA_Check( hipDeviceSynchronize() );
	particleVectors.push_back(pv);
	cellLists.push_back(cl);
	nBounceCalls.push_back(0);

	PVview view(pv, pv->local());
	PinnedBuffer<int> nBoundaryCells(1);
	nBoundaryCells.clear(0);
	SAFE_KERNEL_LAUNCH(
			getBoundaryCells<true>,
			(cl->totcells + 127) / 128, 128, 0, 0,
			view, cl->cellInfo(), nBoundaryCells.devPtr(), nullptr, insideWallChecker.handler() );

	nBoundaryCells.downloadFromDevice(0);

	debug("Found %d boundary cells", nBoundaryCells[0]);
	auto bc = new DeviceBuffer<int>(nBoundaryCells[0]);

	nBoundaryCells.clear(0);
	SAFE_KERNEL_LAUNCH(
			getBoundaryCells<false>,
			(cl->totcells + 127) / 128, 128, 0, 0,
			view, cl->cellInfo(), nBoundaryCells.devPtr(), bc->devPtr(), insideWallChecker.handler() );

	boundaryCells.push_back(bc);
	CUDA_Check( hipDeviceSynchronize() );
}



template<class InsideWallChecker>
void SimpleStationaryWall<InsideWallChecker>::removeInner(ParticleVector* pv)
{
	CUDA_Check( hipDeviceSynchronize() );

	PinnedBuffer<int> nRemaining(1);
	nRemaining.clear(0);

	int oldSize = pv->local()->size();
	if (oldSize == 0) return;

	const int nthreads = 128;
	// Need a different path for objects
	ObjectVector* ov = dynamic_cast<ObjectVector*>(pv);
	if (ov == nullptr)
	{
		PVview view(pv, pv->local());
		PinnedBuffer<Particle> tmp(view.size);

		SAFE_KERNEL_LAUNCH(
				collectRemaining,
				getNblocks(view.size, nthreads), nthreads, 0, 0,
				view, (float4*)tmp.devPtr(), nRemaining.devPtr(), insideWallChecker.handler() );

		nRemaining.downloadFromDevice(0);
		std::swap(pv->local()->coosvels, tmp);
		int oldSize = pv->local()->size();
		pv->local()->resize(nRemaining[0], 0);
	}
	else
	{
		// Prepare temp storage for extra object data
		OVviewWithExtraData ovView(ov, ov->local(), 0);
		DeviceBuffer<char> tmp(ovView.nObjects * ovView.packedObjSize_byte);

		SAFE_KERNEL_LAUNCH(
				packRemainingObjects,
				getNblocks(ovView.nObjects*32, nthreads), nthreads, 0, 0,
				ovView,	tmp.devPtr(), nRemaining.devPtr(), insideWallChecker.handler() );

		// Copy temporary buffers back
		nRemaining.downloadFromDevice(0);
		ov->local()->resize_anew(nRemaining[0]);
		ovView = OVviewWithExtraData(ov, ov->local(), 0);
		SAFE_KERNEL_LAUNCH(
				unpackRemainingObjects,
				getNblocks(ovView.nObjects*32, nthreads), nthreads, 0, 0,
				ovView, tmp.devPtr() );
	}

	pv->haloValid = false;
	pv->redistValid = false;
	pv->cellListStamp++;

	info("Removed inner entities of %s, keeping %d out of %d particles",
			pv->name.c_str(), pv->local()->size(), oldSize);

	CUDA_Check( hipDeviceSynchronize() );
}

template<class InsideWallChecker>
void SimpleStationaryWall<InsideWallChecker>::bounce(float dt, hipStream_t stream)
{
	for (int i=0; i<particleVectors.size(); i++)
	{
		auto pv = particleVectors[i];
		auto cl = cellLists[i];
		auto bc = boundaryCells[i];
		PVview view(pv, pv->local());

		debug2("Bouncing %d %s particles, %d boundary cells",
				pv->local()->size(), pv->name.c_str(), bc->size());

		const int nthreads = 64;
		SAFE_KERNEL_LAUNCH(
				bounceKernel,
				getNblocks(bc->size(), nthreads), nthreads, 0, stream,
				view, bc->devPtr(), bc->size(), cl->cellInfo(), dt, insideWallChecker.handler() );

		CUDA_Check( hipPeekAtLastError() );
		nBounceCalls[i]++;
	}
}

template<class InsideWallChecker>
void SimpleStationaryWall<InsideWallChecker>::check(hipStream_t stream)
{
	const int nthreads = 128;
	for (int i=0; i<particleVectors.size(); i++)
	{
		auto pv = particleVectors[i];
		{
			nInside.clearDevice(stream);
			PVview view(pv, pv->local());
			SAFE_KERNEL_LAUNCH(
					checkInside,
					getNblocks(view.size, nthreads), nthreads, 0, stream,
					view, nInside.devPtr(), insideWallChecker.handler() );

			nInside.downloadFromDevice(stream);

			info("%d particles of %s are inside the wall %s", nInside[0], pv->name.c_str(), name.c_str());
		}
	}
}

template class SimpleStationaryWall<StationaryWall_Sphere>;
template class SimpleStationaryWall<StationaryWall_Cylinder>;
template class SimpleStationaryWall<StationaryWall_SDF>;




