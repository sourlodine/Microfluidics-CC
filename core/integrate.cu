#include "hip/hip_runtime.h"
#include <core/integrate.h>
#include <core/particle_vector.h>
#include <core/object_vector.h>
#include <core/logger.h>
#include <core/hip/hip_vector_types.h>
#include <core/cuda_common.h>

// Workaround for nsight
#ifndef __CUDACC_EXTENDED_LAMBDA__
#define __device__
#endif

/**
 * transform(float4& x, float4& v, const float4 f, const float invm, const float dt):
 *  performs integration
 */
template<typename Transform>
__global__ void integrationKernel(float4* coosvels, const float4* forces, const int n, const float invmass, const float dt, Transform transform)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int pid = gid / 2;
	const int sh  = gid % 2;  // sh = 0 loads coordinate, sh = 1 -- velocity
	if (pid >= n) return;

	float4 val = coosvels[gid]; //readNoCache(coosvels+gid);
	float4 frc = forces[pid];

	// Send velocity to adjacent thread that has the coordinate
	float4 othval;
	othval.x = __shfl_down(val.x, 1);
	othval.y = __shfl_down(val.y, 1);
	othval.z = __shfl_down(val.z, 1);
	othval.w = __shfl_down(val.w, 1);

	// val is coordinate, othval is corresponding velocity
	if (sh == 0)
		transform(val, othval, frc, invmass, dt);

	// val is velocity, othval is rubbish
	if (sh == 1)
		transform(othval, val, frc, invmass, dt);

	coosvels[gid] = val; //writeNoCache(coosvels + gid, val);
}

template<typename Transform>
__global__ void integrateRigidKernel(float4 * coosvels, const float4 * forces, ObjectVector::COMandExtent* props, const int nObj, const int objSize,
		const float invmass, const float dt, Transform transform)
{
	// http://math.stackexchange.com/questions/519200/dot-product-and-cross-product-solving-a-set-of-simultaneous-vector-equations

	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	const int objId = gid >> 5;
	const int tid = gid & 0x1f;
	if (objId >= nObj) return;

	float3 force  = make_float3(0);
	float3 torque = make_float3(0);
	const float3 com = props[objId].com;

	// Find the total force and torque
#pragma unroll 3
	for (int i = tid; i < objSize; i += warpSize)
	{
		const int offset = (objId * objSize + i);

		const float3 frc = make_float3(coosvels[offset]);
		const float3 r   = make_float3(coosvels[offset*2]) - com;

		force += frc;
		torque += cross(r, frc);
	}

	force  = warpReduce( force,  [] (float a, float b) { return a+b; } ) / objSize;
	torque = warpReduce( torque, [] (float a, float b) { return a+b; } ) / objSize;

	force.x  = __shfl(force.x, 0);
	force.y  = __shfl(force.y, 0);
	force.z  = __shfl(force.z, 0);

	torque.x = __shfl(torque.x, 0);
	torque.y = __shfl(torque.y, 0);
	torque.z = __shfl(torque.z, 0);

	// Distribute the force and torque per particle
#pragma unroll 3
	for (int i = tid; i < objSize; i += warpSize)
	{
		const int offset = (objId * objSize + i) * 2;

		float4 r = coosvels[offset];
		float4 v = coosvels[offset+1];

		// Force consists of translational and rotational components
		// first is just average force, second comes from a solution of:
		//
		//  torque = r x f,  f*r = 0
		//
		const float3 f = force + cross(torque, make_float3(r)) / dot(r, r);

		transform(r, v, make_float4(f), invmass, dt);

		coosvels[offset]   = r;
		coosvels[offset+1] = v;
	}
}

//==============================================================================================
//==============================================================================================

__device__ __forceinline__ void _noflow (float4& x, float4& v, const float4 f, const float invm, const float dt)
{
	v.x += f.x*invm*dt;
	v.y += f.y*invm*dt;
	v.z += f.z*invm*dt;

	x.x += v.x*dt;
	x.y += v.y*dt;
	x.z += v.z*dt;
}

__device__ __forceinline__ void _constDP (float4& x, float4& v, const float4 f, const float invm, const float dt, const float3 extraForce)
{
	v.x += (f.x+extraForce.x) * invm*dt;
	v.y += (f.y+extraForce.y) * invm*dt;
	v.z += (f.z+extraForce.z) * invm*dt;

	x.x += v.x*dt;
	x.y += v.y*dt;
	x.z += v.z*dt;
}

/**
 * Free flow
 */
void integrateNoFlow(ParticleVector* pv, const float dt, hipStream_t stream)
{
	auto noflow = [] __device__ (float4& x, float4& v, const float4 f, const float invm, const float dt) {
		_noflow(x, v, f, invm, dt);
	};

	debug2("Integrating %d %s particles, timestep is %f", pv->np, pv->name.c_str(), dt);
	integrationKernel<<< (2*pv->np + 127)/128, 128, 0, stream >>>((float4*)pv->coosvels.devPtr(), (float4*)pv->forces.devPtr(), pv->np, 1.0/pv->mass, dt, noflow);
}

/**
 * Applied additional force to every particle
 */
void integrateConstDP(ParticleVector* pv, const float dt, hipStream_t stream, float3 extraForce)
{
	auto constDP = [extraForce] __device__ (float4& x, float4& v, const float4 f, const float invm, const float dt) {
		_constDP(x, v, f, invm, dt, extraForce);
	};

	debug2("Integrating %d %s particles, timestep is %f", pv->np, pv->name.c_str(), dt);
	integrationKernel<<< (2*pv->np + 127)/128, 128, 0, stream >>>((float4*)pv->coosvels.devPtr(), (float4*)pv->forces.devPtr(), pv->np, 1.0/pv->mass, dt, constDP);
}

/**
 * Rotate with constant angular velocity omega around x0, regardless force
 */
void integrateConstOmega(ParticleVector* pv, const float dt, hipStream_t stream, const float3 omega, const float3 x0)
{
	// https://en.wikipedia.org/wiki/Rodrigues%27_rotation_formula

	const float3 locX0 = x0 - pv->globalDomainStart;

	const float IomegaI = sqrt(dot(omega, omega));
	const float phi     = IomegaI * dt;
	const float sphi    = sin(phi);
	const float cphi    = cos(phi);

	const float3 k = omega / IomegaI;

	auto rotate = [k, sphi, cphi, locX0] __device__ (float4& x, float4& v, const float4 f, const float invm, const float dt) {
		float3 r = make_float3(x) - locX0;
		r = r * cphi + cross(k, r)*sphi * k*dot(k, r) * (1-cphi);
		x.x = r.x;
		x.y = r.y;
		x.z = r.z;
	};

	integrationKernel<<< (2*pv->np + 127)/128, 128, 0, stream >>>((float4*)pv->coosvels.devPtr(), (float4*)pv->forces.devPtr(), pv->np, 1.0/pv->mass, dt, rotate);
}

void integrateRigid(ObjectVector* ov, const float dt, hipStream_t stream, float3 extraForce)
{
	auto noflow = [] __device__ (float4& x, float4& v, const float4 f, const float invm, const float dt) {
		_noflow(x, v, f, invm, dt);
	};

	debug2("Integrating %d objecst %s, timestep is %f", ov->nObjects, ov->name.c_str(), dt);

	const int nthreads = 128;
	integrateRigidKernel<<< (ov->nObjects*32 + nthreads-1)/nthreads, nthreads, 0, stream >>> (
			(float4*)ov->coosvels.devPtr(), (float4*)ov->forces.devPtr(), ov->com_extent.devPtr(),
			ov->nObjects, ov->objSize, 1.0/ov->mass, dt, noflow);
}










