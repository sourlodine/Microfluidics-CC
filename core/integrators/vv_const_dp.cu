#include "hip/hip_runtime.h"
#include "vv_const_dp.h"
#include "integration_kernel.h"

#include <core/logger.h>
#include <core/pvs/particle_vector.h>


void IntegratorVVConstDP::stage1(ParticleVector* pv, hipStream_t stream)
{
//	auto st1 = [=] __device__ (Particle& p, const float3 f, const float invm, const float dt) {
//		p.u += 0.5*(f+extraForce)*invm*dt;
//		p.r += p.u*dt;
//	};
//
//	int nthreads = 128;
//	debug2("Integrating (stage 1) %d %s particles with extra force [%8.5f %8.5f %8.5f], timestep is %f", pv->local()->size(), pv->name.c_str(), dt);
//	integrationKernel<<< getNblocks(2*pv->local()->size(), nthreads), nthreads, 0, stream >>>(
//			(float4*)pv->local()->coosvels.devPtr(), (float4*)pv->local()->forces.devPtr(), pv->local()->size(), 1.0/pv->mass, dt, st1);
//	pv->local()->changedStamp++;
}

void IntegratorVVConstDP::stage2(ParticleVector* pv, hipStream_t stream)
{
	auto ef = extraForce;
	auto st2 = [ef] __device__ (Particle& p, const float3 f, const float invm, const float dt) {
		p.u += (f+ef)*invm*dt;
		p.r += p.u*dt;
	};

	int nthreads = 128;
	debug2("Integrating (stage 2) %d %s particles with extra force [%8.5f %8.5f %8.5f], timestep is %f",
			pv->local()->size(), pv->name.c_str(), extraForce.x, extraForce.y, extraForce.z, dt);

	if (pv->local()->size() > 0)
		integrationKernel<<< getNblocks(2*pv->local()->size(), nthreads), nthreads, 0, stream >>>(
				(float4*)pv->local()->coosvels.devPtr(), (float4*)pv->local()->forces.devPtr(), pv->local()->size(), 1.0/pv->mass, dt, st2);
	pv->local()->changedStamp++;
}
