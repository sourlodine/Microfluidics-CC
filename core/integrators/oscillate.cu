#include "hip/hip_runtime.h"
#include "oscillate.h"
#include "integration_kernel.h"

#include <core/utils/kernel_launch.h>
#include <core/logger.h>
#include <core/pvs/particle_vector.h>


IntegratorOscillate::IntegratorOscillate(std::string name, float dt, float3 vel, int period) :
	Integrator(name, dt),
	vel(vel), period(period)
{
	if (period <= 0)
		die("Oscillating period should be strictly positive");
}

/**
 * Oscillate with cos wave in time, regardless force
 */
void IntegratorOscillate::stage2(ParticleVector* pv, float t, hipStream_t stream)
{
	const auto _vel = vel;
	float cosOmega = cos(2*M_PI * (float)count / period);
	count++;

	auto oscillate = [_vel, cosOmega] __device__ (Particle& p, const float3 f, const float invm, const float dt) {
		p.u = _vel * cosOmega;
		p.r += p.u*dt;
	};

	int nthreads = 128;

	// New particles now become old
	std::swap(pv->local()->coosvels, *pv->local()->extraPerParticle.getData<Particle>("old_particles"));
	PVviewWithOldParticles pvView(pv, pv->local());

	SAFE_KERNEL_LAUNCH(
			integrationKernel,
			getNblocks(2*pvView.size, nthreads), nthreads, 0, stream,
			pvView, dt, oscillate );

	// PV may have changed, invalidate all
	pv->haloValid = false;
	pv->redistValid = false;
	pv->cellListStamp++;
}
