#include "hip/hip_runtime.h"
/*
 * bounce.cu
 *
 *  Created on: Jul 20, 2017
 *      Author: alexeedm
 */

#include <core/bounce.h>
#include <core/particle_vector.h>
#include <core/celllist.h>
#include <core/rigid_object_vector.h>
#include <core/rigid_kernels/bounce.h>


void bounceFromRigidEllipsoid(ParticleVector* pv, CellList* cl, RigidObjectVector* rov, const float dt, bool local)
{
	debug("Bouncing %s particles from %s objects\n", pv->name.c_str(), rov->name.c_str());
	auto activeROV = local ? rov->local() : rov->halo();

	bounceEllipsoid<<< activeROV->nObjects, 128 >>> (
			(float4*)pv->local()->coosvels.devPtr(), pv->mass, activeROV->comAndExtents.devPtr(), activeROV->motions.devPtr(),
			activeROV->nObjects, 1.0f / rov->axes,
			cl->cellsStartSize.devPtr(), cl->cellInfo(), dt);
}
