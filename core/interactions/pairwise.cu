#include "hip/hip_runtime.h"
#include "pairwise.h"

#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/logger.h>

#include "pairwise_kernels.h"

#include "pairwise_interactions/dpd.h"
#include "pairwise_interactions/lj.h"
#include "pairwise_interactions/lj_object_aware.h"

/**
 * Convenience macro wrapper
 *
 * Select one of the available kernels for external interaction depending
 * on the number of particles involved, report it and call
 */
#define DISPATCH_EXTERNAL(P1, P2, P3, TPP, INTERACTION_FUNCTION)                \
do{ debug2("Dispatched to "#TPP" thread(s) per particle variant");              \
	SAFE_KERNEL_LAUNCH(                                                         \
			computeExternalInteractions_##TPP##tpp<P1 COMMA P2 COMMA P3>,       \
			getNblocks(TPP*view.size, nth), nth, 0, stream,                     \
			view, cl2->cellInfo(), rc*rc, INTERACTION_FUNCTION); } while (0)

#define CHOOSE_EXTERNAL(P1, P2, P3, INTERACTION_FUNCTION)                                              \
do{  if (view.size < 1000  ) { DISPATCH_EXTERNAL(P1, P2, P3, 27, INTERACTION_FUNCTION); }              \
else if (view.size < 10000 ) { DISPATCH_EXTERNAL(P1, P2, P3, 9,  INTERACTION_FUNCTION); }              \
else if (view.size < 400000) { DISPATCH_EXTERNAL(P1, P2, P3, 3,  INTERACTION_FUNCTION); }              \
else                         { DISPATCH_EXTERNAL(P1, P2, P3, 1,  INTERACTION_FUNCTION); } } while(0)

/**
 * Compute forces between all the pairs of particles that are closer
 * than #rc to each other.
 *
 * Depending on \p type and whether \p pv1 == \p pv2 call
 * computeSelfInteractions() or computeExternalInteractions_1tpp()
 * (or other variants of external interaction kernels).
 *
 * @tparam PariwiseInteraction is a functor that computes the force
 * given a pair of particles. It has to
 * provide two functions:
 * - This function will be called once before interactions computation
 *   and allows the functor to obtain required variables or data
 *   channels from the two ParticleVector and CellList:
 *   \code setup(ParticleVector* pv1, ParticleVector* pv2, CellList* cl1, CellList* cl2, float t) \endcode
 *
 * - This should be a \c \_\_device\_\_ operator that computes
 *   the force. It will be called for each close enough particle pair:
 *   \code float3 operator()(const Particle dst, int dstId, const Particle src, int srcId) const \endcode
 *   Return value of that call is force acting on the first particle,
 *   force acting on the second one is just opposite.
 */
template<class PariwiseInteraction>
void InteractionPair<PariwiseInteraction>::_compute(InteractionType type,
		ParticleVector* pv1, ParticleVector* pv2, CellList* cl1, CellList* cl2, const float t, hipStream_t stream)
{
	interaction.setup(pv1, pv2, cl1, cl2, t);

	if (type == InteractionType::Regular)
	{
		/*  Self interaction */
		if (pv1 == pv2)
		{
			const int np = pv1->local()->size();
			debug("Computing internal forces for %s (%d particles)", pv1->name.c_str(), np);

			const int nth = 128;

			auto cinfo = cl1->cellInfo();
			SAFE_KERNEL_LAUNCH(
					computeSelfInteractions,
					getNblocks(np, nth), nth, 0, stream,
					np, cinfo, rc*rc, interaction );
		}
		else /*  External interaction */
		{
			const int np1 = pv1->local()->size();
			const int np2 = pv2->local()->size();
			debug("Computing external forces for %s - %s (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), np1, np2);

			PVview view(pv1, pv1->local());
			const int nth = 128;
			if (np1 > 0 && np2 > 0)
				CHOOSE_EXTERNAL(true, true, true, interaction );
		}
	}

	/*  Halo interaction */
	if (type == InteractionType::Halo)
	{
		const int np1 = pv1->halo()->size();  // note halo here
		const int np2 = pv2->local()->size();
		debug("Computing halo forces for %s(halo) - %s (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), np1, np2);

		PVview view(pv1, pv1->halo());
		const int nth = 128;
		if (np1 > 0 && np2 > 0)
			if (dynamic_cast<ObjectVector*>(pv1) == nullptr) // don't need forces for pure particle halo
				CHOOSE_EXTERNAL(false, true, false, interaction );
			else
				CHOOSE_EXTERNAL(true,  true, false, interaction );
	}
}

template class InteractionPair<Pairwise_DPD>;
template class InteractionPair<Pairwise_LJ>;
template class InteractionPair<Pairwise_LJObjectAware>;


