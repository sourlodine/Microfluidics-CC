#include "hip/hip_runtime.h"
#include "lj.h"

#include <core/cuda_common.h>
#include <core/celllist.h>
#include <core/pvs/object_vector.h>

#include "pairwise_engine.h"
#include "wrapper_macro.h"

__device__ inline float3 pairwiseLJ(Particle dst, Particle src, const float sigma, const float epsx24_sigma, const float rc2)
{
	const float3 dr = dst.r - src.r;
	const float rij2 = dot(dr, dr);

	if (rij2 > rc2) return make_float3(0.0f);

	const float rs2 = sigma*sigma / rij2;
	const float rs4 = rs2*rs2;
	const float rs8 = rs4*rs4;
	const float rs14 = rs8*rs4*rs2;

	return dr * epsx24_sigma * (2*rs14 - rs8);
}

__device__ inline float3 pairwiseLJ_objectAware(Particle dst, Particle src,
		bool isDstObj, float3 dstCom,
		bool isSrcObj, float3 srcCom,
		const float sigma, const float epsx24_sigma, const float rc2)
{
	const float3 dr = dst.r - src.r;

	const bool dstSide = dot(dr, dst.r-dstCom) < 0.0f;
	const bool srcSide = dot(dr, srcCom-src.r) < 0.0f;

	if (dstSide && (!isSrcObj)) return make_float3(0.0f);
	if ((!isDstObj) && srcSide) return make_float3(0.0f);
	if (dstSide && srcSide)     return make_float3(0.0f);

	return pairwiseLJ(dst, src, sigma, epsx24_sigma, rc2);
}



InteractionLJ::InteractionLJ(std::string name, float rc, float sigma, float epsilon) :
		Interaction(name, rc), sigma(sigma), epsilon(epsilon)
{ }

void InteractionLJ::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl1, CellList* cl2, const float t, hipStream_t stream)
{
	const float epsx24_sigma = 24.0*epsilon/sigma;
	const float rc2 = rc*rc;

	auto ljCore = [=, *this] __device__ ( Particle dst, Particle src ) {
		return pairwiseLJ( dst, src, sigma, epsx24_sigma, rc2);
	};

	WRAP_INTERACTON(ljCore)
}

/**
 * LJ interaction, to prevent overlap of the rigid objects
 */
InteractionLJ_objectAware::InteractionLJ_objectAware(std::string name, float rc, float sigma, float epsilon) :
		Interaction(name, rc), sigma(sigma), epsilon(epsilon)
{ }

void InteractionLJ_objectAware::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl1, CellList* cl2, const float t, hipStream_t stream)
{
	auto ov1 = dynamic_cast<ObjectVector*>(pv1);
	auto ov2 = dynamic_cast<ObjectVector*>(pv2);
	if (ov1 == nullptr && ov2 == nullptr)
		die("Object-aware LJ interaction can only be used with objects");

	const float epsx24_sigma = 24.0*epsilon/sigma;
	const float rc2 = rc*rc;
	const bool self = (pv1 == pv2);

	const auto view1 = create_OVview(ov1, ov1 ? ov1->local() : nullptr);
	const auto view2 = create_OVview(ov2, ov2 ? ov2->local() : nullptr);

	if (view1.comAndExtents == nullptr && view2.comAndExtents == nullptr)
		warn("Neither of the pvs (%s or %s) has required property 'com_extents', trying to move on",
				pv1->name.c_str(), pv2->name.c_str());

	auto ljCore_Obj = [=, *this] __device__ ( Particle dst, Particle src ) {
		const int dstObjId = dst.s21;
		const int srcObjId = src.s21;

		if (dstObjId == srcObjId && self) return make_float3(0.0f);

		float3 dstCom = make_float3(0.0f);
		float3 srcCom = make_float3(0.0f);
		if (view1.comAndExtents != nullptr) dstCom = view1.comAndExtents[dstObjId].com;
		if (view2.comAndExtents != nullptr) srcCom = view2.comAndExtents[srcObjId].com;

		return pairwiseLJ_objectAware( dst, src,
				(view1.comAndExtents != nullptr), dstCom,
				(view2.comAndExtents != nullptr), srcCom,
				sigma, epsx24_sigma, rc2);
	};

	WRAP_INTERACTON(ljCore_Obj)
}
