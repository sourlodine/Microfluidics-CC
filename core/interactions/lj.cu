#include "hip/hip_runtime.h"
#include "lj.h"

#include <core/cuda_common.h>
#include <core/celllist.h>
#include <core/pvs/object_vector.h>

#include "pairwise_engine.h"
#include "wrapper_macro.h"

__device__ inline float3 pairwiseLJ(Particle dst, Particle src, const float sigma, const float epsx24_sigma, const float rc2)
{
	const float3 dr = dst.r - src.r;
	const float rij2 = dot(dr, dr);

	if (rij2 > rc2) return make_float3(0.0f);

	const float rs2 = sigma*sigma / rij2;
	const float rs4 = rs2*rs2;
	const float rs8 = rs4*rs4;
	const float rs14 = rs8*rs4*rs2;

	return dr * epsx24_sigma * (2*rs14 - rs8);
}

__device__ inline float3 pairwiseLJ_objectAware(Particle dst, Particle src,
		bool isDstObj, float3 dstCom,
		bool isSrcObj, float3 srcCom,
		const float sigma, const float epsx24_sigma, const float rc2)
{
	const float3 dr = dst.r - src.r;

	const bool dstSide = dot(dr, dst.r-dstCom) < 0.0f;
	const bool srcSide = dot(dr, srcCom-src.r) < 0.0f;

	if (dstSide && (!isSrcObj)) return make_float3(0.0f);
	if ((!isDstObj) && srcSide) return make_float3(0.0f);
	if (dstSide && srcSide)     return make_float3(0.0f);

	return pairwiseLJ(dst, src, sigma, epsx24_sigma, rc2);
}



InteractionLJ::InteractionLJ(std::string name, float rc, float sigma, float epsilon) :
		name(name), rc(rc), sigma(sigma), epsilon(epsilon)
{ }

void InteractionLJ::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream)
{
	const float epsx24_sigma = 24.0*epsilon/sigma;
	const float rc2 = rc*rc;
	const bool self = (pv1 == pv2);

	auto ljCore = [=, *this] __device__ ( Particle dst, Particle src ) {
		return pairwiseLJ( dst, src, sigma, epsx24_sigma, rc2);
	};

	WRAP_INTERACTON(ljCore)
}

/**
 * LJ interaction, to prevent overlap of the rigid objects
 */
InteractionLJ_objectAware::InteractionLJ_objectAware(std::string name, float rc, float sigma, float epsilon) :
				name(name), rc(rc), sigma(sigma), epsilon(epsilon)
{ }

void InteractionLJ_objectAware::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream)
{
	auto ov1 = dynamic_cast<ObjectVector*>(pv1);
	auto ov2 = dynamic_cast<ObjectVector*>(pv2);
	if (ov1 == nullptr && ov2 == nullptr)
		die("Object-aware LJ interaction can only be used with objects");

	const float epsx24_sigma = 24.0*epsilon/sigma;
	const float rc2 = rc*rc;
	const bool self = (pv1 == pv2);

	const LocalObjectVector::COMandExtent* dstComExt = (ov1 != nullptr) ? ov1->local()->comAndExtents.devPtr() : nullptr;
	const LocalObjectVector::COMandExtent* srcComExt = (ov2 != nullptr) ? ov2->local()->comAndExtents.devPtr() : nullptr;

	auto ljCore_Obj = [=, *this] __device__ ( Particle dst, Particle src ) {
		const int dstObjId = dst.s21;
		const int srcObjId = src.s21;

		if (dstObjId == srcObjId && self) return make_float3(0.0f);

		float3 dstCom = make_float3(0.0f);
		float3 srcCom = make_float3(0.0f);
		if (dstComExt != nullptr) dstCom = dstComExt[dstObjId].com;
		if (srcComExt != nullptr) srcCom = srcComExt[srcObjId].com;

		return pairwiseLJ_objectAware( dst, src, (dstComExt != nullptr), dstCom, (srcComExt != nullptr), srcCom, sigma, epsx24_sigma, rc2);
	};

	WRAP_INTERACTON(ljCore_Obj)
}
