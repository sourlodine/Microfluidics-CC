#include "particle_exchanger.h"

#include <core/utils/kernel_launch.h>
#include <core/logger.h>
#include <core/utils/cuda_common.h>

#include <algorithm>

ExchangeHelper::ExchangeHelper(std::string name, const int datumSize) :
	name(name), datumSize(datumSize)
{
	recvSizes.  resize_anew(nBuffers);
	recvOffsets.resize_anew(nBuffers+1);

	sendSizes.  resize_anew(nBuffers);
	sendOffsets.resize_anew(nBuffers+1);
}

void ExchangeHelper::makeOffsets(const PinnedBuffer<int>& sz, PinnedBuffer<int>& of)
{
	int n = sz.size();
	if (n == 0) return;

	of[0] = 0;
	for (int i=0; i < n; i++)
		of[i+1] = of[i] + sz[i];
}

ParticleExchanger::ParticleExchanger(MPI_Comm& comm) :
		nActiveNeighbours(26)
{
	MPI_Check( MPI_Comm_dup(comm, &haloComm) );

	int dims[3], periods[3], coords[3];
	MPI_Check( MPI_Cart_get (haloComm, 3, dims, periods, coords) );
	MPI_Check( MPI_Comm_rank(haloComm, &myrank));

	for(int i = 0; i < 27; ++i)
	{
		int d[3] = { i%3 - 1, (i/3) % 3 - 1, i/9 - 1 };

		int coordsNeigh[3];
		for(int c = 0; c < 3; ++c)
			coordsNeigh[c] = coords[c] + d[c];

		MPI_Check( MPI_Cart_rank(haloComm, coordsNeigh, dir2rank + i) );

		dir2sendTag[i] = i;

		int cx = -( i%3 - 1 ) + 1;
		int cy = -( (i/3)%3 - 1 ) + 1;
		int cz = -( i/9 - 1 ) + 1;
		dir2recvTag[i] = (cz*3 + cy)*3 + cx;
	}
}

void ParticleExchanger::init(hipStream_t stream)
{
	// Derived class determines what to send
	for (int i=0; i<helpers.size(); i++)
		if (needExchange(i)) prepareData(i, stream);
}

void ParticleExchanger::finalize(hipStream_t stream)
{
	// Internal functions to exchange data
	for (int i=0; i<helpers.size(); i++)
		if (needExchange(i)) send(helpers[i], stream);

	for (int i=0; i<helpers.size(); i++)
		if (needExchange(i)) recv(helpers[i], stream);


	// Derived class unpack implementation
	for (int i=0; i<helpers.size(); i++)
		if (needExchange(i)) combineAndUploadData(i, stream);
}


int ParticleExchanger::tagByName(std::string name)
{
	// TODO: better tagging policy (unique id?)
	static std::hash<std::string> nameHash;
	return (int)( nameHash(name) % (32767 / 27) );
}


/**
 * helper->recvBuf will contain all the data, ON DEVICE already
 * will set and sync recvSizes and recvOffsets as well
 */
void ParticleExchanger::recv(ExchangeHelper* helper, hipStream_t stream)
{
	std::string pvName = helper->name;

	auto nBuffers = helper->nBuffers;
	auto rSizes   = helper->recvSizes.  hostPtr();
	auto rOffsets = helper->recvOffsets.hostPtr();

	// Receive sizes
	helper->requests.clear();
	helper->recvSizes.clearHost();

	for (int i=0; i < nBuffers; i++)
		if (i != 13 && dir2rank[i] >= 0)
		{
			MPI_Request req;
			const int tag = 27 * tagByName(pvName) + dir2recvTag[i];

			MPI_Check( MPI_Irecv(rSizes + i, 1, MPI_INT, dir2rank[i], tag, haloComm, &req) );
			helper->requests.push_back(req);
		}

	const int nMessages = helper->requests.size(); // 26 for now
	MPI_Check( MPI_Waitall(nMessages, helper->requests.data(), MPI_STATUSES_IGNORE) );

	// Prepare offsets and resize
	helper->makeRecvOffsets();
	int totalRecvd = rOffsets[nBuffers];
	helper->resizeRecvBuf();

	// Now do the actual data recv
	helper->requests.clear();
	for (int i=0; i < nBuffers; i++)
		if (i != 13 && dir2rank[i] >= 0)
		{
			MPI_Request req;
			const int tag = nBuffers * tagByName(pvName) + dir2recvTag[i];

			debug3("Receiving %s entities from rank %d, %d entities (buffer %d)",
					pvName.c_str(), dir2rank[i], rSizes[i], i);

			MPI_Check( MPI_Irecv(
					helper->recvBuf.hostPtr() + rOffsets[i]*helper->datumSize,
					rSizes[i]*helper->datumSize,
					MPI_BYTE, dir2rank[i], tag, haloComm, &req) );

			helper->requests.push_back(req);
		}

	// Start uploading sizes and offsets
	helper->recvSizes.  uploadToDevice(stream);
	helper->recvOffsets.uploadToDevice(stream);

	// Wait for completion
	MPI_Check( MPI_Waitall(nMessages, helper->requests.data(), MPI_STATUSES_IGNORE) );

	// And finally upload received
	helper->recvBuf.uploadToDevice(stream);

	debug("Received total %d %s entities", totalRecvd, pvName.c_str());
}

/**
 * Expects helper->sendSizes and helper->sendOffsets to be ON HOST
 * helper->sendBuf data is ON DEVICE
 */
void ParticleExchanger::send(ExchangeHelper* helper, hipStream_t stream)
{
	std::string pvName = helper->name;

	auto nBuffers = helper->nBuffers;
	auto sSizes   = helper->sendSizes.  hostPtr();
	auto sOffsets = helper->sendOffsets.hostPtr();

	helper->sendBuf.downloadFromDevice(stream);

	MPI_Request req;
	int totSent = 0;
	for (int i=0; i < nBuffers; i++)
		if (i != 13 && dir2rank[i] >= 0)
		{
			debug3("Sending %s entities to rank %d in dircode %d [%2d %2d %2d], %d entities",
					pvName.c_str(), dir2rank[i], i, i%3 - 1, (i/3)%3 - 1, i/9 - 1, sSizes[i]);

			const int tag = nBuffers * tagByName(pvName) + dir2sendTag[i];

			// Send sizes
			MPI_Check( MPI_Isend(sSizes+i, 1, MPI_INT, dir2rank[i], tag, haloComm, &req) );
			MPI_Check( MPI_Request_free(&req) );

			// Send actual data
			MPI_Check( MPI_Isend(
					helper->sendBuf.hostPtr() + sOffsets[i]*helper->datumSize,
					sSizes[i] * helper->datumSize,
					MPI_BYTE, dir2rank[i], tag, haloComm, &req) );
			MPI_Check( MPI_Request_free(&req) );

			totSent += sSizes[i];
		}
	debug("Sent total %d %s entities", totSent, pvName.c_str());
}


