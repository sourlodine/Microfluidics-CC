#include "hip/hip_runtime.h"
#include "object_forces_reverse_exchanger.h"

#include "object_halo_exchanger.h"

#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/logger.h>
#include <core/cuda_common.h>


// TODO: change id scheme
__global__ void addHaloForces(const float4* haloForces, const float4* halo, float4* forces, int objSize, int n)
{
	const int srcId = blockIdx.x*blockDim.x + threadIdx.x;
	if (srcId >= n) return;

	const Particle p(halo[2*srcId], halo[2*srcId+1]);
	const int dstId = p.s22 /* objId */ * objSize + p.s21 /* pid in object */;

	const Float3_int extraFrc = readNoCache(haloForces + srcId);
	Float3_int frc0 = forces[dstId];
	frc0.v += extraFrc.v;

	forces[dstId] = frc0.toFloat4();
}


void ObjectForcesReverseExchanger::attach(ObjectVector* ov)
{
	objects.push_back(ov);
	ExchangeHelper* helper = new ExchangeHelper(ov->name, ov->local()->objSize*sizeof(Force));
	helpers.push_back(helper);
}


void ObjectForcesReverseExchanger::prepareData(int id, hipStream_t stream)
{
	auto ov = objects[id];
	auto helper = helpers[id];
	auto offsets = entangledHaloExchanger->helpers[id]->recvOffsets;

	debug2("Preparing %s forces to sending back", ov->name.c_str());

	for (int i=0; i<27; i++)
		helper->sendBufSizes[i] = offsets[i+1] - offsets[i];
	helper->resizeSendBufs(stream);

	for (int i=0; i<27; i++)
	{
		if (helper->sendBufSizes[i] > 0)
			CUDA_Check( hipMemcpyAsync( helper->sendBufs[i].hostPtr(),
										 ov->halo()->forces.devPtr() + offsets[i]*ov->halo()->objSize,
										 helper->sendBufSizes[i]*sizeof(Force)*ov->halo()->objSize,
										 hipMemcpyHostToDevice, stream ) );
	}
}

void ObjectForcesReverseExchanger::combineAndUploadData(int id, hipStream_t stream)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	for (int i=0; i < helper->recvOffsets.size() - 1; i++)
	{
		const int msize = helper->recvOffsets[i+1] - helper->recvOffsets[i];

		debug3("Updating forces for %d %s objects", msize, ov->name.c_str());

		if (msize > 0)
			CUDA_Check( hipMemcpyAsync(ov->halo()->forces.devPtr() + helper->recvOffsets[i]*ov->halo()->objSize,
										helper->recvBufs[compactedDirs[i]].hostPtr(),
										msize*sizeof(Force)*ov->halo()->objSize,
										hipMemcpyHostToDevice, stream) );
	}

	const int np = helper->recvOffsets[27];
	if (np > 0)
		addHaloForces<<< (np+127)/128, 128, 0, stream >>> (
				(float4*)ov->halo()->forces.devPtr(),    /* add to */
				(float4*)ov->halo()->coosvels.devPtr(),  /* destination id here */
				(float4*)ov->local()->forces.devPtr(),   /* source */
				ov->objSize, np );
}





