#include "hip/hip_runtime.h"
#include "object_forces_reverse_exchanger.h"

#include "object_halo_exchanger.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/rigid_object_vector.h>
#include <core/logger.h>
#include <core/utils/cuda_common.h>


__device__ __forceinline__ void atomicAddNonZero(float4* dest, float3 v)
{
	const float tol = 1e-7;

	float* fdest = (float*)dest;
	if (fabs(v.x) > tol) atomicAdd(fdest,     v.x);
	if (fabs(v.y) > tol) atomicAdd(fdest + 1, v.y);
	if (fabs(v.z) > tol) atomicAdd(fdest + 2, v.z);
}

__global__ void addHaloForces(
		const float4* recvForces, const int* origins,
		float4* forces, int objSize, int packedObjSize)
{
	const int objId = blockIdx.x;

	for (int pid = threadIdx.x; pid < objSize; pid += blockDim.x)
	{
		const int dstId = origins[objId*objSize + pid];
		Float3_int extraFrc( recvForces[objId*packedObjSize + pid] );
	
		atomicAddNonZero(forces + dstId, extraFrc.v);
	}
}

__global__ void addRigidForces(
		const float4* recvForces, const int* origins,
		ROVview view, int packedObjSize)
{
	const int objId = blockIdx.x;

	const int dstObjId = origins[objId*view.objSize] / view.objSize;

	if (threadIdx.x < 2)
	{
		float4 v = recvForces[ objId*packedObjSize + view.objSize + threadIdx.x ];

		if (threadIdx.x == 0)
			atomicAdd(&view.motions[dstObjId].force,  make_float3(v));

		if (threadIdx.x == 1)
			atomicAdd(&view.motions[dstObjId].torque, make_float3(v));
	}
}

__global__ void packRigidForces(ROVview view, float4* output, int packedObjSize)
{
	const int objId = blockIdx.x;

	for (int pid = threadIdx.x; pid < view.objSize; pid += blockDim.x)
		output[objId*view.objSize + pid] = view.forces[objId*view.objSize + pid];

	if (threadIdx.x == 0)
		output[objId*packedObjSize + view.objSize + 0] = make_float4(view.motions[objId].force,  0);

	if (threadIdx.x == 1)
		output[objId*packedObjSize + view.objSize + 1] = make_float4(view.motions[objId].torque, 0);
}


//===============================================================================================
// Member functions
//===============================================================================================

bool ObjectForcesReverseExchanger::needExchange(int id)
{
	return true;
}

void ObjectForcesReverseExchanger::attach(ObjectVector* ov)
{
	objects.push_back(ov);

	int psize = ov->objSize;
	if (dynamic_cast<RigidObjectVector*>(ov) != 0)
		psize += 2;

	ExchangeHelper* helper = new ExchangeHelper(ov->name, psize*sizeof(float4));
	helpers.push_back(helper);
}


void ObjectForcesReverseExchanger::prepareData(int id, hipStream_t stream)
{
	auto ov = objects[id];
	auto helper = helpers[id];
	auto& offsets = entangledHaloExchanger->getRecvOffsets(id);

	debug2("Preparing %s forces to sending back", ov->name.c_str());

	for (int i=0; i < helper->nBuffers; i++)
		helper->sendSizes[i] = offsets[i+1] - offsets[i];

	helper->makeSendOffsets();
	helper->resizeSendBuf();

	auto rov = dynamic_cast<RigidObjectVector*>(ov);
	if (rov != nullptr)
	{
		int psize = rov->objSize + 2;
		ROVview view(rov, rov->halo());

		const int nthreads = 128;
		SAFE_KERNEL_LAUNCH(
				packRigidForces,
				view.nObjects, nthreads, 0, stream,
				view, (float4*)helper->sendBuf.devPtr(), psize);

	}
	else
	{
		CUDA_Check( hipMemcpyAsync( helper->sendBuf.devPtr(),
									 ov->halo()->forces.devPtr(),
									 helper->sendBuf.size(), hipMemcpyDeviceToDevice, stream ) );
	}

	debug2("Will send back forces for %d objects", offsets[helper->nBuffers]);
}

void ObjectForcesReverseExchanger::combineAndUploadData(int id, hipStream_t stream)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	int totalRecvd = helper->recvOffsets[helper->nBuffers];
	auto& origins = entangledHaloExchanger->getOrigins(id);

	debug("Updating forces for %d %s objects", totalRecvd, ov->name.c_str());

	int psize = ov->objSize;
	auto rov = dynamic_cast<RigidObjectVector*>(ov);
	if (rov != nullptr) psize += 2;

	const int nthreads = 128;
	SAFE_KERNEL_LAUNCH(
			addHaloForces,
			totalRecvd, nthreads, 0, stream,
			(const float4*)helper->recvBuf.devPtr(),     /* source */
			(const int*)origins.devPtr(),                /* destination ids here */
			(float4*)ov->local()->forces.devPtr(),       /* add to */
			ov->objSize, psize );

	if (rov != nullptr)
	{
		ROVview view(rov, rov->local());
		SAFE_KERNEL_LAUNCH(
				addRigidForces,
				totalRecvd, nthreads, 0, stream,
				(const float4*)helper->recvBuf.devPtr(),     /* source */
				(const int*)origins.devPtr(),                /* destination ids here */
				view, psize );                               /* add to, packed size */
	}
}





