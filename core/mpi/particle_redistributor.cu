#include "hip/hip_runtime.h"
#include "particle_redistributor.h"

#include <core/utils/kernel_launch.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/utils/cuda_common.h>

#include <core/mpi/valid_cell.h>

template<bool QUERY=false>
__global__ void getExitingParticles(const CellListInfo cinfo, BufferOffsetsSizesWrap dataWrap)
{
	const int gid = blockIdx.x*blockDim.x + threadIdx.x;
	int cid;
	int cx, cy, cz;
	const int3 ncells = cinfo.ncells;

	bool valid = isValidCell(cid, cx, cy, cz, gid, blockIdx.y, cinfo);

	if (!valid) return;

	// The following is called for every outer cell and exactly once for each
	//
	// Now for each cell we check its every particle if it needs to move

	int pstart = cinfo.cellStarts[cid];
	int pend   = cinfo.cellStarts[cid+1];

#pragma unroll 2
	for (int i = 0; i < pend-pstart; i++)
	{
		const int srcId = pstart + i;
		Particle p(cinfo.particles, srcId);

		int3 code = cinfo.getCellIdAlongAxes<false>(make_float3(p.r));

		if (code.x < 0) code.x = 0;
		else if (code.x >= ncells.x) code.x = 2;
		else code.x = 1;

		if (code.y < 0) code.y = 0;
		else if (code.y >= ncells.y) code.y = 2;
		else code.y = 1;

		if (code.z < 0) code.z = 0;
		else if (code.z >= ncells.z) code.z = 2;
		else code.z = 1;

		if (code.x*code.y*code.z != 1) // this means that the particle has to leave
		{
			const int bufId = (code.z*3 + code.y)*3 + code.x;
			const float3 shift{ cinfo.localDomainSize.x*(code.x-1),
								cinfo.localDomainSize.y*(code.y-1),
								cinfo.localDomainSize.z*(code.z-1) };
			p.r -= shift;

			int myid = atomicAdd(dataWrap.sizes + bufId, 1);

			if (QUERY) continue;

			const int dstInd = myid;
			float4* addr = (float4*) ( (Particle*)dataWrap.buffer + dataWrap.offsets[bufId] );
			p.write2Float4(addr, dstInd);

			// mark the particle as exited to assist cell-list building
			cinfo.particles[2*srcId] = Float3_int(make_float3(-1e5), p.i1).toFloat4();
		}
	}
}

//===============================================================================================
// Member functions
//===============================================================================================

bool ParticleRedistributor::needExchange(int id)
{
	return !particles[id]->redistValid;
}

void ParticleRedistributor::attach(ParticleVector* pv, CellList* cl)
{
	particles.push_back(pv);
	cellLists.push_back(cl);

	if (dynamic_cast<PrimaryCellList*>(cl) == nullptr)
		die("Redistributor (for %s) should be used with the primary cell-lists only!", pv->name.c_str());

	auto helper = new ExchangeHelper(pv->name, sizeof(Particle));
	helpers.push_back(helper);

	info("Particle redistributor takes pv %s, base tag %d", pv->name.c_str(), tagByName(pv->name));
}

void ParticleRedistributor::prepareData(int id, hipStream_t stream)
{
	auto pv = particles[id];
	auto cl = cellLists[id];
	auto helper = helpers[id];

	debug2("Preparing %s leaving particles on the device", pv->name.c_str());

	helper->sendSizes.clear(stream);
	if (pv->local()->size() > 0)
	{
		const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
		const int nthreads = 64;
		const dim3 nblocks = dim3(getNblocks(maxdim*maxdim, nthreads), 6, 1);

		SAFE_KERNEL_LAUNCH(
				getExitingParticles<true>,
				nblocks, nthreads, 0, stream,
				cl->cellInfo(), helper->wrapSendData() );

		helper->makeSendOffsets_Dev2Dev(stream);
		helper->resizeSendBuf();

		// Sizes will still remain on host, no need to download again
		helper->sendSizes.clearDevice(stream);
		SAFE_KERNEL_LAUNCH(
				getExitingParticles<false>,
				nblocks, nthreads, 0, stream,
				cl->cellInfo(), helper->wrapSendData() );
	}
}

void ParticleRedistributor::combineAndUploadData(int id, hipStream_t stream)
{
	auto pv = particles[id];
	auto helper = helpers[id];

	int oldsize = pv->local()->size();
	int totalRecvd = helper->recvOffsets[helper->nBuffers];
	pv->local()->resize(oldsize + totalRecvd,  stream);

	if (totalRecvd > 0)
		CUDA_Check( hipMemcpyAsync(
				pv->local()->coosvels.devPtr() + oldsize,
				helper->recvBuf.devPtr(),
				helper->recvBuf.size(), hipMemcpyDeviceToDevice, stream) );

	pv->redistValid = true;
}
