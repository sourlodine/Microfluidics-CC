#include "hip/hip_runtime.h"
#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/cuda_common.h>

#include <core/mpi/object_redistributor.h>
#include <core/mpi/valid_cell.h>

#include <vector>
#include <algorithm>
#include <limits>



__device__ void packExtraData(int objId, int32_t** extraData, int nPtrsPerObj, const int* dataSizes, int32_t* destanation)
{
	int baseId = 0;

	for (int ptrId = 0; ptrId < nPtrsPerObj; ptrId++)
		{
			// dataSizes are in bytes
			const int size = dataSizes[ptrId] / 4;
			for (int i = threadIdx.x; i < size; i += blockDim.x)
				destanation[baseId+i] = extraData[ptrId][objId*size + i];

			baseId += dataSizes[ptrId];
		}
}

__device__ void unpackExtraData(int objId, int32_t** extraData, int nPtrsPerObj, const int* dataSizes, const int32_t* source)
{
	int baseId = 0;

	for (int ptrId = 0; ptrId < nPtrsPerObj; ptrId++)
	{
		// dataSizes are in bytes
		const int size = dataSizes[ptrId] / 4;
		for (int i = threadIdx.x; i < size; i += blockDim.x)
			extraData[ptrId][objId*size + i] = source[baseId+i];

		baseId += dataSizes[ptrId];
	}
}


__global__ void getExitingObjects(const float4* __restrict__ coosvels, const LocalObjectVector::COMandExtent* props, const int nObj, const int objSize,
		const float3 localDomainSize,
		const int64_t dests[27], int bufSizes[27], /*int* haloParticleIds,*/
		const int packedObjSize_byte, int32_t** extraData, int nPtrsPerObj, const int* dataSizes)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	if (objId >= nObj) return;

	// Find to which buffer this object should go
	auto prop = props[objId];
	int cx = 1, cy = 1, cz = 1;

	if (prop.com.x  < -0.5*localDomainSize.x) cx = 0;
	if (prop.com.y  < -0.5*localDomainSize.y) cy = 0;
	if (prop.com.z  < -0.5*localDomainSize.z) cz = 0;

	if (prop.com.x >=  0.5*localDomainSize.x) cx = 2;
	if (prop.com.y >=  0.5*localDomainSize.y) cy = 2;
	if (prop.com.z >=  0.5*localDomainSize.z) cz = 2;

//	if (tid == 0) printf("Obj %d : [%f %f %f] -- [%f %f %f]\n", objId,
//			prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z);


	const int bufId = (cz*3 + cy)*3 + cx;

	__shared__ int shDstObjId;

	const float3 shift{ localDomainSize.x*(cx-1),
						localDomainSize.y*(cy-1),
						localDomainSize.z*(cz-1) };

	__syncthreads();
	if (tid == 0)
		shDstObjId = atomicAdd(bufSizes + bufId, 1);
	__syncthreads();

//		if (tid == 0)
//			if (objId == 5)
//				printf("obj  %d  to halo  %d  [%f %f %f] - [%f %f %f]  %d %d %d\n", objId, bufId,
//						prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z, cx, cy, cz);

	float4* dstAddr = (float4*) (dests[bufId]) + packedObjSize_byte/sizeof(float4) * shDstObjId;

	for (int pid = tid/2; pid < objSize; pid += blockDim.x/2)
	{
		const int srcId = objId * objSize + pid;
		Float3_int data(coosvels[2*srcId + sh]);

		if (sh == 0) data.v -= shift;

		dstAddr[2*pid + sh] = data.toFloat4();
	}

	// Add extra data at the end of the object
	dstAddr += objSize*2;
	packExtraData(objId, extraData, nPtrsPerObj, dataSizes, (int32_t*)dstAddr);
}


__global__ void unpackObject(const float4* from, float4* to, const int objSize, const int packedObjSize_byte, const int nObj,
		int32_t** extraData, int nPtrsPerObj, const int* dataSizes)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	for (int pid = tid/2; pid < objSize; pid += blockDim.x/2)
	{
		const int srcId = objId * packedObjSize_byte/sizeof(float4) + pid*2;
		float4 data = from[srcId + sh];

		to[2*(objId*objSize + pid) + sh] = data;
	}

	unpackExtraData(objId, extraData, nPtrsPerObj, dataSizes, (int32_t*)( ((char*)from) + objId * packedObjSize_byte + objSize*sizeof(Particle) ));
}





void ObjectRedistributor::attach(ObjectVector* ov, float rc)
{
	objects.push_back(ov);

	const float objPerCell = 0.1f;

	const int maxdim = std::max({ov->localDomainSize.x, ov->localDomainSize.y, ov->localDomainSize.z});

	const int sizes[3] = { (int)(4*objPerCell * maxdim*maxdim + 10),
						   (int)(4*objPerCell * maxdim + 10),
						   (int)(4*objPerCell + 10) };


	ExchangeHelper* helper = new ExchangeHelper(ov->name, ov->local()->packedObjSize_bytes, sizes);

	//  Central buffer will be used to move the data around
	// while removing exiting objects
	helper->sendBufs[13].pushStream(stream);
	helper->sendBufs[13].resize( ov->local()->packedObjSize_bytes * (ov->local()->nObjects + 5) * 1.5 );
	helper->sendAddrs[13] = sendBufs[i].devPtr();
	helper->sendAddrs.uploadToDevice();

	ov->halo()->pushStream(helper->stream);
	helpers.push_back(helper);
}


void ObjectRedistributor::prepareData(int id)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	debug2("Preparing %s halo on the device", ov->name.c_str());

	helper->bufSizes.pushStream(defStream);
	helper->bufSizes.clearDevice();

	if ( helper->sendBufs[13].size() < ov->local()->packedObjSize_bytes * ov->local()->nObjects )
	{
		helper->sendBufs[13].pushStream(stream);
		helper->sendBufs[13].resize( ov->local()->packedObjSize_bytes * ov->local()->nObjects * 1.5 );
		helper->sendAddrs[13] = sendBufs[i].devPtr();
		helper->sendAddrs.uploadToDevice();
	}

	const int nthreads = 128;
	if (ov->local()->nObjects > 0)
	{
		int       nPtrs  = ov->local()->extraDataPtrs.size();
		int totSize_byte = ov->local()->packedObjSize_bytes;

		getExitingObjects <<< ov->local()->nObjects, nthreads, 0, defStream >>> (
				(float4*)ov->local()->coosvels.devPtr(), ov->local()->comAndExtents.devPtr(),
				ov->local()->nObjects, ov->local()->objSize, ov->localDomainSize,
				(int64_t*)helper->sendAddrs.devPtr(), helper->bufSizes.devPtr(),
				totSize_byte, ov->local()->extraDataPtrs.devPtr(), nPtrs, ov->local()->extraDataSizes.devPtr());

		// Unpack the central buffer into the object vector itself
		helper->bufSizes.downloadFromDevice();
		int nObjs = helper->bufSizes[13];
		unpackObject<<< nObjs, nthreads, 0, defStream >>>
				(((float4*)helper->sendBufs[13].devPtr(), (float4*)ov->local()->coosvels.devPtr(), ov->local()->objSize, ov->local()->packedObjSize_bytes, nObjs,
				 ov->local()->extraDataPtrs.devPtr(), nPtrs, ov->local()->extraDataSizes.devPtr())
	}

	helper->bufSizes.popStream();
}

void ObjectRedistributor::combineAndUploadData(int id)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	ov->halo()->resize(helper->recvOffsets[27] * ov->halo()->objSize, resizeAnew);
	ov->halo()->resize(helper->recvOffsets[27] * ov->halo()->objSize, resizeAnew);

	const int nthreads = 128;
	for (int i=0; i < 27; i++)
	{
		const int nObjs = helper->recvOffsets[i+1] - helper->recvOffsets[i];
		if (nObjs > 0)
		{
			int        nPtrs = ov->local()->extraDataPtrs.size();
			int totSize_byte = ov->local()->packedObjSize_bytes;

			unpackObject<<< nObjs, nthreads, 0, defStream >>>
					((float4*)helper->recvBufs[i].devPtr(), (float4*)(ov->halo()->coosvels.devPtr() + helper->recvOffsets[i]*nObjs), ov->local()->objSize, totSize_byte, nObjs,
					 ov->halo()->extraDataPtrs.devPtr(), nPtrs, ov->halo()->extraDataSizes.devPtr());
		}
	}
}



