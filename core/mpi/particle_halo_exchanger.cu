#include "hip/hip_runtime.h"
#include "particle_halo_exchanger.h"

#include <core/pvs/particle_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/cuda_common.h>

#include "valid_cell.h"

template<bool QUERY=false>
__global__ void getHalos(const float4* __restrict__ coosvels, const CellListInfo cinfo, const uint* __restrict__ cellsStartSize,
		const int64_t dests[27], int counts[27])
{
	const int gid = blockIdx.x*blockDim.x + threadIdx.x;
	const int tid = threadIdx.x;
	int cid;
	int cx, cy, cz;

	bool valid = isValidCell(cid, cx, cy, cz, gid, blockIdx.y, cinfo);

	if (__all(!valid) && tid > 27) return;

	int2 start_size = valid ? cinfo.decodeStartSize(cellsStartSize[cid]) : make_int2(0, 0);

	// Use shared memory to decrease number of global atomics
	// We're sending to max 7 halos (corner)
	short validHalos[7];
	int haloOffset[7] = {};

	int current = 0;

	// Total number of elements written to halos by this block
	__shared__ int blockSum[27];
	if (tid < 27) blockSum[tid] = 0;

	__syncthreads();

	for (int ix = min(cx, 1); ix <= max(cx, 1); ix++)
		for (int iy = min(cy, 1); iy <= max(cy, 1); iy++)
			for (int iz = min(cz, 1); iz <= max(cz, 1); iz++)
			{
				if (ix == 1 && iy == 1 && iz == 1) continue;

				const int bufId = (iz*3 + iy)*3 + ix;
				validHalos[current] = bufId;
				haloOffset[current] = atomicAdd(blockSum + bufId, start_size.y);
				current++;
			}

	__syncthreads();

	if (tid < 27 && blockSum[tid] > 0)
		blockSum[tid] = atomicAdd(counts + tid, blockSum[tid]);

	if (QUERY) return;

	__syncthreads();

#pragma unroll 3
	for (int i=0; i<current; i++)
	{
		const int bufId = validHalos[i];
		const int myid  = blockSum[bufId] + haloOffset[i];

		const int ix = bufId % 3;
		const int iy = (bufId / 3) % 3;
		const int iz = bufId / 9;
		const float3 shift{ cinfo.localDomainSize.x*(ix-1),
							cinfo.localDomainSize.y*(iy-1),
							cinfo.localDomainSize.z*(iz-1) };

#pragma unroll 2
		for (int i = 0; i < start_size.y; i++)
		{
			const int dstInd = myid         + i;
			const int srcInd = start_size.x + i;

			Particle p(coosvels, srcInd);
			p.r -= shift;

			float4* addr = (float4*)dests[bufId];
			addr[2*dstInd + 0] = p.r2Float4();
			addr[2*dstInd + 1] = p.u2Float4();
		}
	}
}

void ParticleHaloExchanger::attach(ParticleVector* pv, CellList* cl)
{
	particles.push_back(pv);
	cellLists.push_back(cl);

	auto helper = new ExchangeHelper(pv->name, sizeof(Particle));
	helpers.push_back(helper);

	info("Particle halo exchanger takes pv %s, base tag %d", pv->name.c_str(), tagByName(pv->name));
}

void ParticleHaloExchanger::combineAndUploadData(int id, hipStream_t stream)
{
	auto pv = particles[id];
	auto helper = helpers[id];

	pv->halo()->resize(helper->recvOffsets[27], stream, ResizeKind::resizeAnew);

	for (int i=0; i < 27; i++)
	{
		const int msize = helper->recvOffsets[i+1] - helper->recvOffsets[i];
		if (msize > 0)
			CUDA_Check( hipMemcpyAsync(pv->halo()->coosvels.devPtr() + helper->recvOffsets[i], helper->recvBufs[i].hostPtr(),
					msize*sizeof(Particle), hipMemcpyHostToDevice, stream) );
	}
}

void ParticleHaloExchanger::prepareData(int id, hipStream_t stream)
{
	auto pv = particles[id];
	auto cl = cellLists[id];
	auto helper = helpers[id];

	debug2("Preparing %s halo on the device", pv->name.c_str());


	const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
	const int nthreads = 32;
	if (pv->local()->size() > 0)
	{
		helper->sendBufSizes.clearDevice(stream);
		getHalos<true>  <<< dim3((maxdim*maxdim + nthreads - 1) / nthreads, 6, 1),  dim3(nthreads, 1, 1), 0, stream >>> (
				(float4*)pv->local()->coosvels.devPtr(), cl->cellInfo(), cl->cellsStartSize.devPtr(),
				(int64_t*)helper->sendAddrs.devPtr(), helper->sendBufSizes.devPtr() );

		helper->sendBufSizes.downloadFromDevice(stream);
		helper->resizeSendBufs(stream);

		helper->sendBufSizes.clearDevice(stream);
		getHalos<false> <<< dim3((maxdim*maxdim + nthreads - 1) / nthreads, 6, 1),  dim3(nthreads, 1, 1), 0, stream >>> (
				(float4*)pv->local()->coosvels.devPtr(), cl->cellInfo(), cl->cellsStartSize.devPtr(),
				(int64_t*)helper->sendAddrs.devPtr(), helper->sendBufSizes.devPtr() );
	}

	debug2("%s halo prepared", pv->name.c_str());
}




