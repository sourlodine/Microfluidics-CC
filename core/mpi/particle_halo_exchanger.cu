#include "hip/hip_runtime.h"
#include "particle_halo_exchanger.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/particle_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/utils/cuda_common.h>

#include "valid_cell.h"

template<bool QUERY=false>
__global__ void getHalos(const CellListInfo cinfo, char** dests, int* counts)
{
	const int gid = blockIdx.x*blockDim.x + threadIdx.x;
	const int tid = threadIdx.x;
	int cid;
	int cx, cy, cz;

	bool valid = isValidCell(cid, cx, cy, cz, gid, blockIdx.y, cinfo);

	int pstart = valid ? cinfo.cellStarts[cid]   : 0;
	int pend   = valid ? cinfo.cellStarts[cid+1] : 0;

	// Use shared memory to decrease number of global atomics
	// We're sending to max 7 halos (corner)
	short validHalos[7];
	int haloOffset[7] = {};

	int current = 0;

	// Total number of elements written to halos by this block
	__shared__ int blockSum[27];
	if (tid < 27) blockSum[tid] = 0;

	__syncthreads();

	for (int ix = min(cx, 1); ix <= max(cx, 1); ix++)
		for (int iy = min(cy, 1); iy <= max(cy, 1); iy++)
			for (int iz = min(cz, 1); iz <= max(cz, 1); iz++)
			{
				if (ix == 1 && iy == 1 && iz == 1) continue;

				const int bufId = (iz*3 + iy)*3 + ix;
				validHalos[current] = bufId;
				haloOffset[current] = atomicAdd(blockSum + bufId, pend-pstart);
				current++;
			}

	__syncthreads();

	if (tid < 27 && blockSum[tid] > 0)
		blockSum[tid] = atomicAdd(counts + tid, blockSum[tid]);

	if (QUERY) return;

	__syncthreads();

#pragma unroll 3
	for (int i=0; i<current; i++)
	{
		const int bufId = validHalos[i];
		const int myid  = blockSum[bufId] + haloOffset[i];

		const int ix = bufId % 3;
		const int iy = (bufId / 3) % 3;
		const int iz = bufId / 9;
		const float3 shift{ cinfo.localDomainSize.x*(ix-1),
							cinfo.localDomainSize.y*(iy-1),
							cinfo.localDomainSize.z*(iz-1) };

#pragma unroll 2
		for (int i = 0; i < pend-pstart; i++)
		{
			const int dstInd = myid   + i;
			const int srcInd = pstart + i;

			Particle p(cinfo.particles, srcInd);
			p.r -= shift;

			float4* addr = (float4*)dests[bufId];
			addr[2*dstInd + 0] = p.r2Float4();
			addr[2*dstInd + 1] = p.u2Float4();
		}
	}
}

void ParticleHaloExchanger::attach(ParticleVector* pv, CellList* cl)
{
	particles.push_back(pv);
	cellLists.push_back(cl);

	auto helper = new ExchangeHelper(pv->name, sizeof(Particle));
	helpers.push_back(helper);

	info("Particle halo exchanger takes pv %s, base tag %d", pv->name.c_str(), tagByName(pv->name));
}

void ParticleHaloExchanger::combineAndUploadData(int id, hipStream_t stream)
{
	auto pv = particles[id];
	auto helper = helpers[id];

	pv->halo()->resize_anew(helper->recvOffsets[27]);

	for (int i=0; i < 27; i++)
	{
		const int msize = helper->recvOffsets[i+1] - helper->recvOffsets[i];
		if (msize > 0)
			CUDA_Check( hipMemcpyAsync(pv->halo()->coosvels.devPtr() + helper->recvOffsets[i], helper->recvBufs[i].hostPtr(),
					msize*sizeof(Particle), hipMemcpyHostToDevice, stream) );
	}
}

void ParticleHaloExchanger::prepareData(int id, hipStream_t stream)
{
	auto pv = particles[id];
	auto cl = cellLists[id];
	auto helper = helpers[id];

	debug2("Preparing %s halo on the device", pv->name.c_str());

	helper->sendBufSizes.clear(stream);
	if (pv->local()->size() > 0)
	{
		const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
		const int nthreads = 64;
		const dim3 nblocks = dim3(getNblocks(maxdim*maxdim, nthreads), 6, 1);

		SAFE_KERNEL_LAUNCH(
				getHalos<true>,
				nblocks, nthreads, 0, stream,
				cl->cellInfo(),
				helper->sendAddrs.devPtr(), helper->sendBufSizes.devPtr() );

		helper->sendBufSizes.downloadFromDevice(stream);
		helper->resizeSendBufs();

		helper->sendBufSizes.clearDevice(stream);
		SAFE_KERNEL_LAUNCH(
				getHalos<false>,
				nblocks, nthreads, 0, stream,
				cl->cellInfo(),
				helper->sendAddrs.devPtr(), helper->sendBufSizes.devPtr() );
	}

	debug2("%s halo prepared", pv->name.c_str());
}




