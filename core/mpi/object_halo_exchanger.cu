#include "hip/hip_runtime.h"
#include "object_halo_exchanger.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/rigid_object_vector.h>
#include <core/logger.h>
#include <core/utils/cuda_common.h>

template<bool QUERY=false>
__global__ void getObjectHalos(const DomainInfo domain, const OVviewWithExtraData ovView,
		const float rc, BufferOffsetsSizesWrap dataWrap, int* haloParticleIds = nullptr)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	int nHalos = 0;
	short validHalos[7];

	if (objId < ovView.nObjects)
	{
		// Find to which halos this object should go
		auto prop = ovView.comAndExtents[objId];
		int cx = 1, cy = 1, cz = 1;

		if (prop.low.x  < -0.5f*domain.localSize.x + rc) cx = 0;
		if (prop.low.y  < -0.5f*domain.localSize.y + rc) cy = 0;
		if (prop.low.z  < -0.5f*domain.localSize.z + rc) cz = 0;

		if (prop.high.x >  0.5f*domain.localSize.x - rc) cx = 2;
		if (prop.high.y >  0.5f*domain.localSize.y - rc) cy = 2;
		if (prop.high.z >  0.5f*domain.localSize.z - rc) cz = 2;

//			if (tid == 0 && !QUERY) printf("Obj %d : [%f %f %f] -- [%f %f %f]\n", objId,
//			prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z);

		for (int ix = min(cx, 1); ix <= max(cx, 1); ix++)
			for (int iy = min(cy, 1); iy <= max(cy, 1); iy++)
				for (int iz = min(cz, 1); iz <= max(cz, 1); iz++)
				{
					if (ix == 1 && iy == 1 && iz == 1) continue;
					const int bufId = (iz*3 + iy)*3 + ix;
					validHalos[nHalos] = bufId;
					nHalos++;
				}
	}

	// Copy objects to each halo
	// TODO: maybe other loop order?
	__shared__ int shDstObjId;
	for (int i=0; i<nHalos; i++)
	{
		const int bufId = validHalos[i];

		const int ix = bufId % 3;
		const int iy = (bufId / 3) % 3;
		const int iz = bufId / 9;
		const float3 shift{ domain.localSize.x*(ix-1),
							domain.localSize.y*(iy-1),
							domain.localSize.z*(iz-1) };

		__syncthreads();
		if (tid == 0)
			shDstObjId = atomicAdd(dataWrap.sizes + bufId, 1);

		if (QUERY)
			continue;

		__syncthreads();

//		if (tid == 0)
//			printf("obj  %d  to halo  %d\n", objId, bufId);

		int myOffset = dataWrap.offsets[bufId] + shDstObjId;
		float4* dstAddr = (float4*) ( dataWrap.buffer + ovView.packedObjSize_byte * myOffset );
		int* partIdsAddr = haloParticleIds + ovView.objSize * myOffset;

		for (int pid = tid/2; pid < ovView.objSize; pid += blockDim.x/2)
		{
			const int srcId = objId * ovView.objSize + pid;
			Float3_int data(ovView.particles[2*srcId + sh]);

			// Remember your origin, little particle!
			if (sh == 1)
			{
				partIdsAddr[pid] = srcId;

				data.s2 = objId;
				data.s1 = pid;
			}

			if (sh == 0)
				data.v -= shift;

			dstAddr[2*pid + sh] = data.toFloat4();
		}

		// Add extra data at the end of the object
		dstAddr += ovView.objSize*2;
		ovView.packExtraData(objId, (char*)dstAddr);

		if (tid == 0) ovView.applyShift2extraData((char*)dstAddr, shift);
	}
}

__global__ static void unpackObject(const char* from, const int startDstObjId, OVviewWithExtraData ovView)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	const float4* srcAddr = (float4*) (from + ovView.packedObjSize_byte * objId);

	for (int pid = tid/2; pid < ovView.objSize; pid += blockDim.x/2)
	{
		const int dstId = (startDstObjId+objId)*ovView.objSize + pid;
		ovView.particles[2*dstId + sh] = srcAddr[2*pid + sh];
	}

	ovView.unpackExtraData( startDstObjId+objId, (char*)(srcAddr + 2*ovView.objSize));
}

//===============================================================================================
// Member functions
//===============================================================================================

bool ObjectHaloExchanger::needExchange(int id)
{
	return !objects[id]->haloValid;
}

void ObjectHaloExchanger::attach(ObjectVector* ov, float rc)
{
	objects.push_back(ov);
	rcs.push_back(rc);
	ExchangeHelper* helper = new ExchangeHelper(ov->name);
	helpers.push_back(helper);

	origins.push_back(new PinnedBuffer<int>(ov->local()->size()));

	info("Object vector %s (rc %f) was attached to halo exchanger", ov->name.c_str(), rc);
}

void ObjectHaloExchanger::prepareData(int id, hipStream_t stream)
{
	auto ov  = objects[id];
	auto rc  = rcs[id];
	auto helper = helpers[id];
	auto origin = origins[id];

	debug2("Preparing %s halo on the device", ov->name.c_str());

	OVviewWithExtraData ovView(ov, ov->local(), stream);
	helper->setDatumSize(ovView.packedObjSize_byte);

	helper->sendSizes.clear(stream);
	if (ovView.nObjects > 0)
	{
		const int nthreads = 256;

		SAFE_KERNEL_LAUNCH(
				getObjectHalos<true>,
				ovView.nObjects, nthreads, 0, stream,
				ov->domain, ovView, rc, helper->wrapSendData() );

		helper->makeSendOffsets_Dev2Dev(stream);
		helper->resizeSendBuf();

		// 1 int per particle: #objects x objSize x int
		origin->resize_anew(helper->sendOffsets[helper->nBuffers] * ovView.objSize);

		helper->sendSizes.clearDevice(stream);
		SAFE_KERNEL_LAUNCH(
				getObjectHalos<false>,
				ovView.nObjects, nthreads, 0, stream,
				ov->domain, ovView, rc, helper->wrapSendData(), origin->devPtr() );
	}
}

void ObjectHaloExchanger::combineAndUploadData(int id, hipStream_t stream)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	int totalRecvd = helper->recvOffsets[helper->nBuffers];

	// Make sure halo has ALL the extra data of local
	auto& haloMap = ov->halo()->getDataPerObjectMap();

	for (auto& kv : ov->local()->getDataPerObjectMap())
	{
		if (haloMap.find(kv.first) == haloMap.end())
			haloMap[kv.first] = std::unique_ptr<GPUcontainer>(kv.second->produce());
	}

	ov->halo()->resize_anew(totalRecvd * ov->objSize);
	OVviewWithExtraData ovView(ov, ov->halo(), stream);

	const int nthreads = 128;
	SAFE_KERNEL_LAUNCH(
			unpackObject,
			totalRecvd, nthreads, 0, stream,
			helper->recvBuf.devPtr(), 0, ovView );

	ov->haloValid = true;
}

PinnedBuffer<int>& ObjectHaloExchanger::getRecvOffsets(int id)
{
	return helpers[id]->recvOffsets;
}

PinnedBuffer<int>& ObjectHaloExchanger::getOrigins(int id)
{
	return *origins[id];
}




