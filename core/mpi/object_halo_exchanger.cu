#include "hip/hip_runtime.h"
#include <core/particle_vector.h>
#include <core/object_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/cuda_common.h>

#include <core/mpi/object_halo_exchanger.h>
#include <core/mpi/valid_cell.h>

#include <vector>
#include <algorithm>
#include <limits>

__global__ void getObjectHalos(const float4* __restrict__ coosvels, const ObjectVector::COMandExtent* props, const int nObj, const int objSize,
		const int* objParticleIds, const float3 domainSize, const float rc,
		const int64_t dests[27], int bufSizes[27], int* haloParticleIds,
		const int packedObjSize_float4, const int32_t** extraData, int nPtrsPerObj, const int* dataSizes)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	if (objId >= nObj) return;

	int nHalos = 0;
	short validHalos[7];

	// Find to which halos this object should go
	auto prop = props[objId];
	int cx = 1, cy = 1, cz = 1;

	if (prop.low.x  < -0.5*domainSize.x + rc) cx = 0;
	if (prop.low.y  < -0.5*domainSize.y + rc) cy = 0;
	if (prop.low.z  < -0.5*domainSize.z + rc) cz = 0;

	if (prop.high.x >  0.5*domainSize.x - rc) cx = 2;
	if (prop.high.y >  0.5*domainSize.y - rc) cy = 2;
	if (prop.high.z >  0.5*domainSize.z - rc) cz = 2;

//	if (tid == 0) printf("Obj %d : [%f %f %f] -- [%f %f %f]\n", objId,
//			prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z);

	for (int ix = min(cx, 1); ix <= max(cx, 1); ix++)
		for (int iy = min(cy, 1); iy <= max(cy, 1); iy++)
			for (int iz = min(cz, 1); iz <= max(cz, 1); iz++)
			{
				if (ix == 1 && iy == 1 && iz == 1) continue;
				const int bufId = (iz*3 + iy)*3 + ix;
				validHalos[nHalos] = bufId;
				nHalos++;
			}

	// Copy objects to each halo
	// TODO: maybe other loop order?
	__shared__ int shDstObjId;
	for (int i=0; i<nHalos; i++)
	{
		const int bufId = validHalos[i];

		const int ix = bufId % 3;
		const int iy = (bufId / 3) % 3;
		const int iz = bufId / 9;
		const float4 shift{ domainSize.x*(ix-1),
							domainSize.y*(iy-1),
							domainSize.z*(iz-1), 0.0f };

		__syncthreads();
		if (tid == 0)
			shDstObjId = atomicAdd(bufSizes + bufId, 1);
		__syncthreads();

		float4* dstAddr = (float4*) (dests[bufId]) + packedObjSize_float4;

		for (int pid = tid/2; pid < objSize; pid += blockDim.x/2)
		{
			const int srcId = objParticleIds[objId * objSize + pid];
			float4 data = coosvels[2*srcId + sh];

			// Remember your origin, little particle!
			if (sh == 1)
				data.w = __int_as_float(pid);

			if (sh == 0)
				data -= shift;

			dstAddr[2*pid + sh] = data;
		}

		// Add extra data at the end of the object
		dstAddr += objSize*2;
		packExtraData(objId, extraData, nPtrsPerObj, dataSizes, (int32_t*)dstAddr);
	}
}


__global__ void unpackObject(const float4* from, float4* to, const int objSize, const int packedObjSize_float4, const int nObj,
		int32_t** extraData, int nPtrsPerObj, const int* dataSizes)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	for (int pid = tid/2; pid < objSize; pid += blockDim.x/2)
	{
		const int srcId = objParticleIds[objId * packedObjSize_float4 + pid*2];
		float4 data = coosvels[srcId + sh];

		to[objId*objSize + 2*pid + sh] = data;
	}

	unpackExtraData(objId, extraData, nPtrsPerObj, dataSizes);
}

__device__ void packExtraData(int objId, const int32_t** extraData, int nPtrsPerObj, const int* dataSizes, int32_t* destanation)
{
	int baseId = 0;

	for (int ptrId = 0; ptrId < nPtrsPerObj; ptrId++)
		{
			const int size = dataSizes[ptrId];
			for (int i = threadIdx.x; i < size; i += blockDim.x)
				destanation[baseId+i] = extraData[ptrId][objId*size + i];

			baseId += dataSizes[ptrId];
		}
}

__device__ void unpackExtraData(int objId, int32_t** extraData, int nPtrsPerObj, const int* dataSizes, const int32_t* source)
{
	int baseId = 0;

	for (int ptrId = 0; ptrId < nPtrsPerObj; ptrId++)
	{
		const int size = dataSizes[ptrId];
		for (int i = threadIdx.x; i < size; i += blockDim.x)
			extraData[ptrId][objId*size + i] = source[baseId+i];

		baseId += dataSizes[ptrId];
	}
}




void ObjectHaloExchanger::attach(ObjectVector* ov, float rc)
{
	objects.push_back(ov);
	rcs.push_back(rc);

	const int maxdim = std::max({ov->domainSize.x, ov->domainSize.y, ov->domainSize.z});
	const float ndens = (double)ov->local()->size() / (ov->domainSize.x * ov->domainSize.y * ov->domainSize.z);

	int extraSize_bytes = 0;
	for (int i=0; i<ov->extraDataNumPtrs(); i++)
		extraSize_bytes += ov->extraDataSize(i);
	int totSize = ov->objSize + extraSize_bytes/sizeof(Particle);


	const int sizes[3] = { (int)(4*ndens * maxdim*maxdim + 10*totSize),
						   (int)(4*ndens * maxdim + 10*totSize),
						   (int)(4*ndens + 10*totSize) };


	ExchangeHelper* helper = new ExchangeHelper(ov->name, totSize * sizeof(Particle), sizes);
	ov->halo()->pushStream(helper->stream);
	ov->haloForces.pushStream(helper->stream);
	helpers.push_back(helper);

	helper->extraDataPtrs_local.resize(ov->extraDataNumPtrs());
	helper->extraDataPtrs_halo .resize(ov->extraDataNumPtrs());
	helper->extraDataSizes     .resize(ov->extraDataNumPtrs());

	for (int i=0; i<helper->extraDataPtrs.size(); i++)
	{
		helper->extraDataPtrs_local[i] = ov->extraDataPtr_local(i);
		helper->extraDataPtrs_halo[i]  = ov->extraDataPtr_halo(i);
		helper->extraDataSizes[i]      = ov->extraDataSize(i);
	}

	helper->extraDataPtrs_local.uploadToDevice();
	helper->extraDataPtrs_halo .uploadToDevice();
	helper->extraDataSizes     .uploadToDevice();
}


void ObjectHaloExchanger::prepareData(int id)
{
	auto ov = objects[id];
	auto rc = rcs[id];
	auto helper = helpers[id];

	debug2("Preparing %s halo on the device", ov->name.c_str());

	helper->bufSizes.pushStream(defStream);
	helper->bufSizes.clearDevice();
	helper->bufSizes.popStream();

	const int nthreads = 128;
	if (ov->nObjects > 0)
	{
		const int  nPtrs = helper->extraDataPtrs.size();
		int32_t**  dataPtrs  = helper->extraDataPtrs_local. devPtr();
		const int* dataSizes = helper->extraDataSizes.devPtr();

		int extraSize_bytes = 0;
		for (int i=0; i<nPtrs; i++)
			extraSize_bytes += ov->extraDataSize(i);

		int totalObjSize_float4 = ov->objSize*2 + (extraSize_bytes+sizeof(float4)-1)/sizeof(float4);

		getObjectHalos <<< ov->nObjects, nthreads, 0, defStream >>>
				((float4*)ov->local()->coosvels.devPtr(), ov->com_extent.devPtr(), ov->nObjects, ov->objSize, ov->particles2objIds.devPtr(), ov->domainSize, rc,
				 (int64_t*)helper->sendAddrs.devPtr(), helper->bufSizes.devPtr(), ov->haloIds.devPtr(),
				 totalObjSize_float4, dataPtrs, dataSizes);
	}
}

void ObjectHaloExchanger::combineAndUploadData(int id)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	ov->halo()->resize(helper->recvOffsets[27] / sizeof(Particle), resizeAnew);
	ov->halo()->resize(helper->recvOffsets[27] / sizeof(Particle), resizeAnew);

	const int nthreads = 128;
	for (int i=0; i < 27; i++)
	{
		const int msize = helper->recvOffsets[i+1] - helper->recvOffsets[i];
		if (msize > 0)
		{
			const int nPtrs = helper->extraDataPtrs.size();
			const int32_t** dataPtrs  = helper->extraDataPtrs. devPtr();
			const int*      dataSizes = helper->extraDataSizes.devPtr();

			int extraSize_bytes = 0;
			for (int i=0; i<helper->local()->size()trs; i++)
				extraSize_bytes += ov->extraDataSize(i);

			const int nObjs     = msize                  / (ov->objSize*sizeof(Particle) + extraSize_bytes);
			const int objOffset = helper->recvOffsets[i] / (ov->objSize*sizeof(Particle) + extraSize_bytes);

			int totalObjSize_float4 = ov->objSize*2 + (extraSize_bytes+sizeof(float4)-1)/sizeof(float4);

			unpackObject<<< nObjs, nthreads, 0, defStream >>>
					(helper->recvBufs[i].devPtr(), (float4*)(ov->local()->coosvels.devPtr()+ov->objOffset*nObjs), ov->objSize, totalObjSize_float4, nObjs, extraData, dataSizes);
		}
	}
}




//__global__ void addHaloForces(const float4* haloForces, const float4* halo, float4* forces, int n)
//{
//	const int srcId = blockIdx.x*blockDim.x + threadIdx.x;
//	if (srcId >= n) return;
//
//	const int dstId = __float_as_int(halo[2*srcId].w);
//	const float4 frc = readNoCache(haloForces + srcId);
//	forces[dstId] += frc;
//}
//
//void ObjectHaloExchanger::exchangeForces()
//{
//	for (int i=0; i<objects.size(); i++)
//		prepareForces(objects[i], helpers[i]);
//
//	for (int i=0; i<objects.size(); i++)
//		exchange(helpers[i], sizeof(Force));
//
//	for (int i=0; i<objects.size(); i++)
//		uploadForces(objects[i], helpers[i]);
//
//	for (auto helper : helpers)
//		CUDA_Check( hipStreamSynchronize(helper->stream) );
//}
//
//void ObjectHaloExchanger::prepareForces(ObjectVector* ov, HaloHelper* helper)
//{
//	debug2("Preparing %s halo on the device", ov->name.c_str());
//
//	for (int i=0; i<27; i++)
//	{
//		helper->bufSizes[i] = helper->recvOffsets[i+1] - helper->recvOffsets[i];
//		if (helper->bufSizes[i] > 0)
//			CUDA_Check( hipMemcpyAsync(ov->haloForces.devPtr() + helper->recvOffsets[i], helper->sendBufs[i].hostPtr(),
//					helper->bufSizes[i]*sizeof(Force), hipMemcpyHostToDevice, helper->stream) );
//	}
//
//	// implicit synchronization here
//	helper->bufSizes.uploadToDevice();
//}
//
//void ObjectHaloExchanger::uploadForces(ObjectVector* ov, HaloHelper* helper)
//{
//	for (int i=0; i < helper->recvOffsets.size(); i++)
//	{
//		const int msize = helper->recvOffsets[i+1] - helper->recvOffsets[i];
//
//		if (msize > 0)
//			CUDA_Check( hipMemcpyAsync(ov->haloForces.devPtr() + helper->recvOffsets[i], helper->recvBufs[compactedDirs[i]].hostPtr(),
//					msize*sizeof(Force), hipMemcpyHostToDevice, helper->stream) );
//	}
//
//	const int np = helper->recvOffsets[27];
//	addHaloForces<<< (np+127)/128, 128, 0, helper->stream >>> ( (float4*)ov->haloForces.devPtr(), (float4*)ov->halo()->local()->coosvels->devPtr(), (float4*)ov->local()->forces.devPtr(), np);
//}
//

