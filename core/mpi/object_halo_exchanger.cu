#include "hip/hip_runtime.h"
#include <core/particle_vector.h>
#include <core/object_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/cuda_common.h>

#include <core/mpi/object_halo_exchanger.h>
#include <core/mpi/valid_cell.h>

#include <vector>
#include <algorithm>
#include <limits>



__device__ void packExtraData(int objId, int32_t** extraData, int nPtrsPerObj, const int* dataSizes, int32_t* destanation)
{
	int baseId = 0;

	for (int ptrId = 0; ptrId < nPtrsPerObj; ptrId++)
		{
			// dataSizes are in bytes
			const int size = dataSizes[ptrId] / 4;
			for (int i = threadIdx.x; i < size; i += blockDim.x)
				destanation[baseId+i] = extraData[ptrId][objId*size + i];

			baseId += dataSizes[ptrId];
		}
}

__device__ void unpackExtraData(int objId, int32_t** extraData, int nPtrsPerObj, const int* dataSizes, const int32_t* source)
{
	int baseId = 0;

	for (int ptrId = 0; ptrId < nPtrsPerObj; ptrId++)
	{
		// dataSizes are in bytes
		const int size = dataSizes[ptrId] / 4;
		for (int i = threadIdx.x; i < size; i += blockDim.x)
			extraData[ptrId][objId*size + i] = source[baseId+i];

		baseId += dataSizes[ptrId];
	}
}


__global__ void getObjectHalos(const float4* __restrict__ coosvels, const LocalObjectVector::COMandExtent* props, const int nObj, const int objSize,
		const float3 domainSize, const float rc,
		const int64_t dests[27], int bufSizes[27], /*int* haloParticleIds,*/
		const int packedObjSize_byte, int32_t** extraData, int nPtrsPerObj, const int* dataSizes)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	if (objId >= nObj) return;

	int nHalos = 0;
	short validHalos[7];

	// Find to which halos this object should go
	auto prop = props[objId];
	int cx = 1, cy = 1, cz = 1;

	if (prop.low.x  < -0.5*domainSize.x + rc) cx = 0;
	if (prop.low.y  < -0.5*domainSize.y + rc) cy = 0;
	if (prop.low.z  < -0.5*domainSize.z + rc) cz = 0;

	if (prop.high.x >  0.5*domainSize.x - rc) cx = 2;
	if (prop.high.y >  0.5*domainSize.y - rc) cy = 2;
	if (prop.high.z >  0.5*domainSize.z - rc) cz = 2;

//	if (tid == 0) printf("Obj %d : [%f %f %f] -- [%f %f %f]\n", objId,
//			prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z);

	for (int ix = min(cx, 1); ix <= max(cx, 1); ix++)
		for (int iy = min(cy, 1); iy <= max(cy, 1); iy++)
			for (int iz = min(cz, 1); iz <= max(cz, 1); iz++)
			{
				if (ix == 1 && iy == 1 && iz == 1) continue;
				const int bufId = (iz*3 + iy)*3 + ix;
				validHalos[nHalos] = bufId;
				nHalos++;
			}

	// Copy objects to each halo
	// TODO: maybe other loop order?
	__shared__ int shDstObjId;
	for (int i=0; i<nHalos; i++)
	{
		const int bufId = validHalos[i];

		const int ix = bufId % 3;
		const int iy = (bufId / 3) % 3;
		const int iz = bufId / 9;
		const float3 shift{ domainSize.x*(ix-1),
							domainSize.y*(iy-1),
							domainSize.z*(iz-1) };

		__syncthreads();
		if (tid == 0)
			shDstObjId = atomicAdd(bufSizes + bufId, 1);
		__syncthreads();

//		if (tid == 0)
//			if (objId == 5)
//				printf("obj  %d  to halo  %d  [%f %f %f] - [%f %f %f]  %d %d %d\n", objId, bufId,
//						prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z, cx, cy, cz);

		float4* dstAddr = (float4*) (dests[bufId]) + packedObjSize_byte/sizeof(float4) * shDstObjId;

		for (int pid = tid/2; pid < objSize; pid += blockDim.x/2)
		{
			const int srcId = objId * objSize + pid;
			Float3_int data(coosvels[2*srcId + sh]);

			// Remember your origin, little particle!
			if (sh == 1)
				data.s2 = objId;

			if (sh == 0)
				data.v -= shift;

			dstAddr[2*pid + sh] = data.toFloat4();
		}

		// Add extra data at the end of the object
		dstAddr += objSize*2;
		packExtraData(objId, extraData, nPtrsPerObj, dataSizes, (int32_t*)dstAddr);
	}
}


__global__ void unpackObject(const float4* from, float4* to, const int objSize, const int packedObjSize_byte, const int nObj,
		int32_t** extraData, int nPtrsPerObj, const int* dataSizes)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	for (int pid = tid/2; pid < objSize; pid += blockDim.x/2)
	{
		const int srcId = objId * packedObjSize_byte/sizeof(float4) + pid*2;
		float4 data = from[srcId + sh];

		to[2*(objId*objSize + pid) + sh] = data;
	}

	unpackExtraData(objId, extraData, nPtrsPerObj, dataSizes, (int32_t*)( ((char*)from) + objId * packedObjSize_byte + objSize*sizeof(Particle) ));
}





void ObjectHaloExchanger::attach(ObjectVector* ov, float rc)
{
	objects.push_back(ov);
	rcs.push_back(rc);

	const float objPerCell = 0.1f;

	const int maxdim = std::max({ov->domainSize.x, ov->domainSize.y, ov->domainSize.z});

	const int sizes[3] = { (int)(4*objPerCell * maxdim*maxdim + 10),
						   (int)(4*objPerCell * maxdim + 10),
						   (int)(4*objPerCell + 10) };


	ExchangeHelper* helper = new ExchangeHelper(ov->name, ov->local()->packedObjSize_bytes, sizes);
	ov->halo()->pushStream(helper->stream);
	helpers.push_back(helper);
}


void ObjectHaloExchanger::prepareData(int id)
{
	auto ov = objects[id];
	auto rc = rcs[id];
	auto helper = helpers[id];

	debug2("Preparing %s halo on the device", ov->name.c_str());

	helper->bufSizes.pushStream(defStream);
	helper->bufSizes.clearDevice();
	helper->bufSizes.popStream();

	const int nthreads = 128;
	if (ov->local()->nObjects > 0)
	{
		int       nPtrs  = ov->local()->extraDataPtrs.size();
		int totSize_byte = ov->local()->packedObjSize_bytes;

		getObjectHalos <<< ov->local()->nObjects, nthreads, 0, defStream >>> (
				(float4*)ov->local()->coosvels.devPtr(), ov->local()->comAndExtents.devPtr(),
				ov->local()->nObjects, ov->local()->objSize, ov->domainSize, rc,
				(int64_t*)helper->sendAddrs.devPtr(), helper->bufSizes.devPtr(),
				totSize_byte, ov->local()->extraDataPtrs.devPtr(), nPtrs, ov->local()->extraDataSizes.devPtr());
	}
}

void ObjectHaloExchanger::combineAndUploadData(int id)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	ov->halo()->resize(helper->recvOffsets[27] * ov->halo()->objSize, resizeAnew);
	ov->halo()->resize(helper->recvOffsets[27] * ov->halo()->objSize, resizeAnew);

	const int nthreads = 128;
	for (int i=0; i < 27; i++)
	{
		const int nObjs = helper->recvOffsets[i+1] - helper->recvOffsets[i];
		if (nObjs > 0)
		{
			int        nPtrs = ov->local()->extraDataPtrs.size();
			int totSize_byte = ov->local()->packedObjSize_bytes;

			unpackObject<<< nObjs, nthreads, 0, defStream >>>
					((float4*)helper->recvBufs[i].devPtr(), (float4*)(ov->halo()->coosvels.devPtr() + helper->recvOffsets[i]*nObjs), ov->local()->objSize, totSize_byte, nObjs,
					 ov->halo()->extraDataPtrs.devPtr(), nPtrs, ov->halo()->extraDataSizes.devPtr());
		}
	}
}



//__global__ void addHaloForces(const float4* haloForces, const float4* halo, float4* forces, int n)
//{
//	const int srcId = blockIdx.x*blockDim.x + threadIdx.x;
//	if (srcId >= n) return;
//
//	const int dstId = __float_as_int(halo[2*srcId].w);
//	const float4 frc = readNoCache(haloForces + srcId);
//	forces[dstId] += frc;
//}
//
//void ObjectHaloExchanger::exchangeForces()
//{
//	for (int i=0; i<objects.size(); i++)
//		prepareForces(objects[i], helpers[i]);
//
//	for (int i=0; i<objects.size(); i++)
//		exchange(helpers[i], sizeof(Force));
//
//	for (int i=0; i<objects.size(); i++)
//		uploadForces(objects[i], helpers[i]);
//
//	for (auto helper : helpers)
//		CUDA_Check( hipStreamSynchronize(helper->stream) );
//}
//
//void ObjectHaloExchanger::prepareForces(ObjectVector* ov, HaloHelper* helper)
//{
//	debug2("Preparing %s halo on the device", ov->name.c_str());
//
//	for (int i=0; i<27; i++)
//	{
//		helper->bufSizes[i] = helper->recvOffsets[i+1] - helper->recvOffsets[i];
//		if (helper->bufSizes[i] > 0)
//			CUDA_Check( hipMemcpyAsync(ov->haloForces.devPtr() + helper->recvOffsets[i], helper->sendBufs[i].hostPtr(),
//					helper->bufSizes[i]*sizeof(Force), hipMemcpyHostToDevice, helper->stream) );
//	}
//
//	// implicit synchronization here
//	helper->bufSizes.uploadToDevice();
//}
//
//void ObjectHaloExchanger::uploadForces(ObjectVector* ov, HaloHelper* helper)
//{
//	for (int i=0; i < helper->recvOffsets.size(); i++)
//	{
//		const int msize = helper->recvOffsets[i+1] - helper->recvOffsets[i];
//
//		if (msize > 0)
//			CUDA_Check( hipMemcpyAsync(ov->haloForces.devPtr() + helper->recvOffsets[i], helper->recvBufs[compactedDirs[i]].hostPtr(),
//					msize*sizeof(Force), hipMemcpyHostToDevice, helper->stream) );
//	}
//
//	const int np = helper->recvOffsets[27];
//	addHaloForces<<< (np+127)/128, 128, 0, helper->stream >>> ( (float4*)ov->haloForces.devPtr(), (float4*)ov->halo()->local()->coosvels->devPtr(), (float4*)ov->local()->forces.devPtr(), np);
//}
//

