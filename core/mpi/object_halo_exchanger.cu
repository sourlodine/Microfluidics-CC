#include "hip/hip_runtime.h"
#include "object_halo_exchanger.h"

#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/rigid_object_vector.h>
#include <core/logger.h>
#include <core/utils/cuda_common.h>

template<bool QUERY=false>
__global__ void getObjectHalos(const OVviewWithExtraData ovView, const ROVview rovView,
		const float rc, char** dests, int* sendBufSizes, int** haloParticleIds = nullptr)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	int nHalos = 0;
	short validHalos[7];

	if (objId < ovView.nObjects)
	{
		// Find to which halos this object should go
		auto prop = ovView.comAndExtents[objId];
		int cx = 1, cy = 1, cz = 1;

		if (prop.low.x  < -0.5f*ovView.localDomainSize.x + rc) cx = 0;
		if (prop.low.y  < -0.5f*ovView.localDomainSize.y + rc) cy = 0;
		if (prop.low.z  < -0.5f*ovView.localDomainSize.z + rc) cz = 0;

		if (prop.high.x >  0.5f*ovView.localDomainSize.x - rc) cx = 2;
		if (prop.high.y >  0.5f*ovView.localDomainSize.y - rc) cy = 2;
		if (prop.high.z >  0.5f*ovView.localDomainSize.z - rc) cz = 2;

//			if (tid == 0 && !QUERY) printf("Obj %d : [%f %f %f] -- [%f %f %f]\n", objId,
//			prop.low.x, prop.low.y, prop.low.z, prop.high.x, prop.high.y, prop.high.z);

		for (int ix = min(cx, 1); ix <= max(cx, 1); ix++)
			for (int iy = min(cy, 1); iy <= max(cy, 1); iy++)
				for (int iz = min(cz, 1); iz <= max(cz, 1); iz++)
				{
					if (ix == 1 && iy == 1 && iz == 1) continue;
					const int bufId = (iz*3 + iy)*3 + ix;
					validHalos[nHalos] = bufId;
					nHalos++;
				}
	}

	// Copy objects to each halo
	// TODO: maybe other loop order?
	__shared__ int shDstObjId;
	for (int i=0; i<nHalos; i++)
	{
		const int bufId = validHalos[i];

		const int ix = bufId % 3;
		const int iy = (bufId / 3) % 3;
		const int iz = bufId / 9;
		const float3 shift{ ovView.localDomainSize.x*(ix-1),
							ovView.localDomainSize.y*(iy-1),
							ovView.localDomainSize.z*(iz-1) };

		__syncthreads();
		if (tid == 0)
			shDstObjId = atomicAdd(sendBufSizes + bufId, 1);

		if (QUERY)
			continue;

		__syncthreads();

//		if (tid == 0)
//			printf("obj  %d  to halo  %d\n", objId, bufId);

		float4* dstAddr = (float4*) (dests[bufId]) + ovView.packedObjSize_byte/sizeof(float4) * shDstObjId;

		for (int pid = tid/2; pid < ovView.objSize; pid += blockDim.x/2)
		{
			const int srcId = objId * ovView.objSize + pid;
			Float3_int data(ovView.particles[2*srcId + sh]);

			// Remember your origin, little particle!
			if (sh == 1)
			{
				haloParticleIds[bufId][shDstObjId * ovView.objSize + pid] = srcId;

				data.s2 = objId;
				data.s1 = pid;
			}

			if (sh == 0)
				data.v -= shift;

			dstAddr[2*pid + sh] = data.toFloat4();
		}

		// Add extra data at the end of the object
		dstAddr += ovView.objSize*2;
		ovView.packExtraData(objId, (char*)dstAddr);

		if (rovView.objSize == ovView.objSize)
			rovView.applyShift2extraData((char*)dstAddr, shift);
	}
}

__global__ static void unpackObject(const float4* from, const int startDstObjId, OVviewWithExtraData ovView)
{
	const int objId = blockIdx.x;
	const int tid = threadIdx.x;
	const int sh  = tid % 2;

	const float4* srcAddr = from + ovView.packedObjSize_byte/sizeof(float4) * objId;

	for (int pid = tid/2; pid < ovView.objSize; pid += blockDim.x/2)
	{
		const int dstId = (startDstObjId+objId)*ovView.objSize + pid;
		ovView.particles[2*dstId + sh] = srcAddr[2*pid + sh];
	}

	ovView.unpackExtraData( startDstObjId+objId, (char*)(srcAddr + 2*ovView.objSize));
}


void ObjectHaloExchanger::attach(ObjectVector* ov, float rc)
{
	objects.push_back(ov);
	rcs.push_back(rc);
	ExchangeHelper* helper = new ExchangeHelper(ov->name);
	helpers.push_back(helper);

	ExchangeHelper* originHelper = new ExchangeHelper(ov->name, sizeof(int)*ov->objSize);
	originHelpers.push_back(originHelper);

	info("Object vector %s (rc %f) was attached to halo exchanger", ov->name.c_str(), rc);
}


void ObjectHaloExchanger::prepareData(int id, hipStream_t stream)
{
	auto ov  = objects[id];
	auto rc  = rcs[id];
	auto helper = helpers[id];
	auto originHelper = originHelpers[id];

	debug2("Preparing %s halo on the device", ov->name.c_str());

	auto ovView = create_OVviewWithExtraData(ov, ov->local(), stream);
	helper->setDatumSize(ovView.packedObjSize_byte);
	helper->sendBufSizes.clear(stream);

	const int nthreads = 128;
	if (ovView.nObjects > 0)
	{
		// FIXME: this is a hack
		auto rovView = create_ROVview(nullptr, nullptr);
		RigidObjectVector* rov;
		if ( (rov = dynamic_cast<RigidObjectVector*>(ov)) != 0 )
			rovView = create_ROVview(rov, rov->local());

		getObjectHalos<true>  <<< ovView.nObjects, nthreads, 0, stream >>> (
				ovView, rovView, rc, helper->sendAddrs.devPtr(), helper->sendBufSizes.devPtr());

		helper->sendBufSizes.downloadFromDevice(stream);
		for (int i=0; i<helper->sendBufSizes.size(); i++)
			originHelper->sendBufSizes[i] = helper->sendBufSizes[i];

		helper->resizeSendBufs();
		originHelper->resizeSendBufs();

		helper->sendBufSizes.clearDevice(stream);
		getObjectHalos<false> <<< ovView.nObjects, nthreads, 0, stream >>> (
				ovView, rovView, rc, helper->sendAddrs.devPtr(), helper->sendBufSizes.devPtr(), (int**)originHelper->sendAddrs.devPtr());
	}
}

void ObjectHaloExchanger::combineAndUploadData(int id, hipStream_t stream)
{
	auto ov = objects[id];
	auto helper = helpers[id];

	ov->halo()->resize_anew(helper->recvOffsets[27] * ov->objSize);
	auto ovView = create_OVviewWithExtraData(ov, ov->halo(), stream);

	// TODO: unite into one unpack call
	const int nthreads = 128;
	for (int i=0; i < 27; i++)
	{
		const int nObjs = helper->recvOffsets[i+1] - helper->recvOffsets[i];
		if (nObjs > 0)
		{
			helper->recvBufs[i].uploadToDevice(stream);
			unpackObject<<< nObjs, nthreads, 0, stream >>> ( (float4*)helper->recvBufs[i].devPtr(),  helper->recvOffsets[i], ovView );
		}
	}
}

std::vector<int>& ObjectHaloExchanger::getRecvOffsets(int id)
{
	return helpers[id]->recvOffsets;
}

PinnedBuffer<char*>& ObjectHaloExchanger::getOriginAddrs(int id)
{
	return originHelpers[id]->sendAddrs;
}




