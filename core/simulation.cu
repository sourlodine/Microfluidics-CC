#include "hip/hip_runtime.h"
#include <core/simulation.h>
#include <core/integrate.h>
#include <core/interactions.h>
#include <core/redistributor.h>
#include <core/halo_exchanger.h>
#include <core/logger.h>

Simulation::Simulation(int3 nranks3D, float3 globalDomainSize, MPI_Comm& comm, MPI_Comm& interComm) :
nranks3D(nranks3D), globalDomainSize(globalDomainSize), interComm(interComm)
{
	int ranksArr[] = {nranks3D.x, nranks3D.y, nranks3D.z};
	int periods[] = {1, 1, 1};
	int coords[3];

	MPI_Check( MPI_Comm_rank(comm, &rank) );
	MPI_Check( MPI_Cart_create(comm, 3, ranksArr, periods, 0, &cartComm) );
	MPI_Check( MPI_Cart_get(cartComm, 3, ranksArr, periods, coords) );
	rank3D = {coords[0], coords[1], coords[2]};

	subDomainSize = globalDomainSize / make_float3(nranks3D);
	subDomainStart = {subDomainSize.x * coords[0], subDomainSize.y * coords[1], subDomainSize.y * coords[2]};
}

void Simulation::registerParticleVector(ParticleVector* pv, InitialConditions* ic)
{
	std::string name = pv->name;
	particleVectors.push_back(pv);

	if (pvMap.find(name) != pvMap.end())
		die("More than one particle vector is called %s", name.c_str());

	pvMap[name] = particleVectors.size() - 1;
	ic->exec(pv, globalDomainSize, subDomainSize);
}

void Simulation::registerObjectVector  (ObjectVector* ov)
{
	std::string name = ov->name;
	particleVectors.push_back(static_cast<ParticleVector*>(ov));

	if (pvMap.find(name) != pvMap.end())
		die("More than one particle vector is called %s", name.c_str());

	pvMap[name] = particleVectors.size() - 1;
}

void Simulation::registerWall          (Wall* wall)
{
	std::string name = wall->name;

	if (wallMap.find(name) != wallMap.end())
		die("More than one wall is called %s", name.c_str());

	if (pvMap.find(name) != pvMap.end())
		die("Wall has the same name as particle vector: %s", name.c_str());

	wallMap[name] = wall;

	particleVectors.push_back(wall->getFrozen());
	pvMap[name] = particleVectors.size() - 1;
}

void Simulation::registerInteraction   (Interaction* interaction)
{
	std::string name = interaction->name;
	if (interactionMap.find(name) != interactionMap.end())
		die("More than one interaction is called %s", name.c_str());

	interactionMap[name] = interaction;
}

void Simulation::registerIntegrator    (Integrator* integrator)
{
	std::string name = integrator->name;
	if (integratorMap.find(name) != integratorMap.end())
		die("More than one interaction is called %s", name.c_str());

	integratorMap[name] = integrator;
}

void Simulation::registerPlugin(SimulationPlugin* plugin)
{
	plugins.push_back(plugin);
}

void Simulation::setIntegrator(std::string pvName, std::string integratorName)
{
	if (pvMap.find(pvName) == pvMap.end())
		die("No such particle vector: %s", pvName.c_str());

	if (integratorMap.find(integratorName) == integratorMap.end())
		die("No such integrator: %s", integratorName.c_str());

	const int pvId = pvMap[pvName];
	integrators.resize(std::max((int)integrators.size(), pvId+1), nullptr);
	integrators[pvId] = integratorMap[integratorName];
}

void Simulation::setInteraction(std::string pv1Name, std::string pv2Name, std::string interactionName)
{
	if (pvMap.find(pv1Name) == pvMap.end())
		die("No such particle vector: %s", pv1Name.c_str());

	if (pvMap.find(pv2Name) == pvMap.end())
		die("No such particle vector: %s", pv2Name.c_str());

	if (interactionMap.find(interactionName) == interactionMap.end())
		die("No such integrator: %s", interactionName.c_str());

	const int pv1Id = pvMap[pv1Name];
	const int pv2Id = pvMap[pv2Name];

	// Allocate interactionTable
	interactionTable.resize(std::max((int)interactionTable.size(), pv1Id+1));
	auto& interactionVector = interactionTable[pv1Id];
	interactionVector.resize( std::max((int)interactionVector.size(), pv2Id+1), {nullptr, nullptr} );

	// Find interaction
	auto interaction = interactionMap[interactionName];

	cellListTable.resize(std::max((int)cellListTable.size(), pv1Id+1));

	CellList* cl = nullptr;
	for (auto& entry : cellListTable[pv1Id])
	{
		if (fabs(entry->rc - interaction->rc) < 1e-6)
		{
			cl = entry;
			break;
		}
	}
	if (cl == nullptr)
	{
		cl = new CellList(particleVectors[pv1Id], interaction->rc, subDomainStart, subDomainSize);
		cellListTable[pv1Id].push_back(cl);
	}

	interactionTable[pv1Id][pv2Id] = {interaction, cl};
}

// TODO: wall has self-interactions
void Simulation::run(int nsteps)
{
	hipStream_t defStream;
	CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );

	for (auto& pl : plugins)
	{
		pl->setup(this, defStream, cartComm, interComm);
		pl->handshake();
	}

	// TODO: STREAMS FOR CELL-LISTS

	HaloExchanger halo(cartComm);
	Redistributor redist(cartComm);

	cellListTable.resize(particleVectors.size());
	for (int i=0; i<particleVectors.size(); i++)
	{
		if (cellListTable[i].size() > 0)
		{
			auto it = std::max_element(cellListTable[i].begin(), cellListTable[i].end(),
					[] (CellList* cl1, CellList* cl2) { return cl1->rc < cl2->rc; } );
			halo.attach(particleVectors[i], *it);
			redist.attach(particleVectors[i], *it);
		}

		particleVectors[i]->setStream(defStream);
		for (auto& cl : cellListTable[i])
			cl->setStream(defStream);
	}

	float t = 0;
	for (int iter=0; iter<nsteps; iter++)
	{
		//===================================================================================================
		for (auto& pv : particleVectors)
			pv->forces.clear();

		//===================================================================================================
		for (auto& cllist : cellListTable)
			for (auto& cl : cllist)
				cl->build(defStream);

		for (auto& pl : plugins)
			pl->beforeForces(t);

		//===================================================================================================
		for (int i=0; i<interactionTable.size(); i++)
			for (int j=0; j<interactionTable[i].size(); j++)
				if (interactionTable[i][j].first != nullptr)
				{
					if (i == j)
					{
						if (interactionTable[i][j].first != nullptr)
							interactionTable[i][j].first->execSelf(particleVectors[i], interactionTable[i][j].second, t, defStream);
					}
					else
					{
						if (interactionTable[i][j].first != nullptr)
							interactionTable[i][j].first->execExternal(particleVectors[i], particleVectors[j], interactionTable[i][j].second, t, defStream);
					}
				}

		//===================================================================================================
		halo.exchange();

		//===================================================================================================
		for (int i=0; i<interactionTable.size(); i++)
			for (int j=0; j<interactionTable[i].size(); j++)
				if (interactionTable[i][j].first != nullptr)
				{
					if (interactionTable[i][j].first != nullptr)
						interactionTable[i][j].first->execHalo(particleVectors[i], particleVectors[j], interactionTable[i][j].second, t, defStream);
				}

		for (auto& pl : plugins)
			pl->beforeIntegration(t);

		//===================================================================================================
		for (int i=0; i<integrators.size(); i++)
			if (integrators[i] != nullptr)
				integrators[i]->exec(particleVectors[i], defStream);
		CUDA_Check( hipStreamSynchronize(defStream) );

		for (auto& pl : plugins)
			pl->afterIntegration(t);

		//===================================================================================================
		redist.redistribute();
		CUDA_Check( hipStreamSynchronize(defStream) );
	}

	MPI_Check( MPI_Barrier(cartComm) );
}


//===================================================================================================
// Postprocessing
//===================================================================================================

Postprocess::Postprocess(MPI_Comm& comm, MPI_Comm& interComm) : comm(comm), interComm(interComm) {};

void Postprocess::registerPlugin(PostprocessPlugin* plugin)
{
	plugins.push_back(plugin);
}

void Postprocess::run()
{
	for (auto& pl : plugins)
	{
		pl->setup(comm, interComm);
		pl->handshake();
	}

	std::vector<MPI_Request> requests;
	for (auto& pl : plugins)
		requests.push_back(pl->postRecv());

	// TODO: need stopping criterion
	while (true)
	{
		int index;
		MPI_Check( MPI_Waitany(requests.size(), requests.data(), &index, MPI_STATUS_IGNORE) );

		plugins[index]->deserialize();
		requests[index] = plugins[index]->postRecv();
	}
}



//===================================================================================================
// uDeviceX
//===================================================================================================

uDeviceX::uDeviceX(int argc, char** argv, int3 nranks3D, float3 globalDomainSize, Logger& logger, std::string logFileName, int verbosity)
{
	int nranks, rank;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
		printf("ERROR: The MPI library does not have full thread support\n");
		MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, logFileName, verbosity);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );

	MPI_Comm ioComm, compComm, interComm, splitComm;

	if (nranks % 2 != 0)
		die("Number of MPI ranks should be even");

	debug("Program started, splitting commuticator");

	computeTask = (rank+1) % 2;
	MPI_Check( MPI_Comm_split(MPI_COMM_WORLD, computeTask, rank, &splitComm) );

	if (isComputeTask())
	{
		MPI_Check( MPI_Comm_dup(splitComm, &compComm) );
		MPI_Check( MPI_Intercomm_create(compComm, 0, MPI_COMM_WORLD, 0, 0, &interComm) );

		sim = new Simulation(nranks3D, globalDomainSize, compComm, interComm);
	}
	else
	{
		MPI_Check( MPI_Comm_dup(splitComm, &ioComm) );
		MPI_Check( MPI_Intercomm_create(ioComm,   0, MPI_COMM_WORLD, 1, 0, &interComm) );

		post = new Postprocess(ioComm, interComm);
	}
}

bool uDeviceX::isComputeTask()
{
	return computeTask == 0;
}

void uDeviceX::registerJointPlugins(SimulationPlugin* simPl, PostprocessPlugin* postPl)
{
	const int id = pluginId++;

	if (isComputeTask())
	{
		simPl->setId(id);
		sim->registerPlugin(simPl);
	}
	else
	{
		postPl->setId(id);
		post->registerPlugin(postPl);
	}
}

void uDeviceX::run()
{
	if (computeTask)
		sim->run(100000);
	else
		post->run();

	if (computeTask)
	{
		CUDA_Check( hipDeviceSynchronize() );
		CUDA_Check( hipDeviceReset() );
	}

	MPI_Check( MPI_Finalize() );
}




