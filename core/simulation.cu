#include "hip/hip_runtime.h"
#include "simulation.h"

#include <algorithm>

Simulation::Simulation(int3 nranks3D, float3 globalDomainSize, const MPI_Comm& comm, const MPI_Comm& interComm) :
nranks3D(nranks3D), globalDomainSize(globalDomainSize), interComm(interComm), currentTime(0), currentStep(0)
{
	int ranksArr[] = {nranks3D.x, nranks3D.y, nranks3D.z};
	int periods[] = {1, 1, 1};
	int coords[3];

	MPI_Check( MPI_Comm_rank(comm, &rank) );
	MPI_Check( MPI_Cart_create(comm, 3, ranksArr, periods, 0, &cartComm) );
	MPI_Check( MPI_Cart_get(cartComm, 3, ranksArr, periods, coords) );
	rank3D = {coords[0], coords[1], coords[2]};

	localDomainSize = globalDomainSize / make_float3(nranks3D);
	globalDomainStart = {localDomainSize.x * coords[0], localDomainSize.y * coords[1], localDomainSize.z * coords[2]};

	restartFolder  = "./restart/";
	std::string command = "mkdir -p " + restartFolder;
	if (rank == 0)
	{
		if ( system(command.c_str()) != 0 )
		{
			error("Could not create folder for restart files, will try to use ./");
			restartFolder = "./";
		}
	}

	info("Simulation initialized, subdomain size is [%f %f %f], subdomain starts at [%f %f %f]",
			localDomainSize.x,  localDomainSize.y,  localDomainSize.z,
			globalDomainStart.x, globalDomainStart.y, globalDomainStart.z);
}

//================================================================================================
// Registration
//================================================================================================

void Simulation::registerParticleVector(ParticleVector* pv, InitialConditions* ic)
{
	std::string name = pv->name;
	particleVectors.push_back(pv);

	auto ov = dynamic_cast<ObjectVector*>(pv);
	if(ov != nullptr)
		objectVectors.push_back(ov);

	if (pvIdMap.find(name) != pvIdMap.end())
		die("More than one particle vector is called %s", name.c_str());

	if (wallMap.find(name) != wallMap.end())
		die("Particle vector cannot be called as another wall %s", name.c_str());

	pvIdMap[name] = particleVectors.size() - 1;
	ic->exec(cartComm, pv, globalDomainStart, localDomainSize, 0);
}

void Simulation::registerWall(Wall* wall)
{
	std::string name = wall->name;

	if (wallMap.find(name) != wallMap.end())
		die("More than one wall is called %s", name.c_str());

	if (pvIdMap.find(name) != pvIdMap.end())
		die("Wall cannot be called as another particle vector %s", name.c_str());

	wallMap[name] = wall;
	wall->createSdf(cartComm, globalDomainSize, globalDomainStart, localDomainSize);
}

void Simulation::registerInteraction(Interaction* interaction)
{
	std::string name = interaction->name;
	if (interactionMap.find(name) != interactionMap.end())
		die("More than one interaction is called %s", name.c_str());

	interactionMap[name] = interaction;
}

void Simulation::registerIntegrator(Integrator* integrator)
{
	std::string name = integrator->name;
	if (integratorMap.find(name) != integratorMap.end())
		die("More than one interaction is called %s", name.c_str());

	integratorMap[name] = integrator;
}

void Simulation::registerBouncer(Bouncer* bouncer)
{
	std::string name = bouncer->name;
	if (bouncerMap.find(name) != bouncerMap.end())
		die("More than one bouncer is called %s", name.c_str());

	bouncerMap[name] = bouncer;
}

void Simulation::registerPlugin(SimulationPlugin* plugin)
{
	plugins.push_back(plugin);
}

//================================================================================================
// Applying something to something else
//================================================================================================

void Simulation::setIntegrator(std::string integratorName, std::string pvName)
{
	if (integratorMap.find(integratorName) == integratorMap.end())
		die("No such integrator: %s", integratorName.c_str());

	auto pv = getPVbyName(pvName);
	if (pv == nullptr)
		die("No such particle vector: %s", pvName.c_str());

	integrator = integratorMap[integratorName];

	integratorsStage1.push_back([integrator, pv] (hipStream_t stream) {
		integrator->stage1(pv, stream);
	});

	integratorsStage2.push_back([integrator, pv] (hipStream_t stream) {
		integrator->stage1(pv, stream);
	});
}

void Simulation::setInteraction(std::string interactionName, std::string pv1Name, std::string pv2Name)
{
	auto pv1 = getPVbyName(pv1Name);
	if (pv1 == nullptr)
		die("No such particle vector: %s", pv1Name.c_str());

	auto pv2 = getPVbyName(pv2Name);
	if (pv2 == nullptr)
		die("No such particle vector: %s", pv2Name.c_str());

	if (interactionMap.find(interactionName) == interactionMap.end())
		die("No such integrator: %s", interactionName.c_str());
	auto interaction = interactionMap[interactionName];


	float rc = interaction->rc;
	interactionPrototypes.push_back(std::make_tuple(rc, pv1, pv2, interaction));
}

void Simulation::setBouncer(std::string bouncerName, std::string objName, std::string pvName)
{
	auto pv = getPVbyName(pvName);
	if (pv == nullptr)
		die("No such particle vector: %s", pvName.c_str());

	auto ov = dynamic_cast<ObjectVector*> (getPVbyName(objName));
	if (ov == nullptr)
		die("No such object vector: %s", objName.c_str());

	if (bouncerMap.find(bouncerName) == bouncerMap.end())
		die("No such bouncer: %s", bouncerName.c_str());
	auto bouncer = bouncerMap[bouncerName];

	bouncerPrototypes.push_back(std::make_tuple(bouncer, ov, pv));
}

void Simulation::setWallBounce(std::string wallName, std::string pvName)
{
	auto pv = getPVbyName(pvName);
	if (pv == nullptr)
		die("No such particle vector: %s", pvName.c_str());

	if (wallMap.find(wallName) == wallMap.end())
		die("No such wall: %s", wallName.c_str());
	auto wall = wallMap[wallName];

	wallPrototypes.push_back( {wall, pv} );
}


void Simulation::prepareCellLists()
{
	const float rcTolerance = 1e-4;

	std::map<ParticleVector*, std::vector<float>> cutOffMap;

	// Deal with the cell-lists and interactions
	for (auto prototype : interactionPrototypes)
	{
		float rc = std::get<0>(prototype);
		cutOffMap[std::get<1>(prototype)].push_back(rc);
		cutOffMap[std::get<2>(prototype)].push_back(rc);
	}

	for (auto& cutoffs : cutOffMap)
	{
		std::sort(cutoffs.second.begin(), cutoffs.second.end(), [] (float a, float b) { return a > b; });

		auto it = std::unique(cutoffs.second.begin(), cutoffs.second.end(), [=] (float a, float b) { return fabs(a - b) < rcTolerance; });
		cutoffs.second.resize( std::distance(cutoffs.second.begin(), it) );

		bool primary = true;

		// Don't use primary cell-lists with ObjectVectors
		if (dynamic_cast<ObjectVector*>(cutoffs.first) != nullptr)
			primary = false;

		for (auto rc : cutoffs.second)
		{
			cellListMap[cutoffs.first].push_back(first ?
					new PrimaryCellList(cutoffs.first, rc, localDomainSize) :
					new CellList       (cutoffs.first, rc, localDomainSize));
			primary = false;
		}
	}
}

void Simulation::prepareInteractions()
{
	for (auto prototype : interactionPrototypes)
	{
		float rc = std::get<0>(prototype);
		auto pv1 = std::get<1>(prototype);
		auto pv2 = std::get<2>(prototype);

		auto& clVec1 = cellListMap[pv1];
		auto& clVec2 = cellListMap[pv2];

		CellList *cl1, *cl2;

		for (auto cl : clVec1)
			if (fabs(cl->rc - rc) <= rcTolerance)
				cl1 = cl;

		for (auto cl : clVec2)
			if (fabs(cl->rc - rc) <= rcTolerance)
				cl2 = cl;

		auto inter = std::get<3>(prototype);

		regularInteractions.push_back([inter, pv1, pv2, cl1, cl2] (float t, hipStream_t stream) {
			inter->regular(pv1, pv2, cl1, cl2, t, stream);
		});

		haloInteractions.push_back([inter, pv1, pv2, cl1, cl2] (float t, hipStream_t stream) {
			inter->halo(pv1, pv2, cl1, cl2, t, stream);
		});
	}
}

void Simulation::prepareBouncers()
{
	for (auto prototype : bouncerPrototypes)
	{
		auto bouncer = std::get<0>(prototype);
		auto ov = std::get<1>(prototype);
		auto pv = std::get<2>(prototype);

		auto& clVec = cellListMap[pv];

		if (clVec.empty()) continue;

		CellList *cl = clVec[0];

		regularBouncers.push_back([bouncer, ov, pv, cl] (hipStream_t stream) {
			bouncer->bounceLocal(ov, pv, cl, stream);
		});

		haloBouncers.   push_back([bouncer, ov, pv, cl] (hipStream_t stream) {
			bouncer->bounceLocal(ov, pv, cl, stream);
		});
	}
}

void Simulation::prepareWalls()
{
	for (auto prototype : wallPrototypes)
	{
		auto wall = prototype.first;
		auto pv   = prototype.second;

		auto& clVec = cellListMap[pv];

		if (clVec.empty()) continue;

		CellList *cl = clVec[0];

		wall->attach(pv, cl);
	}

	for (auto pv : particleVectors)
		for (auto wall : wallMap)
			if (cellListMap[pv].size() > 0 && pv->name != wall.second->name)
				wall.second->removeInner(pv);
}

void Simulation::init()
{
	prepareCellLists();

	prepareInteractions();
	prepareBouncers();
	prepareWalls();

	debug("Simulation initiated, preparing plugins");
	for (auto& pl : plugins)
	{
		pl->setup(this, cartComm, interComm);
		pl->handshake();
	}

	halo = new ParticleHaloExchanger(cartComm);
	redistributor = new ParticleRedistributor(cartComm);

	debug("Attaching particle vectors to halo exchanger and redistributor");
	for (auto pv : particleVectors)
		if (cellListMap[pv].size() > 0)
			if (dynamic_cast<ObjectVector*>(pv) == nullptr)
			{
				auto cl = cellListMap[pv][0];

				halo->attach         (pv, cl);
				redistributor->attach(pv, cl);
			}
			else
			{
				auto cl = cellListMap[pv][0];
				auto ov = dynamic_cast<ObjectVector*>(pv);

				objHalo->attach        (ov, cl->rc);
				objRedistibutor->attach(ov, cl->rc);
			}

	assemble();
}

void Simulation::assemble()
{
	// XXX: different dt not implemented
	dt = 1.0;
	for (auto integr : integrators)
		if (integr != nullptr)
			dt = min(dt, integr->dt);


	scheduler.addTask("Сell-lists", [&] (hipStream_t stream) {
		for (auto clVec : cellListMap)
			for (auto cl : clVec.second)
				cl->build(stream);
	});

	scheduler.addTask("Clear forces", [&] (hipStream_t stream) {
		for (auto& pv : particleVectors)
			pv->local()->forces.clear(stream);
	});

	scheduler.addTask("Plugins: before forces", [&] (hipStream_t stream) {
		for (auto& pl : plugins)
			{
				pl->setTime(currentTime, currentStep);
				pl->beforeForces(stream);
			}
	});

	scheduler.addTask("Halo init", [&] (hipStream_t stream) {
		halo->init(stream);
	});

	scheduler.addTask("Internal forces", [&] (hipStream_t stream) {
		for (auto& inter : regularInteractions)
			inter(currentTime, stream);
	});

	scheduler.addTask("Plugins: serialize and send", [&] (hipStream_t stream) {
		for (auto& pl : plugins)
			pl->serializeAndSend(stream);
	});

	scheduler.addTask("Halo finalize", [&] (hipStream_t stream) {
		halo->finalize();
	});

	scheduler.addTask("Halo forces", [&] (hipStream_t stream) {
		for (auto& inter : haloInteractions)
			inter(currentTime, stream);
	});

	scheduler.addTask("Accumulate forces", [&] (hipStream_t stream) {
		for (auto clVec : cellListMap)
			for (auto cl : clVec.second)
				cl->addForces(stream);
	});

	scheduler.addTask("Plugins: before integration", [&] (hipStream_t stream) {
		for (auto& pl : plugins)
			pl->beforeIntegration(stream);
	});

	scheduler.addTask("Integration", [&] (hipStream_t stream) {
		for (int i=0; i<integrators.size(); i++)
			if (integrators[i] != nullptr)
				integrators[i]->stage2(particleVectors[i], stream);
	});


	scheduler.addTask("Object halo init", [&] (hipStream_t stream) {
		objHalo->init(stream);
	});
	scheduler.addTask("Object halo finalize", [&] (hipStream_t stream) {
		objHalo->finalize();
	});

	scheduler.addTask("Object bounce", [&] (hipStream_t stream) {
		for (auto bouncer : bouncers)
			bouncer.first->exec(dt, bouncer.second, stream);
	});

	scheduler.addTask("Obj forces exchange: init", [&] (hipStream_t stream) {
		objForceExchanger->init(stream);
	});

	scheduler.addTask("Obj forces exchange: finalize", [&] (hipStream_t stream) {
		objForceExchanger->finalize();
	});

	scheduler.addTask("Wall bounce", [&] (hipStream_t stream) {
		for (auto wall : wallMap)
		{
			wall.second->bounce(dt, stream);
			wall.second->check(stream);
		}
	});

	scheduler.addTask("Plugins: after integration", [&] (hipStream_t stream) {
		for (auto pl : plugins)
			pl->afterIntegration(stream);
	});

	scheduler.addTask("Redistribute init", [&] (hipStream_t stream) {
		redistributor->init(stream);
	});

	scheduler.addTask("Redistribute finalize", [&] (hipStream_t stream) {
		redistributor->finalize();
	});



	scheduler.addDependency("Сell-lists", {"Clear forces", "Halo init", "Object internal forces"}, {});

	scheduler.addDependency("Plugins: before forces", {"Internal forces", "Halo forces", "Object internal forces"}, {});
	scheduler.addDependency("Internal forces", {}, {"Clear forces"});
	scheduler.addDependency("Plugins: serialize and send", {}, {"Internal forces"});
	scheduler.addDependency("Halo init", {"Internal forces"}, {});
	scheduler.addDependency("Halo finalize", {}, {"Halo init"});
	scheduler.addDependency("Halo forces", {}, {"Halo finalize"});
	scheduler.addDependency("Accumulate forces", {"Integration"}, {"Halo forces", "Internal forces"});
	scheduler.addDependency("Plugins: before integration", {"Integration"}, {});

	scheduler.addDependency("Object halo init", {}, {"Integrate"});
	scheduler.addDependency("Object halo finalize", {}, {"Object halo init"});

	scheduler.addDependency("Object bounce", {}, {"Object halo finalize", "Object integration", "Integration"});
	scheduler.addDependency("Obj forces exchange: init", {"Redistribute init"}, {"Object bounce", "Object internal forces"});
	scheduler.addDependency("Obj forces exchange: finalize", {}, {"Obj forces exchange: init"});

	scheduler.addDependency("Plugins: after integration", {}, {"Integration", "Wall bounce", "Send obj forces"});
	scheduler.addDependency("Redistribute init", {}, {"Integration", "Wall bounce", "Send obj forces", "Plugins: after integration"});
	scheduler.addDependency("Redistribute finalize", {}, {"Redistribute init"});

	scheduler.setHighPriority("Object internal forces");
	scheduler.setHighPriority("Object halo init");
	scheduler.setHighPriority("Object halo finalize");
	scheduler.setHighPriority("Object halo forces");
	scheduler.setHighPriority("Object accumulate forces");
	scheduler.setHighPriority("Object integrate");

	scheduler.compile();
}

// TODO: wall has self-interactions
void Simulation::run(int nsteps)
{
	info("Will run %d iterations now", nsteps);
	int begin = currentStep, end = currentStep + nsteps;

	for (currentStep = begin; currentStep < end; currentStep++)
	{
		if (rank == 0)
			info("===============================================================================\nTimestep: %d, simulation time: %f",
					currentStep, currentTime);

		scheduler.run();

		currentTime += dt;
	}

	// Finish the redistribution by rebuilding the primary cell-lists
	for (auto clVec : cellListMap)
		if (clVec.second.size() > 0)
			clVec.second[0]->build(0);

	info("Finished with %d iterations", nsteps);
}

void Simulation::finalize()
{
	MPI_Check( MPI_Barrier(cartComm) );

	debug("Finished, exiting now");

	if (interComm != MPI_COMM_NULL)
	{
		int dummy = -1;
		int tag = 424242;

		MPI_Request req;
		MPI_Check( MPI_Isend(&dummy, 1, MPI_INT, rank, tag, interComm, &req) );
	}
}



