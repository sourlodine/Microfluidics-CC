#include "hip/hip_runtime.h"
#include <core/dpd-rng.h>
#include <core/particle_vector.h>
#include <core/interaction_engine.h>
#include <core/hip/hip_vector_types.h>
#include <core/interactions.h>
#include <core/cuda_common.h>
#include <core/object_vector.h>
#include <core/rbc_vector.h>

//==================================================================================================================
// Interaction wrapper macro
//==================================================================================================================

#define WRAP_INTERACTON(INTERACTION_FUNCTION)                                                                                                                         \
	if (type == InteractionType::Regular)                                                                                                                             \
	{                                                                                                                                                                 \
		/*  Self interaction */                                                                                                                                       \
		if (pv1 == pv2)                                                                                                                                               \
		{                                                                                                                                                             \
			debug2("Computing internal forces for %s (%d particles)", pv1->name.c_str(), pv1->local()->size());                                                       \
                                                                                                                                                                      \
			const int nth = 128;                                                                                                                                      \
			if (pv1->local()->size() > 0)                                                                                                                             \
				computeSelfInteractions<<< (pv1->local()->size() + nth - 1) / nth, nth, 0, stream >>>(                                                                \
						pv1->local()->size(), (float4*)cl->coosvels->devPtr(), (float*)cl->forces->devPtr(),                                                          \
						cl->cellInfo(), cl->cellsStartSize.devPtr(), rc*rc, INTERACTION_FUNCTION);                                                                    \
		}                                                                                                                                                             \
		else /*  External interaction */                                                                                                                              \
		{                                                                                                                                                             \
			debug2("Computing external forces for %s - %s (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), pv1->local()->size(), pv2->local()->size());    \
                                                                                                                                                                      \
			const int nth = 128;                                                                                                                                      \
			if (pv1->local()->size() > 0 && pv2->local()->size() > 0)                                                                                                 \
				computeExternalInteractions<true, true, true> <<< (pv2->local()->size() + nth - 1) / nth, nth, 0, stream >>>(                                         \
						pv2->local()->size(),                                                                                                                         \
						(float4*)pv2->local()->coosvels.devPtr(), (float*)pv2->local()->forces.devPtr(),                                                              \
						(float4*)cl->coosvels->devPtr(), (float*)cl->forces->devPtr(),                                                                                \
						cl->cellInfo(), cl->cellsStartSize.devPtr(),                                                                                                  \
						rc*rc, INTERACTION_FUNCTION);                                                                                                                 \
		}                                                                                                                                                             \
	}                                                                                                                                                                 \
                                                                                                                                                                      \
	/*  Halo interaction */                                                                                                                                           \
	if (type == InteractionType::Halo)                                                                                                                                \
	{                                                                                                                                                                 \
		debug2("Computing halo forces for %s - %s(halo) (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), pv1->local()->size(), pv2->halo()->size());       \
                                                                                                                                                                      \
		const int nth = 128;                                                                                                                                          \
		if (pv1->local()->size() > 0 && pv2->halo()->size() > 0)                                                                                                      \
			computeExternalInteractions<false, true, false> <<< (pv2->halo()->size() + nth - 1) / nth, nth, 0, stream >>>(                                            \
					pv2->halo()->size(),                                                                                                                              \
					(float4*)pv2->halo()->coosvels.devPtr(), nullptr,                                                                                                 \
					(float4*)cl->coosvels->devPtr(), (float*)cl->forces->devPtr(),                                                                                    \
					cl->cellInfo(), cl->cellsStartSize.devPtr(),                                                                                                      \
					rc*rc, INTERACTION_FUNCTION);                                                                                                                     \
	}


//==================================================================================================================
// DPD interactions
//==================================================================================================================

inline __device__ float viscosityKernel(const float x, const float k)
{
	if (fabs(k - 1.0f)   < 1e-6f) return x;
	if (fabs(k - 0.5f)   < 1e-6f) return sqrtf(fabs(x));
	if (fabs(k - 0.25f)  < 1e-6f) return sqrtf(fabs(sqrtf(fabs(x))));
	if (fabs(k - 0.125f) < 1e-6f) return sqrtf(fabs(sqrtf(fabs(sqrtf(fabs(x))))));

    return powf(fabs(x), k);
}

__device__ __forceinline__ float3 pairwiseDPD(
		Particle dst, Particle src,
		const float adpd, const float gammadpd, const float sigmadpd,
		const float rc2, const float invrc, const float k, const float seed)
{
	const float3 dr = dst.r - src.r;
	const float rij2 = dot(dr, dr);
	if (rij2 > rc2) return make_float3(0.0f);

	const float invrij = rsqrtf(max(rij2, 1e-20f));
	const float rij = rij2 * invrij;
	const float argwr = 1.0f - rij*invrc;
	const float wr = viscosityKernel(argwr, k);

	const float3 dr_r = dr * invrij;
	const float3 du = dst.u - src.u;
	const float rdotv = dot(dr_r, du);

	const float myrandnr = Logistic::mean0var1(seed, min(src.i1, dst.i1), max(src.i1, dst.i1));

	const float strength = adpd * argwr - (gammadpd * wr * rdotv + sigmadpd * myrandnr) * wr;

	return dr_r * strength;
}


//==================================================================================================================
// LJ interactions
//==================================================================================================================

__device__ inline float3 pairwiseLJ(Particle dst, Particle src, const float sigma, const float epsx24_sigma, const float rc2)
{
	const float3 dr = dst.r - src.r;
	const float rij2 = dot(dr, dr);

	if (rij2 > rc2) return make_float3(0.0f);

	const float rs2 = sigma*sigma / rij2;
	const float rs4 = rs2*rs2;
	const float rs8 = rs4*rs4;
	const float rs14 = rs8*rs4*rs2;

	return dr * epsx24_sigma * (2*rs14 - rs8);
}

__device__ inline float3 pairwiseLJ_objectAware(Particle dst, Particle src,
		bool isDstObj, float3 dstCom,
		bool isSrcObj, float3 srcCom,
		const float sigma, const float epsx24_sigma, const float rc2)
{
	const float3 dr = dst.r - src.r;

	const bool dstSide = dot(dr, dst.r-dstCom) < 0.0f;
	const bool srcSide = dot(dr, srcCom-src.r) < 0.0f;

	if (dstSide && (!isSrcObj)) return make_float3(0.0f);
	if ((!isDstObj) && srcSide) return make_float3(0.0f);
	if (dstSide && srcSide)     return make_float3(0.0f);

	return pairwiseLJ(dst, src, sigma, epsx24_sigma, rc2);
}

//==================================================================================================================
//==================================================================================================================


/**
 * Regular DPD interaction
 */
InteractionDPD::InteractionDPD(pugi::xml_node node)
{
	name = node.attribute("name").as_string("");
	rc   = node.attribute("rc").as_float(1.0f);

	power = node.attribute("power").as_float(1.0f);
	a     = node.attribute("a")    .as_float(50);
	gamma = node.attribute("gamma").as_float(20);

	const float dt  = node.attribute("dt") .as_float(0.01);
	const float kBT = node.attribute("kbt").as_float(1.0);

	sigma = sqrt(2 * gamma * kBT / dt);
}

void InteractionDPD::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream)
{
	// Better to use random number in the seed instead of periodically changing time
	const float seed = drand48();
	const float rc2 = rc*rc;
	const float rc_1 = 1.0 / rc;
	auto dpdCore = [=, *this] __device__ ( Particle dst, Particle src ) {
		return pairwiseDPD( dst, src, a, gamma, sigma, rc2, rc_1, power, seed);
	};

	WRAP_INTERACTON(dpdCore)
}

/**
 * LJ interaction, to prevent overlap of the rigid objects
 */
InteractionLJ_objectAware::InteractionLJ_objectAware(pugi::xml_node node)
{
	name = node.attribute("name").as_string("");
	rc   = node.attribute("rc").as_float(1.0f);

	epsilon = node.attribute("power").as_float(10.0f);
	sigma   = node.attribute("a")    .as_float(0.5f);
}

void InteractionLJ_objectAware::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream)
{
	auto ov1 = dynamic_cast<ObjectVector*>(pv1);
	auto ov2 = dynamic_cast<ObjectVector*>(pv2);
	if (ov1 == nullptr && ov2 == nullptr)
		die("Object-aware LJ interaction can only be used with objects");

	const float epsx24_sigma = 24.0*epsilon/sigma;
	const float rc2 = rc*rc;
	const bool self = (pv1 == pv2);

	const LocalObjectVector::COMandExtent* dstComExt = (ov1 != nullptr) ? ov1->local()->comAndExtents.devPtr() : nullptr;
	const LocalObjectVector::COMandExtent* srcComExt = (ov2 != nullptr) ? ov2->local()->comAndExtents.devPtr() : nullptr;

	auto ljCore = [=, *this] __device__ ( Particle dst, Particle src ) {
		const int dstObjId = dst.s21;
		const int srcObjId = src.s21;

		if (dstObjId == srcObjId && self) return make_float3(0.0f);

		float3 dstCom = make_float3(0.0f);
		float3 srcCom = make_float3(0.0f);
		if (dstComExt != nullptr) dstCom = dstComExt[dstObjId].com;
		if (srcComExt != nullptr) srcCom = srcComExt[srcObjId].com;

		return pairwiseLJ_objectAware( dst, src, (dstComExt != nullptr), dstCom, (srcComExt != nullptr), srcCom, sigma, epsx24_sigma, rc2);
	};

	WRAP_INTERACTON(ljCore)
}


void InteractionRBCMembrane::_compute(InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream)
{
	if (pv1 != pv2)
		die("Internal RBC forces can't be computed between two different particle vectors");

	auto rbcv = dynamic_cast<RBCvector*>(pv1);
	if (rbcv == nullptr)
		die("Internal RBC forces can only be computed with RBC object vector");

	int nthreads = 128;
	int nRbcs  = rbcv->local()->nObjects;
	int nVerts = rbcv->mesh.nvertices;


	dim3 avThreads(256, 1);
	dim3 avBlocks( 1, nRbcs );
//	computeAreaAndVolume <<< avBlocks, avThreads, 0, stream >>> (
//			(float4*)rbcv->local()->coosvels.devPtr(), rbcv->local()->mesh, nRbcs,
//			rbcv->local()->areas.devPtr(), rbcv->local()->volumes.devPtr());

	int blocks = getNblocks(nRbcs*nVerts*rbcv->mesh.maxDegree, nthreads);

//	computeMembraneForces <<<blocks, nthreads, 0, stream>>> (
//			(float4*)rbcv->local()->coosvels.devPtr(), rbcv->local()->mesh, nRbcs,
//			rbcv->local()->areas.devPtr(), rbcv->local()->volumes.devPtr(),
//			(float4*)rbcv->local()->forces.devPtr());
}











