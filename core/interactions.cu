#include "hip/hip_runtime.h"
#include "dpd-rng.h"
#include "containers.h"
#include <core/interaction_engine.h>
#include <core/hip/hip_vector_types.h>

//==================================================================================================================
// DPD interactions
//==================================================================================================================

template<int s>
inline __device__ float viscosity_function(float x)
{
    return sqrtf(viscosity_function<s - 1>(x));
}

template<> inline __device__ float viscosity_function<1>(float x) { return sqrtf(max(x, 1e-20f)); }
template<> inline __device__ float viscosity_function<0>(float x){ return x; }

__device__ __forceinline__ float3 dpd_interaction(
		const float3 dstCoo, const float3 dstVel, const int dstId,
		const float3 srcCoo, const float3 srcVel, const int srcId,
		const float adpd, const float gammadpd, const float sigmadpd,
		const float rc2, const float invrc, const float seed)
{
	const float3 dr = dstCoo - srcCoo;
	const float rij2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z; // dot(dr, dr)
	if (rij2 > rc2) return make_float3(0.0f);

	const float invrij = rsqrtf(max(rij2, 1e-20f));
	const float rij = rij2 * invrij;
	const float argwr = 1.0f - rij*invrc;
	const float wr = viscosity_function<0>(argwr);

	const float3 dr_r = dr * invrij;
	const float rdotv = dot(dr_r, (dstVel - srcVel));

	const float myrandnr = Logistic::mean0var1(seed, min(srcId, dstId), max(srcId, dstId));

	const float strength = adpd * argwr - (gammadpd * wr * rdotv + sigmadpd * myrandnr) * wr;

	return dr_r * strength;
}


void interactionDPDSelf (ParticleVector* pv, CellList* cl, const float t, hipStream_t stream,
		float adpd, float gammadpd, float sigma_dt, float rc)
{
	auto dpdCore = [=] __device__ ( const float4 dstCoo, const float4 dstVel, const int dstId,
									const float4 srcCoo, const float4 srcVel, const int srcId)
	{
		return dpd_interaction( make_float3(dstCoo), make_float3(dstVel), dstId,
								make_float3(srcCoo), make_float3(srcVel), srcId,
								adpd, gammadpd, sigma_dt, rc*rc, 1.0f/rc, t);
	};

	const int nth = 32 * 4;

	if (pv->np > 0)
	{
		debug2("Computing internal forces for %s (%d particles)", pv->name.c_str(), pv->np);
		computeSelfInteractions<<< (pv->np + nth - 1) / nth, nth, 0, stream >>>(
				(float4*)pv->coosvels.devPtr(), (float*)pv->forces.devPtr(), cl->cellInfo(), cl->cellsStart.devPtr(), pv->np, dpdCore);
	}
}

void interactionDPDHalo (ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream,
		float adpd, float gammadpd, float sigma_dt, float rc)
{
	auto dpdCore = [=] __device__ ( const float4 dstCoo, const float4 dstVel, const int dstId,
									const float4 srcCoo, const float4 srcVel, const int srcId)
	{
		return dpd_interaction( make_float3(dstCoo), make_float3(dstVel), dstId,
								make_float3(srcCoo), make_float3(srcVel), srcId,
								adpd, gammadpd, sigma_dt, rc*rc, 1.0f/rc, t);
	};

	const int nth = 32 * 4;

	if (pv1->np > 0 && pv2->np > 0)
	{
		debug2("Computing halo forces for %s - %s(halo) (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), pv1->np, pv2->halo.size());
		computeExternalInteractions<false, true> <<< (pv2->halo.size() + nth - 1) / nth, nth, 0, stream >>>(
									(float4*)pv2->halo.devPtr(), nullptr, (float4*)pv1->coosvels.devPtr(),
									(float*)pv1->forces.devPtr(), cl->cellInfo(), cl->cellsStart.devPtr(), pv2->halo.size(), dpdCore);
	}
}

void interactionDPDExternal (ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream,
		float adpd, float gammadpd, float sigma_dt, float rc)
{
	auto dpdCore = [=] __device__ ( const float4 dstCoo, const float4 dstVel, const int dstId,
									const float4 srcCoo, const float4 srcVel, const int srcId)
	{
		return dpd_interaction( make_float3(dstCoo), make_float3(dstVel), dstId,
								make_float3(srcCoo), make_float3(srcVel), srcId,
								adpd, gammadpd, sigma_dt, rc*rc, 1.0f/rc, t);
	};

	const int nth = 32 * 4;

	if (pv1->np > 0 && pv2->np > 0)
	{
		debug2("Computing external forces for %s - %s (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), pv1->np, pv2->np);
		computeExternalInteractions<true, true> <<< (pv2->np + nth - 1) / nth, nth, 0, stream >>>(
									(float4*)pv2->coosvels.devPtr(), nullptr, (float4*)pv1->coosvels.devPtr(),
									(float*)pv1->forces.devPtr(), cl->cellInfo(), cl->cellsStart.devPtr(), pv2->np, dpdCore);
	}
}
