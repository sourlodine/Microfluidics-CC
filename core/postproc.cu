#include "postproc.h"

#include <core/logger.h>

Postprocess::Postprocess(MPI_Comm& comm, MPI_Comm& interComm) : comm(comm), interComm(interComm)
{
	info("Postprocessing initialized");
}

void Postprocess::registerPlugin(PostprocessPlugin* plugin)
{
	info("New plugin registered: %s", plugin->name.c_str());
	plugins.push_back(plugin);
}

void Postprocess::run()
{
	for (auto& pl : plugins)
	{
		debug("Setup and handshake of %s", pl->name.c_str());
		pl->setup(comm, interComm);
		pl->handshake();
	}

	// Stopping condition
	int dummy = 0;
	int tag = 424242;
	int rank;

	MPI_Check( MPI_Comm_rank(comm, &rank) );

	MPI_Request endReq;
	MPI_Check( MPI_Irecv(&dummy, 1, MPI_INT, rank, tag, interComm, &endReq) );

	std::vector<MPI_Request> requests;
	for (auto& pl : plugins)
		requests.push_back(pl->waitData());
	requests.push_back(endReq);

	while (true)
	{
		int index;
		MPI_Status stat;
		MPI_Check( MPI_Waitany(requests.size(), requests.data(), &index, &stat) );

		if (index == plugins.size())
		{
			if (dummy != -1)
				die("Something went terribly wrong");

			info("Postprocess got a stopping message and will exit now");
			break;
		}

		debug2("Postprocess got a request from plugin %s, executing now", plugins[index]->name.c_str());
		plugins[index]->recv();
		plugins[index]->deserialize(stat);
		requests[index] = plugins[index]->waitData();
	}
}

