#include "hip/hip_runtime.h"
#include <fstream>
#include <cmath>
#include <texture_types.h>

#include <core/hip/hip_vector_types.h>
#include <core/wall.h>
#include <core/celllist.h>
#include <core/particle_vector.h>
#include <core/bounce.h>


// This should be in hip/hip_vector_types.h, but not there for some reason
//***************************************************************
inline __host__ __device__ int3 operator%(int3 a, int3 b)
{
    return make_int3(a.x % b.x, a.y % b.y, a.z % b.z);
}

inline __host__ __device__ int3 operator/(int3 a, int b)
{
    return make_int3(a.x / b, a.y / b, a.z / b);
}

//***************************************************************


__device__ __forceinline__ float cubicInterpolate1D(float y[4], float mu)
{
	// mu == 0 at y[1], mu == 1 at y[2]
	const float a0 = y[3] - y[2] - y[0] + y[1];
	const float a1 = y[0] - y[1] - a0;
	const float a2 = y[2] - y[0];
	const float a3 = y[1];

	return ((a0*mu + a1)*mu + a2)*mu + a3;
}


__global__ void cubicInterpolate3D(const float* in, int3 inDims, float3 inH, float* out, int3 outDims, float3 outH, float3 offset, float scalingFactor)
{
	// Inspired by http://paulbourke.net/miscellaneous/interpolation/
	// Center of the output domain is in offset
	// Center of the input domain is in (0,0,0)

	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	const int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= outDims.x || iy >= outDims.y || iz >= outDims.z) return;

	float interp2D[4][4];
	float interp1D[4];

	// Coordinates where to interpolate
	float3 outputId  = make_float3(ix, iy, iz);
	float3 outputCoo = outputId*outH;

	float3 inputCoo  = outputCoo + offset;

	// Make sure we're within the region where the the input data is defined
	assert( 0.0f <= inputCoo.x && inputCoo.x <= inDims.x*inH.x &&
			0.0f <= inputCoo.y && inputCoo.y <= inDims.y*inH.y &&
			0.0f <= inputCoo.z && inputCoo.z <= inDims.z*inH.z    );

	// Reference point of the original grid, rounded down
	int3 inputId_down = make_int3( floorf(inputCoo / inH) );
	float3 mu = (inputCoo - make_float3(inputId_down)*inH) / inH;

	// Interpolate along x
	for (int dz = -1; dz <= 2; dz++)
		for (int dy = -1; dy <= 2; dy++)
		{
			float vals[4];

			for (int dx = -1; dx <= 2; dx++)
			{
				int3 delta{dx, dy, dz};
				const int3 curInputId = (inputId_down+delta + inDims) % inDims;

				vals[dx+1] = in[ (curInputId.z*inDims.y + curInputId.y) * inDims.x + curInputId.x ] * scalingFactor;
			}

			interp2D[dz+1][dy+1] = cubicInterpolate1D(vals, mu.x);
		}

	// Interpolate along y
	for (int dz = 0; dz <= 3; dz++)
		interp1D[dz] = cubicInterpolate1D(interp2D[dz], mu.y);

	// Interpolate along z
	out[ (iz*outDims.y + iy) * outDims.x + ix ] = cubicInterpolate1D(interp1D, mu.z);
}

template<typename T>
__device__ __forceinline__ float evalSdf(T x, Wall::SdfInfo sdfInfo)
{
	float3 x3{x.x, x.y, x.z};
	float3 texcoord = floorf((x3 + sdfInfo.extendedDomainSize*0.5f) * sdfInfo.invh);
	float3 lambda = (x3 - (texcoord * sdfInfo.h - sdfInfo.extendedDomainSize*0.5f)) * sdfInfo.invh;

	const float s000 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 0, texcoord.y + 0, texcoord.z + 0);
	const float s001 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 1, texcoord.y + 0, texcoord.z + 0);
	const float s010 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 0, texcoord.y + 1, texcoord.z + 0);
	const float s011 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 1, texcoord.y + 1, texcoord.z + 0);
	const float s100 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 0, texcoord.y + 0, texcoord.z + 1);
	const float s101 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 1, texcoord.y + 0, texcoord.z + 1);
	const float s110 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 0, texcoord.y + 1, texcoord.z + 1);
	const float s111 = tex3D<float>(sdfInfo.sdfTex, texcoord.x + 1, texcoord.y + 1, texcoord.z + 1);

	const float s00x = s000 * (1 - lambda.x) + lambda.x * s001;
	const float s01x = s010 * (1 - lambda.x) + lambda.x * s011;
	const float s10x = s100 * (1 - lambda.x) + lambda.x * s101;
	const float s11x = s110 * (1 - lambda.x) + lambda.x * s111;

	const float s0yx = s00x * (1 - lambda.y) + lambda.y * s01x;
	const float s1yx = s10x * (1 - lambda.y) + lambda.y * s11x;

	const float szyx = s0yx * (1 - lambda.z) + lambda.z * s1yx;

//	printf("[%f %f %f]  [%f %f %f]  [%f %f %f]  = %f  vs  %f\n", x.x, x.y, x.z,  texcoord.x, texcoord.y, texcoord.z,
//			lambda.x, lambda.y, lambda.z, szyx, sqrt(x.x*x.x + x.y*x.y + x.z*x.z) - 5);

	return szyx;
}

// warp-aggregated atomic increment
// https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
// just4fun
__device__ __forceinline__ int atomicAggInc(int *ctr)
{
	int lane_id = (threadIdx.x % 32);

	int mask = __ballot(1);
	// select the leader
	int leader = __ffs(mask) - 1;
	// leader does the update
	int res;
	if(lane_id == leader)
	res = atomicAdd(ctr, __popc(mask));
	// broadcast result
	res = __shfl(res, leader);
	// each thread computes its own value
	return res + __popc(mask & ((1 << lane_id) - 1));
}


__global__ void countFrozen(const float4* pv, const int np, Wall::SdfInfo sdfInfo, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coo = pv[2*pid];

	const float sdf = evalSdf(coo, sdfInfo);

	if (sdf > 0.0f && sdf < 1.2f)
	{
		atomicAggInc(nFrozen);
	}
}

__global__ void collectFrozen(const float4* input, const int np, Wall::SdfInfo sdfInfo,
		float4* remaining, float4* frozen, int* nRemaining, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coo = input[2*pid];
	const float4 vel = input[2*pid+1];

	const float sdf = evalSdf(coo, sdfInfo);

	if (sdf <= 0.0f)
	{
		const int ind = atomicAggInc(nRemaining);
		remaining[2*ind] = coo;
		remaining[2*ind + 1] = vel;
	}

	if (sdf > 0.0f && sdf < 1.2f)
	{
		const int ind = atomicAggInc(nFrozen);
		frozen[2*ind] = coo;
		frozen[2*ind + 1] = make_float4(0.0f, 0.0f, 0.0f, vel.w);
	}
}

__device__ inline bool isCellOnBoundary(float3 cornerCoo, float3 len, Wall::SdfInfo sdfInfo)
{
	// About maximum distance a particle can cover in one step
	const float tol = 0.5f;

#pragma unroll
	for (int i=0; i<2; i++)
#pragma unroll
		for (int j=0; j<2; j++)
#pragma unroll
			for (int k=0; k<2; k++)
			{
				// Value in the cell corner
				const float3 shift = make_float3(i ? len.x : 0.0f, j ? len.y : 0.0f, k ? len.z : 0.0f);
				const float s = evalSdf( cornerCoo + shift,  sdfInfo );

				if (-1.0f - tol < s && s < 0.0f + tol)
					return true;
			}

	return false;
}

__global__ void countBoundaryCells(CellListInfo cinfo, Wall::SdfInfo sdfInfo, int* nBoundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;

	int3 ind;
	cinfo.decode(cid, ind.x, ind.y, ind.z);
	float3 cornerCoo = -0.5f*cinfo.domainSize + make_float3(ind)*cinfo.h;

	if (isCellOnBoundary(cornerCoo, cinfo.h, sdfInfo))
		atomicAggInc(nBoundaryCells);
}

__global__ void getBoundaryCells(CellListInfo cinfo, Wall::SdfInfo sdfInfo,
		int* nBoundaryCells, int* boundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;

	int3 ind;
	cinfo.decode(cid, ind.x, ind.y, ind.z);
	float3 cornerCoo = -0.5f*cinfo.domainSize + make_float3(ind)*cinfo.h;

	if (isCellOnBoundary(cornerCoo, cinfo.h, sdfInfo))
	{
		int id = atomicAggInc(nBoundaryCells);
		boundaryCells[id] = cid;
	}
}

__launch_bounds__(128, 8)
__global__ void bounceSDF(const int* wallCells, const int nWallCells, const uint* __restrict__ cellsStartSize, CellListInfo cinfo,
		Wall::SdfInfo sdfInfo, float4* coosvels, const float dt)
{
	const auto F = [sdfInfo] (const float3 r) {
		return evalSdf(r, sdfInfo);
	};

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nWallCells) return;
	const int cid = wallCells[tid];

	const int2 startSize = cinfo.decodeStartSize(cellsStartSize[cid]);

	for (int pid = startSize.x; pid < startSize.x + startSize.y; pid++)
	{
		float va, vb;

		float3 coo = make_float3(coosvels[2*pid]);
		float3 vel = make_float3(coosvels[2*pid+1]);

		const float alpha = bounceLinSearch(coo, vel, dt, F);

		// Just place the particle almost onto the surface and reverse the velocity
		float beta = alpha - 1e-6f;
		float4 candidate = oldCoo + beta * (coo - oldCoo);

		coosvels[2*pid] = candidate;
		coosvels[2*pid + 1] = -vel;
	}
}

/*
 * We only set a few params here
 */
Wall::Wall(std::string name, std::string sdfFileName, float3 sdfH,  float _creationTime) :
		name(name), sdfFileName(sdfFileName), _creationTime(_creationTime)
{
	sdfInfo.h = sdfH;
	frozen = new ParticleVector(name);
}

void Wall::attach(ParticleVector* pv, CellList* cl)
{
	CUDA_Check( hipDeviceSynchronize() );
	particleVectors.push_back(pv);
	cellLists.push_back(cl);

	const int oldSize = nBoundaryCells.size();
	boundaryCells.resize(oldSize+1);

	nBoundaryCells.resize(oldSize+1);
	nBoundaryCells.hostPtr()[oldSize] = 0;
	nBoundaryCells.uploadToDevice();
	countBoundaryCells<<< (cl->totcells + 127) / 128, 128 >>> (cl->cellInfo(), sdfInfo, nBoundaryCells.devPtr()+oldSize);
	nBoundaryCells.downloadFromDevice();

	debug("Found %d boundary cells", nBoundaryCells.hostPtr()[oldSize]);
	boundaryCells[oldSize].resize(nBoundaryCells.hostPtr()[oldSize]);

	nBoundaryCells.hostPtr()[oldSize] = 0;
	nBoundaryCells.uploadToDevice();
	getBoundaryCells<<< (cl->totcells + 127) / 128, 128 >>> (cl->cellInfo(), sdfInfo,
			nBoundaryCells.devPtr()+oldSize, boundaryCells[oldSize].devPtr());
	CUDA_Check( hipDeviceSynchronize() );
}

void Wall::readHeader(int3& sdfResolution, float3& sdfExtent, int64_t& fullSdfSize_byte, int64_t& endHeader_byte, int rank)
{
	if (rank == 0)
	{
		//printf("'%s'\n", sdfFileName.c_str());
		std::ifstream file(sdfFileName);
		if (!file.good())
			die("File not found or not accessible");

		auto fstart = file.tellg();

		file >> sdfExtent.x >> sdfExtent.y >> sdfExtent.z >>
			sdfResolution.x >> sdfResolution.y >> sdfResolution.z;
		fullSdfSize_byte = (int64_t)sdfResolution.x * sdfResolution.y * sdfResolution.z * sizeof(float);

		info("Using wall file '%s' of size %.2fx%.2fx%.2f and resolution %dx%dx%d", sdfFileName.c_str(),
				sdfExtent.x, sdfExtent.y, sdfExtent.z,
				sdfResolution.x, sdfResolution.y, sdfResolution.z);

		file.seekg( 0, std::ios::end );
		auto fend = file.tellg();

		endHeader_byte = (fend - fstart) - fullSdfSize_byte;

		file.close();
	}

	MPI_Check( MPI_Bcast(&sdfExtent,        3, MPI_FLOAT,     0, wallComm) );
	MPI_Check( MPI_Bcast(&sdfResolution,    3, MPI_INT,       0, wallComm) );
	MPI_Check( MPI_Bcast(&fullSdfSize_byte, 1, MPI_INT64_T,   0, wallComm) );
	MPI_Check( MPI_Bcast(&endHeader_byte,   1, MPI_INT64_T,   0, wallComm) );
}

void Wall::readSdf(int64_t fullSdfSize_byte, int64_t endHeader_byte, int nranks, int rank, std::vector<float>& fullSdfData)
{
	// Read part and allgather
	const int64_t readPerProc_byte = (fullSdfSize_byte + nranks - 1) / (int64_t)nranks;
	std::vector<char> readBuffer(readPerProc_byte);

	// Limits in bytes
	const int64_t readStart = readPerProc_byte * rank + endHeader_byte;
	const int64_t readEnd   = std::min( readStart + readPerProc_byte, fullSdfSize_byte + endHeader_byte);

	MPI_File fh;
	MPI_Status status;
	MPI_Check( MPI_File_open(wallComm, sdfFileName.c_str(), MPI_MODE_RDONLY, MPI_INFO_NULL, &fh) );  // TODO: MPI_Info
	MPI_Check( MPI_File_read_at_all(fh, readStart, readBuffer.data(), readEnd - readStart, MPI_BYTE, &status) );
	// TODO: check that we read just what we asked
	// MPI_Get_count only return int though

	fullSdfData.resize(readPerProc_byte * nranks / sizeof(float));  // May be bigger than fullSdfSize, to make gather easier
	MPI_Check( MPI_Allgather(readBuffer.data(), readPerProc_byte, MPI_BYTE, fullSdfData.data(), readPerProc_byte, MPI_BYTE, wallComm) );
}

void Wall::prepareRelevantSdfPiece(const float* fullSdfData, float3 extendedDomainStart, float3 initialSdfH, int3 initialSdfResolution,
		int3& resolution, float3& offset, PinnedBuffer<float>& localSdfData)
{
	// Find your relevant chunk of data
	// We cannot send big sdf files directly, so we'll carve a piece now

	const int margin = 3; // +2 from cubic interpolation, +1 from possible round-off errors
	const int3 startId = make_int3( floorf( extendedDomainStart                             / initialSdfH) ) - margin;
	const int3 endId   = make_int3( ceilf ((extendedDomainStart+sdfInfo.extendedDomainSize) / initialSdfH) ) + margin;

	float3 startInLocalCoord = make_float3(startId)*initialSdfH - (extendedDomainStart + 0.5*sdfInfo.extendedDomainSize);
	offset = -0.5*sdfInfo.extendedDomainSize - startInLocalCoord;

	int rank;
	MPI_Check( MPI_Comm_rank(wallComm, &rank) );
//	printf("%d:  SDstart [%f %f %f]  sdfH [%f %f %f] startId [%d %d %d], endId [%d %d %d], localstart [%f %f %f]\n",
//				rank,
//				extendedDomainStart.x, extendedDomainStart.y, extendedDomainStart.z,
//				initialSdfH.x, initialSdfH.y, initialSdfH.z,
//				startId.x, startId.y, startId.z,
//				endId.x, endId.y, endId.z,
//				startInLocalCoord.x, startInLocalCoord.y, startInLocalCoord.z);

	resolution = endId - startId;

	localSdfData.resize( resolution.x * resolution.y * resolution.z );
	auto locSdfDataPtr = localSdfData.hostPtr();

//	printf("%d:  input [%d %d %d], initial [%d %d %d], start [%d %d %d]\n",
//			rank, resolution.x, resolution.y, resolution.z,
//			initialSdfResolution.x, initialSdfResolution.y, initialSdfResolution.z,
//			startId.x, startId.y, startId.z);

//#warning "Minus here should be removed"
	for (int k = 0; k < resolution.z; k++)
		for (int j = 0; j < resolution.y; j++)
			for (int i = 0; i < resolution.x; i++)
			{
				const int origIx = (i+startId.x + initialSdfResolution.x) % initialSdfResolution.x;
				const int origIy = (j+startId.y + initialSdfResolution.y) % initialSdfResolution.y;
				const int origIz = (k+startId.z + initialSdfResolution.z) % initialSdfResolution.z;

				// FIXME: AAAAAAAAAAAAAAAAA MINUS
				locSdfDataPtr[ (k*resolution.y + j)*resolution.x + i ] =
						fullSdfData[ (origIz*initialSdfResolution.y + origIy)*initialSdfResolution.x + origIx ];
			}
}

void Wall::createSdf(MPI_Comm& comm, float3 subDomainStart, float3 subDomainSize, float3 globalDomainSize)
{
	debug2("Creating wall");

	CUDA_Check( hipDeviceSynchronize() );
	MPI_Check( MPI_Comm_dup(comm, &wallComm) );

	int nranks, rank;
	int ranks[3], periods[3], coords[3];
	MPI_Check( MPI_Comm_size(wallComm, &nranks) );
	MPI_Check( MPI_Comm_rank(wallComm, &rank) );
	MPI_Check( MPI_Cart_get (wallComm, 3, ranks, periods, coords) );

	int3 initialSdfResolution;
	float3 initialSdfExtent;

	int64_t fullSdfSize_byte;
	int64_t endHeader_byte;

	// Read header
	readHeader(initialSdfResolution, initialSdfExtent, fullSdfSize_byte, endHeader_byte, rank);
	float3 initialSdfH = globalDomainSize / make_float3(initialSdfResolution-1);

	// Read heavy data
	std::vector<float> fullSdfData;
	readSdf(fullSdfSize_byte, endHeader_byte, nranks, rank, fullSdfData);

	// We'll make sdf a bit bigger, so that particles that flew away
	// would also be correctly bounced back
	sdfInfo.extendedDomainSize = subDomainSize + 2.0f*margin3;
	sdfInfo.resolution         = make_int3( ceilf(sdfInfo.extendedDomainSize / sdfInfo.h) );
	sdfInfo.h                  = sdfInfo.extendedDomainSize / make_float3(sdfInfo.resolution-1);
	sdfInfo.invh               = 1.0f / sdfInfo.h;

	const float3 scale3 = globalDomainSize / initialSdfExtent;
	if ( fabs(scale3.x - scale3.y) > 1e-5 || fabs(scale3.x - scale3.z) > 1e-5 )
		die("Sdf size and domain size mismatch");
	const float lenScalingFactor = (scale3.x + scale3.y + scale3.z) / 3;

	int3 resolutionBeforeInterpolation;
	float3 offset;
	PinnedBuffer<float> localSdfData;
	prepareRelevantSdfPiece(fullSdfData.data(), subDomainStart - margin3, initialSdfH, initialSdfResolution,
			resolutionBeforeInterpolation, offset, localSdfData);

	// Interpolate
	sdfRawData.resize(sdfInfo.resolution.x * sdfInfo.resolution.y * sdfInfo.resolution.z);

	dim3 threads(8, 8, 8);
	dim3 blocks((sdfInfo.resolution.x+threads.x-1) / threads.x,
				(sdfInfo.resolution.y+threads.y-1) / threads.y,
				(sdfInfo.resolution.z+threads.z-1) / threads.z);

	localSdfData.uploadToDevice();
	cubicInterpolate3D<<< blocks, threads >>>(localSdfData.devPtr(), resolutionBeforeInterpolation, initialSdfH,
			sdfRawData.devPtr(), sdfInfo.resolution, sdfInfo.h, offset, lenScalingFactor);


	// Prepare array to be transformed into texture
	auto chDesc = hipCreateChannelDesc<float>();
	CUDA_Check( hipMalloc3DArray(&sdfArray, &chDesc, make_hipExtent(sdfInfo.resolution.x, sdfInfo.resolution.y, sdfInfo.resolution.z)) );

	hipMemcpy3DParms copyParams = {};
	copyParams.srcPtr = make_hipPitchedPtr((void*)sdfRawData.devPtr(), sdfInfo.resolution.x*sizeof(float), sdfInfo.resolution.x, sdfInfo.resolution.y);
	copyParams.dstArray = sdfArray;
	copyParams.extent = make_hipExtent(sdfInfo.resolution.x, sdfInfo.resolution.y, sdfInfo.resolution.z);
	copyParams.kind = hipMemcpyDeviceToDevice;

	CUDA_Check( hipMemcpy3D(&copyParams) );

	// Create texture
	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = sdfArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.addressMode[2]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	CUDA_Check( hipCreateTextureObject(&sdfInfo.sdfTex, &resDesc, &texDesc, nullptr) );
}

void Wall::freezeParticles(ParticleVector* pv)
{
	PinnedBuffer<int> nFrozen(1), nRemaining(1), nBoundaryCells(1);

	nFrozen.clear();
	countFrozen<<< (pv->np + 127) / 128, 128 >>>((float4*)pv->coosvels.devPtr(), pv->np, sdfInfo, nFrozen.devPtr());
	nFrozen.downloadFromDevice();

	frozen->resize(nFrozen.hostPtr()[0]);
	frozen->mass = pv->mass;
	frozen->domainSize = pv->domainSize;

	info("Freezing %d pv", nFrozen.hostPtr()[0]);

	nFrozen.   clear();
	nRemaining.clear();

	PinnedBuffer<Particle> tmp(pv->np);
	collectFrozen<<< (pv->np + 127) / 128, 128 >>>( (float4*)pv->coosvels.devPtr(), pv->np, sdfInfo,
			(float4*)tmp.devPtr(), (float4*)frozen->coosvels.devPtr(),
			nRemaining.devPtr(), nFrozen.devPtr());
	nRemaining.downloadFromDevice();
	nFrozen.   downloadFromDevice();


	CUDA_Check( hipStreamSynchronize(0) );
	containerSwap(pv->coosvels, tmp);
	pv->resize(nRemaining.hostPtr()[0]);
	pv->changedStamp++;
	info("Keeping %d pv", nRemaining.hostPtr()[0]);

	CUDA_Check( hipDeviceSynchronize() );
}

void Wall::bounce(float dt, hipStream_t stream)
{
	for (int i=0; i<particleVectors.size(); i++)
	{
		auto pv = particleVectors[i];
		auto cl = cellLists[i];

		debug2("Bouncing %d %s particles", pv->size(), pv->name.c_str());
		bounceKernel<<< (boundaryCells[i].size() + 63) / 64, 64, 0, stream >>>(
				boundaryCells[i].devPtr(), boundaryCells[i].size(), cl->cellsStartSize.devPtr(), cl->cellInfo(),
				sdfInfo, (float4*)pv->coosvels.devPtr(), dt);
	}
}


__global__ void _check(float4* coosvels, int n, Wall::SdfInfo sdfInfo)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= n) return;

	float4 coo = coosvels[2*pid];
	float v = evalSdf(coo, sdfInfo);

	if (v > 0) printf("Check failed %d (in array %d): [%f %f %f] (%f)\n", __float_as_int(coo.w), pid, coo.x, coo.y, coo.z, v);
}

void Wall::check(Particle* parts, int n, hipStream_t stream)
{
	_check<<< (n+127)/128, 128, 0, stream >>> ((float4*)parts, n, sdfInfo);
}

