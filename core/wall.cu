#include "hip/hip_runtime.h"
#include <fstream>
#include <cmath>
#include <texture_types.h>

#include <core/hip/hip_vector_types.h>
#include <core/wall.h>
#include <core/celllist.h>
#include <core/interactions.h>
#include <core/interaction_engine.h>


// This should be in hip/hip_vector_types.h, but not there for some reason
//***************************************************************
inline __host__ __device__ int3 operator%(int3 a, int3 b)
{
    return make_int3(a.x % b.x, a.y % b.y, a.z % b.z);
}

inline __host__ __device__ int3 operator/(int3 a, int b)
{
    return make_int3(a.x / b, a.y / b, a.z / b);
}

//***************************************************************


__device__ __forceinline__ float cubicInterpolate1D(float y[4], float mu)
{
   const float a0 = y[3] - y[2] - y[0] + y[1];
   const float a1 = y[0] - y[1] - a0;
   const float a2 = y[2] - y[0];
   const float a3 = y[1];

   return ((a0*mu + a1)*mu + a2)*mu + a3;
}


__global__ void cubicInterpolate3D(const float* in, int3 inDims, float3 inH, float* out, int3 outDims, float3 outH, float3 offset, float scaling)
{
	// Inspired by http://paulbourke.net/miscellaneous/interpolation/
	// Center of the output domain is in offset
	// Center of the input domain is in (0,0,0)

	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	const int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= outDims.x || iy >= outDims.y || iz >= outDims.z) return;

	float interp2D[4][4];
	float interp1D[4];

	// Coordinates where to interpolate
	float3 tmp = make_float3(ix, iy, iz);
	float3 coos = tmp*outH + offset;

	// Reference point of the original grid
	int3 closestInPoint = min( make_int3( fmaxf( floorf(coos / inH), make_float3(0.0f)) ),  inDims-1);

	// Interpolate along x
	for (int dz = -1; dz <= 2; dz++)
		for (int dy = -1; dy <= 2; dy++)
		{
			float vals[4];

			for (int dx = -1; dx <= 2; dx++)
			{
				int3 delta{dx, dy, dz};
				const int3 curCoos = (closestInPoint+delta + inDims) % inDims;

				vals[dx+1] = in[ (curCoos.z*inDims.y + curCoos.y) * inDims.x + curCoos.x ] * scaling;
			}

			interp2D[dz+1][dy+1] = cubicInterpolate1D(vals, (coos.x - closestInPoint.x*inH.x) / inH.x);
		}

	// Interpolate along y
	for (int dz = 0; dz <= 3; dz++)
		interp1D[dz] = cubicInterpolate1D(interp2D[dz], (coos.y - closestInPoint.y*inH.y) / inH.y);

	// Interpolate along z
	out[ (iz*outDims.y + iy) * outDims.x + ix ] = cubicInterpolate1D(interp1D, (coos.z - closestInPoint.z*inH.z) / inH.z);
}

__forceinline__ __device__ float gRouyTourin(float a, float b, float c, float d, float e, float f)
{
	// Rouy-Tourin scheme
	// http://epubs.siam.org/doi/pdf/10.1137/0729053

	return sqrt(
			max( sqr(max(a, 0.0f)), sqr(min(b, 0.0f)) ) +
			max( sqr(max(c, 0.0f)), sqr(min(d, 0.0f)) ) +
			max( sqr(max(e, 0.0f)), sqr(min(f, 0.0f)) )
			);
}

__global__ void redistance(const float* in, int3 dims, float3 h, float dt, float* out)
{
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	const int iz = blockIdx.z * blockDim.z + threadIdx.z;

	auto sqr  = [](float x) { return x*x; };

	auto encode = [=](int i, int j, int k) {
		i = (i+dims.x) % dims.x;
		j = (j+dims.y) % dims.y;
		k = (k+dims.z) % dims.z;

		return (k*dims.y + j) * dims.x + i;
	};

	const int id0 = encode(ix, iy, iz);

	const float u        = in[id0];

	if (fabs(u) < max(h.x, max(h.y, h.z)))
	{
		out[id0] = in[id0];
		return;
	}

	const float ux_minus = in[encode(ix-1, iy,   iz  )];
	const float ux_plus  = in[encode(ix+1, iy,   iz  )];
	const float uy_minus = in[encode(ix,   iy-1, iz  )];
	const float uy_plus  = in[encode(ix,   iy+1, iz  )];
	const float uz_minus = in[encode(ix,   iy,   iz-1)];
	const float uz_plus  = in[encode(ix,   iy,   iz+1)];

	const float dx_minus = (u - ux_minus) / h.x;
	const float dx_plus  = (ux_plus  - u) / h.x;
	const float dy_minus = (u - uy_minus) / h.y;
	const float dy_plus  = (uy_plus  - u) / h.y;
	const float dz_minus = (u - uz_minus) / h.z;
	const float dz_plus  = (uz_plus  - u) / h.z;

	const float grad = gRouyTourin(dx_minus, dx_plus, dy_minus, dy_plus, dz_minus, dz_plus);

//	if (grad < 0.6f)
//		printf("%f,  x %f %f %f,  y %f %f %f,  z %f %f %f,  %d %d %d\n", grad,
//				ux_minus, u, ux_plus,  uy_minus, u, uy_plus,  uz_minus, u, uz_plus, ix, iy, iz);

	const float S = fabs(u) / sqrt( u*u + sqr(grad * h.x) );
	out[id0] = u + S * dt * (1-grad);
}

template<typename T>
__device__ __forceinline__ float evalSdf(hipTextureObject_t sdfTex, T x, float3 subDomainSize, float3 h, float3 invH)
{
	float3 x3{x.x, x.y, x.z};
	float3 texcoord = floorf((x3 + subDomainSize*0.5f) * invH);
	float3 lambda = (x3 - (texcoord * h - subDomainSize*0.5f)) * invH;

	const float s000 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 0, texcoord.z + 0);
	const float s001 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 0, texcoord.z + 0);
	const float s010 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 1, texcoord.z + 0);
	const float s011 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 1, texcoord.z + 0);
	const float s100 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 0, texcoord.z + 1);
	const float s101 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 0, texcoord.z + 1);
	const float s110 = tex3D<float>(sdfTex, texcoord.x + 0, texcoord.y + 1, texcoord.z + 1);
	const float s111 = tex3D<float>(sdfTex, texcoord.x + 1, texcoord.y + 1, texcoord.z + 1);

	const float s00x = s000 * (1 - lambda.x) + lambda.x * s001;
	const float s01x = s010 * (1 - lambda.x) + lambda.x * s011;
	const float s10x = s100 * (1 - lambda.x) + lambda.x * s101;
	const float s11x = s110 * (1 - lambda.x) + lambda.x * s111;

	const float s0yx = s00x * (1 - lambda.y) + lambda.y * s01x;
	const float s1yx = s10x * (1 - lambda.y) + lambda.y * s11x;

	const float szyx = s0yx * (1 - lambda.z) + lambda.z * s1yx;

//	printf("[%f %f %f]  [%f %f %f]  [%f %f %f]  = %f  vs  %f\n", x.x, x.y, x.z,  texcoord.x, texcoord.y, texcoord.z,
//			lambda.x, lambda.y, lambda.z, szyx, sqrt(x.x*x.x + x.y*x.y + x.z*x.z) - 5);

	return szyx;
}

// warp-aggregated atomic increment
// https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
__device__ __forceinline__ int atomicAggInc(int *ctr)
{
	int lane_id = (threadIdx.x % 32);

	int mask = __ballot(1);
	// select the leader
	int leader = __ffs(mask) - 1;
	// leader does the update
	int res;
	if(lane_id == leader)
	res = atomicAdd(ctr, __popc(mask));
	// broadcast result
	res = __shfl(res, leader);
	// each thread computes its own value
	return res + __popc(mask & ((1 << lane_id) - 1));
}


__global__ void countFrozen(const float4* pv, const int np, hipTextureObject_t sdfTex, float3 subDomainSize, float3 h, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coos = pv[2*pid];

	const float sdf = evalSdf(sdfTex, coos, subDomainSize, h, 1.0f / h);

	if (sdf > 0.0f && sdf < 1.2f)
	{
		atomicAggInc(nFrozen);
	}
}

__global__ void collectFrozen(hipTextureObject_t sdfTex, float3 subDomainSize, float3 h, const int np,
		const float4* input, float4* remaining, float4* frozen, int* nRemaining, int* nFrozen)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coos = input[2*pid];
	const float4 vels = input[2*pid+1];

	const float sdf = evalSdf(sdfTex, coos, subDomainSize, h, 1.0f / h);

	if (sdf <= 0.0f)
	{
		const int ind = atomicAggInc(nRemaining);
		remaining[2*ind] = coos;
		remaining[2*ind + 1] = vels;
	}

	if (sdf > 0.0f && sdf < 1.2f)
	{
		const int ind = atomicAggInc(nFrozen);
		frozen[2*ind] = coos;
		frozen[2*ind + 1] = vels;
	}
}

__global__ void countBoundaryCells(CellListInfo cinfo, hipTextureObject_t sdfTex,
		const float3 subDomainSize, const float3 h, int* nBoundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;
	int ix, iy, iz;

	cinfo.decode(cid, ix, iy, iz);

	const float3 invH = 1.0f / h;

	const float cx = cinfo.domainStart.x + ix*cinfo.rc;
	const float cy = cinfo.domainStart.y + iy*cinfo.rc;
	const float cz = cinfo.domainStart.z + iz*cinfo.rc;

	const float l = cinfo.rc;
	const float s000 = evalSdf(sdfTex, make_float3(cx,   cy,   cz),   subDomainSize, h, invH);
	const float s001 = evalSdf(sdfTex, make_float3(cx,   cy,   cz+l), subDomainSize, h, invH);
	const float s010 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz),   subDomainSize, h, invH);
	const float s011 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz+l), subDomainSize, h, invH);
	const float s100 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz),   subDomainSize, h, invH);
	const float s101 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz+l), subDomainSize, h, invH);
	const float s110 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz),   subDomainSize, h, invH);
	const float s111 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz+l), subDomainSize, h, invH);

	if ( (0.1f > s000 && s000 > -1.1f) || (0.1f > s001 && s001 > -1.1f) ||
		 (0.1f > s010 && s010 > -1.1f) || (0.1f > s011 && s011 > -1.1f) ||
		 (0.1f > s100 && s100 > -1.1f) || (0.1f > s101 && s101 > -1.1f) ||
		 (0.1f > s110 && s110 > -1.1f) || (0.1f > s111 && s111 > -1.1f) )
	{
		atomicAggInc(nBoundaryCells);
	}
}

__global__ void getBoundaryCells(CellListInfo cinfo, hipTextureObject_t sdfTex,
		const float3 subDomainSize, const float3 h, int* nBoundaryCells, int* boundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;

	int ix, iy, iz;

	cinfo.decode(cid, ix, iy, iz);

	const float3 invH = 1.0f / h;

	const float cx = cinfo.domainStart.x + ix*cinfo.rc;
	const float cy = cinfo.domainStart.y + iy*cinfo.rc;
	const float cz = cinfo.domainStart.z + iz*cinfo.rc;

	const float l = cinfo.rc;
	const float s000 = evalSdf(sdfTex, make_float3(cx,   cy,   cz),   subDomainSize, h, invH);
	const float s001 = evalSdf(sdfTex, make_float3(cx,   cy,   cz+l), subDomainSize, h, invH);
	const float s010 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz),   subDomainSize, h, invH);
	const float s011 = evalSdf(sdfTex, make_float3(cx,   cy+l, cz+l), subDomainSize, h, invH);
	const float s100 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz),   subDomainSize, h, invH);
	const float s101 = evalSdf(sdfTex, make_float3(cx+l, cy,   cz+l), subDomainSize, h, invH);
	const float s110 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz),   subDomainSize, h, invH);
	const float s111 = evalSdf(sdfTex, make_float3(cx+l, cy+l, cz+l), subDomainSize, h, invH);

	if ( (0.1f > s000 && s000 > -1.1f) || (0.1f > s001 && s001 > -1.1f) ||
		 (0.1f > s010 && s010 > -1.1f) || (0.1f > s011 && s011 > -1.1f) ||
		 (0.1f > s100 && s100 > -1.1f) || (0.1f > s101 && s101 > -1.1f) ||
		 (0.1f > s110 && s110 > -1.1f) || (0.1f > s111 && s111 > -1.1f) )
	{
		int id = atomicAggInc(nBoundaryCells);
		boundaryCells[id] = cid;
	}
}

__launch_bounds__(128, 8)
__global__ void bounceKernel(const int* wallCells, const int nWallCells, const int* __restrict__ cellsStart, CellListInfo cinfo, const float4* accs,
		hipTextureObject_t sdfTex, const float3 subDomainSize, const float3 h, const float3 invH, float4* xyzouvwo, const float dt)
{
	const int maxNIters = 20;
	const float tolerance = 5e-6;

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nWallCells) return;
	const int cid = wallCells[tid];

	const int2 startSize = cinfo.decodeStartSize(cellsStart[cid]);

	for (int pid = startSize.x; pid < startSize.x + startSize.y; pid++)
	{
		float va, vb;

		float4 coo = xyzouvwo[2*pid];
		float4 vel = xyzouvwo[2*pid+1];

		// Warning - this is only valid for VV
		float4 oldCoo = coo - dt*vel;

		vb = evalSdf(sdfTex, coo, subDomainSize, h, invH);
		if (vb < 0.0f) continue; // if inside - continue

		va = evalSdf(sdfTex, oldCoo, subDomainSize, h, invH);
		assert( va < 0.0f ); // Accuracy issues here!

		// Determine where we cross
		// Interpolation search

		float3 a{oldCoo.x, oldCoo.y, oldCoo.z};
		float3 b{coo.x, coo.y, coo.z};
		float3 mid;
		float vmid;

		int iters;

		for (iters=0; iters<maxNIters; iters++)
		{
			const float lambda = min(max((vb / (vb - va)), 0.01f), 0.99f);  // va*l + (1-l)*vb = 0
			mid = a*lambda + b*(1.0f - lambda);
			vmid = evalSdf(sdfTex, mid, subDomainSize, h, invH);

			if (va * vmid < 0.0f)
			{
				vb = vmid;
				b = mid;
			}
			else
			{
				va = vmid;
				a = mid;
			}

			if (fabs(vmid) < tolerance) break;
		}
		assert(fabs(vmid) < tolerance);

		// Final intersection at old*alpha + new*(1-alpha)
		const float alpha = (oldCoo.x - mid.x) / (oldCoo.x - coo.x);

		// Travel along alpha*(new - old), then bounces back along -(1-alpha)*(new - old)
		float beta = 2*alpha - 1;

		// In the corners long bounce may place the particle into another wall
		// Need to find a safe step in that case
		float4 candidate = oldCoo + beta * (coo - oldCoo);

		for (int i=0; i<maxNIters; i++)
		{
			if ( (evalSdf(sdfTex, candidate, subDomainSize, h, invH)) < 0.0f ) break;

			beta *= 0.5;
			candidate = oldCoo - beta * (coo - oldCoo);
		}

		// Not sure why, but this assertion always fails
		// even though everything seems allright
		//assert(vcandidate < 1.0f);

		xyzouvwo[2*pid] = candidate;
		xyzouvwo[2*pid + 1] = -vel;
	}
}

__global__ void checkKernel(float4* data, const int n, hipTextureObject_t sdfTex, const float3 subDomainSize, const float3 h, const float3 invH)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= n) return;

	float4 coo = data[2*pid];
	float v = evalSdf(sdfTex, coo, subDomainSize, h, invH);

	if (v > 0.0f)
		printf("CHECK! %d:  [%f %f %f] -> %f\n", __float_as_int(coo.w), coo.x, coo.y, coo.z, v);
}

void Wall::_check()
{
	for (auto& pv : particleVectors)
	{
		checkKernel<<< (pv->np + 127) / 128, 128 >>>( (float4*)pv->coosvels.constDevPtr(), pv->np, sdfTex, subDomainSize, sdfH, 1.0 / sdfH);
	}
}


/*
 * We only set a few params here
 */
Wall::Wall(std::string name, std::string sdfFileName, float3 sdfH,  float _creationTime) :
		name(name), sdfFileName(sdfFileName), sdfH(sdfH), _creationTime(_creationTime), frozen(name)
{ }

void Wall::attach(ParticleVector* pv, CellList* cl)
{
	particleVectors.push_back(pv);
	cellLists.push_back(cl);

	const int oldSize = nBoundaryCells.size();
	boundaryCells.resize(oldSize+1);
	nBoundaryCells.resize(oldSize+1, resizePreserve);
	nBoundaryCells.hostPtr()[oldSize] = 0;

	countBoundaryCells<<< (cl->totcells + 127) / 128, 128 >>> (cl->cellInfo(), sdfTex, subDomainSize, sdfH, nBoundaryCells.devPtr()+oldSize);

	info("Found %d boundary cells", nBoundaryCells.constHostPtr()[oldSize]);
	boundaryCells[oldSize].resize(nBoundaryCells.constHostPtr()[oldSize]);

	nBoundaryCells.hostPtr()[oldSize] = 0;
	getBoundaryCells<<< (cl->totcells + 127) / 128, 128 >>> (cl->cellInfo(), sdfTex, subDomainSize, sdfH,
			nBoundaryCells.devPtr()+oldSize, boundaryCells[oldSize].devPtr());
}

void Wall::readSdf(int64_t fullSdfSize_byte, int64_t endHeader_byte, int nranks, int rank, std::vector<float>& fullSdfData)
{
	// Read part and allgather
	const int64_t readPerProc_byte = (fullSdfSize_byte + nranks - 1) / (int64_t)nranks;
	std::vector<char> readBuffer(readPerProc_byte);

	// Limits in bytes
	const int64_t readStart = readPerProc_byte * rank + endHeader_byte;
	const int64_t readEnd   = std::min( readStart + readPerProc_byte, fullSdfSize_byte + endHeader_byte);

	MPI_File fh;
	MPI_Status status;
	MPI_Check( MPI_File_open(wallComm, sdfFileName.c_str(), MPI_MODE_RDONLY, MPI_INFO_NULL, &fh) );  // TODO: MPI_Info
	MPI_Check( MPI_File_read_at_all(fh, readStart, readBuffer.data(), readEnd - readStart, MPI_BYTE, &status) );
	// TODO: check that we read just what we asked
	// MPI_Get_count only return int though

	fullSdfData.resize(readPerProc_byte * nranks / sizeof(float));  // May be bigger than fullSdfSize, to make gather easier
	MPI_Check( MPI_Allgather(readBuffer.data(), readPerProc_byte, MPI_BYTE, fullSdfData.data(), readPerProc_byte, MPI_BYTE, wallComm) );
}

void Wall::readHeader(int3& sdfResolution, float3& sdfExtent, int64_t& fullSdfSize_byte, int64_t& endHeader_byte, int rank)
{
	if (rank == 0)
	{
		//printf("'%s'\n", sdfFileName.c_str());
		std::ifstream file(sdfFileName);
		if (!file.good())
			die("File not found or not accessible");

		auto fstart = file.tellg();

		file >> sdfExtent.x >> sdfExtent.y >> sdfExtent.z >>
			sdfResolution.x >> sdfResolution.y >> sdfResolution.z;
		fullSdfSize_byte = (int64_t)sdfResolution.x * sdfResolution.y * sdfResolution.z * sizeof(float);

		info("Using wall file '%s' of size %.2fx%.2fx%.2f and resolution %dx%dx%d", sdfFileName.c_str(),
				sdfExtent.x, sdfExtent.y, sdfExtent.z,
				sdfResolution.x, sdfResolution.y, sdfResolution.z);

		file.seekg( 0, std::ios::end );
		auto fend = file.tellg();

		endHeader_byte = (fend - fstart) - fullSdfSize_byte;

		file.close();
	}

	MPI_Check( MPI_Bcast(&sdfExtent,        3, MPI_FLOAT,     0, wallComm) );
	MPI_Check( MPI_Bcast(&sdfResolution,    3, MPI_INT,       0, wallComm) );
	MPI_Check( MPI_Bcast(&fullSdfSize_byte, 1, MPI_INT64_T,   0, wallComm) );
	MPI_Check( MPI_Bcast(&endHeader_byte,   1, MPI_INT64_T,   0, wallComm) );
}


void Wall::create(MPI_Comm& comm, float3 subDomainStart, float3 subDomainSize, float3 globalDomainSize, ParticleVector* pv, CellList* cl)
{
	MPI_Check( MPI_Comm_dup(comm, &wallComm) );

	int nranks, rank;
	int ranks[3], periods[3], coords[3];
	MPI_Check( MPI_Comm_size(wallComm, &nranks) );
	MPI_Check( MPI_Comm_rank(wallComm, &rank) );
	MPI_Check( MPI_Cart_get (wallComm, 3, ranks, periods, coords) );

	int3 initialSdfResolution;
	float3 initialSdfExtent;

	int64_t fullSdfSize_byte;
	int64_t endHeader_byte;

	// Read header
	readHeader(initialSdfResolution, initialSdfExtent, fullSdfSize_byte, endHeader_byte, rank);

	std::vector<float> fullSdfData;
	// Read heavy data
	readSdf(fullSdfSize_byte, endHeader_byte, nranks, rank, fullSdfData);


	subDomainResolution = make_int3(ceilf(subDomainSize / sdfH));
	sdfH = subDomainSize / make_float3(subDomainResolution);

	// Find your relevant chunk of data
	const float3 scale3 = globalDomainSize / initialSdfExtent;
	if ( fabs(scale3.x - scale3.y) > 1e-5 || fabs(scale3.x - scale3.z) > 1e-5 )
		die("Sdf size and domain size mismatch");
	const float scale = (scale3.x + scale3.y + scale3.z) / 3;

	const int margin = 3; // +2 from cubic interpolation, +1 from possible round-off errors
	float3 initialH = globalDomainSize / make_float3(initialSdfResolution-1);

	const int3 startId = make_int3( floorf(subDomainStart / initialH) );
	const int3 endId   = make_int3( ceilf((subDomainStart + subDomainSize) / initialH) );

	float3 startInLocalCoord = make_float3(startId - margin)*initialH - (subDomainStart + 0.5*subDomainSize);
	//float3 endInLocalCoord   = make_float3(endId   + margin)*initialH - (subDomainStart + 0.5*subDomainSize);
	const int3 inputResolution = (endId - startId) + make_int3(2*margin);

	PinnedBuffer<float> inputSdfData ( inputResolution.x * inputResolution.y * inputResolution.z );
	auto inpSdfDataPtr = inputSdfData.hostPtr();

	for (int k = 0; k < inputResolution.z; k++)
		for (int j = 0; j < inputResolution.y; j++)
			for (int i = 0; i < inputResolution.x; i++)
			{
				const int origIx = (i+startId.x + initialSdfResolution.x) % initialSdfResolution.x;
				const int origIy = (j+startId.y + initialSdfResolution.y) % initialSdfResolution.y;
				const int origIz = (k+startId.z + initialSdfResolution.z) % initialSdfResolution.z;

				inpSdfDataPtr[ (k*inputResolution.y + j)*inputResolution.x + i ] =
						fullSdfData[ (origIz*initialSdfResolution.y + origIy)*initialSdfResolution.x + origIx ];
			}

	// Compute offset
	float3 offset = startInLocalCoord - 0.5*subDomainSize;

	// Interpolate
	sdfRawData.resize(subDomainResolution.x * subDomainResolution.y * subDomainResolution.z);

	dim3 threads(8, 8, 8);
	dim3 blocks((subDomainResolution.x+threads.x-1) / threads.x,
				(subDomainResolution.y+threads.y-1) / threads.y,
				(subDomainResolution.z+threads.z-1) / threads.z);

	float lenScalingFactor = scale;
	cubicInterpolate3D<<< blocks, threads >>>(inputSdfData.constDevPtr(), inputResolution, initialH, sdfRawData.devPtr(), subDomainResolution, sdfH, offset, lenScalingFactor);

	// Redistance
	// Need 2 arrays for redistancing

//	DeviceBuffer<float> tmp(sdfData.size);
//	const float redistDt = 0.1;
//	for (float t = 0; t < 200; t+=redistDt)
//	{
//		redistance<<< blocks, threads >>>(sdfData.constDevPtr(), resolution, h, redistDt, tmp.devPtr());
//		swap(sdfData, tmp);
//	}

	// Prepare array to be transformed into texture
	auto chDesc = hipCreateChannelDesc<float>();
	CUDA_Check( hipMalloc3DArray(&sdfArray, &chDesc, make_hipExtent(subDomainResolution.x, subDomainResolution.y, subDomainResolution.z)) );

	hipMemcpy3DParms copyParams = {};
	copyParams.srcPtr = make_hipPitchedPtr((void*)sdfRawData.constDevPtr(), subDomainResolution.x*sizeof(float), subDomainResolution.y, subDomainResolution.z);
	copyParams.dstArray = sdfArray;
	copyParams.extent = make_hipExtent(subDomainResolution.x, subDomainResolution.y, subDomainResolution.z);
	copyParams.kind = hipMemcpyDeviceToDevice;

	CUDA_Check( hipMemcpy3D(&copyParams) );

	// Create texture
	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = sdfArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.addressMode[2]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	CUDA_Check( hipCreateTextureObject(&sdfTex, &resDesc, &texDesc, nullptr) );

	PinnedBuffer<int> nFrozen(1), nRemaining(1), nBoundaryCells(1);

	nFrozen.clear();
	countFrozen<<< (pv->np + 127) / 128, 128 >>>((float4*)pv->coosvels.constDevPtr(), pv->np, sdfTex, subDomainSize, sdfH, nFrozen.devPtr());

	frozen.resize(nFrozen.constHostPtr()[0]);
	info("Freezing %d pv", nFrozen.constHostPtr()[0]);

	nFrozen.   clear();
	nRemaining.clear();
	collectFrozen<<< (pv->np + 127) / 128, 128 >>>(sdfTex, subDomainSize, sdfH, pv->np,
			(float4*)pv->coosvels.constDevPtr(), (float4*)pv->pingPongBuf.devPtr(), (float4*)frozen.coosvels.devPtr(),
			nRemaining.devPtr(), nFrozen.devPtr());

	swap(pv->coosvels, pv->pingPongBuf);
	pv->resize(nRemaining.constHostPtr()[0]);
	info("Keeping %d pv", nRemaining.constHostPtr()[0]);

	CUDA_Check( hipDeviceSynchronize() );
}

void Wall::bounce(hipStream_t stream)
{
	for (int i=0; i<particleVectors.size(); i++)
	{
		auto pv = particleVectors[i];
		auto cl = cellLists[i];

		bounceKernel<<< (boundaryCells[i].size() + 63) / 64, 64, 0, stream >>>(
				boundaryCells[i].constDevPtr(), boundaryCells[i].size(), cl->cellsStart.constDevPtr(), cl->cellInfo(), (float4*)pv->forces.constDevPtr(),
				sdfTex, subDomainSize, sdfH, 1.0 / sdfH, (float4*)pv->coosvels.devPtr(), dt);
	}
}

