#include "hip/hip_runtime.h"
#include <fstream>
#include <cmath>
#include <texture_types.h>

#include <core/cuda_common.h>
#include <core/wall.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/bounce_solver.h>

#include <core/cuda-rng.h>

#include "sdf_kernels.h"

//===============================================================================================
// Interpolation kernels
//===============================================================================================

__device__ __forceinline__ float cubicInterpolate1D(float y[4], float mu)
{
	// mu == 0 at y[1], mu == 1 at y[2]
	const float a0 = y[3] - y[2] - y[0] + y[1];
	const float a1 = y[0] - y[1] - a0;
	const float a2 = y[2] - y[0];
	const float a3 = y[1];

	return ((a0*mu + a1)*mu + a2)*mu + a3;
}

__global__ void cubicInterpolate3D(const float* in, int3 inDims, float3 inH, float* out, int3 outDims, float3 outH, float3 offset, float scalingFactor)
{
	// Inspired by http://paulbourke.net/miscellaneous/interpolation/
	// Center of the output domain is in offset
	// Center of the input domain is in (0,0,0)

	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	const int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= outDims.x || iy >= outDims.y || iz >= outDims.z) return;

	float interp2D[4][4];
	float interp1D[4];

	// Coordinates where to interpolate
	float3 outputId  = make_float3(ix, iy, iz);
	float3 outputCoo = outputId*outH;

	float3 inputCoo  = outputCoo + offset;

	// Make sure we're within the region where the the input data is defined
	assert( 0.0f <= inputCoo.x && inputCoo.x <= inDims.x*inH.x &&
			0.0f <= inputCoo.y && inputCoo.y <= inDims.y*inH.y &&
			0.0f <= inputCoo.z && inputCoo.z <= inDims.z*inH.z    );

	// Reference point of the original grid, rounded down
	int3 inputId_down = make_int3( floorf(inputCoo / inH) );
	float3 mu = (inputCoo - make_float3(inputId_down)*inH) / inH;

	// Interpolate along x
	for (int dz = -1; dz <= 2; dz++)
		for (int dy = -1; dy <= 2; dy++)
		{
			float vals[4];

			for (int dx = -1; dx <= 2; dx++)
			{
				int3 delta{dx, dy, dz};
				const int3 curInputId = (inputId_down+delta + inDims) % inDims;

				vals[dx+1] = in[ (curInputId.z*inDims.y + curInputId.y) * inDims.x + curInputId.x ] * scalingFactor;
			}

			interp2D[dz+1][dy+1] = cubicInterpolate1D(vals, mu.x);
		}

	// Interpolate along y
	for (int dz = 0; dz <= 3; dz++)
		interp1D[dz] = cubicInterpolate1D(interp2D[dz], mu.y);

	// Interpolate along z
	out[ (iz*outDims.y + iy) * outDims.x + ix ] = cubicInterpolate1D(interp1D, mu.z);
}

//===============================================================================================
// Removing kernels
//===============================================================================================

__global__ void countRemaining(const float4* pv, const int np, Wall::SdfInfo sdfInfo, int* nRemaining)
{
	const float tolerance = 1e-6f;

	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coo = pv[2*pid];
	const float sdf = evalSdf(coo, sdfInfo);

	if (sdf <= -tolerance)
		atomicAggInc(nRemaining);
}

__global__ void collectRemaining(const float4* input, const int np, Wall::SdfInfo sdfInfo,
		float4* remaining, int* nRemaining)
{
	const float tolerance = 1e-6f;

	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	const float4 coo = input[2*pid];
	const float4 vel = input[2*pid+1];

	const float sdf = evalSdf(coo, sdfInfo);

	if (sdf <= -tolerance)
	{
		const int ind = atomicAggInc(nRemaining);
		remaining[2*ind] = coo;
		remaining[2*ind + 1] = vel;
	}
}

__global__ void collectRemainingObjects(const float4* input, const int nObjects, const int objSize, Wall::SdfInfo sdfInfo,
		float4* remaining, int* nRemaining)
{
	const float tolerance = 1e-6f;

	// One warp per object
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int objId = gid / warpSize;
	const int tid = gid % warpSize;

	if (objId >= nObjects) return;

	bool isRemaining = true;
	for (int i=tid; i<objSize; i++)
	{
		Particle p(input, objId*objSize + i);
		if (evalSdf(p.r, sdfInfo) <= -tolerance)
		{
			isRemaining = false;
			break;
		}
	}

	if (!isRemaining) return;

	int dstId = atomicAdd(nRemaining, objSize);

	for (int i=tid; i<objSize; i++)
	{
		Particle p(input, objId*objSize + i);
		float4* dstAddr = remaining + 2*(dstId + i);
		dstAddr[0] = p.r2Float4();
		dstAddr[1] = p.u2Float4();
	}
}

//===============================================================================================
// Boundary walls kernels
//===============================================================================================

__device__ inline bool isCellOnBoundary(float3 cornerCoo, float3 len, Wall::SdfInfo sdfInfo)
{
	// About maximum distance a particle can cover in one step
	const float tol = 0.25f;

#pragma unroll
	for (int i=0; i<2; i++)
#pragma unroll
		for (int j=0; j<2; j++)
#pragma unroll
			for (int k=0; k<2; k++)
			{
				// Value in the cell corner
				const float3 shift = make_float3(i ? len.x : 0.0f, j ? len.y : 0.0f, k ? len.z : 0.0f);
				const float s = evalSdf( cornerCoo + shift,  sdfInfo );

				if (-1.0f - tol < s && s < 0.0f + tol)
					return true;
			}

	return false;
}

__global__ void countBoundaryCells(CellListInfo cinfo, Wall::SdfInfo sdfInfo, int* nBoundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;

	int3 ind;
	cinfo.decode(cid, ind.x, ind.y, ind.z);
	float3 cornerCoo = -0.5f*cinfo.localDomainSize + make_float3(ind)*cinfo.h;

	if (isCellOnBoundary(cornerCoo, cinfo.h, sdfInfo))
		atomicAggInc(nBoundaryCells);
}

__global__ void getBoundaryCells(CellListInfo cinfo, Wall::SdfInfo sdfInfo,
		int* nBoundaryCells, int* boundaryCells)
{
	const int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if (cid >= cinfo.totcells) return;

	int3 ind;
	cinfo.decode(cid, ind.x, ind.y, ind.z);
	float3 cornerCoo = -0.5f*cinfo.localDomainSize + make_float3(ind)*cinfo.h;

	if (isCellOnBoundary(cornerCoo, cinfo.h, sdfInfo))
	{
		int id = atomicAggInc(nBoundaryCells);
		boundaryCells[id] = cid;
	}
}

//===============================================================================================
// SDF bouncing kernel
//===============================================================================================

__global__ void bounceSDF(const int* wallCells, const int nWallCells, const uint* __restrict__ cellsStartSize, CellListInfo cinfo,
		Wall::SdfInfo sdfInfo, float4* coosvels, const float dt)
{
	const int maxIters = 50;
	const float corrStep = (1.0f / (float)maxIters) * dt;

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nWallCells) return;
	const int cid = wallCells[tid];

	const int2 startSize = cinfo.decodeStartSize(cellsStartSize[cid]);

	for (int pid = startSize.x; pid < startSize.x + startSize.y; pid++)
	{
		Particle p(coosvels[2*pid], coosvels[2*pid+1]);
		if (evalSdf(p.r, sdfInfo) <= 0.0f) continue;

		float3 oldCoo = p.r - p.u*dt;

		for (int i=0; i<maxIters; i++)
		{
			if (evalSdf(oldCoo, sdfInfo) < 0.0f) break;
			oldCoo -= p.u*corrStep;
		}

		const float alpha = solveLinSearch([=] (float lambda) { return evalSdf(oldCoo + (p.r-oldCoo)*lambda, sdfInfo); });
		float3 candidate = (alpha >= 0.0f) ? oldCoo + alpha * (p.r - oldCoo) : oldCoo;

		if (evalSdf(candidate, sdfInfo) >= 0.0f)
			for (int i=0; i<maxIters; i++)
			{
				float3 rndShift;
				rndShift.x = Saru::mean0var1(p.r.x - floorf(p.r.x), p.i1+i, p.i1*p.i1);
				rndShift.y = Saru::mean0var1(rndShift.x,            p.i1+i, p.i1*p.i1);
				rndShift.z = Saru::mean0var1(rndShift.y,            p.i1+i, p.i1*p.i1);
	
				if (evalSdf(candidate + 5.0f*rndShift*dt, sdfInfo) < 0.0f)
				{
					candidate += 5.0f*rndShift*dt;
					break;
				}
			}

		coosvels[2*pid]     = Float3_int(candidate, p.i1).toFloat4();
		coosvels[2*pid + 1] = Float3_int(-p.u, p.i2).toFloat4();
	}
}

__global__ void checkInside(const float4* coosvels, int np, Wall::SdfInfo sdfInfo, int* nInside)
{
	const int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid >= np) return;

	float4 coo = coosvels[2*pid];
	float v = evalSdf(coo, sdfInfo);

	if (v > 0)
		atomicAggInc(nInside);
}

/*
 * We only set a few params here
 */
Wall::Wall(std::string name, std::string sdfFileName, float3 sdfH) :
		name(name), sdfFileName(sdfFileName), nInside(1)
{
	sdfInfo.h = sdfH;
}

void Wall::attach(ParticleVector* pv, CellList* cl)
{
	CUDA_Check( hipDeviceSynchronize() );
	particleVectors.push_back(pv);
	cellLists.push_back(cl);

	const int oldSize = nBoundaryCells.size();
	boundaryCells.resize(oldSize+1);

	nBoundaryCells.resize(oldSize+1, 0);
	nBoundaryCells.hostPtr()[oldSize] = 0;
	nBoundaryCells.uploadToDevice(0);
	countBoundaryCells<<< (cl->totcells + 127) / 128, 128, 0, 0 >>> (cl->cellInfo(), sdfInfo, nBoundaryCells.devPtr()+oldSize);
	nBoundaryCells.downloadFromDevice(0);

	debug("Found %d boundary cells", nBoundaryCells.hostPtr()[oldSize]);
	boundaryCells[oldSize].resize(nBoundaryCells.hostPtr()[oldSize], 0);

	nBoundaryCells.hostPtr()[oldSize] = 0;
	nBoundaryCells.uploadToDevice(0);
	getBoundaryCells<<< (cl->totcells + 127) / 128, 128, 0, 0 >>> (cl->cellInfo(), sdfInfo,
			nBoundaryCells.devPtr()+oldSize, boundaryCells[oldSize].devPtr());
	CUDA_Check( hipDeviceSynchronize() );
}

void Wall::readHeader(int3& sdfResolution, float3& sdfExtent, int64_t& fullSdfSize_byte, int64_t& endHeader_byte, int rank)
{
	if (rank == 0)
	{
		//printf("'%s'\n", sdfFileName.c_str());
		std::ifstream file(sdfFileName);
		if (!file.good())
			die("File not found or not accessible");

		auto fstart = file.tellg();

		file >> sdfExtent.x >> sdfExtent.y >> sdfExtent.z >>
			sdfResolution.x >> sdfResolution.y >> sdfResolution.z;
		fullSdfSize_byte = (int64_t)sdfResolution.x * sdfResolution.y * sdfResolution.z * sizeof(float);

		info("Using wall file '%s' of size %.2fx%.2fx%.2f and resolution %dx%dx%d", sdfFileName.c_str(),
				sdfExtent.x, sdfExtent.y, sdfExtent.z,
				sdfResolution.x, sdfResolution.y, sdfResolution.z);

		file.seekg( 0, std::ios::end );
		auto fend = file.tellg();

		endHeader_byte = (fend - fstart) - fullSdfSize_byte;

		file.close();
	}

	MPI_Check( MPI_Bcast(&sdfExtent,        3, MPI_FLOAT,     0, wallComm) );
	MPI_Check( MPI_Bcast(&sdfResolution,    3, MPI_INT,       0, wallComm) );
	MPI_Check( MPI_Bcast(&fullSdfSize_byte, 1, MPI_INT64_T,   0, wallComm) );
	MPI_Check( MPI_Bcast(&endHeader_byte,   1, MPI_INT64_T,   0, wallComm) );
}

void Wall::readSdf(int64_t fullSdfSize_byte, int64_t endHeader_byte, int nranks, int rank, std::vector<float>& fullSdfData)
{
	// Read part and allgather
	const int64_t readPerProc_byte = (fullSdfSize_byte + nranks - 1) / (int64_t)nranks;
	std::vector<char> readBuffer(readPerProc_byte);

	// Limits in bytes
	const int64_t readStart = readPerProc_byte * rank + endHeader_byte;
	const int64_t readEnd   = std::min( readStart + readPerProc_byte, fullSdfSize_byte + endHeader_byte);

	MPI_File fh;
	MPI_Status status;
	MPI_Check( MPI_File_open(wallComm, sdfFileName.c_str(), MPI_MODE_RDONLY, MPI_INFO_NULL, &fh) );  // TODO: MPI_Info
	MPI_Check( MPI_File_read_at_all(fh, readStart, readBuffer.data(), readEnd - readStart, MPI_BYTE, &status) );
	// TODO: check that we read just what we asked
	// MPI_Get_count only return int though

	fullSdfData.resize(readPerProc_byte * nranks / sizeof(float));  // May be bigger than fullSdfSize, to make gather easier
	MPI_Check( MPI_Allgather(readBuffer.data(), readPerProc_byte, MPI_BYTE, fullSdfData.data(), readPerProc_byte, MPI_BYTE, wallComm) );
}

void Wall::prepareRelevantSdfPiece(const float* fullSdfData, float3 extendedDomainStart, float3 initialSdfH, int3 initialSdfResolution,
		int3& resolution, float3& offset, PinnedBuffer<float>& localSdfData)
{
	// Find your relevant chunk of data
	// We cannot send big sdf files directly, so we'll carve a piece now

	const int margin = 3; // +2 from cubic interpolation, +1 from possible round-off errors
	const int3 startId = make_int3( floorf( extendedDomainStart                             / initialSdfH) ) - margin;
	const int3 endId   = make_int3( ceilf ((extendedDomainStart+sdfInfo.extendedDomainSize) / initialSdfH) ) + margin;

	float3 startInLocalCoord = make_float3(startId)*initialSdfH - (extendedDomainStart + 0.5*sdfInfo.extendedDomainSize);
	offset = -0.5*sdfInfo.extendedDomainSize - startInLocalCoord;

	int rank;
	MPI_Check( MPI_Comm_rank(wallComm, &rank) );
//	printf("%d:  SDstart [%f %f %f]  sdfH [%f %f %f] startId [%d %d %d], endId [%d %d %d], localstart [%f %f %f]\n",
//				rank,
//				extendedDomainStart.x, extendedDomainStart.y, extendedDomainStart.z,
//				initialSdfH.x, initialSdfH.y, initialSdfH.z,
//				startId.x, startId.y, startId.z,
//				endId.x, endId.y, endId.z,
//				startInLocalCoord.x, startInLocalCoord.y, startInLocalCoord.z);

	resolution = endId - startId;

	localSdfData.resize( resolution.x * resolution.y * resolution.z, 0 );
	auto locSdfDataPtr = localSdfData.hostPtr();

//	printf("%d:  input [%d %d %d], initial [%d %d %d], start [%d %d %d]\n",
//			rank, resolution.x, resolution.y, resolution.z,
//			initialSdfResolution.x, initialSdfResolution.y, initialSdfResolution.z,
//			startId.x, startId.y, startId.z);

//#warning "Minus here should be removed"
	for (int k = 0; k < resolution.z; k++)
		for (int j = 0; j < resolution.y; j++)
			for (int i = 0; i < resolution.x; i++)
			{
				const int origIx = (i+startId.x + initialSdfResolution.x) % initialSdfResolution.x;
				const int origIy = (j+startId.y + initialSdfResolution.y) % initialSdfResolution.y;
				const int origIz = (k+startId.z + initialSdfResolution.z) % initialSdfResolution.z;

				locSdfDataPtr[ (k*resolution.y + j)*resolution.x + i ] =
						fullSdfData[ (origIz*initialSdfResolution.y + origIy)*initialSdfResolution.x + origIx ];
			}
}

void Wall::createSdf(MPI_Comm& comm, float3 globalDomainSize, float3 globalDomainStart, float3 localDomainSize)
{
	info("Creating wall %s", name.c_str());

	CUDA_Check( hipDeviceSynchronize() );
	MPI_Check( MPI_Comm_dup(comm, &wallComm) );

	int nranks, rank;
	int ranks[3], periods[3], coords[3];
	MPI_Check( MPI_Comm_size(wallComm, &nranks) );
	MPI_Check( MPI_Comm_rank(wallComm, &rank) );
	MPI_Check( MPI_Cart_get (wallComm, 3, ranks, periods, coords) );

	int3 initialSdfResolution;
	float3 initialSdfExtent;

	int64_t fullSdfSize_byte;
	int64_t endHeader_byte;

	// Read header
	readHeader(initialSdfResolution, initialSdfExtent, fullSdfSize_byte, endHeader_byte, rank);
	float3 initialSdfH = globalDomainSize / make_float3(initialSdfResolution-1);

	// Read heavy data
	std::vector<float> fullSdfData;
	readSdf(fullSdfSize_byte, endHeader_byte, nranks, rank, fullSdfData);

	// We'll make sdf a bit bigger, so that particles that flew away
	// would also be correctly bounced back
	sdfInfo.extendedDomainSize = localDomainSize + 2.0f*margin3;
	sdfInfo.resolution         = make_int3( ceilf(sdfInfo.extendedDomainSize / sdfInfo.h) );
	sdfInfo.h                  = sdfInfo.extendedDomainSize / make_float3(sdfInfo.resolution-1);
	sdfInfo.invh               = 1.0f / sdfInfo.h;

	const float3 scale3 = globalDomainSize / initialSdfExtent;
	if ( fabs(scale3.x - scale3.y) > 1e-5 || fabs(scale3.x - scale3.z) > 1e-5 )
		die("Sdf size and domain size mismatch");
	const float lenScalingFactor = (scale3.x + scale3.y + scale3.z) / 3;

	int3 resolutionBeforeInterpolation;
	float3 offset;
	PinnedBuffer<float> localSdfData;
	prepareRelevantSdfPiece(fullSdfData.data(), globalDomainStart - margin3, initialSdfH, initialSdfResolution,
			resolutionBeforeInterpolation, offset, localSdfData);

	// Interpolate
	sdfRawData.resize(sdfInfo.resolution.x * sdfInfo.resolution.y * sdfInfo.resolution.z, 0);

	dim3 threads(8, 8, 8);
	dim3 blocks((sdfInfo.resolution.x+threads.x-1) / threads.x,
				(sdfInfo.resolution.y+threads.y-1) / threads.y,
				(sdfInfo.resolution.z+threads.z-1) / threads.z);

	localSdfData.uploadToDevice(0);
	cubicInterpolate3D<<< blocks, threads >>>(localSdfData.devPtr(), resolutionBeforeInterpolation, initialSdfH,
			sdfRawData.devPtr(), sdfInfo.resolution, sdfInfo.h, offset, lenScalingFactor);


	// Prepare array to be transformed into texture
	auto chDesc = hipCreateChannelDesc<float>();
	CUDA_Check( hipMalloc3DArray(&sdfArray, &chDesc, make_hipExtent(sdfInfo.resolution.x, sdfInfo.resolution.y, sdfInfo.resolution.z)) );

	hipMemcpy3DParms copyParams = {};
	copyParams.srcPtr = make_hipPitchedPtr((void*)sdfRawData.devPtr(), sdfInfo.resolution.x*sizeof(float), sdfInfo.resolution.x, sdfInfo.resolution.y);
	copyParams.dstArray = sdfArray;
	copyParams.extent = make_hipExtent(sdfInfo.resolution.x, sdfInfo.resolution.y, sdfInfo.resolution.z);
	copyParams.kind = hipMemcpyDeviceToDevice;

	CUDA_Check( hipMemcpy3D(&copyParams) );

	// Create texture
	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = sdfArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.addressMode[2]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	CUDA_Check( hipCreateTextureObject(&sdfInfo.sdfTex, &resDesc, &texDesc, nullptr) );

	CUDA_Check( hipDeviceSynchronize() );
}

void Wall::removeInner(ParticleVector* pv)
{
	CUDA_Check( hipDeviceSynchronize() );

	PinnedBuffer<int> nRemaining(1);
	nRemaining.clear(0);
	PinnedBuffer<Particle> tmp(pv->local()->size(), 0);

	const int nthreads = 128;
	// Need a different path for objects
	ObjectVector* ov = dynamic_cast<ObjectVector*>(pv);
	if (ov == nullptr)
	{
		collectRemaining<<< getNblocks(pv->local()->size(), nthreads), nthreads, 0, 0 >>>(
				(float4*)pv->local()->coosvels.devPtr(), pv->local()->size(), sdfInfo,
				(float4*)tmp.devPtr(), nRemaining.devPtr() );
	}
	else
	{
		collectRemainingObjects<<<  getNblocks(ov->local()->nObjects*32, nthreads), nthreads, 0, 0 >>> (
				(float4*)ov->local()->coosvels.devPtr(), ov->local()->nObjects, ov->objSize, sdfInfo,
				(float4*)tmp.devPtr(), nRemaining.devPtr() );
	}

	nRemaining.downloadFromDevice(0);
	containerSwap(pv->local()->coosvels, tmp, 0);
	pv->local()->resize(nRemaining[0], 0);
	pv->local()->changedStamp++;
	info("Keeping %d particles of %s", nRemaining[0], pv->name.c_str());

	CUDA_Check( hipDeviceSynchronize() );
}

void Wall::bounce(float dt, hipStream_t stream)
{
	for (int i=0; i<particleVectors.size(); i++)
	{
		auto pv = particleVectors[i];
		auto cl = cellLists[i];

		debug2("Bouncing %d %s particles", pv->local()->size(), pv->name.c_str());

		const int nthreads = 64;
		bounceSDF<<< getNblocks(boundaryCells[i].size(), nthreads), nthreads, 0, stream >>>(
				boundaryCells[i].devPtr(), boundaryCells[i].size(), cl->cellsStartSize.devPtr(), cl->cellInfo(),
				sdfInfo, (float4*)pv->local()->coosvels.devPtr(), dt);
	}
}

void Wall::check(hipStream_t stream)
{
	const int nthreads = 128;
	for (auto pv : particleVectors)
	{
		nInside.clearDevice(stream);
		checkInside<<< getNblocks(pv->local()->size(), nthreads), nthreads, 0, stream >>> (
				(float4*)pv->local()->coosvels.devPtr(), pv->local()->size(), sdfInfo, nInside.devPtr());
		nInside.downloadFromDevice(stream);

		if (nInside[0] > 0)
			warn ("%d particles of %s are inside the wall %s", nInside[0], pv->name.c_str(), name.c_str());
		else
			debug("%d particles of %s are inside the wall %s", nInside[0], pv->name.c_str(), name.c_str());
	}
}




