#include "hip/hip_runtime.h"
/*
 *  test.cu
 *  Part of CTC/logistic_rng/
 *
 *  Created and authored by Yu-Hang Tang on 2015-03-20.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

>>>>>>> origin
#include <cstdio>
#include <cstdlib>
#include <climits>
#include "logistic.h"

__global__ void generate( float *output, float trunk, int n_particle )
{
    for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_particle; i += gridDim.x * blockDim.x ) {
        for( int j = 0; j < n_particle; j++ ) {
            output[ i + j * n_particle ] = logistic<11, float>( trunk, i, j );
        }
    }
}

int main()
{
    int n = 100;
    
    srand( 0 );
    float trunk = double( rand() ) / RAND_MAX;

    float *output;
    hipHostMalloc( &output, n * n * sizeof( float ) );
    
    generate <<< 1, 512>>>( output, trunk, n );
    hipDeviceSynchronize();
    
    float *p = output;
    for( int i = 0; i < n; i++ ) {
        for( int j = 0; j < n; j++ ) {
            printf( "%f%c ", *p++, ( j == n - 1 ) ? '\n' : ' ' );
        }
    }
}
