#include "hip/hip_runtime.h"
/*
 *  main-cuda.cpp
 *  ctc phenix
 *
 *  Created by Dmitry Alexeev on Nov 10, 2014
 *  Copyright 2014 ETH Zurich. All rights reserved.
 *
 */


/*
 *  main.cpp
 *  ctc local
 *
 *  Created by Dmitry Alexeev on Nov 5, 2014
 *  Copyright 2014 ETH Zurich. All rights reserved.
 *
 */

#include "rbc-cuda.h"
#include "timer.h"
#include "misc.h"
#include "cuda-common.h"


#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>


using namespace std;

__global__ void _update_pos(real * const xyzuvw, const real f, const int n, const real L)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < n)
	{
		for(int c = 0; c < 3; ++c)
		{
			const real xold = xyzuvw[c + 6 * tid];

			real xnew = xold + f * xyzuvw[3 + c + 6 * tid];
			xnew -= L * floor((xnew + 0.5 * L) / L);

			xyzuvw[c + 6 * tid] = xnew;
		}
	}
}

__global__ void _update_vel(real * const xyzuvw, const real * const axayaz, const real f, const int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < n)
	{
		for(int c = 0; c < 3; ++c)
		{
			const real vold = xyzuvw[3 + c + 6 * tid];

			real vnew = vold + f * axayaz[c + 3 * tid];

			xyzuvw[3 + c + 6 * tid] = vnew;
		}
	}
}

__global__ void _diag_kbt(const real * const xyzuvw, real * const diag, const int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < n)
		diag[tid] =
				pow(xyzuvw[3 + 6 * tid], 2) +
				pow(xyzuvw[4 + 6 * tid], 2) +
				pow(xyzuvw[5 + 6 * tid], 2);
}

__global__ void _diag_p(const real * const xyzuvw, real * const diag, const int n, const int c)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < n)
		diag[tid] = xyzuvw[3 + c + 6 * tid];
}

void vmd_xyz(const char * path, real* _xyzuvw, const int n, bool append)
{
	real* xyzuvw = new real[6*n];
	gpuErrchk( hipMemcpy(xyzuvw, _xyzuvw, 6*n * sizeof(real), hipMemcpyDeviceToHost) );

	FILE * f = fopen(path, append ? "a" : "w");

	if (f == NULL)
	{
		printf("I could not open the file <%s>\n", path);
		printf("Aborting now.\n");
		abort();
	}

	fprintf(f, "%d\n", n);
	fprintf(f, "mymolecule\n");

	for(int i = 0; i < n; ++i)
		fprintf(f, "1 %f %f %f\n",
				(real)xyzuvw[0 + 6 * i],
				(real)xyzuvw[1 + 6 * i],
				(real)xyzuvw[2 + 6 * i]);

	fclose(f);
	delete[] xyzuvw;

	printf("vmd_xyz: wrote to <%s>\n", path);
}

void vmd_xyz_3comp(const char * path, real* _xyz, const int n, bool append)
{
	real* xyz = new real[3*n];
	gpuErrchk( hipMemcpy(xyz, _xyz, 3*n * sizeof(real), hipMemcpyDeviceToHost) );

	FILE * f = fopen(path, append ? "a" : "w");

	if (f == NULL)
	{
		printf("I could not open the file <%s>\n", path);
		printf("Aborting now.\n");
		abort();
	}

	fprintf(f, "%d\n", n);
	fprintf(f, "mymolecule\n");

	for(int i = 0; i < n; ++i)
		fprintf(f, "%d %f %f %f\n", i,
				(real)xyz[0 + 3 * i],
				(real)xyz[1 + 3 * i],
				(real)xyz[2 + 3 * i]);

	fclose(f);

	printf("vmd_xyz: wrote to <%s>\n", path);
}

class SimRBC
{
	int nparticles;
	const real L;
	real *xyzuvw, *fxfyfz;
	int ncells;
	hipEvent_t start, stop;

public:

	SimRBC(const real L, int ncells): L(L), ncells(ncells)
{
		CudaRBC::Extent extent;
		CudaRBC::Extent *devExtents, *hstExtents;
		CudaRBC::setup(nparticles, extent);

		gpuErrchk( hipMalloc(&xyzuvw, ncells * 6*nparticles*sizeof(real)) );
		gpuErrchk( hipMalloc(&fxfyfz, ncells * 3*nparticles*sizeof(real)) );
		gpuErrchk( hipMalloc(&devExtents, ncells * sizeof(CudaRBC::Extent)) );
		hstExtents = new CudaRBC::Extent[ncells];

		float A[4][4];
		memset(&A[0][0], 0, 16*sizeof(float));
		A[0][0] = A[1][1] = A[2][2] = A[3][3] = 1;

		for (int i=0; i<ncells; i++)
		{
			A[0][0] = A[1][1] = A[2][2] = 1 + 0.2*(drand48() - 0.5);
			A[2][3] += 4;
			CudaRBC::initialize(xyzuvw + i * 6*nparticles, A);
		}
		printf("initialized\n");

		CudaRBC::extent_nohost(0, ncells, xyzuvw, devExtents);
		gpuErrchk( hipMemcpy(hstExtents, devExtents, ncells * sizeof(CudaRBC::Extent), hipMemcpyDeviceToHost) );
		hipDeviceSynchronize();
		for (int i=0; i<ncells; i++)
		{
			printf("#%.3d:  [%.3f  %.3f], [%.3f  %.3f], [%.3f  %.3f]\n", i,
					hstExtents[i].xmin, hstExtents[i].xmax,
					hstExtents[i].ymin, hstExtents[i].ymax,
					hstExtents[i].zmin, hstExtents[i].zmax);
		}

		hipEventCreate(&start);
		hipEventCreate(&stop);
}

	void _diag(FILE ** fs, const int nfs, real t)
	{

	}

	void _f()
	{
		gpuErrchk( hipMemset(fxfyfz, 0, ncells * 3*nparticles * sizeof(real)) );

		hipEventRecord(start);
		//for (int i=0; i<ncells; i++)
			CudaRBC::forces_nohost(0, ncells, xyzuvw, fxfyfz);
		hipEventRecord(stop);
	};

	void run(const real tend, const real dt)
	{
		vmd_xyz("ic.xyz", xyzuvw, nparticles, false);

		FILE * fdiags[2] = {stdout, fopen("diag.txt", "w") };

		const size_t nt = (int)(tend / dt);

		_f();

		Timer tm;
		tm.start();

		float tottime = 0;

		for(int it = 0; it < nt; ++it)
		{
			//			if (it % 200 == 0)
			//			{
			//				real t = it * dt;
			//				_diag(fdiags, 2, t);
			//			}

			_update_vel<<<(ncells*nparticles + 127) / 128, 128>>>(xyzuvw, fxfyfz, dt * 0.5, ncells*nparticles);


			_update_pos<<<(ncells*nparticles + 127) / 128, 128>>>(xyzuvw, dt, ncells*nparticles, L);


			_f();

			_update_vel<<<(ncells*nparticles + 127) / 128, 128>>>(xyzuvw, fxfyfz, dt * 0.5, ncells*nparticles);

			float interval;
			hipEventSynchronize(stop);
			hipEventElapsedTime(&interval, start, stop);
			tottime += interval;

			if (it % 20 == 0)
			{
				vmd_xyz("evolution.xyz", xyzuvw, ncells*nparticles, it > 0);
				//vmd_xyz_3comp("force.xyz", fxfyfz, nparticles, it > 0);
			}
		}

		printf("Avg time per step is %.4f  ms, forces took %.5f ms\n", tm.elapsed() / 1e6 / nt, tottime / nt);

		fclose(fdiags[1]);
	}
};

int main()
{
	printf("hello rbc-gpu test\n");

	real L = 10000; //  /Volumes/Phenix/CTC/vanilla-rbc/evolution.xyz

	SimRBC sim(L, 50);

	sim.run(1, 0.001);

	return 0;
}

