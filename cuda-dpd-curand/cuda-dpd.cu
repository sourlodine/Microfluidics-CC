#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

//#define _CHECK_

const int collapsefactor = 1;
 
struct InfoDPD
{
    int3 ncells;
    int np, nsamples, rsamples_start;
    float3 domainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
    float *xyzuvw, *axayaz, *rsamples;
};

__constant__ InfoDPD info;
 
#include "cell-lists.h"

const int xbs = 16;
const int ybs = 6;

__device__ void _ftable(
    float p1[3][ybs], float p2[3][xbs], float v1[3][ybs], float v2[3][xbs],
    const int np1, const int np2, const int nonzero_start, const int rsamples_start,
    float a1[3][ybs], float a2[3][xbs])
{
    assert(np2 <= xbs);
    assert(np1 <= ybs);
    assert(np1 <= xbs * ybs);
    assert(blockDim.x == xbs && xbs == xbs);
    assert(blockDim.y == ybs);

    const int lx = threadIdx.x;
    const int ly = threadIdx.y;

    float xmyforce = 0, ymyforce = 0, zmyforce = 0;
    
    {
	const bool valid = (lx < np2 && ly < np1) * (lx > ly + nonzero_start);
	
	float xr = p1[0][ly] - p2[0][lx];
	float yr = p1[1][ly] - p2[1][lx];
	float zr = p1[2][ly] - p2[2][lx];
				
	xr -= info.domainsize.x * floorf(0.5f + xr / info.domainsize.x);
	yr -= info.domainsize.y * floorf(0.5f + yr / info.domainsize.y);
	zr -= info.domainsize.z * floorf(0.5f + zr / info.domainsize.z);

	const float rij2 = xr * xr + yr * yr + zr * zr;
	const float invrij = rsqrtf(rij2);
	const float rij = rij2 * invrij;
	const float wr = max((float)0, 1 - rij * info.invrc);
	
	xr *= invrij;
	yr *= invrij;
	zr *= invrij;

	const float rdotv = xr * (v1[0][ly] - v2[0][lx]) + yr * (v1[1][ly] - v2[1][lx]) + zr * (v1[2][ly] - v2[2][lx]);

	int entry = lx + np2 * ly;
	const float myrandnr = info.rsamples[(info.rsamples_start + rsamples_start + entry) % info.nsamples];
#if 0
	assert(myrandnr != -313);
	info.rsamples[(info.rsamples_start + rsamples_start + entry) % info.nsamples] = -313;
#endif
	
	const float strength = (info.aij - info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;

	if (valid)
	{
#ifdef _CHECK_
	    xmyforce = (rij2 < 1);
	    ymyforce = 0;
	    zmyforce = 0;
#else		    
	    xmyforce = strength * xr;
	    ymyforce = strength * yr;
	    zmyforce = strength * zr;
#endif
	}
    }

    {
	assert(xbs == 16 && warpSize == 32);

	float xmysum = xmyforce, ymysum = ymyforce, zmysum = zmyforce;
	
#pragma unroll
	for(int l = 1; l < 16; l <<= 1)
	{
	    const float xother = __shfl_xor(xmysum, l);
	    const float yother = __shfl_xor(ymysum, l);
	    const float zother = __shfl_xor(zmysum, l);

	    xmysum += xother;
	    ymysum += yother;
	    zmysum += zother;
	}

	if (lx == 0 && ly < np1)
	{
	    a1[0][ly] = xmysum;
	    a1[1][ly] = ymysum;
	    a1[2][ly] = zmysum;
	}
    }

    {
	assert(xbs == 16 && warpSize == 32 && ybs == 6);

	__shared__ float buf[3][3][16];

	xmyforce += __shfl_xor(xmyforce, 16);
	ymyforce += __shfl_xor(ymyforce, 16);
	zmyforce += __shfl_xor(zmyforce, 16);
	
	if ((ly & 1) == 0)
	{
	    const int entry = ly >> 1;
	    buf[0][entry][lx] = xmyforce;
	    buf[1][entry][lx] = ymyforce;
	    buf[2][entry][lx] = zmyforce;
	}

	__syncthreads();

	if (lx < np2 && threadIdx.y == 0)
	{
	    a2[0][lx] = buf[0][0][lx] + buf[0][1][lx] + buf[0][2][lx];
	    a2[1][lx] = buf[1][0][lx] + buf[1][1][lx] + buf[1][2][lx];
	    a2[2][lx] = buf[2][0][lx] + buf[2][1][lx] + buf[2][2][lx];
	}
    }
}

texture<float, hipTextureType1D> texParticles;

__device__ void _cellscells(const int p1start[4], const int p1count[4], const int p2start[4][4], const int p2scan[4][4],
			    const int maxnp1, const int maxnp2, const bool self, int rsamples_start,
			    float * const axayaz)
{
    __shared__ float pva1[9][ybs], pva2[9][xbs];

    const int l = threadIdx.x + blockDim.x * threadIdx.y;
    const int BS = blockDim.x * blockDim.y;
      
    for(int ty = 0; ty < maxnp1; ty += ybs)
    {
	int np1[4];
	
#pragma unroll
	for(int i = 0; i < 4; ++i)
	    np1[i] = max(0, min(ybs, p1count[i] - ty));
	
#pragma unroll
	for(int i = 0; i < 4; ++i)
	    assert(BS >= np1[i] * 6);

	float pva1contrib[4];
#pragma unroll
	for(int i = 0; i < 4; ++i)
	    if (l < np1[i] * 6)
		pva1contrib[i] = tex1Dfetch(texParticles, 6 * (p1start[i] + ty) + l);

	float pva1result[4] = {0, 0, 0, 0};
	
	for(int tx = 0; tx < maxnp2; tx += xbs)
	{
	    int np2[4];
#pragma unroll
	    for(int i = 0; i < 4; ++i)
		np2[i] = max(0, min(xbs, p2scan[i][3] - tx));

	    float pva2contrib[4];
#pragma unroll
	    for(int i = 0; i < 4; ++i)
	    	if (l < np2[i] * 6)
		{
		    const int d = l / 6;
		    const int s = tx + d;
		    const int c = l % 6;
		    const int entry = (s >= p2scan[i][0]) + (s >= p2scan[i][1]) + (s >= p2scan[i][2]);
		    const int pid = s - (entry ? p2scan[i][entry - 1] : 0) + p2start[i][entry];

		    pva2contrib[i] = tex1Dfetch(texParticles, c + 6 * pid);
		}

	    float pva2result[4] = {0, 0, 0, 0};
	   
#pragma unroll
	    for(int i = 0; i < 4; ++i)
	    {
		if (np1[i] * np2[i] == 0)
		    continue;
		
		if (l < np1[i] * 6)
		    pva1[l % 6][l / 6] = pva1contrib[i];

		assert(np2[i] * 6 <= BS);
		assert(BS >= np2[i] * 3);
	   
		if (l < np2[i] * 6)
		    pva2[l % 6][l / 6] = pva2contrib[i];

		__syncthreads();

		_ftable(pva1, pva2, &pva1[3], &pva2[3], np1[i], np2[i], i == 0 ? ty - tx : -30000, rsamples_start, &pva1[6], &pva2[6]);

		__syncthreads();
		
		rsamples_start += np1[i] * np2[i];

		if (l < np1[i] * 3)
		    pva1result[i] += pva1[6 + (l % 3)][l / 3];
		
		if (l < np2[i] * 3)
		    pva2result[i] = pva2[6 + (l % 3)][l / 3];
		
		__syncthreads();
	    }
	    
#pragma unroll
	    for(int i = 0; i < 4; ++i)
	    {
		if (l < np2[i] * 3)
		{
		    const int s = l / 3;
		    const int d = tx + s;
		    const int c = l % 3;
		    const int entry = (d >= p2scan[i][0]) + (d >= p2scan[i][1]) + (d >= p2scan[i][2]);
		    const int pid = d - (entry ? p2scan[i][entry - 1] : 0) + p2start[i][entry];
#ifdef _CHECK_
		    atomicAdd(axayaz + c + 3 * pid, pva2result[i]);
#else
		    atomicAdd(axayaz + c + 3 * pid, -pva2result[i]);
#endif
		}
	    }
	}

#pragma unroll
	for(int i = 0; i < 4; ++i)
	    assert(np1[i] * 3 <= BS);

	float oldval[4];
#pragma unroll
	for(int i = 0; i < 4; ++i)
	    if (l < np1[i] * 3)
		oldval[i] = axayaz[l + 3 * (p1start[i] + ty)];

#pragma unroll
	for(int i = 0; i < 4; ++i)
	    if (l < np1[i] * 3)
		axayaz[l + 3 * (p1start[i] + ty)] = pva1result[i] + oldval[i];
    }
}

__device__ int _cid(int shiftcode)
{
    int3 indx = make_int3(blockIdx.x, blockIdx.y, blockIdx.z); 
	    
    indx.x += (shiftcode & 1);
    indx.y += ((shiftcode >> 1) & 1);
    indx.z += ((shiftcode >> 2) & 1);
	    
    indx.x = (indx.x + info.ncells.x) % info.ncells.x;
    indx.y = (indx.y + info.ncells.y) % info.ncells.y;
    indx.z = (indx.z + info.ncells.z) % info.ncells.z;

    return indx.x + info.ncells.x * (indx.y + info.ncells.y * indx.z);//encode(indx.x, indx.y, indx.z);
}

__constant__ int edgeslutcount[4] = {4, 4, 3, 3};
__constant__ int edgeslutstart[4] = {0, 4, 8, 11};
__constant__ int edgeslut[14] = {0, 1, 2, 7, 2, 4, 6, 7, 4, 5, 7, 4, 0, 7};

texture<int, hipTextureType1D> texStart, texEnd;

__global__ void _dpd_forces(float * tmp, int * consumed)
{
    const int idbuf = (blockIdx.x & 1) | ((blockIdx.y & 1) << 1) | ((blockIdx.z & 1) << 2);
    float * const axayaz = tmp + 3 * info.np * idbuf;
    
    const bool master = threadIdx.x + threadIdx.y == 0;
    const int l = threadIdx.x + blockDim.x * threadIdx.y;

    __shared__ int offsetrsamples, rconsumption, maxnp1, maxnp2;
    __shared__ int p1starts[4], p1counts[4];
    __shared__ int p2starts[4][4], p2scans[4][4];

    if (master)
	rconsumption = 0;

    if (l < 4 * 4)
    {
	const int i = l / 4;
	const int j = l % 4;

	if (j == 0)
	{
	    const int cid1 = _cid(i);
	    p1starts[i] = tex1Dfetch(texStart, cid1);
	    p1counts[i] = tex1Dfetch(texEnd, cid1);
	    
	    assert( tex1Dfetch(texEnd, cid1) - tex1Dfetch(texStart, cid1) >= 0);
	}
		
	if (j < edgeslutcount[i])
	{
	    const int cid2 = _cid(edgeslut[j + edgeslutstart[i]]);
	    
	    p2starts[i][j] = tex1Dfetch(texStart, cid2);
	    p2scans[i][j] = tex1Dfetch(texEnd, cid2);

	    assert( tex1Dfetch(texEnd, cid2) - tex1Dfetch(texStart, cid2) >= 0);
	}
	else
	    p2scans[i][j] = p2starts[i][j] = 0;

	if (j == 0)
	    p1counts[i] -= p1starts[i];
	
	int myp1count = __shfl(p1counts[i], i * 4 + 0);
	myp1count = max(myp1count, __shfl_xor(myp1count, 8));
	myp1count = max(myp1count, __shfl_xor(myp1count, 4));
	
	if (master)
	    maxnp1 = myp1count;
			
	int entryscan = p2scans[i][j] - p2starts[i][j];
	
	entryscan += (j >= 1) * __shfl_up(entryscan, 1);
	entryscan += (j >= 2) * __shfl_up(entryscan, 2);
	p2scans[i][j] = entryscan;
	
	const int r0 = entryscan * p1counts[i];
	const int e1m = __shfl_xor(entryscan, 4);
	const int e1r = __shfl_xor(r0, 4);
	const int m1 = max(entryscan, e1m);
	const int r1 = r0 + e1r;
	const int e2m = __shfl_xor(m1, 8);
	const int e2r = __shfl_xor(r1, 8);
	const int m2 = max(m1, e2m);
	const int r2 = r1 + e2r;
	
	if (l == 3)
	{
	    maxnp2 = m2;
	    rconsumption = r2;
	    offsetrsamples = atomicAdd(consumed, rconsumption);
	}
    }

    __syncthreads();    

    
    if (offsetrsamples + rconsumption >= info.nsamples)
	return;
    
    _cellscells(p1starts, p1counts, p2starts, p2scans, maxnp1, maxnp2, true, offsetrsamples, axayaz);
}

__global__ void _reduce(float * tmp)
{
    assert(gridDim.x * blockDim.x >= info.np);
    
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < info.np * 3)
    {
	const int nbufs = 8;

	float s = 0;
	for(int idbuf = 0; idbuf < nbufs ; ++idbuf)
	    s += tmp[tid + 3 * info.np * idbuf];
	
	info.axayaz[tid] = s;
    }
}

#include <cmath>
#include <unistd.h>

#include <thrust/device_vector.h>
using namespace thrust;

#include "profiler-dpd.h"
#include "rring-buffer.h"

#define CUDA_CHECK(ans) do { cudaAssert((ans), __FILE__, __LINE__); } while(0)
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
	fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	sleep(5);
	if (abort) exit(code);
    }
}

template<typename T> T * _ptr(device_vector<T>& v) { return raw_pointer_cast(v.data()); }

ProfilerDPD * myprof = NULL;
RRingBuffer * rrbuf = NULL;

void forces_dpd_cuda(float * const _xyzuvw, float * const _axayaz,
		     int * const order, const int np,
		     const float rc,
		     const float XL, const float YL, const float ZL,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
		     float * const _rsamples, int nsamples)
{
    static bool initialized = false;

    if (!initialized)
    {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	if (!prop.canMapHostMemory)
	{
	    printf("Capability zero-copy not there! Aborting now.\n");
	    abort();
	}
	else
	    hipSetDeviceFlags(hipDeviceMapHost);

//	CUDA_CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));  
	
	initialized = true;
    }

    if (rrbuf == NULL)
	rrbuf = new RRingBuffer(50 * np * 3 * collapsefactor * collapsefactor * collapsefactor);

    if (myprof == NULL)
#ifdef _PROFILE_
	myprof = new ProfilerDPD(true);
#else
    myprof = new ProfilerDPD(false);
#endif
    
    int nx = (int)ceil(XL / (collapsefactor *rc));
    int ny = (int)ceil(YL / (collapsefactor *rc));
    int nz = (int)ceil(ZL / (collapsefactor *rc));
    const int ncells = nx * ny * nz;
        
    device_vector<float> xyzuvw(_xyzuvw, _xyzuvw + np * 6), axayaz(np * 3);
    
    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.np = np;
    c.domainsize = make_float3(XL, YL, ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
    c.xyzuvw = _ptr(xyzuvw);
    c.axayaz = _ptr(axayaz);
    c.nsamples = rrbuf->nsamples();
    c.rsamples = rrbuf->buffer();
    c.rsamples_start = rrbuf->start();

    device_vector<float> rsamples;
    if (_rsamples != NULL)
    {
	rsamples.resize(nsamples);
	copy(_rsamples, _rsamples + nsamples, rsamples.begin());

	c.nsamples = nsamples;
	c.rsamples = _ptr(rsamples);
	c.rsamples_start = 0;
    }
    else
	nsamples = rrbuf->nsamples();
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(info), &c, sizeof(c)));

    device_vector<int> starts(ncells + 1), ends(ncells + 1);
    build_clists(_ptr(xyzuvw), np, rc, c.ncells.x, c.ncells.y, c.ncells.z,
		 c.domainstart.x, c.domainstart.y, c.domainstart.z,
		 order, _ptr(starts), _ptr(ends));

    {
	size_t textureoffset = 0;
	hipChannelFormatDesc fmt =  hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
	texStart.channelDesc = fmt;
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;
	hipBindTexture(&textureoffset, &texStart, _ptr(starts), &fmt, sizeof(int) * (ncells + 1));

	texEnd.channelDesc = fmt;
	texEnd.filterMode = hipFilterModePoint;
	texEnd.mipmapFilterMode = hipFilterModePoint;
	texEnd.normalized = 0;
	hipBindTexture(&textureoffset, &texEnd, _ptr(ends), &fmt, sizeof(int) * (ncells + 1));
	
	fmt =  hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	texParticles.channelDesc = fmt;
	texParticles.filterMode = hipFilterModePoint;
	texParticles.mipmapFilterMode = hipFilterModePoint;
	texParticles.normalized = 0;
	hipBindTexture(&textureoffset, &texParticles, c.xyzuvw, &fmt, sizeof(float) * 6 * np);
    }
    
    int * consumed = NULL;
    hipHostAlloc((void **)&consumed, sizeof(int), hipHostMallocMapped);
    assert(consumed != NULL);
    *consumed = 0;

    {
	float * tmp;

	const int nreplica = 24;

	CUDA_CHECK(hipMalloc(&tmp, sizeof(float) * np * nreplica));
	CUDA_CHECK(hipMemset(tmp, 0, sizeof(float) * np * nreplica));
	
	int * dconsumed = NULL;
	hipHostGetDevicePointer(&dconsumed, consumed, 0);

	myprof->start();
	
	_dpd_forces<<<dim3(c.ncells.x, c.ncells.y, c.ncells.z), dim3(xbs, ybs, 1)>>>(tmp, dconsumed);

	myprof->force();
	CUDA_CHECK(hipPeekAtLastError());

	_reduce<<<(3 * np + 127) / 128, 128>>>(tmp);
	myprof->reduce();
	CUDA_CHECK(hipPeekAtLastError());
	
	CUDA_CHECK(hipFree(tmp));
	
	if (*consumed >= nsamples)
	{
	    printf("done with code %d: consumed: %d\n", 7, *consumed);
	    printf("not a nice situation.\n");
	    abort();
	}
    }
	
    myprof->report();
    
    if (_rsamples == NULL)
	rrbuf->update(*consumed);
    
    hipHostFree(consumed);
   
    copy(xyzuvw.begin(), xyzuvw.end(), _xyzuvw);
    copy(axayaz.begin(), axayaz.end(), _axayaz);
    
#ifdef _CHECK_
    CUDA_CHECK(hipDeviceSynchronize());
    
    for(int i = 0; i < np; ++i)
    {
	printf("pid %d -> %f %f %f\n", i, (float)axayaz[0 + 3 * i], (float)axayaz[1 + 3* i], (float)axayaz[2 + 3 *i]);

	int cnt = 0;
	float fc = 0;
	printf("devi coords are %f %f %f\n", (float)xyzuvw[0 + 6 * i], (float)xyzuvw[1 + 6 * i], (float)xyzuvw[2 + 6 * i]);
	printf("host coords are %f %f %f\n", (float)_xyzuvw[0 + 6 * i], (float)_xyzuvw[1 + 6 * i], (float)_xyzuvw[2 + 6 * i]);
	
	for(int j = 0; j < np; ++j)
	{
	    if (i == j)
		continue;
 
	    float xr = _xyzuvw[0 + 6 *i] - _xyzuvw[0 + 6 * j];
	    float yr = _xyzuvw[1 + 6 *i] - _xyzuvw[1 + 6 * j];
	    float zr = _xyzuvw[2 + 6 *i] - _xyzuvw[2 + 6 * j];

	    xr -= c.domainsize.x *  ::floor(0.5f + xr / c.domainsize.x);
	    yr -= c.domainsize.y *  ::floor(0.5f + yr / c.domainsize.y);
	    zr -= c.domainsize.z *  ::floor(0.5f + zr / c.domainsize.z);

	    const float rij2 = xr * xr + yr * yr + zr * zr;
	    const float invrij = rsqrtf(rij2);
	    const float rij = rij2 * invrij;
	    const float wr = max((float)0, 1 - rij * c.invrc);
	
	    const bool collision =  rij2 < 1;

	    if (collision)
		fc += wr;//	printf("ref p %d colliding with %d\n", i, j);
	    
	    cnt += collision;
	}
	printf("i found %d host interactions and with cuda i found %d\n", cnt, (int)axayaz[0 + 3 * i]);
	assert(cnt == (float)axayaz[0 + 3 * i]);
	printf("fc aij ref %f vs cuda %e\n", fc,  (float)axayaz[1 + 3 * i]);
	assert(fabs(fc - (float)axayaz[1 + 3 * i]) < 1e-4);
    }
    
    printf("test done.\n");
    sleep(1);
    exit(0);
#endif
}

void forces_dpd_cuda(float * const xp, float * const yp, float * const zp,
		     float * const xv, float * const yv, float * const zv,
		     float * const xa, float * const ya, float * const za,
		     int * const order, const int np,
		     const float rc,
		     const float LX, const float LY, const float LZ,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
		     float * const rsamples, int nsamples)
{
    float * pv = new float[6 * np];

    for(int i = 0; i < np; ++i)
    {
	pv[0 + 6 * i] = xp[i];
	pv[1 + 6 * i] = yp[i];
	pv[2 + 6 * i] = zp[i];
	pv[3 + 6 * i] = xv[i];
	pv[4 + 6 * i] = yv[i];
	pv[5 + 6 * i] = zv[i];
    }

    float * a = new float[3 * np];
    
    forces_dpd_cuda(pv, a, order, np, rc, LX, LY, LZ,
		    aij, gamma, sigma, invsqrtdt, rsamples,  nsamples);

    for(int i = 0; i < np; ++i)
    {
	xp[i] = pv[0 + 6 * i]; 
	yp[i] = pv[1 + 6 * i]; 
	zp[i] = pv[2 + 6 * i]; 
	xv[i] = pv[3 + 6 * i]; 
	yv[i] = pv[4 + 6 * i]; 
	zv[i] = pv[5 + 6 * i];
    }

    delete [] pv;
     
    for(int i = 0; i < np; ++i)
    {
	xa[i] = a[0 + 3 * i];
	ya[i] = a[1 + 3 * i];
	za[i] = a[2 + 3 * i];
    }

    delete [] a;
}