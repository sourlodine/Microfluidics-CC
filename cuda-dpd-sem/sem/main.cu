#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cmath>
#include <cstdio>
#include <cassert>

#include <algorithm>

#include <thrust/device_vector.h>

#include "cuda-sem.h"
#include "../hacks.h"

__global__ void _update_pos(float * const xyzuvw, const float f, const int n, const float L)
{
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n)
    {
	for(int c = 0; c < 3; ++c)
	{
	    const float xold = xyzuvw[c + 6 * tid];
	    
	    float xnew = xold + f * xyzuvw[3 + c + 6 * tid];
	    xnew -= L * floor((xnew + 0.5 * L) / L);
	    
	    xyzuvw[c + 6 * tid] = xnew;
	}
    }
}

__global__ void _update_vel(float * const xyzuvw, const float * const axayaz, const float f, const int n)
{
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n)
    {
	for(int c = 0; c < 3; ++c)
	{
	    const float vold = xyzuvw[3 + c + 6 * tid];
	    
	    float vnew = vold + f * axayaz[c + 3 * tid];
	    	    
	    xyzuvw[3 + c + 6 * tid] = vnew;
	}
    }
}

__global__ void _diag_kbt(const float * const xyzuvw, float * const diag, const int n)
{
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n)
	diag[tid] =
	    pow(xyzuvw[3 + 6 * tid], 2) +
	    pow(xyzuvw[4 + 6 * tid], 2) +
	    pow(xyzuvw[5 + 6 * tid], 2);
}

__global__ void _diag_p(const float * const xyzuvw, float * const diag, const int n, const int c)
{
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n)
	diag[tid] = xyzuvw[3 + c + 6 * tid];
}

using namespace thrust;

void vmd_xyz(const char * path, device_vector<float>& _xyzuvw, const int n, bool append)
{
    host_vector<float> xyzuvw(_xyzuvw);
    
    FILE * f = fopen(path, append ? "a" : "w");

    if (f == NULL)
    {
	printf("I could not open the file <%s>\n", path);
	printf("Aborting now.\n");
	abort();
    }
    
    fprintf(f, "%d\n", n);
    fprintf(f, "mymolecule\n");
    
    for(int i = 0; i < n; ++i)
	fprintf(f, "1 %f %f %f\n",
		(float)xyzuvw[0 + 6 * i],
		(float)xyzuvw[1 + 6 * i],
		(float)xyzuvw[2 + 6 * i]);
    
    fclose(f);

    printf("vmd_xyz: wrote to <%s>\n", path);
}

class SimSEM
{
    const int n;
    const float L;
    device_vector<float> xyzuvw, axayaz, diag;

public:

    SimSEM(const int n, const float L): n(n), L(L), xyzuvw(6 * n), axayaz(3 * n), diag(n)
	{
	    srand48(6516L);
	    for(int i = 0; i < n; ++i)
	    {
		xyzuvw[0 + 6 *i] = -L * 0.5f +  drand48() * L;
		xyzuvw[1 + 6 *i] = -L * 0.5f +  drand48() * L;
		xyzuvw[2 + 6 *i] = -L * 0.5f +  drand48() * L;	
	    }
	}

    void _diag(FILE ** fs, const int nfs, float t)
	{
	    _diag_kbt<<< (n + 127) / 128, 128 >>>(_ptr(xyzuvw), _ptr(diag), n);
	    const float sv2 = reduce(diag.begin(), diag.end());
	    float T = 0.5 * sv2 / (n * 3. / 2);
	    
	    float p[3];
	    for(int c = 0; c < 3; ++c)
	    {
		_diag_p<<< (n + 127) / 128, 128 >>>(_ptr(xyzuvw), _ptr(diag), n, 0);
		p[c] = reduce(diag.begin(), diag.end());
	    }

	    for(int i = 0; i < nfs; ++i)
	    {
		FILE * f = fs[i];
		
		if (ftell(f) == 0)
		    fprintf(f, "TIME\tkBT\tX-MOMENTUM\tY-MOMENTUM\tZ-MOMENTUM\n");

		fprintf(f, "%s %+e\t%+e\t%+e\t%+e\t%+e\n", (f == stdout ? "DIAG:" : ""), t, T, p[0], p[1], p[2]);
	    }
	}

     void _f(const float dt)
	{
	    //np,  rc,  LX, LY, LZ,  gamma, temp, dt,   u0,    rho,  req, D
	    //1e3, 1.0, 10, 10, 10,  80,    0.1,  0.01, 0.001, 1.5,  0.85, 0.0001
	    const float rcutoff = 2.5, gamma = 80, temp = 0.1, u0 = 0.001, rho = 1.5, req = 0.85, D = .0001, rc = 1;
	 
	    forces_sem_cuda_nohost(_ptr(xyzuvw), _ptr(axayaz), NULL,
		    n, rcutoff, L, L, L, gamma, temp, dt, u0, rho, req, D, rc);
	};
    
    void run(const double tend, const double dt)
	{
	    vmd_xyz("ic.xyz", xyzuvw, n, false);

	    FILE * fdiags[2] = {stdout, fopen("diag.txt", "w") };

	    const size_t nt = (int)(tend / dt);

	    _f(dt);
	    
	    for(int it = 0; it < nt; ++it)
	    {
		if (it % 200 == 0)
		{
		    float t = it * dt;
		    _diag(fdiags, 2, t);
		}
		
		_update_vel<<<(n + 127) / 128, 128>>>(_ptr(xyzuvw), _ptr(axayaz), dt * 0.5, n);
		
		_update_pos<<<(n + 127) / 128, 128>>>(_ptr(xyzuvw), dt, n, L);
		
		_f(dt);

		_update_vel<<<(n + 127) / 128, 128>>>(_ptr(xyzuvw), _ptr(axayaz), dt * 0.5, n);
		
		if (it % 200 == 0)
		    vmd_xyz("evolution.xyz", xyzuvw, n, it > 0);
	    }

	    fclose(fdiags[1]);
	}
};

int main()
{
    printf("hello gpu only test\n");
    
    float L = 20;

    const float Nm = .25;
    const int n = L * L * L * Nm;

    SimSEM sim(n, L);
       
    sim.run(8, 0.01);
    
    return 0;
}
