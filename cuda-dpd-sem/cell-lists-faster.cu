#include "hip/hip_runtime.h"
/*
 *  cell-lists-faster.cu
 *  Part of CTC/cuda-dpd-sem/
 *
 *  Created and authored by Diego Rossinelli on 2014-08-07.
 *  Edited by Massimo Bernaschi on 2014-03-30.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdio>
#include <cassert>

//#define TEST_MAURO

texture<float, hipTextureType1D> texParticlesCLS;

__device__ int  blockscount = 0;
 
template<int ILP, int SLOTS, int WARPS>
__global__ void yzhistogram(const int np,
			    const float invrc, const int3 ncells, 
			    const float3 domainstart,
			    int * const yzcid,
			    int * const localoffsets,
			    int * const global_yzhisto,
			    int * const global_yzscan,
                            int * const max_yzcount,
			    int * const gmemhisto)
{
    extern __shared__ int shmemhisto[];

    assert(blockDim.y == 1);
    assert(blockDim.x == warpSize * WARPS);

    const int tid = threadIdx.x;
#ifndef TEST_MAURO
    const int slot = tid % (SLOTS);
#endif
    const int gsize = gridDim.x * blockDim.x;
    const int nhisto = ncells.y * ncells.z;

    const int tile = blockIdx.x * blockDim.x;
    
    if (tile >= np)
	return;
        
 
    float y[ILP], z[ILP];
#ifdef TEST_MAURO
    float x[ILP];
#endif
    for(int j = 0; j < ILP; ++j)
    {
	const int g = tile + tid + gsize * j;

	y[j] = z[j] = -1;

	if (g < np)
	{
#ifdef TEST_MAURO
	    x[j] = tex1Dfetch(texParticlesCLS, 0 + 6 * g); 
#endif
	    y[j] = tex1Dfetch(texParticlesCLS, 1 + 6 * g); 
	    z[j] = tex1Dfetch(texParticlesCLS, 2 + 6 * g); 
	}
    }

//    __syncthreads();
	
    int entries[ILP];
#ifndef TEST_MAURO
    int offset[ILP];
#endif
    for(int j = 0; j < ILP; ++j)
    {
	const int g = tile + tid + gsize * j;
	    
#ifdef TEST_MAURO
	int xcid = min(ncells.x - 1, max(0, (int)floor(invrc * (x[j] - domainstart.x))));
#endif
	int ycid = min(ncells.y - 1, max(0, (int)(floor(y[j] - domainstart.y) * invrc)));
	int zcid = min(ncells.z - 1, max(0, (int)(floor(z[j] - domainstart.z) * invrc)));
	    
	assert(ycid >= 0 && ycid < ncells.y);
	assert(zcid >= 0 && zcid < ncells.z);

	entries[j] = -1;
#ifndef TEST_MAURO
	offset[j] = -1;
#endif
	if (g < np)
	{
	    entries[j] =  ycid + ncells.y * zcid;
#ifndef TEST_MAURO
	    offset[j] = atomicAdd(gmemhisto + (blockIdx.x*nhisto*SLOTS) + entries[j] + slot * nhisto, 1);
#else
	    localoffsets[g] = (atomicAdd(global_yzhisto + entries[j], 1) & 0x00FFFFFFFF) | (xcid << 24);
	    yzcid[g] = entries[j];
#endif
	}
    }
#ifndef TEST_MAURO
    __syncthreads();

    for(int i = tid ; i < SLOTS * nhisto; i += blockDim.x)
        shmemhisto[i] = gmemhisto[i + (blockIdx.x * nhisto * SLOTS)];

    __syncthreads();
	
    for(int s = 1; s < SLOTS; ++s)
    {
	for(int i = tid ; i < nhisto; i += blockDim.x)
	    shmemhisto[i + s * nhisto] += shmemhisto[i + (s - 1) * nhisto];

	__syncthreads();
    }

    if (slot > 0)
	for(int j = 0; j < ILP; ++j)
	    offset[j] += shmemhisto[entries[j] + (slot - 1) * nhisto];
	
    __syncthreads();
	
    for(int i = tid ; i < nhisto; i += blockDim.x)
	shmemhisto[i] = atomicAdd(global_yzhisto + i, shmemhisto[i + (SLOTS - 1) * nhisto]);

    __syncthreads();

    for(int j = 0; j < ILP; ++j)
    {
	const int g = tile + tid + gsize * j;
	    
	if (g < np)
	{
	    yzcid[g] = entries[j];
	    localoffsets[g] = offset[j] + shmemhisto[entries[j]];
	}
    }
#endif
    //__threadfence();
    __syncthreads();
    __shared__ bool lastone;

    if (tid == 0)
    {
	lastone = gridDim.x - 1 == atomicAdd(&blockscount, 1);
	
	if (lastone)
	    blockscount = 0;
    }

    __syncthreads();
    if (lastone)
    {
	for(int i = tid ; i < nhisto; i += blockDim.x)
	    shmemhisto[i] = global_yzhisto[i];

	if (max_yzcount != NULL)
	{
	    __syncthreads();

	    int mymax = 0;
	    for(int i = tid ; i < nhisto; i += blockDim.x)
		mymax = max(shmemhisto[i], mymax);

	    for(int L = 16; L > 0; L >>=1)
		mymax = max(__shfl_xor(mymax, L), mymax);

	    __shared__ int maxies[WARPS];
	
	    if (tid % warpSize == 0)
		maxies[tid / warpSize] = mymax;
	
	    __syncthreads();

	    mymax = 0;
	
	    if (tid < WARPS)
		mymax = maxies[tid];

	    for(int L = 16; L > 0; L >>=1)
		mymax = max(__shfl_xor(mymax, L), mymax);

	    if (tid == 0)
		*max_yzcount = mymax;
	}
	
	const int bwork = blockDim.x * ILP;
	for(int tile = 0; tile < nhisto; tile += bwork)
	{
	    const int n = min(bwork, nhisto - tile);

	    __syncthreads();
	    
	    if (tile > 0 && tid == 0)
		shmemhisto[tile] += shmemhisto[tile - 1];
	    
	    for(int l = 1; l < n; l <<= 1)
	    {
		__syncthreads();
		
		int tmp[ILP];

		for(int j = 0; j < ILP; ++j)
		{
		    const int d = tid + j * blockDim.x;
		    
		    tmp[j] = 0;

		    if (d >= l && d < n) 
			tmp[j] = shmemhisto[d + tile] + shmemhisto[d + tile - l];
		}

		__syncthreads();

		for(int j = 0; j < ILP; ++j)
		{
		    const int d = tid + j * blockDim.x;

		    if (d >= l && d < n) 
			shmemhisto[d + tile] = tmp[j];
		}
	    }
	}

	__syncthreads();

	for(int i = tid ; i < nhisto; i += blockDim.x)
	    global_yzscan[i] = i == 0 ? 0 : shmemhisto[i - 1];
    }
}

texture<int, hipTextureType1D> texScanYZ;

template<int ILP>
__global__ void yzscatter(const int * const localoffsets,
			  const int * const yzcids,
			  const int np,
			  int * const outid)
{
    for(int j = 0; j < ILP; ++j)
    {
	const int g = threadIdx.x + blockDim.x * (j + ILP * blockIdx.x);

	if (g < np)
	{
	    const int yzcid = yzcids[g];
	    const int localoffset = localoffsets[g];
	    const int base = tex1Dfetch(texScanYZ, yzcid);
	
#ifndef TEST_MAURO
	    const int entry = base + localoffset;
	    outid[entry] = g;
#else
	    const int entry = base + (localoffset & 0x00FFFFFF);
	    outid[entry] = (g & 0x00FFFFFF) | (localoffset & 0xFF000000);
#endif
	}
    }
}

texture<int, hipTextureType1D> texCountYZ;

template<int YCPB>
__global__ void xgather(const int * const ids, const int np, const float invrc, const int3 ncells, const float3 domainstart,
			int * const starts, int * const counts,
			float * const xyzuvw, const int bufsize, int * const order, int *loffs)
{
    assert(gridDim.x == 1 && gridDim.y == ncells.y / YCPB && gridDim.z == ncells.z);
    assert(blockDim.x == warpSize);
    assert(blockDim.y == YCPB);
    
    extern __shared__ volatile int allhisto[];
    volatile int * const xhisto = &allhisto[ncells.x * threadIdx.y];
#ifndef TEST_MAURO
    volatile int * const loffset = &allhisto[YCPB * ncells.x + bufsize * threadIdx.y];
    volatile int * const reordered = &allhisto[YCPB * ncells.x + bufsize * (YCPB + threadIdx.y)];
#else
    volatile int * const reordered = &allhisto[YCPB * ncells.x + bufsize * threadIdx.y];
#endif
    const int tid = threadIdx.x;
    const int ycid = threadIdx.y + YCPB * blockIdx.y;

    if (ycid >= ncells.y)
	return;
    
    const int yzcid = ycid + ncells.y * blockIdx.z;
    const int start = tex1Dfetch(texScanYZ, yzcid);
    const int count = tex1Dfetch(texCountYZ, yzcid);
#ifdef TEST_MAURO
    loffs += start;
#endif
    if (count > bufsize)
    {
	//asm("trap ;");
	return; //something went wrong
    }
    
    for(int i = tid; i < ncells.x; i += warpSize)
	xhisto[i] = 0;
 
    for(int i = tid; i < count; i += warpSize)
    {
	const int g = start + i;
#ifndef TEST_MAURO
 	const int id = ids[g];
	const float x = tex1Dfetch(texParticlesCLS, 6 * id);
	const int xcid = min(ncells.x - 1, max(0, (int)floor(invrc * (x - domainstart.x))));
#else
 	int id = ids[g];
	const int xcid = id >> 24;
	id &= 0x00FFFFFF;
#endif
	const int val = atomicAdd((int *)(xhisto + xcid), 1);
	assert(xcid < ncells.x);
	assert(i < bufsize);
#ifndef TEST_MAURO	
	loffset[i] = val |  (xcid << 16);
#else
	loffs[i] = val |  (xcid << 16);
#endif
    }
    
    for(int i = tid; i < ncells.x; i += warpSize)
	counts[i + ncells.x * yzcid] = xhisto[i];

    for(int base = 0; base < ncells.x; base += warpSize)
    {
	const int n = min(warpSize, ncells.x - base);
	const int g = base + tid;
	
	int val = (tid == 0 && base > 0) ? xhisto[g - 1] : 0;

	if (tid < n)
	    val += xhisto[g];

	for(int l = 1; l < n; l <<= 1)
	    val += (tid >= l) * __shfl_up(val, l);

	if (tid < n)
	    xhisto[g] = val;
    }

    for(int i = tid; i < ncells.x; i += warpSize)
	starts[i + ncells.x * yzcid] = start + (i == 0 ? 0 : xhisto[i - 1]);
 
    if (yzcid == ncells.y * ncells.z - 1 && tid == 0)
    {
	starts[ncells.x * ncells.y * ncells.z] = start + xhisto[ncells.x - 1];
	counts[ncells.x * ncells.y * ncells.z] = 0;
    }

    for(int i = tid; i < count; i += warpSize)
    {
#ifndef TEST_MAURO	
	const int entry = loffset[i];
#else
	const int entry = loffs[i];
#endif
	const int xcid = entry >> 16;
	assert(xcid < ncells.x);
	const int loff = entry & 0xffff;

	const int dest = (xcid == 0 ? 0 : xhisto[xcid - 1]) + loff;

#ifndef TEST_MAURO
	reordered[dest] = ids[start + i];
#else
	reordered[dest] = ids[start + i] & 0x00FFFFFF;
#endif
    }

    const int nfloats = count * 6;
    const int base = 6 * start;
    
    //const int mystart = (32 - (base & 0x1f) + tid) & 0x1f;
    //for(int i = mystart; i < nfloats; i += warpSize)
    for(int i = tid; i < nfloats; i += warpSize)
    {
	const int c = i % 6;
	const int p = reordered[i / 6];
	assert(i / 6 < bufsize);
	
	xyzuvw[base + i] = tex1Dfetch(texParticlesCLS, c + 6 * p);
    }

    if (order != NULL)
	for(int i = tid; i < count; i += warpSize)
	    order[start + i] = reordered[i];
}


#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

using namespace thrust;

#include "hacks.h"

#include <utility>

struct FailureTest
{
    int bufsize;
    int * maxstripe, * dmaxstripe;

    FailureTest() : maxstripe(NULL), dmaxstripe(NULL)
	{
	    hipDeviceProp_t prop;
	    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
	    if (!prop.canMapHostMemory)
	    {
		printf("Capability zero-copy not there! Aborting now.\n");
		abort();
	    }
	    else
	    { 
		hipSetDeviceFlags(hipDeviceMapHost);
		hipError_t status = hipGetLastError ( );
		hipError_t status2 = hipPeekAtLastError();

		//printf("attempting to set MapHost..status:  %d -> %d\n", status == hipSuccess, status2 == hipSuccess);
	    }
	}

    static void callback_crash(hipStream_t stream, hipError_t status, void*  userData )
	{
	    FailureTest& f = *(FailureTest *)userData;
	    
	    if (*f.maxstripe > f.bufsize)
	    {
		printf("Ouch .. I would need to rollback. Maxstripe: %d, bufsize: %d\n", *f.maxstripe, f.bufsize);
		printf("Too late to recover this. Aborting now.\n");
		abort();
	    }
	}

    void reset() 
	{
	    if (maxstripe == NULL)
	    {
		CUDA_CHECK(hipHostAlloc((void **)&maxstripe, sizeof(int), hipHostMallocMapped));
		assert(maxstripe != NULL);
		
		CUDA_CHECK(hipHostGetDevicePointer(&dmaxstripe, maxstripe, 0));
	    }
	    
	    *maxstripe = 0; 
	}
} static failuretest;

struct is_gzero
{
    __host__ __device__
    bool operator()(const int &x)
	{
	    return  x > 0;
	}
};

bool clists_perfmon = false;
bool clists_robust = true;

float * xyzuvw_internal_copy = NULL;
int *loffsets = NULL, *yzcid = NULL, *outid = NULL, *dyzscan = NULL, *yzhisto = NULL, *gmemhistos = NULL;

hipEvent_t evstart, evacquire, evscatter, evgather;

bool initialized = false;
int old_np = 0, old_yzncells = 0, old_gmemhistos_size = 0;

void build_clists(float * const device_xyzuvw, int np, const float rc, 
		  const int xcells, const int ycells, const int zcells,
		  const float xdomainstart, const float ydomainstart, const float zdomainstart,
		  int * const order, int * device_cellsstart, int * device_cellscount,
		  std::pair<int, int *> * nonemptycells, hipStream_t stream, const float * const src_device_xyzuvw)
{
    assert(np > 0);
    
    const float3 domainstart = make_float3(xdomainstart, ydomainstart, zdomainstart);
    const int3 ncells = make_int3(xcells, ycells, zcells);
    const int yzncells = ycells * zcells;
    const float densitynumber = np / (float)(ncells.x * ncells.y * ncells.z);
    int xbufsize = (int)(ncells.x * densitynumber * 2);

 
     
    if (!initialized)
    {
	CUDA_CHECK(hipEventCreate(&evacquire));
	CUDA_CHECK(hipEventCreate(&evstart));
	CUDA_CHECK(hipEventCreate(&evscatter));
	CUDA_CHECK(hipEventCreate(&evgather));
	    
	texScanYZ.channelDesc = hipCreateChannelDesc<int>();
	texScanYZ.filterMode = hipFilterModePoint;
	texScanYZ.mipmapFilterMode = hipFilterModePoint;
	texScanYZ.normalized = 0;
    
	texCountYZ.channelDesc = hipCreateChannelDesc<int>();
	texCountYZ.filterMode = hipFilterModePoint;
	texCountYZ.mipmapFilterMode = hipFilterModePoint;
	texCountYZ.normalized = 0;

	texParticlesCLS.channelDesc = hipCreateChannelDesc<float>();
	texParticlesCLS.filterMode = hipFilterModePoint;
	texParticlesCLS.mipmapFilterMode = hipFilterModePoint;
	texParticlesCLS.normalized = 0;
	
	initialized = true;
    }

    if (old_np < np)
    {
	if (old_np > 0)
	{
	    CUDA_CHECK(hipFree(xyzuvw_internal_copy));
	    CUDA_CHECK(hipFree(loffsets));
	    CUDA_CHECK(hipFree(yzcid));
	    CUDA_CHECK(hipFree(outid));
	}

	CUDA_CHECK(hipMalloc(&xyzuvw_internal_copy, sizeof(float) * 6 * np));
	CUDA_CHECK(hipMalloc(&loffsets, sizeof(int) * np));
	CUDA_CHECK(hipMalloc(&yzcid, sizeof(int) * np));
	CUDA_CHECK(hipMalloc(&outid, sizeof(int) * np));

	old_np = np;
    }
    
    if (old_yzncells < yzncells)
    {
	if (old_yzncells > 0)
	{
	    CUDA_CHECK(hipFree(dyzscan));
	    CUDA_CHECK(hipFree(yzhisto));
	}

	CUDA_CHECK(hipMalloc(&dyzscan, sizeof(int) * yzncells));
	CUDA_CHECK(hipMalloc(&yzhisto, sizeof(int) * yzncells));
	
	old_yzncells = yzncells;
    }
      
    failuretest.reset(); 
    assert(failuretest.maxstripe != NULL);
    
    const float * xyzuvw_copy = xyzuvw_internal_copy;

    if (src_device_xyzuvw)
	xyzuvw_copy = src_device_xyzuvw;
    else
	CUDA_CHECK(hipMemcpyAsync(xyzuvw_internal_copy, device_xyzuvw, sizeof(float) * 6 * np, hipMemcpyDeviceToDevice, stream));
 
    CUDA_CHECK(hipMemsetAsync(yzhisto, 0, sizeof(int) * yzncells, stream));
  
    size_t textureoffset = 0;
    CUDA_CHECK(hipBindTexture(&textureoffset, &texParticlesCLS, xyzuvw_copy, &texParticlesCLS.channelDesc, sizeof(float) * 6 * np));
    CUDA_CHECK(hipBindTexture(&textureoffset, &texScanYZ, dyzscan, &texScanYZ.channelDesc, sizeof(int) * ncells.y * ncells.z));
    CUDA_CHECK(hipBindTexture(&textureoffset, &texCountYZ, yzhisto, &texCountYZ.channelDesc, sizeof(int) * ncells.y * ncells.z));
  
    if (clists_perfmon)
	CUDA_CHECK(hipEventRecord(evstart));
//fprintf(stdout, " densitynumber * 2=%f\n",  densitynumber * 2);
    {
	static const int ILP = 4;
#ifndef TEST_MAURO
	static const int SLOTS = 3;
#else
	static const int SLOTS = 1;
#endif
	static const int WARPS = 32;
	
	const int blocksize = 32 * WARPS;
	const int nblocks = (np + blocksize * ILP - 1)/ (blocksize * ILP);
	const int shmem_fp = sizeof(int) * ncells.y * ncells.z * SLOTS;

	if(nblocks * shmem_fp > old_gmemhistos_size)
	{
	    if(old_gmemhistos_size > 0)
		CUDA_CHECK(hipFree(gmemhistos));

	    CUDA_CHECK(hipMalloc(&gmemhistos, nblocks * shmem_fp));

	    old_gmemhistos_size = nblocks * shmem_fp;
        }

        CUDA_CHECK(hipMemsetAsync(gmemhistos, 0, nblocks*shmem_fp, stream));
		
	*failuretest.maxstripe = 0;
	
	if (shmem_fp <= 32 * 1024) {
            //fprintf(stderr, "yzhistogram<%d, %d, %d><<<%d, %d, %zu>>>\n", ILP, SLOTS, WARPS, nblocks, blocksize, sizeof(int) * ncells.y * ncells.z * SLOTS);
	    yzhistogram<ILP, SLOTS, WARPS><<<nblocks, blocksize, sizeof(int) * ncells.y * ncells.z * SLOTS, stream>>>
                (np, 1 / rc, ncells, domainstart, yzcid,  loffsets, yzhisto, dyzscan, failuretest.dmaxstripe, gmemhistos);
	} else
	{
	    static const int SLOTS = 1;
	    
	    //printf("SHMEM: %.2f kB\n", (float)(sizeof(int) * ncells.y * ncells.z * SLOTS) / 1024.);
	    
	    yzhistogram<ILP, SLOTS, WARPS><<<nblocks, blocksize, sizeof(int) * ncells.y * ncells.z * SLOTS, stream>>>
                (np, 1 / rc, ncells, domainstart, yzcid,  loffsets, yzhisto, dyzscan, failuretest.dmaxstripe, gmemhistos);
	}

	CUDA_CHECK(hipPeekAtLastError());
    }

    CUDA_CHECK(hipEventRecord(evacquire));
    
    {
	static const int ILP = 4;
	yzscatter<ILP><<<(np + 256 * ILP - 1) / (256 * ILP), 256, 0, stream>>>(loffsets, yzcid, np, outid);
    }
    
    CUDA_CHECK(hipEventSynchronize(evacquire));

    {
	xbufsize = *failuretest.maxstripe;
#ifndef TEST_MAURO
	static const int YCPB = 2;
	int shmem_fp = sizeof(int) * (ncells.x  + 2 * xbufsize) * YCPB;
#else
	static const int YCPB = 4;
	int shmem_fp = sizeof(int) * (ncells.x  + xbufsize) * YCPB;
#endif
	if(shmem_fp < 48 * 1024) {
            //printf("line %d: xgather<%d><<<(%d,%d,%d), (%d,%d), %d>>>\n", __LINE__, YCPB, 1, ncells.y / YCPB, ncells.z, 32, YCPB, shmem_fp);
	    xgather<YCPB><<< dim3(1, (ncells.y +YCPB-1)/ YCPB, ncells.z), dim3(32, YCPB), shmem_fp, stream>>>
		(outid, np, 1 / rc, ncells, domainstart, device_cellsstart, device_cellscount, device_xyzuvw, xbufsize,
		 order, loffsets);
	} else
	{
	    static const int YCPB = 1;

	    shmem_fp = sizeof(int) * (ncells.x  + 2 * xbufsize) * YCPB;
	    
	    assert(shmem_fp < 48 * 1024);
	    
	    xgather<YCPB><<< dim3(1, ncells.y / YCPB, ncells.z), dim3(32, YCPB), shmem_fp, stream>>>
		(outid, np, 1 / rc, ncells, domainstart, device_cellsstart, device_cellscount, device_xyzuvw, xbufsize,
		 order, loffsets);
	}
    }
    
    if (clists_perfmon)
	CUDA_CHECK(hipEventRecord(evscatter));
    
    if (!clists_robust)
    {
	failuretest.bufsize = xbufsize;
	CUDA_CHECK(hipStreamAddCallback(stream, failuretest.callback_crash, &failuretest, 0));
    }
    else
    {
	CUDA_CHECK(hipEventSynchronize(evacquire));

	if (*failuretest.maxstripe > xbufsize)
	{
	    //we should not be here anymore, after assignement at line 526
	    assert(false);
	    
	    CUDA_CHECK(hipDeviceSynchronize());
	    
	    printf("Ooops: maxstripe %d > bufsize %d.\nRecovering now...\n", *failuretest.maxstripe, xbufsize);
	    printf("density number223332 is %f\n", densitynumber);
	    const int xbufsize = *failuretest.maxstripe;
	    
	    xgather<1><<< dim3(1, ncells.y, ncells.z), dim3(32), sizeof(int) * (ncells.x  + 2 * xbufsize), stream>>>
		(outid, np, 1 / rc, ncells, domainstart, device_cellsstart, device_cellscount, device_xyzuvw, xbufsize,
		 order, loffsets);

	    hipError_t status = hipPeekAtLastError();

	    if (status != hipSuccess)
	    {
		printf("Could not roll back. Aborting now.\n");
		abort();
	    }
	    else
		printf("Recovery succeeded.\n");
	}
    }

    if (clists_perfmon)
    {
	CUDA_CHECK(hipEventRecord(evgather));
    
	CUDA_CHECK(hipEventSynchronize(evgather));
   
	CUDA_CHECK(hipPeekAtLastError());
	float tacquirems;
	CUDA_CHECK(hipEventElapsedTime(&tacquirems, evstart, evacquire));
	float tscatterms;
	CUDA_CHECK(hipEventElapsedTime(&tscatterms, evacquire, evscatter));
	float tgatherms;
	CUDA_CHECK(hipEventElapsedTime(&tgatherms, evscatter, evgather));
	float ttotalms;
	CUDA_CHECK(hipEventElapsedTime(&ttotalms, evstart, evgather));
    
	printf("acquiring time... %f ms\n", tacquirems);
	printf("scattering time... %f ms\n", tscatterms);
	printf("gathering time... %f ms\n", tgatherms);
	printf("total time ... %f ms\n", ttotalms);
	printf("one 2read-1write sweep should take about %.3f ms\n", 1e3 * np * 3 * 4/ (90.0 * 1024 * 1024 * 1024));
	printf("maxstripe was %d and bufsize is %d\n", *failuretest.maxstripe, xbufsize);
    }

    if (nonemptycells != NULL)
    {
	assert(nonemptycells->second != NULL);

	const int ntotcells = ncells.x * ncells.y * ncells.z;
	const int nonempties = copy_if(counting_iterator<int>(0), counting_iterator<int>(ntotcells), 
				       device_ptr<int>(device_cellscount), device_ptr<int>(nonemptycells->second), is_gzero())
	    - device_ptr<int>(nonemptycells->second);
	
	nonemptycells->first = nonempties;
    }

    CUDA_CHECK(hipUnbindTexture(texParticlesCLS));
    CUDA_CHECK(hipUnbindTexture(texScanYZ));
    CUDA_CHECK(hipUnbindTexture(texCountYZ));
}
