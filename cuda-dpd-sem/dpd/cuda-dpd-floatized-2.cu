#include "hip/hip_runtime.h"
/*
 *  cuda-dpd.cu
 *  Part of CTC/cuda-dpd-sem/dpd/
 *
 *  Evaluation of DPD force WITHOUT using Newton's 3rd law
 *  Created and authored by Yu-Hang Tang on 2015-03-18.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdio>
#include <cassert>

#include "../tiny-float.h"
#include "../dpd-rng.h"
#include "../hacks.h"

#define USE_TEXOBJ 0

struct InfoDPD {
    int3 ncells;
    float ncell_x, ncell_y;
    float3 domainsize, invdomainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
    float * axayaz;
    float seed;
	#if (USE_TEXOBJ&1)
    hipTextureObject_t txoParticles2;
    hipTextureObject_t txoStart, txoCount;
	#endif
};

__constant__ InfoDPD info;

#if !(USE_TEXOBJ&2)
texture<float4, hipTextureType1D> texParticles2;
texture<uint, hipTextureType1D> texStart, texCount;
#endif
#if (USE_TEXOBJ&1)
template<typename TYPE> struct texture_object {
	hipTextureObject_t txo;
	hipResourceDesc res_desc;
	hipTextureDesc tex_desc;
	TYPE *ptr_;
	long long n_;

	texture_object() : txo(0), ptr_(NULL), n_(0LL) {}

	inline operator hipTextureObject_t () { return txo; };

	inline hipTextureObject_t rebind( TYPE *ptr, const long long n ) {
		if ( ptr != ptr_ || ( ptr == ptr_ && n > n_ ) ) {
			if ( txo ) CUDA_CHECK( hipDestroyTextureObject( txo ) );
			ptr_ = ptr;
			n_ = n;
			res_desc.resType = hipResourceTypeLinear;
			res_desc.res.linear.desc = hipCreateChannelDesc<TYPE>();
			res_desc.res.linear.devPtr = ptr_;
			res_desc.res.linear.sizeInBytes = sizeof( TYPE ) * n_;
			tex_desc.readMode = hipReadModeElementType;
			CUDA_CHECK( hipCreateTextureObject( &txo, &res_desc, &tex_desc, NULL ) );
		}
		return txo;
	}
};

texture_object<float2> txoParticles2;
texture_object<uint> txoStart, txoCount;
#endif

#define _XCPB_ 2
#define _YCPB_ 2
#define _ZCPB_ 1
#define CPB (_XCPB_ * _YCPB_ * _ZCPB_)
//#define  _TIME_PROFILE_

#define LETS_MAKE_IT_MESSY

template<int s>
__device__ float viscosity_function( float x )
{
    return sqrtf( viscosity_function < s - 1 > ( x ) );
}

template<> __device__ float viscosity_function<0>( float x )
{
    return x;
}

// 31+56 FLOPS
__device__ float3 _dpd_interaction( const uint dpid, const float4 xdest, const float4 udest, const uint spid )
{
    const int sentry = xscale( spid, 2.f ); // 1 FLOP
	#if (USE_TEXOBJ&2)
    const float2 stmp0 = tex1Dfetch<float2>( info.txoParticles2, sentry           );
    const float2 stmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( sentry, 1 ) );
    const float2 stmp2 = tex1Dfetch<float2>( info.txoParticles2, xadd( sentry, 2 ) );
	#else
    const float4 xtmp = tex1Dfetch( texParticles2, sentry           );
    const float4 utmp = tex1Dfetch( texParticles2, xadd( sentry, 1 ) ); // 1 FLOP
    #endif

    const float _xr = xdest.x - xtmp.x; // 1 FLOP
    const float _yr = xdest.y - xtmp.y; // 1 FLOP
    const float _zr = xdest.z - xtmp.z; // 1 FLOP

    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr; // 5 FLOPS
    assert( rij2 < 1.f );

    const float invrij = rsqrtf( rij2 ); // 1 FLOP
    const float rij = rij2 * invrij; // 1 FLOP
    const float wc = 1.f - rij; // 1 FLOP
    const float wr = viscosity_function < -VISCOSITY_S_LEVEL > ( wc ); // 0 FLOP

    const float xr = _xr * invrij; // 1 FLOP
    const float yr = _yr * invrij; // 1 FLOP
    const float zr = _zr * invrij; // 1 FLOP

    const float rdotv =
        xr * ( udest.x - utmp.x ) +
        yr * ( udest.y - utmp.y ) +
        zr * ( udest.z - utmp.z );  // 8 FLOPS

    const float myrandnr = Logistic::mean0var1( info.seed, xmin(spid,dpid), xmax(spid,dpid) );  // 54+2 FLOP

    const float strength = info.aij * wc - ( info.gamma * wr * rdotv + info.sigmaf * myrandnr ) * wr; // 7 FLOPS

    return make_float3( strength * xr, strength * yr, strength * zr );
}

__device__ float3 _dpd_interaction(const int dpid, const float4 xdest, const float4 udest, const float4 xsrc, const float4 usrc, const int spid)
{
    const float _xr = xdest.x - xsrc.x;
    const float _yr = xdest.y - xsrc.y;
    const float _zr = xdest.z - xsrc.z;

    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
    assert(rij2 < 1);

    const float invrij = rsqrtf(rij2);
    const float rij = rij2 * invrij;
    const float argwr = 1 - rij;
    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

    const float xr = _xr * invrij;
    const float yr = _yr * invrij;
    const float zr = _zr * invrij;

    const float rdotv =
	xr * (udest.x - usrc.x) +
	yr * (udest.y - usrc.y) +
	zr * (udest.z - usrc.z);

    const float myrandnr = Logistic::mean0var1(info.seed, min(spid, dpid), max(spid, dpid));

    const float strength = info.aij * argwr - (info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;

    return make_float3(strength * xr, strength * yr, strength * zr);
}

template<uint COLS, uint ROWS, uint NSRCMAX>
__device__ void core( const uint nsrc, const uint2 * const starts_and_scans,
                      const uint ndst, const uint dststart )
{
	uint srccount = 0;
    assert( ndst == ROWS );

    const uint tid = threadIdx.x;
    const uint wid = threadIdx.y;
    const uint slot = tid / COLS;
    const uint subtid = tid % COLS;

    const uint dpid = xadd( dststart, slot ); // 1 FLOP
    const int entry = xscale( dpid, 2.f ); // 1 FLOP
	#if (USE_TEXOBJ&2)
    const float2 dtmp0 = tex1Dfetch<float2>( info.txoParticles2,       entry      );
    const float2 dtmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 1 ) );
    const float2 dtmp2 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 2 ) );
    const float3 xdest = make_float3( dtmp0.x, dtmp0.y, dtmp1.x );
    const float3 udest = make_float3( dtmp1.y, dtmp2.x, dtmp2.y );
	#else
    const float4 xdest = tex1Dfetch( texParticles2,       entry      );
    const float4 udest = tex1Dfetch( texParticles2, xadd( entry, 1 ) ); // 1 FLOP
	#endif

    float xforce = 0, yforce = 0, zforce = 0;

    for(uint s = 0; s < nsrc; s = xadd( s, COLS ) )
	{
    	const uint pid  = xadd( s, subtid ); // 1 FLOP
#ifdef LETS_MAKE_IT_MESSY
		uint spid;
		asm( "{ .reg .pred p, q;"
			 "  .reg .f32  key;"
			 "  .reg .f32  scan3, scan6, scan9, scan18;"
			 "  .reg .f32  mystart, myscan;"
			 "  .reg .s32  array;"
			 "  .reg .f32  array_f;"
			 "   mov.b32           array_f, %4;"
			 "   mul.f32           array_f, array_f, 256.0;"
			 "   mov.b32           array, array_f;"
			 "   ld.shared.f32     scan9,  [array +  9*8 + 4];"
			 "   ld.shared.f32     scan18, [array + 18*8 + 4];"
			 "   setp.ge.f32       p, %1, scan9;"
			 "   setp.ge.f32       q, %1, scan18;"
			 "   selp.f32          key, %2, 0.0, p;"
			 "@q add.f32           key, key, %2;"
			 "   mov.b32           array_f, array;"
			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
			 "   mov.b32 array,    array_f;"
			 "   ld.shared.f32     scan3, [array + 3*8 + 4];"
			 "   ld.shared.f32     scan6, [array + 6*8 + 4];"
			 "   setp.ge.f32       p, %1, scan3;"
			 "   setp.ge.f32       q, %1, scan6;"
			 "@p add.f32           key, key, %3;"
			 "@q add.f32           key, key, %3;"
			 "   mov.b32           array_f, %4;"
			 "   mul.f32           array_f, array_f, 256.0;"
			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
			 "   mov.b32           array, array_f;"
			 "   ld.shared.v2.f32 {mystart, myscan}, [array];"
			 "   add.f32           mystart, mystart, %1;"
			 "   sub.f32           mystart, mystart, myscan;"
	         "   mov.b32           %0, mystart;"
	         "}" : "=r"(spid) : "f"(u2f(pid)), "f"(u2f(9u)), "f"(u2f(3u)), "f"(u2f(wid)) );
		// 15 FLOPS
#else
		const uint key9 = xadd( xsel_ge( pid, scan[ 9u            ].y, 9u, 0u ), xsel_ge( pid, scan[ 18u           ].y, 9u, 0u ) );
		const uint key3 = xadd( xsel_ge( pid, scan[ xadd(key9,3u) ].y, 3u, 0u ), xsel_ge( pid, scan[ xadd(key9,6u) ].y, 3u, 0u ) );
		const uint key  = xadd( key9, key3 );
		const uint spid = xsub( xadd( pid, starts_and_scans[key].x ), starts_and_scans[key].y );
#endif


		#if (USE_TEXOBJ&2)
		const int sentry = xscale( spid, 3.f );
		const float2 stmp0 = tex1Dfetch<float2>( info.txoParticles2,       sentry      );
		const float2 stmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( sentry, 1 ) );
		#else
		const int sentry = xscale( spid, 2.f ); // 1 FLOP
		const float4 xtmp = tex1Dfetch( texParticles2, sentry );
		#endif

		const float xdiff = xdest.x - xtmp.x; // 1 FLOP
		const float ydiff = xdest.y - xtmp.y; // 1 FLOP
		const float zdiff = xdest.z - xtmp.z; // 1 FLOP
#ifdef LETS_MAKE_IT_MESSY
		asm("{ .reg .pred p;"
			"  .reg .f32 srccount_f;"
			"   mov.b32 srccount_f, %0;"
			"   setp.lt.f32 p, %1, %2;"
			"   setp.lt.and.f32 p, %3, 1.0, p;"
			"   setp.ne.and.f32 p, %4, %5, p;"
			"   @p st.shared.u32 [%6], %8;"
			"   @p add.f32 srccount_f, srccount_f, %7;"
			"   mov.b32 %0, srccount_f;"
			"}" : "+r"(srccount) :
			"f"( u2f(pid) ), "f"(u2f(nsrc)), "f"(xdiff * xdiff + ydiff * ydiff + zdiff * zdiff), "f"(u2f(dpid)), "f"(u2f(spid)),
			"r"( xmad(tid,4.f,xmad(wid,128.f,xmad(srccount,512.f,1024u))) ), "f"(u2f(1u)), "r"(spid) : "memory" );
		// 3+(?1)+11
#else
		// 14 FLOPS
		const float interacting = xfcmp_lt(pid, nsrc )
				                * xfcmp_lt( xdiff * xdiff + ydiff * ydiff + zdiff * zdiff, 1.f )
				                * xfcmp_ne( dpid, spid ) ;
		if (interacting) {
			srcids[srccount] = spid;
			srccount = xadd( srccount, 1u ); // 1 FLOP
		}
#endif
		if ( srccount == NSRCMAX ) {
			srccount = xsub( srccount, 1u ); // 1 FLOP
			const float4 utmp = tex1Dfetch( texParticles2, xmad( spid, 2.f, 1u ) );
			const float3 f = _dpd_interaction( dpid, xdest, udest, xtmp, utmp, spid ); // 87 FLOPS

			xforce += f.x; // 1 FLOP
			yforce += f.y; // 1 FLOP
			zforce += f.z; // 1 FLOP
		}
		// 1 FLOP for s++
	}

#pragma unroll 4
	for( uint i = 0; i < srccount; i = xadd( i, 1u ) ) {
#ifdef LETS_MAKE_IT_MESSY
		uint spid;
		asm("ld.shared.u32 %0, [%1];" : "=r"(spid) : "r"( xmad(tid,4.f,xmad(wid,128.f,xmad(i,512.f,1024u))) ) ); // 6 FLOPS
		const float3 f = _dpd_interaction( dpid, xdest, udest, spid ); // 87 FLOPS
#else
		const float3 f = _dpd_interaction( dpid, xdest, udest, srcids[i] ); // 87 FLOPS
#endif
        xforce += f.x; // 1 FLOP
        yforce += f.y; // 1 FLOP
        zforce += f.z; // 1 FLOP

        // 1 FLOP for i++
    }

    for( uint L = COLS / 2; L > 0; L >>= 1 ) {
        xforce += __shfl_xor( xforce, L ); // 1 FLOP
        yforce += __shfl_xor( yforce, L ); // 1 FLOP
        zforce += __shfl_xor( zforce, L ); // 1 FLOP
    }

#ifdef LETS_MAKE_IT_MESSY
    float fcontrib;
    asm("{   .reg .pred isy, isz;"
    	"     setp.f32.eq isy, %1, %5;"
    	"     setp.f32.eq isz, %1, %6;"
    	"     selp.f32 %0, %2, %3, !isy;"
    	"@isz mov.b32 %0, %4;"
    	"}" : "=f"(fcontrib) : "f"(u2f(subtid)), "f"(xforce), "f"(yforce), "f"(zforce), "f"(u2f(1u)), "f"(u2f(2u)) );
    // 2 FLOPS
#else
    //const float fcontrib = xsel_eq( subtid, 0u, xforce, xsel_eq( subtid, 1u, yforce, zforce ) ); // 2 FLOPS
#endif

    if( subtid < 3.f )
        info.axayaz[ xmad( dpid, 3.f, subtid ) ] = fcontrib;  // 2 FLOPS
}

template<uint COLS, uint ROWS, uint NSRCMAX>
__device__ void core_ilp( const uint nsrc, const uint2 * const starts_and_scans,
                          const uint ndst, const uint dststart )
{
    const uint tid    = threadIdx.x;
    const uint wid    = threadIdx.y;
    const uint slot   = tid / COLS;
    const uint subtid = tid % COLS;

    const uint dpid = xadd( dststart, slot ); // 1 FLOP
    const int entry = xscale( dpid, 2.f ); // 1 FLOP
	#if (USE_TEXOBJ&2)
	const float2 dtmp0 = tex1Dfetch<float2>( info.txoParticles2,       entry      );
	const float2 dtmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 1 ) );
	const float2 dtmp2 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 2 ) );
    const float3 xdest = make_float3( dtmp0.x, dtmp0.y, dtmp1.x );
    const float3 udest = make_float3( dtmp1.y, dtmp2.x, dtmp2.y );
	#else
	const float4 xdest = tex1Dfetch( texParticles2,       entry      );
	const float4 udest = tex1Dfetch( texParticles2, xadd( entry, 1 ) ); // 1 FLOP
	#endif

    float xforce = 0, yforce = 0, zforce = 0;

    for( uint s = 0; s < nsrc; s = xadd( s, NSRCMAX * COLS ) ) {
        uint spids[NSRCMAX];
		#pragma unroll
        for( uint i = 0; i < NSRCMAX; ++i ) {
            const uint pid  = xadd( s, xmad( i, float(COLS), subtid ) );
#ifdef LETS_MAKE_IT_MESSY
    		uint spid;
    		asm( "{ .reg .pred p, q;"
    			 "  .reg .f32  key;"
    			 "  .reg .f32  scan3, scan6, scan9, scan18;"
    			 "  .reg .f32  mystart, myscan;"
    			 "  .reg .s32  array;"
    			 "  .reg .f32  array_f;"
    			 "   mov.b32           array_f, %4;"
    			 "   mul.f32           array_f, array_f, 256.0;"
    			 "   mov.b32           array, array_f;"
    			 "   ld.shared.f32     scan9,  [array +  9*8 + 4];"
    			 "   ld.shared.f32     scan18, [array + 18*8 + 4];"
    			 "   setp.ge.f32       p, %1, scan9;"
    			 "   setp.ge.f32       q, %1, scan18;"
    			 "   selp.f32          key, %2, 0.0, p;"
    			 "@q add.f32           key, key, %2;"
    			 "   mov.b32           array_f, array;"
    			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
    			 "   mov.b32 array,    array_f;"
    			 "   ld.shared.f32     scan3, [array + 3*8 + 4];"
    			 "   ld.shared.f32     scan6, [array + 6*8 + 4];"
    			 "   setp.ge.f32       p, %1, scan3;"
    			 "   setp.ge.f32       q, %1, scan6;"
    			 "@p add.f32           key, key, %3;"
    			 "@q add.f32           key, key, %3;"
    			 "   mov.b32           array_f, %4;"
    			 "   mul.f32           array_f, array_f, 256.0;"
    			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
    			 "   mov.b32           array, array_f;"
    			 "   ld.shared.v2.f32 {mystart, myscan}, [array];"
    			 "   add.f32           mystart, mystart, %1;"
    			 "   sub.f32           mystart, mystart, myscan;"
    	         "   mov.b32           %0, mystart;"
    	         "}" : "=r"(spid) : "f"(u2f(pid)), "f"(u2f(9u)), "f"(u2f(3u)), "f"(u2f(wid)) );
    		// 15 FLOPS
            spids[i] = spid;
#else
    		const uint key9 = xadd( xsel_ge( pid, scan[ 9             ], 9u, 0u ), xsel_ge( pid, scan[ 18            ], 9u, 0u ) );
    		const uint key3 = xadd( xsel_ge( pid, scan[ xadd(key9,3u) ], 3u, 0u ), xsel_ge( pid, scan[ xadd(key9,6u) ], 3u, 0u ) );
    		const uint key  = xadd( key9, key3 );
            spids[i] = xsub( xadd( pid, starts_and_scans[key].x ), starts_and_scans[key].y );
#endif
        }

        uint interacting[NSRCMAX];
		#pragma unroll
        for( uint i = 0; i < NSRCMAX; ++i ) {
			#if (USE_TEXOBJ&2)
			const float2 stmp0 = tex1Dfetch<float2>( info.txoParticles2,       sentry      );
			const float2 stmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( sentry, 1 ) );
			#else
            const int sentry = xscale( spids[i], 2.f ); // 1 FLOP
			const float4 xtmp = tex1Dfetch( texParticles2,       sentry      );
			#endif

            const float xdiff = xdest.x - xtmp.x; // 1 FLOP
            const float ydiff = xdest.y - xtmp.y; // 1 FLOP
            const float zdiff = xdest.z - xtmp.z; // 1 FLOP
#ifdef LETS_MAKE_IT_MESSY
			uint interacting_one;
            asm("{ .reg .pred p;"
				"   setp.lt.f32 p, %1, %2;"
				"   setp.lt.and.f32 p, %3, 1.0, p;"
				"   set.ne.and.u32.f32 %0, %4, %5, p;"
				"   }" : "=r"(interacting_one)  : "f"(u2f(xadd( s, xmad( i, float(COLS), subtid ) ))), "f"(u2f(nsrc)), "f"(xdiff * xdiff + ydiff * ydiff + zdiff * zdiff), "f"(u2f(dpid)), "f"(u2f(spids[i])) );
            // 12 FLOPS
            interacting[i] = interacting_one;
#else
            interacting[i] = xfcmp_lt( xadd( s, xmad( i, float(COLS), subtid ) ), nsrc )
            		       * xfcmp_lt( xdiff * xdiff + ydiff * ydiff + zdiff * zdiff, 1.f )
            		       * xfcmp_ne( dpid, spids[i] );
#endif
        }

		#pragma unroll
        for( uint i = 0; i < NSRCMAX; ++i ) {
            if( interacting[i] ) {
                const float3 f = _dpd_interaction( dpid, xdest, udest, spids[i] ); // 88 FLOPS

                xforce += f.x; // 1 FLOP
                yforce += f.y; // 1 FLOP
                zforce += f.z; // 1 FLOP
            }
        }

        // 1 FLOP for s += NSRCMAX * COLS;
    }

    for( uint L = COLS / 2; L > 0; L >>= 1 ) {
        xforce += __shfl_xor( xforce, L ); // 1 FLOP
        yforce += __shfl_xor( yforce, L ); // 1 FLOP
        zforce += __shfl_xor( zforce, L ); // 1 FLOP
    }

#ifdef LETS_MAKE_IT_MESSY
    float fcontrib;
    asm("{   .reg .pred isy, isz;"
    	"     setp.f32.eq isy, %1, %5;"
    	"     setp.f32.eq isz, %1, %6;"
    	"     selp.f32 %0, %2, %3, !isy;"
    	"@isz mov.b32 %0, %4;"
    	"}" : "=f"(fcontrib) : "f"(u2f(subtid)), "f"(xforce), "f"(yforce), "f"(zforce), "f"(u2f(1u)), "f"(u2f(2u)) );
    // 2 FLOPS
#else
    const float fcontrib = xsel_eq( subtid, 0u, xforce, xsel_eq( subtid, 1u, yforce, zforce ) );  // 2 FLOPS
#endif

    if( subtid < 3u )
        info.axayaz[ xmad( dpid, 3.f, subtid ) ] = fcontrib;  // 2 FLOPS
}

__global__ __launch_bounds__( 32 * CPB, 16 )
void _dpd_forces_floatized()
{
    assert( blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1 );

    const uint tid = threadIdx.x;
    const uint wid = threadIdx.y;

    __shared__ volatile uint2 starts_and_scans[CPB*3][32];

    uint mycount = 0, myscan = 0;
    const int dx = ( tid ) % 3;
    const int dy = ( ( tid / 3 ) ) % 3;
    const int dz = ( ( tid / 9 ) ) % 3;

    if( tid < 27 ) {

        int xcid = blockIdx.x * _XCPB_ + ( ( threadIdx.y ) % _XCPB_ ) + dx - 1;
        int ycid = blockIdx.y * _YCPB_ + ( ( threadIdx.y / _XCPB_ ) % _YCPB_ ) + dy - 1;
        int zcid = blockIdx.z * _ZCPB_ + ( ( threadIdx.y / ( _XCPB_ * _YCPB_ ) ) % _ZCPB_ ) + dz - 1;

        const bool valid_cid =
                ( xcid >= 0 ) && ( xcid < info.ncells.x ) &&
                ( ycid >= 0 ) && ( ycid < info.ncells.y ) &&
                ( zcid >= 0 ) && ( zcid < info.ncells.z );

        xcid = xmin( xsub( info.ncells.x, 1 ), max( 0, xcid ) ); // 2 FLOPS
        ycid = xmin( xsub( info.ncells.y, 1 ), max( 0, ycid ) ); // 2 FLOPS
        zcid = xmin( xsub( info.ncells.z, 1 ), max( 0, zcid ) ); // 2 FLOPS

        const int cid = max( 0, ( zcid * info.ncells.y + ycid ) * info.ncells.x + xcid );
		#if (USE_TEXOBJ&2)
        starts_and_scans[wid][tid].x = tex1Dfetch<uint>( info.txoStart, cid );
        myscan = mycount = valid_cid ? tex1Dfetch<uint>( info.txoCount, cid ) : 0u;
		#else
        starts_and_scans[wid][tid].x = tex1Dfetch( texStart, cid );
        myscan = mycount = valid_cid ? tex1Dfetch( texCount, cid ) : 0u;
		#endif
    }

	#pragma unroll
    for( int L = 1; L < 32; L <<= 1 ) {
    	uint theirscan = __shfl_up( myscan, L );
    	myscan = xadd( myscan, xsel_ge( tid, i2u(L), theirscan, 0u ) ); // 2 FLOPS
    }

    if( tid < 28 )
    	starts_and_scans[wid][tid].y = xsub( myscan, mycount ); // 1 FLOP

    const uint nsrc = starts_and_scans[wid][27].y;
    const uint dststart = starts_and_scans[wid][1 + 3 + 9].x;
    const uint ndst = xsub( starts_and_scans[wid][1 + 3 + 9 + 1].y, starts_and_scans[wid][1 + 3 + 9].y ); // 1 FLOP
    const uint ndst4 = ( ndst >> 2 ) << 2;

    for( uint d = 0; d < ndst4; d = xadd( d, 4u ) ) // 1 FLOP
        core<8, 4, 4>( nsrc, ( const uint2 * )starts_and_scans[wid], 4, xadd( dststart, d ) ); // 1 FLOP

    uint d = ndst4;
    if( xadd( d, 2u ) <= ndst ) { // 1 FLOPS
        core<16, 2, 4>( nsrc, ( const uint2 * )starts_and_scans[wid], 2, xadd( dststart, d ) ); // 1 FLOP
        d = xadd( d, 2u ); // 1 FLOP
    }

    if( d < ndst )
        core_ilp<32, 1, 2>( nsrc, ( const uint2 * )starts_and_scans[wid], 1, xadd( dststart, d ) ); // 1 FLOP
}

__global__ void copy( float *v4, const float *v3, const int n ) {
	for(int i=blockIdx.x*blockDim.x+threadIdx.x;i<n;i+=blockDim.x*gridDim.x) {
			v4[i*8+0] = v3[i*6+0];
			v4[i*8+1] = v3[i*6+1];
			v4[i*8+2] = v3[i*6+2];
			v4[i*8+4] = v3[i*6+3];
			v4[i*8+5] = v3[i*6+4];
			v4[i*8+6] = v3[i*6+5];
	}
}


#ifdef _COUNT_FLOPS
struct _dpd_interaction_flops_counter {
	const static unsigned long long FLOPS = 31ULL + Logistic::mean0var1_flops_counter::FLOPS;
};

template<uint COLS, uint ROWS, uint NSRCMAX>
__device__ void core_flops_counter( unsigned long long *FLOPS, const uint nsrc, const uint2 * const starts_and_scans,
        const uint ndst, const uint dststart )
{
	uint srccount = 0;
    assert( ndst == ROWS );

    const uint tid = threadIdx.x;
    const uint wid = threadIdx.y;
    const uint slot = tid / COLS;
    const uint subtid = tid % COLS;

    const uint dpid = xadd( dststart, slot ); // 1 FLOP
    const int entry = xscale( dpid, 2.f ); // 1 FLOP
	#if (USE_TEXOBJ&2)
    const float2 dtmp0 = tex1Dfetch<float2>( info.txoParticles2,       entry      );
    const float2 dtmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 1 ) );
    const float2 dtmp2 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 2 ) );
    const float3 xdest = make_float3( dtmp0.x, dtmp0.y, dtmp1.x );
    const float3 udest = make_float3( dtmp1.y, dtmp2.x, dtmp2.y );
	#else
    const float4 xdest = tex1Dfetch( texParticles2,       entry      );
    const float4 udest = tex1Dfetch( texParticles2, xadd( entry, 1 ) ); // 1 FLOP
	#endif

    atomicAdd( FLOPS, 3ULL );

    float xforce = 0, yforce = 0, zforce = 0;

    for(uint s = 0; s < nsrc; s = xadd( s, COLS ) )
	{
    	const uint pid  = xadd( s, subtid ); // 1 FLOP
#ifdef LETS_MAKE_IT_MESSY
		uint spid;
		asm( "{ .reg .pred p, q;"
			 "  .reg .f32  key;"
			 "  .reg .f32  scan3, scan6, scan9, scan18;"
			 "  .reg .f32  mystart, myscan;"
			 "  .reg .s32  array;"
			 "  .reg .f32  array_f;"
			 "   mov.b32           array_f, %4;"
			 "   mul.f32           array_f, array_f, 256.0;"
			 "   mov.b32           array, array_f;"
			 "   ld.shared.f32     scan9,  [array +  9*8 + 4];"
			 "   ld.shared.f32     scan18, [array + 18*8 + 4];"
			 "   setp.ge.f32       p, %1, scan9;"
			 "   setp.ge.f32       q, %1, scan18;"
			 "   selp.f32          key, %2, 0.0, p;"
			 "@q add.f32           key, key, %2;"
			 "   mov.b32           array_f, array;"
			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
			 "   mov.b32 array,    array_f;"
			 "   ld.shared.f32     scan3, [array + 3*8 + 4];"
			 "   ld.shared.f32     scan6, [array + 6*8 + 4];"
			 "   setp.ge.f32       p, %1, scan3;"
			 "   setp.ge.f32       q, %1, scan6;"
			 "@p add.f32           key, key, %3;"
			 "@q add.f32           key, key, %3;"
			 "   mov.b32           array_f, %4;"
			 "   mul.f32           array_f, array_f, 256.0;"
			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
			 "   mov.b32           array, array_f;"
			 "   ld.shared.v2.f32 {mystart, myscan}, [array];"
			 "   add.f32           mystart, mystart, %1;"
			 "   sub.f32           mystart, mystart, myscan;"
	         "   mov.b32           %0, mystart;"
	         "}" : "=r"(spid) : "f"(u2f(pid)), "f"(u2f(9u)), "f"(u2f(3u)), "f"(u2f(wid)) );
		// 15 FLOPS
#else
		const uint key9 = xadd( xsel_ge( pid, scan[ 9u            ].y, 9u, 0u ), xsel_ge( pid, scan[ 18u           ].y, 9u, 0u ) );
		const uint key3 = xadd( xsel_ge( pid, scan[ xadd(key9,3u) ].y, 3u, 0u ), xsel_ge( pid, scan[ xadd(key9,6u) ].y, 3u, 0u ) );
		const uint key  = xadd( key9, key3 );
		const uint spid = xsub( xadd( pid, starts_and_scans[key].x ), starts_and_scans[key].y );
#endif

		#if (USE_TEXOBJ&2)
		const int sentry = xscale( spid, 3.f );
		const float2 stmp0 = tex1Dfetch<float2>( info.txoParticles2,       sentry      );
		const float2 stmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( sentry, 1 ) );
		#else
		const int sentry = xscale( spid, 2.f ); // 1 FLOP
		const float4 xtmp = tex1Dfetch( texParticles2, sentry );
		#endif

		const float xdiff = xdest.x - xtmp.x; // 1 FLOP
		const float ydiff = xdest.y - xtmp.y; // 1 FLOP
		const float zdiff = xdest.z - xtmp.z; // 1 FLOP
		atomicAdd( FLOPS, 20ULL );
#if 0
		asm("{ .reg .pred p;"
			"  .reg .f32 srccount_f;"
			"   mov.b32 srccount_f, %0;"
			"   setp.lt.f32 p, %1, %2;"
			"   setp.lt.and.f32 p, %3, 1.0, p;"
			"   setp.ne.and.f32 p, %4, %5, p;"
			"   @p st.shared.u32 [%6], %8;"
			"   @p add.f32 srccount_f, srccount_f, %7;"
			"   mov.b32 %0, srccount_f;"
			"}" : "+r"(srccount) :
			"f"( u2f(pid) ), "f"(u2f(nsrc)), "f"(xdiff * xdiff + ydiff * ydiff + zdiff * zdiff), "f"(u2f(dpid)), "f"(u2f(spid)),
			"r"( xmad(tid,4.f,xmad(wid,128.f,xmad(srccount,512.f,1024u))) ), "f"(u2f(1u)), "r"(spid) : "memory" );
		// 3+(?1)+11
#else
		// 14 FLOPS
		const float interacting = xfcmp_lt(pid, nsrc )
				                * xfcmp_lt( xdiff * xdiff + ydiff * ydiff + zdiff * zdiff, 1.f )
				                * xfcmp_ne( dpid, spid ) ;
		atomicAdd( FLOPS, 14ULL );
		if (interacting) {
//			srcids[srccount] = spid;
			srccount = xadd( srccount, 1u ); // 1 FLOP
			atomicAdd( FLOPS, 1ULL );
		}
#endif
		if ( srccount == NSRCMAX ) {
			srccount = xsub( srccount, 1u ); // 1 FLOP
			// why do we reload spid? it's right there in register
			float3 f;// = _dpd_interaction( dpid, xdest, udest, spid ); // 87 FLOPS

			xforce += f.x; // 1 FLOP
			yforce += f.y; // 1 FLOP
			zforce += f.z; // 1 FLOP
			atomicAdd( FLOPS, 4ULL + _dpd_interaction_flops_counter::FLOPS );
		}
		// 1 FLOP for s++
		atomicAdd( FLOPS, 1ULL );
	}

#pragma unroll 4
	for( uint i = 0; i < srccount; i = xadd( i, 1u ) ) {
#ifdef LETS_MAKE_IT_MESSY
		uint spid;
		asm("ld.shared.u32 %0, [%1];" : "=r"(spid) : "r"( xmad(tid,4.f,xmad(wid,128.f,xmad(i,512.f,1024u))) ) ); // 6 FLOPS
		float3 f;// = _dpd_interaction( dpid, xdest, udest, spid ); // 87 FLOPS
		atomicAdd( FLOPS, 6 + _dpd_interaction_flops_counter::FLOPS );
#else
		const float3 f = _dpd_interaction( dpid, xdest, udest, srcids[i] ); // 87 FLOPS
#endif
        xforce += f.x; // 1 FLOP
        yforce += f.y; // 1 FLOP
        zforce += f.z; // 1 FLOP

        // 1 FLOP for i++
        atomicAdd( FLOPS, 4ULL );
    }

    for( uint L = COLS / 2; L > 0; L >>= 1 ) {
        xforce += __shfl_xor( xforce, L ); // 1 FLOP
        yforce += __shfl_xor( yforce, L ); // 1 FLOP
        zforce += __shfl_xor( zforce, L ); // 1 FLOP
        atomicAdd( FLOPS, 3ULL );
    }

#ifdef LETS_MAKE_IT_MESSY
    float fcontrib;
    asm("{   .reg .pred isy, isz;"
    	"     setp.f32.eq isy, %1, %5;"
    	"     setp.f32.eq isz, %1, %6;"
    	"     selp.f32 %0, %2, %3, !isy;"
    	"@isz mov.b32 %0, %4;"
    	"}" : "=f"(fcontrib) : "f"(u2f(subtid)), "f"(xforce), "f"(yforce), "f"(zforce), "f"(u2f(1u)), "f"(u2f(2u)) );
    // 2 FLOPS
    atomicAdd( FLOPS, 2ULL );
#else
    //const float fcontrib = xsel_eq( subtid, 0u, xforce, xsel_eq( subtid, 1u, yforce, zforce ) ); // 2 FLOPS
#endif

    if( subtid < 3.f ) {
        //info.axayaz[ xmad( dpid, 3.f, subtid ) ] = fcontrib;  // 2 FLOPS
        atomicAdd( FLOPS, 2ULL );
    }
}

template<uint COLS, uint ROWS, uint NSRCMAX>
__device__ void core_ilp_flops_counter( unsigned long long *FLOPS, const uint nsrc, const uint2 * const starts_and_scans,
        const uint ndst, const uint dststart )
{
    const uint tid    = threadIdx.x;
    const uint wid    = threadIdx.y;
    const uint slot   = tid / COLS;
    const uint subtid = tid % COLS;

    const uint dpid = xadd( dststart, slot ); // 1 FLOP
    const int entry = xscale( dpid, 2.f ); // 1 FLOP
	#if (USE_TEXOBJ&2)
	const float2 dtmp0 = tex1Dfetch<float2>( info.txoParticles2,       entry      );
	const float2 dtmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 1 ) );
	const float2 dtmp2 = tex1Dfetch<float2>( info.txoParticles2, xadd( entry, 2 ) );
    const float3 xdest = make_float3( dtmp0.x, dtmp0.y, dtmp1.x );
    const float3 udest = make_float3( dtmp1.y, dtmp2.x, dtmp2.y );
	#else
	const float4 xdest = tex1Dfetch( texParticles2,       entry      );
	const float4 udest = tex1Dfetch( texParticles2, xadd( entry, 1 ) ); // 1 FLOP
	#endif

	atomicAdd( FLOPS, 3ULL );

    float xforce = 0, yforce = 0, zforce = 0;

    for( uint s = 0; s < nsrc; s = xadd( s, NSRCMAX * COLS ) ) {
        uint spids[NSRCMAX];
		#pragma unroll
        for( uint i = 0; i < NSRCMAX; ++i ) {
            const uint pid  = xadd( s, xmad( i, float(COLS), subtid ) );
            atomicAdd( FLOPS, 3ULL );
#ifdef LETS_MAKE_IT_MESSY
    		uint spid;
    		asm( "{ .reg .pred p, q;"
    			 "  .reg .f32  key;"
    			 "  .reg .f32  scan3, scan6, scan9, scan18;"
    			 "  .reg .f32  mystart, myscan;"
    			 "  .reg .s32  array;"
    			 "  .reg .f32  array_f;"
    			 "   mov.b32           array_f, %4;"
    			 "   mul.f32           array_f, array_f, 256.0;"
    			 "   mov.b32           array, array_f;"
    			 "   ld.shared.f32     scan9,  [array +  9*8 + 4];"
    			 "   ld.shared.f32     scan18, [array + 18*8 + 4];"
    			 "   setp.ge.f32       p, %1, scan9;"
    			 "   setp.ge.f32       q, %1, scan18;"
    			 "   selp.f32          key, %2, 0.0, p;"
    			 "@q add.f32           key, key, %2;"
    			 "   mov.b32           array_f, array;"
    			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
    			 "   mov.b32 array,    array_f;"
    			 "   ld.shared.f32     scan3, [array + 3*8 + 4];"
    			 "   ld.shared.f32     scan6, [array + 6*8 + 4];"
    			 "   setp.ge.f32       p, %1, scan3;"
    			 "   setp.ge.f32       q, %1, scan6;"
    			 "@p add.f32           key, key, %3;"
    			 "@q add.f32           key, key, %3;"
    			 "   mov.b32           array_f, %4;"
    			 "   mul.f32           array_f, array_f, 256.0;"
    			 "   fma.f32.rm        array_f, key, 8.0, array_f;"
    			 "   mov.b32           array, array_f;"
    			 "   ld.shared.v2.f32 {mystart, myscan}, [array];"
    			 "   add.f32           mystart, mystart, %1;"
    			 "   sub.f32           mystart, mystart, myscan;"
    	         "   mov.b32           %0, mystart;"
    	         "}" : "=r"(spid) : "f"(u2f(pid)), "f"(u2f(9u)), "f"(u2f(3u)), "f"(u2f(wid)) );
    		// 15 FLOPS
            spids[i] = spid;
            atomicAdd( FLOPS, 15ULL );
#else
    		const uint key9 = xadd( xsel_ge( pid, scan[ 9             ], 9u, 0u ), xsel_ge( pid, scan[ 18            ], 9u, 0u ) );
    		const uint key3 = xadd( xsel_ge( pid, scan[ xadd(key9,3u) ], 3u, 0u ), xsel_ge( pid, scan[ xadd(key9,6u) ], 3u, 0u ) );
    		const uint key  = xadd( key9, key3 );
            spids[i] = xsub( xadd( pid, starts_and_scans[key].x ), starts_and_scans[key].y );
#endif
        }

        uint interacting[NSRCMAX];
		#pragma unroll
        for( uint i = 0; i < NSRCMAX; ++i ) {
			#if (USE_TEXOBJ&2)
			const float2 stmp0 = tex1Dfetch<float2>( info.txoParticles2,       sentry      );
			const float2 stmp1 = tex1Dfetch<float2>( info.txoParticles2, xadd( sentry, 1 ) );
			#else
            const int sentry = xscale( spids[i], 2.f ); // 1 FLOP
			const float4 xtmp = tex1Dfetch( texParticles2,       sentry      );
			#endif

            const float xdiff = xdest.x - xtmp.x; // 1 FLOP
            const float ydiff = xdest.y - xtmp.y; // 1 FLOP
            const float zdiff = xdest.z - xtmp.z; // 1 FLOP
            atomicAdd( FLOPS, 4ULL );

#ifdef LETS_MAKE_IT_MESSY
			uint interacting_one;
            asm("{ .reg .pred p;"
				"   setp.lt.f32 p, %1, %2;"
				"   setp.lt.and.f32 p, %3, 1.0, p;"
				"   set.ne.and.u32.f32 %0, %4, %5, p;"
				"   }" : "=r"(interacting_one)  : "f"(u2f(xadd( s, xmad( i, float(COLS), subtid ) ))), "f"(u2f(nsrc)), "f"(xdiff * xdiff + ydiff * ydiff + zdiff * zdiff), "f"(u2f(dpid)), "f"(u2f(spids[i])) );
            // 12 FLOPS
            interacting[i] = interacting_one;
            atomicAdd( FLOPS, 12ULL );
#else
            interacting[i] = xfcmp_lt( xadd( s, xmad( i, float(COLS), subtid ) ), nsrc )
            		       * xfcmp_lt( xdiff * xdiff + ydiff * ydiff + zdiff * zdiff, 1.f )
            		       * xfcmp_ne( dpid, spids[i] );
#endif
        }

		#pragma unroll
        for( uint i = 0; i < NSRCMAX; ++i ) {
            if( interacting[i] ) {
                float3 f;// = _dpd_interaction( dpid, xdest, udest, spids[i] ); // 88 FLOPS

                xforce += f.x; // 1 FLOP
                yforce += f.y; // 1 FLOP
                zforce += f.z; // 1 FLOP
                atomicAdd( FLOPS, 3ULL + _dpd_interaction_flops_counter::FLOPS );
            }
        }

        // 1 FLOP for s += NSRCMAX * COLS;
        atomicAdd( FLOPS, 1ULL );
    }

    for( uint L = COLS / 2; L > 0; L >>= 1 ) {
        xforce += __shfl_xor( xforce, L ); // 1 FLOP
        yforce += __shfl_xor( yforce, L ); // 1 FLOP
        zforce += __shfl_xor( zforce, L ); // 1 FLOP
        atomicAdd( FLOPS, 3ULL );
    }

#ifdef LETS_MAKE_IT_MESSY
    float fcontrib;
    asm("{   .reg .pred isy, isz;"
    	"     setp.f32.eq isy, %1, %5;"
    	"     setp.f32.eq isz, %1, %6;"
    	"     selp.f32 %0, %2, %3, !isy;"
    	"@isz mov.b32 %0, %4;"
    	"}" : "=f"(fcontrib) : "f"(u2f(subtid)), "f"(xforce), "f"(yforce), "f"(zforce), "f"(u2f(1u)), "f"(u2f(2u)) );
    // 2 FLOPS
    atomicAdd( FLOPS, 2ULL );
#else
    const float fcontrib = xsel_eq( subtid, 0u, xforce, xsel_eq( subtid, 1u, yforce, zforce ) );  // 2 FLOPS
#endif

    if( subtid < 3u ) {
        //info.axayaz[ xmad( dpid, 3.f, subtid ) ] = fcontrib;  // 2 FLOPS
    	atomicAdd( FLOPS, 2ULL );
    }
}

__global__ __launch_bounds__( 32 * CPB, 16 )
void _dpd_forces_floatized_flops_counter(unsigned long long *FLOPS)
{
    assert( blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1 );

    const uint tid = threadIdx.x;
    const uint wid = threadIdx.y;

    __shared__ volatile uint2 starts_and_scans[CPB*3][32];

    uint mycount = 0, myscan = 0;
    const int dx = ( tid ) % 3;
    const int dy = ( ( tid / 3 ) ) % 3;
    const int dz = ( ( tid / 9 ) ) % 3;

    if( tid < 27 ) {

        int xcid = blockIdx.x * _XCPB_ + ( ( threadIdx.y ) % _XCPB_ ) + dx - 1;
        int ycid = blockIdx.y * _YCPB_ + ( ( threadIdx.y / _XCPB_ ) % _YCPB_ ) + dy - 1;
        int zcid = blockIdx.z * _ZCPB_ + ( ( threadIdx.y / ( _XCPB_ * _YCPB_ ) ) % _ZCPB_ ) + dz - 1;

        const bool valid_cid =
                ( xcid >= 0 ) && ( xcid < info.ncells.x ) &&
                ( ycid >= 0 ) && ( ycid < info.ncells.y ) &&
                ( zcid >= 0 ) && ( zcid < info.ncells.z );

        xcid = xmin( xsub( info.ncells.x, 1 ), max( 0, xcid ) ); // 2 FLOPS
        ycid = xmin( xsub( info.ncells.y, 1 ), max( 0, ycid ) ); // 2 FLOPS
        zcid = xmin( xsub( info.ncells.z, 1 ), max( 0, zcid ) ); // 2 FLOPS
        atomicAdd( FLOPS, 6ULL );

        const int cid = max( 0, ( zcid * info.ncells.y + ycid ) * info.ncells.x + xcid );
		#if (USE_TEXOBJ&2)
        starts_and_scans[wid][tid].x = tex1Dfetch<uint>( info.txoStart, cid );
        myscan = mycount = valid_cid ? tex1Dfetch<uint>( info.txoCount, cid ) : 0u;
		#else
        starts_and_scans[wid][tid].x = tex1Dfetch( texStart, cid );
        myscan = mycount = valid_cid ? tex1Dfetch( texCount, cid ) : 0u;
		#endif
    }

	#pragma unroll
    for( int L = 1; L < 32; L <<= 1 ) {
    	uint theirscan = __shfl_up( myscan, L );
    	myscan = xadd( myscan, xsel_ge( tid, i2u(L), theirscan, 0u ) ); // 2 FLOPS
    	atomicAdd( FLOPS, 2ULL );
    }

    if( tid < 28 ) {
    	starts_and_scans[wid][tid].y = xsub( myscan, mycount ); // 1 FLOP
    	atomicAdd( FLOPS, 1ULL );
    }

    const uint nsrc = starts_and_scans[wid][27].y;
    const uint dststart = starts_and_scans[wid][1 + 3 + 9].x;
    const uint ndst = xsub( starts_and_scans[wid][1 + 3 + 9 + 1].y, starts_and_scans[wid][1 + 3 + 9].y ); // 1 FLOP
    const uint ndst4 = ( ndst >> 2 ) << 2;
    atomicAdd( FLOPS, 1ULL );

    for( uint d = 0; d < ndst4; d = xadd( d, 4u ) ) { // 1 FLOP
        core_flops_counter<8, 4, 4>( FLOPS, nsrc, ( const uint2 * )starts_and_scans[wid], 4, xadd( dststart, d ) ); // 1 FLOP
        atomicAdd( FLOPS, 2ULL );
    }

    uint d = ndst4;
    if( xadd( d, 2u ) <= ndst ) { // 1 FLOPS
        core_flops_counter<16, 2, 4>( FLOPS, nsrc, ( const uint2 * )starts_and_scans[wid], 2, xadd( dststart, d ) ); // 1 FLOP
        d = xadd( d, 2u ); // 1 FLOP
        atomicAdd( FLOPS, 3ULL );
    }

    if( d < ndst ) {
        core_ilp_flops_counter<32, 1, 2>( FLOPS, nsrc, ( const uint2 * )starts_and_scans[wid], 1, xadd( dststart, d ) ); // 1 FLOP
        atomicAdd( FLOPS, 1ULL );
    }
}

__global__ void reset_flops( unsigned long long *FLOPS ) {
	*FLOPS = 0ULL;
}

__global__ void print_flops( unsigned long long *FLOPS ) {
	printf("FLOPS count: %llu\n", *FLOPS);
}
#endif

bool fdpd_init = false;

#include "../hacks.h"
#ifdef _TIME_PROFILE_
static hipEvent_t evstart, evstop;
#endif

void forces_dpd_cuda_nohost( const float * const xyzuvw, float * const axayaz,  const int np,
                             const int * const cellsstart, const int * const cellscount,
                             const float rc,
                             const float XL, const float YL, const float ZL,
                             const float aij,
                             const float gamma,
                             const float sigma,
                             const float invsqrtdt,
                             const float seed, hipStream_t stream )
{
	if( np == 0 ) {
        printf( "WARNING: forces_dpd_cuda_nohost called with np = %d\n", np );
        return;
    }

    int nx = ( int )ceil( XL / rc );
    int ny = ( int )ceil( YL / rc );
    int nz = ( int )ceil( ZL / rc );
    const int ncells = nx * ny * nz;

	#if !(USE_TEXOBJ&2)
    size_t textureoffset;
	static float *xyz_o_uvw_o;
	static int last_size;
	if (!xyz_o_uvw_o || last_size < np ) {
			if (xyz_o_uvw_o) hipFree(xyz_o_uvw_o);
			hipMalloc(&xyz_o_uvw_o,sizeof(float)*8*np);
			last_size = np;
	}
    copy<<<64,512,0,stream>>>( xyz_o_uvw_o, xyzuvw, np );
    CUDA_CHECK( hipBindTexture( &textureoffset, &texParticles2, xyz_o_uvw_o, &texParticles2.channelDesc, sizeof( float ) * 8 * np ) );
    assert( textureoffset == 0 );
    CUDA_CHECK( hipBindTexture( &textureoffset, &texStart, cellsstart, &texStart.channelDesc, sizeof( uint ) * ncells ) );
    assert( textureoffset == 0 );
    CUDA_CHECK( hipBindTexture( &textureoffset, &texCount, cellscount, &texCount.channelDesc, sizeof( uint ) * ncells ) );
    assert( textureoffset == 0 );
	#endif

    InfoDPD c;
    c.ncells = make_int3( nx, ny, nz );
    c.ncell_x = nx;
    c.ncell_y = ny;
    c.domainsize = make_float3( XL, YL, ZL );
    c.invdomainsize = make_float3( 1 / XL, 1 / YL, 1 / ZL );
    c.domainstart = make_float3( -XL * 0.5, -YL * 0.5, -ZL * 0.5 );
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
    c.axayaz = axayaz;
    c.seed = seed;
	#if (USE_TEXOBJ&1)
    c.txoParticles2 = txoParticles2.rebind( (float2*)const_cast<float*>(xyzuvw), 3 * np );
    c.txoStart = txoStart.rebind( (uint*)const_cast<int*>(cellsstart), ncells );
    c.txoCount = txoCount.rebind( (uint*)const_cast<int*>(cellscount), ncells );
	#endif

	if( !fdpd_init ) {
		#if !(USE_TEXOBJ&2)
        texStart.channelDesc = hipCreateChannelDesc<uint>();
        texStart.filterMode = hipFilterModePoint;
        texStart.mipmapFilterMode = hipFilterModePoint;
        texStart.normalized = 0;

        texCount.channelDesc = hipCreateChannelDesc<uint>();
        texCount.filterMode = hipFilterModePoint;
        texCount.mipmapFilterMode = hipFilterModePoint;
        texCount.normalized = 0;

        texParticles2.channelDesc = hipCreateChannelDesc<float4>();
        texParticles2.filterMode = hipFilterModePoint;
        texParticles2.mipmapFilterMode = hipFilterModePoint;
        texParticles2.normalized = 0;
		#endif

	void ( *dpdkernel )() =  _dpd_forces_floatized;

        CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>( *dpdkernel), hipFuncCachePreferShared ) );

#ifdef _TIME_PROFILE_
        CUDA_CHECK( hipEventCreate( &evstart ) );
        CUDA_CHECK( hipEventCreate( &evstop ) );
#endif
        fdpd_init = true;
    }

    CUDA_CHECK( hipMemcpyToSymbolAsync(HIP_SYMBOL( info), &c, sizeof( c ), 0, hipMemcpyHostToDevice, stream ) );

    static int cetriolo = 0;
    cetriolo++;

#ifdef _TIME_PROFILE_
    if( cetriolo % 500 == 0 )
        CUDA_CHECK( hipEventRecord( evstart ) );
#endif
    _dpd_forces_floatized <<< dim3( c.ncells.x / _XCPB_,
                          c.ncells.y / _YCPB_,
                          c.ncells.z / _ZCPB_ ), dim3( 32, CPB ), 0, stream >>> ();

#ifdef _COUNT_FLOPS
    {
		static int nstep = 0;
		if ( ++nstep > 6950 ) {
			static unsigned long long *FLOPS;
			if (!FLOPS) hipMalloc( &FLOPS, 128 * sizeof(unsigned long long) );
			reset_flops<<<1,1,0,stream>>>(FLOPS);
			_dpd_forces_floatized_flops_counter <<< dim3( c.ncells.x / _XCPB_,
									  c.ncells.y / _YCPB_,
									  c.ncells.z / _ZCPB_ ), dim3( 32, CPB ), 0, stream >>> ( FLOPS );
			print_flops<<<1,1,0,stream>>>(FLOPS);
			//count FLOPS
			//report data to scree
		}
    }
#endif

    CUDA_CHECK( hipPeekAtLastError() );
}
