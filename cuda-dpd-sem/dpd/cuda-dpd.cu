#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

#include "../saru.cuh"

#ifndef NDEBUG
//#define _CHECK_
#endif

struct InfoDPD
{
    int3 ncells;
    float3 domainsize, invdomainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
};

__constant__ InfoDPD info;

texture<float2, hipTextureType1D> texParticles2;
texture<int, hipTextureType1D> texStart, texCount;
 
#define COLS 16
#define ROWS (32 / COLS)
#define _XCPB_ 4
#define _YCPB_ 1
#define _ZCPB_ 1
#define CPB (_XCPB_ * _YCPB_ * _ZCPB_)

#if 1
__device__ float3 _dpd_interaction(const int idtimestep, const int dpid, const float3 xdest, const float3 udest, const int spid)
{
    const int sentry = 3 * spid;
    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);
    const float2 stmp2 = tex1Dfetch(texParticles2, sentry + 2);
    
    const float _xr = xdest.x - stmp0.x;
    const float _yr = xdest.y - stmp0.y;
    const float _zr = xdest.z - stmp1.x;
   
    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
    assert(rij2 < 1);
    
    const float invrij = rsqrtf(rij2);
    const float rij = rij2 * invrij;
    const float argwr = max((float)0, 1 - rij * info.invrc);
    const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));
    
    const float xr = _xr * invrij;
    const float yr = _yr * invrij;
    const float zr = _zr * invrij;
    
    const float rdotv = 
	xr * (udest.x - stmp1.y) +
	yr * (udest.y - stmp2.x) +
	zr * (udest.z - stmp2.y);
    
    const float mysaru = saru(min(spid, dpid), max(spid, dpid), idtimestep);
    const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
    
    const float strength = info.aij * argwr - (info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;
    
    return make_float3(strength * xr, strength * yr, strength * zr);
}

#define _SRCIDS_LMEM_
#define _NSRCMAX_ 3

__global__ __launch_bounds__(32 * CPB, 16) 
void _dpd_forces_saru(float * const axayaz, const int idtimestep)
{
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int tid = threadIdx.x; 
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;

    enum { NSRCMAX = _NSRCMAX_ };

#ifdef _SRCIDS_LMEM_ 
    int srcids[NSRCMAX];
    for(int i = 0; i < NSRCMAX; ++i)
	srcids[i] = 0;
#else
    __shared__ int srcids[NSRCMAX][CPB][32];
#endif    

    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0, myscan = 0; 
    if (tid < 27)
    {
	const int dx = (tid) % 3;
	const int dy = ((tid / 3)) % 3; 
	const int dz = ((tid / 9)) % 3;

	int xcid = blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1;
	int ycid = blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1;
	int zcid = blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1;
	
	const bool valid_cid = 
	    xcid >= 0 && xcid < info.ncells.x &&
	    ycid >= 0 && ycid < info.ncells.y &&
	    zcid >= 0 && zcid < info.ncells.z ;
	
	xcid = min(info.ncells.x - 1, max(0, xcid));
	ycid = min(info.ncells.y - 1, max(0, ycid));
	zcid = min(info.ncells.z - 1, max(0, zcid));
	
	const int cid = max(0, xcid + info.ncells.x * (ycid + info.ncells.y * zcid));
	
	starts[wid][tid] = tex1Dfetch(texStart, cid);
	
	myscan = mycount = valid_cid * tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	myscan += (tid >= L) * __shfl_up(myscan, L) ;

    if (tid < 28)
	scan[wid][tid] = myscan - mycount;

    const int dststart = starts[wid][1 + 3 + 9];
    const int nsrc = scan[wid][27], ndst = scan[wid][1 + 3 + 9 + 1] - scan[wid][1 + 3 + 9];
 
    for(int d = 0; d < ndst; d += ROWS)
    {
	int srccount = 0;
	
	const int np1 = min(ndst - d, ROWS);

	if (slot >= np1)
	    continue;

	const int dpid = dststart + d + slot;
	const int entry = 3 * dpid;
	
	const float2 dtmp0 = tex1Dfetch(texParticles2, entry);
	const float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
	const float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
	const float3 xdest = make_float3(dtmp0.x, dtmp0.y, dtmp1.x);
	const float3 udest = make_float3(dtmp1.y, dtmp2.x, dtmp2.y);
	
	float xforce = 0, yforce = 0, zforce = 0;

	for(int s = 0; s < nsrc; s += COLS)
	{
	    // const int np2 = min(nsrc - s, COLS);
	    
	    //if (s + subtid >= nsrc)
	    //continue;
	    
	    const int pid = s + subtid;
	    const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
	    const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
	    const int key = key9 + key3;	    
	   
	    const int spid = pid - scan[wid][key] + starts[wid][key];

	    //if (dpid == spid)
	    //continue;

	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);

	    const float xdiff = xdest.x - stmp0.x;
	    const float ydiff = xdest.y - stmp0.y;
	    const float zdiff = xdest.z - stmp1.x;
	    const bool interacting = (s + subtid < nsrc) && (dpid != spid) && (xdiff * xdiff + ydiff * ydiff + zdiff * zdiff < 1);
	    
	    //xforce += interacting * 1e-5f;
	    //if (interacting)
#ifdef _SRCIDS_LMEM_
		srcids[srccount] = spid;
#else
	srcids[srccount][wid][tid] = spid;
#endif

	srccount += interacting;
	    
	if (srccount == NSRCMAX)
	{
	    const float3 f = _dpd_interaction(idtimestep, dpid, xdest, udest, 
#ifdef _SRCIDS_LMEM_
					      srcids[NSRCMAX - 1]);
#else
	    srcids[NSRCMAX - 1][wid][tid]);
#endif
	srccount = NSRCMAX - 1;
	
	xforce += f.x; yforce += f.y; zforce += f.z;
	}
    }
    

#pragma unroll _NSRCMAX_
    for(int i = 0; i < srccount; ++i)
    {
	    const float3 f = _dpd_interaction(idtimestep, dpid, xdest, udest, 
#ifdef _SRCIDS_LMEM_
					      srcids[i]);
#else
	    srcids[i][wid][tid]);
#endif
    
    xforce += f.x; yforce += f.y; zforce += f.z;
}
	
	for(int L = COLS / 2; L > 0; L >>=1)
	{
	    xforce += __shfl_xor(xforce, L);
	    yforce += __shfl_xor(yforce, L);
	    zforce += __shfl_xor(zforce, L);
	    }

	const float fcontrib = (subtid == 0) * xforce + (subtid == 1) * yforce + (subtid == 2) * zforce;

	if (slot < np1 && subtid < 3)
	    axayaz[subtid + 3 * dpid] = fcontrib;
    }
}

#else
__global__ __launch_bounds__(32 * CPB, 16) 
    void _dpd_forces_saru(float * const axayaz,
			  const int idtimestep)
{
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int tid = threadIdx.x; 
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;
     
    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0, myscan = 0; 
    if (tid < 27)
    {
	const int dx = (tid) % 3;
	const int dy = ((tid / 3)) % 3; 
	const int dz = ((tid / 9)) % 3;

	int xcid = blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1;
	int ycid = blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1;
	int zcid = blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1;
	
	const bool valid_cid = 
	    xcid >= 0 && xcid < info.ncells.x &&
	    ycid >= 0 && ycid < info.ncells.y &&
	    zcid >= 0 && zcid < info.ncells.z ;
	
	xcid = min(info.ncells.x - 1, max(0, xcid));
	ycid = min(info.ncells.y - 1, max(0, ycid));
	zcid = min(info.ncells.z - 1, max(0, zcid));
	
	const int cid = max(0, xcid + info.ncells.x * (ycid + info.ncells.y * zcid));
	
	starts[wid][tid] = tex1Dfetch(texStart, cid);
	
	myscan = mycount = valid_cid * tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	myscan += (tid >= L) * __shfl_up(myscan, L) ;

    if (tid < 28)
	scan[wid][tid] = myscan - mycount;

    const int dststart = starts[wid][1 + 3 + 9];
    const int nsrc = scan[wid][27], ndst = scan[wid][1 + 3 + 9 + 1] - scan[wid][1 + 3 + 9];
 
    for(int d = 0; d < ndst; d += ROWS)
    {
	const int np1 = min(ndst - d, ROWS);

	const int dpid = dststart + d + slot;
	const int entry = 3 * dpid;
	float2 dtmp0 = tex1Dfetch(texParticles2, entry);
	float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
	float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
	
	float xforce = 0, yforce = 0, zforce = 0;

	for(int s = 0; s < nsrc; s += COLS)
	{
	    const int np2 = min(nsrc - s, COLS);
  
	    const int pid = s + subtid;
	    const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
	    const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
	    const int key = key9 + key3;	    
	   
	    const int spid = pid - scan[wid][key] + starts[wid][key];
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);
	    const float2 stmp2 = tex1Dfetch(texParticles2, sentry + 2);

#ifndef NDEBUG
	    {
		const int key1 = (pid >= scan[wid][key9 + key3 + 1]) + (pid >= scan[wid][key9 + key3 + 2]);
		const int keyref = key9 + key3 + key1;
		assert(keyref >= 0 && keyref < 27);
		assert(pid >= scan[wid][keyref]);
		assert(pid < scan[wid][keyref + 1] || pid >= nsrc);

		const int spidref = pid - scan[wid][keyref] + starts[wid][keyref];
		assert(spidref == spid || pid >= nsrc);
	    }
#endif
	    
	    {
		const float xdiff = dtmp0.x - stmp0.x;
		const float ydiff = dtmp0.y - stmp0.y;
		const float zdiff = dtmp1.x - stmp1.x;

#ifndef _NONPERIODIC_KERNEL_
		asdasda
		const float _xr = xdiff - info.domainsize.x * floorf(0.5f + xdiff * info.invdomainsize.x);
		const float _yr = ydiff - info.domainsize.y * floorf(0.5f + ydiff * info.invdomainsize.y);
		const float _zr = zdiff - info.domainsize.z * floorf(0.5f + zdiff * info.invdomainsize.z);
#else
		const float _xr = xdiff;
		const float _yr = ydiff;
		const float _zr = zdiff;
#endif
		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		const float invrij = rsqrtf(rij2);
		const float rij = rij2 * invrij;
		const float argwr = max((float)0, 1 - rij * info.invrc);
		const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;
		
		const float rdotv = 
		    xr * (dtmp1.y - stmp1.y) +
		    yr * (dtmp2.x - stmp2.x) +
		    zr * (dtmp2.y - stmp2.y);
		  
		const float mysaru = saru(min(spid, dpid), max(spid, dpid), idtimestep);
		const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
		 
		const float strength = info.aij * argwr - (info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;
		const bool valid = (dpid != spid) && (slot < np1) && (subtid < np2);
		
		if (valid)
		{
#ifdef _CHECK_
		    xforce += (rij2 < 1);
		    yforce += wr;
		    zforce += 0;
#else		    	     
		    xforce += strength * xr;
		    yforce += strength * yr;
		    zforce += strength * zr;
#endif
		}
	    } 
	}
	
	for(int L = COLS / 2; L > 0; L >>=1)
	{
	    xforce += __shfl_xor(xforce, L);
	    yforce += __shfl_xor(yforce, L);
	    zforce += __shfl_xor(zforce, L);
	}

	const int c = (subtid % 3);       
	const float fcontrib = (c == 0) * xforce + (c == 1) * yforce + (c == 2) * zforce;//f[subtid % 3];
	const int dstpid = dststart + d + slot;

	if (slot < np1)
	    axayaz[c + 3 * dstpid] = fcontrib;
    }
}
#endif


__global__ __launch_bounds__(32 * CPB, 16) 
    void inspect_dpd_forces_saru(const int nparticles, int2 * const entries, const int nentries)
{
    assert(nentries = COLS * nparticles);
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int tid = threadIdx.x; 
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;
 
    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0, myscan = 0; 
    if (tid < 27)
    {
	const int dx = (tid) % 3;
	const int dy = ((tid / 3)) % 3; 
	const int dz = ((tid / 9)) % 3;

	int xcid = blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1;
	int ycid = blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1;
	int zcid = blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1;
	
	const bool valid_cid = 
	    xcid >= 0 && xcid < info.ncells.x &&
	    ycid >= 0 && ycid < info.ncells.y &&
	    zcid >= 0 && zcid < info.ncells.z ;
	
	xcid = min(info.ncells.x - 1, max(0, xcid));
	ycid = min(info.ncells.y - 1, max(0, ycid));
	zcid = min(info.ncells.z - 1, max(0, zcid));
	
	const int cid = max(0, xcid + info.ncells.x * (ycid + info.ncells.y * zcid));
	
	starts[wid][tid] = tex1Dfetch(texStart, cid);
	
	myscan = mycount = valid_cid * tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	myscan += (tid >= L) * __shfl_up(myscan, L) ;

    if (tid < 28)
	scan[wid][tid] = myscan - mycount;

    const int dststart = starts[wid][1 + 3 + 9];
    const int nsrc = scan[wid][27], ndst = scan[wid][1 + 3 + 9 + 1] - scan[wid][1 + 3 + 9];
 
    for(int d = 0; d < ndst; d += ROWS)
    {
	int srccount = 0;
	
	const int np1 = min(ndst - d, ROWS);

	const int dpid = dststart + d + slot;
	const int entry = 3 * dpid;
	
	const float2 dtmp0 = tex1Dfetch(texParticles2, entry);
	const float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
	const float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
	const float3 xdest = make_float3(dtmp0.x, dtmp0.y, dtmp1.x);
	const float3 udest = make_float3(dtmp1.y, dtmp2.x, dtmp2.y);
	
	int ninteractions = 0, npotentialinteractions = 0;
	
	for(int s = 0; s < nsrc; s += COLS)
	{
	    const int np2 = min(nsrc - s, COLS);
  
	    const int pid = s + subtid;
	    const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
	    const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
	    const int key = key9 + key3;	    
	   
	    const int spid = pid - scan[wid][key] + starts[wid][key];
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);

	    const float xdiff = xdest.x - stmp0.x;
	    const float ydiff = xdest.y - stmp0.y;
	    const float zdiff = xdest.z - stmp1.x;
	    const bool interacting = (dpid != spid) && (slot < np1) && (subtid < np2) &&
		(xdiff * xdiff + ydiff * ydiff + zdiff * zdiff < 1);
    
	    ninteractions += (int)(interacting);
	    npotentialinteractions += 1;
	}

	if (slot < np1)
	    entries[subtid + COLS * dpid] = make_int2(ninteractions, npotentialinteractions);
    }
}


bool fdpd_init = false;

#include "../hacks.h"

static hipEvent_t evstart, evstop;

void forces_dpd_cuda_nohost(const float * const xyzuvw, float * const axayaz,  const int np,
			    const int * const cellsstart, const int * const cellscount, 
			    const float rc,
			    const float XL, const float YL, const float ZL,
			    const float aij,
			    const float gamma,
			    const float sigma,
			    const float invsqrtdt,
			    const int saru_tag, hipStream_t stream)
{
    if (np == 0)
    {
	printf("WARNING: forces_dpd_cuda_nohost called with np = %d\n", np);
	return;
    }

    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;

    if (!fdpd_init)
    {
	texStart.channelDesc = hipCreateChannelDesc<int>();
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;
    
	texCount.channelDesc = hipCreateChannelDesc<int>();
	texCount.filterMode = hipFilterModePoint;
	texCount.mipmapFilterMode = hipFilterModePoint;
	texCount.normalized = 0;

	texParticles2.channelDesc = hipCreateChannelDesc<float2>();
	texParticles2.filterMode = hipFilterModePoint;
	texParticles2.mipmapFilterMode = hipFilterModePoint;
	texParticles2.normalized = 0;

	void (*dpdkernel)(float * const axayaz, const int idtimestep) =  _dpd_forces_saru;
#ifdef _SRCIDS_LMEM_
	CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*dpdkernel), hipFuncCachePreferL1));
#else
	CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*dpdkernel), hipFuncCachePreferNone));
#endif

	CUDA_CHECK(hipEventCreate(&evstart));
	CUDA_CHECK(hipEventCreate(&evstop));

	fdpd_init = true;
    }

    size_t textureoffset;
    CUDA_CHECK(hipBindTexture(&textureoffset, &texParticles2, xyzuvw, &texParticles2.channelDesc, sizeof(float) * 6 * np));
    assert(textureoffset == 0);
    CUDA_CHECK(hipBindTexture(&textureoffset, &texStart, cellsstart, &texStart.channelDesc, sizeof(int) * ncells));
    assert(textureoffset == 0);
    CUDA_CHECK(hipBindTexture(&textureoffset, &texCount, cellscount, &texCount.channelDesc, sizeof(int) * ncells));
    assert(textureoffset == 0);
      
    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.domainsize = make_float3(XL, YL, ZL);
    c.invdomainsize = make_float3(1 / XL, 1 / YL, 1 / ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
      
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &c, sizeof(c), 0, hipMemcpyHostToDevice, stream));

    static int cetriolo = 0;
    cetriolo++;

#if 0
    {
	

	if (cetriolo % 1000 == 0)
	{
	    int2 * data;
	    size_t nentries = np * COLS;
	    CUDA_CHECK(hipHostAlloc(&data, sizeof(int2) * nentries, hipHostMallocMapped));
	    memset(data, 0xff, sizeof(int2) * nentries);
	    
	    int * devptr;
	    CUDA_CHECK(hipHostGetDevicePointer(&devptr, data, 0));

	    inspect_dpd_forces_saru<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB), 0, stream>>>(
				np, data, nentries);

	    CUDA_CHECK(hipDeviceSynchronize());
	    

	    char path2report[2000];
	    sprintf(path2report, "inspection-%d-tstep.txt", cetriolo);
	    FILE * f = fopen(path2report, "w");
	    assert(f);
	       for(int i = 0, c = 0; i < np; ++i)
	    {
		fprintf(f, "pid %05d: ", i);
		
		int s = 0, pot = 0;
		for(int j = 0; j < COLS; ++j, ++c)
		{
		    fprintf(f, "%02d ", data[c].x);
		    s += data[c].x;
		    pot += data[c].y;
		}
		
		fprintf(f, " sum: %02d pot: %d\n", s, (pot + COLS - 1) / (COLS));
		}
	    fclose(f);
	    
	    CUDA_CHECK(hipHostFree(data));
	    printf("inspection concluded.\n");
	}
    }
#endif

    if (cetriolo % 500 == 0)
	CUDA_CHECK(hipEventRecord(evstart));
      
    _dpd_forces_saru<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB), 0, stream>>>(axayaz, saru_tag);

    if (cetriolo % 500 == 0)
    {
	CUDA_CHECK(hipEventRecord(evstop));
	CUDA_CHECK(hipEventSynchronize(evstop));
	
	float tms;
	CUDA_CHECK(hipEventElapsedTime(&tms, evstart, evstop));
	printf("elapsed time for DPD-BULK kernel: %.2f ms\n", tms);
    }

    CUDA_CHECK(hipPeekAtLastError());	
}

#include <cmath>
#include <unistd.h>

//#include <thrust/device_vector.h>
//using namespace thrust;

#include "../profiler-dpd.h"
#include "../cell-lists.h"




int fdpd_oldnp = 0, fdpd_oldnc = 0;

float * fdpd_xyzuvw = NULL, * fdpd_axayaz = NULL;
int * fdpd_start = NULL, * fdpd_count = NULL;

void forces_dpd_cuda_aos(float * const _xyzuvw, float * const _axayaz,
		     int * const order, const int np,
		     const float rc,
		     const float XL, const float YL, const float ZL,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
			 const int saru_tag,
			 const bool nohost)
{
    if (np == 0)
    {
	printf("WARNING: forces_dpd_cuda_aos called with np = %d\n", np);
	return;
    }

    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;

    if (!fdpd_init)
    {
	texStart.channelDesc = hipCreateChannelDesc<int>();
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;
    
	texCount.channelDesc = hipCreateChannelDesc<int>();
	texCount.filterMode = hipFilterModePoint;
	texCount.mipmapFilterMode = hipFilterModePoint;
	texCount.normalized = 0;

	texParticles2.channelDesc = hipCreateChannelDesc<float2>();
	texParticles2.filterMode = hipFilterModePoint;
	texParticles2.mipmapFilterMode = hipFilterModePoint;
	texParticles2.normalized = 0;

	fdpd_init = true;
    }
    
    if (fdpd_oldnp < np)
    {
	if (fdpd_oldnp > 0)
	{
	    CUDA_CHECK(hipFree(fdpd_xyzuvw));
	    CUDA_CHECK(hipFree(fdpd_axayaz));
	}

	CUDA_CHECK(hipMalloc(&fdpd_xyzuvw, sizeof(float) * 6 * np));
	CUDA_CHECK(hipMalloc(&fdpd_axayaz, sizeof(float) * 3 * np));

	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texParticles2, fdpd_xyzuvw, &texParticles2.channelDesc, sizeof(float) * 6 * np));
	
	fdpd_oldnp = np;
    }

    if (fdpd_oldnc < ncells)
    {
	if (fdpd_oldnc > 0)
	{
	    CUDA_CHECK(hipFree(fdpd_start));
	    CUDA_CHECK(hipFree(fdpd_count));
	}

	CUDA_CHECK(hipMalloc(&fdpd_start, sizeof(int) * ncells));
	CUDA_CHECK(hipMalloc(&fdpd_count, sizeof(int) * ncells));

	size_t textureoffset = 0;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texStart, fdpd_start, &texStart.channelDesc, sizeof(int) * ncells));
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCount, fdpd_count, &texCount.channelDesc, sizeof(int) * ncells));
	
	fdpd_oldnc = ncells;
    }

    CUDA_CHECK(hipMemcpyAsync(fdpd_xyzuvw, _xyzuvw, sizeof(float) * np * 6, nohost ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice, 0));
    
    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.domainsize = make_float3(XL, YL, ZL);
    c.invdomainsize = make_float3(1 / XL, 1 / YL, 1 / ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
        
    build_clists(fdpd_xyzuvw, np, rc, c.ncells.x, c.ncells.y, c.ncells.z,
		 c.domainstart.x, c.domainstart.y, c.domainstart.z,
		 order, fdpd_start, fdpd_count, NULL);

    //TextureWrap texStart(_ptr(starts), ncells), texCount(_ptr(counts), ncells);
    //TextureWrap texParticles((float2*)_ptr(xyzuvw), 3 * np);
    
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &c, sizeof(c), 0));
   
    ProfilerDPD::singletone().start();

    if (saru_tag >= 0)
	saru_tid = saru_tag;
    
    _dpd_forces_saru<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB)>>>(fdpd_axayaz, saru_tid);
 
    ++saru_tid;

    CUDA_CHECK(hipPeekAtLastError());
	
    ProfilerDPD::singletone().force();
    
//copy xyzuvw as well?!?
    if (nohost)
    {
	CUDA_CHECK(hipMemcpy(_xyzuvw, fdpd_xyzuvw, sizeof(float) * 6 * np, hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(_axayaz, fdpd_axayaz, sizeof(float) * 3 * np, hipMemcpyDeviceToDevice));
    }
    else
	CUDA_CHECK(hipMemcpy(_axayaz, fdpd_axayaz, sizeof(float) * 3 * np, hipMemcpyDeviceToHost));

    ProfilerDPD::singletone().report();

    //copy(axayaz.begin(), axayaz.end(), _axayaz);
     
#ifdef _CHECK_
    CUDA_CHECK(hipDeviceSynchronize());
    
    for(int ii = 0; ii < np; ++ii)
    { 
	printf("pid %d -> %f %f %f\n", ii, (float)axayaz[0 + 3 * ii], (float)axayaz[1 + 3* ii], (float)axayaz[2 + 3 *ii]);

	int cnt = 0;
	float fc = 0;
	const int i = order[ii];
	printf("devi coords are %f %f %f\n", (float)xyzuvw[0 + 6 * ii], (float)xyzuvw[1 + 6 * ii], (float)xyzuvw[2 + 6 * ii]);
	printf("host coords are %f %f %f\n", (float)_xyzuvw[0 + 6 * i], (float)_xyzuvw[1 + 6 * i], (float)_xyzuvw[2 + 6 * i]);
	
	for(int j = 0; j < np; ++j)
	{
	    if (i == j) 
		continue;
 
	    float xr = _xyzuvw[0 + 6 *i] - _xyzuvw[0 + 6 * j];
	    float yr = _xyzuvw[1 + 6 *i] - _xyzuvw[1 + 6 * j];
	    float zr = _xyzuvw[2 + 6 *i] - _xyzuvw[2 + 6 * j];

	    xr -= c.domainsize.x *  ::floor(0.5f + xr / c.domainsize.x);
	    yr -= c.domainsize.y *  ::floor(0.5f + yr / c.domainsize.y);
	    zr -= c.domainsize.z *  ::floor(0.5f + zr / c.domainsize.z);

	    const float rij2 = xr * xr + yr * yr + zr * zr;
	    const float invrij = rsqrtf(rij2);
	    const float rij = rij2 * invrij;
	    const float wr = max((float)0, 1 - rij * c.invrc);
	
	    const bool collision =  rij2 < 1;

	    if (collision)
		fc += wr;//	printf("ref p %d colliding with %d\n", i, j);
	    
	    cnt += collision;
	}
	printf("i found %d host interactions and with cuda i found %d\n", cnt, (int)axayaz[0 + 3 * ii]);
	assert(cnt == (float)axayaz[0 + 3 * ii]);
	printf("fc aij ref %f vs cuda %e\n", fc,  (float)axayaz[1 + 3 * ii]);
	assert(fabs(fc - (float)axayaz[1 + 3 * ii]) < 1e-4);
    }
    
    printf("test done.\n");
    sleep(1);
    exit(0);
#endif
}

int * fdpd_order = NULL;
float * fdpd_pv = NULL, *fdpd_a = NULL;

void forces_dpd_cuda(const float * const xp, const float * const yp, const float * const zp,
		     const float * const xv, const float * const yv, const float * const zv,
		     float * const xa, float * const ya, float * const za,
		     const int np,
		     const float rc,
		     const float LX, const float LY, const float LZ,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
		     const int input_saru_tag)
{
    if (np <= 0) return;

    if (np > fdpd_oldnp)
    {
	if (fdpd_oldnp > 0)
	{
	    CUDA_CHECK(hipHostFree(fdpd_pv));
	    CUDA_CHECK(hipHostFree(fdpd_order));
	    CUDA_CHECK(hipHostFree(fdpd_a));
	}

	CUDA_CHECK(hipHostAlloc(&fdpd_pv, sizeof(float) * np * 6, hipHostMallocDefault));
	CUDA_CHECK(hipHostAlloc(&fdpd_order, sizeof(int) * np, hipHostMallocDefault));
	CUDA_CHECK(hipHostAlloc(&fdpd_a, sizeof(float) * np * 3, hipHostMallocDefault));

	//this will be done by forces_dpd_cuda
	//fdpd_oldnp = np;
    }
    
    for(int i = 0; i < np; ++i)
    {
	fdpd_pv[0 + 6 * i] = xp[i];
	fdpd_pv[1 + 6 * i] = yp[i];
	fdpd_pv[2 + 6 * i] = zp[i];
	fdpd_pv[3 + 6 * i] = xv[i];
	fdpd_pv[4 + 6 * i] = yv[i];
	fdpd_pv[5 + 6 * i] = zv[i];
    }

    forces_dpd_cuda_aos(fdpd_pv, fdpd_a, fdpd_order, np, rc, LX, LY, LZ,
			aij, gamma, sigma, invsqrtdt, input_saru_tag, false);
    
    //delete [] pv;
     
    for(int i = 0; i < np; ++i)
    {
	xa[fdpd_order[i]] += fdpd_a[0 + 3 * i];
	ya[fdpd_order[i]] += fdpd_a[1 + 3 * i];
	za[fdpd_order[i]] += fdpd_a[2 + 3 * i];
    }

    //delete [] a;

    //delete [] order;
}