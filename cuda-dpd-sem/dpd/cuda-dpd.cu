#include "hip/hip_runtime.h"
/*
 *  cuda-dpd.cu
 *  Part of CTC/cuda-dpd-sem/dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-02-26.
 *  Major editing by Mauro Bisson on 2015-04-01.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cstdio>
#include <cassert>

#include "cuda-dpd.h"
#include "../dpd-rng.h"

struct InfoDPD
{
    int3 ncells;
    float3 domainsize, invdomainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
    float * axayaz;
    float seed;
};

__constant__ InfoDPD info;

texture<float2, hipTextureType1D> texParticles2;
texture<int, hipTextureType1D> texStart, texCount;

#define _XCPB_ 2
#define _YCPB_ 2
#define _ZCPB_ 1
#define CPB (_XCPB_ * _YCPB_ * _ZCPB_)
//#define  _TIME_PROFILE_
//#define _INSPECT_

__device__ float3 _dpd_interaction(const int dpid, const float3 xdest, const float3 udest, const int spid, const float2 stmp0, const float2 stmp1)
{
    const int sentry = 3 * spid;
    //const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
    //const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);
    const float2 stmp2 = tex1Dfetch(texParticles2, sentry + 2);

    const float _xr = xdest.x - stmp0.x;
    const float _yr = xdest.y - stmp0.y;
    const float _zr = xdest.z - stmp1.x;
    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
    assert(rij2 < 1);

    const float invrij = rsqrtf(rij2);
    const float rij = rij2 * invrij;
    const float argwr = 1 - rij;
    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

    const float xr = _xr * invrij;
    const float yr = _yr * invrij;
    const float zr = _zr * invrij;

    const float rdotv =
	xr * (udest.x - stmp1.y) +
	yr * (udest.y - stmp2.x) +
	zr * (udest.z - stmp2.y);

    const float myrandnr = Logistic::mean0var1(info.seed, min(spid, dpid), max(spid, dpid));

    const float strength = info.aij * argwr - (info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;

    return make_float3(strength * xr, strength * yr, strength * zr);
}

#define __IMOD(x,y) ((x)-((x)/(y))*(y))

template<int COLS, int ROWS>
__global__ void _dpd_forces_new2()
{
    int mycount=0, myscan=0;

    __shared__ int volatile starts[CPB][16], scan[CPB][16];

    if (threadIdx.x < 14)
    {
	const int cbase = blockIdx.x * blockDim.y + threadIdx.y;

	int dx, dy, dz;
	dx = dy = dz = threadIdx.x / 3;
	dx = threadIdx.x - dx * 3 - 1;
	dy = __IMOD(dy, 3) - 1;
	dz = __IMOD(dz / 3, 3) - 1;

	int cid = cbase + dz * info.ncells.x * info.ncells.y + dy * info.ncells.x + dx;

	const bool valid_cid = (cid >= 0) && (cid < info.ncells.x * info.ncells.y * info.ncells.z);

	starts[threadIdx.y][threadIdx.x] = (valid_cid) ? tex1Dfetch(texStart, cid) : 0;

	myscan = mycount = (valid_cid) ? tex1Dfetch(texCount, cid) : 0;
    }

#pragma unroll
    for(int L = 1; L < 16; L <<= 1)
	myscan += (threadIdx.x >= L) * __shfl_up(myscan, L);

    if (threadIdx.x < 15)
	scan[threadIdx.y][threadIdx.x] = myscan - mycount;

    const int subtid = threadIdx.x % COLS;
    const int slot = threadIdx.x / COLS;

    const int dststart = starts[threadIdx.y][13];
    const int lastdst = dststart + scan[threadIdx.y][14] - scan[threadIdx.y][13];

    const int nsrc = scan[threadIdx.y][14];
    const int nsrcext = scan[threadIdx.y][13];

    for(int pid = subtid; pid < nsrc; pid += COLS)
    {
	const int key9 = 9 * (pid >= scan[threadIdx.y][9]);

	int key3 = 3 * (pid >= scan[threadIdx.y][key9 + 3]);
	key3 += (key9 < 9) ? 3 * (pid >= scan[threadIdx.y][key9 + 6]) : 0;

	int spid = pid - scan[threadIdx.y][key3 + key9] + starts[threadIdx.y][key3 + key9];

	const int sentry = 3 * spid;
	const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);

	for(int dpid = dststart + slot; dpid < lastdst; dpid += ROWS)
	{
	    float3 xdest, udest;

	    float2 dtmp0 = tex1Dfetch(texParticles2, 3 * dpid);
	    xdest.x = dtmp0.x;
	    xdest.y = dtmp0.y;

	    dtmp0 = tex1Dfetch(texParticles2, 3 * dpid + 1);
	    xdest.z = dtmp0.x;
	    udest.x = dtmp0.y;

	    dtmp0 = tex1Dfetch(texParticles2, 3 * dpid + 2);
	    udest.y = dtmp0.x;
	    udest.z = dtmp0.y;

	    const float d2 =
		(xdest.x - stmp0.x) * (xdest.x - stmp0.x) +
		(xdest.y - stmp0.y) * (xdest.y - stmp0.y) +
		(xdest.z - stmp1.x) * (xdest.z - stmp1.x);

	    if ((dpid != spid) && (d2 < 1.0f))
	    {
		const float3 f = _dpd_interaction(dpid, xdest, udest, spid, stmp0, stmp1);

		atomicAdd(info.axayaz + 3*dpid    , f.x);
		atomicAdd(info.axayaz + 3*dpid + 1, f.y);
		atomicAdd(info.axayaz + 3*dpid + 2, f.z);

		if (pid < nsrcext)
		{
		    atomicAdd(info.axayaz + 3*spid    , -f.x);
		    atomicAdd(info.axayaz + 3*spid + 1, -f.y);
		    atomicAdd(info.axayaz + 3*spid + 2, -f.z);
		}
	    }
	}
    }
}

#if 1

__device__ float3 _dpd_interaction(const int dpid, const float3 xdest, const float3 udest, const int spid)
{
    const int sentry = 3 * spid;
    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);
    const float2 stmp2 = tex1Dfetch(texParticles2, sentry + 2);

    const float _xr = xdest.x - stmp0.x;
    const float _yr = xdest.y - stmp0.y;
    const float _zr = xdest.z - stmp1.x;

    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
    assert(rij2 < 1);

    const float invrij = rsqrtf(rij2);
    const float rij = rij2 * invrij;
    const float argwr = 1 - rij;
    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

    const float xr = _xr * invrij;
    const float yr = _yr * invrij;
    const float zr = _zr * invrij;

    const float rdotv =
	xr * (udest.x - stmp1.y) +
	yr * (udest.y - stmp2.x) +
	zr * (udest.z - stmp2.y);

    const float myrandnr = Logistic::mean0var1(info.seed, min(spid, dpid), max(spid, dpid));

    const float strength = info.aij * argwr - (info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;

    return make_float3(strength * xr, strength * yr, strength * zr);
}

template<int COLS, int ROWS, int NSRCMAX>
__device__ void core(const int nsrc, const int * const scan, const int * const starts,
		     const int ndst, const int dststart)
{
   int srcids[NSRCMAX];
    for(int i = 0; i < NSRCMAX; ++i)
	srcids[i] = 0;

    int srccount = 0;
    assert(ndst == ROWS);

    const int tid = threadIdx.x;
    const int slot = tid / COLS;
    const int subtid = tid % COLS;

    const int dpid = dststart + slot;
    const int entry = 3 * dpid;
    const float2 dtmp0 = tex1Dfetch(texParticles2, entry);
    const float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
    const float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
    const float3 xdest = make_float3(dtmp0.x, dtmp0.y, dtmp1.x);
    const float3 udest = make_float3(dtmp1.y, dtmp2.x, dtmp2.y);

    float xforce = 0, yforce = 0, zforce = 0;

    for(int s = 0; s < nsrc; s += COLS)
    {
	const int pid = s + subtid;
	const int key9 = 9 * ((pid >= scan[9]) + (pid >= scan[18]));
	const int key3 = 3 * ((pid >= scan[key9 + 3]) + (pid >= scan[key9 + 6]));
	const int key = key9 + key3;

	const int spid = pid - scan[key] + starts[key];

	const int sentry = 3 * spid;
	const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);

	const float xdiff = xdest.x - stmp0.x;
	const float ydiff = xdest.y - stmp0.y;
	const float zdiff = xdest.z - stmp1.x;
	const bool interacting = (s + subtid < nsrc) && (dpid != spid) && (xdiff * xdiff + ydiff * ydiff + zdiff * zdiff < 1);

	srcids[srccount] = spid;
	srccount += interacting;

	if (srccount == NSRCMAX)
	{
	    const float3 f = _dpd_interaction(dpid, xdest, udest, srcids[NSRCMAX - 1]);

	    xforce += f.x;
	    yforce += f.y;
	    zforce += f.z;

	    srccount = NSRCMAX - 1;
	}
    }

#pragma unroll 4
    for(int i = 0; i < srccount; ++i)
    {
	const float3 f = _dpd_interaction(dpid, xdest, udest, srcids[i]);

	xforce += f.x;
	yforce += f.y;
	zforce += f.z;
    }

    for(int L = COLS / 2; L > 0; L >>=1)
    {
	xforce += __shfl_xor(xforce, L);
	yforce += __shfl_xor(yforce, L);
	zforce += __shfl_xor(zforce, L);
    }

    const float fcontrib = (subtid == 0) * xforce + (subtid == 1) * yforce + (subtid == 2) * zforce;

    if (subtid < 3)
	info.axayaz[subtid + 3 * dpid] = fcontrib;
}

template<int COLS, int ROWS, int NSRCMAX>
__device__ void core_ilp(const int nsrc, const int * const scan, const int * const starts,
		     const int ndst, const int dststart)
{
    const int tid = threadIdx.x;
    const int slot = tid / COLS;
    const int subtid = tid % COLS;

    const int dpid = dststart + slot;
    const int entry = 3 * dpid;
    const float2 dtmp0 = tex1Dfetch(texParticles2, entry);
    const float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
    const float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
    const float3 xdest = make_float3(dtmp0.x, dtmp0.y, dtmp1.x);
    const float3 udest = make_float3(dtmp1.y, dtmp2.x, dtmp2.y);

    float xforce = 0, yforce = 0, zforce = 0;

    for(int s = 0; s < nsrc; s += NSRCMAX * COLS)
    {
	int spids[NSRCMAX];
#pragma unroll
	for(int i = 0; i < NSRCMAX; ++i)
	{
	    const int pid = s + i * COLS + subtid;
	    const int key9 = 9 * ((pid >= scan[9]) + (pid >= scan[18]));
	    const int key3 = 3 * ((pid >= scan[key9 + 3]) + (pid >= scan[key9 + 6]));
	    const int key = key9 + key3;

	    spids[i] = pid - scan[key] + starts[key];
	}

	bool interacting[NSRCMAX];
#pragma unroll
	for(int i = 0; i < NSRCMAX; ++i)
	{
	    const int sentry = 3 * spids[i];
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);

	    const float xdiff = xdest.x - stmp0.x;
	    const float ydiff = xdest.y - stmp0.y;
	    const float zdiff = xdest.z - stmp1.x;
	    interacting[i] = (s + i * COLS + subtid < nsrc) && (dpid != spids[i]) && (xdiff * xdiff + ydiff * ydiff + zdiff * zdiff < 1);
	}

#pragma unroll
	for(int i = 0; i < NSRCMAX; ++i)
	{
	    if (interacting[i])
	    {
		const float3 f = _dpd_interaction(dpid, xdest, udest, spids[i]);

		xforce += f.x;
		yforce += f.y;
		zforce += f.z;
	    }
	}
    }

    for(int L = COLS / 2; L > 0; L >>=1)
    {
	xforce += __shfl_xor(xforce, L);
	yforce += __shfl_xor(yforce, L);
	zforce += __shfl_xor(zforce, L);
    }

    const float fcontrib = (subtid == 0) * xforce + (subtid == 1) * yforce + (subtid == 2) * zforce;

    if (subtid < 3)
	info.axayaz[subtid + 3 * dpid] = fcontrib;
}
__global__ __launch_bounds__(32 * CPB, 16)
void _dpd_forces()
{
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);

    const int tid = threadIdx.x;
    const int wid = threadIdx.y;

    __shared__ volatile int starts[CPB][32], scan[CPB][32];

    int mycount = 0, myscan = 0;
    if (tid < 27)
    {
	const int dx = (tid) % 3;
	const int dy = ((tid / 3)) % 3;
	const int dz = ((tid / 9)) % 3;

	int xcid = blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1;
	int ycid = blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1;
	int zcid = blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1;

	const bool valid_cid =
	    xcid >= 0 && xcid < info.ncells.x &&
	    ycid >= 0 && ycid < info.ncells.y &&
	    zcid >= 0 && zcid < info.ncells.z ;

	xcid = min(info.ncells.x - 1, max(0, xcid));
	ycid = min(info.ncells.y - 1, max(0, ycid));
	zcid = min(info.ncells.z - 1, max(0, zcid));

	const int cid = max(0, xcid + info.ncells.x * (ycid + info.ncells.y * zcid));

	starts[wid][tid] = tex1Dfetch(texStart, cid);

	myscan = mycount = valid_cid * tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	myscan += (tid >= L) * __shfl_up(myscan, L) ;

    if (tid < 28)
	scan[wid][tid] = myscan - mycount;

    const int nsrc = scan[wid][27];
    const int dststart = starts[wid][1 + 3 + 9];
    const int ndst = scan[wid][1 + 3 + 9 + 1] - scan[wid][1 + 3 + 9];
    const int ndst4 = (ndst >> 2) << 2;

    for(int d = 0; d < ndst4; d += 4)
	core<8, 4, 4>(nsrc, (const int *)scan[wid], (const int *)starts[wid], 4, dststart + d);

    int d = ndst4;
    if (d + 2 <= ndst)
    {
	core<16, 2, 4>(nsrc, (const int *)scan[wid],  (const int *)starts[wid], 2, dststart + d);
	d += 2;
    }

    if (d < ndst)
	core_ilp<32, 1, 2>(nsrc, (const int *)scan[wid], (const int *)starts[wid], 1, dststart + d);
}

#else
__global__ __launch_bounds__(32 * CPB, 16)
    void _dpd_forces()
{
    const int COLS = 32;
    const int ROWS = 1;
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int tid = threadIdx.x;
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;

    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0, myscan = 0;
    if (tid < 27)
    {
	const int dx = (tid) % 3;
	const int dy = ((tid / 3)) % 3;
	const int dz = ((tid / 9)) % 3;

	int xcid = blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1;
	int ycid = blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1;
	int zcid = blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1;

	const bool valid_cid =
	    xcid >= 0 && xcid < info.ncells.x &&
	    ycid >= 0 && ycid < info.ncells.y &&
	    zcid >= 0 && zcid < info.ncells.z ;

	xcid = min(info.ncells.x - 1, max(0, xcid));
	ycid = min(info.ncells.y - 1, max(0, ycid));
	zcid = min(info.ncells.z - 1, max(0, zcid));

	const int cid = max(0, xcid + info.ncells.x * (ycid + info.ncells.y * zcid));

	starts[wid][tid] = tex1Dfetch(texStart, cid);

	myscan = mycount = valid_cid * tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	myscan += (tid >= L) * __shfl_up(myscan, L) ;

    if (tid < 28)
	scan[wid][tid] = myscan - mycount;

    const int dststart = starts[wid][1 + 3 + 9];
    const int nsrc = scan[wid][27], ndst = scan[wid][1 + 3 + 9 + 1] - scan[wid][1 + 3 + 9];

    for(int d = 0; d < ndst; d += ROWS)
    {
	const int np1 = min(ndst - d, ROWS);

	const int dpid = dststart + d + slot;
	const int entry = 3 * dpid;
	float2 dtmp0 = tex1Dfetch(texParticles2, entry);
	float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
	float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);

	float xforce = 0, yforce = 0, zforce = 0;

	for(int s = 0; s < nsrc; s += COLS)
	{
	    const int np2 = min(nsrc - s, COLS);

	    const int pid = s + subtid;
	    const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
	    const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
	    const int key = key9 + key3;

	    const int spid = pid - scan[wid][key] + starts[wid][key];
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);
	    const float2 stmp2 = tex1Dfetch(texParticles2, sentry + 2);

#ifndef NDEBUG
	    {
		const int key1 = (pid >= scan[wid][key9 + key3 + 1]) + (pid >= scan[wid][key9 + key3 + 2]);
		const int keyref = key9 + key3 + key1;
		assert(keyref >= 0 && keyref < 27);
		assert(pid >= scan[wid][keyref]);
		assert(pid < scan[wid][keyref + 1] || pid >= nsrc);

		const int spidref = pid - scan[wid][keyref] + starts[wid][keyref];
		assert(spidref == spid || pid >= nsrc);
	    }
#endif

	    {
		const float xdiff = dtmp0.x - stmp0.x;
		const float ydiff = dtmp0.y - stmp0.y;
		const float zdiff = dtmp1.x - stmp1.x;

#ifndef _NONPERIODIC_KERNEL_
		asdasda
		const float _xr = xdiff - info.domainsize.x * floorf(0.5f + xdiff * info.invdomainsize.x);
		const float _yr = ydiff - info.domainsize.y * floorf(0.5f + ydiff * info.invdomainsize.y);
		const float _zr = zdiff - info.domainsize.z * floorf(0.5f + zdiff * info.invdomainsize.z);
#else
		const float _xr = xdiff;
		const float _yr = ydiff;
		const float _zr = zdiff;
#endif
		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		const float invrij = rsqrtf(rij2);
		const float rij = rij2 * invrij;
		const float argwr = max((float)0, 1 - rij * info.invrc);
		const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);
		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv =
		    xr * (dtmp1.y - stmp1.y) +
		    yr * (dtmp2.x - stmp2.x) +
		    zr * (dtmp2.y - stmp2.y);

		const float myrandnr = Logistic::mean0var1(info.seed, min(spid, dpid), max(spid, dpid));

		const float strength = info.aij * argwr - (info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;
		const bool valid = (dpid != spid) && (slot < np1) && (subtid < np2);

		if (valid)
		{
#ifdef _CHECK_
		    xforce += (rij2 < 1);
		    yforce += wr;
		    zforce += 0;
#else
		    xforce += strength * xr;
		    yforce += strength * yr;
		    zforce += strength * zr;
#endif
		}
	    }
	}

	for(int L = COLS / 2; L > 0; L >>=1)
	{
	    xforce += __shfl_xor(xforce, L);
	    yforce += __shfl_xor(yforce, L);
	    zforce += __shfl_xor(zforce, L);
	}

	const int c = (subtid % 3);
	const float fcontrib = (c == 0) * xforce + (c == 1) * yforce + (c == 2) * zforce;//f[subtid % 3];
	const int dstpid = dststart + d + slot;

	if (slot < np1)
	    info.axayaz[c + 3 * dstpid] = fcontrib;
    }
}
#endif


#ifdef _INSPECT_
__global__ __launch_bounds__(32 * CPB, 8)
    void inspect_dpd_forces(const int COLS, const int ROWS, const int nparticles, int2 * const entries, const int nentries)
{
    assert(nentries = COLS * nparticles);
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int tid = threadIdx.x;
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;

    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0, myscan = 0;
    if (tid < 27)
    {
	const int dx = (tid) % 3;
	const int dy = ((tid / 3)) % 3;
	const int dz = ((tid / 9)) % 3;

	int xcid = blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1;
	int ycid = blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1;
	int zcid = blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1;

	const bool valid_cid =
	    xcid >= 0 && xcid < info.ncells.x &&
	    ycid >= 0 && ycid < info.ncells.y &&
	    zcid >= 0 && zcid < info.ncells.z ;

	xcid = min(info.ncells.x - 1, max(0, xcid));
	ycid = min(info.ncells.y - 1, max(0, ycid));
	zcid = min(info.ncells.z - 1, max(0, zcid));

	const int cid = max(0, xcid + info.ncells.x * (ycid + info.ncells.y * zcid));

	starts[wid][tid] = tex1Dfetch(texStart, cid);

	myscan = mycount = valid_cid * tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	myscan += (tid >= L) * __shfl_up(myscan, L) ;

    if (tid < 28)
	scan[wid][tid] = myscan - mycount;

    const int dststart = starts[wid][1 + 3 + 9];
    const int nsrc = scan[wid][27], ndst = scan[wid][1 + 3 + 9 + 1] - scan[wid][1 + 3 + 9];

    for(int d = 0; d < ndst; d += ROWS)
    {
	//int srccount = 0;

	const int np1 = min(ndst - d, ROWS);

	const int dpid = dststart + d + slot;
	const int entry = 3 * dpid;

	const float2 dtmp0 = tex1Dfetch(texParticles2, entry);
	const float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
	const float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
	const float3 xdest = make_float3(dtmp0.x, dtmp0.y, dtmp1.x);
	const float3 udest = make_float3(dtmp1.y, dtmp2.x, dtmp2.y);

	int ninteractions = 0, npotentialinteractions = 0;

	for(int s = 0; s < nsrc; s += COLS)
	{
	    const int np2 = min(nsrc - s, COLS);

	    const int pid = s + subtid;
	    const int key9 = 9 * ((pid >= scan[wid][9]) + (pid >= scan[wid][18]));
	    const int key3 = 3 * ((pid >= scan[wid][key9 + 3]) + (pid >= scan[wid][key9 + 6]));
	    const int key = key9 + key3;

	    const int spid = pid - scan[wid][key] + starts[wid][key];
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);

	    const float xdiff = xdest.x - stmp0.x;
	    const float ydiff = xdest.y - stmp0.y;
	    const float zdiff = xdest.z - stmp1.x;
	    const bool interacting = (dpid != spid) && (slot < np1) && (subtid < np2) &&
		(xdiff * xdiff + ydiff * ydiff + zdiff * zdiff < 1);

	    ninteractions += (int)(interacting);
	    npotentialinteractions += 1;
	}

	if (slot < np1)
	    entries[subtid + COLS * dpid] = make_int2(ninteractions, npotentialinteractions);
    }
}
#endif

bool fdpd_init = false;

#include "../hacks.h"
#ifdef _TIME_PROFILE_
static hipEvent_t evstart, evstop;
#endif

void forces_dpd_cuda_nohost(const float * const xyzuvw, float * const axayaz,  const int np,
			    const int * const cellsstart, const int * const cellscount,
			    const float rc,
			    const float XL, const float YL, const float ZL,
			    const float aij,
			    const float gamma,
			    const float sigma,
			    const float invsqrtdt,
			    const float seed, hipStream_t stream)
{
    if (np == 0)
    {
	printf("WARNING: forces_dpd_cuda_nohost called with np = %d\n", np);
	return;
    }

    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;

    if (!fdpd_init)
    {
	texStart.channelDesc = hipCreateChannelDesc<int>();
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;

	texCount.channelDesc = hipCreateChannelDesc<int>();
	texCount.filterMode = hipFilterModePoint;
	texCount.mipmapFilterMode = hipFilterModePoint;
	texCount.normalized = 0;

	texParticles2.channelDesc = hipCreateChannelDesc<float2>();
	texParticles2.filterMode = hipFilterModePoint;
	texParticles2.mipmapFilterMode = hipFilterModePoint;
	texParticles2.normalized = 0;

	void (*dpdkernel)() =  _dpd_forces_new2<32, 1>; //_dpd_forces;

	CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*dpdkernel), hipFuncCachePreferL1));

#ifdef _TIME_PROFILE_
	CUDA_CHECK(hipEventCreate(&evstart));
	CUDA_CHECK(hipEventCreate(&evstop));
#endif
	fdpd_init = true;
    }

    size_t textureoffset;
    CUDA_CHECK(hipBindTexture(&textureoffset, &texParticles2, xyzuvw, &texParticles2.channelDesc, sizeof(float) * 6 * np));
    assert(textureoffset == 0);
    CUDA_CHECK(hipBindTexture(&textureoffset, &texStart, cellsstart, &texStart.channelDesc, sizeof(int) * ncells));
    assert(textureoffset == 0);
    CUDA_CHECK(hipBindTexture(&textureoffset, &texCount, cellscount, &texCount.channelDesc, sizeof(int) * ncells));
    assert(textureoffset == 0);

    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.domainsize = make_float3(XL, YL, ZL);
    c.invdomainsize = make_float3(1 / XL, 1 / YL, 1 / ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
    c.axayaz = axayaz;
    c.seed = seed;

    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &c, sizeof(c), 0, hipMemcpyHostToDevice, stream));

    static int cetriolo = 0;
    cetriolo++;

#ifdef _INSPECT_
    {
	//inspect irregularity of the computation,
	//report data to file
	if (cetriolo % 1000 == 0)
	{
	    enum { COLS = 16, ROWS = 2 };

	    const size_t nentries = np * COLS;

	    int2 * data;
	    CUDA_CHECK(hipHostAlloc(&data, sizeof(int2) * nentries, hipHostMallocMapped));
	    memset(data, 0xff, sizeof(int2) * nentries);

	    int * devptr;
	    CUDA_CHECK(hipHostGetDevicePointer(&devptr, data, 0));

	    inspect_dpd_forces<<<dim3(c.ncells.x / _XCPB_, c.ncells.y / _YCPB_, c.ncells.z / _ZCPB_), dim3(32, CPB), 0, stream>>>
		(COLS, ROWS, np, data, nentries);

	    CUDA_CHECK(hipDeviceSynchronize());

	    char path2report[2000];
	    sprintf(path2report, "inspection-%d-tstep.txt", cetriolo);

	    FILE * f = fopen(path2report, "w");
	    assert(f);

	    for(int i = 0, c = 0; i < np; ++i)
	    {
		fprintf(f, "pid %05d: ", i);

		int s = 0, pot = 0;
		for(int j = 0; j < COLS; ++j, ++c)
		{
		    fprintf(f, "%02d ", data[c].x);
		    s += data[c].x;
		    pot += data[c].y;
		}

		fprintf(f, " sum: %02d pot: %d\n", s, (pot + COLS - 1) / (COLS));
	    }

	    fclose(f);

	    CUDA_CHECK(hipHostFree(data));
	    printf("inspection saved to %s.\n", path2report);
	}
    }
#endif

#ifdef _TIME_PROFILE_
    if (cetriolo % 500 == 0)
	CUDA_CHECK(hipEventRecord(evstart));
#endif
	_dpd_forces_new2<32, 1>/*, 3>*/<<<(c.ncells.x*c.ncells.y*c.ncells.z+CPB-1)/CPB, dim3(32, CPB), 0, stream>>>();
/*_dpd_forces<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB), 0, stream>>>();
*/
#ifdef _TIME_PROFILE_
    if (cetriolo % 500 == 0)
    {
	CUDA_CHECK(hipEventRecord(evstop));
	CUDA_CHECK(hipEventSynchronize(evstop));

	float tms;
	CUDA_CHECK(hipEventElapsedTime(&tms, evstart, evstop));
	printf("elapsed time for DPD-BULK kernel: %.2f ms\n", tms);
    }
#endif

    CUDA_CHECK(hipPeekAtLastError());
}

#include <cmath>
#include <unistd.h>

#include "../cell-lists.h"

int fdpd_oldnp = 0, fdpd_oldnc = 0;

float * fdpd_xyzuvw = NULL, * fdpd_axayaz = NULL;
int * fdpd_start = NULL, * fdpd_count = NULL;

void forces_dpd_cuda_aos(float * const _xyzuvw, float * const _axayaz,
		     int * const order, const int np,
		     const float rc,
		     const float XL, const float YL, const float ZL,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
			 const float seed,
			 const bool nohost)
{
    if (np == 0)
    {
	printf("WARNING: forces_dpd_cuda_aos called with np = %d\n", np);
	return;
    }

    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;

    if (!fdpd_init)
    {
	texStart.channelDesc = hipCreateChannelDesc<int>();
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;

	texCount.channelDesc = hipCreateChannelDesc<int>();
	texCount.filterMode = hipFilterModePoint;
	texCount.mipmapFilterMode = hipFilterModePoint;
	texCount.normalized = 0;

	texParticles2.channelDesc = hipCreateChannelDesc<float2>();
	texParticles2.filterMode = hipFilterModePoint;
	texParticles2.mipmapFilterMode = hipFilterModePoint;
	texParticles2.normalized = 0;

	fdpd_init = true;
    }

    if (fdpd_oldnp < np)
    {
	if (fdpd_oldnp > 0)
	{
	    CUDA_CHECK(hipFree(fdpd_xyzuvw));
	    CUDA_CHECK(hipFree(fdpd_axayaz));
	}

	CUDA_CHECK(hipMalloc(&fdpd_xyzuvw, sizeof(float) * 6 * np));
	CUDA_CHECK(hipMalloc(&fdpd_axayaz, sizeof(float) * 3 * np));

	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texParticles2, fdpd_xyzuvw, &texParticles2.channelDesc, sizeof(float) * 6 * np));

	fdpd_oldnp = np;
    }

    if (fdpd_oldnc < ncells)
    {
	if (fdpd_oldnc > 0)
	{
	    CUDA_CHECK(hipFree(fdpd_start));
	    CUDA_CHECK(hipFree(fdpd_count));
	}

	CUDA_CHECK(hipMalloc(&fdpd_start, sizeof(int) * ncells));
	CUDA_CHECK(hipMalloc(&fdpd_count, sizeof(int) * ncells));

	size_t textureoffset = 0;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texStart, fdpd_start, &texStart.channelDesc, sizeof(int) * ncells));
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCount, fdpd_count, &texCount.channelDesc, sizeof(int) * ncells));

	fdpd_oldnc = ncells;
    }

    CUDA_CHECK(hipMemcpyAsync(fdpd_xyzuvw, _xyzuvw, sizeof(float) * np * 6, nohost ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice, 0));

    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.domainsize = make_float3(XL, YL, ZL);
    c.invdomainsize = make_float3(1 / XL, 1 / YL, 1 / ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
    c.axayaz = fdpd_axayaz;
    c.seed = seed;

    build_clists(fdpd_xyzuvw, np, rc, c.ncells.x, c.ncells.y, c.ncells.z,
		 c.domainstart.x, c.domainstart.y, c.domainstart.z,
		 order, fdpd_start, fdpd_count, NULL);

    //TextureWrap texStart(_ptr(starts), ncells), texCount(_ptr(counts), ncells);
    //TextureWrap texParticles((float2*)_ptr(xyzuvw), 3 * np);

    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &c, sizeof(c), 0));

    _dpd_forces<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB)>>>();

    CUDA_CHECK(hipPeekAtLastError());

//copy xyzuvw as well?!?
    if (nohost)
    {
	CUDA_CHECK(hipMemcpy(_xyzuvw, fdpd_xyzuvw, sizeof(float) * 6 * np, hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(_axayaz, fdpd_axayaz, sizeof(float) * 3 * np, hipMemcpyDeviceToDevice));
    }
    else
	CUDA_CHECK(hipMemcpy(_axayaz, fdpd_axayaz, sizeof(float) * 3 * np, hipMemcpyDeviceToHost));

#ifdef _CHECK_
    CUDA_CHECK(hipDeviceSynchronize());

    for(int ii = 0; ii < np; ++ii)
    {
	printf("pid %d -> %f %f %f\n", ii, (float)axayaz[0 + 3 * ii], (float)axayaz[1 + 3* ii], (float)axayaz[2 + 3 *ii]);

	int cnt = 0;
	float fc = 0;
	const int i = order[ii];
	printf("devi coords are %f %f %f\n", (float)xyzuvw[0 + 6 * ii], (float)xyzuvw[1 + 6 * ii], (float)xyzuvw[2 + 6 * ii]);
	printf("host coords are %f %f %f\n", (float)_xyzuvw[0 + 6 * i], (float)_xyzuvw[1 + 6 * i], (float)_xyzuvw[2 + 6 * i]);

	for(int j = 0; j < np; ++j)
	{
	    if (i == j)
		continue;

	    float xr = _xyzuvw[0 + 6 *i] - _xyzuvw[0 + 6 * j];
	    float yr = _xyzuvw[1 + 6 *i] - _xyzuvw[1 + 6 * j];
	    float zr = _xyzuvw[2 + 6 *i] - _xyzuvw[2 + 6 * j];

	    xr -= c.domainsize.x *  ::floor(0.5f + xr / c.domainsize.x);
	    yr -= c.domainsize.y *  ::floor(0.5f + yr / c.domainsize.y);
	    zr -= c.domainsize.z *  ::floor(0.5f + zr / c.domainsize.z);

	    const float rij2 = xr * xr + yr * yr + zr * zr;
	    const float invrij = rsqrtf(rij2);
	    const float rij = rij2 * invrij;
	    const float wr = max((float)0, 1 - rij * c.invrc);

	    const bool collision =  rij2 < 1;

	    if (collision)
		fc += wr;//	printf("ref p %d colliding with %d\n", i, j);

	    cnt += collision;
	}
	printf("i found %d host interactions and with cuda i found %d\n", cnt, (int)axayaz[0 + 3 * ii]);
	assert(cnt == (float)axayaz[0 + 3 * ii]);
	printf("fc aij ref %f vs cuda %e\n", fc,  (float)axayaz[1 + 3 * ii]);
	assert(fabs(fc - (float)axayaz[1 + 3 * ii]) < 1e-4);
    }

    printf("test done.\n");
    sleep(1);
    exit(0);
#endif
}


int * fdpd_order = NULL;
float * fdpd_pv = NULL, *fdpd_a = NULL;

void forces_dpd_cuda(const float * const xp, const float * const yp, const float * const zp,
		     const float * const xv, const float * const yv, const float * const zv,
		     float * const xa, float * const ya, float * const za,
		     const int np,
		     const float rc,
		     const float LX, const float LY, const float LZ,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
		     const float seed)
{
    if (np <= 0) return;

    if (np > fdpd_oldnp)
    {
	if (fdpd_oldnp > 0)
	{
	    CUDA_CHECK(hipHostFree(fdpd_pv));
	    CUDA_CHECK(hipHostFree(fdpd_order));
	    CUDA_CHECK(hipHostFree(fdpd_a));
	}

	CUDA_CHECK(hipHostAlloc(&fdpd_pv, sizeof(float) * np * 6, hipHostMallocDefault));
	CUDA_CHECK(hipHostAlloc(&fdpd_order, sizeof(int) * np, hipHostMallocDefault));
	CUDA_CHECK(hipHostAlloc(&fdpd_a, sizeof(float) * np * 3, hipHostMallocDefault));

	//this will be done by forces_dpd_cuda
	//fdpd_oldnp = np;
    }

    for(int i = 0; i < np; ++i)
    {
	fdpd_pv[0 + 6 * i] = xp[i];
	fdpd_pv[1 + 6 * i] = yp[i];
	fdpd_pv[2 + 6 * i] = zp[i];
	fdpd_pv[3 + 6 * i] = xv[i];
	fdpd_pv[4 + 6 * i] = yv[i];
	fdpd_pv[5 + 6 * i] = zv[i];
    }

    forces_dpd_cuda_aos(fdpd_pv, fdpd_a, fdpd_order, np, rc, LX, LY, LZ,
			aij, gamma, sigma, invsqrtdt, seed, false);

    //delete [] pv;

    for(int i = 0; i < np; ++i)
    {
	xa[fdpd_order[i]] += fdpd_a[0 + 3 * i];
	ya[fdpd_order[i]] += fdpd_a[1 + 3 * i];
	za[fdpd_order[i]] += fdpd_a[2 + 3 * i];
    }

    //delete [] a;

    //delete [] order;
}
