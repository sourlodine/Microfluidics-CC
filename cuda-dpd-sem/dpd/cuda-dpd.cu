#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

#include "../saru.cuh"

#ifndef NDEBUG
//#define _CHECK_
#endif

struct InfoDPD
{
    int3 ncells;
    float3 domainsize, invdomainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
};

__constant__ InfoDPD info;

texture<float2, hipTextureType1D> texParticles2;
texture<int, hipTextureType1D> texStart, texCount;
 
#define COLS 32
#define ROWS (32 / COLS)
#define _XCPB_ 2
#define _YCPB_ 2
#define _ZCPB_ 1
#define CPB (_XCPB_ * _YCPB_ * _ZCPB_)

__global__ __launch_bounds__(32 * CPB, 16) 
    void _dpd_forces_saru(float * const axayaz,
			  const int idtimestep)
{
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int tid = threadIdx.x; 
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;
     
    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0; 
    if (tid < 27)
    {
	const int dx = (1 + tid) % 3;
	const int dy = (1 + (tid / 3)) % 3; 
	const int dz = (1 + (tid / 9)) % 3;

	const int xcid = (blockIdx.x * _XCPB_ + ((threadIdx.y) % _XCPB_) + dx - 1 + info.ncells.x) % info.ncells.x;
	const int ycid = (blockIdx.y * _YCPB_ + ((threadIdx.y / _XCPB_) % _YCPB_) + dy - 1 + info.ncells.y) % info.ncells.y;
	const int zcid = (blockIdx.z * _ZCPB_ + ((threadIdx.y / (_XCPB_ * _YCPB_)) % _ZCPB_) + dz - 1 + info.ncells.z) % info.ncells.z;
	const int cid = xcid + info.ncells.x * (ycid + info.ncells.y * zcid);

	starts[wid][tid] = tex1Dfetch(texStart, cid);
	mycount = tex1Dfetch(texCount, cid);
    }

    for(int L = 1; L < 32; L <<= 1)
	mycount += (tid >= L) * __shfl_up(mycount, L) ;

    if (tid < 27)
	scan[wid][tid] = mycount;

    const int dststart = starts[wid][0];
    const int nsrc = scan[wid][26], ndst = scan[wid][0];
 
    for(int d = 0; d < ndst; d += ROWS)
    {
	const int np1 = min(ndst - d, ROWS);

	const int dpid = dststart + d + slot;
	const int entry = 3 * dpid;
	float2 dtmp0 = tex1Dfetch(texParticles2, entry);
	float2 dtmp1 = tex1Dfetch(texParticles2, entry + 1);
	float2 dtmp2 = tex1Dfetch(texParticles2, entry + 2);
	
	float xforce = 0, yforce = 0, zforce = 0;

	for(int s = 0; s < nsrc; s += COLS)
	{
	    const int np2 = min(nsrc - s, COLS);
  
	    const int pid = s + subtid;
	    const int key9 = 9 * (pid >= scan[wid][8]) + 9 * (pid >= scan[wid][17]);
	    const int key3 = 3 * (pid >= scan[wid][key9 + 2]) + 3 * (pid >= scan[wid][key9 + 5]);
	    const int key1 = (pid >= scan[wid][key9 + key3]) + (pid >= scan[wid][key9 + key3 + 1]);
	    const int key = key9 + key3 + key1;
	    assert(subtid >= np2 || pid >= (key ? scan[wid][key - 1] : 0) && pid < scan[wid][key]);

	    const int spid = starts[wid][key] + pid - (key ? scan[wid][key - 1] : 0);
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch(texParticles2, sentry);
	    const float2 stmp1 = tex1Dfetch(texParticles2, sentry + 1);
	    const float2 stmp2 = tex1Dfetch(texParticles2, sentry + 2);
	    
	    {
		const float xdiff = dtmp0.x - stmp0.x;
		const float ydiff = dtmp0.y - stmp0.y;
		const float zdiff = dtmp1.x - stmp1.x;

#ifndef _NONPERIODIC_KERNEL_
		asdasda
		const float _xr = xdiff - info.domainsize.x * floorf(0.5f + xdiff * info.invdomainsize.x);
		const float _yr = ydiff - info.domainsize.y * floorf(0.5f + ydiff * info.invdomainsize.y);
		const float _zr = zdiff - info.domainsize.z * floorf(0.5f + zdiff * info.invdomainsize.z);
#else
		const float _xr = xdiff;
		const float _yr = ydiff;
		const float _zr = zdiff;
#endif
		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		const float invrij = rsqrtf(rij2);
		const float rij = rij2 * invrij;
		const float wr = max((float)0, 1 - rij * info.invrc);
		
		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;
		
		const float rdotv = 
		    xr * (dtmp1.y - stmp1.y) +
		    yr * (dtmp2.x - stmp2.x) +
		    zr * (dtmp2.y - stmp2.y);
		  
		const float mysaru = saru(min(spid, dpid), max(spid, dpid), idtimestep);
		const float myrandnr = 3.464101615f * mysaru - 1.732050807f;
		 
		const float strength = (info.aij - info.gamma * wr * rdotv + info.sigmaf * myrandnr) * wr;
		const bool valid = (d + slot != s + subtid) && (slot < np1) && (subtid < np2);
		
		if (valid)
		{
#ifdef _CHECK_
		    xforce += (rij2 < 1);
		    yforce += wr;
		    zforce += 0;
#else		    	     
		    xforce += strength * xr;
		    yforce += strength * yr;
		    zforce += strength * zr;
#endif
		}
	    } 
	}
	
	for(int L = COLS / 2; L > 0; L >>=1)
	{
	    xforce += __shfl_xor(xforce, L);
	    yforce += __shfl_xor(yforce, L);
	    zforce += __shfl_xor(zforce, L);
	}

	const int c = (subtid % 3);       
	const float fcontrib = (c == 0) * xforce + (c == 1) * yforce + (c == 2) * zforce;//f[subtid % 3];
	const int dstpid = dststart + d + slot;


	if (slot < np1)
	    axayaz[c + 3 * dstpid] = fcontrib;
    }
}

bool fdpd_init = false;

#include "../hacks.h"

void forces_dpd_cuda_nohost(const float * const xyzuvw, float * const axayaz,  const int np,
			    const int * const cellsstart, const int * const cellscount, 
			    const float rc,
			    const float XL, const float YL, const float ZL,
			    const float aij,
			    const float gamma,
			    const float sigma,
			    const float invsqrtdt,
			    const int saru_tag, hipStream_t stream)
{
    if (np == 0)
    {
	printf("WARNING: forces_dpd_cuda_nohost called with np = %d\n", np);
	return;
    }

    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;

    if (!fdpd_init)
    {
	texStart.channelDesc = hipCreateChannelDesc<int>();
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;
    
	texCount.channelDesc = hipCreateChannelDesc<int>();
	texCount.filterMode = hipFilterModePoint;
	texCount.mipmapFilterMode = hipFilterModePoint;
	texCount.normalized = 0;

	texParticles2.channelDesc = hipCreateChannelDesc<float2>();
	texParticles2.filterMode = hipFilterModePoint;
	texParticles2.mipmapFilterMode = hipFilterModePoint;
	texParticles2.normalized = 0;

	fdpd_init = true;
    }

    size_t textureoffset;
    CUDA_CHECK(hipBindTexture(&textureoffset, &texParticles2, xyzuvw, &texParticles2.channelDesc, sizeof(float) * 6 * np));
    assert(textureoffset == 0);
    CUDA_CHECK(hipBindTexture(&textureoffset, &texStart, cellsstart, &texStart.channelDesc, sizeof(int) * ncells));
    assert(textureoffset == 0);
    CUDA_CHECK(hipBindTexture(&textureoffset, &texCount, cellscount, &texCount.channelDesc, sizeof(int) * ncells));
    assert(textureoffset == 0);
      
    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.domainsize = make_float3(XL, YL, ZL);
    c.invdomainsize = make_float3(1 / XL, 1 / YL, 1 / ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
      
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &c, sizeof(c)));
   
    _dpd_forces_saru<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB), 0, stream>>>(axayaz, saru_tag);

    CUDA_CHECK(hipPeekAtLastError());	
}

#include <cmath>
#include <unistd.h>

//#include <thrust/device_vector.h>
//using namespace thrust;

#include "../profiler-dpd.h"
#include "../cell-lists.h"




int fdpd_oldnp = 0, fdpd_oldnc = 0;

float * fdpd_xyzuvw = NULL, * fdpd_axayaz = NULL;
int * fdpd_start = NULL, * fdpd_count = NULL;

void forces_dpd_cuda_aos(float * const _xyzuvw, float * const _axayaz,
		     int * const order, const int np,
		     const float rc,
		     const float XL, const float YL, const float ZL,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
			 const int saru_tag,
			 const bool nohost)
{
    if (np == 0)
    {
	printf("WARNING: forces_dpd_cuda_aos called with np = %d\n", np);
	return;
    }

    int nx = (int)ceil(XL / rc);
    int ny = (int)ceil(YL / rc);
    int nz = (int)ceil(ZL / rc);
    const int ncells = nx * ny * nz;

    if (!fdpd_init)
    {
	texStart.channelDesc = hipCreateChannelDesc<int>();
	texStart.filterMode = hipFilterModePoint;
	texStart.mipmapFilterMode = hipFilterModePoint;
	texStart.normalized = 0;
    
	texCount.channelDesc = hipCreateChannelDesc<int>();
	texCount.filterMode = hipFilterModePoint;
	texCount.mipmapFilterMode = hipFilterModePoint;
	texCount.normalized = 0;

	texParticles2.channelDesc = hipCreateChannelDesc<float2>();
	texParticles2.filterMode = hipFilterModePoint;
	texParticles2.mipmapFilterMode = hipFilterModePoint;
	texParticles2.normalized = 0;

	fdpd_init = true;
    }
    
    if (fdpd_oldnp < np)
    {
	if (fdpd_oldnp > 0)
	{
	    CUDA_CHECK(hipFree(fdpd_xyzuvw));
	    CUDA_CHECK(hipFree(fdpd_axayaz));
	}

	CUDA_CHECK(hipMalloc(&fdpd_xyzuvw, sizeof(float) * 6 * np));
	CUDA_CHECK(hipMalloc(&fdpd_axayaz, sizeof(float) * 3 * np));

	size_t textureoffset;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texParticles2, fdpd_xyzuvw, &texParticles2.channelDesc, sizeof(float) * 6 * np));
	
	fdpd_oldnp = np;
    }

    if (fdpd_oldnc < ncells)
    {
	if (fdpd_oldnc > 0)
	{
	    CUDA_CHECK(hipFree(fdpd_start));
	    CUDA_CHECK(hipFree(fdpd_count));
	}

	CUDA_CHECK(hipMalloc(&fdpd_start, sizeof(int) * ncells));
	CUDA_CHECK(hipMalloc(&fdpd_count, sizeof(int) * ncells));

	size_t textureoffset = 0;
	CUDA_CHECK(hipBindTexture(&textureoffset, &texStart, fdpd_start, &texStart.channelDesc, sizeof(int) * ncells));
	CUDA_CHECK(hipBindTexture(&textureoffset, &texCount, fdpd_count, &texCount.channelDesc, sizeof(int) * ncells));
	
	fdpd_oldnc = ncells;
    }

    CUDA_CHECK(hipMemcpyAsync(fdpd_xyzuvw, _xyzuvw, sizeof(float) * np * 6, nohost ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice));
    
    InfoDPD c;
    c.ncells = make_int3(nx, ny, nz);
    c.domainsize = make_float3(XL, YL, ZL);
    c.invdomainsize = make_float3(1 / XL, 1 / YL, 1 / ZL);
    c.domainstart = make_float3(-XL * 0.5, -YL * 0.5, -ZL * 0.5);
    c.invrc = 1.f / rc;
    c.aij = aij;
    c.gamma = gamma;
    c.sigmaf = sigma * invsqrtdt;
        
    build_clists(fdpd_xyzuvw, np, rc, c.ncells.x, c.ncells.y, c.ncells.z,
		 c.domainstart.x, c.domainstart.y, c.domainstart.z,
		 order, fdpd_start, fdpd_count, NULL);

    //TextureWrap texStart(_ptr(starts), ncells), texCount(_ptr(counts), ncells);
    //TextureWrap texParticles((float2*)_ptr(xyzuvw), 3 * np);
    
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &c, sizeof(c)));
   
    ProfilerDPD::singletone().start();

    if (saru_tag >= 0)
	saru_tid = saru_tag;
    
    _dpd_forces_saru<<<dim3(c.ncells.x / _XCPB_,
			    c.ncells.y / _YCPB_,
			    c.ncells.z / _ZCPB_), dim3(32, CPB)>>>(fdpd_axayaz, saru_tid);
 
    ++saru_tid;

    CUDA_CHECK(hipPeekAtLastError());
	
    ProfilerDPD::singletone().force();
    
//copy xyzuvw as well?!?
    if (nohost)
    {
	CUDA_CHECK(hipMemcpy(_xyzuvw, fdpd_xyzuvw, sizeof(float) * 6 * np, hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(_axayaz, fdpd_axayaz, sizeof(float) * 3 * np, hipMemcpyDeviceToDevice));
    }
    else
	CUDA_CHECK(hipMemcpy(_axayaz, fdpd_axayaz, sizeof(float) * 3 * np, hipMemcpyDeviceToHost));

    ProfilerDPD::singletone().report();

    //copy(axayaz.begin(), axayaz.end(), _axayaz);
     
#ifdef _CHECK_
    CUDA_CHECK(hipDeviceSynchronize());
    
    for(int ii = 0; ii < np; ++ii)
    { 
	printf("pid %d -> %f %f %f\n", ii, (float)axayaz[0 + 3 * ii], (float)axayaz[1 + 3* ii], (float)axayaz[2 + 3 *ii]);

	int cnt = 0;
	float fc = 0;
	const int i = order[ii];
	printf("devi coords are %f %f %f\n", (float)xyzuvw[0 + 6 * ii], (float)xyzuvw[1 + 6 * ii], (float)xyzuvw[2 + 6 * ii]);
	printf("host coords are %f %f %f\n", (float)_xyzuvw[0 + 6 * i], (float)_xyzuvw[1 + 6 * i], (float)_xyzuvw[2 + 6 * i]);
	
	for(int j = 0; j < np; ++j)
	{
	    if (i == j) 
		continue;
 
	    float xr = _xyzuvw[0 + 6 *i] - _xyzuvw[0 + 6 * j];
	    float yr = _xyzuvw[1 + 6 *i] - _xyzuvw[1 + 6 * j];
	    float zr = _xyzuvw[2 + 6 *i] - _xyzuvw[2 + 6 * j];

	    xr -= c.domainsize.x *  ::floor(0.5f + xr / c.domainsize.x);
	    yr -= c.domainsize.y *  ::floor(0.5f + yr / c.domainsize.y);
	    zr -= c.domainsize.z *  ::floor(0.5f + zr / c.domainsize.z);

	    const float rij2 = xr * xr + yr * yr + zr * zr;
	    const float invrij = rsqrtf(rij2);
	    const float rij = rij2 * invrij;
	    const float wr = max((float)0, 1 - rij * c.invrc);
	
	    const bool collision =  rij2 < 1;

	    if (collision)
		fc += wr;//	printf("ref p %d colliding with %d\n", i, j);
	    
	    cnt += collision;
	}
	printf("i found %d host interactions and with cuda i found %d\n", cnt, (int)axayaz[0 + 3 * ii]);
	assert(cnt == (float)axayaz[0 + 3 * ii]);
	printf("fc aij ref %f vs cuda %e\n", fc,  (float)axayaz[1 + 3 * ii]);
	assert(fabs(fc - (float)axayaz[1 + 3 * ii]) < 1e-4);
    }
    
    printf("test done.\n");
    sleep(1);
    exit(0);
#endif
}

int * fdpd_order = NULL;
float * fdpd_pv = NULL, *fdpd_a = NULL;

void forces_dpd_cuda(const float * const xp, const float * const yp, const float * const zp,
		     const float * const xv, const float * const yv, const float * const zv,
		     float * const xa, float * const ya, float * const za,
		     const int np,
		     const float rc,
		     const float LX, const float LY, const float LZ,
		     const float aij,
		     const float gamma,
		     const float sigma,
		     const float invsqrtdt,
		     const int input_saru_tag)
{
    if (np <= 0) return;

    if (np > fdpd_oldnp)
    {
	if (fdpd_oldnp > 0)
	{
	    CUDA_CHECK(hipHostFree(fdpd_pv));
	    CUDA_CHECK(hipHostFree(fdpd_order));
	    CUDA_CHECK(hipHostFree(fdpd_a));
	}

	CUDA_CHECK(hipHostAlloc(&fdpd_pv, sizeof(float) * np * 6, hipHostMallocDefault));
	CUDA_CHECK(hipHostAlloc(&fdpd_order, sizeof(int) * np, hipHostMallocDefault));
	CUDA_CHECK(hipHostAlloc(&fdpd_a, sizeof(float) * np * 3, hipHostMallocDefault));

	//this will be done by forces_dpd_cuda
	//fdpd_oldnp = np;
    }
    
    for(int i = 0; i < np; ++i)
    {
	fdpd_pv[0 + 6 * i] = xp[i];
	fdpd_pv[1 + 6 * i] = yp[i];
	fdpd_pv[2 + 6 * i] = zp[i];
	fdpd_pv[3 + 6 * i] = xv[i];
	fdpd_pv[4 + 6 * i] = yv[i];
	fdpd_pv[5 + 6 * i] = zv[i];
    }

    forces_dpd_cuda_aos(fdpd_pv, fdpd_a, fdpd_order, np, rc, LX, LY, LZ,
			aij, gamma, sigma, invsqrtdt, input_saru_tag, false);
    
    //delete [] pv;
     
    for(int i = 0; i < np; ++i)
    {
	xa[fdpd_order[i]] += fdpd_a[0 + 3 * i];
	ya[fdpd_order[i]] += fdpd_a[1 + 3 * i];
	za[fdpd_order[i]] += fdpd_a[2 + 3 * i];
    }

    //delete [] a;

    //delete [] order;
}