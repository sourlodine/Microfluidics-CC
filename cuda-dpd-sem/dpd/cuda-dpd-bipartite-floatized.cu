#include "hip/hip_runtime.h"
/*
 *  cuda-dpd-bipartite.cu
 *  Part of CTC/cuda-dpd-sem/dpd/
 *
 *  Created and authored by Yu-Hang Tang on 2015-03-18.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cassert>

#include "../dpd-rng.h"
#include "cuda-dpd.h"

struct BipartiteInfoDPD {
    int3 ncells;
    float3 domainsize, invdomainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
};

__constant__ BipartiteInfoDPD bipart_info;

#ifndef NDEBUG
//#define _CHECK_
#endif

const static uint COLS = 8;
const static uint ROWS = (32 / COLS);
const static uint CPB = 4;

#include "../hacks.h"

__global__
void _bipartite_dpd_directforces( float * const axayaz, const int np, const int np_src,
                                  const float seed, const int mask, const float * xyzuvw, const float * xyzuvw_src,
                                  const float invrc, const float aij, const float gamma, const float sigmaf )
{
    assert( blockDim.x % warpSize == 0 );
    assert( blockDim.x * gridDim.x >= np );

    const int tid = threadIdx.x % warpSize;
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;
    const bool valid = pid < np;

    float xp, yp, zp, up, vp, wp;

    if( valid ) {
        xp = xyzuvw[0 + pid * 6];
        yp = xyzuvw[1 + pid * 6];
        zp = xyzuvw[2 + pid * 6];
        up = xyzuvw[3 + pid * 6];
        vp = xyzuvw[4 + pid * 6];
        wp = xyzuvw[5 + pid * 6];
    }

    float xforce = 0, yforce = 0, zforce = 0;

    for( int s = 0; s < np_src; s += warpSize ) {
        float my_xq, my_yq, my_zq, my_uq, my_vq, my_wq;

        const int batchsize = min( warpSize, np_src - s );

        if( tid < batchsize ) {
            my_xq = xyzuvw_src[0 + ( tid + s ) * 6];
            my_yq = xyzuvw_src[1 + ( tid + s ) * 6];
            my_zq = xyzuvw_src[2 + ( tid + s ) * 6];
            my_uq = xyzuvw_src[3 + ( tid + s ) * 6];
            my_vq = xyzuvw_src[4 + ( tid + s ) * 6];
            my_wq = xyzuvw_src[5 + ( tid + s ) * 6];
        }

        for( int l = 0; l < batchsize; ++l ) {
            const float xq = __shfl( my_xq, l );
            const float yq = __shfl( my_yq, l );
            const float zq = __shfl( my_zq, l );
            const float uq = __shfl( my_uq, l );
            const float vq = __shfl( my_vq, l );
            const float wq = __shfl( my_wq, l );

            //necessary to force the execution shuffles here below
            //__syncthreads();

            //if (valid)
            {
                const float _xr = xp - xq;
                const float _yr = yp - yq;
                const float _zr = zp - zq;

                const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

                const float invrij = rsqrtf( rij2 );

                const float rij = rij2 * invrij;
                const float argwr = max( ( float )0, 1 - rij * invrc );
                const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

                const float xr = _xr * invrij;
                const float yr = _yr * invrij;
                const float zr = _zr * invrij;

                const float rdotv =
                    xr * ( up - uq ) +
                    yr * ( vp - vq ) +
                    zr * ( wp - wq );

                const int spid = s + l;
                const int dpid = pid;

                const int arg1 = mask * dpid + ( 1 - mask ) * spid;
                const int arg2 = mask * spid + ( 1 - mask ) * dpid;
                const float myrandnr = Logistic::mean0var1( seed, arg1, arg2 );

                const float strength = aij * argwr + ( - gamma * wr * rdotv + sigmaf * myrandnr ) * wr;
                //if (valid && spid < np_src)
                {
                    xforce += strength * xr;
                    yforce += strength * yr;
                    zforce += strength * zr;
                }
            }
        }
    }

    if( valid ) {
        assert( !isnan( xforce ) );
        assert( !isnan( yforce ) );
        assert( !isnan( zforce ) );

        axayaz[0 + 3 * pid] = xforce;
        axayaz[1 + 3 * pid] = yforce;
        axayaz[2 + 3 * pid] = zforce;
    }
}

void directforces_dpd_cuda_bipartite_nohost(
    const float * const xyzuvw, float * const axayaz, const int np,
    const float * const xyzuvw_src, const int np_src,
    const float aij, const float gamma, const float sigma, const float invsqrtdt,
    const float seed, const int mask, hipStream_t stream )
{
    if( np == 0 || np_src == 0 ) {
        printf( "warning: directforces_dpd_cuda_bipartite_nohost called with ZERO!\n" );
        return;
    }

    _bipartite_dpd_directforces <<< ( np + 127 ) / 128, 128, 0, stream >>> ( axayaz, np, np_src, seed, mask,
            xyzuvw, xyzuvw_src, 1, aij, gamma, sigma * invsqrtdt );

    CUDA_CHECK( hipPeekAtLastError() );
}

__global__ __launch_bounds__( 32 * CPB, 16 )
void _dpd_bipforces( const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
                     hipTextureObject_t texSrcStart,  hipTextureObject_t texSrcParticles, const int np_src, const int3 halo_ncells,
                     const float aij, const float gamma, const float sigmaf,
                     const float seed, const uint mask, float * const axayaz )
{
    assert( warpSize == COLS * ROWS );
    assert( blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1 );
    assert( ROWS * 3 <= warpSize );

    const uint tid = threadIdx.x;
    const uint subtid = tid % COLS;
    const uint slot = tid / COLS;
    const uint wid = threadIdx.y;

    __shared__ uint volatile starts[CPB][32], scan[CPB][32];

    const int mycid = xmad( blockIdx.x, float(CPB), threadIdx.y );

    if( mycid >= halo_ncells.x * halo_ncells.y * halo_ncells.z ) return;

    uint mycount = 0, myscan = 0;
    if( tid < 27 ) {
        const int dx = ( tid ) % 3;
        const int dy = ( ( tid / 3 ) ) % 3;
        const int dz = ( ( tid / 9 ) ) % 3;

        int xcid = ( mycid % halo_ncells.x ) + dx - 1;
        int ycid = ( ( mycid / halo_ncells.x ) % halo_ncells.y ) + dy - 1;
        int zcid = ( ( mycid / halo_ncells.x / halo_ncells.y ) % halo_ncells.z ) + dz - 1;

        const bool valid_cid =
                ( xcid >= 0 ) && ( xcid < halo_ncells.x ) &&
                ( ycid >= 0 ) && ( ycid < halo_ncells.y ) &&
                ( zcid >= 0 ) && ( zcid < halo_ncells.z );

        xcid = xmin( xsub( halo_ncells.x, 1 ), max( 0, xcid ) );
        ycid = xmin( xsub( halo_ncells.y, 1 ), max( 0, ycid ) );
        zcid = xmin( xsub( halo_ncells.z, 1 ), max( 0, zcid ) );

        const int cid = max( 0, xcid + halo_ncells.x * ( ycid + halo_ncells.y * zcid ) );

        starts[wid][tid] = tex1Dfetch<uint>( texSrcStart, cid );

        myscan = mycount = valid_cid ? ( tex1Dfetch<uint>( texSrcStart, cid + 1 ) - tex1Dfetch<uint>( texSrcStart, cid ) ) : 0u;
    }

    for( int L = 1; L < 32; L <<= 1 )
	//int or float yuhang?
        myscan += ( tid >= L ) * __shfl_up((int) myscan, L ) ;

    if( tid < 28 )
        scan[wid][tid] = myscan - mycount;

    const uint dststart = tex1Dfetch<uint>( texDstStart, mycid );
    const uint nsrc = scan[wid][27];
    const uint ndst = xsub( tex1Dfetch<uint>( texDstStart, xadd( mycid, 1 ) ), tex1Dfetch<uint>( texDstStart, mycid ) );

    for( uint d = 0u; d < ndst; d += ROWS ) {
        const uint np1 = xmin( ndst - d, ROWS );
        const uint dpid = xadd( xadd( dststart, d ), slot );
        const int entry = xscale( dpid, 3.f );
        float2 dtmp0 = xyzuvw[entry];
        float2 dtmp1 = xyzuvw[xadd(entry, 1)];
        float2 dtmp2 = xyzuvw[xadd(entry, 2)];

        float3 f = make_float3(0,0,0);

        for( uint s = 0; s < nsrc; s += COLS ) {
            const uint np2 = xmin( xsub(nsrc, s), COLS );
            const uint pid = xadd( s, subtid );
            const uint key9 = xadd( xsel_ge( pid, scan[wid][9            ], 9u, 0u ), xsel_ge( pid, scan[wid][18           ], 9u, 0u ) );
            const uint key3 = xadd( xsel_ge( pid, scan[wid][xadd(key9,3u)], 3u, 0u ), xsel_ge( pid, scan[wid][xadd(key9,6u)], 3u, 0u ) );
            const uint key  = xadd( key9, key3 );
            const uint spid = xsub( xadd( starts[wid][key], pid ), scan[wid][key] );
            const int sentry = xscale( spid, 3.f );

            const float2 stmp0 = tex1Dfetch<float2>( texSrcParticles, sentry );
            const float2 stmp1 = tex1Dfetch<float2>( texSrcParticles, xadd( sentry, 1 ) );
            const float2 stmp2 = tex1Dfetch<float2>( texSrcParticles, xadd( sentry, 2 ) );

            {
                const float _xr = dtmp0.x - stmp0.x;
                const float _yr = dtmp0.y - stmp0.y;
                const float _zr = dtmp1.x - stmp1.x;

                const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
                const float invrij = rsqrtf( rij2 );
                const float rij = rij2 * invrij;
                const float argwr = max( ( float )0, 1 - rij );
                const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

                const float xr = _xr * invrij;
                const float yr = _yr * invrij;
                const float zr = _zr * invrij;

                const float rdotv =
                    xr * ( dtmp1.y - stmp1.y ) +
                    yr * ( dtmp2.x - stmp2.x ) +
                    zr * ( dtmp2.y - stmp2.y );

                const uint arg1 = xsel_gt( mask, 0u, dpid, spid );
                const uint arg2 = xsel_gt( mask, 0u, spid, dpid );
                const float myrandnr = Logistic::mean0var1( seed, arg1, arg2 );

                const float strength = aij * argwr + ( - gamma * wr * rdotv + sigmaf * myrandnr ) * wr;
                const bool valid = xfcmp_lt( slot, np1 ) * xfcmp_lt( subtid, np2 );

                assert( ( dpid >= 0 && dpid < np && spid >= 0 && spid < np_src ) || ! valid );

                if( valid ) {
                    f.x += strength * xr;
                    f.y += strength * yr;
                    f.z += strength * zr;
                }
            }
        }

        for( uint L = COLS / 2u; L > 0u; L >>= 1 ) {
			f.x += __shfl_xor( f.x, L );
			f.y += __shfl_xor( f.y, L );
			f.z += __shfl_xor( f.z, L );
        }

        const uint c = ( subtid % 3u );
        const float fcontrib = xsel_eq( c, 0u, f.x, xsel_eq( c, 1u, f.y, f.z ) );
        const uint dstpid = xadd( xadd( dststart, d ), slot );

        if( slot < np1 )
            axayaz[ xmad( dstpid, 3.f, c ) ] = fcontrib;
    }
}

void forces_dpd_cuda_bipartite_nohost( hipStream_t stream, const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
                                       hipTextureObject_t texSrcStart, hipTextureObject_t texSrcParticles, const int np_src,
                                       const int3 halo_ncells,
                                       const float aij, const float gamma, const float sigmaf,
                                       const float seed, const int mask, float * const axayaz )
{
    const int ncells = halo_ncells.x * halo_ncells.y * halo_ncells.z;

    static bool fbip_init = false;

    if( !fbip_init ) {
        CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>( _dpd_bipforces), hipFuncCachePreferL1 ) );

        fbip_init = true;
    }

    _dpd_bipforces <<< ( ncells + CPB - 1 ) / CPB, dim3( 32, CPB ), 0, stream >>> (
        xyzuvw, np, texDstStart, texSrcStart, texSrcParticles, np_src,
        halo_ncells, aij, gamma, sigmaf, seed, mask,
        axayaz );
}
