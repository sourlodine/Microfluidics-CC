#include "hip/hip_runtime.h"
#include <cassert>

#include "../dpd-rng.h"

struct BipartiteInfoDPD
{
    int3 ncells;
    float3 domainsize, invdomainsize, domainstart;
    float invrc, aij, gamma, sigmaf;
};

__constant__ BipartiteInfoDPD bipart_info;

#ifndef NDEBUG
//#define _CHECK_
#endif
 
#define COLS 8
#define ROWS (32 / COLS)
#define CPB 4

#include "../hacks.h"

__global__
void _bipartite_dpd_directforces(float * const axayaz, const int np, const int np_src,
				 const float seed, const bool mask, const float * xyzuvw, const float * xyzuvw_src,
				 const float invrc, const float aij, const float gamma, const float sigmaf)
{
    assert(blockDim.x % warpSize == 0);
    assert(blockDim.x * gridDim.x >= np);
    
    const int tid = threadIdx.x % warpSize;
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;
    const bool valid = pid < np;

    float xp, yp, zp, up, vp, wp;

    if (valid)
    {
	xp = xyzuvw[0 + pid * 6];
	yp = xyzuvw[1 + pid * 6];
	zp = xyzuvw[2 + pid * 6];
	up = xyzuvw[3 + pid * 6];
	vp = xyzuvw[4 + pid * 6];
	wp = xyzuvw[5 + pid * 6];
    }

    float xforce = 0, yforce = 0, zforce = 0;
    
    for(int s = 0; s < np_src; s += warpSize)
    {
	float my_xq, my_yq, my_zq, my_uq, my_vq, my_wq;

	const int batchsize = min(warpSize, np_src - s);

	if (tid < batchsize)
	{
	    my_xq = xyzuvw_src[0 + (tid + s) * 6];
	    my_yq = xyzuvw_src[1 + (tid + s) * 6];
	    my_zq = xyzuvw_src[2 + (tid + s) * 6];
	    my_uq = xyzuvw_src[3 + (tid + s) * 6];
	    my_vq = xyzuvw_src[4 + (tid + s) * 6];
	    my_wq = xyzuvw_src[5 + (tid + s) * 6];
	}
	
	for(int l = 0; l < batchsize; ++l)
	{
	    const float xq = __shfl(my_xq, l);
	    const float yq = __shfl(my_yq, l);
	    const float zq = __shfl(my_zq, l);
	    const float uq = __shfl(my_uq, l);
	    const float vq = __shfl(my_vq, l);
	    const float wq = __shfl(my_wq, l);

	    //necessary to force the execution shuffles here below
	    //__syncthreads();
	    
	    //if (valid)
	    {
		const float _xr = xp - xq;
		const float _yr = yp - yq;
		const float _zr = zp - zq;
		
		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		
		const float invrij = rsqrtf(rij2);
		 
		const float rij = rij2 * invrij;
		const float argwr = max((float)0, 1 - rij * invrc);
		const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv = 
		    xr * (up - uq) +
		    yr * (vp - vq) +
		    zr * (wp - wq);
		
		const int spid = s + l;
		const int dpid = pid;
		const float myrandnr = Logistic::mean0var1(seed, mask ? dpid : spid, mask ? spid : dpid);
		
		const float strength = aij * argwr + (- gamma * wr * rdotv + sigmaf * myrandnr) * wr;
		//if (valid && spid < np_src)
		{
		    xforce += strength * xr;
		    yforce += strength * yr;
		    zforce += strength * zr;
		}
	    }
	}
    }

    if (valid)
    {
	assert(!isnan(xforce));
	assert(!isnan(yforce));
	assert(!isnan(zforce));
    
	axayaz[0 + 3 * pid] = xforce;
	axayaz[1 + 3 * pid] = yforce;
	axayaz[2 + 3 * pid] = zforce;
    }
}

void directforces_dpd_cuda_bipartite_nohost(
    const float * const xyzuvw, float * const axayaz, const int np,
    const float * const xyzuvw_src, const int np_src,
    const float aij, const float gamma, const float sigma, const float invsqrtdt,
    const float seed, const bool mask, hipStream_t stream)
{
    if (np == 0 || np_src == 0)
    {
	printf("warning: directforces_dpd_cuda_bipartite_nohost called with ZERO!\n");
	return;
    }
 
    _bipartite_dpd_directforces<<<(np + 127) / 128, 128, 0, stream>>>(axayaz, np, np_src, seed, mask,
								      xyzuvw, xyzuvw_src, 1, aij, gamma, sigma * invsqrtdt);
   
    CUDA_CHECK(hipPeekAtLastError());
}

__global__ __launch_bounds__(32 * CPB, 16) 
    void _dpd_bipforces(const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
			  hipTextureObject_t texSrcStart,  hipTextureObject_t texSrcParticles, const int np_src, const int3 halo_ncells,
			  const float aij, const float gamma, const float sigmaf,
			  const float seed, const bool mask, float * const axayaz)
{
    assert(warpSize == COLS * ROWS);
    assert(blockDim.x == warpSize && blockDim.y == CPB && blockDim.z == 1);
    assert(ROWS * 3 <= warpSize);

    const int mycid = blockIdx.x * CPB + threadIdx.y;

    if (mycid >= halo_ncells.x * halo_ncells.y * halo_ncells.z)
	return;

    const int xmycid = mycid % halo_ncells.x;
    const int ymycid = (mycid / halo_ncells.x) % halo_ncells.y;
    const int zmycid = (mycid / halo_ncells.x / halo_ncells.y) % halo_ncells.z;

    const int tid = threadIdx.x; 
    const int subtid = tid % COLS;
    const int slot = tid / COLS;
    const int wid = threadIdx.y;
     
    __shared__ int volatile starts[CPB][32], scan[CPB][32];

    int mycount = 0; 
    if (tid < 27)
    {
	const int dx = (1 + tid) % 3;
	const int dy = (1 + (tid / 3)) % 3; 
	const int dz = (1 + (tid / 9)) % 3;

	const int xcid = xmycid + dx - 1;
	const int ycid = ymycid + dy - 1;
	const int zcid = zmycid + dz - 1;
	
	const bool bad_cid =
	    xcid < 0 || xcid >= halo_ncells.x ||
	    ycid < 0 || ycid >= halo_ncells.y ||
	    zcid < 0 || zcid >= halo_ncells.z ;
	    
	const int cid = xcid + halo_ncells.x * (ycid + halo_ncells.y * zcid);

	starts[wid][tid] = bad_cid ? -10000 : tex1Dfetch<int>(texSrcStart, cid);
	mycount = bad_cid ? 0 : (tex1Dfetch<int>(texSrcStart, cid + 1) - tex1Dfetch<int>(texSrcStart, cid));
    }

    for(int L = 1; L < 32; L <<= 1)
	mycount += (tid >= L) * __shfl_up(mycount, L) ;

    if (tid < 27)
	scan[wid][tid] = mycount;

    const int dststart = tex1Dfetch<int>(texDstStart, mycid);
    const int nsrc = scan[wid][26], ndst = tex1Dfetch<int>(texDstStart, mycid + 1) - tex1Dfetch<int>(texDstStart, mycid);
    
    for(int d = 0; d < ndst; d += ROWS)
    {
	const int np1 = min(ndst - d, ROWS);

	const int dpid = dststart + d + slot;

	const int entry = 3 * dpid;
	float2 dtmp0 = xyzuvw[entry];
	float2 dtmp1 = xyzuvw[entry + 1];
	float2 dtmp2 = xyzuvw[entry + 2];
	
	float f[3] = {0, 0, 0};

	for(int s = 0; s < nsrc; s += COLS)
	{
	    const int np2 = min(nsrc - s, COLS);
  
	    const int pid = s + subtid;
	    const int key9 = 9 * (pid >= scan[wid][8]) + 9 * (pid >= scan[wid][17]);
	    const int key3 = 3 * (pid >= scan[wid][key9 + 2]) + 3 * (pid >= scan[wid][key9 + 5]);
	    const int key1 = (pid >= scan[wid][key9 + key3]) + (pid >= scan[wid][key9 + key3 + 1]);
	    const int key = key9 + key3 + key1;
	    assert(key >= 0 && key < 27);
	    assert(subtid >= np2 || pid >= (key ? scan[wid][key - 1] : 0) && pid < scan[wid][key]);

	    const int spid = starts[wid][key] + pid - (key ? scan[wid][key - 1] : 0);
	    assert(subtid >= np2 || starts[wid][key] >= 0);
	    
	    const int sentry = 3 * spid;
	    const float2 stmp0 = tex1Dfetch<float2>(texSrcParticles, sentry);
	    const float2 stmp1 = tex1Dfetch<float2>(texSrcParticles, sentry + 1);
	    const float2 stmp2 = tex1Dfetch<float2>(texSrcParticles, sentry + 2);
	    
	    {
		const float xforce = f[0];
		const float yforce = f[1];
		const float zforce = f[2];
			    
		const float _xr = dtmp0.x - stmp0.x;
		const float _yr = dtmp0.y - stmp0.y;
		const float _zr = dtmp1.x - stmp1.x;

		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
		const float invrij = rsqrtf(rij2);
		const float rij = rij2 * invrij;
		const float argwr = max((float)0, 1 - rij);
		const float wr = powf(argwr, powf(0.5f, -VISCOSITY_S_LEVEL));

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;
		
		const float rdotv = 
		    xr * (dtmp1.y - stmp1.y) +
		    yr * (dtmp2.x - stmp2.x) +
		    zr * (dtmp2.y - stmp2.y);
	
		const float myrandnr = Logistic::mean0var1(seed, mask ? dpid : spid, mask ? spid : dpid);

		const float strength = aij * argwr + (- gamma * wr * rdotv + sigmaf * myrandnr) * wr;
		const bool valid = (slot < np1) && (subtid < np2);

		assert( (dpid >= 0 && dpid < np && spid >= 0 && spid < np_src) || ! valid); 
		
		if (valid)
		{
		    f[0] = xforce + strength * xr;
		    f[1] = yforce + strength * yr;
		    f[2] = zforce + strength * zr;
		}
	    } 
	}
		
	for(int L = COLS / 2; L > 0; L >>=1)
	    for(int c = 0; c < 3; ++c)
		f[c] += __shfl_xor(f[c], L);

	const float fcontrib = f[subtid % 3];
	const int dstpid = dststart + d + slot;
	const int c = (subtid % 3);

	if (slot < np1)
	    axayaz[c + 3 * dstpid] = fcontrib;
    } 
}

void forces_dpd_cuda_bipartite_nohost(hipStream_t stream, const float2 * const xyzuvw, const int np, hipTextureObject_t texDstStart,
					    hipTextureObject_t texSrcStart, hipTextureObject_t texSrcParticles, const int np_src,
					    const int3 halo_ncells,
					    const float aij, const float gamma, const float sigmaf,
					    const float seed, const bool mask, float * const axayaz)
{ 
    const int ncells = halo_ncells.x * halo_ncells.y * halo_ncells.z;
    
    _dpd_bipforces<<<(ncells + CPB - 1) / CPB, dim3(32, CPB), 0, stream>>>(
	xyzuvw, np, texDstStart, texSrcStart, texSrcParticles, np_src,
	halo_ncells, aij, gamma, sigmaf, seed, mask,
	axayaz);
}