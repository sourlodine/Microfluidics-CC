#include "hip/hip_runtime.h"
#include "dumpavg.h"
#include "simple_serializer.h"
#include "utils.h"

#include <core/utils/kernel_launch.h>
#include <core/simulation.h>
#include <core/pvs/particle_vector.h>
#include <core/celllist.h>
#include <core/utils/cuda_common.h>


__global__ void sample(PVview pvView, CellListInfo cinfo, float* avgDensity, float3* avgMomentum, float3* avgForce)
{
	const int pid = threadIdx.x + blockIdx.x*blockDim.x;
	if (pid >= pvView.size) return;

	const float4 coo = pvView.particles[2*pid];
	const int cid = cinfo.getCellId(coo);

	if (avgDensity != nullptr)
		atomicAdd(avgDensity + cid, pvView.mass);

	if (avgMomentum != nullptr)
		atomicAdd(avgMomentum + cid, make_float3(pvView.particles[2*pid+1] * pvView.mass));

	if (avgForce != nullptr)
		atomicAdd( avgForce + cid, make_float3(pvView.forces[pid]) );
}

__global__ void scaleVec(int n, float3* vectorField, const float* density)
{
	const int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < n)
		vectorField[id] /= (density[id] + 1e-6f);
}

__global__ void scaleDensity(int n, float* density, const float factor)
{
	const int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < n)
		density[id] *= factor;
}

Avg3DPlugin::Avg3DPlugin(std::string name, std::string pvNames, int sampleEvery, int dumpEvery, float3 binSize,
			bool needMomentum, bool needForce) :
	SimulationPlugin(name), pvNames(pvNames),
	sampleEvery(sampleEvery), dumpEvery(dumpEvery), binSize(binSize),
	needDensity(true), needMomentum(needMomentum), needForce(needForce),
	nSamples(0)
{ }

void Avg3DPlugin::setup(Simulation* sim, const MPI_Comm& comm, const MPI_Comm& interComm)
{
	SimulationPlugin::setup(sim, comm, interComm);

	// TODO: this should be reworked if the domains are allowed to have different size
	resolution = make_int3( floorf(sim->domain.localSize / binSize) );
	binSize = sim->domain.localSize / make_float3(resolution);

	const int total = resolution.x * resolution.y * resolution.z;
	if (needDensity)  density .resize_anew(total);
	if (needMomentum) momentum.resize_anew(total);
	if (needForce)    force   .resize_anew(total);

	auto splitPvNames = splitByDelim(pvNames);

	density.clear(0);
	momentum.clear(0);
	force.clear(0);

	for (auto& nm : splitPvNames)
		particleVectors.push_back(sim->getPVbyNameOrDie(nm));

	info("Plugin %s initialized for the following particle vectors: %s", name.c_str(), pvNames.c_str());
}



void Avg3DPlugin::afterIntegration(hipStream_t stream)
{
	if (currentTimeStep % sampleEvery != 0 || currentTimeStep == 0) return;

	debug2("Plugin %s is sampling now", name.c_str());

	for (auto pv : particleVectors)
	{
		CellListInfo cinfo(binSize, pv->domain.localSize);
		PVview pvView(pv, pv->local());

		const int nthreads = 128;
		SAFE_KERNEL_LAUNCH(
				sample,
				getNblocks(pvView.size, nthreads), nthreads, 0, stream,
				pvView, cinfo,
				needDensity  ? density .devPtr() : nullptr,
				needMomentum ? momentum.devPtr() : nullptr,
				needForce    ? force   .devPtr() : nullptr );
	}

	nSamples++;
}

void Avg3DPlugin::serializeAndSend(hipStream_t stream)
{
	if (currentTimeStep % dumpEvery != 0 || currentTimeStep == 0) return;

	const int nthreads = 128;
	// Order is important here! First mom and frc, only then dens
	if (needMomentum)
	{
		int sz = momentum.size();
		SAFE_KERNEL_LAUNCH(
				scaleVec,
				getNblocks(sz, nthreads), nthreads, 0, stream,
				sz, momentum.devPtr(), density.devPtr() );

		momentum.downloadFromDevice(stream);
		momentum.clearDevice(stream);
	}

	if (needForce)
	{
		int sz = force.size();
		SAFE_KERNEL_LAUNCH(
				scaleVec,
				getNblocks(sz, nthreads), nthreads, 0, stream,
				sz, force.devPtr(), density.devPtr() );

		force.downloadFromDevice(stream);
		force.clearDevice(stream);
	}

	if (needDensity)
	{
		int sz = density.size();
		SAFE_KERNEL_LAUNCH(
				scaleDensity,
				getNblocks(sz, nthreads), nthreads, 0, stream,
				sz, density.devPtr(), 1.0 / (nSamples * binSize.x*binSize.y*binSize.z) );

		density.downloadFromDevice(stream);
		density.clearDevice(stream);
	}

	debug2("Plugin %s is sending now data", name.c_str());
	SimpleSerializer::serialize(sendBuffer, currentTime, density, momentum, force);
	send(sendBuffer);

	nSamples = 0;
}

void Avg3DPlugin::handshake()
{
	std::vector<char> data;
	SimpleSerializer::serialize(data, sim->nranks3D, resolution, binSize, needDensity, needMomentum, needForce);
	send(data.data(), data.size());

	debug2("Plugin %s was set up to sample%s%s%s for the following PVs: %s. Local resolution %dx%dx%d", name.c_str(),
			needDensity ? " density" : "", needMomentum ? " momentum" : "", needForce ? " force" : "", pvNames.c_str(),
			resolution.x, resolution.y, resolution.z);
}




Avg3DDumper::Avg3DDumper(std::string name, std::string path) :
		PostprocessPlugin(name), path(path) { }

void Avg3DDumper::handshake()
{
	auto req = waitData();
	MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );
	recv();

	SimpleSerializer::deserialize(data, nranks3D, resolution, h, needDensity, needMomentum, needForce);
	int totalPoints = resolution.x * resolution.y * resolution.z;

	std::vector<std::string> channelNames;
	std::vector<XDMFDumper::ChannelType> channelTypes;

	// For current time
	data.resize(sizeof(float));
	if (needDensity)
	{
		channelNames.push_back("density");
		channelTypes.push_back(XDMFDumper::ChannelType::Scalar);
		density.resize(totalPoints);
	}
	if (needMomentum)
	{
		channelNames.push_back("momentum");
		channelTypes.push_back(XDMFDumper::ChannelType::Vector);
		momentum.resize(totalPoints);
	}
	if (needForce)
	{
		channelNames.push_back("force");
		channelTypes.push_back(XDMFDumper::ChannelType::Vector);
		force.resize(totalPoints);
	}

	debug2("Plugin %s was set up to dump%s%s%s. Resolution %dx%dx%d. Path %s", name.c_str(),
			needDensity ? " density" : "", needMomentum ? " momentum" : "", needForce ? " force" : "",
			resolution.x, resolution.y, resolution.z, path.c_str());

	dumper = new XDMFDumper(comm, nranks3D, path, resolution, h, channelNames, channelTypes);
}

void Avg3DDumper::deserialize(MPI_Status& stat)
{
	float t;
	SimpleSerializer::deserialize(data, t, density, momentum, force);

	std::vector<const float*> channels;
	if (needDensity)  channels.push_back(density.data());
	if (needMomentum) channels.push_back((const float*)momentum.data());
	if (needForce)    channels.push_back((const float*)force.data());

	debug2("Plugin %s will dump right now", name.c_str());
	dumper->dump(channels, t);
}

