#include "hip/hip_runtime.h"
#include "dumpavg.h"
#include "simple_serializer.h"
#include "../core/simulation.h"
#include "../core/containers.h"
#include "../core/celllist.h"
#include "../core/hip/hip_vector_types.h"
#include <sstream>

__global__ void sample(int np, const float4* coosvels, const float4* forces,
		const float mass, CellListInfo cinfo, float* avgDensity, float3* avgMomentum, float3* avgForce)
{
	const int pid = threadIdx.x + blockIdx.x*blockDim.x;
	if (pid >= np) return;

	const float4 coo = coosvels[2*pid];
	const int cid = cinfo.getCellId(coo);

	if (avgDensity != nullptr)
		atomicAdd(avgDensity+cid, mass);

	if (avgMomentum != nullptr)
	{
		const float3 momentum = make_float3(coosvels[2*pid+1] * mass);
		atomicAdd( (float*)(avgMomentum + cid)  , momentum.x);
		atomicAdd( (float*)(avgMomentum + cid)+1, momentum.y);
		atomicAdd( (float*)(avgMomentum + cid)+2, momentum.z);
	}

	if (avgForce != nullptr)
	{
		const float3 frc = make_float3(forces[pid]);
		atomicAdd( (float*)(avgForce + cid)  , frc.x);
		atomicAdd( (float*)(avgForce + cid)+1, frc.y);
		atomicAdd( (float*)(avgForce + cid)+2, frc.z);
	}
}

__global__ void scaleVec(int n, float3* vectorField, const float* density, const float factor)
{
	const int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < n)
		vectorField[id] *= factor * __frcp_rn(density[id]);
}

__global__ void scaleDensity(int n, float* density, const float factor)
{
	const int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < n)
		density[id] *= factor;
}

Avg3DPlugin::Avg3DPlugin(std::string name, std::string pvNames, int sampleEvery, int dumpEvery, int3 resolution,
			bool needDensity, bool needMomentum, bool needForce) :
	SimulationPlugin(name), pvNames(pvNames),
	sampleEvery(sampleEvery), dumpEvery(dumpEvery), resolution(resolution),
	needDensity(needDensity), needMomentum(needMomentum), needForce(needForce),
	nSamples(0)
{
	// TODO: this should be reworked if the domains are allowed to have different size

	const int total = resolution.x * resolution.y * resolution.z;
	if (needDensity)  density .resize(total);
	if (needMomentum) momentum.resize(total);
	if (needForce)    force   .resize(total);
}

void Avg3DPlugin::setup(Simulation* sim, hipStream_t stream, const MPI_Comm& comm, const MPI_Comm& interComm)
{
	SimulationPlugin::setup(sim, stream, comm, interComm);

	std::stringstream sstream(pvNames);
	std::string pvName;
	std::vector<std::string> splitPvNames;

	while(std::getline(sstream, pvName, ','))
	{
		splitPvNames.push_back(pvName);
	}

	h = sim->subDomainSize / make_float3(resolution);

	density.pushStream(stream);
	density.clearDevice();

	momentum.pushStream(stream);
	momentum.clearDevice();

	force.pushStream(stream);
	force.clearDevice();

	for (auto& nm : splitPvNames)
	{
		auto& pvMap = sim->getPvMap();
		auto pvIter = pvMap.find(nm);
		if (pvIter == pvMap.end())
			die("No such particle vector registered: %s", nm.c_str());

		auto pv = sim->getParticleVectors()[pvIter->second];
		particleVectors.push_back(pv);
	}

	info("Plugin %s was set up for the following particle vectors: %s", name.c_str(), pvNames.c_str());
}



void Avg3DPlugin::afterIntegration()
{
	if (currentTimeStep % sampleEvery != 0 || currentTimeStep == 0) return;

	debug2("Plugin %s is sampling now", name.c_str());

	for (auto pv : particleVectors)
	{
		CellListInfo cinfo(h, pv->domainStart, pv->domainLength);

		sample<<< (pv->np+127) / 128, 128, 0, stream >>> (
				pv->np, (float4*)pv->coosvels.devPtr(), (float4*)pv->forces.devPtr(),
				pv->mass, cinfo,
				needDensity  ? density .devPtr() : nullptr,
				needMomentum ? momentum.devPtr() : nullptr,
				needForce    ? force   .devPtr() : nullptr );
	}

	nSamples++;
}

void Avg3DPlugin::serializeAndSend()
{
	if (currentTimeStep % dumpEvery != 0 || currentTimeStep == 0) return;

	// Order is important here! First mom and frc, only then dens
	if (needMomentum)
	{
		int sz = momentum.size();
		scaleVec<<< (sz+127)/128, 128, 0, stream >>> ( sz, momentum.devPtr(), density.devPtr(), 1.0/nSamples);
		momentum.downloadFromDevice();
		momentum.clearDevice();
	}

	if (needForce)
	{
		int sz = force.size();
		scaleVec<<< (sz+127)/128, 128, 0, stream >>> ( sz, force.devPtr(),    density.devPtr(), 1.0/nSamples);
		force.downloadFromDevice();
		force.clearDevice();
	}

	if (needDensity)
	{
		int sz = density.size();
		scaleDensity<<< (sz+127)/128, 128, 0, stream >>> ( sz, density.devPtr(), 1.0 / (nSamples * h.x*h.y*h.z) );
		density.downloadFromDevice();
		density.clearDevice();
	}

	debug2("Plugin %s is sending now data", name.c_str());
	SimpleSerializer::serialize(sendBuffer, currentTime, density, momentum, force);
	send(sendBuffer.hostPtr(), sendBuffer.size());

	nSamples = 0;
}

void Avg3DPlugin::handshake()
{
	HostBuffer<char> data;
	SimpleSerializer::serialize(data, resolution, h, needDensity, needMomentum, needForce);

	MPI_Check( MPI_Send(data.hostPtr(), data.size(), MPI_BYTE, rank, id, interComm) );

	debug2("Plugin %s was set up to sample%s%s%s for the following PVs: %s. Resolution %dx%dx%d", name.c_str(),
			needDensity ? " density" : "", needMomentum ? " momentum" : "", needForce ? " force" : "", pvNames.c_str(),
			resolution.x, resolution.y, resolution.z);
}





Avg3DDumper::Avg3DDumper(std::string name, std::string path, int3 nranks3D) :
		PostprocessPlugin(name), path(path), nranks3D(nranks3D) { }

void Avg3DDumper::handshake()
{
	HostBuffer<char> buf(1000);
	MPI_Check( MPI_Recv(buf.hostPtr(), buf.size(), MPI_BYTE, rank, id, interComm, MPI_STATUS_IGNORE) );
	SimpleSerializer::deserialize(buf, resolution, h, needDensity, needMomentum, needForce);
	int totalPoints = resolution.x * resolution.y * resolution.z;

	std::vector<std::string> channelNames;
	std::vector<XDMFDumper::ChannelType> channelTypes;

	// For current time
	data.resize(sizeof(float));
	if (needDensity)
	{
		channelNames.push_back("density");
		channelTypes.push_back(XDMFDumper::ChannelType::Scalar);
		density.resize(totalPoints);
	}
	if (needMomentum)
	{
		channelNames.push_back("momentum");
		channelTypes.push_back(XDMFDumper::ChannelType::Vector);
		momentum.resize(totalPoints);
	}
	if (needForce)
	{
		channelNames.push_back("force");
		channelTypes.push_back(XDMFDumper::ChannelType::Vector);
		force.resize(totalPoints);
	}

	float t;
	data.resize(SimpleSerializer::totSize(t, density, momentum, force));

	debug2("Plugin %s was set up to dump%s%s%s. Resolution %dx%dx%d. Path %s", name.c_str(),
			needDensity ? " density" : "", needMomentum ? " momentum" : "", needForce ? " force" : "",
			resolution.x, resolution.y, resolution.z, path.c_str());

	dumper = new XDMFDumper(comm, nranks3D, path, resolution, h, channelNames, channelTypes);

	size = data.size();
}

void Avg3DDumper::deserialize(MPI_Status& stat)
{
	float t;
	SimpleSerializer::deserialize(data, t, density, momentum, force);

	std::vector<const float*> channels;
	if (needDensity)  channels.push_back(density.hostPtr());
	if (needMomentum) channels.push_back((const float*)momentum.hostPtr());
	if (needForce)    channels.push_back((const float*)force.hostPtr());

	debug2("Plugin %s will dump right now", name.c_str());
	dumper->dump(channels, t);
}

