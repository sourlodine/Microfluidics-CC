#include "hip/hip_runtime.h"
#include <core/interactions/rod.h>
#include <core/logger.h>
#include <core/pvs/rod_vector.h>
#include <core/utils/hip/hip_vector_types.h>
#include <core/utils/quaternion.h>
#include <plugins/utils/xyz.h>

#include <vector>
#include <functional>
#include <gtest/gtest.h>

Logger logger;

#define FMT "%+6e"
#define SEP "\t"

using real = double;
using real2 = double2;
using real3 = double3;
using real4 = double4;

static real2 make_real2(float2 v) { return {(real) v.x, (real) v.y}; }
static real3 make_real3(float3 v) { return {(real) v.x, (real) v.y, (real) v.z}; }

using CenterLineFunc = std::function<real3(real)>;

static void initialFlagellum(int n, std::vector<real3>& positions, CenterLineFunc centerLine)
{
    positions.resize(5 * n + 1);
    real h = 1.0 / n;

    for (int i = 0; i < n; ++i) {
        real3 r = centerLine(i*h);

        positions[i * 5 + 0] = r;
        positions[i * 5 + 1] = r;
        positions[i * 5 + 2] = r;
        positions[i * 5 + 3] = r;
        positions[i * 5 + 4] = r;
    }

    positions[5*n] = centerLine(1.f);
}

static void getTransformation(real3 t0, real3 t1, real4& Q)
{
    Q = getQfrom(t0, t1);
    auto t0t1 = cross(t0, t1);
    if (length(t0t1) > 1e-6)
        t0t1 = normalize(t0t1);

    real err_t0_t1   = length(t1 - rotate(t0, Q));
    real err_t01_t01 = length(t0t1 - rotate(t0t1, Q));

    ASSERT_LE(err_t01_t01, 1e-6f);
    ASSERT_LE(err_t0_t1, 1e-6);
}

static void initialFrame(real3 t0, real3& u, real3& v)
{
    t0 = normalize(t0);
    u = anyOrthogonal(t0);
    u = normalize(u);
    v = normalize(cross(t0, u));
}

static void transportBishopFrame(const std::vector<real3>& positions, std::vector<real3>& frames)
{
    int n = (positions.size() - 1) / 5;
    
    for (int i = 1; i < n; ++i)
    {
        auto r0 = positions[5*(i-1)];
        auto r1 = positions[5*(i)];
        auto r2 = positions[5*(i+1)];
        
        auto t0 = normalize(r1-r0);
        auto t1 = normalize(r2-r1);

        real4 Q;
        getTransformation(t0, t1, Q);
        auto u0 = frames[2*(i-1) + 0];
        auto u1 = rotate(u0, Q);
        auto v1 = cross(t1, u1);
        frames[2*i + 0] = u1;
        frames[2*i + 1] = v1;
    }
}

static real bendingEnergy(const float2 B[2], float2 omega_eq, const std::vector<real3>& positions)
{
    int n = (positions.size() - 1) / 5;

    real Etot = 0;
    
    for (int i = 1; i < n; ++i)
    {
        auto r0 = positions[5*(i-1)];
        auto r1 = positions[5*(i)];
        auto r2 = positions[5*(i+1)];

        auto e0 = r1-r0;
        auto e1 = r2-r1;

        auto t0 = normalize(e0);
        auto t1 = normalize(e1);
        
        auto dp0 = positions[5*(i-1) + 2] - positions[5*(i-1) + 1];
        auto dp1 = positions[5*i     + 2] - positions[5*i     + 1];

        auto dp0Perp = dp0 - dot(dp0, t0) * t0;
        auto dp1Perp = dp1 - dot(dp1, t1) * t1;
        
        auto m10 = normalize(dp0Perp);
        auto m20 = cross(t0, m10);

        auto m11 = normalize(dp1Perp);
        auto m21 = cross(t1, m11);
        
        real denom = length(e0) * length(e1) + dot(e0, e1);
        auto bicur = (2.f / denom) * cross(e0, e1);
        
        real dp0Perpinv = 1.0 / length(dp0Perp);
        real dp1Perpinv = 1.0 / length(dp1Perp);

        real2 om0 = {+dp0Perpinv * dot(bicur, cross(t0, dp0)),
                     -dp0Perpinv * dot(bicur, dp0)};
        real2 om1 = {+dp1Perpinv * dot(bicur, cross(t1, dp1)),
                     -dp1Perpinv * dot(bicur, dp1)};
                                       
        
        om0 -= make_real2(omega_eq);
        om1 -= make_real2(omega_eq);

        real l = 0.5 * (length(e0) + length(e1));

        real2 Bom0 {dot(om0, make_real2(B[0])),
                    dot(om0, make_real2(B[1]))};
        
        real2 Bom1 {dot(om1, make_real2(B[0])),
                    dot(om1, make_real2(B[1]))};

        real E = (dot(Bom0, om0) + dot(Bom1, om1)) / (2.0 * l);
        Etot += E;
    }

    return Etot;
}

inline real safeDiffTheta(real t0, real t1)
{
    auto dth = t1 - t0;
    if (dth >  M_PI) dth -= 2.0 * M_PI;
    if (dth < -M_PI) dth += 2.0 * M_PI;
    return dth;
}

static real twistEnergy(real kTwist, real tau0, const std::vector<real3>& positions, const std::vector<real3>& frames)
{
    int n = (positions.size() - 1) / 5;

    real Etot = 0;
    
    for (int i = 1; i < n; ++i)
    {
        auto r0 = positions[5*(i-1)];
        auto r1 = positions[5*(i)];
        auto r2 = positions[5*(i+1)];

        auto u0 = frames[2*(i-1)   ];
        auto v0 = frames[2*(i-1) + 1];

        auto u1 = frames[2*i    ];
        auto v1 = frames[2*i + 1];
        
        auto dp0 = positions[5*(i-1) + 2] - positions[5*(i-1) + 1];
        auto dp1 = positions[5*i     + 2] - positions[5*i     + 1];
        
        auto e0 = r1-r0;
        auto e1 = r2-r1;
        auto l = 0.5 * (length(e0) + length(e1));

        auto theta0 = atan2(dot(dp0, v0), dot(dp0, u0));
        auto theta1 = atan2(dot(dp1, v1), dot(dp1, u1));

        auto tau = safeDiffTheta(theta0, theta1) / l;
        auto dtau = tau - tau0;
        
        auto E = kTwist * l * dtau * dtau;

        Etot += E;
    }

    return Etot;
}

static void bendingForces(real h, const float2 B[2], float2 omega_eq, const std::vector<real3>& positions, std::vector<real3>& forces)
{
    auto perturbed = positions;
    auto E0 = bendingEnergy(B, omega_eq, positions);

    auto computeEnergy = [&]() {
        return bendingEnergy(B, omega_eq, perturbed);
    };
    
    for (size_t i = 0; i < positions.size(); ++i)
    {
        auto computeForce = [&](real3 dir) {
            const auto r = positions[i];
            perturbed[i] = r + (h/2) * dir;
            auto Ep = computeEnergy();
            perturbed[i] = r - (h/2) * dir;
            auto Em = computeEnergy();
            perturbed[i] = r;
            return - (Ep - Em) / h;
        };

        forces[i].x = computeForce({1.0, 0.0, 0.0});
        forces[i].y = computeForce({0.0, 1.0, 0.0});
        forces[i].z = computeForce({0.0, 0.0, 1.0});
    }
}

static void twistForces(real h, float kt, float tau0, const std::vector<real3>& positions, std::vector<real3>& forces)
{
    auto perturbed = positions;
    int nSegments = (positions.size() - 1) / 5;
    
    std::vector<real3> frames(2*nSegments);

    auto compEnergy = [&]() {
                          initialFrame(perturbed[5]-perturbed[0], frames[0], frames[1]);
                          transportBishopFrame(perturbed, frames);
                          return twistEnergy(kt, tau0, perturbed, frames);
                      };
    
    for (size_t i = 0; i < positions.size(); ++i)
    {
        auto computeForce = [&](real3 dir) {
            const auto r = positions[i];
            perturbed[i] = r + (h/2) * dir;
            auto Ep = compEnergy();
            perturbed[i] = r - (h/2) * dir;
            auto Em = compEnergy();
            perturbed[i] = r;
            return - (Ep - Em) / h;
        };

        forces[i].x = computeForce({1.0, 0.0, 0.0});
        forces[i].y = computeForce({0.0, 1.0, 0.0});
        forces[i].z = computeForce({0.0, 0.0, 1.0});
    }
}

static void setCrosses(const std::vector<real3>& frames, std::vector<real3>& positions)
{
    int n = (positions.size() - 1) / 5;
    for (int i = 0; i < n; ++i)
    {
        auto u = frames[2*i+0];
        auto v = frames[2*i+1];
        auto r0 = positions[5*i+0];
        auto r1 = positions[5*i+5];
        auto dr = 0.5f * (r1 - r0);
        real a = length(dr);
        auto c = 0.5f * (r0 + r1);

        positions[i*5+1] = c - a * u;
        positions[i*5+2] = c + a * u;
        positions[i*5+3] = c - a * v;
        positions[i*5+4] = c + a * v;
    }
}

template <class CenterLine>
static void initializeRef(CenterLine centerLine, int nSegments, std::vector<real3>& positions, std::vector<real3>& frames)
{
    initialFlagellum(nSegments, positions, centerLine);

    frames.resize(2*nSegments);
    initialFrame(positions[5]-positions[0],
                 frames[0], frames[1]);

    transportBishopFrame(positions, frames);
    setCrosses(frames, positions);
}

static void copyToRv(const std::vector<real3>& positions, RodVector& rod)
{
    auto& pos = rod.local()->positions ();
    auto& vel = rod.local()->velocities();

    for (int i = 0; i < positions.size(); ++i)
    {
        Particle p;
        p.r = make_float3(positions[i]);
        p.u = make_float3(0);
        p.setId(i);
        pos[i] = p.r2Float4();
        vel[i] = p.u2Float4();
    }
    pos.uploadToDevice(defaultStream);
    vel.uploadToDevice(defaultStream);    
}

template <class CenterLine>
static double testBishopFrame(CenterLine centerLine)
{
    YmrState state(DomainInfo(), 0.f);
    int nSegments {200};
    
    std::vector<real3> refPositions, refFrames;
    RodVector rod(&state, "rod", 1.f, nSegments, 1);

    initializeRef(centerLine, nSegments, refPositions, refFrames);
    copyToRv(refPositions, rod);
    
    rod.updateBishopFrame(defaultStream);

    HostBuffer<float3> frames;
    frames.copy(rod.local()->bishopFrames, defaultStream);
    CUDA_Check( hipDeviceSynchronize() );

    double Linfty = 0;
    for (int i = 0; i < refFrames.size() / 2; ++i)
    {
        real3 a = refFrames[2*i];
        real3 b = make_real3(frames[i]);
        auto diff = a - b;
        double err = std::max(std::max(fabs(diff.x), fabs(diff.y)), fabs(diff.z));

        Linfty = std::max(Linfty, err);
    }
    return Linfty;
}

TEST (FLAGELLA, BishopFrames_straight)
{
    real height = 1.0;
    
    auto centerLine = [&](real s) -> real3 {
                          return {(real)0.0, (real)0.0, s*height};
                      };

    auto err = testBishopFrame(centerLine);
    ASSERT_LE(err, 1e-5);
}

TEST (FLAGELLA, BishopFrames_circle)
{
    real radius = 0.5;

    auto centerLine = [&](real s) -> real3 {
                          real theta = s * 2 * M_PI;
                          real x = radius * cos(theta);
                          real y = radius * sin(theta);
                          return {x, y, 0.f};
                      };

    auto err = testBishopFrame(centerLine);
    ASSERT_LE(err, 3e-5);
}

TEST (FLAGELLA, BishopFrames_helix)
{
    real pitch  = 1.0;
    real radius = 0.5;
    real height = 1.0;
    
    auto centerLine = [&](real s) -> real3 {
                          real z = s * height;
                          real theta = 2 * M_PI * z / pitch;
                          real x = radius * cos(theta);
                          real y = radius * sin(theta);
                          return {x, y, z};
                      };

    auto err = testBishopFrame(centerLine);
    ASSERT_LE(err, 2e-5);
}


template <class CenterLine>
static double testTwistForces(float kt, float tau0, CenterLine centerLine, int nSegments, real h)
{
    YmrState state(DomainInfo(), 0.f);

    RodParameters params;
    params.kBending = {0.f, 0.f, 0.f};
    params.omegaEq = {{0.f, 0.f}};
    params.kTwist = kt;
    params.tauEq = {tau0};
    params.groundE = {0.f};
    params.a0 = params.l0 = 0.f;
    params.kBounds = 0.f;
    params.kVisc = 0.f;
    
    std::vector<real3> refPositions, refFrames, refForces;
    RodVector rod(&state, "rod", 1.f, nSegments, 1);
    InteractionRod interactions(&state, "rod_interaction", params);
    initializeRef(centerLine, nSegments, refPositions, refFrames);
    copyToRv(refPositions, rod);


    refForces.resize(refPositions.size());
    twistForces(h, kt, tau0, refPositions, refForces);

    rod.local()->forces().clear(defaultStream);
    interactions.setPrerequisites(&rod, &rod, nullptr, nullptr);
    interactions.local(&rod, &rod, nullptr, nullptr, defaultStream);

    HostBuffer<Force> forces;
    forces.copy(rod.local()->forces(), defaultStream);
    CUDA_Check( hipDeviceSynchronize() );

    double Linfty = 0;
    for (int i = 0; i < refForces.size(); ++i)
    {
        real3 a = refForces[i];
        real3 b = make_real3(forces[i].f);
        real3 diff = a - b;
        double err = std::max(std::max(fabs(diff.x), fabs(diff.y)), fabs(diff.z));

        // if ((i % 5) == 0) printf("%03d ---------- \n", i/5);
        // if ((i % 5) == 0)
        //     printf(FMT SEP FMT SEP FMT SEP SEP
        //            FMT SEP FMT SEP FMT SEP SEP
        //            FMT SEP FMT "\n",
        //            a.x, a.y, a.z,
        //            b.x, b.y, b.z,
        //            length(a), length(b));
        
        Linfty = std::max(Linfty, err);
    }
    return Linfty;
}

template <class CenterLine>
static double testBendingForces(float3 B, float2 omega, CenterLine centerLine, int nSegments, real h)
{
    YmrState state(DomainInfo(), 0.f);

    RodParameters params;
    params.kBending = B;
    params.omegaEq = {omega};
    params.kTwist = 0.f;
    params.tauEq = {0.f};
    params.groundE = {0.f};
    params.a0 = params.l0 = 1.f; // set to 1.f so that omegaEq is the one entered
    params.kBounds = 0.f;
    params.kVisc = 0.f;
    
    std::vector<real3> refPositions, refFrames, refForces;
    RodVector rod(&state, "rod", 1.f, nSegments, 1);
    InteractionRod interactions(&state, "rod_interaction", params);
    initializeRef(centerLine, nSegments, refPositions, refFrames);
    copyToRv(refPositions, rod);


    refForces.resize(refPositions.size());
    const float2 B_[2] {{B.x, B.y}, {B.y, B.z}};
    bendingForces(h, B_, omega, refPositions, refForces);

    rod.local()->forces().clear(defaultStream);
    interactions.setPrerequisites(&rod, &rod, nullptr, nullptr);
    interactions.local(&rod, &rod, nullptr, nullptr, defaultStream);

    HostBuffer<Force> forces;
    forces.copy(rod.local()->forces(), defaultStream);
    CUDA_Check( hipDeviceSynchronize() );

    double Linfty = 0;
    for (int i = 0; i < refForces.size(); ++i)
    {
        real3 a = refForces[i];
        real3 b = make_real3(forces[i].f);
        real3 diff = a - b;
        double err = std::max(std::max(fabs(diff.x), fabs(diff.y)), fabs(diff.z));
        
        // if ((i % 5) == 0) printf("%03d ---------- \n", i/5);
        // if ((i % 5) == 0)
        //     printf(FMT SEP FMT SEP FMT SEP SEP
        //            FMT SEP FMT SEP FMT SEP SEP
        //            FMT SEP FMT "\n",
        //            a.x, a.y, a.z,
        //            b.x, b.y, b.z,
        //            length(a), length(b));

        Linfty = std::max(Linfty, err);
    }
    return Linfty;
}


TEST (FLAGELLA, twistForces_straight)
{
    real height = 5.0;
    real h = 1e-6;
    
    auto centerLine = [&](real s) -> real3 {
                          return {0.f, 0.f, s*height};
                      };

    auto err = testTwistForces(1.f, 0.1f, centerLine, 50, h);
    ASSERT_LE(err, 1e-5);
}

TEST (FLAGELLA, twistForces_helix)
{
    real pitch  = 1.0;
    real radius = 0.5;
    real height = 1.0;
    real h = 1e-7;
    
    auto centerLine = [&](real s) -> real3 {
                          real z = s * height;
                          real theta = 2 * M_PI * z / pitch;
                          real x = radius * cos(theta);
                          real y = radius * sin(theta);
                          return {x, y, z};
                      };

    auto err = testTwistForces(1.f, 0.1f, centerLine, 50, h);
    ASSERT_LE(err, 1e-3);
}


TEST (FLAGELLA, bendingForces_straight)
{
    real height = 5.0;
    real h = 1e-4;
    
    auto centerLine = [&](real s) -> real3 {
                          return {0.f, 0.f, s*height};
                      };

    auto err = testBendingForces({1.0f, 0.0f, 0.5f}, {0.1f, 0.2f}, centerLine, 10, h);
    ASSERT_LE(err, 5e-4);
}

TEST (FLAGELLA, bendingForces_circle)
{
    real radius = 4.0;
    real h = 1e-4;
    
    auto centerLine = [&](real s) -> real3 {
                          real theta = s * 2 * M_PI;
                          real x = radius * cos(theta);
                          real y = radius * sin(theta);
                          return {x, y, 0.f};
                      };


    float3 B {1.0f, 0.0f, 1.0f};
    float2 omega {0.f, 0.f};
    
    auto err = testBendingForces(B, omega, centerLine, 10, h);
    ASSERT_LE(err, 1e-3);
}

TEST (FLAGELLA, bendingForces_helix)
{
    real pitch  = 1.0;
    real radius = 0.5;
    real height = 1.0;
    real h = 1e-3;
    
    auto centerLine = [&](real s) -> real3 {
                          real z = s * height;
                          real theta = 2 * M_PI * z / pitch;
                          real x = radius * cos(theta);
                          real y = radius * sin(theta);
                          return {x, y, z};
                      };

    float3 B {1.0f, 0.0f, 1.0f};
    float2 omega {0.f, 0.f};
    
    auto err = testBendingForces(B, omega, centerLine, 10, h);
    ASSERT_LE(err, 1e-3);
}



int main(int argc, char **argv)
{
    MPI_Init(&argc, &argv);

    logger.init(MPI_COMM_WORLD, "flagella.log", 9);
    
    testing::InitGoogleTest(&argc, argv);
    auto ret = RUN_ALL_TESTS();

    MPI_Finalize();
    return ret;
}
