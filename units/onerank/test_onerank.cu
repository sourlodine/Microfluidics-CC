#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public

#include <gtest/gtest.h>

#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>
#include <core/celllist.h>
#include <core/domain.h>
#include <core/exchangers/api.h>
#include <core/logger.h>
#include <core/integrators/factory.h>
#include <core/interactions/dpd.h>

#include "../timer.h"
#include <unistd.h>

Logger logger;

const float dt = 0.0025;
const float kBT = 0.0; // to get rid of rng
const float gammadpd = 20;
const float adpd = 50;
const float powerdpd = 1.0;

const float sigma = sqrt(2 * gammadpd * kBT);
const float sigmaf = sigma / sqrt(dt);


void makeCells(float4*& pos, float4*& vel,
               float4*& posBuffer, float4*& velBuffer,
               int *cellsStartSize, int *cellsSize,
               int np, CellListInfo cinfo)
{
    for (int i = 0; i < cinfo.totcells+1; i++)
        cellsSize[i] = 0;

    for (int i = 0; i < np; i++)
        cellsSize[cinfo.getCellId(make_float3(pos[i]))]++;

    cellsStartSize[0] = 0;
    for (int i = 1; i <= cinfo.totcells; i++)
        cellsStartSize[i] = cellsSize[i-1] + cellsStartSize[i-1];

    for (int i = 0; i < np; i++)
    {
        const int cid = cinfo.getCellId(make_float3(pos[i]));
        posBuffer[cellsStartSize[cid]] = pos[i];
        velBuffer[cellsStartSize[cid]] = vel[i];
        cellsStartSize[cid]++;
    }

    for (int i = 0; i < cinfo.totcells; i++)
        cellsStartSize[i] -= cellsSize[i];

    std::swap(pos, posBuffer);
    std::swap(vel, velBuffer);
}

void integrate(float4* pos, float4 *vel, Force* accs,
               int np, float dt, CellListInfo cinfo, DomainInfo dinfo)
{
    float3 dstart = dinfo.globalStart;
    float3 dlength = dinfo.localSize;
    
    for (int i = 0; i < np; i++)
    {
        auto& r = pos[i];
        auto& u = vel[i];
        u.x += accs[i].f.x * dt;
        u.y += accs[i].f.y * dt;
        u.z += accs[i].f.z * dt;

        r.x += u.x * dt;
        r.y += u.y * dt;
        r.z += u.z * dt;
        
        if (r.x >  dstart.x+dlength.x) r.x -= dlength.x;
        if (r.x <= dstart.x)	       r.x += dlength.x;

        if (r.y >  dstart.y+dlength.y) r.y -= dlength.y;
        if (r.y <= dstart.y)	       r.y += dlength.y;

        if (r.z >  dstart.z+dlength.z) r.z -= dlength.z;
        if (r.z <= dstart.z)	       r.z += dlength.z;
    }
}


template<typename T>
T minabs(T arg)
{
    return arg;
}

template<typename T, typename... Args>
T minabs(T arg, Args... other)
{
    const T v = minabs(other...	);
    return (std::abs(arg) < std::abs(v)) ? arg : v;
}


void forces(const float4 *pos, const float4 *vel, Force *accs,
            const int *cellsStartSize, const int *cellsSize,
            CellListInfo cinfo, DomainInfo dinfo)
{
    float3 dlength = dinfo.localSize;
    
    auto addForce = [=] (int dstId, int srcId, Force& a)
    {
        Particle pdst(pos[dstId], vel[dstId]);
        Particle psrc(pos[srcId], vel[srcId]);
        
        float _xr = pdst.r.x - psrc.r.x;
        float _yr = pdst.r.y - psrc.r.y;
        float _zr = pdst.r.z - psrc.r.z;

        _xr = minabs(_xr, _xr - dlength.x, _xr + dlength.x);
        _yr = minabs(_yr, _yr - dlength.y, _yr + dlength.y);
        _zr = minabs(_zr, _zr - dlength.z, _zr + dlength.z);

        const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

        if (rij2 > 1.0f) return;
        //assert(rij2 < 1);

        const float invrij = 1.0f / sqrt(rij2);
        const float rij = rij2 * invrij;
        const float argwr = 1.0f - rij;
        const float wr = pow(argwr, powerdpd);

        const float xr = _xr * invrij;
        const float yr = _yr * invrij;
        const float zr = _zr * invrij;

        const float rdotv =
        xr * (pdst.u.x - psrc.u.x) +
        yr * (pdst.u.y - psrc.u.y) +
        zr * (pdst.u.z - psrc.u.z);

        const float myrandnr = 0;//Logistic::mean0var1(1, min(srcId, dstId), max(srcId, dstId));

        const float strength = adpd * argwr - (gammadpd * wr * rdotv + sigmaf * myrandnr) * wr;

        a.f.x += strength * xr;
        a.f.y += strength * yr;
        a.f.z += strength * zr;
    };

    const int3 ncells = cinfo.ncells;

#pragma omp parallel for collapse(3)
    for (int cx = 0; cx < ncells.x; cx++)
        for (int cy = 0; cy < ncells.y; cy++)
            for (int cz = 0; cz < ncells.z; cz++)
            {
                const int cid = cinfo.encode(cx, cy, cz);

                for (int dstId = cellsStartSize[cid]; dstId < cellsStartSize[cid] + cellsSize[cid]; dstId++)
                {
                    Force f (make_float4(0.f, 0.f, 0.f, 0.f));

                    for (int dx = -1; dx <= 1; dx++)
                        for (int dy = -1; dy <= 1; dy++)
                            for (int dz = -1; dz <= 1; dz++)
                            {
                                int ncx, ncy, ncz;
                                ncx = (cx+dx + ncells.x) % ncells.x;
                                ncy = (cy+dy + ncells.y) % ncells.y;
                                ncz = (cz+dz + ncells.z) % ncells.z;

                                const int srcCid = cinfo.encode(ncx, ncy, ncz);
                                if (srcCid >= cinfo.totcells || srcCid < 0) continue;

                                for (int srcId = cellsStartSize[srcCid]; srcId < cellsStartSize[srcCid] + cellsSize[srcCid]; srcId++)
                                {
                                    if (dstId != srcId)
                                        addForce(dstId, srcId, f);

                                    //printf("%d  %f %f %f\n", dstId, a.a[0], a.a[1], a.a[2]);
                                }
                            }

                    accs[dstId].f.x = f.f.x;
                    accs[dstId].f.y = f.f.y;
                    accs[dstId].f.z = f.f.z;
                }
            }
}

void execute(float3 length, int niters, double& l2, double& linf)
{
    hipStream_t defStream;
    CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );
    
    // Initial cells
    
    float3 domainStart = -length / 2.0f;
    const float rc = 1.0f;
    const float mass = 1.0f;

    DomainInfo domainInfo;
    domainInfo.localSize = length;
    domainInfo.globalStart.x = -0.5f * length.x;
    domainInfo.globalStart.y = -0.5f * length.y;
    domainInfo.globalStart.z = -0.5f * length.z;

    YmrState state(domainInfo, dt);
    
    ParticleVector pv(&state, "pv", mass);
    PrimaryCellList cells(&pv, rc, length);
    const int3 ncells = cells.ncells;

    const int ndens = 8;
    pv.local()->resize(ncells.x*ncells.y*ncells.z * ndens, defStream);
    auto& pos = pv.local()->positions();
    auto& vel = pv.local()->velocities();

    srand48(0);
    
    printf("initializing...\n");

    int c = 0;
    for (int i = 0; i < ncells.x; i++)
        for (int j = 0; j < ncells.y; j++)
            for (int k = 0; k < ncells.z; k++)
                for (int l = 0; l < ndens; l++)
                {
                    Particle p;
                    
                    p.r.x = i + drand48() + domainStart.x;
                    p.r.y = j + drand48() + domainStart.y;
                    p.r.z = k + drand48() + domainStart.z;                    

                    p.u.x = 0*(drand48() - 0.5);
                    p.u.y = 0*(drand48() - 0.5);
                    p.u.z = 0*(drand48() - 0.5);
                    p.setId(c);

                    pos[c] = p.r2Float4();
                    vel[c] = p.u2Float4();
                    c++;
                }


    pos.uploadToDevice(defStream);
    vel.uploadToDevice(defStream);
    pv.local()->forces().clear(defStream);

    HostBuffer<float4> positions(pv.local()->size()), velocities(pv.local()->size());
    std::copy(pos.begin(), pos.end(), positions .begin());
    std::copy(vel.begin(), vel.end(), velocities.begin());

    auto haloExchanger = std::make_unique<ParticleHaloExchanger>();
    haloExchanger->attach(&pv, &cells, {});
    SingleNodeEngine haloEngine(std::move(haloExchanger));

    auto redistributor = std::make_unique<ParticleRedistributor>();
    redistributor->attach(&pv, &cells);
    SingleNodeEngine redistEngine(std::move(redistributor));

    InteractionDPD dpd(&state, "dpd", rc, adpd, gammadpd, kBT, powerdpd);

    auto integrator = IntegratorFactory::createVV(&state, "vv");
    
    CUDA_Check( hipStreamSynchronize(defStream) );

    printf("GPU execution\n");

    Timer tm;
    tm.start();

    for (int i = 0; i < niters; i++)
    {
        state.currentStep = i;
        state.currentTime = i * dt;
        
        pv.local()->forces().clear(defStream);
        cells.build(defStream);

        haloEngine.init(defStream);
        
        dpd.setPrerequisites(&pv, &pv, &cells, &cells);
        dpd.local(&pv, &pv, &cells, &cells, defStream);

        haloEngine.finalize(defStream);

        dpd.halo(&pv, &pv, &cells, &cells, defStream);

        integrator->setPrerequisites(&pv);
        integrator->stage2(&pv, defStream);
        
        CUDA_Check( hipStreamSynchronize(defStream) );

        redistEngine.init(defStream);
        redistEngine.finalize(defStream);

        CUDA_Check( hipStreamSynchronize(defStream) );
    }

    double elapsed = tm.elapsed() * 1e-9;

    printf("Finished in %f s, 1 step took %f ms\n", elapsed, elapsed / niters * 1000.0);

    cells.build(defStream);

    int np = positions.size();
    int totcells = cells.totcells;

    HostBuffer<float4> posBuffer(np), velBuffer(np);
    HostBuffer<Force> accs(np);
    HostBuffer<int>   cellsStartSize(totcells+1), cellsSize(totcells+1);
    
    printf("CPU execution\n");
    
    for (int i = 0; i < niters; i++)
    {
        printf("%d...", i);
        fflush(stdout);

        makeCells(positions.hostptr, velocities.hostptr,
                  posBuffer.hostptr, velBuffer.hostptr,
                  cellsStartSize.data(), cellsSize.data(), np, cells.cellInfo());

        forces(positions.data(), velocities.data(),
               accs.data(), cellsStartSize.data(), cellsSize.data(), cells.cellInfo(), domainInfo);

        integrate(positions.data(), velocities.data(), accs.data(), np, dt, cells.cellInfo(), domainInfo);
    }

    printf("\nDone, checking\n");
    printf("NP:  %d,  ref  %d\n", pv.local()->size(), np);


    pos.downloadFromDevice(defStream, ContainersSynch::Asynch);
    vel.downloadFromDevice(defStream, ContainersSynch::Synch);

    std::vector<int> gpuid(np), cpuid(np);
    for (int i = 0; i < np; i++)
    {
        Particle pg(pos[i], vel[i]);
        Particle pc(positions[i], velocities[i]);
        
        gpuid[pg.getId()] = i;
        cpuid[pc.getId()] = i;
    }


    l2 = 0;
    linf = -1;

    for (int i = 0; i < np; i++)
    {
        Particle cpuP(positions[cpuid[i]], velocities[cpuid[i]]);
        Particle gpuP(pos[gpuid[i]], vel[gpuid[i]]);

        double perr = -1;


        double3 err = {
            fabs(cpuP.r.x - gpuP.r.x) + fabs(cpuP.u.x - gpuP.u.x),
            fabs(cpuP.r.y - gpuP.r.y) + fabs(cpuP.u.y - gpuP.u.y),
            fabs(cpuP.r.z - gpuP.r.z) + fabs(cpuP.u.z - gpuP.u.z)};
            
        linf = max(linf, max(err.x, max(err.y, err.z)));
        perr = max(perr, max(err.x, max(err.y, err.z)));
        l2 += err.x * err.x + err.y * err.y + err.z * err.z;

        if (perr > 0.01)
        {
            printf("id %8d diff %8e  [%12f %12f %12f  %8d] [%12f %12f %12f]\n"
                   "                           ref [%12f %12f %12f  %8d] [%12f %12f %12f] \n\n", i, perr,
                   gpuP.r.x, gpuP.r.y, gpuP.r.z, gpuP.i1,
                   gpuP.u.x, gpuP.u.y, gpuP.u.z,
                   cpuP.r.x, cpuP.r.y, cpuP.r.z, cpuP.i1,
                   cpuP.u.x, cpuP.u.y, cpuP.u.z);
        }
    }

    l2 = sqrt(l2 / pv.local()->size());
    printf("L2   norm: %f\n", l2);
    printf("Linf norm: %f\n", linf);
}

TEST (ONE_RANK, small)
{
    double l2, linf, tol;
    int niters = 50;
    float3 length{8, 8, 8};
    tol = 0.001;
    
    execute(length, niters, l2, linf);

    ASSERT_LE(l2,   tol);
    ASSERT_LE(linf, tol);
}

TEST (ONE_RANK, big)
{
    double l2, linf, tol;
    int niters = 3;
    float3 length{32, 32, 32};
    tol = 0.00002;
    
    execute(length, niters, l2, linf);

    ASSERT_LE(l2,   tol);
    ASSERT_LE(linf, tol);
}

int main(int argc, char ** argv)
{
    int provided, required = MPI_THREAD_FUNNELED;
    MPI_Init_thread(&argc, &argv, required, &provided);

    if (provided < required) {
        printf("ERROR: The MPI library does not have required thread support\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    logger.init(MPI_COMM_WORLD, "onerank.log", 9);

    testing::InitGoogleTest(&argc, argv);
    auto retval = RUN_ALL_TESTS();
    
    MPI_Finalize();
    return retval;
}
