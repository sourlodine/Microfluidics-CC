#include "hip/hip_runtime.h"
#include "velocity_control.h"
#include <plugins/simple_serializer.h>
#include <core/datatypes.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace velocity_control_kernels {

static __device__ bool is_inside(float3 r, float3 low, float3 high)
{
    return
        low.x <= r.x && r.x <= high.x &&
        low.y <= r.y && r.y <= high.y &&
        low.z <= r.z && r.z <= high.z;
}

__global__ void addForce(PVview view, DomainInfo domain, float3 low, float3 high, float3 force)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    Particle p;
    p.readCoordinate(view.particles, gid);
    float3 gr = domain.local2global(p.r);

    if (is_inside(gr, low, high))
        view.forces[gid] += make_float4(force, 0.0f);
}

__global__ void sumVelocity(PVview view, DomainInfo domain, float3 low, float3 high, float3 *totVel, int *nSamples)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    Particle p(view.particles, gid);
    float3 gr = domain.local2global(p.r);

    if (is_inside(gr, low, high))
        atomicAggInc(nSamples);
    else
        p.u = make_float3(0.0f);

    float3 u = warpReduce(p.u, [](float a, float b) { return a+b; });
    if (threadIdx.x % warpSize == 0 && dot(u, u) > 1e-8)
        atomicAdd(totVel, u);
}

}

SimulationVelocityControl::SimulationVelocityControl(const YmrState *state, std::string name, std::vector<std::string> pvNames,
                                                     float3 low, float3 high,
                                                     int sampleEvery, int tuneEvery, int dumpEvery,
                                                     float3 targetVel, float Kp, float Ki, float Kd) :
    SimulationPlugin(state, name),
    pvNames(pvNames),
    low(low),
    high(high),
    currentVel(make_float3(0,0,0)),
    targetVel(targetVel),
    sampleEvery(sampleEvery),
    tuneEvery(tuneEvery),
    dumpEvery(dumpEvery), 
    force(make_float3(0, 0, 0)),
    pid(make_float3(0, 0, 0), Kp, Ki, Kd),
    accumulatedTotVel({0,0,0})
{}


void SimulationVelocityControl::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    for (auto &pvName : pvNames)
        pvs.push_back(simulation->getPVbyNameOrDie(pvName));
}

void SimulationVelocityControl::beforeForces(hipStream_t stream)
{
    for (auto &pv : pvs)
    {
        PVview view(pv, pv->local());
        const int nthreads = 128;

        SAFE_KERNEL_LAUNCH
            (velocity_control_kernels::addForce,
             getNblocks(view.size, nthreads), nthreads, 0, stream,
             view, state->domain, low, high, force );
    }
}

void SimulationVelocityControl::sampleOnePv(ParticleVector *pv, hipStream_t stream) {
    PVview pvView(pv, pv->local());
    const int nthreads = 128;
 
    SAFE_KERNEL_LAUNCH
        (velocity_control_kernels::sumVelocity,
         getNblocks(pvView.size, nthreads), nthreads, 0, stream,
         pvView, state->domain, low, high, totVel.devPtr(), nSamples.devPtr());
}

void SimulationVelocityControl::afterIntegration(hipStream_t stream)
{
    if (currentTimeStep % sampleEvery == 0 && currentTimeStep != 0)
    {
        debug2("Velocity control %s is sampling now", name.c_str());

        totVel.clearDevice(stream);
        for (auto &pv : pvs) sampleOnePv(pv, stream);
        totVel.downloadFromDevice(stream);
        accumulatedTotVel.x += totVel[0].x;
        accumulatedTotVel.y += totVel[0].y;
        accumulatedTotVel.z += totVel[0].z;
    }
    
    if (currentTimeStep % tuneEvery != 0 || currentTimeStep == 0) return;
    
    nSamples.downloadFromDevice(stream);
    nSamples.clearDevice(stream);
    
    long nSamples_loc, nSamples_tot = 0;
    double3 totVel_tot = make_double3(0,0,0);  

    nSamples_loc = nSamples[0];
    
    MPI_Check( MPI_Allreduce(&nSamples_loc,        &nSamples_tot, 1, MPI_LONG,   MPI_SUM, comm) );
    MPI_Check( MPI_Allreduce(&accumulatedTotVel,   &totVel_tot,   3, MPI_DOUBLE, MPI_SUM, comm) );

    currentVel = nSamples_tot ? make_float3(totVel_tot / nSamples_tot) : make_float3(0.f, 0.f, 0.f);
    force = pid.update(targetVel - currentVel);
    accumulatedTotVel = {0,0,0};
}

void SimulationVelocityControl::serializeAndSend(hipStream_t stream)
{
    if (currentTimeStep % dumpEvery != 0 || currentTimeStep == 0) return;

    waitPrevSend();
    SimpleSerializer::serialize(sendBuffer, currentTime, currentTimeStep, currentVel, force);
    send(sendBuffer);
}




PostprocessVelocityControl::PostprocessVelocityControl(std::string name, std::string filename) :
    PostprocessPlugin(name)
{
    fdump = fopen(filename.c_str(), "w");
    if (!fdump) die("Could not open file '%s'", filename.c_str());
    fprintf(fdump, "# time time_step velocity force\n");
}

PostprocessVelocityControl::~PostprocessVelocityControl()
{
    fclose(fdump);
}

void PostprocessVelocityControl::deserialize(MPI_Status& stat)
{
    int currentTimeStep;
    float currentTime;
    float3 vel, force;

    SimpleSerializer::deserialize(data, currentTime, currentTimeStep, vel, force);

    if (rank == 0) {
        fprintf(fdump,
                "%g %d "
                "%g %g %g "
                "%g %g %g\n",
                currentTime, currentTimeStep,
                vel.x, vel.y, vel.z,
                force.x, force.y, force.z
                );
        
        fflush(fdump);
    }
}
