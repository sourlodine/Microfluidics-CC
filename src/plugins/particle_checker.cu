#include "hip/hip_runtime.h"
#include "particle_checker.h"
#include "utils/time_stamp.h"

#include <core/datatypes.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace ParticleCheckerKernels
{
__device__ inline bool checkFinite(float3 v)
{
    return isfinite(v.x) && isfinite(v.y) && isfinite(v.z);
}

__device__ inline bool withinBounds(float3 v, float3 bounds)
{
    return
        (fabs(v.x) < bounds.x) &&
        (fabs(v.y) < bounds.y) &&
        (fabs(v.z) < bounds.z);
}

__global__ void checkParticles(PVview view, DomainInfo domain, float dtInv, ParticleCheckerPlugin::ParticleStatus *status)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;

    if (pid >= view.size) return;

    auto pos = make_float3(view.readPosition(pid));
    auto vel = make_float3(view.readVelocity(pid));

    if (!checkFinite(pos) || !checkFinite(vel))
    {
        *status = {ParticleCheckerPlugin::Status::Nan, pid};
        return; // TODO: atomics?
    }

    float3 boundsPos = 1.5f  * domain.localSize; // particle should not be further that in a neighbouring domain
    float3 boundsVel = dtInv * domain.localSize; // particle should not travel more than one domain size per iteration

    if (!withinBounds(pos, boundsPos) || !withinBounds(vel, boundsVel))
    {
        *status = {ParticleCheckerPlugin::Status::Out, pid};
        return;
    }
}
} // namespace ParticleCheckerKernels
    
ParticleCheckerPlugin::ParticleCheckerPlugin(const YmrState *state, std::string name, int checkEvery) :
    SimulationPlugin(state, name),
    checkEvery(checkEvery)
{}

ParticleCheckerPlugin::~ParticleCheckerPlugin() = default;

void ParticleCheckerPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    pvs = simulation->getParticleVectors();

    statuses.resize_anew(pvs.size());

    for (auto& s : statuses)
        s = {Status::Ok, 0};
    statuses.uploadToDevice(defaultStream);
}

void ParticleCheckerPlugin::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(state, checkEvery)) return;

    const int nthreads = 128;

    auto dtInv = 1.0f / state->dt;
    auto domain = state->domain;
    
    for (size_t i = 0; i < pvs.size(); ++i)
    {
        auto pv = pvs[i];
        PVview view(pv, pv->local());

        SAFE_KERNEL_LAUNCH(
            ParticleCheckerKernels::checkParticles,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, domain, dtInv, statuses.devPtr() + i );
    }

    statuses.downloadFromDevice(stream, ContainersSynch::Synch);

    for (size_t i = 0; i < pvs.size(); ++i)
    {
        const auto& s = statuses[i];
        if (s.status == Status::Ok) continue;

        // from now we know we will fail; download particles and print error
        auto pv = pvs[i];
        auto lpv = pv->local();
        lpv->positions ().downloadFromDevice(stream, ContainersSynch::Asynch);
        lpv->velocities().downloadFromDevice(stream, ContainersSynch::Synch);

        auto p = Particle(lpv->positions ()[s.id],
                          lpv->velocities()[s.id]);

        die("Bad particle with id %ld, position %g %g %g, velocity %g %g %g : %s",
            p.getId(), p.r.x, p.r.y, p.r.z, p.u.x, p.u.y, p.u.z,
            s.status == Status::Nan ? "non finite number" : "out of bounds");
    }
}

