#include "hip/hip_runtime.h"
#include "particle_checker.h"
#include "utils/time_stamp.h"

#include <core/datatypes.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace ParticleCheckerKernels
{
__device__ inline bool checkFinite(float3 v)
{
    return isfinite(v.x) && isfinite(v.y) && isfinite(v.z);
}

__device__ inline bool withinBounds(float3 v, float3 bounds)
{
    return
        (fabs(v.x) < bounds.x) &&
        (fabs(v.y) < bounds.y) &&
        (fabs(v.z) < bounds.z);
}

__global__ void checkParticles(PVview view, DomainInfo domain, float dtInv, ParticleCheckerPlugin::ParticleStatus *status)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;

    if (pid >= view.size) return;

    auto pos = make_float3(view.readPosition(pid));
    auto vel = make_float3(view.readVelocity(pid));

    if (!checkFinite(pos) || !checkFinite(vel))
    {
        auto tag = atomicExch(&status->tag, ParticleCheckerPlugin::BAD);

        if (tag == ParticleCheckerPlugin::GOOD)
        {
            status->id   = pid;
            status->info = ParticleCheckerPlugin::Info::Nan;
        }
        return;
    }

    float3 boundsPos = 1.5f  * domain.localSize; // particle should not be further that in a neighbouring domain
    float3 boundsVel = dtInv * domain.localSize; // particle should not travel more than one domain size per iteration

    if (!withinBounds(pos, boundsPos) || !withinBounds(vel, boundsVel))
    {
        auto tag = atomicExch(&status->tag, ParticleCheckerPlugin::BAD);

        if (tag == ParticleCheckerPlugin::GOOD)
        {
            status->id   = pid;
            status->info = ParticleCheckerPlugin::Info::Out;
        }
        return;
    }
}
} // namespace ParticleCheckerKernels
    
ParticleCheckerPlugin::ParticleCheckerPlugin(const YmrState *state, std::string name, int checkEvery) :
    SimulationPlugin(state, name),
    checkEvery(checkEvery)
{}

ParticleCheckerPlugin::~ParticleCheckerPlugin() = default;

void ParticleCheckerPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    pvs = simulation->getParticleVectors();

    statuses.resize_anew(pvs.size());

    for (auto& s : statuses)
        s = {GOOD, 0, Info::Ok};
    statuses.uploadToDevice(defaultStream);
}

void ParticleCheckerPlugin::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(state, checkEvery)) return;

    const int nthreads = 128;

    auto dt     = state->dt;
    auto dtInv  = 1.0f / max(1e-6f, dt);
    auto domain = state->domain;
    
    for (size_t i = 0; i < pvs.size(); ++i)
    {
        auto pv = pvs[i];
        PVview view(pv, pv->local());

        SAFE_KERNEL_LAUNCH(
            ParticleCheckerKernels::checkParticles,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, domain, dtInv, statuses.devPtr() + i );
    }

    statuses.downloadFromDevice(stream, ContainersSynch::Synch);

    for (size_t i = 0; i < pvs.size(); ++i)
    {
        const auto& s = statuses[i];
        if (s.tag == GOOD) continue;

        // from now we know we will fail; download particles and print error
        auto pv = pvs[i];
        auto lpv = pv->local();
        lpv->positions ().downloadFromDevice(stream, ContainersSynch::Asynch);
        lpv->velocities().downloadFromDevice(stream, ContainersSynch::Synch);

        auto p = Particle(lpv->positions ()[s.id],
                          lpv->velocities()[s.id]);

        const char *infoStr = s.info == Info::Nan ? "non finite number" : "out of bounds";
        
        die("Bad particle in '%s' with id %ld, position %g %g %g, velocity %g %g %g : %s",
            pv->name.c_str(), p.getId(), p.r.x, p.r.y, p.r.z, p.u.x, p.u.y, p.u.z, infoStr);
    }
}

