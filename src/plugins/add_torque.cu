#include "hip/hip_runtime.h"
#include "add_torque.h"

#include <core/pvs/rigid_object_vector.h>
#include <core/pvs/views/rov.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace AddTorqueKernels
{

__global__ void addTorque(ROVview view, float3 torque)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.nObjects) return;

    view.motions[gid].torque += torque;
}

} // namespace AddTorqueKernels

AddTorquePlugin::AddTorquePlugin(const YmrState *state, std::string name, std::string rovName, float3 torque) :
    SimulationPlugin(state, name),
    rovName(rovName),
    torque(torque)
{}

void AddTorquePlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    rov = dynamic_cast<RigidObjectVector*>( simulation->getOVbyNameOrDie(rovName) );
    if (rov == nullptr)
        die("Need rigid object vector to add torque, plugin '%s', OV name '%s'",
            name.c_str(), rovName.c_str());

    info("Objects '%s' will experience external torque [%f %f %f]", 
            rovName.c_str(), torque.x, torque.y, torque.z);
}

void AddTorquePlugin::beforeForces(hipStream_t stream)
{
    ROVview view(rov, rov->local());
    const int nthreads = 128;

    SAFE_KERNEL_LAUNCH(
            AddTorqueKernels::addTorque,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, torque );
}

