#include "hip/hip_runtime.h"
#include "average_flow.h"

#include <core/utils/kernel_launch.h>
#include <core/simulation.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/celllist.h>
#include <core/utils/cuda_common.h>

#include "simple_serializer.h"
#include "sampling_helpers.h"

namespace average_flow_kernels {

__global__ void sample(
        PVview pvView, CellListInfo cinfo,
        float* avgDensity,
        ChannelsInfo channelsInfo)
{
    const int pid = threadIdx.x + blockIdx.x*blockDim.x;
    if (pid >= pvView.size) return;

    Particle p(pvView.particles, pid);

    int cid = cinfo.getCellId(p.r);

    atomicAdd(avgDensity + cid, 1);

    sampling_helpers_kernels::sampleChannels(pid, cid, channelsInfo);
}

}

Average3D::Average3D(std::string name,
        std::vector<std::string> pvNames,
        std::vector<std::string> channelNames, std::vector<Average3D::ChannelType> channelTypes,
        int sampleEvery, int dumpEvery, float3 binSize) :
    SimulationPlugin(name), pvNames(pvNames),
    sampleEvery(sampleEvery), dumpEvery(dumpEvery), binSize(binSize),
    nSamples(0)
{
    channelsInfo.n = channelTypes.size();
    channelsInfo.types.resize_anew(channelsInfo.n);
    channelsInfo.average.resize(channelsInfo.n);
    channelsInfo.averagePtrs.resize_anew(channelsInfo.n);
    channelsInfo.dataPtrs.resize_anew(channelsInfo.n);

    for (int i=0; i<channelsInfo.n; i++)
        channelsInfo.types[i] = channelTypes[i];

    channelsInfo.names = channelNames;
}

void Average3D::setup(Simulation* sim, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(sim, comm, interComm);

    domain = sim->domain;
    // TODO: this should be reworked if the domains are allowed to have different size
    resolution = make_int3( floorf(domain.localSize / binSize) );
    binSize = domain.localSize / make_float3(resolution);

    const int total = resolution.x * resolution.y * resolution.z;

    density.resize_anew(total);
    density.clear(0);
    std::string allChannels("density");
    for (int i=0; i<channelsInfo.n; i++)
    {
        if      (channelsInfo.types[i] == Average3D::ChannelType::Scalar)  channelsInfo.average[i].resize_anew(1*total);
        else if (channelsInfo.types[i] == Average3D::ChannelType::Tensor6) channelsInfo.average[i].resize_anew(6*total);
        else                                                               channelsInfo.average[i].resize_anew(3*total);

        channelsInfo.average[i].clear(0);
        channelsInfo.averagePtrs[i] = channelsInfo.average[i].devPtr();

        allChannels += ", " + channelsInfo.names[i];
    }

    channelsInfo.averagePtrs.uploadToDevice(0);
    channelsInfo.types.uploadToDevice(0);


    for (const auto& pvName : pvNames)
        pvs.push_back(sim->getPVbyNameOrDie(pvName));

    info("Plugin '%s' initialized for the %d PVs and channels %s, resolution %dx%dx%d",
         name.c_str(), pvs.size(), allChannels.c_str(),
         resolution.x, resolution.y, resolution.z);
}

void Average3D::sampleOnePv(ParticleVector *pv, hipStream_t stream)
{
    CellListInfo cinfo(binSize, pv->domain.localSize);
    PVview pvView(pv, pv->local());
    ChannelsInfo gpuInfo(channelsInfo, pv, stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
            average_flow_kernels::sample,
            getNblocks(pvView.size, nthreads), nthreads, 0, stream,
            pvView, cinfo, density.devPtr(), gpuInfo);
}

void Average3D::afterIntegration(hipStream_t stream)
{
    if (currentTimeStep % sampleEvery != 0 || currentTimeStep == 0) return;

    debug2("Plugin %s is sampling now", name.c_str());

    for (auto& pv : pvs) sampleOnePv(pv, stream);    

    nSamples++;
}

void Average3D::scaleSampled(hipStream_t stream)
{
    const int nthreads = 128;
    // Order is important here! First channels, only then dens

    for (int i=0; i<channelsInfo.n; i++)
    {
        auto& data = channelsInfo.average[i];
        int sz = density.size();
        int components = 3;
        if (channelsInfo.types[i] == ChannelType::Scalar)  components = 1;
        if (channelsInfo.types[i] == ChannelType::Tensor6) components = 6;

        SAFE_KERNEL_LAUNCH(
                sampling_helpers_kernels::scaleVec,
                getNblocks(sz, nthreads), nthreads, 0, stream,
                sz, components, data.devPtr(), density.devPtr() );

        data.downloadFromDevice(stream, ContainersSynch::Asynch);
        data.clearDevice(stream);
    }

    int sz = density.size();
    SAFE_KERNEL_LAUNCH(
            sampling_helpers_kernels::scaleDensity,
            getNblocks(sz, nthreads), nthreads, 0, stream,
            sz, density.devPtr(), /* pv->mass */ 1.0 / (nSamples * binSize.x*binSize.y*binSize.z) );

    density.downloadFromDevice(stream, ContainersSynch::Synch);
    density.clearDevice(stream);

    nSamples = 0;
}

void Average3D::serializeAndSend(hipStream_t stream)
{
    if (currentTimeStep % dumpEvery != 0 || currentTimeStep == 0) return;
    if (nSamples == 0) return;
    
    scaleSampled(stream);

    debug2("Plugin '%s' is now packing the data", name.c_str());
    SimpleSerializer::serialize(sendBuffer, currentTime, density, channelsInfo.average);
    send(sendBuffer);
}

void Average3D::handshake()
{
    std::vector<char> data;
    std::vector<int> sizes;

    for (auto t : channelsInfo.types)
        switch (t)
        {
            case ChannelType::Scalar:
                sizes.push_back(1);
                break;
            case ChannelType::Tensor6:
                sizes.push_back(6);
                break;
            default:
                sizes.push_back(3);
                break;
        }
    
    SimpleSerializer::serialize(data, sim->nranks3D, sim->rank3D, resolution, binSize, sizes, channelsInfo.names);
    send(data);
}

