#include "hip/hip_runtime.h"
#include "object_vector.h"
#include "views/ov.h"
#include "restart/helpers.h"
#include "checkpoint/helpers.h"

#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/folders.h>
#include <mirheo/core/xdmf/xdmf.h>

#include <limits>

namespace mirheo
{

constexpr const char *RestartOVIdentifier = "OV";

namespace ObjectVectorKernels
{

__global__ void minMaxCom(OVview ovView)
{
    const int gid    = threadIdx.x + blockDim.x * blockIdx.x;
    const int objId  = gid / warpSize;
    const int laneId = gid % warpSize;
    if (objId >= ovView.nObjects) return;

    real3 mymin = make_real3(+1e10_r);
    real3 mymax = make_real3(-1e10_r);
    real3 mycom = make_real3(0.0_r);

#pragma unroll 3
    for (int i = laneId; i < ovView.objSize; i += warpSize)
    {
        const int offset = objId * ovView.objSize + i;

        const real3 coo = make_real3(ovView.readPosition(offset));

        mymin = math::min(mymin, coo);
        mymax = math::max(mymax, coo);
        mycom += coo;
    }

    mycom = warpReduce( mycom, [] (real a, real b) { return a+b; } );
    mymin = warpReduce( mymin, [] (real a, real b) { return math::min(a, b); } );
    mymax = warpReduce( mymax, [] (real a, real b) { return math::max(a, b); } );

    if (laneId == 0)
        ovView.comAndExtents[objId] = {mycom / ovView.objSize, mymin, mymax};
}

} // namespace ObjectVectorKernels


LocalObjectVector::LocalObjectVector(ParticleVector *pv, int objSize, int nObjects) :
    LocalParticleVector(pv, objSize*nObjects), objSize(objSize), nObjects(nObjects)
{
    if (objSize <= 0)
        die("Object vector should contain at least one particle per object instead of %d", objSize);

    resize_anew(nObjects*objSize);
}

LocalObjectVector::~LocalObjectVector() = default;

void swap(LocalObjectVector& a, LocalObjectVector& b)
{
    swap(static_cast<LocalParticleVector &>(a), static_cast<LocalParticleVector &>(b));
    std::swap(a.nObjects, b.nObjects);
    std::swap(a.objSize,  b.objSize);
    swap(a.dataPerObject, b.dataPerObject);
}

void LocalObjectVector::resize(int np, hipStream_t stream)
{
    nObjects = getNobjects(np);
    LocalParticleVector::resize(np, stream);
    dataPerObject.resize(nObjects, stream);
}

void LocalObjectVector::resize_anew(int np)
{
    nObjects = getNobjects(np);
    LocalParticleVector::resize_anew(np);
    dataPerObject.resize_anew(nObjects);
}

void LocalObjectVector::computeGlobalIds(MPI_Comm comm, hipStream_t stream)
{
    LocalParticleVector::computeGlobalIds(comm, stream);

    if (np == 0) return;

    Particle p0( positions()[0], velocities()[0]);
    int64_t rankStart = p0.getId();
    
    if ((rankStart % objSize) != 0)
        die("Something went wrong when computing ids of '%s':"
            "got rankStart = '%ld' while objectSize is '%d'",
            pv->name.c_str(), rankStart, objSize);

    auto& ids = *dataPerObject.getData<int64_t>(ChannelNames::globalIds);
    int64_t id = (int64_t) (rankStart / objSize);
    
    for (auto& i : ids)
        i = id++;

    ids.uploadToDevice(stream);
}

PinnedBuffer<real4>* LocalObjectVector::getMeshVertices(__UNUSED hipStream_t stream)
{
    return &positions();
}

PinnedBuffer<real4>* LocalObjectVector::getOldMeshVertices(__UNUSED hipStream_t stream)
{
    return dataPerParticle.getData<real4>(ChannelNames::oldPositions);
}

PinnedBuffer<Force>* LocalObjectVector::getMeshForces(__UNUSED hipStream_t stream)
{
    return &forces();
}

int LocalObjectVector::getNobjects(int np) const
{
    if (np % objSize != 0)
        die("Incorrect number of particles in object: given %d, must be a multiple of %d", np, objSize);

    return np / objSize;
}


ObjectVector::ObjectVector(const MirState *state, std::string name, real mass, int objSize, int nObjects) :
    ObjectVector( state, name, mass, objSize,
                  std::make_unique<LocalObjectVector>(this, objSize, nObjects),
                  std::make_unique<LocalObjectVector>(this, objSize, 0) )
{}

ObjectVector::ObjectVector(const MirState *state, std::string name, real mass, int objSize,
                           std::unique_ptr<LocalParticleVector>&& local,
                           std::unique_ptr<LocalParticleVector>&& halo) :
    ParticleVector(state, name, mass, std::move(local), std::move(halo)),
    objSize(objSize)
{
    // center of mass and extents are not to be sent around
    // it's cheaper to compute them on site
    requireDataPerObject<COMandExtent>(ChannelNames::comExtents, DataManager::PersistenceMode::None);

    // object ids must always follow objects
    requireDataPerObject<int64_t>(ChannelNames::globalIds, DataManager::PersistenceMode::Active);
}

ObjectVector::~ObjectVector() = default;

void ObjectVector::findExtentAndCOM(hipStream_t stream, ParticleVectorLocality locality)
{
    auto lov = get(locality);

    debug("Computing COM and extent OV '%s' (%s)",
          name.c_str(), getParticleVectorLocalityStr(locality).c_str());

    OVview view(this, lov);
    
    constexpr int warpSize = 32;
    const int nthreads = 128;
    const int nblocks = getNblocks(view.nObjects * warpSize, nthreads);
    
    SAFE_KERNEL_LAUNCH(
            ObjectVectorKernels::minMaxCom,
            nblocks, nthreads, 0, stream,
            view );
}

static std::vector<real3> getCom(DomainInfo domain,
                                  const PinnedBuffer<COMandExtent>& com_extents)
{
    int n = com_extents.size();
    std::vector<real3> pos(n);

    for (int i = 0; i < n; ++i) {
        auto r = com_extents[i].com;
        pos[i] = domain.local2global(r);
    }

    return pos;
}

void ObjectVector::_checkpointObjectData(MPI_Comm comm, const std::string& path, int checkpointId)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointNameWithId(path, RestartOVIdentifier, "", checkpointId);
    info("Checkpoint for object vector '%s', writing to file %s",
         name.c_str(), filename.c_str());

    auto coms_extents = local()->dataPerObject.getData<COMandExtent>(ChannelNames::comExtents);

    coms_extents->downloadFromDevice(defaultStream, ContainersSynch::Synch);
    
    auto positions = std::make_shared<std::vector<real3>>(getCom(state->domain, *coms_extents));

    XDMF::VertexGrid grid(positions, comm);

    auto channels = CheckpointHelpers::extractShiftPersistentData(state->domain,
                                                                  local()->dataPerObject);
    
    XDMF::write(filename, &grid, channels, comm);

    createCheckpointSymlink(comm, path, RestartOVIdentifier, "xmf", checkpointId);

    debug("Checkpoint for object vector '%s' successfully written", name.c_str());
}

void ObjectVector::_restartObjectData(MPI_Comm comm, const std::string& path,
                                      const ObjectVector::ExchMapSize& ms)
{
    constexpr int objChunkSize = 1; // only one datum per object
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointName(path, RestartOVIdentifier, "xmf");
    info("Restarting object vector %s from file %s", name.c_str(), filename.c_str());

    auto listData = RestartHelpers::readData(filename, comm, objChunkSize);

    // remove positions from the read data (artificial for non rov)
    RestartHelpers::extractChannel<real3> (ChannelNames::XDMF::position, listData);
    
    RestartHelpers::exchangeListData(comm, ms.map, listData, objChunkSize);
    RestartHelpers::requireExtraDataPerObject(listData, this);

    auto& dataPerObject = local()->dataPerObject;
    dataPerObject.resize_anew(ms.newSize);

    RestartHelpers::copyAndShiftListData(state->domain, listData, dataPerObject);
    
    info("Successfully read object infos of '%s'", name.c_str());
}

void ObjectVector::checkpoint(MPI_Comm comm, const std::string& path, int checkpointId)
{
    _checkpointParticleData(comm, path, checkpointId);
    _checkpointObjectData  (comm, path, checkpointId);
}

void ObjectVector::restart(MPI_Comm comm, const std::string& path)
{
    auto ms = _restartParticleData(comm, path, objSize);
    _restartObjectData(comm, path, ms);
    
    local()->resize(ms.newSize * objSize, defaultStream);
}

} // namespace mirheo
