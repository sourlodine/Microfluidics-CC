#include "hip/hip_runtime.h"
#include "membrane.h"

#include "membrane/kernels/common.h"
#include "membrane/kernels/dihedral/kantor.h"
#include "membrane/kernels/dihedral/juelicher.h"
#include "membrane/kernels/triangle/lim.h"
#include "membrane/kernels/triangle/wlc.h"
#include "membrane/impl.h"

#include <mirheo/core/mirheo_undump.h>
#include <mirheo/core/pvs/membrane_vector.h>
#include <mirheo/core/pvs/views/ov.h>
#include <mirheo/core/utils/config.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/variant_foreach.h>

namespace mirheo
{

namespace MembraneInteractionKernels
{
__global__ void computeAreaAndVolume(OVviewWithAreaVolume view, MeshView mesh)
{
    const int objId = blockIdx.x;
    const int offset = objId * mesh.nvertices;
    real2 a_v = make_real2(0.0_r);

    for (int i = threadIdx.x; i < mesh.ntriangles; i += blockDim.x) {
        const int3 ids = mesh.triangles[i];

        const auto v0 = make_mReal3(make_real3( view.readPosition(offset + ids.x) ));
        const auto v1 = make_mReal3(make_real3( view.readPosition(offset + ids.y) ));
        const auto v2 = make_mReal3(make_real3( view.readPosition(offset + ids.z) ));

        a_v.x += triangleArea(v0, v1, v2);
        a_v.y += triangleSignedVolume(v0, v1, v2);
    }

    a_v = warpReduce( a_v, [] (real a, real b) { return a+b; } );

    if (laneId() == 0)
        atomicAdd(&view.area_volumes[objId], a_v);
}
} // namespace MembraneInteractionKernels

MembraneInteraction::MembraneInteraction(const MirState *state, std::string name, CommonMembraneParameters commonParams,
                                         VarBendingParams varBendingParams, VarShearParams varShearParams,
                                         bool stressFree, real growUntil, VarMembraneFilter varFilter) :
    Interaction(state, name, /* default cutoff rc */ 1.0)
{
    mpark::visit([&](auto bendingParams, auto shearParams, auto filter)
    {                     
        using FilterType    = decltype(filter);
        using DihedralForce = typename decltype(bendingParams)::DihedralForce;
        
        if (stressFree)
        {
            using TriangleForce = typename decltype(shearParams)::TriangleForce <StressFreeState::Active>;
            
            impl = std::make_unique<MembraneInteractionImpl<TriangleForce, DihedralForce, FilterType>>
                (state, name, commonParams, shearParams, bendingParams, growUntil, filter);
        }
        else
        {
            using TriangleForce = typename decltype(shearParams)::TriangleForce <StressFreeState::Inactive>;
            
            impl = std::make_unique<MembraneInteractionImpl<TriangleForce, DihedralForce, FilterType>>
                (state, name, commonParams, shearParams, bendingParams, growUntil, filter);
        }
        
    }, varBendingParams, varShearParams, varFilter);
}

namespace {
    struct MembraneImplVisitor
    {
        template <typename BendingParams, typename ShearParams, typename FilterType>
        void operator()()
        {
            using DihedralForce = typename BendingParams::DihedralForce;

            {
                using TriangleForce = typename ShearParams::TriangleForce <StressFreeState::Active>;
                using Impl = MembraneInteractionImpl<TriangleForce, DihedralForce, FilterType>;
                if (Impl::getTypeName() == implTypeName) {
                    *impl = std::make_unique<Impl>(state, undumper, implConfig);
                    return;
                }
            }
            {
                using TriangleForce = typename ShearParams::TriangleForce <StressFreeState::Inactive>;
                using Impl = MembraneInteractionImpl<TriangleForce, DihedralForce, FilterType>;
                if (Impl::getTypeName() == implTypeName)
                    *impl = std::make_unique<Impl>(state, undumper, implConfig);
            }
        }

        const MirState *state;
        Undumper& undumper;
        const ConfigDictionary& implConfig;
        const std::string& implTypeName;
        std::unique_ptr<Interaction> *impl;
    };
} // anonymous namespace

MembraneInteraction::MembraneInteraction(const MirState *state, Undumper& undumper, const ConfigDictionary& config) :
    Interaction(state, undumper, config)
{
    const ConfigDictionary& implConfig =
            undumper.getContext().getCompObjectConfig("InteractionImpl", getName());
    const std::string& implTypeName = implConfig["__type"].getString();

    /// Check all possible template combinations and match with the `implTypeName`.
    variantForeach<MembraneImplVisitor, VarBendingParams, VarShearParams, VarMembraneFilter>(
            MembraneImplVisitor{state, undumper, implConfig, implTypeName, &impl});

    if (!impl)
        die("Unrecognized impl type \"%s\".", implTypeName.c_str());
}


MembraneInteraction::~MembraneInteraction() = default;

void MembraneInteraction::saveSnapshotAndRegister(Dumper& dumper)
{
    dumper.registerObject<MembraneInteraction>(this, _saveSnapshotWithImpl(
                dumper, "MembraneInteraction"));
}

void MembraneInteraction::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    if (pv1 != pv2)
        die("Internal membrane forces can't be computed between two different particle vectors");

    auto ov = dynamic_cast<MembraneVector*>(pv1);
    if (ov == nullptr)
        die("Internal membrane forces can only be computed with a MembraneVector");

    ov->requireDataPerObject<real2>(ChannelNames::areaVolumes, DataManager::PersistenceMode::None);

    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}

void MembraneInteraction::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    if (impl.get() == nullptr)
        die("%s needs a concrete implementation, none was provided", getCName());

    precomputeQuantities(pv1, stream);
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void MembraneInteraction::halo(ParticleVector *pv1,
                               __UNUSED ParticleVector *pv2,
                               __UNUSED CellList *cl1,
                               __UNUSED CellList *cl2,
                               __UNUSED hipStream_t stream)
{
    debug("Not computing internal membrane forces between local and halo membranes of '%s'",
          pv1->getCName());
}

bool MembraneInteraction::isSelfObjectInteraction() const
{
    return true;
}

void MembraneInteraction::precomputeQuantities(ParticleVector *pv1, hipStream_t stream)
{
    auto ov = dynamic_cast<MembraneVector *>(pv1);

    if (ov->objSize != ov->mesh->getNvertices())
        die("Object size of '%s' (%d) and number of vertices (%d) mismatch",
            ov->getCName(), ov->objSize, ov->mesh->getNvertices());

    debug("Computing areas and volumes for %d cells of '%s'",
          ov->local()->nObjects, ov->getCName());

    OVviewWithAreaVolume view(ov, ov->local());

    MembraneMeshView mesh(static_cast<MembraneMesh*>(ov->mesh.get()));

    ov->local()
        ->dataPerObject.getData<real2>(ChannelNames::areaVolumes)
        ->clearDevice(stream);
    
    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(MembraneInteractionKernels::computeAreaAndVolume,
                       view.nObjects, nthreads, 0, stream,
                       view, mesh);
}

} // namespace mirheo
