#include "hip/hip_runtime.h"
#include "membrane.h"

#include "membrane/kernels/common.h"
#include "membrane/kernels/dihedral/kantor.h"
#include "membrane/kernels/dihedral/juelicher.h"
#include "membrane/kernels/triangle/lim.h"
#include "membrane/kernels/triangle/wlc.h"
#include "membrane/impl.h"

#include <mirheo/core/pvs/membrane_vector.h>
#include <mirheo/core/pvs/views/ov.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>

namespace mirheo
{


namespace MembraneInteractionKernels
{
__global__ void computeAreaAndVolume(OVviewWithAreaVolume view, MeshView mesh)
{
    const int objId = blockIdx.x;
    const int offset = objId * mesh.nvertices;
    real2 a_v = make_real2(0.0_r);

    for (int i = threadIdx.x; i < mesh.ntriangles; i += blockDim.x) {
        const int3 ids = mesh.triangles[i];

        const auto v0 = make_mReal3(make_real3( view.readPosition(offset + ids.x) ));
        const auto v1 = make_mReal3(make_real3( view.readPosition(offset + ids.y) ));
        const auto v2 = make_mReal3(make_real3( view.readPosition(offset + ids.z) ));

        a_v.x += triangleArea(v0, v1, v2);
        a_v.y += triangleSignedVolume(v0, v1, v2);
    }

    a_v = warpReduce( a_v, [] (real a, real b) { return a+b; } );

    if (laneId() == 0)
        atomicAdd(&view.area_volumes[objId], a_v);
}
} // namespace MembraneInteractionKernels

MembraneInteraction::MembraneInteraction(const MirState *state, std::string name, CommonMembraneParameters commonParams,
                                         VarBendingParams bendingParams, VarShearParams shearParams,
                                         bool stressFree, real growUntil) :
    Interaction(state, name, /* default cutoff rc */ 1.0)
{
    mpark::visit([&](auto bePrms, auto shPrms)
    {                     
        using DihedralForce = typename decltype(bePrms)::DihedralForce;
        
        if (stressFree)
        {
            using TriangleForce = typename decltype(shPrms)::TriangleForce <StressFreeState::Active>;
            
            impl = std::make_unique<MembraneInteractionImpl<TriangleForce, DihedralForce>>
                (state, name, commonParams, shPrms, bePrms, growUntil);
        }
        else                         
        {
            using TriangleForce = typename decltype(shPrms)::TriangleForce <StressFreeState::Inactive>;
            
            impl = std::make_unique<MembraneInteractionImpl<TriangleForce, DihedralForce>>
                (state, name, commonParams, shPrms, bePrms, growUntil);
        }
        
    }, bendingParams, shearParams);
}

MembraneInteraction::~MembraneInteraction() = default;

void MembraneInteraction::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    if (pv1 != pv2)
        die("Internal membrane forces can't be computed between two different particle vectors");

    auto ov = dynamic_cast<MembraneVector*>(pv1);
    if (ov == nullptr)
        die("Internal membrane forces can only be computed with a MembraneVector");

    ov->requireDataPerObject<real2>(ChannelNames::areaVolumes, DataManager::PersistenceMode::None);

    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}

void MembraneInteraction::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    if (impl.get() == nullptr)
        die("%s needs a concrete implementation, none was provided", name.c_str());

    precomputeQuantities(pv1, stream);
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void MembraneInteraction::halo(ParticleVector *pv1,
                               __UNUSED ParticleVector *pv2,
                               __UNUSED CellList *cl1,
                               __UNUSED CellList *cl2,
                               __UNUSED hipStream_t stream)
{
    debug("Not computing internal membrane forces between local and halo membranes of '%s'",
          pv1->name.c_str());
}

bool MembraneInteraction::isSelfObjectInteraction() const
{
    return true;
}

void MembraneInteraction::precomputeQuantities(ParticleVector *pv1, hipStream_t stream)
{
    auto ov = dynamic_cast<MembraneVector *>(pv1);

    if (ov->objSize != ov->mesh->getNvertices())
        die("Object size of '%s' (%d) and number of vertices (%d) mismatch",
            ov->name.c_str(), ov->objSize, ov->mesh->getNvertices());

    debug("Computing areas and volumes for %d cells of '%s'",
          ov->local()->nObjects, ov->name.c_str());

    OVviewWithAreaVolume view(ov, ov->local());

    MembraneMeshView mesh(static_cast<MembraneMesh*>(ov->mesh.get()));

    ov->local()
        ->dataPerObject.getData<real2>(ChannelNames::areaVolumes)
        ->clearDevice(stream);
    
    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(MembraneInteractionKernels::computeAreaAndVolume,
                       view.nObjects, nthreads, 0, stream,
                       view, mesh);
}

} // namespace mirheo
