#include "pairwise.h"
#include "pairwise/factory_helper.h"
#include "pairwise/impl.h"
#include "pairwise/impl.stress.h"
#include "pairwise/kernels/density.h"
#include "pairwise/kernels/density_kernels.h"
#include "pairwise/kernels/dpd.h"
#include "pairwise/kernels/mdpd.h"
#include "pairwise/kernels/pressure_EOS.h"
#include "pairwise/kernels/repulsive_lj.h"
#include "pairwise/kernels/sdpd.h"
#include "pairwise/kernels/type_traits.h"
#include <mirheo/core/utils/config.h>

#include <memory>

namespace mirheo
{

template <class KernelType>
static std::unique_ptr<Interaction>
createPairwiseFromKernel(const MirState *state, const std::string& name, real rc,
                         const KernelType& kernel,const VarStressParams& varStressParams)
{
    if (mpark::holds_alternative<StressActiveParams>(varStressParams))
    {
        const auto stressParams = mpark::get<StressActiveParams>(varStressParams);
        return std::make_unique<PairwiseInteractionWithStressImpl<KernelType>>(state, name, rc, stressParams.period, kernel);
    }
    else
    {
        return std::make_unique<PairwiseInteractionImpl<KernelType>>(state, name, rc, kernel);
    }
}

template <class KernelType>
static std::unique_ptr<Interaction>
createPairwiseFromKernelNoStress(const MirState *state, const std::string& name, real rc,
                         const KernelType& kernel, const VarStressParams& varStressParams)
{
    if (mpark::holds_alternative<StressActiveParams>(varStressParams))
        die("Incompatible interaction output: '%s' can not output stresses.", name.c_str());
    
    return std::make_unique<PairwiseInteractionImpl<KernelType>>(state, name, rc, kernel);
}


template <class Parameters>
static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, real rc, const Parameters& params, const VarStressParams& varStressParams)
{
    using KernelType = typename Parameters::KernelType;
    KernelType kernel(rc, params, state->dt);

    return createPairwiseFromKernel(state, name, rc, kernel, varStressParams);
}


static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, real rc, const LJParams& params, const VarStressParams& varStressParams)
{
    return mpark::visit([&](auto& awareParams)
    {
        using AwareType = typename std::remove_reference<decltype(awareParams)>::type::KernelType;
        
        AwareType awareness(awareParams);
        PairwiseRepulsiveLJ<AwareType> lj(rc, params.epsilon, params.sigma, params.maxForce, awareness);

        return createPairwiseFromKernel(state, name, rc, lj, varStressParams);
    }, params.varLJAwarenessParams);
}

static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, real rc, const DensityParams& params, const VarStressParams& varStressParams)
{
    return mpark::visit([&](auto& densityKernelParams)
    {
        using DensityKernelType = typename std::remove_reference<decltype(densityKernelParams)>::type::KernelType;
        
        DensityKernelType densityKernel;
        PairwiseDensity<DensityKernelType> density(rc, densityKernel);

        return createPairwiseFromKernelNoStress(state, name, rc, density, varStressParams);
    }, params.varDensityKernelParams);
}

static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, real rc, const SDPDParams& params, const VarStressParams& varStressParams)
{
    return mpark::visit([&](auto& densityKernelParams, auto& EOSParams)
    {
        using DensityKernelType = typename std::remove_reference<decltype(densityKernelParams)>::type::KernelType;
        using EOSKernelType     = typename std::remove_reference<decltype(EOSParams          )>::type::KernelType;
        
        DensityKernelType density;
        EOSKernelType pressure(EOSParams);

        PairwiseSDPD<EOSKernelType, DensityKernelType> sdpd(rc, pressure, density, params.viscosity, params.kBT, state->dt);
        
        return createPairwiseFromKernel(state, name, rc, sdpd, varStressParams);
    }, params.varDensityKernelParams, params.varEOSParams);
}


PairwiseInteraction::PairwiseInteraction(const MirState *state, const std::string& name, real rc,
                                         const VarPairwiseParams& varParams, const VarStressParams& varStressParams) :
    Interaction(state, name, rc),
    varParams(varParams),
    varStressParams(varStressParams)
{
    impl = mpark::visit([&](const auto& params)
    {
        return createPairwiseFromParams(state, name, rc, params, varStressParams);
    }, varParams);
}

PairwiseInteraction::~PairwiseInteraction() = default;

void PairwiseInteraction::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}
    
void PairwiseInteraction::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void PairwiseInteraction::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}


Interaction::Stage PairwiseInteraction::getStage() const
{
    return impl->getStage();
}

std::vector<Interaction::InteractionChannel> PairwiseInteraction::getInputChannels() const
{
    return impl->getInputChannels();
}

std::vector<Interaction::InteractionChannel> PairwiseInteraction::getOutputChannels() const
{
    return impl->getOutputChannels();
}

void PairwiseInteraction::checkpoint(MPI_Comm comm, const std::string& path, int checkpointId)
{
    return impl->checkpoint(comm, path, checkpointId);
}

void PairwiseInteraction::restart(MPI_Comm comm, const std::string& path)
{
    return impl->restart(comm, path);
}

ConfigDictionary PairwiseInteraction::writeSnapshot(Dumper&) const
{
    return {
        {"__category",      "Interaction"},
        {"__type",          "PairwiseInteraction"},
        {"varParams",       varParams},
        {"varStressParams", varStressParams},
    };
}

template <class Params>
static void readSpecificParams(Params& p, ParametersWrap& desc)
{
    using namespace FactoryHelper;
    readParams(p, desc, {ParamsReader::Mode::DefaultIfNotFound});
}

static void readSpecificParams(LJParams& p, ParametersWrap& desc)
{
    using namespace FactoryHelper;
    const ParamsReader reader{ParamsReader::Mode::DefaultIfNotFound};
    
    readParams(p, desc, reader);

    mpark::visit([&](auto& awareParams)
    {
        readParams(awareParams, desc, reader);
    }, p.varLJAwarenessParams);
}

static void readSpecificParams(DensityParams& p, ParametersWrap& desc)
{
    using namespace FactoryHelper;
    const ParamsReader reader{ParamsReader::Mode::DefaultIfNotFound};
    
    readParams(p, desc, reader);

    mpark::visit([&](auto& densityParams)
    {
        readParams(densityParams, desc, reader);
    }, p.varDensityKernelParams);
}

static void readSpecificParams(SDPDParams& p, ParametersWrap& desc)
{
    using namespace FactoryHelper;
    const ParamsReader reader{ParamsReader::Mode::DefaultIfNotFound};
    
    readParams(p, desc, reader);

    mpark::visit([&](auto& eosParams)
    {
        readParams(eosParams, desc, reader);
    }, p.varEOSParams);

    mpark::visit([&](auto& densityParams)
    {
        readParams(densityParams, desc, reader);
    }, p.varDensityKernelParams);
}


struct SpecificPairInfo
{
    ParticleVector *pv1, *pv2;
    Interaction *impl;
};

template <class KernelType>
static void setSpecificFromKernel(const KernelType& kernel, const VarStressParams& varStressParams, SpecificPairInfo info)
{
    if (mpark::holds_alternative<StressActiveParams>(varStressParams))
    {
        if (auto ptr = dynamic_cast<PairwiseInteractionWithStressImpl<KernelType>*>(info.impl))
        {
            ptr->setSpecificPair(info.pv1->getName(), info.pv2->getName(), kernel);
        }
        else
        {
            die("Internal error: could not convert to the given kernel");
        }
    }
    else
    {
        if (auto ptr = dynamic_cast<PairwiseInteractionImpl<KernelType>*>(info.impl))
        {
            ptr->setSpecificPair(info.pv1->getName(), info.pv2->getName(), kernel);
        }
        else
        {
            die("Internal error: could not convert to the given kernel");
        }
    }
}

template <class KernelType>
static void setSpecificFromKernelNoStress(const KernelType& kernel, const VarStressParams& varStressParams, SpecificPairInfo info)
{
    if (mpark::holds_alternative<StressActiveParams>(varStressParams))
        die("Incompatible interaction output: can not output stresses.");

    if (auto ptr = dynamic_cast<PairwiseInteractionImpl<KernelType>*>(info.impl))
    {
        ptr->setSpecificPair(info.pv1->getName(), info.pv2->getName(), kernel);
    }
    else
    {
        die("Internal error: could not convert to the given kernel");
    }
}

template <class Parameters>
static void setSpecificFromParams(const MirState *state, real rc, const Parameters& params,
                                  const VarStressParams& varStressParams, SpecificPairInfo info)
{
    using KernelType = typename Parameters::KernelType;
    KernelType kernel(rc, params, state->dt);

    setSpecificFromKernel(kernel, varStressParams, info);
}


static void setSpecificFromParams(__UNUSED const MirState *state, real rc, const LJParams& params,
                                  const VarStressParams& varStressParams, SpecificPairInfo info)
{
    mpark::visit([&](auto& awareParams)
    {
        using AwareType = typename std::remove_reference<decltype(awareParams)>::type::KernelType;
        
        AwareType awareness(awareParams);
        PairwiseRepulsiveLJ<AwareType> lj(rc, params.epsilon, params.sigma, params.maxForce, awareness);

        setSpecificFromKernel(lj, varStressParams, info);
    }, params.varLJAwarenessParams);
}

static void setSpecificFromParams(__UNUSED const MirState *state, real rc, const DensityParams& params,
                                  const VarStressParams& varStressParams, SpecificPairInfo info)
{
    mpark::visit([&](auto& densityKernelParams)
    {
        using DensityKernelType = typename std::remove_reference<decltype(densityKernelParams)>::type::KernelType;
        
        DensityKernelType densityKernel;
        PairwiseDensity<DensityKernelType> density(rc, densityKernel);

        setSpecificFromKernelNoStress(density, varStressParams, info);
    }, params.varDensityKernelParams);
}

static void setSpecificFromParams(const MirState *state, real rc, const SDPDParams& params,
                                  const VarStressParams& varStressParams, SpecificPairInfo info)
{
    mpark::visit([&](auto& densityKernelParams, auto& EOSParams)
    {
        using DensityKernelType = typename std::remove_reference<decltype(densityKernelParams)>::type::KernelType;
        using EOSKernelType     = typename std::remove_reference<decltype(EOSParams          )>::type::KernelType;
        
        DensityKernelType density;
        EOSKernelType pressure(EOSParams);

        PairwiseSDPD<EOSKernelType, DensityKernelType> sdpd(rc, pressure, density, params.viscosity, params.kBT, state->dt);
        
        setSpecificFromKernel(sdpd, varStressParams, info);
    }, params.varDensityKernelParams, params.varEOSParams);
}


void PairwiseInteraction::setSpecificPair(ParticleVector *pv1, ParticleVector *pv2, const ParametersWrap::MapParams& mapParams)
{
    auto varParamsSpecific = varParams;
    ParametersWrap desc(mapParams);

    const SpecificPairInfo info {pv1, pv2, impl.get()};

    mpark::visit([&](auto& params)
    {
        readSpecificParams(params, desc);
        setSpecificFromParams(getState(), rc, params, varStressParams, info);
    }, varParamsSpecific);
    
    desc.checkAllRead();
}

} // namespace mirheo
