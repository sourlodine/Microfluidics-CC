#include "translate.h"
#include "integration_kernel.h"

#include <mirheo/core/logger.h>
#include <mirheo/core/pvs/particle_vector.h>

namespace mirheo
{


/**
 * @param vel Move with this velocity
 */
IntegratorTranslate::IntegratorTranslate(const MirState *state, std::string name, real3 vel) :
    Integrator(state, name),
    vel(vel)
{}

IntegratorTranslate::~IntegratorTranslate() = default;

void IntegratorTranslate::stage2(ParticleVector *pv, hipStream_t stream)
{
    const auto _vel = vel;

    auto translate = [_vel] __device__ (Particle& p, const real3 f, const real invm, const real dt) {
        p.u = _vel;
        p.r += p.u*dt;
    };

    integrate(pv, getState()->dt, translate, stream);
    invalidatePV(pv);
}

} // namespace mirheo
