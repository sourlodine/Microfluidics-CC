#include "hip/hip_runtime.h"
#include "wall_with_velocity.h"

#include "common_kernels.h"
#include "stationary_walls/box.h"
#include "stationary_walls/cylinder.h"
#include "stationary_walls/plane.h"
#include "stationary_walls/sdf.h"
#include "stationary_walls/sphere.h"
#include "velocity_field/oscillate.h"
#include "velocity_field/rotate.h"
#include "velocity_field/translate.h"

#include <mirheo/core/celllist.h>
#include <mirheo/core/logger.h>
#include <mirheo/core/pvs/object_vector.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/root_finder.h>

#include <cassert>
#include <cmath>
#include <fstream>
#include <texture_types.h>

namespace mirheo
{


template<typename VelocityField>
__global__ void imposeVelField(PVview view, const VelocityField velField)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    Particle p(view.readParticle(pid));

    p.u = velField(p.r);

    view.writeParticle(pid, p);
}

//===============================================================================================
// Member functions
//===============================================================================================

template<class InsideWallChecker, class VelocityField>
WallWithVelocity<InsideWallChecker, VelocityField>::WallWithVelocity
(std::string name, const MirState *state, InsideWallChecker&& insideWallChecker, VelocityField&& velField) :
    SimpleStationaryWall<InsideWallChecker>(name, state, std::move(insideWallChecker)),
    velField(std::move(velField))
{}


template<class InsideWallChecker, class VelocityField>
void WallWithVelocity<InsideWallChecker, VelocityField>::setup(MPI_Comm& comm)
{
    info("Setting up wall %s", this->name.c_str());

    CUDA_Check( hipDeviceSynchronize() );

    this->insideWallChecker.setup(comm, this->getState()->domain);
    velField.setup(this->getState()->currentTime, this->getState()->domain);

    CUDA_Check( hipDeviceSynchronize() );
}

template<class InsideWallChecker, class VelocityField>
void WallWithVelocity<InsideWallChecker, VelocityField>::attachFrozen(ParticleVector* pv)
{
    SimpleStationaryWall<InsideWallChecker>::attachFrozen(pv);

    const int nthreads = 128;
    PVview view(pv, pv->local());
    SAFE_KERNEL_LAUNCH(
            imposeVelField,
            getNblocks(view.size, nthreads), nthreads, 0, 0,
            view, velField.handler() );

    CUDA_Check( hipDeviceSynchronize() );
}

template<class InsideWallChecker, class VelocityField>
void WallWithVelocity<InsideWallChecker, VelocityField>::bounce(hipStream_t stream)
{
    real t  = this->getState()->currentTime;
    real dt = this->getState()->dt;
    
    velField.setup(t, this->getState()->domain);
    this->bounceForce.clear(stream);

    for (size_t i = 0; i < this->particleVectors.size(); ++i)
    {
        auto  pv = this->particleVectors[i];
        auto  cl = this->cellLists[i];
        auto& bc = this->boundaryCells[i];
        auto view = cl->CellList::getView<PVviewWithOldParticles>();

        debug2("Bouncing %d %s particles with wall velocity, %d boundary cells",
               pv->local()->size(), pv->name.c_str(), bc.size());

        const int nthreads = 64;
        SAFE_KERNEL_LAUNCH(
                BounceKernels::sdfBounce,
                getNblocks(bc.size(), nthreads), nthreads, 0, stream,
                view, cl->cellInfo(), bc.devPtr(), bc.size(), dt,
                this->insideWallChecker.handler(),
                velField.handler(),
                this->bounceForce.devPtr());

        CUDA_Check( hipPeekAtLastError() );
    }
}


template class WallWithVelocity<StationaryWall_Sphere,   VelocityField_Rotate>;
template class WallWithVelocity<StationaryWall_Cylinder, VelocityField_Rotate>;
template class WallWithVelocity<StationaryWall_Plane,    VelocityField_Translate>;
template class WallWithVelocity<StationaryWall_Plane,    VelocityField_Oscillate>;

} // namespace mirheo
