#include "hip/hip_runtime.h"
#include "dump_obj_stats.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/pvs/rigid_object_vector.h>
#include <mirheo/core/pvs/views/ov.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/folders.h>
#include <mirheo/core/utils/hip/hip_vector_types.h>
#include <mirheo/core/utils/kernel_launch.h>

namespace mirheo
{

namespace ObjStatsPluginKernels
{

__global__ void collectObjStats(OVview view, RigidMotion *motionStats)
{
    const int objId  = blockIdx.x;
    const int tid    = threadIdx.x;
    const int laneId = tid % warpSize;

    RigidMotion local = {0};

    const real3 com = view.comAndExtents[objId].com;
    
    for (int i = tid; i < view.objSize; i += blockDim.x)
    {
        const int pid = objId * view.objSize + i;
        const Particle p = view.readParticle(pid);
        const real3 f = make_real3(view.forces[pid]);

        const real3 dr = p.r - com;
        
        local.vel    += p.u;
        local.omega  += cross(dr, p.u);
        local.force  += f;
        local.torque += cross(dr, f);
    }

    auto add = [](const RigidReal& a, const RigidReal& b) {return a+b;};

    warpReduce(local.vel,    add);
    warpReduce(local.omega,  add);
    warpReduce(local.force,  add);
    warpReduce(local.torque, add);

    if (laneId == 0)
    {
        atomicAdd( &motionStats[objId].vel,   local.vel   / view.objSize);
        atomicAdd( &motionStats[objId].omega, local.omega / view.objSize);

        atomicAdd( &motionStats[objId].force,  local.force );
        atomicAdd( &motionStats[objId].torque, local.torque);
    }
}

} // namespace ObjStatsPluginKernels

ObjStatsPlugin::ObjStatsPlugin(const MirState *state, std::string name, std::string ovName, int dumpEvery) :
    SimulationPlugin(state, name),
    ovName(ovName),
    dumpEvery(dumpEvery)
{}

void ObjStatsPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    ov = simulation->getOVbyNameOrDie(ovName);
    info("Plugin '%s' initialized for object vector '%s'", name.c_str(), ovName.c_str());
}

void ObjStatsPlugin::handshake()
{
    SimpleSerializer::serialize(sendBuffer, ovName);
    send(sendBuffer);
}

void ObjStatsPlugin::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(state, dumpEvery)) return;

    auto lov = ov->local();
    
    ids .copy( *lov->dataPerObject.getData<int64_t>     (ChannelNames::globalIds),  stream );
    coms.copy( *lov->dataPerObject.getData<COMandExtent>(ChannelNames::comExtents), stream );

    if (auto rov = dynamic_cast<RigidObjectVector*>(ov))
    {
        auto& oldMotions = *rov->local()->dataPerObject.getData<RigidMotion> (ChannelNames::oldMotions);
        motions.copy(oldMotions, stream);
        isRov = true;
    }
    else
    {
        const int nthreads = 128;
        OVview view(ov, lov);
        motionStats.resize_anew(view.nObjects);

        motionStats.clear(stream);

        SAFE_KERNEL_LAUNCH(
            ObjStatsPluginKernels::collectObjStats,
            view.nObjects, nthreads, 0, stream,
            view, motionStats.devPtr());

        motions.copy(motionStats, stream);
        isRov = false;
    }

    if (lov->dataPerObject.checkChannelExists(ChannelNames::membraneTypeId))
    {
        typeIds.copy( *lov->dataPerObject.getData<int>(ChannelNames::membraneTypeId), stream);
        hasTypeIds = true;
    }
    
    savedTime = state->currentTime;
    needToSend = true;
}

void ObjStatsPlugin::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (!needToSend) return;

    debug2("Plugin %s is sending now data", name.c_str());

    waitPrevSend();
    SimpleSerializer::serialize(sendBuffer, savedTime, state->domain, isRov, ids, coms, motions, hasTypeIds, typeIds);
    send(sendBuffer);
    
    needToSend=false;
}

//=================================================================================

static void writeStats(MPI_Comm comm, DomainInfo domain, MPI_File& fout, real curTime, const std::vector<int64_t>& ids,
                       const std::vector<COMandExtent>& coms, const std::vector<RigidMotion>& motions, bool isRov,
                       bool hasTypeIds, const std::vector<int>& typeIds)
{
    const int np = ids.size();

    std::stringstream ss;
    ss.setf(std::ios::fixed, std::ios::floatfield);
    ss.precision(5);

    for (int i = 0; i < np; ++i)
    {
        auto com = coms[i].com;
        com = domain.local2global(com);

        ss << ids[i] << " " << curTime << "   "
           << std::setw(10) << com.x << " "
           << std::setw(10) << com.y << " "
           << std::setw(10) << com.z;

        const auto& motion = motions[i];

        if (isRov)
        {
            ss << "    "
               << std::setw(10) << motion.q.x << " "
               << std::setw(10) << motion.q.y << " "
               << std::setw(10) << motion.q.z << " "
               << std::setw(10) << motion.q.w;
        }

        ss << "    "   
           << std::setw(10) << motion.vel.x << " "
           << std::setw(10) << motion.vel.y << " "
           << std::setw(10) << motion.vel.z << "    "
            
           << std::setw(10) << motion.omega.x << " "
           << std::setw(10) << motion.omega.y << " "
           << std::setw(10) << motion.omega.z << "    "
            
           << std::setw(10) << motion.force.x << " "
           << std::setw(10) << motion.force.y << " "
           << std::setw(10) << motion.force.z << "    "
            
           << std::setw(10) << motion.torque.x << " "
           << std::setw(10) << motion.torque.y << " "
           << std::setw(10) << motion.torque.z;

        if (hasTypeIds)
            ss << "    "  << typeIds[i];

        ss << std::endl;
    }

    const std::string content = ss.str();

    MPI_Offset offset = 0, size;
    MPI_Check( MPI_File_get_size(fout, &size) );
    MPI_Check( MPI_Barrier(comm) );

    const MPI_Offset len = content.size();
    MPI_Check( MPI_Exscan(&len, &offset, 1, MPI_OFFSET, MPI_SUM, comm) );

    MPI_Check( MPI_File_write_at_all(fout, offset + size, content.c_str(), len, MPI_CHAR, MPI_STATUS_IGNORE) );
    MPI_Check( MPI_Barrier(comm) );
}

//=================================================================================


ObjStatsDumper::ObjStatsDumper(std::string name, std::string path) :
    PostprocessPlugin(name),
    path(makePath(path))
{}

ObjStatsDumper::~ObjStatsDumper()
{
    if (activated)
        MPI_Check( MPI_File_close(&fout) );
}

void ObjStatsDumper::setup(const MPI_Comm& comm, const MPI_Comm& interComm)
{
    PostprocessPlugin::setup(comm, interComm);
    activated = createFoldersCollective(comm, path);
}

void ObjStatsDumper::handshake()
{
    auto req = waitData();
    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );
    recv();

    std::string ovName;
    SimpleSerializer::deserialize(data, ovName);

    if (activated)
    {
        const std::string fname = path + ovName + ".txt";
        MPI_Check( MPI_File_open(comm, fname.c_str(), MPI_MODE_CREATE | MPI_MODE_DELETE_ON_CLOSE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fout) );
        MPI_Check( MPI_File_close(&fout) );
        MPI_Check( MPI_File_open(comm, fname.c_str(), MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fout) );
    }
}


void ObjStatsDumper::deserialize()
{
    MirState::TimeType curTime;
    DomainInfo domain;
    std::vector<int64_t> ids;
    std::vector<COMandExtent> coms;
    std::vector<RigidMotion> motions;
    std::vector<int> typeIds;
    bool isRov;
    bool hasTypeIds;

    SimpleSerializer::deserialize(data, curTime, domain, isRov, ids, coms, motions, hasTypeIds, typeIds);

    if (activated)
        writeStats(comm, domain, fout, curTime, ids, coms, motions, isRov, hasTypeIds, typeIds);
}

} // namespace mirheo
