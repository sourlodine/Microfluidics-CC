#include "hip/hip_runtime.h"
#include "dump_particles_rod.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/pvs/rod_vector.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/cuda_common.h>

namespace mirheo
{

namespace DumpParticlesRodKernels
{

template <typename T>
__global__ void copyRodQuantities(int numBiSegmentsPerObject, int objSize, int nObjects, const T *rodData, T *particleData)
{
    constexpr int stride = 5;
    const int pid = threadIdx.x + blockIdx.x * blockDim.x;

    const int objId        = pid / objSize;
    const int localPartId  = pid % objSize;
    const int localBisegId = math::min(localPartId / stride, numBiSegmentsPerObject); // min because of last particle

    const int bid = objId * numBiSegmentsPerObject + localBisegId;

    if (objId < nObjects)
        particleData[pid] = rodData[bid];
}

} // namespace DumpParticlesRodKernels


ParticleWithRodQuantitiesSenderPlugin::
ParticleWithRodQuantitiesSenderPlugin(const MirState *state, std::string name, std::string pvName, int dumpEvery,
                                      const std::vector<std::string>& channelNames) :
    ParticleSenderPlugin(state, name, pvName, dumpEvery, channelNames)
{}

void ParticleWithRodQuantitiesSenderPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv = simulation->getPVbyNameOrDie(pvName);

    rv = dynamic_cast<RodVector*>(pv);

    if (rv == nullptr)
        die("Plugin '%s' expects a rod vector; given '%s'",
            name.c_str(), pvName.c_str());
    
    info("Plugin %s initialized for the following particle vector: %s", name.c_str(), pvName.c_str());
}

void ParticleWithRodQuantitiesSenderPlugin::beforeForces(hipStream_t stream)
{
    if (!isTimeEvery(state, dumpEvery)) return;

    positions .genericCopy(&pv->local()->positions() , stream);
    velocities.genericCopy(&pv->local()->velocities(), stream);

    auto& partManager  = pv->local()->dataPerParticle;
    auto& bisegManager = rv->local()->dataPerBisegment;

    for (size_t i = 0; i < channelNames.size(); ++i)
    {
        auto name = channelNames[i];
        if (partManager.checkChannelExists(name))
        {
            auto srcContainer = partManager.getGenericData(name);
            channelData[i].genericCopy(srcContainer, stream);
        }
        else
        {
            auto& desc = bisegManager.getChannelDescOrDie(name);
            auto& partData = channelRodData[name];
            
            mpark::visit([&](auto srcPinnedBuffer)
            {
                using Type = typename std::remove_pointer<decltype(srcPinnedBuffer)>::type::value_type;

                const int nparticles = rv->local()->size();
                const int objSize  = rv->objSize;
                const int nObjects = rv->local()->nObjects;
                
                const size_t sizeReals = pv->local()->size() * sizeof(Type) / sizeof(real);
                partData.resize_anew(sizeReals);

                constexpr int nthreads = 128;
                const int nblocks = getNblocks(nparticles, nthreads);
                
                SAFE_KERNEL_LAUNCH(
                    DumpParticlesRodKernels::copyRodQuantities,
                    nblocks, nthreads, 0, stream,
                    rv->local()->getNumSegmentsPerRod(), objSize, nObjects,
                    srcPinnedBuffer->devPtr(), reinterpret_cast<Type*>(partData.devPtr()));
            }, desc.varDataPtr);

            channelData[i].genericCopy(&partData, stream);
        }
    }
}

} // namespace mirheo
