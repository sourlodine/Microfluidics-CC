#include "hip/hip_runtime.h"
#include "stats.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/datatypes.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/config.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/mpi_types.h>

namespace mirheo
{

namespace StatsKernels
{
using Stats::ReductionType;

__global__ void totalMomentumEnergy(PVview view, ReductionType *momentum, ReductionType *energy, real* maxvel)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    real3 vel, myMomentum;
    real myEnergy = 0._r, myMaxIvelI;
    vel = myMomentum = make_real3(0._r);

    if (tid < view.size)
    {
        vel        = make_real3(view.readVelocity(tid));
        myMomentum = vel * view.mass;
        myEnergy   = dot(vel, vel) * view.mass * 0.5_r;
    }
    
    myMomentum = warpReduce(myMomentum, [](real a, real b) { return a+b; });
    myEnergy   = warpReduce(myEnergy,   [](real a, real b) { return a+b; });
    
    myMaxIvelI = warpReduce(length(vel), [](real a, real b) { return math::max(a, b); });

    if (laneId() == 0)
    {
        atomicAdd(momentum+0, (ReductionType)myMomentum.x);
        atomicAdd(momentum+1, (ReductionType)myMomentum.y);
        atomicAdd(momentum+2, (ReductionType)myMomentum.z);
        atomicAdd(energy,     (ReductionType)myEnergy);

        atomicMax((int*)maxvel, __float_as_int(myMaxIvelI));
    }
}
} // namespace StatsKernels
    
SimulationStats::SimulationStats(const MirState *state, std::string name, int fetchEvery) :
    SimulationPlugin(state, name),
    fetchEvery(fetchEvery)
{
    timer.start();
}

SimulationStats::~SimulationStats() = default;

void SimulationStats::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    pvs = simulation->getParticleVectors();
}

void SimulationStats::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(getState(), fetchEvery)) return;

    momentum.clear(stream);
    energy  .clear(stream);
    maxvel  .clear(stream);

    nparticles = 0;
    for (auto& pv : pvs)
    {
        PVview view(pv, pv->local());

        SAFE_KERNEL_LAUNCH(
                StatsKernels::totalMomentumEnergy,
                getNblocks(view.size, 128), 128, 0, stream,
                view, momentum.devPtr(), energy.devPtr(), maxvel.devPtr() );

        nparticles += view.size;
    }

    momentum.downloadFromDevice(stream, ContainersSynch::Asynch);
    energy  .downloadFromDevice(stream, ContainersSynch::Asynch);
    maxvel  .downloadFromDevice(stream);

    needToDump = true;
}

void SimulationStats::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (needToDump)
    {
        const real tm = timer.elapsedAndReset() / (getState()->currentStep < fetchEvery ? 1.0_r : fetchEvery);
        waitPrevSend();
        SimpleSerializer::serialize(sendBuffer, tm, getState()->currentTime, getState()->currentStep, nparticles, momentum, energy, maxvel);
        send(sendBuffer);
        needToDump = false;
    }
}

ConfigDictionary SimulationStats::writeSnapshot(Dumper&) const {
    return {
        {"__category", "SimulationPlugin"},
        {"__type",     "SimulationStats"},
        {"fetchEvery", fetchEvery},
    };
}

PostprocessStats::PostprocessStats(std::string name, std::string filename) :
    PostprocessPlugin(name),
    filename(filename)
{
    if (filename != "")
    {
        auto status = fdump.open(filename, "w");
        if (status != FileWrapper::Status::Success)
            die("Could not open file '%s'", filename.c_str());

        fprintf(fdump.get(), "# time  kBT  vx vy vz  max(abs(v)) num_particles simulation_time_per_step(ms)\n");
    }
}

void PostprocessStats::deserialize()
{
    MirState::TimeType currentTime;
    MirState::StepType currentTimeStep;
    real realTime;
    Stats::CountType nparticles, maxNparticles, minNparticles;

    std::vector<Stats::ReductionType> momentum, energy;
    std::vector<real> maxvel;

    SimpleSerializer::deserialize(data, realTime, currentTime, currentTimeStep, nparticles, momentum, energy, maxvel);

    MPI_Check( MPI_Reduce(&nparticles, &minNparticles, 1, getMPIIntType<Stats::CountType>(), MPI_MIN, 0, comm) );
    MPI_Check( MPI_Reduce(&nparticles, &maxNparticles, 1, getMPIIntType<Stats::CountType>(), MPI_MAX, 0, comm) );
    
    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &nparticles,     &nparticles,     1, getMPIIntType<Stats::CountType>(),       MPI_SUM, 0, comm) );
    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : energy.data(),   energy.data(),   1, getMPIFloatType<Stats::ReductionType>(), MPI_SUM, 0, comm) );
    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : momentum.data(), momentum.data(), 3, getMPIFloatType<Stats::ReductionType>(), MPI_SUM, 0, comm) );

    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : maxvel.data(),   maxvel.data(),   1, getMPIFloatType<real>(), MPI_MAX, 0, comm) );

    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &realTime,       &realTime,       1, getMPIFloatType<real>(), MPI_MAX, 0, comm) );

    if (rank == 0)
    {
        const double invNparticles = nparticles > 0 ? 1.0 / nparticles : 0.0;
        momentum[0] *= invNparticles;
        momentum[1] *= invNparticles;
        momentum[2] *= invNparticles;
        const Stats::ReductionType temperature = energy[0] * invNparticles * (2.0/3.0);

        printf("Stats at timestep %lld (simulation time %f):\n", currentTimeStep, currentTime);
        printf("\tOne timestep takes %.2f ms", realTime);
        printf("\tNumber of particles (total, min/proc, max/proc): %llu,  %llu,  %llu\n", nparticles, minNparticles, maxNparticles);
        printf("\tAverage momentum: [%e %e %e]\n", momentum[0], momentum[1], momentum[2]);
        printf("\tMax velocity magnitude: %f\n", maxvel[0]);
        printf("\tTemperature: %.4f\n\n", temperature);

        if (fdump.get())
        {
            fprintf(fdump.get(), "%g %g %g %g %g %g %llu %g\n", currentTime,
                    temperature, momentum[0], momentum[1], momentum[2],
                    maxvel[0], nparticles, realTime);
            fflush(fdump.get());
        }
    }
}

ConfigDictionary PostprocessStats::writeSnapshot(Dumper&) const {
    return {
        {"__category", "PostprocessPlugin"},
        {"__type",     "PostprocessStats"},
        {"filename",   filename},
    };
}

} // namespace mirheo
