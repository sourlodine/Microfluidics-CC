#include "hip/hip_runtime.h"
#include "wall_force_collector.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/datatypes.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/walls/interface.h>

namespace mirheo
{

namespace WallForceCollector
{
__global__ void totalForce(PVview view, double3 *totalForce)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    real3 f {0._r, 0._r, 0._r};
    
    if (tid < view.size)
        f = make_real3(view.forces[tid]);

    f = warpReduce(f, [](real a, real b) { return a + b; });

    if (laneId() == 0)
        atomicAdd(totalForce, make_double3(f));
}
} //namespace WallForceCollector


WallForceCollectorPlugin::WallForceCollectorPlugin(const MirState *state, std::string name,
                                                   std::string wallName, std::string frozenPvName,
                                                   int sampleEvery, int dumpEvery) :
    SimulationPlugin(state, name),
    sampleEvery(sampleEvery),
    dumpEvery(dumpEvery),
    wallName(wallName),
    frozenPvName(frozenPvName)
{}

WallForceCollectorPlugin::~WallForceCollectorPlugin() = default;


void WallForceCollectorPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    wall = dynamic_cast<SDF_basedWall*>(simulation->getWallByNameOrDie(wallName));

    if (wall == nullptr)
        die("Plugin '%s' expects a SDF based wall (got '%s')\n", name.c_str(), wallName.c_str());

    pv = simulation->getPVbyNameOrDie(frozenPvName);

    bounceForceBuffer = wall->getCurrentBounceForce();
}

void WallForceCollectorPlugin::afterIntegration(hipStream_t stream)
{   
    if (isTimeEvery(getState(), sampleEvery))
    {
        pvForceBuffer.clear(stream);

        PVview view(pv, pv->local());
        const int nthreads = 128;

        SAFE_KERNEL_LAUNCH(
            WallForceCollector::totalForce,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, pvForceBuffer.devPtr() );

        pvForceBuffer     .downloadFromDevice(stream);
        bounceForceBuffer->downloadFromDevice(stream);

        totalForce += pvForceBuffer[0];
        totalForce += (*bounceForceBuffer)[0];

        ++nsamples;
    }
    
    needToDump = (isTimeEvery(getState(), dumpEvery) && nsamples > 0);
}

void WallForceCollectorPlugin::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (needToDump)
    {
        waitPrevSend();
        SimpleSerializer::serialize(sendBuffer, getState()->currentTime, nsamples, totalForce);
        send(sendBuffer);
        needToDump = false;
        nsamples   = 0;
        totalForce = make_double3(0, 0, 0);
    }
}

WallForceDumperPlugin::WallForceDumperPlugin(std::string name, std::string filename) :
    PostprocessPlugin(name)
{
    auto status = fdump.open(filename, "w");
    if (status != FileWrapper::Status::Success)
        die("Could not open file '%s'", filename.c_str());
}

void WallForceDumperPlugin::deserialize()
{
    MirState::TimeType currentTime;
    int nsamples;
    double localForce[3], totalForce[3] = {0.0, 0.0, 0.0};

    SimpleSerializer::deserialize(data, currentTime, nsamples, localForce);
    
    MPI_Check( MPI_Reduce(localForce, totalForce, 3, MPI_DOUBLE, MPI_SUM, 0, comm) );

    if (rank == 0)
    {
        totalForce[0] /= (double)nsamples;
        totalForce[1] /= (double)nsamples;
        totalForce[2] /= (double)nsamples;

        fprintf(fdump.get(), "%g %g %g %g\n",
                currentTime, totalForce[0], totalForce[1], totalForce[2]);
        fflush(fdump.get());
    }
}

} // namespace mirheo
