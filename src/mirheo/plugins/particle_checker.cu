#include "hip/hip_runtime.h"
#include "particle_checker.h"
#include "utils/time_stamp.h"

#include <mirheo/core/datatypes.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/rigid_object_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/pvs/views/rov.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/types/str.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/strprintf.h>

namespace mirheo
{

namespace ParticleCheckerKernels
{
template<typename R3>
__device__ static inline bool isFinite(R3 v)
{
    return isfinite(v.x) && isfinite(v.y) && isfinite(v.z);
}

template<typename R3>
__device__ static inline bool withinBounds(R3 v, real3 bounds)
{
    return
        (math::abs(v.x) < bounds.x) &&
        (math::abs(v.y) < bounds.y) &&
        (math::abs(v.z) < bounds.z);
}

__device__ static inline void setBadStatus(int pid, ParticleCheckerPlugin::Info info, ParticleCheckerPlugin::ParticleStatus *status)
{
    const auto tag = atomicExch(&status->tag, ParticleCheckerPlugin::BadTag);

    if (tag == ParticleCheckerPlugin::GoodTag)
    {
        status->id   = pid;
        status->info = info;
    }
}

__global__ void checkForces(PVview view, ParticleCheckerPlugin::ParticleStatus *status)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;

    if (pid >= view.size) return;

    const auto force = make_real3(view.forces[pid]);

    if (!isFinite(force))
        setBadStatus(pid, ParticleCheckerPlugin::Info::Nan, status);
}

__global__ void checkParticles(PVview view, DomainInfo domain, real dtInv, ParticleCheckerPlugin::ParticleStatus *status)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;

    if (pid >= view.size) return;

    const auto pos = make_real3(view.readPosition(pid));
    const auto vel = make_real3(view.readVelocity(pid));

    if (!isFinite(pos) || !isFinite(vel))
    {
        setBadStatus(pid, ParticleCheckerPlugin::Info::Nan, status);
        return;
    }

    const real3 boundsPos = 1.5_r * domain.localSize; // particle should not be further than one neighbouring domain
    const real3 boundsVel = dtInv * domain.localSize; // particle should not travel more than one domain size per iteration

    if (!withinBounds(pos, boundsPos) || !withinBounds(vel, boundsVel))
    {
        setBadStatus(pid, ParticleCheckerPlugin::Info::Out, status);
        return;
    }
}

__global__ void checkRigidForces(ROVview view, ParticleCheckerPlugin::ParticleStatus *status)
{
    const int objId = blockIdx.x * blockDim.x + threadIdx.x;

    if (objId >= view.nObjects) return;

    const auto m = view.motions[objId];

    if (!isFinite(m.force) || !isFinite(m.torque))
        setBadStatus(objId, ParticleCheckerPlugin::Info::Nan, status);
}

__global__ void checkRigidMotions(ROVview view, DomainInfo domain, real dtInv, ParticleCheckerPlugin::ParticleStatus *status)
{
    const int objId = blockIdx.x * blockDim.x + threadIdx.x;

    if (objId >= view.nObjects) return;

    const auto m = view.motions[objId];

    if (!isFinite(m.r) || !isFinite(m.vel) || !isFinite(m.omega))
    {
        setBadStatus(objId, ParticleCheckerPlugin::Info::Nan, status);
        return;
    }

    const real3 boundsPos   = 1.5_r * domain.localSize; // objects should not be further than one neighbouring domain
    const real3 boundsVel   = dtInv * domain.localSize; // objects should not travel more than one domain size per iteration
    const real3 boundsOmega = make_real3(dtInv * M_PI); // objects should not rotate more than half a turn per iteration

    if (!withinBounds(m.r, boundsPos) || !withinBounds(m.vel, boundsVel), !withinBounds(m.omega, boundsOmega))
    {
        setBadStatus(objId, ParticleCheckerPlugin::Info::Out, status);
        return;
    }
}

} // namespace ParticleCheckerKernels

constexpr int ParticleCheckerPlugin::NotRov_;

ParticleCheckerPlugin::ParticleCheckerPlugin(const MirState *state, std::string name, int checkEvery) :
    SimulationPlugin(state, name),
    checkEvery_(checkEvery)
{}

ParticleCheckerPlugin::~ParticleCheckerPlugin() = default;

void ParticleCheckerPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    pvs_ = simulation->getParticleVectors();
    rovStatusIds_.clear();
    
    int numRovs {0};
    for (auto pv : pvs_)
    {
        int id = NotRov_;
        if (dynamic_cast<RigidObjectVector*>(pv))
        {
            id = pvs_.size() + numRovs++;
        }
        rovStatusIds_.push_back(id);
    }

    statuses_.resize_anew(pvs_.size() + numRovs);

    for (auto& s : statuses_)
        s = {GoodTag, 0, Info::Ok};
    statuses_.uploadToDevice(defaultStream);
}

void ParticleCheckerPlugin::beforeIntegration(hipStream_t stream)
{
    if (!isTimeEvery(getState(), checkEvery_)) return;

    constexpr int nthreads = 128;
    
    for (size_t i = 0; i < pvs_.size(); ++i)
    {
        auto pv = pvs_[i];
        PVview view(pv, pv->local());

        SAFE_KERNEL_LAUNCH(
            ParticleCheckerKernels::checkForces,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, statuses_.devPtr() + i );

        if (auto rov = dynamic_cast<RigidObjectVector*>(pv))
        {
            ROVview view(rov, rov->local());

            SAFE_KERNEL_LAUNCH(
                ParticleCheckerKernels::checkRigidForces,
                getNblocks(view.nObjects, nthreads), nthreads, 0, stream,
                view, statuses_.devPtr() + rovStatusIds_[i] );
        }
    }

    dieIfBadStatus(stream, "force");
}

void ParticleCheckerPlugin::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(getState(), checkEvery_)) return;

    constexpr int nthreads = 128;

    const real dt     = getState()->dt;
    const real dtInv  = 1.0_r / math::max(1e-6_r, dt);
    const auto domain = getState()->domain;
    
    for (size_t i = 0; i < pvs_.size(); ++i)
    {
        auto pv = pvs_[i];
        PVview view(pv, pv->local());

        SAFE_KERNEL_LAUNCH(
            ParticleCheckerKernels::checkParticles,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, domain, dtInv, statuses_.devPtr() + i );

        if (auto rov = dynamic_cast<RigidObjectVector*>(pv))
        {
            ROVview view(rov, rov->local());

            SAFE_KERNEL_LAUNCH(
                ParticleCheckerKernels::checkRigidMotions,
                getNblocks(view.nObjects, nthreads), nthreads, 0, stream,
                view, domain, dtInv, statuses_.devPtr() + rovStatusIds_[i] );
        }
    }

    dieIfBadStatus(stream, "particle");
}

static inline void downloadAllFields(hipStream_t stream, const DataManager& manager)
{
    for (auto entry : manager.getSortedChannels())
    {
        auto desc = entry.second;
        mpark::visit([stream](auto pinnedBuffPtr)
        {
            pinnedBuffPtr->downloadFromDevice(stream, ContainersSynch::Asynch);
        }, desc->varDataPtr);
    }
    CUDA_Check( hipStreamSynchronize(stream) );
}

static inline std::string listOtherFieldValues(const DataManager& manager, int id)
{
    std::string fieldValues;
    
    for (auto entry : manager.getSortedChannels())
    {
        const auto& name = entry.first;
        const auto desc = entry.second;
            
        if (name == ChannelNames::positions ||
            name == ChannelNames::velocities)
            continue;
            
        mpark::visit([&](auto pinnedBuffPtr)
        {
            const auto val = (*pinnedBuffPtr)[id];
            fieldValues += '\t' + name + " : " + printToStr(val) + '\n';
        }, desc->varDataPtr);
    }
    return fieldValues;    
}

static inline std::string infoToStr(ParticleCheckerPlugin::Info info)
{
    using Info = ParticleCheckerPlugin::Info;
    if (info == Info::Nan) return "not a finite number";
    if (info == Info::Out) return "out of bounds";
    return "no error detected";
}

void ParticleCheckerPlugin::dieIfBadStatus(hipStream_t stream, const std::string& identifier)
{
    statuses_.downloadFromDevice(stream, ContainersSynch::Synch);
    const auto domain = getState()->domain;

    bool failing {false};
    std::string allErrors;

    for (size_t i = 0; i < pvs_.size(); ++i)
    {
        const auto& partStatus = statuses_[i];
        if (partStatus.tag == GoodTag) continue;

        const int partId = partStatus.id;

        // from now we know we will fail; download particles and print error
        auto pv = pvs_[i];
        auto lpv = pv->local();

        downloadAllFields(stream, lpv->dataPerParticle);

        const auto p = Particle(lpv->positions ()[partId],
                                lpv->velocities()[partId]);

        const auto infoStr = infoToStr(partStatus.info);

        const real3 lr = p.r;
        const real3 gr = domain.local2global(lr);

        allErrors += strprintf("\n\tBad %s in '%s' with id %ld, local position %g %g %g, global position %g %g %g, velocity %g %g %g : %s\n",
                               identifier.c_str(),
                               pv->getCName(), p.getId(),
                               lr.x, lr.y, lr.z, gr.x, gr.y, gr.z,
                               p.u.x, p.u.y, p.u.z, infoStr.c_str());

        allErrors += listOtherFieldValues(lpv->dataPerParticle, partId);
        
        failing = true;
    }

    for (size_t i = 0; i < pvs_.size(); ++i)
    {
        const int rovSId = rovStatusIds_[i];
        if (rovSId == NotRov_) continue;
        
        const auto& rovStatus = statuses_[rovSId];
        if (rovStatus.tag == GoodTag) continue;

        const int rovId = rovStatus.id;

        // from now we know we will fail; download particles and print error
        auto rov = dynamic_cast<RigidObjectVector*>(pvs_[i]);
        auto lrov = rov->local();

        downloadAllFields(stream, lrov->dataPerObject);

        const auto infoStr = infoToStr(rovStatus.info);
        
        allErrors += strprintf("\n\tBad %s in rov '%s' : %s\n",
                               identifier.c_str(), rov->getCName(), infoStr.c_str());

        allErrors += listOtherFieldValues(lrov->dataPerObject, rovId);
        
        failing = true;
    }

    if (failing)
        die("Particle checker has found bad particles: %s", allErrors.c_str());
}

} // namespace mirheo
