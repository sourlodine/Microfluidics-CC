#include "hip/hip_runtime.h"
#include "add_torque.h"

#include <mirheo/core/pvs/rigid_object_vector.h>
#include <mirheo/core/pvs/views/rov.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>

namespace AddTorqueKernels
{

__global__ void addTorque(ROVview view, real3 torque)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.nObjects) return;

    view.motions[gid].torque += torque;
}

} // namespace AddTorqueKernels

AddTorquePlugin::AddTorquePlugin(const MirState *state, std::string name, std::string rovName, real3 torque) :
    SimulationPlugin(state, name),
    rovName(rovName),
    torque(torque)
{}

void AddTorquePlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    rov = dynamic_cast<RigidObjectVector*>( simulation->getOVbyNameOrDie(rovName) );
    if (rov == nullptr)
        die("Need rigid object vector to add torque, plugin '%s', OV name '%s'",
            name.c_str(), rovName.c_str());

    info("Objects '%s' will experience external torque [%f %f %f]", 
            rovName.c_str(), torque.x, torque.y, torque.z);
}

void AddTorquePlugin::beforeForces(hipStream_t stream)
{
    ROVview view(rov, rov->local());
    const int nthreads = 128;

    SAFE_KERNEL_LAUNCH(
            AddTorqueKernels::addTorque,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, torque );
}

