#include <core/pvs/membrane_vector.h>
#include <core/pvs/views/ov.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

#include "membrane_juelicher.h"
#include "membrane/bending_juelicher.h"

InteractionMembraneJuelicher::InteractionMembraneJuelicher(const YmrState *state, std::string name,
                                                           MembraneParameters parameters,
                                                           JuelicherBendingParameters bendingParameters,
                                                           bool stressFree, float growUntil) :
    InteractionMembrane(state, name, parameters, stressFree, growUntil),
    bendingParameters(bendingParameters)
{}


InteractionMembraneJuelicher::~InteractionMembraneJuelicher() = default;
    
void InteractionMembraneJuelicher::setPrerequisites(ParticleVector* pv1, ParticleVector* pv2)
{
    InteractionMembrane::setPrerequisites(pv1, pv2);

    auto ov = dynamic_cast<MembraneVector*>(pv1);
    
    ov->requireDataPerObject<float>("lenThetaTot", ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);

    ov->requireDataPerParticle<float>("areas", ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
    ov->requireDataPerParticle<float>("meanCurvatures", ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
}


static bendingJuelicher::GPU_BendingParams setJuelicherBendingParams(float scale, JuelicherBendingParameters& p)
{
    bendingJuelicher::GPU_BendingParams devP;

    devP.kb     = p.kb  * scale*scale;
    devP.kad_pi = p.kad * M_PI * scale*scale;

    devP.H0  = p.C0 / (2 * scale);
    devP.DA0 = p.DA0 * scale*scale;

    return devP;
}

void InteractionMembraneJuelicher::bendingForces(float scale, MembraneVector *ov, MembraneMeshView mesh, hipStream_t stream)
{
    ov->local()->extraPerObject.getData<float>("lenThetaTot")->clearDevice(stream);

    OVviewWithJuelicherQuants view(ov, ov->local());
    
    const int nthreads = 128;    

    {
        dim3 threads(nthreads, 1);
        dim3 blocks(getNblocks(mesh.nvertices, nthreads), view.nObjects);
        
        SAFE_KERNEL_LAUNCH(
            bendingJuelicher::computeAreasAndCurvatures,
            blocks, threads, 0, stream,
            view, mesh );
    }

    {
        auto devParams = setJuelicherBendingParams(scale, bendingParameters);
        
        const int blocks = getNblocks(view.size, nthreads);
    
        SAFE_KERNEL_LAUNCH(
            bendingJuelicher::computeBendingForces,
            blocks, nthreads, 0, stream,
            view, mesh, devParams );
    }
}
