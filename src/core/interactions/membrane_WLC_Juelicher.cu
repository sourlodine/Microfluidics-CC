#include "hip/hip_runtime.h"
#include "membrane_WLC_Juelicher.h"
#include "membrane.impl.h"
#include "membrane/common.h"
#include "membrane/dihedral/juelicher.h"
#include "membrane/triangle/wlc.h"

#include <core/utils/make_unique.h>

namespace InteractionMembraneJuelicherKernels
{
__device__ inline float compute_lenTheta(float3 v0, float3 v1, float3 v2, float3 v3)
{
    float len = length(v2 - v0);
    float theta = supplementaryDihedralAngle(v0, v1, v2, v3);
    return len * theta;
}

__global__ void computeAreasAndCurvatures(OVviewWithJuelicherQuants view, MembraneMeshView mesh)
{
    int rbcId = blockIdx.y;
    int idv0  = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = rbcId * mesh.nvertices;

    float lenTheta = 0;
    
    if (idv0 < mesh.nvertices)
    {        
        int startId = mesh.maxDegree * idv0;
        int degree = mesh.degrees[idv0];
        
        int idv1 = mesh.adjacent[startId];
        int idv2 = mesh.adjacent[startId+1];
        
        float3 v0 = fetchPosition(view, offset + idv0);
        float3 v1 = fetchPosition(view, offset + idv1);
        float3 v2 = fetchPosition(view, offset + idv2);
        
        float area = 0;    
        
#pragma unroll 2
        for (int i = 0; i < degree; i++) {
            
            int idv3 = mesh.adjacent[startId + (i+2) % degree];
            float3 v3 = fetchPosition(view, offset + idv3);
            
            area     += 0.3333333f * triangleArea(v0, v1, v2);
            lenTheta += compute_lenTheta(v0, v1, v2, v3);
            
            v1 = v2;
            v2 = v3;
        }
        
        view.vertexAreas          [offset + idv0] = area;
        view.vertexMeanCurvatures [offset + idv0] = lenTheta / (4 * area);
    }
    
    lenTheta = warpReduce( lenTheta, [] (float a, float b) { return a+b; } );

    if (__laneid() == 0)
        atomicAdd(&view.lenThetaTot[rbcId], lenTheta);
}
} // namespace InteractionMembraneJuelicherKernels

InteractionMembraneWLCJuelicher::InteractionMembraneWLCJuelicher(const YmrState *state, std::string name,
                                                                 MembraneParameters parameters, JuelicherBendingParameters juelicherParams,
                                                                 bool stressFree, float growUntil) :
    InteractionMembrane(state, name)
{
    // TODO
    WLCParameters wlc;
    wlc.x0       = parameters.x0;
    wlc.ks       = parameters.ks;
    wlc.mpow     = parameters.mpow;
    wlc.ka       = parameters.ka;
    wlc.kd       = parameters.kd;
    wlc.totArea0 = parameters.totArea0;

    impl = std::make_unique<InteractionMembraneImpl<TriangleWLC, DihedralJuelicher>>
        (state, name, parameters, wlc, juelicherParams, stressFree, growUntil);
}

InteractionMembraneWLCJuelicher::~InteractionMembraneWLCJuelicher() = default;

void InteractionMembraneWLCJuelicher::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    InteractionMembrane::setPrerequisites(pv1, pv2, cl1, cl2);

    auto ov = dynamic_cast<MembraneVector*>(pv1);
    
    ov->requireDataPerObject<float>(ChannelNames::lenThetaTot, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);

    ov->requireDataPerParticle<float>(ChannelNames::areas, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
    ov->requireDataPerParticle<float>(ChannelNames::meanCurvatures, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
}

void InteractionMembraneWLCJuelicher::precomputeQuantities(ParticleVector *pv1, hipStream_t stream)
{
    auto ov = dynamic_cast<MembraneVector *>(pv1);

    debug("Computing vertex areas and curvatures for %d cells of '%s'",
          ov->local()->nObjects, ov->name.c_str());

    OVviewWithJuelicherQuants view(ov, ov->local());

    MembraneMeshView mesh(static_cast<MembraneMesh*>(ov->mesh.get()));

    const int nthreads = 128;    

    dim3 threads(nthreads, 1);
    dim3 blocks(getNblocks(mesh.nvertices, nthreads), view.nObjects);
        
    SAFE_KERNEL_LAUNCH(
        InteractionMembraneJuelicherKernels::computeAreasAndCurvatures,
        blocks, threads, 0, stream,
        view, mesh );
}
