#include "rod.h"
#include "rod.impl.h"


InteractionRod::InteractionRod(const YmrState *state, std::string name, RodParameters parameters) :
    Interaction(state, name, /*rc*/ 1.f)
{
    impl = std::make_unique<InteractionRodImpl>(state, name, parameters);
}

InteractionRod::~InteractionRod() = default;

void InteractionRod::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    if (pv1 != pv2)
        die("Internal rod forces can't be computed between two different particle vectors");

    auto rv = dynamic_cast<RodVector*>(pv1);
    if (rv == nullptr)
        die("Internal rod forces can only be computed with a RodVector");

    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}

void InteractionRod::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    if (impl.get() == nullptr)
        die("%s needs a concrete implementation, none was provided", name.c_str());

    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionRod::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    debug("Not computing internal rod forces between local and halo rods of '%s'", pv1->name.c_str());
}

bool InteractionRod::isSelfObjectInteraction() const
{
    return true;
}
