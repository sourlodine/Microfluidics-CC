#include "hip/hip_runtime.h"
#include "membrane.new.h"

#include "membrane/common.h"

#include <core/pvs/membrane_vector.h>
#include <core/pvs/views/ov.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>


namespace MembraneKernels
{
__global__ void computeAreaAndVolume(OVviewWithAreaVolume view, MeshView mesh)
{
    int objId = blockIdx.x;
    int offset = objId * mesh.nvertices;
    float2 a_v = make_float2(0.0f);

    for(int i = threadIdx.x; i < mesh.ntriangles; i += blockDim.x) {        
        int3 ids = mesh.triangles[i];

        float3 v0 = f4tof3( view.particles[ 2 * (offset + ids.x) ] );
        float3 v1 = f4tof3( view.particles[ 2 * (offset + ids.y) ] );
        float3 v2 = f4tof3( view.particles[ 2 * (offset + ids.z) ] );

        a_v.x += triangleArea(v0, v1, v2);
        a_v.y += triangleSignedVolume(v0, v1, v2);
    }

    a_v = warpReduce( a_v, [] (float a, float b) { return a+b; } );

    if (__laneid() == 0)
        atomicAdd(&view.area_volumes[objId], a_v);
}
} // namespace MembraneKernels

InteractionMembraneNew::InteractionMembraneNew(const YmrState *state, std::string name) :
    Interaction(state, name, /* default cutoff rc */ 1.0),
    impl(nullptr)
{}

InteractionMembraneNew::~InteractionMembraneNew() = default;

void InteractionMembraneNew::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    if (pv1 != pv2)
        die("Internal membrane forces can't be computed between two different particle vectors");

    auto ov = dynamic_cast<MembraneVector*>(pv1);
    if (ov == nullptr)
        die("Internal membrane forces can only be computed with a MembraneVector");

    ov->requireDataPerObject<float2>(ChannelNames::areaVolumes,
                                     ExtraDataManager::CommunicationMode::None,
                                     ExtraDataManager::PersistenceMode::None);
}

void InteractionMembraneNew::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    precomputeQuantities(pv1, stream);
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionMembraneNew::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    debug("Not computing internal membrane forces between local and halo membranes of '%s'",
          pv1->name.c_str());
}
    
void InteractionMembraneNew::precomputeQuantities(ParticleVector *pv1, hipStream_t stream)
{
    auto ov = dynamic_cast<MembraneVector *>(pv1);

    if (ov->objSize != ov->mesh->getNvertices())
        die("Object size of '%s' (%d) and number of vertices (%d) mismatch",
            ov->name.c_str(), ov->objSize, ov->mesh->getNvertices());

    debug("Computing areas and volumes for %d cells of '%s'",
          ov->local()->nObjects, ov->name.c_str());

    OVviewWithAreaVolume view(ov, ov->local());

    MembraneMeshView mesh(static_cast<MembraneMesh*>(ov->mesh.get()));

    ov->local()
        ->extraPerObject.getData<float2>(ChannelNames::areaVolumes)
        ->clearDevice(stream);
    
    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(MembraneKernels::computeAreaAndVolume,
                       view.nObjects, nthreads, 0, stream,
                       view, mesh);
}
