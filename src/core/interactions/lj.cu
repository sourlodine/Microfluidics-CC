#include "lj.h"
#include "pairwise.h"
#include "pairwise_interactions/lj.h"
#include "pairwise_interactions/lj_object_aware.h"

#include <core/celllist.h>

#include <memory>

InteractionLJ::InteractionLJ(const YmrState *state, std::string name, float rc, float epsilon, float sigma, float maxForce, bool objectAware, bool allocate) :
    Interaction(state, name, rc),
    objectAware(objectAware)
{
    if (!allocate) return;

    if (objectAware) {
        Pairwise_LJObjectAware lj(rc, epsilon, sigma, maxForce);
        impl = std::make_unique<InteractionPair<Pairwise_LJObjectAware>> (state, name, rc, lj);
    }
    else {
        Pairwise_LJ lj(rc, epsilon, sigma, maxForce);
        impl = std::make_unique<InteractionPair<Pairwise_LJ>> (state, name, rc, lj);
    }
}

InteractionLJ::InteractionLJ(const YmrState *state, std::string name, float rc, float epsilon, float sigma, float maxForce, bool objectAware) :
    InteractionLJ(state, name, rc, epsilon, sigma, maxForce, objectAware, true)
{}

InteractionLJ::~InteractionLJ() = default;

void InteractionLJ::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}

std::vector<Interaction::InteractionChannel> InteractionLJ::getFinalOutputChannels() const
{
    return impl->getFinalOutputChannels();
}

void InteractionLJ::local(ParticleVector *pv1, ParticleVector *pv2,
                          CellList *cl1, CellList *cl2,
                          hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionLJ::halo(ParticleVector *pv1, ParticleVector *pv2,
                         CellList *cl1, CellList *cl2,
                         hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionLJ::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
                                    float epsilon, float sigma, float maxForce)
{
    if (objectAware) {
        Pairwise_LJObjectAware lj(rc, epsilon, sigma, maxForce);
        auto ptr = static_cast< InteractionPair<Pairwise_LJObjectAware>* >(impl.get());
        ptr->setSpecificPair(pv1->name, pv2->name, lj);
    }
    else {
        Pairwise_LJ lj(rc, epsilon, sigma, maxForce);
        auto ptr = static_cast< InteractionPair<Pairwise_LJ>* >(impl.get());
        ptr->setSpecificPair(pv1->name, pv2->name, lj);
    }
}

