#include "lj.h"
#include <memory>
#include "pairwise.h"
#include "pairwise_interactions/lj.h"
#include "pairwise_interactions/lj_object_aware.h"


InteractionLJ::InteractionLJ(const YmrState *state, std::string name, float rc, float epsilon, float sigma, float maxForce, bool objectAware, bool allocate) :
    Interaction(state, name, rc),
    objectAware(objectAware)
{
    if (!allocate) return;

    if (objectAware) {
        Pairwise_LJObjectAware lj(rc, epsilon, sigma, maxForce);
        impl = std::make_unique<InteractionPair<Pairwise_LJObjectAware>> (state, name, rc, lj);
    }
    else {
        Pairwise_LJ lj(rc, epsilon, sigma, maxForce);
        impl = std::make_unique<InteractionPair<Pairwise_LJ>> (state, name, rc, lj);
    }
}

InteractionLJ::InteractionLJ(const YmrState *state, std::string name, float rc, float epsilon, float sigma, float maxForce, bool objectAware) :
    InteractionLJ(state, name, rc, epsilon, sigma, maxForce, objectAware, true)
{}

InteractionLJ::~InteractionLJ() = default;

void InteractionLJ::setPrerequisites(ParticleVector* pv1, ParticleVector* pv2)
{
    impl->setPrerequisites(pv1, pv2);
}

void InteractionLJ::initStep(ParticleVector *pv1, ParticleVector *pv2, hipStream_t stream)
{
    impl->initStep(pv1, pv2, stream);
}

void InteractionLJ::regular(ParticleVector *pv1, ParticleVector *pv2,
                            CellList *cl1, CellList *cl2,
                            hipStream_t stream)
{
    impl->regular(pv1, pv2, cl1, cl2, stream);
}

void InteractionLJ::halo(ParticleVector *pv1, ParticleVector *pv2,
                         CellList *cl1, CellList *cl2,
                         hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionLJ::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
                                    float epsilon, float sigma, float maxForce)
{
    if (objectAware) {
        Pairwise_LJObjectAware lj(rc, epsilon, sigma, maxForce);
        auto ptr = static_cast< InteractionPair<Pairwise_LJObjectAware>* >(impl.get());
        ptr->setSpecificPair(pv1->name, pv2->name, lj);
    }
    else {
        Pairwise_LJ lj(rc, epsilon, sigma, maxForce);
        auto ptr = static_cast< InteractionPair<Pairwise_LJ>* >(impl.get());
        ptr->setSpecificPair(pv1->name, pv2->name, lj);
    }
}

