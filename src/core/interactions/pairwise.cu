#include "pairwise.h"
#include "pairwise/factory_helper.h"
#include "pairwise/impl.h"
#include "pairwise/impl.stress.h"
#include "pairwise/kernels/density.h"
#include "pairwise/kernels/density_kernels.h"
#include "pairwise/kernels/dpd.h"
#include "pairwise/kernels/lj.h"
#include "pairwise/kernels/mdpd.h"
#include "pairwise/kernels/pressure_EOS.h"
#include "pairwise/kernels/sdpd.h"
#include "pairwise/kernels/type_traits.h"

#include <memory>

template <class KernelType>
static std::unique_ptr<Interaction>
createPairwiseFromKernel(const MirState *state, const std::string& name, float rc,
                         const typename std::enable_if<outputsForce<KernelType>::value, KernelType>::type& kernel,
                         const VarStressParams& varStressParams)
{
    if (mpark::holds_alternative<StressActiveParams>(varStressParams))
    {
        const auto stressParams = mpark::get<StressActiveParams>(varStressParams);
        return std::make_unique<InteractionPair_withStress<KernelType>>(state, name, rc, stressParams.period, kernel);
    }
    else
    {
        return std::make_unique<InteractionPair<KernelType>>(state, name, rc, kernel);
    }
}

template <class KernelType>
static std::unique_ptr<Interaction>
createPairwiseFromKernel(const MirState *state, const std::string& name, float rc, const KernelType& kernel, const VarStressParams& varStressParams)
{
    if (mpark::holds_alternative<StressActiveParams>(varStressParams))
        die("Incompatible interaction output: '%s' can not output stresses.", name.c_str());
    
    return std::make_unique<InteractionPair<KernelType>>(state, name, rc, kernel);
}


template <class Parameters>
static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, float rc, const Parameters& params, const VarStressParams& varStressParams)
{
    using KernelType = typename Parameters::KernelType;
    KernelType kernel(rc, params, state->dt);

    return createPairwiseFromKernel(state, name, rc, kernel, varStressParams);
}


std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, float rc, const LJParams& params, const VarStressParams& varStressParams)
{
    return mpark::visit([&](auto& awareParams)
    {
        using AwareType = typename std::remove_reference<decltype(awareParams)>::type::KernelType;
        
        AwareType awareness(awareParams);
        PairwiseLJ<AwareType> lj(rc, params.epsilon, params.sigma, params.maxForce, awareness);

        return createPairwiseFromKernel(state, name, rc, lj, varStressParams);
    }, params.varLJAwarenessParams);
}

static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, float rc, const DensityParams& params, const VarStressParams& varStressParams)
{
    return mpark::visit([&](auto& densityKernelParams)
    {
        using DensityKernelType = typename std::remove_reference<decltype(densityKernelParams)>::type::KernelType;
        
        DensityKernelType densityKernel;
        PairwiseDensity<DensityKernelType> density(rc, densityKernel);

        return createPairwiseFromKernel(state, name, rc, density, varStressParams);
    }, params.varDensityKernelParams);
}

static std::unique_ptr<Interaction>
createPairwiseFromParams(const MirState *state, const std::string& name, float rc, const SDPDParams& params, const VarStressParams& varStressParams)
{
    return mpark::visit([&](auto& densityKernelParams, auto& EOSParams)
    {
        using DensityKernelType = typename std::remove_reference<decltype(densityKernelParams)>::type::KernelType;
        using EOSKernelType     = typename std::remove_reference<decltype(EOSParams          )>::type::KernelType;
        
        DensityKernelType density;
        EOSKernelType pressure(EOSParams);

        PairwiseSDPD<EOSKernelType, DensityKernelType> sdpd(rc, pressure, density, params.viscosity, params.kBT, state->dt);
        
        return createPairwiseFromKernel(state, name, rc, sdpd, varStressParams);
    }, params.varDensityKernelParams, params.varEOSParams);
}


PairwiseInteraction::PairwiseInteraction(const MirState *state, const std::string& name, float rc,
                                         const VarPairwiseParams& varParams, const VarStressParams& varStressParams) :
    Interaction(state, name, rc),
    varParams(varParams),
    varStressParams(varStressParams)
{
    impl = mpark::visit([&](const auto& params)
    {
        return createPairwiseFromParams(state, name, rc, params, varStressParams);
    }, varParams);
}

PairwiseInteraction::~PairwiseInteraction() = default;

void PairwiseInteraction::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}
    
void PairwiseInteraction::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void PairwiseInteraction::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}


Interaction::Stage PairwiseInteraction::getStage() const
{
    return impl->getStage();
}

std::vector<Interaction::InteractionChannel> PairwiseInteraction::getInputChannels() const
{
    return impl->getInputChannels();
}

std::vector<Interaction::InteractionChannel> PairwiseInteraction::getOutputChannels() const
{
    return impl->getOutputChannels();
}

void PairwiseInteraction::checkpoint(MPI_Comm comm, const std::string& path, int checkpointId)
{
    return impl->checkpoint(comm, path, checkpointId);
}

void PairwiseInteraction::restart(MPI_Comm comm, const std::string& path)
{
    return impl->restart(comm, path);
}

void PairwiseInteraction::setSpecificPair(__UNUSED const VarPairwiseParams& varParamsSpecific)
{
    
}
