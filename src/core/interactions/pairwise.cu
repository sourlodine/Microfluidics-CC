#include "hip/hip_runtime.h"
#include "pairwise.h"

#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>
#include <core/celllist.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/logger.h>

#include "pairwise_kernels.h"

#include "pairwise_interactions/stress_wrapper.h"
#include "pairwise_interactions/dpd.h"
#include "pairwise_interactions/lj.h"
#include "pairwise_interactions/lj_object_aware.h"

#include "pairwise_interactions/norandom_dpd.h"


/**
 * Convenience macro wrapper
 *
 * Select one of the available kernels for external interaction depending
 * on the number of particles involved, report it and call
 */
#define DISPATCH_EXTERNAL(P1, P2, P3, TPP, INTERACTION_FUNCTION)                \
do{ debug2("Dispatched to "#TPP" thread(s) per particle variant");              \
    SAFE_KERNEL_LAUNCH(                                                         \
            computeExternalInteractions_##TPP##tpp<P1 COMMA P2 COMMA P3>,       \
            getNblocks(TPP*dstView.size, nth), nth, 0, stream,                  \
            dstView, cl2->cellInfo(), rc*rc, INTERACTION_FUNCTION); } while (0)

#define CHOOSE_EXTERNAL(P1, P2, P3, INTERACTION_FUNCTION)                                              \
do{  if (dstView.size < 1000  ) { DISPATCH_EXTERNAL(P1, P2, P3, 27, INTERACTION_FUNCTION); }           \
else if (dstView.size < 10000 ) { DISPATCH_EXTERNAL(P1, P2, P3, 9,  INTERACTION_FUNCTION); }           \
else if (dstView.size < 400000) { DISPATCH_EXTERNAL(P1, P2, P3, 3,  INTERACTION_FUNCTION); }           \
else                            { DISPATCH_EXTERNAL(P1, P2, P3, 1,  INTERACTION_FUNCTION); } } while(0)

/**
 * Interface to _compute() with local interactions.
 */
template <class PariwiseInteraction>
void InteractionPair<PariwiseInteraction>::regular(ParticleVector *pv1,
                                                   ParticleVector *pv2,
                                                   CellList *cl1, CellList *cl2,
                                                   hipStream_t stream)
{
    // if (pv1->local()->size() < pv2->local()->size())
    _compute(InteractionType::Regular, pv1, pv2, cl1, cl2, state->currentTime, stream);
    // else
    //    _compute(InteractionType::Regular, pv2, pv1, cl2, cl1, state->currentTime, stream);
}

/**
 * Interface to _compute() with halo interactions.
 *
 * The following cases exist:
 * - If one of \p pv1 or \p pv2 is ObjectVector, then only call to the _compute()
 *   needed: for halo ObjectVector another ParticleVector (or ObjectVector).
 *   This is because ObjectVector will collect the forces from remote processors,
 *   so we don't need to compute them twice.
 *
 * - Both are ParticleVector. Then if they are different, two _compute() calls
 *   are made such that halo1 \<-\> local2 and halo2 \<-\> local1. If \p pv1 and
 *   \p pv2 are the same, only one call is needed
 */
template <class PairwiseInteraction>
void InteractionPair<PairwiseInteraction>::halo(ParticleVector *pv1,
                                                ParticleVector *pv2,
                                                CellList *cl1, CellList *cl2,
                                                hipStream_t stream)
{
    auto isov1 = dynamic_cast<ObjectVector *>(pv1) != nullptr;
    auto isov2 = dynamic_cast<ObjectVector *>(pv2) != nullptr;

    float t = state->currentTime;
    
    // Two object vectors. Compute just one interaction, doesn't matter which
    if (isov1 && isov2) {
        _compute(InteractionType::Halo, pv1, pv2, cl1, cl2, t, stream);
        return;
    }

    // One object vector. Compute just one interaction, with OV as the first
    // argument
    if (isov1) {
        _compute(InteractionType::Halo, pv1, pv2, cl1, cl2, t, stream);
        return;
    }

    if (isov2) {
        _compute(InteractionType::Halo, pv2, pv1, cl2, cl1, t, stream);
        return;
    }

    // Both are particle vectors. Compute one interaction if pv1 == pv2 and two
    // otherwise
    _compute(InteractionType::Halo, pv1, pv2, cl1, cl2, t, stream);
    if (pv1 != pv2)
        _compute(InteractionType::Halo, pv2, pv1, cl2, cl1, t, stream);
}

/**
 * Compute forces between all the pairs of particles that are closer
 * than #rc to each other.
 *
 * Depending on \p type and whether \p pv1 == \p pv2 call
 * computeSelfInteractions() or computeExternalInteractions_1tpp()
 * (or other variants of external interaction kernels).
 *
 * @tparam PariwiseInteraction is a functor that computes the force
 * given a pair of particles. It has to
 * provide two functions:
 * - This function will be called once before interactions computation
 *   and allows the functor to obtain required variables or data
 *   channels from the two ParticleVector and CellList:
 *   \code setup(ParticleVector* pv1, ParticleVector* pv2, CellList* cl1, CellList* cl2, float t) \endcode
 *
 * - This should be a \c \_\_device\_\_ operator that computes
 *   the force. It will be called for each close enough particle pair:
 *   \code float3 operator()(const Particle dst, int dstId, const Particle src, int srcId) const \endcode
 *   Return value of that call is force acting on the first particle,
 *   force acting on the second one is just opposite.
 */
template<class PairwiseInteraction>
void InteractionPair<PairwiseInteraction>::_compute(InteractionType type,
        ParticleVector* pv1, ParticleVector* pv2, CellList* cl1, CellList* cl2, const float t, hipStream_t stream)
{
    auto& pair = getPairwiseInteraction(pv1->name, pv2->name);

    if (type == InteractionType::Regular)
    {
        pair.setup(pv1->local(), pv2->local(), cl1, cl2, t);

        /*  Self interaction */
        if (pv1 == pv2)
        {
            const int np = pv1->local()->size();
            debug("Computing internal forces for %s (%d particles)", pv1->name.c_str(), np);

            const int nth = 128;

            auto cinfo = cl1->cellInfo();
            SAFE_KERNEL_LAUNCH(
                    computeSelfInteractions,
                    getNblocks(np, nth), nth, 0, stream,
                    np, cinfo, rc*rc, pair);
        }
        else /*  External interaction */
        {
            const int np1 = pv1->local()->size();
            const int np2 = pv2->local()->size();
            debug("Computing external forces for %s - %s (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), np1, np2);

            auto dstView = cl1->getView<PVview>();

            const int nth = 128;
            if (np1 > 0 && np2 > 0)
                CHOOSE_EXTERNAL(InteractionOut::NeedAcc, InteractionOut::NeedAcc, InteractionMode::RowWise, pair);
        }
    }

    /*  Halo interaction */
    if (type == InteractionType::Halo)
    {
        pair.setup(pv1->halo(), pv2->local(), cl1, cl2, t);

        const int np1 = pv1->halo()->size();  // note halo here
        const int np2 = pv2->local()->size();
        debug("Computing halo forces for %s(halo) - %s (%d - %d particles)", pv1->name.c_str(), pv2->name.c_str(), np1, np2);

        PVview dstView(pv1, pv1->halo());
        const int nth = 128;
        if (np1 > 0 && np2 > 0)
            if (dynamic_cast<ObjectVector*>(pv1) == nullptr) // don't need forces for pure particle halo
                CHOOSE_EXTERNAL(InteractionOut::NoAcc,   InteractionOut::NeedAcc, InteractionMode::Dilute, pair );
            else
                CHOOSE_EXTERNAL(InteractionOut::NeedAcc, InteractionOut::NeedAcc, InteractionMode::Dilute, pair );
    }
}

template<class PairwiseInteraction>
void InteractionPair<PairwiseInteraction>::setSpecificPair(std::string pv1name, std::string pv2name, PairwiseInteraction pair)
{
    intMap.insert({{pv1name, pv2name}, pair});
    intMap.insert({{pv2name, pv1name}, pair});
}

template<class PairwiseInteraction>
PairwiseInteraction& InteractionPair<PairwiseInteraction>::getPairwiseInteraction(std::string pv1name, std::string pv2name)
{
    auto it = intMap.find({pv1name, pv2name});
    if (it != intMap.end()) {
        debug("Using SPECIFIC parameters for PV pair '%s' -- '%s'", pv1name.c_str(), pv2name.c_str());
        return it->second;
    }
    else {
        debug("Using default parameters for PV pair '%s' -- '%s'", pv1name.c_str(), pv2name.c_str());
        return defaultPair;
    }
}

// for testing purpose
template class InteractionPair<Pairwise_Norandom_DPD>;

template class InteractionPair<Pairwise_DPD>;
template class InteractionPair<Pairwise_LJ>;
template class InteractionPair<Pairwise_LJObjectAware>;

template class InteractionPair<PairwiseStressWrapper<Pairwise_DPD>>;
template class InteractionPair<PairwiseStressWrapper<Pairwise_LJ>>;
template class InteractionPair<PairwiseStressWrapper<Pairwise_LJObjectAware>>;


