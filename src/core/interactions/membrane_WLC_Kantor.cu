#include "membrane_WLC_Kantor.h"
#include "membrane.impl.h"
#include "membrane/dihedral/kantor.h"
#include "membrane/triangle/wlc.h"

#include <core/utils/make_unique.h>

InteractionMembraneWLCKantor::InteractionMembraneWLCKantor(const YmrState *state, std::string name,
                                                           MembraneParameters parameters, KantorBendingParameters kantorParams,
                                                           bool stressFree, float growUntil) :
    InteractionMembrane(state, name)
{
    // TODO
    WLCParameters wlc;
    wlc.x0       = parameters.x0;
    wlc.ks       = parameters.ks;
    wlc.mpow     = parameters.mpow;
    wlc.kd       = parameters.kd;
    wlc.totArea0 = parameters.totArea0;

    if (stressFree)
        impl = std::make_unique<InteractionMembraneImpl<TriangleWLCForce<true>, DihedralKantor>>
            (state, name, parameters, wlc, kantorParams, growUntil);
    else
        impl = std::make_unique<InteractionMembraneImpl<TriangleWLCForce<false>, DihedralKantor>>
            (state, name, parameters, wlc, kantorParams, growUntil);

}

InteractionMembraneWLCKantor::~InteractionMembraneWLCKantor() = default;

