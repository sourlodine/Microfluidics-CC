#include "mdpd.h"
#include "pairwise.h"
#include "pairwise_interactions/density.h"
#include "pairwise_interactions/mdpd.h"

#include <core/celllist.h>
#include <core/utils/common.h>
#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>

#include <memory>



InteractionDensity::InteractionDensity(const YmrState *state, std::string name, float rc) :
    Interaction(state, name, rc)
{
    Pairwise_density density(rc);
    impl = std::make_unique<InteractionPair<Pairwise_density>> (state, name, rc, density);
}

InteractionDensity::~InteractionDensity() = default;

void InteractionDensity::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);

    pv1->requireDataPerParticle<float>(ChannelNames::densities, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle<float>(ChannelNames::densities, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
    
    cl1->requireExtraDataPerParticle<float>(ChannelNames::densities);
    cl2->requireExtraDataPerParticle<float>(ChannelNames::densities);
}

std::vector<Interaction::InteractionChannel> InteractionDensity::getIntermediateOutputChannels() const
{
    return {{ChannelNames::densities, Interaction::alwaysActive}};
}
std::vector<Interaction::InteractionChannel> InteractionDensity::getFinalOutputChannels() const
{
    return {};
}

void InteractionDensity::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionDensity::halo (ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}






InteractionMDPD::InteractionMDPD(const YmrState *state, std::string name, float rc, float rd, float a, float b, float gamma, float kbt, float power, bool allocateImpl) :
    Interaction(state, name, rc),
    rd(rd), a(a), b(b), gamma(gamma), kbt(kbt), power(power)
{
    if (allocateImpl) {
        Pairwise_MDPD mdpd(rc, rd, a, b, gamma, kbt, state->dt, power);
        impl = std::make_unique<InteractionPair<Pairwise_MDPD>> (state, name, rc, mdpd);
    }
}

InteractionMDPD::InteractionMDPD(const YmrState *state, std::string name, float rc, float rd, float a, float b, float gamma, float kbt, float power) :
    InteractionMDPD(state, name, rc, rd, a, b, gamma, kbt, power, true)
{}

InteractionMDPD::~InteractionMDPD() = default;

void InteractionMDPD::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);

    pv1->requireDataPerParticle<float>(ChannelNames::densities, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle<float>(ChannelNames::densities, ExtraDataManager::CommunicationMode::None, ExtraDataManager::PersistenceMode::None);
    
    cl1->requireExtraDataPerParticle<float>(ChannelNames::densities);
    cl2->requireExtraDataPerParticle<float>(ChannelNames::densities);
}

std::vector<Interaction::InteractionChannel> InteractionMDPD::getIntermediateInputChannels() const
{
    return {{ChannelNames::densities, Interaction::alwaysActive}};
}

std::vector<Interaction::InteractionChannel> InteractionMDPD::getFinalOutputChannels() const
{
    return impl->getFinalOutputChannels();
}

void InteractionMDPD::local(ParticleVector *pv1, ParticleVector *pv2,
                            CellList *cl1, CellList *cl2,
                            hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionMDPD::halo(ParticleVector *pv1, ParticleVector *pv2,
                           CellList *cl1, CellList *cl2,
                           hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionMDPD::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
                                      float a, float b, float gamma, float kbt, float power)
{
    if (a     == Default) a     = this->a;
    if (b     == Default) b     = this->b;
    if (gamma == Default) gamma = this->gamma;
    if (kbt   == Default) kbt   = this->kbt;
    if (power == Default) power = this->power;

    Pairwise_MDPD mdpd(this->rc, this->rd, a, b, gamma, kbt, state->dt, power);
    auto ptr = static_cast< InteractionPair<Pairwise_MDPD>* >(impl.get());
    
    ptr->setSpecificPair(pv1->name, pv2->name, mdpd);
}


