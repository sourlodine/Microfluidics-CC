#include "mdpd.h"
#include "pairwise.h"
#include "pairwise_interactions/mdpd.h"

#include <core/celllist.h>
#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>

#include <memory>

InteractionMDPD::InteractionMDPD(const YmrState *state, std::string name, float rc, float rd, float a, float b, float gamma, float kbt, float power, bool allocateImpl) :
    Interaction(state, name, rc),
    rd(rd), a(a), b(b), gamma(gamma), kbt(kbt), power(power)
{
    if (allocateImpl) {
        Pairwise_MDPD mdpd(rc, rd, a, b, gamma, kbt, state->dt, power);
        impl = std::make_unique<InteractionPair<Pairwise_MDPD>> (state, name, rc, mdpd);
    }
}

InteractionMDPD::InteractionMDPD(const YmrState *state, std::string name, float rc, float rd, float a, float b, float gamma, float kbt, float power) :
    InteractionMDPD(state, name, rc, rd, a, b, gamma, kbt, power, true)
{}

InteractionMDPD::~InteractionMDPD() = default;

void InteractionMDPD::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);

    // cl1->requireExtraDataPerParticle(ChannelNames::densities, Celllist::InteractionOutput::Intermediate);
    // cl2->requireExtraDataPerParticle(ChannelNames::densities, Celllist::InteractionOutput::Intermediate);
    
    cl1->setNeededForOutput();
    cl2->setNeededForOutput();
}

void InteractionMDPD::local(ParticleVector *pv1, ParticleVector *pv2,
                            CellList *cl1, CellList *cl2,
                            hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionMDPD::halo(ParticleVector *pv1, ParticleVector *pv2,
                           CellList *cl1, CellList *cl2,
                           hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionMDPD::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
                                      float a, float b, float gamma, float kbt, float power)
{
    if (a     == Default) a     = this->a;
    if (b     == Default) b     = this->b;
    if (gamma == Default) gamma = this->gamma;
    if (kbt   == Default) kbt   = this->kbt;
    if (power == Default) power = this->power;

    Pairwise_MDPD mdpd(this->rc, this->rd, a, b, gamma, kbt, state->dt, power);
    auto ptr = static_cast< InteractionPair<Pairwise_MDPD>* >(impl.get());
    
    ptr->setSpecificPair(pv1->name, pv2->name, mdpd);
}


