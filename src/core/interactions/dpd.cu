#include "dpd.h"
#include <memory>
#include "pairwise.h"
#include "pairwise_interactions/dpd.h"

#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>


InteractionDPD::InteractionDPD(const YmrState *state, std::string name, float rc, float a, float gamma, float kbt, float power, bool allocateImpl) :
    Interaction(state, name, rc),
    a(a), gamma(gamma), kbt(kbt), power(power)
{
    if (allocateImpl) {
        Pairwise_DPD dpd(rc, a, gamma, kbt, state->dt, power);
        impl = std::make_unique<InteractionPair<Pairwise_DPD>> (state, name, rc, dpd);
    }
}

InteractionDPD::InteractionDPD(const YmrState *state, std::string name, float rc, float a, float gamma, float kbt, float power) :
    InteractionDPD(state, name, rc, a, gamma, kbt, power, true)
{}

InteractionDPD::~InteractionDPD() = default;

void InteractionDPD::setPrerequisites(ParticleVector* pv1, ParticleVector* pv2)
{
    impl->setPrerequisites(pv1, pv2);
}

void InteractionDPD::initStep(ParticleVector *pv1, ParticleVector *pv2, hipStream_t stream)
{
    impl->initStep(pv1, pv2, stream);
}

void InteractionDPD::regular(ParticleVector *pv1, ParticleVector *pv2,
                             CellList *cl1, CellList *cl2,
                             hipStream_t stream)
{
    impl->regular(pv1, pv2, cl1, cl2, stream);
}

void InteractionDPD::halo(ParticleVector *pv1, ParticleVector *pv2,
                          CellList *cl1, CellList *cl2,
                          hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionDPD::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
        float a, float gamma, float kbt, float power)
{
    if (a     == Default) a     = this->a;
    if (gamma == Default) gamma = this->gamma;
    if (kbt   == Default) kbt   = this->kbt;
    if (power == Default) power = this->power;

    Pairwise_DPD dpd(this->rc, a, gamma, kbt, state->dt, power);
    auto ptr = static_cast< InteractionPair<Pairwise_DPD>* >(impl.get());
    
    ptr->setSpecificPair(pv1->name, pv2->name, dpd);
}


