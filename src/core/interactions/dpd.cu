#include "dpd.h"
#include <memory>
#include "pairwise.h"
#include "pairwise_interactions/dpd.h"

#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>


InteractionDPD::InteractionDPD(std::string name, const YmrState *state, float rc, float a, float gamma, float kbt, float dt, float power, bool allocateImpl) :
    Interaction(name, state, rc),
    a(a), gamma(gamma), kbt(kbt), dt(dt), power(power)
{
    if (allocateImpl) {
        Pairwise_DPD dpd(rc, a, gamma, kbt, dt, power);
        impl = std::make_unique<InteractionPair<Pairwise_DPD>> (name, state, rc, dpd);
    }
}

InteractionDPD::InteractionDPD(std::string name, const YmrState *state, float rc, float a, float gamma, float kbt, float dt, float power) :
    InteractionDPD(name, state, rc, a, gamma, kbt, dt, power, true)
{}

InteractionDPD::~InteractionDPD() = default;

void InteractionDPD::setPrerequisites(ParticleVector* pv1, ParticleVector* pv2)
{
    impl->setPrerequisites(pv1, pv2);
}

void InteractionDPD::regular(ParticleVector* pv1, ParticleVector* pv2,
                             CellList* cl1, CellList* cl2,
                             const float t, hipStream_t stream)
{
    impl->regular(pv1, pv2, cl1, cl2, t, stream);
}

void InteractionDPD::halo   (ParticleVector* pv1, ParticleVector* pv2,
                             CellList* cl1, CellList* cl2,
                             const float t, hipStream_t stream)
{
    impl->halo   (pv1, pv2, cl1, cl2, t, stream);
}

void InteractionDPD::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
        float a, float gamma, float kbt, float dt, float power)
{
    if (a     == Default) a     = this->a;
    if (gamma == Default) gamma = this->gamma;
    if (kbt   == Default) kbt   = this->kbt;
    if (dt    == Default) dt    = this->dt;
    if (power == Default) power = this->power;

    Pairwise_DPD dpd(this->rc, a, gamma, kbt, dt, power);
    auto ptr = static_cast< InteractionPair<Pairwise_DPD>* >(impl.get());
    
    ptr->setSpecificPair(pv1->name, pv2->name, dpd);
}


