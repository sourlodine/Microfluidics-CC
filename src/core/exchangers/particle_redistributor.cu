#include "hip/hip_runtime.h"
#include "particle_redistributor.h"

#include "exchange_helpers.h"
#include "utils/common.h"
#include "utils/face_dispatch.h"
#include "utils/fragments_mapping.h"

#include <core/celllist.h>
#include <core/pvs/packers/particles.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

enum class PackMode
{
    Query, Pack
};

namespace ParticleRedistributorKernels
{
inline __device__ int encodeCellId1d(int cid, int ncells)
{
    if      (cid <  0     ) return -1;
    else if (cid >= ncells) return  1;
    else                    return  0;
}

inline __device__ int3 encodeCellId(int3 cid, int3 ncells)
{
    cid.x = encodeCellId1d(cid.x, ncells.x);
    cid.y = encodeCellId1d(cid.y, ncells.y);
    cid.z = encodeCellId1d(cid.z, ncells.z);
    return cid;
}

inline __device__ bool hasToLeave(int3 dir)
{
    return dir.x != 0 || dir.y != 0 || dir.z != 0;
}

template <PackMode packMode>
__global__ void getExitingParticles(CellListInfo cinfo, PVview view, DomainInfo domain,
                                    ParticlePackerHandler packer, BufferOffsetsSizesWrap dataWrap)
{
    const int gid = blockIdx.x*blockDim.x + threadIdx.x;
    const int faceId = blockIdx.y;
    int cid;
    int dx, dy, dz;

    bool valid = distributeThreadsToFaceCell(cid, dx, dy, dz, gid, faceId, cinfo);

    if (!valid) return;

    // The following is called for every outer cell and exactly once for each
    // Now for each cell we check its every particle if it needs to move

    int pstart = cinfo.cellStarts[cid];
    int pend   = cinfo.cellStarts[cid+1];

#pragma unroll 2
    for (int i = 0; i < pend-pstart; i++)
    {
        const int srcId = pstart + i;
        Particle p;
        view.readPosition(p, srcId);

        int3 dir = cinfo.getCellIdAlongAxes<CellListsProjection::NoClamp>(p.r);

        dir = encodeCellId(dir, cinfo.ncells);

        if (p.isMarked()) continue;
        
        if (hasToLeave(dir))
        {
            const int bufId = FragmentMapping::getId(dir);

            int myId = atomicAdd(dataWrap.sizes + bufId, 1);

            if (packMode == PackMode::Query)
            {
                continue;
            }
            else
            {
                auto shift = ExchangersCommon::getShift(domain.localSize, dir);

                const int numElements = dataWrap.offsets[bufId+1] - dataWrap.offsets[bufId];

                auto buffer = dataWrap.getBuffer(bufId);

                packer.particles.packShift(srcId, myId, buffer, numElements, shift);
                
                // mark the particle as exited to assist cell-list building
                Float3_int pos = p.r2Float3_int();
                pos.mark();
                view.writePosition(srcId, pos.toFloat4());
            }
        }
    }
}

__global__ void unpackParticles(int startDstId, BufferOffsetsSizesWrap dataWrap,
                                ParticlePackerHandler packer)
{
    const int bufId = blockIdx.x;

    const int numElements = dataWrap.sizes[bufId];

    for (int pid = threadIdx.x; pid < numElements; pid += blockDim.x)
    {
        const int dstId = startDstId + dataWrap.offsets[bufId] + pid;
        const auto buffer = dataWrap.getBuffer(bufId);
        
        packer.particles.unpack(pid, dstId, buffer, numElements);
    }
}

} // namespace ParticleRedistributorKernels

//===============================================================================================
// Member functions
//===============================================================================================

ParticleRedistributor::ParticleRedistributor() = default;
ParticleRedistributor::~ParticleRedistributor() = default;

bool ParticleRedistributor::needExchange(int id)
{
    return !particles[id]->redistValid;
}

void ParticleRedistributor::attach(ParticleVector *pv, CellList *cl)
{
    int id = particles.size();
    particles.push_back(pv);
    cellLists.push_back(cl);

    if (dynamic_cast<PrimaryCellList*>(cl) == nullptr)
        die("Redistributor (for %s) must be used with a primary cell-list", pv->name.c_str());

    PackPredicate predicate = [](const DataManager::NamedChannelDesc& namedDesc)
    {
        return (namedDesc.second->persistence == DataManager::PersistenceMode::Active) ||
            namedDesc.first == ChannelNames::positions;
    };

    auto packer = std::make_unique<ParticlePacker>(predicate);
    auto helper = std::make_unique<ExchangeHelper>(pv->name, id, packer.get());

    packers.push_back(std::move(packer));
    helpers.push_back(std::move(helper));

    info("Particle redistributor takes pv '%s'", pv->name.c_str());
}

void ParticleRedistributor::prepareSizes(int id, hipStream_t stream)
{
    auto pv = particles[id];
    auto cl = cellLists[id];
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto lpv = pv->local();
    
    debug2("Counting leaving particles of '%s'", pv->name.c_str());

    helper->send.sizes.clear(stream);

    packer->update(lpv, stream);

    if (lpv->size() > 0)
    {
        const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
        const int nthreads = 64;
        const dim3 nblocks = dim3(getNblocks(maxdim*maxdim, nthreads), 6, 1);

        SAFE_KERNEL_LAUNCH(
            ParticleRedistributorKernels::getExitingParticles<PackMode::Query>,
            nblocks, nthreads, 0, stream,
            cl->cellInfo(), cl->getView<PVview>(),
            pv->state->domain, packer->handler(),
            helper->wrapSendData() );
    }
    helper->computeSendOffsets_Dev2Dev(stream);
}

void ParticleRedistributor::prepareData(int id, hipStream_t stream)
{
    auto pv = particles[id];
    auto cl = cellLists[id];
    auto helper = helpers[id].get();
    auto packer = packers[id].get();

    debug2("Downloading %d leaving particles of '%s'",
           helper->send.offsets[helper->nBuffers], pv->name.c_str());

    if (pv->local()->size() > 0)
    {
        const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
        const int nthreads = 64;
        const dim3 nblocks = dim3(getNblocks(maxdim*maxdim, nthreads), 6, 1);
        
        helper->resizeSendBuf();
        
        // Sizes will still remain on host, no need to download again
        helper->send.sizes.clearDevice(stream);
        
        SAFE_KERNEL_LAUNCH(
            ParticleRedistributorKernels::getExitingParticles<PackMode::Pack>,
            nblocks, nthreads, 0, stream,
            cl->cellInfo(), cl->getView<PVview>(),
            pv->state->domain, packer->handler(),
            helper->wrapSendData() );
    }
}

void ParticleRedistributor::combineAndUploadData(int id, hipStream_t stream)
{
    auto pv = particles[id];
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto lpv = pv->local();
    
    int oldSize = lpv->size();
    int totalRecvd = helper->recv.offsets[helper->nBuffers];
    lpv->resize(oldSize + totalRecvd, stream);

    if (totalRecvd > 0)
    {
        const int nthreads = 64;
        const int nblocks  = helper->nBuffers - 1;

        packer->update(lpv, stream);
        
        SAFE_KERNEL_LAUNCH(
            ParticleRedistributorKernels::unpackParticles,
            nblocks, nthreads, 0, stream,
            oldSize, helper->wrapRecvData(), packer->handler());

        // Particles may have migrated, rebuild cell-lists
        pv->cellListStamp++;
    }

    pv->redistValid = true;
}
