#include "hip/hip_runtime.h"
#include "object_halo_extra_exchanger.h"
#include "object_halo_exchanger.h"
#include "exchange_helpers.h"

#include <core/logger.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/packers/objects.h>
#include <core/pvs/views/ov.h>
#include <core/utils/kernel_launch.h>

namespace ObjectHaloExtraExchangerKernels
{
__global__ void pack(const OVview view, ObjectPackerHandler packer,
                     const MapEntry *map, BufferOffsetsSizesWrap dataWrap)
{
    int tid         = threadIdx.x;
    int dstObjId    = blockIdx.x;
    int numElements = gridDim.x;

    auto mapEntry = map[dstObjId];

    const int bufId    = mapEntry.getBufId();
    const int srcObjId = mapEntry.getId();
    
    auto buffer = dataWrap.buffer + dataWrap.offsetsBytes[bufId];

    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < view.objSize; pid += blockDim.x)
    {
        int srcId = srcObjId * view.objSize + pid;
        int dstId = dstObjId * view.objSize + pid;

        offsetBytes = packer.particles.pack(srcId, dstId, buffer,
                                            numElements * view.objSize);
    }

    buffer += offsetBytes;
    if (tid == 0)
        packer.objects.pack(srcObjId, dstObjId, buffer, numElements);    
}

__global__ void unpack(const char *buffer, int startDstObjId,
                       OVview view, ObjectPackerHandler packer)
{
    const int objId = blockIdx.x;
    const int tid   = threadIdx.x;
    const int numElements = gridDim.x;

    const int srcObjId = objId;
    const int dstObjId = objId + startDstObjId;
    
    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < view.objSize; pid += blockDim.x)
    {
        const int dstPid = dstObjId * view.objSize + pid;
        const int srcPid = srcObjId * view.objSize + pid;
        offsetBytes = packer.particles.unpack(srcPid, dstPid, buffer,
                                              numElements * view.objSize);
    }

    buffer += offsetBytes;
    
    if (tid == 0)
        packer.objects.unpack(srcObjId, dstObjId, buffer, numElements);
}
} // namespace ObjectHaloExtraExchangerKernels


ObjectExtraExchanger::ObjectExtraExchanger(ObjectHaloExchanger *entangledHaloExchanger) :
    entangledHaloExchanger(entangledHaloExchanger)
{}

ObjectExtraExchanger::~ObjectExtraExchanger() = default;

bool ObjectExtraExchanger::needExchange(int id)
{
    return true;
}

void ObjectExtraExchanger::attach(ObjectVector *ov, const std::vector<std::string>& extraChannelNames)
{
    int id = objects.size();
    objects.push_back(ov);

    PackPredicate predicate = [extraChannelNames](const DataManager::NamedChannelDesc& namedDesc)
    {
        return std::find(extraChannelNames.begin(),
                         extraChannelNames.end(),
                         namedDesc.first)
            != extraChannelNames.end();
    };
    
    auto   packer = std::make_unique<ObjectPacker>(predicate);
    auto unpacker = std::make_unique<ObjectPacker>(predicate);
    auto   helper = std::make_unique<ExchangeHelper>(ov->name, id, packer.get());

    packers  .push_back(std::move(  packer));
    unpackers.push_back(std::move(unpacker));
    helpers  .push_back(std::move(  helper));
}

void ObjectExtraExchanger::prepareSizes(int id, hipStream_t stream)
{
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto ov = objects[id];

    packer->update(ov->local(), stream);

    const auto& offsets = entangledHaloExchanger->getSendOffsets(id);

    for (int i = 0; i < helper->nBuffers; ++i)
        helper->send.sizes[i] = offsets[i+1] - offsets[i];
}

void ObjectExtraExchanger::prepareData(int id, hipStream_t stream)
{
    auto ov     = objects[id];
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto& map   = entangledHaloExchanger->getMap(id);

    helper->computeSendOffsets();
    helper->send.uploadInfosToDevice(stream);
    helper->resizeSendBuf();

    OVview ovView(ov, ov->local());

    const int nthreads = 256;
    
    SAFE_KERNEL_LAUNCH(
        ObjectHaloExtraExchangerKernels::pack,
        map.size(), nthreads, 0, stream,
        ovView, packer->handler(), map.devPtr(),
        helper->wrapSendData() );
}

void ObjectExtraExchanger::combineAndUploadData(int id, hipStream_t stream)
{
    auto ov       = objects[id];
    auto helper   = helpers[id].get();
    auto unpacker = unpackers[id].get();

    int totalRecvd = helper->recv.offsets[helper->nBuffers];

    ov->halo()->resize_anew(totalRecvd * ov->objSize);
    OVview ovView(ov, ov->local());
    
    unpacker->update(ov->halo(), stream);

    // TODO different streams
    for (int bufId = 0; bufId < helper->nBuffers; ++bufId)
    {
        int nObjs = helper->recv.sizes[bufId];

        if (bufId == helper->bulkId || nObjs == 0) continue;

        const int nthreads = 256;
        
        SAFE_KERNEL_LAUNCH(
            ObjectHaloExtraExchangerKernels::unpack,
            nObjs, nthreads, 0, stream,
            helper->recv.buffer.devPtr() + helper->recv.offsetsBytes[bufId],
            helper->recv.offsets[bufId],
            ovView, unpacker->handler() );
    }
}
