#include "hip/hip_runtime.h"
#include "object_halo_extra_exchanger.h"
#include "object_halo_exchanger.h"
#include "exchange_helpers.h"
#include "utils/stream_pool.h"

#include <core/logger.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/packers/objects.h>
#include <core/utils/kernel_launch.h>

namespace ObjectHaloExtraExchangerKernels
{
__global__ void pack(ObjectPackerHandler packer, const MapEntry *map,
                     BufferOffsetsSizesWrap dataWrap)
{
    const int tid         = threadIdx.x;
    const int objId       = blockIdx.x;
    const int numElements = gridDim.x;
    const int objSize     = packer.objSize;

    auto mapEntry = map[objId];

    const int bufId    = mapEntry.getBufId();
    const int srcObjId = mapEntry.getId();
    const int dstObjId = objId - dataWrap.offsets[bufId];
    
    auto buffer = dataWrap.getBuffer(bufId);

    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < objSize; pid += blockDim.x)
    {
        int srcId = srcObjId * objSize + pid;
        int dstId = dstObjId * objSize + pid;

        offsetBytes = packer.particles.pack(srcId, dstId, buffer,
                                            numElements * objSize);
    }

    buffer += offsetBytes;
    if (tid == 0)
        packer.objects.pack(srcObjId, dstObjId, buffer, numElements);
}

__global__ void unpack(const char *buffer, int startDstObjId,
                       ObjectPackerHandler packer)
{
    const int objId = blockIdx.x;
    const int tid   = threadIdx.x;
    const int numElements = gridDim.x;
    const int objSize = packer.objSize;

    const int srcObjId = objId;
    const int dstObjId = objId + startDstObjId;
    
    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < objSize; pid += blockDim.x)
    {
        const int dstPid = dstObjId * objSize + pid;
        const int srcPid = srcObjId * objSize + pid;
        offsetBytes = packer.particles.unpack(srcPid, dstPid, buffer,
                                              numElements * objSize);
    }

    buffer += offsetBytes;
    
    if (tid == 0)
        packer.objects.unpack(srcObjId, dstObjId, buffer, numElements);
}
} // namespace ObjectHaloExtraExchangerKernels


ObjectExtraExchanger::ObjectExtraExchanger(ObjectHaloExchanger *entangledHaloExchanger) :
    entangledHaloExchanger(entangledHaloExchanger)
{}

ObjectExtraExchanger::~ObjectExtraExchanger() = default;

bool ObjectExtraExchanger::needExchange(int id)
{
    return true;
}

void ObjectExtraExchanger::attach(ObjectVector *ov, const std::vector<std::string>& extraChannelNames)
{
    int id = objects.size();
    objects.push_back(ov);

    PackPredicate predicate = [extraChannelNames](const DataManager::NamedChannelDesc& namedDesc)
    {
        return std::find(extraChannelNames.begin(),
                         extraChannelNames.end(),
                         namedDesc.first)
            != extraChannelNames.end();
    };
    
    auto   packer = std::make_unique<ObjectPacker>(predicate);
    auto unpacker = std::make_unique<ObjectPacker>(predicate);
    auto   helper = std::make_unique<ExchangeHelper>(ov->name, id, packer.get());
    auto       sp = std::make_unique<StreamPool>(helper->nBuffers);

    packers  .push_back(std::move(  packer));
    unpackers.push_back(std::move(unpacker));
    helpers  .push_back(std::move(  helper));
    streamPools.push_back(std::move(sp));
}

void ObjectExtraExchanger::prepareSizes(int id, hipStream_t stream)
{
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto ov = objects[id];

    packer->update(ov->local(), stream);

    const auto& offsets = entangledHaloExchanger->getSendOffsets(id);

    for (int i = 0; i < helper->nBuffers; ++i)
        helper->send.sizes[i] = offsets[i+1] - offsets[i];
}

void ObjectExtraExchanger::prepareData(int id, hipStream_t stream)
{
    auto ov     = objects[id];
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    const auto& map = entangledHaloExchanger->getMap(id);

    helper->computeSendOffsets();
    helper->send.uploadInfosToDevice(stream);
    helper->resizeSendBuf();

    const int nthreads = 256;
    
    SAFE_KERNEL_LAUNCH(
        ObjectHaloExtraExchangerKernels::pack,
        map.size(), nthreads, 0, stream,
        packer->handler(), map.devPtr(),
        helper->wrapSendData() );
}

void ObjectExtraExchanger::combineAndUploadData(int id, hipStream_t stream)
{
    auto ov       = objects[id];
    auto hov      = ov->halo();
    auto helper   = helpers[id].get();
    auto unpacker = unpackers[id].get();
    auto streamPool = streamPools[id].get();
    
    int totalRecvd = helper->recv.offsets[helper->nBuffers];

    hov->resize_anew(totalRecvd * ov->objSize);
    unpacker->update(hov, stream);

    streamPool->setStart(stream);
    
    for (int bufId = 0; bufId < helper->nBuffers; ++bufId)
    {
        int nObjs = helper->recv.sizes[bufId];

        if (bufId == helper->bulkId || nObjs == 0) continue;

        const int nthreads = 256;
        
        SAFE_KERNEL_LAUNCH(
            ObjectHaloExtraExchangerKernels::unpack,
            nObjs, nthreads, 0, streamPool->get(bufId),
            helper->recv.getBufferDevPtr(bufId),
            helper->recv.offsets[bufId],
            unpacker->handler() );
    }

    streamPool->setEnd(stream);
}
