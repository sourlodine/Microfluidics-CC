#include "hip/hip_runtime.h"
#include "particles.h"
#include "common.h"
#include "shifter.h"

#include "../exchange_helpers.h"

#include <core/pvs/particle_vector.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

#include <type_traits>

namespace ParticlePackerKernels
{
template <typename T>
__global__ void packToBuffer(int n, const MapEntry *map, const size_t *offsetsBytes, const int *offsets,
                             const T *srcData, Shifter shift, char *buffer)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > n) return;

    auto m = map[i];
    int bufId = m.getBufId();
    int  srcId = m.getId();

    auto dstData = reinterpret_cast<T*>(buffer + offsetsBytes[bufId]);
    int dstId = i - offsets[bufId];

    dstData[dstId] = shift(srcData[srcId], bufId);
}

template <typename T>
__global__ void unpackFromBuffer(int nBuffers, const int *offsets, int n, const char *buffer,
                                 const size_t *offsetsBytes, T *dstData)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ int sharedOffsets[];

    for (int i = threadIdx.x; i < nBuffers; i += blockDim.x)
        sharedOffsets[i] = offsets[i];
    __syncthreads();

    if (i > n) return;
    
    int bufId = dispatchThreadsPerBuffer(nBuffers, sharedOffsets, i);
    int pid = i - sharedOffsets[bufId];
    
    auto srcData = reinterpret_cast<const T*> (buffer + offsetsBytes[bufId]);

    dstData[i] = srcData[pid];
}

} // namespace ParticlePackerKernels

ParticlesPacker::ParticlesPacker(ParticleVector *pv, PackPredicate predicate) :
    Packer(pv, predicate)
{}

size_t ParticlesPacker::getPackedSizeBytes(int n) const
{
    return _getPackedSizeBytes(pv->local()->dataPerParticle, n);
}

void ParticlesPacker::packToBuffer(const LocalParticleVector *lpv, const DeviceBuffer<MapEntry>& map,
                                   BufferInfos *helper, const std::vector<std::string>& alreadyPacked, hipStream_t stream)
{
    auto& manager = lpv->dataPerParticle;

    int nBuffers = helper->sizes.size();
    
    offsetsBytes.copyFromDevice(helper->offsetsBytes, stream);

    // advance offsets to skip the already packed data
    for (auto name : alreadyPacked)
    {
        auto& desc = manager.getChannelDescOrDie(name);
        auto advanceOffset = [&](auto pinnedBuffPtr)
        {
            using T = typename std::remove_pointer<decltype(pinnedBuffPtr)>::type::value_type;
            updateOffsets<T>(nBuffers, helper->sizes.devPtr(), offsetsBytes.devPtr(), stream);
        };
        mpark::visit(advanceOffset, desc.varDataPtr);
    }
    
    for (const auto& name_desc : manager.getSortedChannels())
    {
        if (!predicate(name_desc)) continue;
        auto& desc = name_desc.second;

        bool isAlreadPacked = std::find(alreadyPacked.begin(), alreadyPacked.end(), name_desc.first) != alreadyPacked.end();
        if (isAlreadPacked) continue;

        Shifter shift(desc->shiftTypeSize > 0, pv->state->domain);

        auto packChannel = [&](auto pinnedBuffPtr)
        {
            using T = typename std::remove_pointer<decltype(pinnedBuffPtr)>::type::value_type;

            int n = map.size();
            const int nthreads = 128;

            SAFE_KERNEL_LAUNCH(
                ParticlePackerKernels::packToBuffer,
                getNblocks(n, nthreads), nthreads, 0, stream,
                n, map.devPtr(), offsetsBytes.devPtr(), helper->offsets.devPtr(),
                pinnedBuffPtr->devPtr(), shift, helper->buffer.devPtr());

            updateOffsets<T>(nBuffers, helper->sizes.devPtr(), offsetsBytes.devPtr(), stream);
        };
        
        mpark::visit(packChannel, desc->varDataPtr);
    }
}

void ParticlesPacker::unpackFromBuffer(LocalParticleVector *lpv, const BufferInfos *helper, int oldSize, hipStream_t stream)
{
    auto& manager = lpv->dataPerParticle;

    offsetsBytes.copyFromDevice(helper->offsetsBytes, stream);

    int nBuffers  = helper->sizes.size();
    int nIncoming = helper->offsets[nBuffers];
    
    for (const auto& name_desc : manager.getSortedChannels())
    {
        if (!predicate(name_desc)) continue;
        auto& desc = name_desc.second;

        auto unpackChannel = [&](auto pinnedBuffPtr)
        {
            using T = typename std::remove_pointer<decltype(pinnedBuffPtr)>::type::value_type;

            const int nthreads = 128;
            const size_t sharedMem = nBuffers * sizeof(int);

            SAFE_KERNEL_LAUNCH(
                ParticlePackerKernels::unpackFromBuffer,
                getNblocks(nIncoming, nthreads), nthreads, sharedMem, stream,
                nBuffers, helper->offsets.devPtr(), nIncoming, helper->buffer.devPtr(),
                offsetsBytes.devPtr(), pinnedBuffPtr->devPtr() + oldSize);

            updateOffsets<T>(nBuffers, helper->sizes.devPtr(), offsetsBytes.devPtr(), stream);
        };
        
        mpark::visit(unpackChannel, desc->varDataPtr);
    }
}
