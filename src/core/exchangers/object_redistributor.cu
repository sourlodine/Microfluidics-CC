#include "hip/hip_runtime.h"
#include "object_redistributor.h"

#include "exchange_helpers.h"
#include "utils/common.h"
#include "utils/fragments_mapping.h"
#include "utils/stream_pool.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/views/ov.h>
#include <core/pvs/packers/objects.h>
#include <core/logger.h>
#include <core/utils/cuda_common.h>

enum class PackMode
{
    Query, Pack
};

namespace ObjecRedistributorKernels
{

template <PackMode packMode>
__global__ void getExitingObjects(DomainInfo domain, OVview view,
                                  ObjectPackerHandler packer, BufferOffsetsSizesWrap dataWrap)
{
    const int objId = blockIdx.x;
    const int tid   = threadIdx.x;
    
    // Find to which buffer this object should go
    auto prop = view.comAndExtents[objId];
    auto dir  = ExchangersCommon::getDirection(prop.com, domain.localSize);

    const int bufId = FragmentMapping::getId(dir);

    __shared__ int shDstObjId;

    __syncthreads();
    
    if (tid == 0)
        shDstObjId = atomicAdd(dataWrap.sizes + bufId, 1);

    if (packMode == PackMode::Query)
    {
        return;
    }
    else
    {
        __syncthreads();
        
        auto shift = ExchangersCommon::getShift(domain.localSize, dir);

        auto buffer = dataWrap.getBuffer(bufId);
        int numElements = dataWrap.offsets[bufId+1] - dataWrap.offsets[bufId];

        size_t offsetBytes = 0;
        
        for (int pid = tid; pid < view.objSize; pid += blockDim.x)
        {
            const int srcPid = objId      * view.objSize + pid;
            const int dstPid = shDstObjId * view.objSize + pid;
            
            offsetBytes = packer.particles.packShift(srcPid, dstPid, buffer,
                                                     numElements * view.objSize, shift);
        }

        buffer += offsetBytes;
        
        if (tid == 0)
            packer.objects.packShift(objId, shDstObjId, buffer, numElements, shift);
    }
}

__global__ void unpackObjects(const char *buffer, int startDstObjId,
                              ObjectPackerHandler packer)
{
    const int objId = blockIdx.x;
    const int tid   = threadIdx.x;
    const int numElements = gridDim.x;
    const int objSize = packer.objSize;

    const int srcObjId = objId;
    const int dstObjId = objId + startDstObjId;
    
    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < objSize; pid += blockDim.x)
    {
        const int dstPid = dstObjId * objSize + pid;
        const int srcPid = srcObjId * objSize + pid;
        offsetBytes = packer.particles.unpack(srcPid, dstPid, buffer,
                                              numElements * objSize);
    }

    buffer += offsetBytes;
    
    if (tid == 0)
        packer.objects.unpack(srcObjId, dstObjId, buffer, numElements);
}

} // namespace ObjecRedistributorKernels

ObjectRedistributor::ObjectRedistributor() = default;
ObjectRedistributor::~ObjectRedistributor() = default;

bool ObjectRedistributor::needExchange(int id)
{
    return !objects[id]->redistValid;
}

void ObjectRedistributor::attach(ObjectVector *ov)
{
    int id = objects.size();
    objects.push_back(ov);

    PackPredicate predicate = [](const DataManager::NamedChannelDesc& namedDesc)
    {
        return (namedDesc.second->persistence == DataManager::PersistenceMode::Persistent) ||
            (namedDesc.first == ChannelNames::positions);
    };
    
    auto packer = std::make_unique<ObjectPacker>(predicate);
    auto helper = std::make_unique<ExchangeHelper>(ov->name, id, packer.get());
    auto sp     = std::make_unique<StreamPool>(helper->nBuffers);
    
    packers.push_back(std::move(packer));
    helpers.push_back(std::move(helper));
    streamPools.push_back(std::move(sp));

    info("The Object vector '%s' was attached to redistributor", ov->name.c_str());
}


void ObjectRedistributor::prepareSizes(int id, hipStream_t stream)
{
    auto ov  = objects[id];
    auto lov = ov->local();
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto bulkId = helper->bulkId;
    
    ov->findExtentAndCOM(stream, ParticleVectorType::Local);
    
    OVview ovView(ov, lov);

    debug2("Counting exiting objects of '%s'", ov->name.c_str());

    // Prepare sizes
    helper->send.sizes.clear(stream);
    packer->update(lov, stream);
    
    if (ovView.nObjects > 0)
    {
        const int nthreads = 256;
        const int nblocks  = ovView.nObjects;
        
        SAFE_KERNEL_LAUNCH(
            ObjecRedistributorKernels::getExitingObjects<PackMode::Query>,
            nblocks, nthreads, 0, stream,
            ov->state->domain, ovView, packer->handler(), helper->wrapSendData() );

        helper->computeSendOffsets_Dev2Dev(stream);
    }

    int nObjs = helper->send.sizes[bulkId];
    debug2("%d objects of '%s' will leave", ovView.nObjects - nObjs, ov->name.c_str());

    // Early termination support
    if (nObjs == ovView.nObjects)
    {
        helper->send.sizes[bulkId] = 0;
        helper->computeSendOffsets();
        helper->send.uploadInfosToDevice(stream);
        helper->resizeSendBuf();
    }
}

void ObjectRedistributor::prepareData(int id, hipStream_t stream)
{
    auto ov  = objects[id];
    auto lov = ov->local();
    auto helper = helpers[id].get();
    auto bulkId = helper->bulkId;
    auto packer = packers[id].get();

    OVview ovView(ov, lov);

    int nObjsBulk = helper->send.sizes[bulkId];

    // Early termination - no redistribution
    if (helper->send.offsets[helper->nBuffers] == 0)
    {
        debug2("No objects of '%s' leaving, no need to rebuild the object vector",
               ov->name.c_str());
        return;
    }

    debug2("Downloading %d leaving objects of '%s'", ovView.nObjects - nObjsBulk,
           ov->name.c_str());

    // Gather data
    helper->resizeSendBuf();
    helper->send.sizes.clearDevice(stream);
    
    const int nthreads = 256;
    const int nblocks  = ovView.nObjects;

    SAFE_KERNEL_LAUNCH(
        ObjecRedistributorKernels::getExitingObjects<PackMode::Pack>,
        nblocks, nthreads, 0, stream,
        ov->state->domain, ovView, packer->handler(), helper->wrapSendData() );    

    // Unpack the central buffer into the object vector itself
    // Renew view, as the ObjectVector may have resized
    lov->resize_anew(nObjsBulk * ov->objSize);
    packer->update(lov, stream);

    SAFE_KERNEL_LAUNCH(
         ObjecRedistributorKernels::unpackObjects,
         nObjsBulk, nthreads, 0, stream,
         helper->send.getBufferDevPtr(bulkId), 0,
         packer->handler() );
    
    helper->send.sizes[bulkId] = 0;
    helper->computeSendOffsets();
    helper->send.uploadInfosToDevice(stream);
    helper->resizeSendBuf(); // relying here on the fact that bulkId is the last one
}

void ObjectRedistributor::combineAndUploadData(int id, hipStream_t stream)
{
    auto ov     = objects[id];
    auto lov    = ov->local();
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto streamPool = streamPools[id].get();

    int oldNObjs = lov->nObjects;
    int objSize = ov->objSize;

    int totalRecvd = helper->recv.offsets[helper->nBuffers];

    lov->resize((oldNObjs + totalRecvd) * objSize, stream);
    packer->update(lov, stream);

    streamPool->setStart(stream);
    
    for (int bufId = 0; bufId < helper->nBuffers; ++bufId)
    {
        int nObjs = helper->recv.sizes[bufId];

        if (bufId == helper->bulkId || nObjs == 0) continue;

        const int nthreads = 256;
        
        SAFE_KERNEL_LAUNCH(
            ObjecRedistributorKernels::unpackObjects,
            nObjs, nthreads, 0, streamPool->get(bufId),
            helper->recv.getBufferDevPtr(bufId),
            oldNObjs + helper->recv.offsets[bufId],
            packer->handler() );
    }

    streamPool->setEnd(stream);

    ov->redistValid = true;

    // Particles may have migrated, rebuild cell-lists
    if (totalRecvd > 0)
        ov->cellListStamp++;
}
