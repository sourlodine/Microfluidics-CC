#include "hip/hip_runtime.h"
#include "object_reverse_exchanger.h"
#include "exchange_helpers.h"
#include "object_halo_exchanger.h"
#include "utils/stream_pool.h"

#include <core/logger.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/packers/objects.h>
#include <core/utils/kernel_launch.h>

namespace ObjectReverseExchangerKernels
{

__global__ void reversePack(char *buffer, int startDstObjId,
                            ObjectPackerHandler packer)
{
    const int objId = blockIdx.x;
    const int tid   = threadIdx.x;
    const int numElements = gridDim.x;
    const int objSize = packer.objSize;

    const int dstObjId = objId;
    const int srcObjId = objId + startDstObjId;
    
    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < objSize; pid += blockDim.x)
    {
        const int dstPid = dstObjId * objSize + pid;
        const int srcPid = srcObjId * objSize + pid;
        offsetBytes = packer.particles.pack(srcPid, dstPid, buffer,
                                            numElements * objSize);
    }

    buffer += offsetBytes;
    
    if (tid == 0)
        packer.objects.pack(srcObjId, dstObjId, buffer, numElements);
}

__global__ void reverseUnpackAndAdd(ObjectPackerHandler packer, const MapEntry *map,
                                    BufferOffsetsSizesWrap dataWrap)
{
    constexpr float eps = 1e-6f;
    const int tid         = threadIdx.x;
    const int objId       = blockIdx.x;
    const int numElements = gridDim.x;
    const int objSize = packer.objSize;

    auto mapEntry = map[objId];
    const int bufId    = mapEntry.getBufId();
    const int dstObjId = mapEntry.getId();
    const int srcObjId = objId - dataWrap.offsets[bufId];
    
    auto buffer = dataWrap.getBuffer(bufId);

    size_t offsetBytes = 0;
    
    for (int pid = tid; pid < objSize; pid += blockDim.x)
    {
        int srcId = srcObjId * objSize + pid;
        int dstId = dstObjId * objSize + pid;

        offsetBytes = packer.particles.
            unpackAtomicAddNonZero(srcId, dstId, buffer,
                                   numElements * objSize, eps);
    }

    buffer += offsetBytes;
    if (tid == 0)
        packer.objects.unpackAtomicAddNonZero(srcObjId, dstObjId, buffer, numElements, eps);    
}

} // namespace ObjectReverseExchangerKernels


ObjectReverseExchanger::ObjectReverseExchanger(ObjectHaloExchanger *entangledHaloExchanger) :
    entangledHaloExchanger(entangledHaloExchanger)
{}

ObjectReverseExchanger::~ObjectReverseExchanger() = default;

void ObjectReverseExchanger::attach(ObjectVector *ov, std::vector<std::string> channelNames)
{
    int id = objects.size();
    objects.push_back(ov);

    PackPredicate predicate = [channelNames](const DataManager::NamedChannelDesc& namedDesc)
    {
        return std::find(channelNames.begin(),
                         channelNames.end(),
                         namedDesc.first)
            != channelNames.end();
    };

    auto   packer = std::make_unique<ObjectPacker>(predicate);
    auto unpacker = std::make_unique<ObjectPacker>(predicate);
    auto   helper = std::make_unique<ExchangeHelper>(ov->name, id, packer.get());
    auto       sp = std::make_unique<StreamPool>(helper->nBuffers);
    
    packers  .push_back(std::move(  packer));
    unpackers.push_back(std::move(unpacker));
    helpers  .push_back(std::move(  helper));

    streamPools.push_back(std::move(sp));
}

bool ObjectReverseExchanger::needExchange(int id)
{
    return true;
}

void ObjectReverseExchanger::prepareSizes(int id, hipStream_t stream)
{
    auto  helper  = helpers[id].get();
    auto& offsets = entangledHaloExchanger->getRecvOffsets(id);
    
    for (int i = 0; i < helper->nBuffers; ++i)
        helper->send.sizes[i] = offsets[i+1] - offsets[i];
}

void ObjectReverseExchanger::prepareData(int id, hipStream_t stream)
{
    auto ov     = objects[id];
    auto hov    = ov->halo();
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    auto streamPool = streamPools[id].get();
    
    debug2("Preparing '%s' data to reverse send", ov->name.c_str());

    packer->update(hov, stream);

    helper->computeSendOffsets();
    helper->send.uploadInfosToDevice(stream);
    helper->resizeSendBuf();

    streamPool->setStart(stream);
    
    for (int bufId = 0; bufId < helper->nBuffers; ++bufId)
    {
        int nObjs = helper->send.sizes[bufId];

        if (bufId == helper->bulkId || nObjs == 0) continue;

        const int nthreads = 256;
        
        SAFE_KERNEL_LAUNCH(
            ObjectReverseExchangerKernels::reversePack,
            nObjs, nthreads, 0, streamPool->get(bufId),
            helper->send.getBufferDevPtr(bufId),
            helper->send.offsets[bufId],
            packer->handler() );
    }

    streamPool->setEnd(stream);
    
    debug2("Will send back data for %d objects", helper->send.offsets[helper->nBuffers]);
}

void ObjectReverseExchanger::combineAndUploadData(int id, hipStream_t stream)
{
    auto ov       = objects[id];
    auto lov      = ov->local();
    auto helper   =   helpers[id].get();
    auto unpacker = unpackers[id].get();

    unpacker->update(lov, stream);
    
    int totalRecvd = helper->recv.offsets[helper->nBuffers];
    auto& map = entangledHaloExchanger->getMap(id);
    
    debug("Updating data for %d '%s' objects", totalRecvd, ov->name.c_str());

    const int nthreads = 256;
        
    SAFE_KERNEL_LAUNCH(
        ObjectReverseExchangerKernels::reverseUnpackAndAdd,
        map.size(), nthreads, 0, stream,
        unpacker->handler(), map.devPtr(),
        helper->wrapRecvData());
}
