#include "hip/hip_runtime.h"
#include "object_reverse_exchanger.h"
#include "object_halo_exchanger.h"
#include "exchange_helpers.h"
#include "utils/common.h"

#include <core/logger.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/packers/objects.h>
#include <core/utils/kernel_launch.h>

namespace ObjectReverseExchangerKernels
{

template <class PackerHandler>
__global__ void reversePack(BufferOffsetsSizesWrap dataWrap, PackerHandler packer)
{
    
    const int objId = blockIdx.x;
    const int tid   = threadIdx.x;

    extern __shared__ int offsets[];

    const int nBuffers = dataWrap.nBuffers;

    for (int i = tid; i < nBuffers + 1; i += blockDim.x)
        offsets[i] = dataWrap.offsets[i];
    __syncthreads();

    const int bufId = dispatchThreadsPerBuffer(nBuffers, offsets, objId);
    auto buffer = dataWrap.getBuffer(bufId);
    const int numElements = dataWrap.sizes[bufId];

    const int dstObjId = objId - offsets[bufId];
    const int srcObjId = objId;

    packer.blockPack(numElements, buffer, srcObjId, dstObjId);
}

template <class PackerHandler>
__global__ void reverseUnpackAndAdd(PackerHandler packer, const MapEntry *map,
                                    BufferOffsetsSizesWrap dataWrap)
{
    constexpr float eps = 1e-6f;
    const int objId       = blockIdx.x;
    const int numElements = gridDim.x;
    
    auto mapEntry = map[objId];
    const int bufId    = mapEntry.getBufId();
    const int dstObjId = mapEntry.getId();
    const int srcObjId = objId - dataWrap.offsets[bufId];
    
    auto buffer = dataWrap.getBuffer(bufId);

    packer.blockUnpackAddNonZero(numElements, buffer, srcObjId, dstObjId, eps);
}

} // namespace ObjectReverseExchangerKernels


ObjectReverseExchanger::ObjectReverseExchanger(ObjectHaloExchanger *entangledHaloExchanger) :
    entangledHaloExchanger(entangledHaloExchanger)
{}

ObjectReverseExchanger::~ObjectReverseExchanger() = default;

void ObjectReverseExchanger::attach(ObjectVector *ov, std::vector<std::string> channelNames)
{
    int id = objects.size();
    objects.push_back(ov);

    PackPredicate predicate = [channelNames](const DataManager::NamedChannelDesc& namedDesc)
    {
        return std::find(channelNames.begin(),
                         channelNames.end(),
                         namedDesc.first)
            != channelNames.end();
    };

    auto   packer = std::make_unique<ObjectPacker>(predicate);
    auto unpacker = std::make_unique<ObjectPacker>(predicate);
    auto   helper = std::make_unique<ExchangeHelper>(ov->name, id, packer.get());
    
    packers  .push_back(std::move(  packer));
    unpackers.push_back(std::move(unpacker));
    helpers  .push_back(std::move(  helper));
}

bool ObjectReverseExchanger::needExchange(int id)
{
    return true;
}

void ObjectReverseExchanger::prepareSizes(int id, hipStream_t stream)
{
    auto  helper  = helpers[id].get();
    auto& offsets = entangledHaloExchanger->getRecvOffsets(id);
    
    for (int i = 0; i < helper->nBuffers; ++i)
        helper->send.sizes[i] = offsets[i+1] - offsets[i];
}

void ObjectReverseExchanger::prepareData(int id, hipStream_t stream)
{
    auto ov     = objects[id];
    auto hov    = ov->halo();
    auto helper = helpers[id].get();
    auto packer = packers[id].get();
    
    debug2("Preparing '%s' data to reverse send", ov->name.c_str());

    packer->update(hov, stream);

    helper->computeSendOffsets();
    helper->send.uploadInfosToDevice(stream);
    helper->resizeSendBuf();

    const auto& offsets = helper->send.offsets;
    const int nSendObj = offsets[helper->nBuffers];
    
    const int nthreads = 256;
    const int nblocks = nSendObj;

    const size_t shMemSize = offsets.size() * sizeof(offsets[0]);

    mpark::visit([&](auto packerHandler)
    {
        SAFE_KERNEL_LAUNCH(
            ObjectReverseExchangerKernels::reversePack,
            nblocks, nthreads, shMemSize, stream,
            helper->wrapSendData(), packerHandler );
    }, ExchangersCommon::getHandler(packer));
    
    debug2("Will send back data for %d objects", nSendObj);
}

void ObjectReverseExchanger::combineAndUploadData(int id, hipStream_t stream)
{
    auto ov       = objects[id];
    auto lov      = ov->local();
    auto helper   =   helpers[id].get();
    auto unpacker = unpackers[id].get();

    unpacker->update(lov, stream);
    
    int totalRecvd = helper->recv.offsets[helper->nBuffers];
    auto& map = entangledHaloExchanger->getMap(id);
    
    debug("Updating data for %d '%s' objects", totalRecvd, ov->name.c_str());

    const int nthreads = 256;
        
    mpark::visit([&](auto unpackerHandler)
    {
        SAFE_KERNEL_LAUNCH(
            ObjectReverseExchangerKernels::reverseUnpackAndAdd,
            map.size(), nthreads, 0, stream,
            unpackerHandler, map.devPtr(),
            helper->wrapRecvData());
    }, ExchangersCommon::getHandler(unpacker));
}
