#include "hip/hip_runtime.h"
#include "const_omega.h"
#include "integration_kernel.h"

#include <core/utils/kernel_launch.h>
#include <core/logger.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>


IntegratorConstOmega::IntegratorConstOmega(const YmrState *state, std::string name, float3 center, float3 omega) :
    Integrator(state, name),
    center(center), omega(omega)
{}

IntegratorConstOmega::~IntegratorConstOmega() = default;

void IntegratorConstOmega::stage1(ParticleVector *pv, hipStream_t stream)
{}

void IntegratorConstOmega::stage2(ParticleVector *pv, hipStream_t stream)
{
    const auto domain = state->domain;
    const auto _center = center;
    const auto _omega = omega;

    auto rotate = [domain, _center, _omega] __device__ (Particle& p, const float3 f, const float invm, const float dt) {
        float3 gr = domain.local2global(p.r);
        float3 gr_c = gr - _center;
        p.u = cross(_omega, gr_c);
        float IrI = length(gr_c);
        gr_c += p.u*dt;

        gr_c = normalize(gr_c) * IrI;
        p.r = domain.global2local(gr_c + _center);
    };

    int nthreads = 128;

    // New particles now become old
    std::swap(pv->local()->coosvels, *pv->local()->extraPerParticle.getData<Particle>(ChannelNames::oldParts));
    PVviewWithOldParticles pvView(pv, pv->local());

    SAFE_KERNEL_LAUNCH(
            integrationKernel,
            getNblocks(2*pvView.size, nthreads), nthreads, 0, stream,
            pvView, state->dt, rotate );

    // PV may have changed, invalidate all
    pv->haloValid = false;
    pv->redistValid = false;
    pv->cellListStamp++;
}
