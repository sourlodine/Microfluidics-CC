#include "hip/hip_runtime.h"
#include "translate.h"
#include "integration_kernel.h"

#include <core/logger.h>
#include <core/pvs/particle_vector.h>


/**
 * @param vel Move with this velocity
 */
IntegratorTranslate::IntegratorTranslate(const YmrState *state, std::string name, float3 vel) :
    Integrator(state, name),
    vel(vel)
{}

IntegratorTranslate::~IntegratorTranslate() = default;

void IntegratorTranslate::stage2(ParticleVector *pv, hipStream_t stream)
{
    const auto _vel = vel;

    auto translate = [_vel] __device__ (Particle& p, const float3 f, const float invm, const float dt) {
        p.u = _vel;
        p.r += p.u*dt;
    };

    integrate(pv, state->dt, translate, stream);
    invalidatePV(pv);
}
