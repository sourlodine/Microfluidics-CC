#include "hip/hip_runtime.h"
#include "rigid_vv.h"

#include <core/logger.h>
#include <core/pvs/rigid_object_vector.h>
#include <core/pvs/views/rov.h>
#include <core/rigid_kernels/integration.h>
#include <core/utils/kernel_launch.h>

IntegratorVVRigid::IntegratorVVRigid(const MirState *state, std::string name) :
    Integrator(state, name)
{}

IntegratorVVRigid::~IntegratorVVRigid() = default;

/**
 * Can only be applied to RigidObjectVector and requires it to have
 * \c old_motions data channel per particle
 */
void IntegratorVVRigid::setPrerequisites(ParticleVector* pv)
{
    auto ov = dynamic_cast<RigidObjectVector*> (pv);
    if (ov == nullptr)
        die("Rigid integration only works with rigid objects, can't work with %s", pv->name.c_str());

    ov->requireDataPerObject<RigidMotion>(ChannelNames::oldMotions, DataManager::PersistenceMode::None);
    warn("Only objects with diagonal inertia tensors are supported now for rigid integration");
}


// FIXME: split VV into two stages
void IntegratorVVRigid::stage1(ParticleVector *pv, hipStream_t stream)
{}




static void collectRigidForces(const ROVviewWithOldMotion& view, hipStream_t stream)
{
    const int nthreads = 128;
    const int nblocks = getNblocks(2*view.size, nthreads);
    
    SAFE_KERNEL_LAUNCH(
        RigidIntegrationKernels::collectRigidForces,
        nblocks, nthreads, 0, stream,
        view );
}

static void integrateRigidMotions(const ROVviewWithOldMotion& view, float dt, hipStream_t stream)
{
    const int nthreads = 64;
    const int nblocks = getNblocks(view.nObjects, nthreads);
    
    SAFE_KERNEL_LAUNCH(
        RigidIntegrationKernels::integrateRigidMotion,
        nblocks, nthreads, 0, stream,
        view, dt );
}

static void setRigidParticlesFromMotions(const ROVviewWithOldMotion& view, const PinnedBuffer<float4>& initialPositions, hipStream_t stream)
{
    const int nthreads = 128;
    const int nblocks = getNblocks(view.size, nthreads);
        
    SAFE_KERNEL_LAUNCH(
        RigidIntegrationKernels::applyRigidMotion<RigidIntegrationKernels::ApplyRigidMotion::PositionsAndVelocities>,
        nblocks, nthreads, 0, stream,
        view, initialPositions.devPtr() );
}

static void clearRigidForces(const ROVviewWithOldMotion& view, hipStream_t stream)
{
    const int nthreads = 64;
    const int nblocks = getNblocks(view.nObjects, nthreads);

    SAFE_KERNEL_LAUNCH(
        RigidIntegrationKernels::clearRigidForces,
        nblocks, nthreads, 0, stream,
        view );
}


void IntegratorVVRigid::stage2(ParticleVector *pv, hipStream_t stream)
{
    const float dt = state->dt;
    auto rov = dynamic_cast<RigidObjectVector*> (pv);

    debug("Integrating %d rigid objects %s (total %d particles), timestep is %f",
          rov->local()->nObjects, rov->name.c_str(), rov->local()->size(), dt);

    const ROVviewWithOldMotion rovView(rov, rov->local());

    collectRigidForces           (rovView,                        stream);
    integrateRigidMotions        (rovView, dt,                    stream);    
    setRigidParticlesFromMotions (rovView, rov->initialPositions, stream);
    clearRigidForces             (rovView,                        stream);

    invalidatePV(pv);
}

