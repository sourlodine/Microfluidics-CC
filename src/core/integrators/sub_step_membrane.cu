#include "sub_step_membrane.h"

#include <core/utils/kernel_launch.h>
#include <core/logger.h>
#include <core/pvs/particle_vector.h>
#include <core/interactions/membrane.h>


IntegratorSubStepMembrane::IntegratorSubStepMembrane(const YmrState *state, std::string name, int substeps, Interaction *fastForces) :
    Integrator(state, name), substeps(substeps),
    subIntegrator(new IntegratorVV<Forcing_None>(state, name + "_sub", Forcing_None()))
{
    this->fastForces = dynamic_cast<InteractionMembrane*>(fastForces);
    
    if ( this->fastForces == nullptr )
        die("IntegratorSubStepMembrane expects an interaction of type <InteractionMembrane>.");

    subIntegrator->dt = dt / substeps;
}

IntegratorSubStepMembrane::~IntegratorSubStepMembrane() = default;

void IntegratorSubStepMembrane::stage1(ParticleVector *pv, hipStream_t stream)
{}

void IntegratorSubStepMembrane::stage2(ParticleVector *pv, hipStream_t stream)
{
    // save "slow forces"
    slowForces.copy(pv->local()->forces, stream);
    
    // save previous positions
    previousPositions.copyFromDevice(pv->local()->coosvels, stream);

    // advance with internal vv integrator
    for (int substep = 0; substep < substeps; ++ substep) {

        if (substep != 0)
            pv->local()->forces.copy(slowForces, stream);

        // TODO was , t + substep * dt / substeps
        fastForces->regular(pv, pv, nullptr, nullptr, stream);
        
        subIntegrator->stage2(pv, stream);
    }
    
    // restore previous positions into old_particles channel
    pv->local()->extraPerParticle.getData<Particle>("old_particles")->copy(previousPositions, stream);

    // PV may have changed, invalidate all
    pv->haloValid = false;
    pv->redistValid = false;
    pv->cellListStamp++;
}

void IntegratorSubStepMembrane::setPrerequisites(ParticleVector* pv)
{
    fastForces->setPrerequisites(pv, pv);
}
