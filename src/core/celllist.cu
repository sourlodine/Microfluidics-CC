#include "hip/hip_runtime.h"
#include "celllist.h"

#include <core/logger.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>
#include <core/utils/type_map.h>

#include <extern/cub/cub/device/device_scan.cuh>

namespace CellListKernels
{

enum {INVALID = -1};

inline __device__ bool outgoingParticle(float4 pos)
{
    return Float3_int(pos).isMarked();
}

__global__ void computeCellSizes(PVview view, CellListInfo cinfo)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    float4 coo = view.readPositionNoCache(pid);
    int cid = cinfo.getCellId(coo);

    // XXX: relying here only on redistribution
    if ( !outgoingParticle(coo) )
        atomicAdd(cinfo.cellSizes + cid, 1);
}

__global__ void reorderParticles(PVview view, CellListInfo cinfo, float4 *outParticles)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int pid = gid / 2;
    const int sh  = gid % 2;  // sh = 0 copies coordinates, sh = 1 -- velocity
    if (pid >= view.size) return;

    int dstId;

    // this is to allow more cache for atomics
    // loads / stores here need no cache
    float4 val = readNoCache(view.particles+gid);

    int cid;
    if (sh == 0)
    {
        cid = cinfo.getCellId(val);

        //  XXX: relying here only on redistribution
        if ( !outgoingParticle(val) )
            dstId = cinfo.cellStarts[cid] + atomicAdd(cinfo.cellSizes + cid, 1);
        else
            dstId = -1;
    }

    int otherDst = warpShflUp(dstId, 1);
    if (sh == 1)
        dstId = otherDst;

    if (dstId >= 0)
    {
        writeNoCache(outParticles + 2*dstId+sh, val);
        if (sh == 0) cinfo.order[pid] = dstId;
    }
    else if (sh == 0)
        cinfo.order[pid] = INVALID;
}

template <typename T>
__global__ void reorderExtraDataPerParticle(int n, const T *inExtraData, CellListInfo cinfo, T *outExtraData)
{
    int srcId = blockIdx.x * blockDim.x + threadIdx.x;
    if (srcId >= n) return;

    int dstId = cinfo.order[srcId];
    if (dstId != INVALID)
        outExtraData[dstId] = inExtraData[srcId];
}

template <typename T>
__global__ void accumulateKernel(int n, T *dst, CellListInfo cinfo, const T *src)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= n) return;

    int srcId = cinfo.order[pid];

    assert(srcId != INVALID);
    dst[pid] += src[srcId];
}

} // namespace CellListKernels

//=================================================================================
// Info
//=================================================================================

CellListInfo::CellListInfo(float rc, float3 localDomainSize) :
        rc(rc), h(make_float3(rc)), localDomainSize(localDomainSize)
{
    ncells = make_int3( floorf(localDomainSize / rc + 1e-6) );
    float3 h = make_float3(localDomainSize) / make_float3(ncells);
    invh = 1.0f / h;
    this->rc = std::min( {h.x, h.y, h.z} );

    totcells = ncells.x * ncells.y * ncells.z;
}

CellListInfo::CellListInfo(float3 h, float3 localDomainSize) :
        h(h), invh(1.0f/h), localDomainSize(localDomainSize)
{
    rc = std::min( {h.x, h.y, h.z} );
    ncells = make_int3( ceilf(localDomainSize / h - 1e-6f) );
    totcells = ncells.x * ncells.y * ncells.z;
}

//=================================================================================
// Basic cell-lists
//=================================================================================

CellList::CellList(ParticleVector *pv, float rc, float3 localDomainSize) :
        CellListInfo(rc, localDomainSize), pv(pv),
        particlesDataContainer(new LocalParticleVector(nullptr))
{
    localPV = particlesDataContainer.get();
    
    cellSizes. resize_anew(totcells + 1);
    cellStarts.resize_anew(totcells + 1);

    cellSizes. clear(0);
    cellStarts.clear(0);
    CUDA_Check( hipStreamSynchronize(0) );

    debug("Initialized %s cell-list with %dx%dx%d cells and cut-off %f", pv->name.c_str(), ncells.x, ncells.y, ncells.z, this->rc);
}

CellList::CellList(ParticleVector *pv, int3 resolution, float3 localDomainSize) :
        CellListInfo(localDomainSize / make_float3(resolution), localDomainSize), pv(pv),
        particlesDataContainer(new LocalParticleVector(nullptr))
{
    localPV = particlesDataContainer.get();
    
    cellSizes. resize_anew(totcells + 1);
    cellStarts.resize_anew(totcells + 1);

    cellSizes. clear(0);
    cellStarts.clear(0);
    CUDA_Check( hipStreamSynchronize(0) );

    debug("Initialized %s cell-list with %dx%dx%d cells and cut-off %f", pv->name.c_str(), ncells.x, ncells.y, ncells.z, this->rc);
}

CellList::~CellList() = default;

bool CellList::_checkNeedBuild() const
{
    if (changedStamp == pv->cellListStamp)
    {
        debug2("%s is already up-to-date, building skipped", makeName().c_str());
        return false;
    }

    if (pv->local()->size() == 0)
    {
        debug2("%s consists of no particles, building skipped", makeName().c_str());
        return false;
    }

    return true;
}

void CellList::_updateExtraDataChannels(hipStream_t stream)
{
    auto& pvManager        = pv->local()->extraPerParticle;
    auto& containerManager = particlesDataContainer->extraPerParticle;
    int np = pv->local()->size();

    for (const auto& namedChannel : pvManager.getSortedChannels()) {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != ExtraDataManager::PersistenceMode::Persistent) continue;

        mpark::visit([&](auto pinnedBuff) {
                         using T = typename std::remove_reference< decltype(pinnedBuff->hostPtr()[0]) >::type;

                         if (!containerManager.checkChannelExists(name))
                             containerManager.createData<T>(name, np);
                         
                     }, desc->varDataPtr);
    }
}

void CellList::_computeCellSizes(hipStream_t stream)
{
    debug2("%s : Computing cell sizes for %d particles", makeName().c_str(), pv->local()->size());
    cellSizes.clear(stream);

    PVview view(pv, pv->local());

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
            CellListKernels::computeCellSizes,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, cellInfo() );
}

void CellList::_computeCellStarts(hipStream_t stream)
{
    // Scan is always working with the same number of cells
    // Memory requirements can't change
    size_t bufSize = scanBuffer.size();
    
    if (bufSize == 0)
    {
        hipcub::DeviceScan::ExclusiveSum(nullptr, bufSize, cellSizes.devPtr(), cellStarts.devPtr(), totcells+1, stream);
        scanBuffer.resize_anew(bufSize);
    }
    hipcub::DeviceScan::ExclusiveSum(scanBuffer.devPtr(), bufSize,
                                  cellSizes.devPtr(), cellStarts.devPtr(), totcells+1, stream);
}

void CellList::_reorderData(hipStream_t stream)
{
    debug2("Reordering %d %s particles", pv->local()->size(), pv->name.c_str());

    PVview view(pv, pv->local());

    order.resize_anew(view.size);
    particlesDataContainer->resize_anew(view.size);
    cellSizes.clear(stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
        CellListKernels::reorderParticles,
        getNblocks(2*view.size, nthreads), nthreads, 0, stream,
        view, cellInfo(), (float4*)particlesDataContainer->coosvels.devPtr() );
}

void CellList::_reorderExtraDataEntry(const std::string& channelName,
                                      const ExtraDataManager::ChannelDescription *channelDesc,
                                      hipStream_t stream)
{
    const auto& dstDesc = particlesDataContainer->extraPerParticle.getChannelDescOrDie(channelName);
    int np = pv->local()->size();

    mpark::visit([&](auto srcPinnedBuff) {
                     auto dstPinnedBuff = mpark::get<decltype(srcPinnedBuff)>(dstDesc.varDataPtr);

                     const int nthreads = 128;

                     SAFE_KERNEL_LAUNCH(
                         CellListKernels::reorderExtraDataPerParticle,
                         getNblocks(np, nthreads), nthreads, 0, stream,
                         np, srcPinnedBuff->devPtr(), this->cellInfo(), dstPinnedBuff->devPtr() );

                 }, channelDesc->varDataPtr);
}

void CellList::_reorderPersistentData(hipStream_t stream)
{
    auto srcExtraData = &pv->local()->extraPerParticle;
    
    for (const auto& namedChannel : srcExtraData->getSortedChannels()) {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != ExtraDataManager::PersistenceMode::Persistent) continue;
        _reorderExtraDataEntry(name, desc, stream);
    }
}

void CellList::_build(hipStream_t stream)
{
    _computeCellSizes(stream);
    _computeCellStarts(stream);
    _reorderData(stream);
    _reorderPersistentData(stream);
    
    changedStamp = pv->cellListStamp;
}

CellListInfo CellList::cellInfo()
{
    CellListInfo::cellSizes  = cellSizes.devPtr();
    CellListInfo::cellStarts = cellStarts.devPtr();
    CellListInfo::order      = order.devPtr();

    return *((CellListInfo*)this);
}

void CellList::build(hipStream_t stream)
{
    _updateExtraDataChannels(stream);
        
    if (!_checkNeedBuild()) return;
    
    debug("building %s", makeName().c_str());
    
    _build(stream);
}

// use SFINAE to discard types without operator+
static void accumulateIfHasAddOperator(GPUcontainer *src,
                                       GPUcontainer *dst,
                                       int n, CellListInfo cinfo,
                                       hipStream_t stream)
{
    die("Cannot accumulate entries: operator+ not supported for this type");
}

template <typename T>
static auto accumulateIfHasAddOperator(PinnedBuffer<T> *src,
                                       PinnedBuffer<T> *dst,
                                       int n, CellListInfo cinfo,
                                       hipStream_t stream)
    -> decltype(T() + T())
{
    const int nthreads = 128;
    
    SAFE_KERNEL_LAUNCH(
        CellListKernels::accumulateKernel,
        getNblocks(n, nthreads), nthreads, 0, stream,
        n, dst->devPtr(), cinfo, src->devPtr() );

    return T();
}

void CellList::_accumulateExtraData(const std::string& channelName, hipStream_t stream)
{
    int n = pv->local()->size();    

    const auto& pvManager   = pv->local()->extraPerParticle;
    const auto& contManager = localPV->extraPerParticle;

    const auto& pvDesc   = pvManager  .getChannelDescOrDie(channelName);
    const auto& contDesc = contManager.getChannelDescOrDie(channelName);

    mpark::visit([&](auto srcPinnedBuff) {
                     auto dstPinnedBuff = mpark::get<decltype(srcPinnedBuff)>(pvDesc.varDataPtr);

                     accumulateIfHasAddOperator(srcPinnedBuff, dstPinnedBuff, n, this->cellInfo(), stream);

                 }, contDesc.varDataPtr);
}

void CellList::accumulateChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (const auto& channelName : channelNames) {
        debug2("%s : accumulating channel '%s'", makeName().c_str(), channelName.c_str());

        _accumulateExtraData(channelName, stream);
    }
}

void CellList::gatherChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (auto& channelName : channelNames) {

        debug("%s : gathering channel '%s'", makeName().c_str(), channelName.c_str());
        
        auto& desc = localPV->extraPerParticle.getChannelDescOrDie(channelName);
        _reorderExtraDataEntry(channelName, &desc, stream);

        // invalidate particle vector halo if any entry is active
        pv->haloValid = false;
    }
}

void CellList::clearChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (const auto& channelName : channelNames) {
        debug2("%s : clearing channel '%s'", makeName().c_str(), channelName.c_str());
        localPV->extraPerParticle.getGenericData(channelName)->clearDevice(stream);
    }
}

LocalParticleVector* CellList::getLocalParticleVector() {return localPV;}

std::string CellList::makeName() const
{
    return "Cell List '" + pv->name + "' (rc " + std::to_string(rc) + ")";
}


//=================================================================================
// Primary cell-lists
//=================================================================================

PrimaryCellList::PrimaryCellList(ParticleVector *pv, float rc, float3 localDomainSize) :
        CellList(pv, rc, localDomainSize)
{
    localPV = pv->local();

    if (dynamic_cast<ObjectVector*>(pv) != nullptr)
        error("Using primary cell-lists with objects is STRONGLY discouraged. This will very likely result in an error");
}

PrimaryCellList::PrimaryCellList(ParticleVector *pv, int3 resolution, float3 localDomainSize) :
        CellList(pv, resolution, localDomainSize)
{
    localPV = pv->local();

    if (dynamic_cast<ObjectVector*>(pv) != nullptr)
        error("Using primary cell-lists with objects is STRONGLY discouraged. This will very likely result in an error");
}

PrimaryCellList::~PrimaryCellList() = default;

void PrimaryCellList::build(hipStream_t stream)
{
	// Reqired here to avoid ptr swap if building didn't actually happen
    if (!_checkNeedBuild()) return;

    CellList::build(stream);

    if (pv->local()->size() == 0)
    {
        debug2("%s consists of no particles, cell-list building skipped", pv->name.c_str());
        return;
    }
    
    // Now we need the new size of particles array.
    int newSize;
    CUDA_Check( hipMemcpyAsync(&newSize, cellStarts.devPtr() + totcells, sizeof(int), hipMemcpyDeviceToHost, stream) );
    CUDA_Check( hipStreamSynchronize(stream) );

    debug2("%s : reordering completed, new size of %s particle vector is %d",
           makeName().c_str(), pv->name.c_str(), newSize);

    particlesDataContainer->resize(newSize, stream);

    std::swap(pv->local()->coosvels, particlesDataContainer->coosvels);
    _swapPersistentExtraData();
    
    pv->local()->resize(newSize, stream);
}

void PrimaryCellList::accumulateChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{}

void PrimaryCellList::gatherChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    // do not need to reorder data, but still invalidate halo
    if (!channelNames.empty())
        pv->haloValid = false;
}


template <typename T>
static void swap(const std::string& channelName, ExtraDataManager& pvManager, ExtraDataManager& containerManager)
{
    std::swap(*pvManager       .getData<T>(channelName),
              *containerManager.getData<T>(channelName));
}

void PrimaryCellList::_swapPersistentExtraData()
{
    auto& pvManager        = pv->local()->extraPerParticle;
    auto& containerManager = particlesDataContainer->extraPerParticle;
    
    for (const auto& namedChannel : pvManager.getSortedChannels()) {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != ExtraDataManager::PersistenceMode::Persistent) continue;

        const auto& descCont = containerManager.getChannelDescOrDie(name);

        mpark::visit([&](auto pinnedBufferPv) {
                         auto pinnedBufferCont = mpark::get<decltype(pinnedBufferPv)>(descCont.varDataPtr);
                         std::swap(*pinnedBufferPv, *pinnedBufferCont);
                     }, desc->varDataPtr);
    }
}

std::string PrimaryCellList::makeName() const
{
    return "Primary " + CellList::makeName();
}
