#include "hip/hip_runtime.h"
#include "restart_helpers.h"

namespace restart_helpers
{
    static void sendData(const std::vector<std::vector<Particle>> &sendBufs, std::vector<MPI_Request> &reqs,
                         MPI_Comm comm, MPI_Datatype type)
    {
        for (int i = 0; i < sendBufs.size(); i++) {
            debug3("Sending %d paricles to rank %d", sendBufs[i].size(), i);
            MPI_Check( MPI_Isend(sendBufs[i].data(), sendBufs[i].size(), type, i, 0, comm, reqs.data()+i) );
        }
    }

    static void recvData(int size, std::vector<Particle> &all, MPI_Comm comm, MPI_Datatype type)
    {
        all.resize(0);
        for (int i = 0; i < size; i++) {
            MPI_Status status;
            int msize;
            std::vector<Particle> recvBuf;
        
            MPI_Check( MPI_Probe(MPI_ANY_SOURCE, 0, comm, &status) );
            MPI_Check( MPI_Get_count(&status, type, &msize) );

            recvBuf.resize(msize);

            debug3("Receiving %d particles from ???", msize);
            MPI_Check( MPI_Recv(recvBuf.data(), msize, type, status.MPI_SOURCE, 0, comm, MPI_STATUS_IGNORE) );

            all.insert(all.end(), recvBuf.begin(), recvBuf.end());
        }
    }
    
    void exchangeParticles(const DomainInfo &domain, MPI_Comm comm, std::vector<Particle> &parts)
    {
        int size;
        int dims[3], periods[3], coords[3];
        MPI_Check( MPI_Comm_size(comm, &size) );
        MPI_Check( MPI_Cart_get(comm, 3, dims, periods, coords) );

        MPI_Datatype ptype;
        MPI_Check( MPI_Type_contiguous(sizeof(Particle), MPI_CHAR, &ptype) );
        MPI_Check( MPI_Type_commit(&ptype) );

        // Find where to send the read particles
        std::vector<std::vector<Particle>> sendBufs(size);

        for (auto& p : parts) {
            int3 procId3 = make_int3(floorf(p.r / domain.localSize));

            if (procId3.x >= dims[0] || procId3.y >= dims[1] || procId3.z >= dims[2])
                continue;

            int procId;
            MPI_Check( MPI_Cart_rank(comm, (int*)&procId3, &procId) );
            sendBufs[procId].push_back(p);
        }

        std::vector<MPI_Request> reqs(size);
        
        sendData(sendBufs, reqs, comm, ptype);
        recvData(size, parts, comm, ptype);

        MPI_Check( MPI_Waitall(reqs.size(), reqs.data(), MPI_STATUSES_IGNORE) );
        MPI_Check( MPI_Type_free(&ptype) );
    }
    
    void copyShiftCoordinates(const DomainInfo &domain, const std::vector<Particle> &parts, LocalParticleVector *local)
    {
        local->resize(parts.size(), 0);

        for (int i = 0; i < parts.size(); i++) {
            auto p = parts[i];
            p.r = domain.global2local(p.r);
            local->coosvels[i] = p;
        }
    }
}
