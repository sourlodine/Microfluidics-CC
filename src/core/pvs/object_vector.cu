#include "hip/hip_runtime.h"
#include "object_vector.h"
#include "views/ov.h"

#include <core/utils/kernel_launch.h>
#include <core/utils/cuda_common.h>
#include <core/utils/folders.h>
#include <core/xdmf/xdmf.h>

#include "restart_helpers.h"

namespace ObjectVectorKernels
{

__global__ void minMaxCom(OVview ovView)
{
    const int gid = threadIdx.x + blockDim.x * blockIdx.x;
    const int objId = gid >> 5;
    const int tid = gid & 0x1f;
    if (objId >= ovView.nObjects) return;

    float3 mymin = make_float3( 1e+10f);
    float3 mymax = make_float3(-1e+10f);
    float3 mycom = make_float3(0);

#pragma unroll 3
    for (int i = tid; i < ovView.objSize; i += warpSize)
    {
        const int offset = objId * ovView.objSize + i;

        const float3 coo = make_float3(ovView.readPosition(offset));

        mymin = fminf(mymin, coo);
        mymax = fmaxf(mymax, coo);
        mycom += coo;
    }

    mycom = warpReduce( mycom, [] (float a, float b) { return a+b; } );
    mymin = warpReduce( mymin, [] (float a, float b) { return fmin(a, b); } );
    mymax = warpReduce( mymax, [] (float a, float b) { return fmax(a, b); } );

    if (tid == 0)
        ovView.comAndExtents[objId] = {mycom / ovView.objSize, mymin, mymax};
}

} // namespace ObjectVectorKernels


LocalObjectVector::LocalObjectVector(ParticleVector *pv, int objSize, int nObjects) :
    LocalParticleVector(pv, objSize*nObjects), objSize(objSize), nObjects(nObjects)
{
    if (objSize <= 0)
        die("Object vector should contain at least one particle per object instead of %d", objSize);

    resize_anew(nObjects*objSize);
}

LocalObjectVector::~LocalObjectVector() = default;

void LocalObjectVector::resize(int np, hipStream_t stream)
{
    nObjects = getNobjects(np);
    LocalParticleVector::resize(np, stream);
    dataPerObject.resize(nObjects, stream);
}

void LocalObjectVector::resize_anew(int np)
{
    nObjects = getNobjects(np);
    LocalParticleVector::resize_anew(np);
    dataPerObject.resize_anew(nObjects);
}

void LocalObjectVector::computeGlobalIds(MPI_Comm comm, hipStream_t stream)
{
    LocalParticleVector::computeGlobalIds(comm, stream);

    if (np == 0) return;

    Particle p0( positions()[0], velocities()[0]);
    int64_t rankStart = p0.getId();
    
    if ((rankStart % objSize) != 0)
        die("Something went wrong when computing ids of '%s':"
            "got rankStart = '%ld' while objectSize is '%d'",
            pv->name.c_str(), rankStart, objSize);

    auto& ids = *dataPerObject.getData<int64_t>(ChannelNames::globalIds);
    int64_t id = (int64_t) (rankStart / objSize);
    
    for (auto& i : ids)
        i = id++;

    ids.uploadToDevice(stream);
}

PinnedBuffer<float4>* LocalObjectVector::getMeshVertices(hipStream_t stream)
{
    return &positions();
}

PinnedBuffer<float4>* LocalObjectVector::getOldMeshVertices(hipStream_t stream)
{
    return dataPerParticle.getData<float4>(ChannelNames::oldPositions);
}

PinnedBuffer<Force>* LocalObjectVector::getMeshForces(hipStream_t stream)
{
    return &forces();
}

int LocalObjectVector::getNobjects(int np) const
{
    if (np % objSize != 0)
        die("Incorrect number of particles in object: given %d, must be a multiple of %d", np, objSize);

    return np / objSize;
}


ObjectVector::ObjectVector(const YmrState *state, std::string name, float mass, int objSize, int nObjects) :
    ObjectVector( state, name, mass, objSize,
                  std::make_unique<LocalObjectVector>(this, objSize, nObjects),
                  std::make_unique<LocalObjectVector>(this, objSize, 0) )
{}

ObjectVector::ObjectVector(const YmrState *state, std::string name, float mass, int objSize,
                           std::unique_ptr<LocalParticleVector>&& local,
                           std::unique_ptr<LocalParticleVector>&& halo) :
    ParticleVector(state, name, mass, std::move(local), std::move(halo)),
    objSize(objSize)
{
    // center of mass and extents are not to be sent around
    // it's cheaper to compute them on site
    requireDataPerObject<COMandExtent>(ChannelNames::comExtents, DataManager::PersistenceMode::None);

    // object ids must always follow objects
    requireDataPerObject<int64_t>(ChannelNames::globalIds, DataManager::PersistenceMode::Persistent);
}

ObjectVector::~ObjectVector() = default;

void ObjectVector::findExtentAndCOM(hipStream_t stream, ParticleVectorType type)
{
    bool isLocal = (type == ParticleVectorType::Local);
    auto lov = isLocal ? local() : halo();

    debug("Computing COM and extent OV '%s' (%s)", name.c_str(), isLocal ? "local" : "halo");

    const int nthreads = 128;
    OVview ovView(this, lov);
    SAFE_KERNEL_LAUNCH(
            ObjectVectorKernels::minMaxCom,
            (ovView.nObjects*32 + nthreads-1)/nthreads, nthreads, 0, stream,
            ovView );
}

void ObjectVector::_getRestartExchangeMap(MPI_Comm comm, const std::vector<float4>& pos, std::vector<int>& map)
{
    int dims[3], periods[3], coords[3];
    MPI_Check( MPI_Cart_get(comm, 3, dims, periods, coords) );

    int nObjs = pos.size() / objSize;
    map.resize(nObjs);
    
    for (int i = 0, k = 0; i < nObjs; ++i) {
        auto com = make_float3(0);

        for (int j = 0; j < objSize; ++j, ++k)
            com += make_float3(pos[k]);

        com /= objSize;

        int3 procId3 = make_int3(floorf(com / state->domain.localSize));

        if (procId3.x >= dims[0] || procId3.y >= dims[1] || procId3.z >= dims[2]) {
            map[i] = -1;
            continue;
        }
        
        int procId;
        MPI_Check( MPI_Cart_rank(comm, (int*)&procId3, &procId) );
        map[i] = procId;
    }
}


std::vector<int> ObjectVector::_restartParticleData(MPI_Comm comm, std::string path)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointName(path, "PV", "xmf");
    info("Restarting object vector %s from file %s", name.c_str(), filename.c_str());

    XDMF::readParticleData(filename, comm, this, objSize);

    std::vector<float4> pos4(local()->size()), vel4(local()->size());
    std::vector<int> map;
    
    std::copy(local()->positions ().begin(), local()->positions ().end(), pos4.begin());
    std::copy(local()->velocities().begin(), local()->velocities().end(), vel4.begin());
    
    _getRestartExchangeMap(comm, pos4, map);
    RestartHelpers::exchangeData(comm, map, pos4, objSize);
    RestartHelpers::exchangeData(comm, map, vel4, objSize);
    RestartHelpers::copyShiftCoordinates(state->domain, pos4, vel4, local());

    local()->positions ().uploadToDevice(defaultStream);
    local()->velocities().uploadToDevice(defaultStream);
    
    CUDA_Check( hipDeviceSynchronize() );

    info("Successfully read %d particles", local()->size());

    return map;
}

static void splitCom(DomainInfo domain, const PinnedBuffer<COMandExtent>& com_extents, std::vector<float> &positions)
{
    int n = com_extents.size();
    positions.resize(3 * n);

    float3 *pos = (float3*) positions.data();
    
    for (int i = 0; i < n; ++i) {
        auto r = com_extents[i].com;
        pos[i] = domain.local2global(r);
    }
}

void ObjectVector::_extractPersistentExtraObjectData(std::vector<XDMF::Channel>& channels, const std::set<std::string>& blackList)
{
    auto& extraData = local()->dataPerObject;
    _extractPersistentExtraData(extraData, channels, blackList);
}

void ObjectVector::_checkpointObjectData(MPI_Comm comm, std::string path, int checkpointId)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointNameWithId(path, "OV", "", checkpointId);
    info("Checkpoint for object vector '%s', writing to file %s", name.c_str(), filename.c_str());

    auto coms_extents = local()->dataPerObject.getData<COMandExtent>(ChannelNames::comExtents);

    coms_extents->downloadFromDevice(defaultStream, ContainersSynch::Synch);
    
    auto positions = std::make_shared<std::vector<float>>();

    splitCom(state->domain, *coms_extents, *positions);

    XDMF::VertexGrid grid(positions, comm);

    std::vector<XDMF::Channel> channels;

    _extractPersistentExtraObjectData(channels);
    
    XDMF::write(filename, &grid, channels, comm);

    createCheckpointSymlink(comm, path, "OV", "xmf", checkpointId);

    debug("Checkpoint for object vector '%s' successfully written", name.c_str());
}

void ObjectVector::_restartObjectData(MPI_Comm comm, std::string path, const std::vector<int>& map)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointName(path, "OV", "xmf");
    info("Restarting object vector %s from file %s", name.c_str(), filename.c_str());

    XDMF::readObjectData(filename, comm, this);

    auto loc_ids = local()->dataPerObject.getData<int64_t>(ChannelNames::globalIds);
    
    std::vector<int> ids(loc_ids->size());
    std::copy(loc_ids->begin(), loc_ids->end(), ids.begin());
    
    RestartHelpers::exchangeData(comm, map, ids, 1);

    loc_ids->resize_anew(ids.size());
    std::copy(ids.begin(), ids.end(), loc_ids->begin());

    loc_ids->uploadToDevice(defaultStream);
    CUDA_Check( hipDeviceSynchronize() );

    info("Successfully read %d object infos", loc_ids->size());
}

void ObjectVector::checkpoint(MPI_Comm comm, std::string path, int checkpointId)
{
    _checkpointParticleData(comm, path, checkpointId);
    _checkpointObjectData  (comm, path, checkpointId);
}

void ObjectVector::restart(MPI_Comm comm, std::string path)
{
    auto map = _restartParticleData(comm, path);
    _restartObjectData(comm, path, map);
}
