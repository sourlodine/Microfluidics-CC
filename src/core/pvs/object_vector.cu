#include "hip/hip_runtime.h"
#include "object_vector.h"
#include "views/ov.h"

#include <core/utils/kernel_launch.h>
#include <core/utils/cuda_common.h>
#include "core/xdmf/xdmf.h"
#include "restart_helpers.h"

__global__ void min_max_com(OVview ovView)
{
    const int gid = threadIdx.x + blockDim.x * blockIdx.x;
    const int objId = gid >> 5;
    const int tid = gid & 0x1f;
    if (objId >= ovView.nObjects) return;

    float3 mymin = make_float3( 1e+10f);
    float3 mymax = make_float3(-1e+10f);
    float3 mycom = make_float3(0);

#pragma unroll 3
    for (int i = tid; i < ovView.objSize; i += warpSize)
    {
        const int offset = (objId * ovView.objSize + i) * 2;

        const float3 coo = make_float3(ovView.particles[offset]);

        mymin = fminf(mymin, coo);
        mymax = fmaxf(mymax, coo);
        mycom += coo;
    }

    mycom = warpReduce( mycom, [] (float a, float b) { return a+b; } );
    mymin = warpReduce( mymin, [] (float a, float b) { return fmin(a, b); } );
    mymax = warpReduce( mymax, [] (float a, float b) { return fmax(a, b); } );

    if (tid == 0)
        ovView.comAndExtents[objId] = {mycom / ovView.objSize, mymin, mymax};
}

void ObjectVector::findExtentAndCOM(hipStream_t stream, ParticleVectorType type)
{
    bool isLocal = (type == ParticleVectorType::Local);
    auto lov = isLocal ? local() : halo();

    if (lov->comExtentValid)
    {
        debug("COM and extent computation for %s OV '%s' skipped",
                isLocal ? "local" : "halo", name.c_str());
        return;
    }

    debug("Computing COM and extent OV '%s' (%s)", name.c_str(), isLocal ? "local" : "halo");

    const int nthreads = 128;
    OVview ovView(this, lov);
    SAFE_KERNEL_LAUNCH(
            min_max_com,
            (ovView.nObjects*32 + nthreads-1)/nthreads, nthreads, 0, stream,
            ovView );
}


void ObjectVector::restart(MPI_Comm comm, std::string path)
{
    CUDA_Check( hipDeviceSynchronize() );

    std::string filename = path + "/" + name + ".xmf";
    info("Restarting object vector %s from file %s", name.c_str(), filename.c_str());

    XDMF::read(filename, comm, this, objSize);

    std::vector<Particle> parts(local()->coosvels.begin(), local()->coosvels.end());

    restart_helpers::exchangeParticlesChunks(domain, comm, parts, objSize);

    restart_helpers::copyShiftCoordinates(domain, parts, local());

    local()->coosvels.uploadToDevice(0);

    CUDA_Check( hipDeviceSynchronize() );

    info("Successfully read %d particles", local()->coosvels.size());
}
