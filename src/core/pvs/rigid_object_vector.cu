#include "hip/hip_runtime.h"
#include "restart/helpers.h"
#include "checkpoint/helpers.h"
#include "rigid_object_vector.h"
#include "views/rov.h"

#include <core/rigid_kernels/integration.h>
#include <core/utils/folders.h>
#include <core/utils/kernel_launch.h>
#include <core/xdmf/type_map.h>
#include <core/xdmf/xdmf.h>


LocalRigidObjectVector::LocalRigidObjectVector(ParticleVector* pv, int objSize, int nObjects) :
    LocalObjectVector(pv, objSize, nObjects)
{}

PinnedBuffer<float4>* LocalRigidObjectVector::getMeshVertices(hipStream_t stream)
{
    auto ov = dynamic_cast<RigidObjectVector*>(pv);
    auto& mesh = ov->mesh;
    meshVertices.resize_anew(nObjects * mesh->getNvertices());

    ROVview fakeView(ov, this);
    fakeView.objSize   = mesh->getNvertices();
    fakeView.size      = mesh->getNvertices() * nObjects;
    fakeView.positions = meshVertices.devPtr();

    const int nthreads = 128;
    
    SAFE_KERNEL_LAUNCH(
            RigidIntegrationKernels::applyRigidMotion
                <RigidIntegrationKernels::ApplyRigidMotion::PositionsOnly>,
            getNblocks(fakeView.size, nthreads), nthreads, 0, stream,
            fakeView, ov->mesh->vertexCoordinates.devPtr() );

    return &meshVertices;
}

PinnedBuffer<float4>* LocalRigidObjectVector::getOldMeshVertices(hipStream_t stream)
{
    auto ov = dynamic_cast<RigidObjectVector*>(pv);
    auto& mesh = ov->mesh;
    meshOldVertices.resize_anew(nObjects * mesh->getNvertices());

    // Overwrite particles with vertices
    // Overwrite motions with the old_motions
    ROVview fakeView(ov, this);
    fakeView.objSize   = mesh->getNvertices();
    fakeView.size      = mesh->getNvertices() * nObjects;
    fakeView.positions = meshOldVertices.devPtr();
    fakeView.motions   = dataPerObject.getData<RigidMotion>(ChannelNames::oldMotions)->devPtr();

    const int nthreads = 128;
    
    SAFE_KERNEL_LAUNCH(
            RigidIntegrationKernels::applyRigidMotion
                <RigidIntegrationKernels::ApplyRigidMotion::PositionsOnly>,
            getNblocks(fakeView.size, nthreads), nthreads, 0, stream,
            fakeView, ov->mesh->vertexCoordinates.devPtr() );

    return &meshOldVertices;
}

PinnedBuffer<Force>* LocalRigidObjectVector::getMeshForces(hipStream_t stream)
{
    auto ov = dynamic_cast<ObjectVector*>(pv);
    meshForces.resize_anew(nObjects * ov->mesh->getNvertices());
    return &meshForces;
}




RigidObjectVector::RigidObjectVector(const MirState *state, std::string name, float partMass,
                                     float3 J, const int objSize,
                                     std::shared_ptr<Mesh> mesh, const int nObjects) :
    ObjectVector( state, name, partMass, objSize,
                  std::make_unique<LocalRigidObjectVector>(this, objSize, nObjects),
                  std::make_unique<LocalRigidObjectVector>(this, objSize, 0) ),
    J(J)
{
    this->mesh = std::move(mesh);

    if (length(J) < 1e-5)
        die("Wrong momentum of inertia: [%f %f %f]", J.x, J.y, J.z);

    if (J.x < 0 || J.y < 0 || J.z < 0)
        die("Inertia tensor must be positive; got [%f %f %f]", J.x, J.y, J.z);


    // rigid motion must be exchanged and shifted
    requireDataPerObject<RigidMotion>(ChannelNames::motions,
                                      DataManager::PersistenceMode::Active,
                                      DataManager::ShiftMode::Active);

    requireDataPerObject<RigidMotion>(ChannelNames::oldMotions,
                                      DataManager::PersistenceMode::None);
}

RigidObjectVector::RigidObjectVector(const MirState *state, std::string name, float partMass,
                                     PyTypes::float3 J, const int objSize,
                                     std::shared_ptr<Mesh> mesh, const int nObjects) :
    RigidObjectVector( state, name, partMass, make_float3(J), objSize, mesh, nObjects )
{}

RigidObjectVector::~RigidObjectVector() = default;

void RigidObjectVector::_checkpointObjectData(MPI_Comm comm, std::string path, int checkpointId)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointNameWithId(path, "ROV", "", checkpointId);
    info("Checkpoint for rigid object vector '%s', writing to file %s",
         name.c_str(), filename.c_str());

    auto motions = local()->dataPerObject.getData<RigidMotion>(ChannelNames::motions);

    motions->downloadFromDevice(defaultStream, ContainersSynch::Synch);
    
    auto positions = std::make_shared<std::vector<float3>>();
    std::vector<RigidReal4> quaternion;
    std::vector<RigidReal3> vel, omega, force, torque;
    
    std::tie(*positions, quaternion, vel, omega, force, torque)
        = CheckpointHelpers::splitAndShiftMotions(state->domain, *motions);

    XDMF::VertexGrid grid(positions, comm);    

    auto rigidType = XDMF::getNumberType<RigidReal>();

    const std::set<std::string> blackList {ChannelNames::motions};
    
    auto channels = CheckpointHelpers::extractShiftPersistentData(state->domain,
                                                                  local()->dataPerObject,
                                                                  blackList);

    channels.emplace_back(ChannelNames::XDMF::Motions::quaternion, quaternion .data(),
                          XDMF::Channel::DataForm::Quaternion,
                          rigidType, DataTypeWrapper<RigidReal4>());
    
    channels.emplace_back(ChannelNames::XDMF::Motions::velocity,   vel.data(),
                          XDMF::Channel::DataForm::Vector,
                          rigidType, DataTypeWrapper<RigidReal3>());

    channels.emplace_back(ChannelNames::XDMF::Motions::omega,      omega.data(),
                          XDMF::Channel::DataForm::Vector,
                          rigidType, DataTypeWrapper<RigidReal3>());
    
    channels.emplace_back(ChannelNames::XDMF::Motions::force,      force.data(),
                          XDMF::Channel::DataForm::Vector,
                          rigidType, DataTypeWrapper<RigidReal3>());
    
    channels.emplace_back(ChannelNames::XDMF::Motions::torque,     torque.data(),
                          XDMF::Channel::DataForm::Vector,
                          rigidType, DataTypeWrapper<RigidReal3>());
    
    XDMF::write(filename, &grid, channels, comm);

    createCheckpointSymlink(comm, path, "ROV", "xmf", checkpointId);

    debug("Checkpoint for object vector '%s' successfully written", name.c_str());
}

void RigidObjectVector::_restartObjectData(MPI_Comm comm, std::string path,
                                           const RigidObjectVector::ExchMapSize& ms)
{
    using namespace RestartHelpers;
    constexpr int objChunkSize = 1; // only one datum per object
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointName(path, "ROV", "xmf");
    info("Restarting rigid object vector %s from file %s", name.c_str(), filename.c_str());

    auto listData = readData(filename, comm, objChunkSize);

    namespace ChNames = ChannelNames::XDMF;
    auto pos        = extractChannel<float3>     (ChNames::position,            listData);
    auto quaternion = extractChannel<RigidReal4> (ChNames::Motions::quaternion, listData);
    auto vel        = extractChannel<RigidReal3> (ChNames::Motions::velocity,   listData);
    auto omega      = extractChannel<RigidReal3> (ChNames::Motions::omega,      listData);
    auto force      = extractChannel<RigidReal3> (ChNames::Motions::force,      listData);
    auto torque     = extractChannel<RigidReal3> (ChNames::Motions::torque,     listData);

    auto motions = combineMotions(pos, quaternion, vel, omega, force, torque);
    
    auto& dataPerObject = local()->dataPerObject;
    dataPerObject.resize_anew(ms.newSize);

    exchangeData    (comm, ms.map, motions,  objChunkSize);
    exchangeListData(comm, ms.map, listData, objChunkSize);

    shiftElementsGlobal2Local(motions, state->domain);

    auto& dstMotions = *dataPerObject.getData<RigidMotion>(ChannelNames::motions);

    std::copy(motions.begin(), motions.end(), dstMotions.begin());
    dstMotions.uploadToDevice(defaultStream);
    
    for (auto& entry : listData)
    {
        auto channelDesc = &dataPerObject.getChannelDescOrDie(entry.name);
        
        mpark::visit([&](const auto& data)
        {
            using T = typename std::remove_reference<decltype(data)>::type::value_type;
            auto dstPtr = dataPerObject.getData<T>(entry.name);

            if (channelDesc->needShift())
                RestartHelpers::shiftElementsGlobal2Local(data, state->domain);

            std::copy(data.begin(), data.end(), dstPtr->begin());
            dstPtr->uploadToDevice(defaultStream);
        }, entry.data);
    }
    
    info("Successfully read object infos of '%s'", name.c_str());
}
