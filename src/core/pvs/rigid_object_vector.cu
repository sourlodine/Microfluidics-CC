#include "hip/hip_runtime.h"
#include "restart_helpers.h"
#include "rigid_object_vector.h"
#include "views/rov.h"

#include <core/rigid_kernels/integration.h>
#include <core/utils/folders.h>
#include <core/utils/kernel_launch.h>
#include <core/xdmf/type_map.h>
#include <core/xdmf/xdmf.h>


LocalRigidObjectVector::LocalRigidObjectVector(ParticleVector* pv, int objSize, int nObjects) :
    LocalObjectVector(pv, objSize, nObjects)
{}

PinnedBuffer<float4>* LocalRigidObjectVector::getMeshVertices(hipStream_t stream)
{
    auto ov = dynamic_cast<RigidObjectVector*>(pv);
    auto& mesh = ov->mesh;
    meshVertices.resize_anew(nObjects * mesh->getNvertices());

    ROVview fakeView(ov, this);
    fakeView.objSize   = mesh->getNvertices();
    fakeView.size      = mesh->getNvertices() * nObjects;
    fakeView.positions = meshVertices.devPtr();

    const int nthreads = 128;
    
    SAFE_KERNEL_LAUNCH(
            RigidIntegrationKernels::applyRigidMotion
                <RigidIntegrationKernels::ApplyRigidMotion::PositionsOnly>,
            getNblocks(fakeView.size, nthreads), nthreads, 0, stream,
            fakeView, ov->mesh->vertexCoordinates.devPtr() );

    return &meshVertices;
}

PinnedBuffer<float4>* LocalRigidObjectVector::getOldMeshVertices(hipStream_t stream)
{
    auto ov = dynamic_cast<RigidObjectVector*>(pv);
    auto& mesh = ov->mesh;
    meshOldVertices.resize_anew(nObjects * mesh->getNvertices());

    // Overwrite particles with vertices
    // Overwrite motions with the old_motions
    ROVview fakeView(ov, this);
    fakeView.objSize   = mesh->getNvertices();
    fakeView.size      = mesh->getNvertices() * nObjects;
    fakeView.positions = meshOldVertices.devPtr();
    fakeView.motions   = dataPerObject.getData<RigidMotion>(ChannelNames::oldMotions)->devPtr();

    const int nthreads = 128;
    
    SAFE_KERNEL_LAUNCH(
            RigidIntegrationKernels::applyRigidMotion
                <RigidIntegrationKernels::ApplyRigidMotion::PositionsOnly>,
            getNblocks(fakeView.size, nthreads), nthreads, 0, stream,
            fakeView, ov->mesh->vertexCoordinates.devPtr() );

    return &meshOldVertices;
}

PinnedBuffer<Force>* LocalRigidObjectVector::getMeshForces(hipStream_t stream)
{
    auto ov = dynamic_cast<ObjectVector*>(pv);
    meshForces.resize_anew(nObjects * ov->mesh->getNvertices());
    return &meshForces;
}




RigidObjectVector::RigidObjectVector(const YmrState *state, std::string name, float partMass,
                                     float3 J, const int objSize,
                                     std::shared_ptr<Mesh> mesh, const int nObjects) :
    ObjectVector( state, name, partMass, objSize,
                  std::make_unique<LocalRigidObjectVector>(this, objSize, nObjects),
                  std::make_unique<LocalRigidObjectVector>(this, objSize, 0) ),
    J(J)
{
    this->mesh = std::move(mesh);

    if (length(J) < 1e-5)
        die("Wrong momentum of inertia: [%f %f %f]", J.x, J.y, J.z);

    if (J.x < 0 || J.y < 0 || J.z < 0)
        die("Inertia tensor must be positive; got [%f %f %f]", J.x, J.y, J.z);


    // rigid motion must be exchanged and shifted
    requireDataPerObject<RigidMotion>(ChannelNames::motions,
                                      DataManager::PersistenceMode::Persistent,
                                      sizeof(RigidReal));

    requireDataPerObject<RigidMotion>(ChannelNames::oldMotions,
                                      DataManager::PersistenceMode::None);
}

RigidObjectVector::RigidObjectVector(const YmrState *state, std::string name, float partMass,
                                     PyTypes::float3 J, const int objSize,
                                     std::shared_ptr<Mesh> mesh, const int nObjects) :
    RigidObjectVector( state, name, partMass, make_float3(J), objSize, mesh, nObjects )
{}

RigidObjectVector::~RigidObjectVector() = default;

// TODO refactor this

static void splitMotions(DomainInfo domain, const PinnedBuffer<RigidMotion>& motions,
                         std::vector<float> &pos, std::vector<RigidReal4> &quaternion,
                         std::vector<RigidReal3> &vel, std::vector<RigidReal3> &omega,
                         std::vector<RigidReal3> &force, std::vector<RigidReal3> &torque)
{
    int n = motions.size();
    pos  .resize(3*n); quaternion.resize(n);
    vel  .resize(n);        omega.resize(n);
    force.resize(n);       torque.resize(n);

    float3 *pos3 = (float3*) pos.data();
    
    for (int i = 0; i < n; ++i) {
        auto m = motions[i];
        pos3[i] = domain.local2global(make_float3(m.r));
        quaternion[i] = m.q;
        vel[i] = m.vel;
        omega[i] = m.omega;
        force[i] = m.force;
        torque[i] = m.torque;
    }
}

void RigidObjectVector::_checkpointObjectData(MPI_Comm comm, std::string path, int checkpointId)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointNameWithId(path, "ROV", "", checkpointId);
    info("Checkpoint for rigid object vector '%s', writing to file %s", name.c_str(), filename.c_str());

    auto motions = local()->dataPerObject.getData<RigidMotion>(ChannelNames::motions);

    motions->downloadFromDevice(defaultStream, ContainersSynch::Synch);
    
    auto positions = std::make_shared<std::vector<float>>();
    std::vector<RigidReal4> quaternion;
    std::vector<RigidReal3> vel, omega, force, torque;
    
    splitMotions(state->domain, *motions, *positions, quaternion, vel, omega, force, torque);

    XDMF::VertexGrid grid(positions, comm);    

    auto rigidType = XDMF::getNumberType<RigidReal>();

    std::vector<XDMF::Channel> channels = {
        XDMF::Channel( "quaternion", quaternion .data(), XDMF::Channel::DataForm::Quaternion, rigidType, DataTypeWrapper<RigidReal4>() ),
        XDMF::Channel( "velocity",   vel        .data(), XDMF::Channel::DataForm::Vector,     rigidType, DataTypeWrapper<RigidReal3>() ),
        XDMF::Channel( "omega",      omega      .data(), XDMF::Channel::DataForm::Vector,     rigidType, DataTypeWrapper<RigidReal3>() ),
        XDMF::Channel( "force",      force      .data(), XDMF::Channel::DataForm::Vector,     rigidType, DataTypeWrapper<RigidReal3>() ),
        XDMF::Channel( "torque",     torque     .data(), XDMF::Channel::DataForm::Vector,     rigidType, DataTypeWrapper<RigidReal3>() )
    };         

    _extractPersistentExtraObjectData(channels, /* blacklist */ {ChannelNames::motions} );
    
    XDMF::write(filename, &grid, channels, comm);

    createCheckpointSymlink(comm, path, "ROV", "xmf", checkpointId);

    debug("Checkpoint for object vector '%s' successfully written", name.c_str());
}

static void shiftCoordinates(const DomainInfo& domain, std::vector<RigidMotion>& motions)
{
    for (auto& m : motions)
        m.r = make_rigidReal3( domain.global2local(make_float3(m.r)) );
}

void RigidObjectVector::_restartObjectData(MPI_Comm comm, std::string path, const std::vector<int>& map)
{
    CUDA_Check( hipDeviceSynchronize() );

    auto filename = createCheckpointName(path, "ROV", "xmf");
    info("Restarting rigid object vector %s from file %s", name.c_str(), filename.c_str());

    XDMF::readRigidObjectData(filename, comm, this);

    auto loc_ids     = local()->dataPerObject.getData<int64_t>(ChannelNames::globalIds);
    auto loc_motions = local()->dataPerObject.getData<RigidMotion>(ChannelNames::motions);
    
    std::vector<int64_t>         ids(loc_ids->size());
    std::vector<RigidMotion> motions(loc_motions->size());
    
    std::copy(loc_ids    ->begin(), loc_ids    ->end(), ids.begin());
    std::copy(loc_motions->begin(), loc_motions->end(), motions.begin());
    
    RestartHelpers::exchangeData(comm, map, ids, 1);
    RestartHelpers::exchangeData(comm, map, motions, 1);

    shiftCoordinates(state->domain, motions);
    
    loc_ids->resize_anew(ids.size());
    loc_motions->resize_anew(motions.size());

    std::copy(ids.begin(), ids.end(), loc_ids->begin());
    std::copy(motions.begin(), motions.end(), loc_motions->begin());

    loc_ids->uploadToDevice(defaultStream);
    loc_motions->uploadToDevice(defaultStream);
    CUDA_Check( hipDeviceSynchronize() );

    info("Successfully read %d object infos", loc_motions->size());
}
