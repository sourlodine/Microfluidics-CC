#include "hip/hip_runtime.h"
#include "mesh.h"

#include <fstream>
#include <unordered_map>
#include <map>
#include <vector>

#include <core/utils/cuda_common.h>

/// Read off mesh
Mesh::Mesh(std::string fname)
{
    std::ifstream fin(fname);
    if (!fin.good())
        die("Mesh file '%s' not found", fname.c_str());

    debug("Reading mesh from file '%s'", fname.c_str());

    std::string line;
    std::getline(fin, line); // OFF header

    int nedges;
    fin >> nvertices >> ntriangles >> nedges;
    std::getline(fin, line); // Finish with this line

    // Read the vertex coordinates
    vertexCoordinates.resize_anew(nvertices);
    for (int i=0; i<nvertices; i++)
        fin >> vertexCoordinates[i].x >> vertexCoordinates[i].y >> vertexCoordinates[i].z;

    // Read the connectivity data
    triangles.resize_anew(ntriangles);
    for (int i=0; i<ntriangles; i++)
    {
        int number;
        fin >> number;
        if (number != 3)
            die("Bad mesh file '%s' on line %d, number of face vertices is %d instead of 3",
                    fname.c_str(), 3 /* header */ + nvertices + i, number);

        fin >> triangles[i].x >> triangles[i].y >> triangles[i].z;

        auto check = [&] (int tr) {
            if (tr < 0 || tr >= nvertices)
                die("Bad triangle indices in mesh '%s' on line %d", fname.c_str(), 3 /* header */ + nvertices + i);
        };

        check(triangles[i].x);
        check(triangles[i].y);
        check(triangles[i].z);
    }


    vertexCoordinates.uploadToDevice(0);
    triangles.uploadToDevice(0);

    _computeMaxDegree();
}

void Mesh::_computeMaxDegree()
{
    std::vector<int> degrees(nvertices);

    for (auto t : triangles) {
        degrees[t.x] ++;
        degrees[t.y] ++;
        degrees[t.z] ++;
    }

    maxDegree = *std::max_element(degrees.begin(), degrees.end());
    fprintf(stderr, "max degree is %d\n", maxDegree);
}

MembraneMesh::MembraneMesh(std::string fname) : Mesh(fname)
{
    findAdjacent();

    initialLengths.resize_anew(nvertices * maxDegree);

    for (int i=0; i<nvertices*maxDegree; i++)
    {
        if (adjacent[i] >= 0)
            initialLengths[i] = length(vertexCoordinates[i / maxDegree] - vertexCoordinates[adjacent[i]]);
    }

    initialLengths.uploadToDevice(0);
}


void MembraneMesh::findAdjacent()
{
    std::vector< std::map<int, int> > adjacentPairs(nvertices);

    for(int i = 0; i < triangles.size(); ++i)
    {
        const int tri[3] = {triangles[i].x, triangles[i].y, triangles[i].z};

        for(int d = 0; d < 3; ++d)
            adjacentPairs[tri[d]][tri[(d + 1) % 3]] = tri[(d + 2) % 3];
    }

    degrees.resize_anew(nvertices);
    for(int i = 0; i < nvertices; ++i)
        degrees[i] = adjacentPairs[i].size();

    auto it = std::max_element(degrees.hostPtr(), degrees.hostPtr() + nvertices);
    const int curMaxDegree = *it;

    if (curMaxDegree > maxDegree)
        die("Degree of vertex %d is %d > %d (max degree supported)", (int)(it - degrees.hostPtr()), curMaxDegree, maxDegree);

    debug("Max degree of mesh vertices is %d", curMaxDegree);

    // Find first (nearest) neighbors of each vertex
    adjacent.resize_anew(ntriangles * maxDegree);
    for (int i=0; i<adjacent.size(); i++)
        adjacent[i] = -1;

    for(int v = 0; v < nvertices; ++v)
    {
        auto& l = adjacentPairs[v];

        adjacent[0 + maxDegree * v] = l.begin()->first;
        int last = adjacent[1 + maxDegree * v] = l.begin()->second;

        for(int i = 2; i < l.size(); ++i)
        {
            assert(l.find(last) != l.end());

            int tmp = adjacent[i + maxDegree * v] = l.find(last)->second;
            last = tmp;
        }
    }


    // Find distance 2 neighbors of each vertex
    adjacent_second.resize_anew(ntriangles * maxDegree);
    for (int i=0; i<adjacent_second.size(); i++)
        adjacent_second[i] = -1;

    // Get all the vertex neighbors from already compiled adjacent array
    auto extract_neighbors = [&] (const int v) {

        std::vector<int> myneighbors;
        for(int c = 0; c < maxDegree; ++c)
        {
            const int val = adjacent[c + maxDegree * v];
            if (val == -1)
                break;

            myneighbors.push_back(val);
        }

        return myneighbors;
    };

    for(int v = 0; v < nvertices; ++v)
    {
        auto myneighbors = extract_neighbors(v);

        for(int i = 0; i < myneighbors.size(); ++i)
        {
            auto s1 = extract_neighbors(myneighbors[i]);
            std::sort(s1.begin(), s1.end());

            auto s2 = extract_neighbors(myneighbors[(i + 1) % myneighbors.size()]);
            std::sort(s2.begin(), s2.end());

            std::vector<int> result(s1.size() + s2.size());

            const int nterms = std::set_intersection(s1.begin(), s1.end(), s2.begin(), s2.end(),
                    result.begin()) - result.begin();

            assert(nterms == 2);

            const int myguy = result[0] == v;

            adjacent_second[i + maxDegree * v] = result[myguy];
        }
    }


    for(int v = 0; v < nvertices; ++v)
    {
        for (int i=0; i<maxDegree; i++)
            if (adjacent[v*maxDegree + i] == -1)
            {
                adjacent[v*maxDegree + i] = adjacent[v*maxDegree];
                break;
            }

        for (int i=0; i<maxDegree; i++)
            if (adjacent_second[v*maxDegree + i] == -1)
            {
                adjacent_second[v*maxDegree + i] = adjacent_second[v*maxDegree];
                break;
            }
    }

    adjacent.uploadToDevice(0);
    adjacent_second.uploadToDevice(0);
    degrees.uploadToDevice(0);
}



