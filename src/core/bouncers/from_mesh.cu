#include "hip/hip_runtime.h"
#include "from_mesh.h"
#include "drivers/mesh.h"
#include "kernels/api.h"

#include <core/celllist.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/ov.h>
#include <core/rigid/operations.h>
#include <core/utils/kernel_launch.h>

/**
 * Create the bouncer
 * @param name unique bouncer name
 * @param kBT temperature which will be used to create a particle
 * velocity after the bounce, @see performBouncing()
 */
BounceFromMesh::BounceFromMesh(const MirState *state, const std::string& name, VarBounceKernel varBounceKernel) :
    Bouncer(state, name),
    varBounceKernel(varBounceKernel)
{}

BounceFromMesh::~BounceFromMesh() = default;

/**
 * @param ov will need an 'old_particles' per PARTICLE channel keeping positions
 * from the previous timestep.
 * This channel has to be communicated with the objects
 */
void BounceFromMesh::setup(ObjectVector *ov)
{
    Bouncer::setup(ov);

    // If the object is rigid, we need to collect the forces into the RigidMotion
    rov = dynamic_cast<RigidObjectVector*> (ov);

    // for NON-rigid objects:
    //
    // old positions HAVE to be known when the mesh travels to other ranks
    // shift HAS be applied as well
    //
    // for Rigid:
    // old motions HAVE to be there and communicated and shifted

    if (rov == nullptr)
        ov->requireDataPerParticle<float4> (ChannelNames::oldPositions, DataManager::PersistenceMode::Active, DataManager::ShiftMode::Active);
    else
        ov->requireDataPerObject<RigidMotion> (ChannelNames::oldMotions, DataManager::PersistenceMode::Active, DataManager::ShiftMode::Active);
}

void BounceFromMesh::setPrerequisites(ParticleVector *pv)
{
    // do not set it to persistent because bounce happens after integration
    pv->requireDataPerParticle<float4> (ChannelNames::oldPositions, DataManager::PersistenceMode::None, DataManager::ShiftMode::Active);
}

std::vector<std::string> BounceFromMesh::getChannelsToBeExchanged() const
{
    if (rov)
        return {ChannelNames::motions, ChannelNames::oldMotions};
    else
        return {ChannelNames::oldPositions};
}

std::vector<std::string> BounceFromMesh::getChannelsToBeSentBack() const
{
    if (rov)
        return {ChannelNames::motions};
    else
        // return {ChannelNames::forces};
        return {};
}

/**
 * Bounce particles from objects with meshes
 */
void BounceFromMesh::exec(ParticleVector *pv, CellList *cl, ParticleVectorLocality locality, hipStream_t stream)
{
    auto activeOV = ov->get(locality);

    debug("Bouncing %d '%s' particles from %d '%s' objects (%s)",
          pv->local()->size(), pv->name.c_str(),
          activeOV->nObjects,  ov->name.c_str(),
          getParticleVectorLocalityStr(locality).c_str());

    ov->findExtentAndCOM(stream, locality);

    const int totalTriangles = ov->mesh->getNtriangles() * activeOV->nObjects;

    // Set maximum possible number of _coarse_ and _fine_ collisions with triangles
    // In case of crash, the estimate should be increased
    const int maxCoarseCollisions = coarseCollisionsPerTri * totalTriangles;
    coarseTable.collisionTable.resize_anew(maxCoarseCollisions);
    coarseTable.nCollisions.clear(stream);
    MeshBounceKernels::TriangleTable devCoarseTable { maxCoarseCollisions,
                                                      coarseTable.nCollisions.devPtr(),
                                                      coarseTable.collisionTable.devPtr() };
    
    int maxFineCollisions = fineCollisionsPerTri * totalTriangles;
    fineTable.collisionTable.resize_anew(maxFineCollisions);
    fineTable.nCollisions.clear(stream);
    MeshBounceKernels::TriangleTable devFineTable { maxFineCollisions,
                                                    fineTable.nCollisions.devPtr(),
                                                    fineTable.collisionTable.devPtr() };

    // Setup collision times array. For speed and simplicity initial time will be 0,
    // and after the collisions detected its i-th element will be t_i-1.0f, where 0 <= t_i <= 1
    // is the collision time, or 0 if no collision with the particle found
    collisionTimes.resize_anew(pv->local()->size());
    collisionTimes.clear(stream);

    const int nthreads = 128;

    // FIXME this is a hack
    if (rov)
    {
        if (locality == ParticleVectorLocality::Local)
            rov->local()->getMeshForces(stream)->clear(stream);
        else
            rov->halo()-> getMeshForces(stream)->clear(stream);
    }


    OVviewWithNewOldVertices vertexView(ov, activeOV, stream);
    PVviewWithOldParticles pvView(pv, pv->local());

    // Step 1, find all the candidate collisions
    SAFE_KERNEL_LAUNCH(
            MeshBounceKernels::findBouncesInMesh,
            getNblocks(totalTriangles, nthreads), nthreads, 0, stream,
            vertexView, pvView, ov->mesh.get(), cl->cellInfo(), devCoarseTable );

    coarseTable.nCollisions.downloadFromDevice(stream);
    debug("Found %d triangle collision candidates", coarseTable.nCollisions[0]);

    if (coarseTable.nCollisions[0] > maxCoarseCollisions)
        die("Found too many triangle collision candidates (coarse) (%d, max %d),"
            "something may be broken or you need to increase the estimate",
            coarseTable.nCollisions[0], maxCoarseCollisions);

    // Step 2, filter the candidates
    SAFE_KERNEL_LAUNCH(
            MeshBounceKernels::refineCollisions,
            getNblocks(coarseTable.nCollisions[0], nthreads), nthreads, 0, stream,
            vertexView, pvView, ov->mesh.get(),
            coarseTable.nCollisions[0], devCoarseTable.indices,
            devFineTable, collisionTimes.devPtr() );

    fineTable.nCollisions.downloadFromDevice(stream);
    debug("Found %d precise triangle collisions", fineTable.nCollisions[0]);

    if (fineTable.nCollisions[0] > maxFineCollisions)
        die("Found too many triangle collisions (precise) (%d, max %d),"
            "something may be broken or you need to increase the estimate",
            fineTable.nCollisions[0], maxFineCollisions);

    // Step 3, resolve the collisions    
    mpark::visit([&](auto& bounceKernel)
    {
        bounceKernel.update(rng);
        
        SAFE_KERNEL_LAUNCH(
            MeshBounceKernels::performBouncingTriangle,
            getNblocks(fineTable.nCollisions[0], nthreads), nthreads, 0, stream,
            vertexView, pvView, ov->mesh.get(),
            fineTable.nCollisions[0], devFineTable.indices, collisionTimes.devPtr(),
            state->dt, bounceKernel );

    }, varBounceKernel);

    if (rov)
    {
        // make a fake view with vertices instead of particles
        ROVview view(rov, rov->get(locality));
        view.objSize   = ov->mesh->getNvertices();
        view.size      = view.nObjects * view.objSize;
        view.positions = vertexView.vertices;
        view.forces    = vertexView.vertexForces;

        RigidOperations::collectRigidForces(view, stream);
    }
}
