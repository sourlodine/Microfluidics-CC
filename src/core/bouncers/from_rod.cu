#include "hip/hip_runtime.h"
#include "from_rod.h"
#include "kernels/rod.h"

#include <core/celllist.h>
#include <core/pvs/rod_vector.h>
#include <core/pvs/views/rv.h>
#include <core/utils/kernel_launch.h>

BounceFromRod::BounceFromRod(const YmrState *state, std::string name, float radius, float kbT) :
    Bouncer(state, name),
    radius(radius),
    kbT(kbT)
{}

BounceFromRod::~BounceFromRod() = default;

void BounceFromRod::setup(ObjectVector *ov)
{
    Bouncer::setup(ov);

    rv = dynamic_cast<RodVector*> (ov);

    if (rv == nullptr)
        die("bounce from rod must be used with a rod vector");

    ov->requireDataPerParticle<float4> (ChannelNames::oldPositions, DataManager::PersistenceMode::Persistent, sizeof(float));
}

std::vector<std::string> BounceFromRod::getChannelsToBeExchanged() const
{
    return {ChannelNames::oldPositions};
}

void BounceFromRod::exec(ParticleVector *pv, CellList *cl, bool local, hipStream_t stream)
{
    auto activeRV = local ? rv->local() : rv->halo();

    debug("Bouncing %d '%s' particles from %d '%s' rods (%s)",
          pv->local()->size(), pv->name.c_str(),
          activeRV->nObjects,  rv->name.c_str(),
          local ? "local" : "halo");

    rv->findExtentAndCOM(stream, local ? ParticleVectorType::Local : ParticleVectorType::Halo);

    int totalSegments = activeRV->getNumSegmentsPerRod() * activeRV->nObjects;

    // Set maximum possible number of collisions with segments
    // In case of crash, the estimate should be increased
    int maxCollisions = collisionsPerSeg * totalSegments;
    table.collisionTable.resize_anew(maxCollisions);
    table.nCollisions.clear(stream);
    RodBounceKernels::SegmentTable devCollisionTable { maxCollisions,
                                                       table.nCollisions.devPtr(),
                                                       table.collisionTable.devPtr() };


    // Setup collision times array. For speed and simplicity initial time will be 0,
    // and after the collisions detected its i-th element will be t_i-1.0f, where 0 <= t_i <= 1
    // is the collision time, or 0 if no collision with the particle found
    collisionTimes.resize_anew(pv->local()->size());
    collisionTimes.clear(stream);

    const int nthreads = 128;

    activeRV->forces().clear(stream);

    RVviewWithOldParticles rvView(rv, activeRV);
    PVviewWithOldParticles pvView(pv, pv->local());

    // Step 1, find all the candidate collisions
    SAFE_KERNEL_LAUNCH(
            RodBounceKernels::findBounces,
            getNblocks(totalSegments, nthreads), nthreads, 0, stream,
            rvView, radius, pvView, cl->cellInfo(), devCollisionTable, collisionTimes.devPtr() );

    table.nCollisions.downloadFromDevice(stream);
    int nCollisions = table.nCollisions[0];
    debug("Found %d rod collision candidates", nCollisions);

    if (table.nCollisions[0] > maxCollisions)
        die("Found too many rod collisions (%d),"
            "something may be broken or you need to increase the estimate", nCollisions);

    // Step 2, resolve the collisions
    SAFE_KERNEL_LAUNCH(
            RodBounceKernels::performBouncing,
            getNblocks(table.nCollisions[0], nthreads), nthreads, 0, stream,
            rvView, radius, pvView, nCollisions, devCollisionTable.indices, collisionTimes.devPtr(),
            state->dt, kbT, drand48(), drand48() );
}
