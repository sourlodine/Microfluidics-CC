#include "hip/hip_runtime.h"
#include "from_file.h"

#include <fstream>
#include <texture_types.h>
#include <core/utils/kernel_launch.h>
#include <core/utils/cuda_common.h>

namespace InterpolateKernels
{
__device__ inline float cubicInterpolate1D(float y[4], float mu)
{
    // mu == 0 at y[1], mu == 1 at y[2]
    const float a0 = -0.5f*y[0] + 1.5f*y[1] - 1.5f*y[2] + 0.5f*y[3];
    const float a1 = y[0] - 2.5f*y[1] + 2.0f*y[2] - 0.5f*y[3];
    const float a2 = -0.5f*y[0] + 0.5f*y[2];
    const float a3 = y[1];

    return ((a0*mu + a1)*mu + a2)*mu + a3;
}

__global__ void cubicInterpolate3D(const float* in, int3 inDims, float3 inH, float* out, int3 outDims, float3 outH, float3 offset, float scalingFactor)
{
    // Inspired by http://paulbourke.net/miscellaneous/interpolation/
    // Origin of the output domain is in offset
    // Origin of the input domain is in (0,0,0)

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= outDims.x || iy >= outDims.y || iz >= outDims.z) return;

    float interp2D[4][4];
    float interp1D[4];

    // Coordinates where to interpolate
    float3 outputId  = make_float3(ix, iy, iz);
    float3 outputCoo = outputId*outH;

    float3 inputCoo  = outputCoo + offset;

    // Make sure we're within the region where the input data is defined
    assert( 0.0f <= inputCoo.x && inputCoo.x <= inDims.x*inH.x &&
            0.0f <= inputCoo.y && inputCoo.y <= inDims.y*inH.y &&
            0.0f <= inputCoo.z && inputCoo.z <= inDims.z*inH.z    );

    // Reference point of the original grid, rounded down
    int3 inputId_down = make_int3( floorf(inputCoo / inH) );
    float3 mu = (inputCoo - make_float3(inputId_down)*inH) / inH;

    // Interpolate along x
    for (int dz = -1; dz <= 2; dz++)
        for (int dy = -1; dy <= 2; dy++)
        {
            float vals[4];

            for (int dx = -1; dx <= 2; dx++)
            {
                int3 delta{dx, dy, dz};
                const int3 curInputId = (inputId_down+delta + inDims) % inDims;

                vals[dx+1] = in[ (curInputId.z*inDims.y + curInputId.y) * inDims.x + curInputId.x ] * scalingFactor;
            }

            interp2D[dz+1][dy+1] = cubicInterpolate1D(vals, mu.x);
        }

    // Interpolate along y
    for (int dz = 0; dz <= 3; dz++)
        interp1D[dz] = cubicInterpolate1D(interp2D[dz], mu.y);

    // Interpolate along z
    out[ (iz*outDims.y + iy) * outDims.x + ix ] = cubicInterpolate1D(interp1D, mu.z);    
}



__device__ inline float interpolationKernel(float3 x, float3 x0)
{
    //const int p = 8;
    const float3 r = x-x0;
    const float l2 = dot(r, r);
    const float l4 = l2*l2;
    
    return l4*l4;
}

__global__ void inverseDistanceWeightedInterpolation(const float* in, int3 inDims, float3 inH, float* out, int3 outDims, float3 outH, float3 offset, float scalingFactor)
{
    // Inspired by http://paulbourke.net/miscellaneous/interpolation/
    // Origin of the output domain is in offset
    // Origin of the input domain is in (0,0,0)

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= outDims.x || iy >= outDims.y || iz >= outDims.z) return;

    // Coordinates where to interpolate
    float3 outputId  = make_float3(ix, iy, iz);
    float3 outputCoo = outputId*outH;

    float3 inputCoo  = outputCoo + offset;

    // Make sure we're within the region where the input data is defined
    assert( 0.0f <= inputCoo.x && inputCoo.x <= inDims.x*inH.x &&
            0.0f <= inputCoo.y && inputCoo.y <= inDims.y*inH.y &&
            0.0f <= inputCoo.z && inputCoo.z <= inDims.z*inH.z    );

    // Reference point of the original grid, rounded down
    int3 inputId_down = make_int3( floorf(inputCoo / inH) );
    
    float nominator = 0, denominator = 0;

    // Interpolate along x
    for (int dz = -1; dz <= 2; dz++)
        for (int dy = -1; dy <= 2; dy++)
            for (int dx = -1; dx <= 2; dx++)
            {
                int3 delta{dx, dy, dz};
                const int3 curInputId = (inputId_down+delta + inDims) % inDims;
                const float3 curInputCoo = make_float3(curInputId)*inH;
                
                const float k = interpolationKernel(inputCoo, curInputCoo);
                nominator += in[ (curInputId.z*inDims.y + curInputId.y) * inDims.x + curInputId.x ] * k;
                denominator += k;
            }

    out[ (iz*outDims.y + iy) * outDims.x + ix ] = scalingFactor * nominator / denominator;
}
} // InterpolateKernels

static void readHeader(const std::string fileName, const MPI_Comm& comm, int3& sdfResolution, float3& sdfExtent, int64_t& fullSdfSize_byte, int64_t& endHeader_byte, int rank)
{
    if (rank == 0)
    {
        std::ifstream file(fileName);
        if (!file.good())
            die("File not found or not accessible");

        auto fstart = file.tellg();

        file >> sdfExtent.x >> sdfExtent.y >> sdfExtent.z >>
            sdfResolution.x >> sdfResolution.y >> sdfResolution.z;
        fullSdfSize_byte = (int64_t)sdfResolution.x * sdfResolution.y * sdfResolution.z * sizeof(float);

        info("Using field file '%s' of size %.2fx%.2fx%.2f and resolution %dx%dx%d", fileName.c_str(),
                sdfExtent.x, sdfExtent.y, sdfExtent.z,
                sdfResolution.x, sdfResolution.y, sdfResolution.z);

        file.seekg( 0, std::ios::end );
        auto fend = file.tellg();

        endHeader_byte = (fend - fstart) - fullSdfSize_byte;

        file.close();
    }

    MPI_Check( MPI_Bcast(&sdfExtent,        3, MPI_FLOAT,     0, comm) );
    MPI_Check( MPI_Bcast(&sdfResolution,    3, MPI_INT,       0, comm) );
    MPI_Check( MPI_Bcast(&fullSdfSize_byte, 1, MPI_INT64_T,   0, comm) );
    MPI_Check( MPI_Bcast(&endHeader_byte,   1, MPI_INT64_T,   0, comm) );
}

static void readSdf(const std::string fileName, const MPI_Comm& comm, int64_t fullSdfSize_byte, int64_t endHeader_byte, int nranks, int rank, std::vector<float>& fullSdfData)
{
    // Read part and allgather
    const int64_t readPerProc_byte = (fullSdfSize_byte + nranks - 1) / (int64_t)nranks;
    std::vector<char> readBuffer(readPerProc_byte);

    // Limits in bytes
    const int64_t readStart = readPerProc_byte * rank + endHeader_byte;
    const int64_t readEnd   = std::min( readStart + readPerProc_byte, fullSdfSize_byte + endHeader_byte);

    MPI_File fh;
    MPI_Status status;
    MPI_Check( MPI_File_open(comm, fileName.c_str(), MPI_MODE_RDONLY, MPI_INFO_NULL, &fh) );  // TODO: MPI_Info
    MPI_Check( MPI_File_read_at_all(fh, readStart, readBuffer.data(), readEnd - readStart, MPI_BYTE, &status) );
    // TODO: check that we read just what we asked
    // MPI_Get_count only return int though

    fullSdfData.resize(readPerProc_byte * nranks / sizeof(float));  // May be bigger than fullSdfSize, to make gather easier
    MPI_Check( MPI_Allgather(readBuffer.data(), readPerProc_byte, MPI_BYTE, fullSdfData.data(), readPerProc_byte, MPI_BYTE, comm) );
}

static void prepareRelevantSdfPiece(const float* fullSdfData, float3 extendedDomainStart, float3 extendedDomainSize,
                                    float3 initialSdfH, int3 initialSdfResolution,
                                    int3& resolution, float3& offset, PinnedBuffer<float>& localSdfData)
{
    // Find your relevant chunk of data
    // We cannot send big sdf files directly, so we'll carve a piece now

    const int margin = 3; // +2 from cubic interpolation, +1 from possible round-off errors
    const int3 startId = make_int3( floorf( extendedDomainStart                     / initialSdfH) ) - margin;
    const int3 endId   = make_int3( ceilf ((extendedDomainStart+extendedDomainSize) / initialSdfH) ) + margin;

    float3 startInLocalCoord = make_float3(startId)*initialSdfH - (extendedDomainStart + 0.5*extendedDomainSize);
    offset = -0.5*extendedDomainSize - startInLocalCoord;

    resolution = endId - startId;

    localSdfData.resize( resolution.x * resolution.y * resolution.z, 0 );
    auto locSdfDataPtr = localSdfData.hostPtr();

    for (int k = 0; k < resolution.z; k++)
        for (int j = 0; j < resolution.y; j++)
            for (int i = 0; i < resolution.x; i++)
            {
                const int origIx = (i+startId.x + initialSdfResolution.x) % initialSdfResolution.x;
                const int origIy = (j+startId.y + initialSdfResolution.y) % initialSdfResolution.y;
                const int origIz = (k+startId.z + initialSdfResolution.z) % initialSdfResolution.z;

                locSdfDataPtr[ (k*resolution.y + j)*resolution.x + i ] =
                        fullSdfData[ (origIz*initialSdfResolution.y + origIy)*initialSdfResolution.x + origIx ];
            }
}

FieldFromFile::FieldFromFile(const YmrState *state, std::string name, std::string fieldFileName, float3 h) :
    Field(state, name, h),
    fieldFileName(fieldFileName)
{}

FieldFromFile::~FieldFromFile() = default;

FieldFromFile::FieldFromFile(FieldFromFile&&) = default;

void FieldFromFile::setup(const MPI_Comm& comm)
{
    info("Setting up field from %s", fieldFileName.c_str());

    const auto domain = state->domain;
    
    CUDA_Check( hipDeviceSynchronize() );

    int nranks, rank;
    int ranks[3], periods[3], coords[3];
    MPI_Check( MPI_Comm_size(comm, &nranks) );
    MPI_Check( MPI_Comm_rank(comm, &rank) );
    MPI_Check( MPI_Cart_get (comm, 3, ranks, periods, coords) );

    int3 initialSdfResolution;
    float3 initialSdfExtent;

    int64_t fullSdfSize_byte;
    int64_t endHeader_byte;

    // Read header
    readHeader(fieldFileName, comm, initialSdfResolution, initialSdfExtent, fullSdfSize_byte, endHeader_byte, rank);
    float3 initialSdfH = domain.globalSize / make_float3(initialSdfResolution-1);

    // Read heavy data
    std::vector<float> fullSdfData;
    readSdf(fieldFileName, comm, fullSdfSize_byte, endHeader_byte, nranks, rank, fullSdfData);

    const float3 scale3 = domain.globalSize / initialSdfExtent;
    if ( fabs(scale3.x - scale3.y) > 1e-5 || fabs(scale3.x - scale3.z) > 1e-5 )
        die("Sdf size and domain size mismatch");
    const float lenScalingFactor = (scale3.x + scale3.y + scale3.z) / 3;

    int3 resolutionBeforeInterpolation;
    float3 offset;
    PinnedBuffer<float> localData;
    prepareRelevantSdfPiece(fullSdfData.data(), domain.globalStart - margin3, extendedDomainSize,
                            initialSdfH, initialSdfResolution,
                            resolutionBeforeInterpolation, offset, localData);

    // Interpolate
    DeviceBuffer<float> fieldRawData (resolution.x * resolution.y * resolution.z);

    dim3 threads(8, 8, 8);
    dim3 blocks((resolution.x+threads.x-1) / threads.x,
                (resolution.y+threads.y-1) / threads.y,
                (resolution.z+threads.z-1) / threads.z);

    localData.uploadToDevice(defaultStream);
    SAFE_KERNEL_LAUNCH(
            InterpolateKernels::cubicInterpolate3D,
            blocks, threads, 0, defaultStream,
            localData.devPtr(), resolutionBeforeInterpolation, initialSdfH,
            fieldRawData.devPtr(), resolution, h, offset, lenScalingFactor );

    setupArrayTexture(fieldRawData.devPtr());
}
