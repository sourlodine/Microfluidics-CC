#include "hip/hip_runtime.h"
#include "freeze_particles.h"

#include <core/logger.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

#include <core/walls/simple_stationary_wall.h>


template<bool QUERY>
__global__ void collectFrozen(PVview view, float* sdfs, float minVal, float maxVal, float4* frozen, int* nFrozen)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    Particle p(view.particles, pid);
    p.u = make_float3(0);

    const float val = sdfs[pid];
    
    if (val > minVal && val < maxVal)
    {
        const int ind = atomicAggInc(nFrozen);

        if (!QUERY)
            p.write2Float4(frozen, ind);
    }
}

void freezeParticlesInWall(SDF_basedWall* wall, ParticleVector* pv, float minVal, float maxVal)
{
    CUDA_Check( hipDeviceSynchronize() );

    DeviceBuffer<float> sdfs(pv->local()->size());

    wall->sdfPerParticle(pv->local(), &sdfs, nullptr, 0);

    PinnedBuffer<int> nFrozen(1);

    PVview view(pv, pv->local());
    const int nthreads = 128;
    const int nblocks = getNblocks(view.size, nthreads);

    nFrozen.clear(0);
    SAFE_KERNEL_LAUNCH(collectFrozen<true>,
                nblocks, nthreads, 0, 0,
                view, sdfs.devPtr(), minVal, maxVal,
                nullptr, nFrozen.devPtr());

    nFrozen.downloadFromDevice(0);

    PinnedBuffer<Particle> frozen(nFrozen[0]);
    info("Freezing %d particles", nFrozen[0]);
    pv->local()->resize(nFrozen[0], 0);

    nFrozen.clear(0);
    SAFE_KERNEL_LAUNCH(collectFrozen<false>,
            nblocks, nthreads, 0, 0,
            view, sdfs.devPtr(), minVal, maxVal,
            (float4*)frozen.devPtr(), nFrozen.devPtr());
    
    CUDA_Check( hipDeviceSynchronize() );
    std::swap(frozen, pv->local()->coosvels);
}


