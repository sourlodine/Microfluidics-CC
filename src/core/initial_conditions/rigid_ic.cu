#include "hip/hip_runtime.h"
#include "rigid_ic.h"

#include <random>
#include <fstream>

#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>
#include <core/integrators/rigid_vv.h>

#include <core/rigid_kernels/rigid_motion.h>

RigidIC::RigidIC(std::string xyzfname, std::string icfname) :
    icfname(icfname), xyzfname(xyzfname)
{   }

RigidIC::~RigidIC() = default;


void static readXYZ(std::string fname, PinnedBuffer<float4>& positions, hipStream_t stream)
{
	int n;
	float dummy;
	std::string line;

	std::ifstream fin(fname);
	if (!fin.good())
		die("XYZ ellipsoid file %s not found", fname.c_str());
	fin >> n;

	// skip the comment line
	std::getline(fin, line);
	std::getline(fin, line);

	positions.resize_anew(n);
	for (int i=0; i<n; i++)
		fin >> dummy >> positions[i].x >>positions[i].y >>positions[i].z;

	positions.uploadToDevice(stream);
}

void RigidIC::exec(const MPI_Comm& comm, ParticleVector* pv, DomainInfo domain, hipStream_t stream)
{
	auto ov = dynamic_cast<RigidObjectVector*>(pv);
	if (ov == nullptr)
		die("Can only generate rigid object vector");

	pv->domain = domain;

	readXYZ(xyzfname, ov->initialPositions, stream);
	if (ov->objSize != ov->initialPositions.size())
		die("Object size and XYZ initial conditions don't match in size for %s", ov->name.c_str());

	std::ifstream fic(icfname);
	int nObjs=0;

	HostBuffer<RigidMotion> motions;

	while (true)
	{
		RigidMotion motion{};

		fic >> motion.r.x >> motion.r.y >> motion.r.z;
		fic >> motion.q.x >> motion.q.y >> motion.q.z >> motion.q.w;

		if (fic.fail()) break;

		motion.q = normalize(motion.q);

		if (ov->domain.globalStart.x <= motion.r.x && motion.r.x < ov->domain.globalStart.x + ov->domain.localSize.x &&
		    ov->domain.globalStart.y <= motion.r.y && motion.r.y < ov->domain.globalStart.y + ov->domain.localSize.y &&
		    ov->domain.globalStart.z <= motion.r.z && motion.r.z < ov->domain.globalStart.z + ov->domain.localSize.z)
		{
			motion.r = make_rigidReal3( ov->domain.global2local(make_float3(motion.r)) );
			motions.resize(nObjs + 1);
			motions[nObjs] = motion;
			nObjs++;
		}
	}

	ov->local()->resize_anew(nObjs * ov->objSize);

	auto ovMotions = ov->local()->extraPerObject.getData<RigidMotion>("motions");
	ovMotions->copy(motions);
	ovMotions->uploadToDevice(stream);

	// Set ids
	int totalCount=0; // TODO: int64!
	MPI_Check( MPI_Exscan(&nObjs, &totalCount, 1, MPI_INT, MPI_SUM, comm) );

	auto ids = ov->local()->extraPerObject.getData<int>("ids");
	for (int i=0; i<nObjs; i++)
		(*ids)[i] = totalCount + i;


	for (int i=0; i < ov->local()->size(); i++)
	{
		Particle p(make_float4(0), make_float4(0));
		p.i1 = totalCount*ov->objSize + i;
		ov->local()->coosvels[i] = p;
	}

	ids->uploadToDevice(stream);
	ov->local()->coosvels.uploadToDevice(stream);
	ov->local()->extraPerParticle.getData<Particle>("old_particles")->copy(ov->local()->coosvels, stream);

	info("Read %d %s objects", nObjs, ov->name.c_str());

	// Do the initial rotation
	ov->requireDataPerObject<RigidMotion>("old_motions", false);
	ov->local()->forces.clear(stream);
	IntegratorVVRigid integrator("dummy", 0.0f);
	integrator.stage2(pv, 0, stream);
}

