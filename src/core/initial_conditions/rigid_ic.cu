#include "hip/hip_runtime.h"
#include "rigid_ic.h"

#include <random>
#include <fstream>

#include <core/pvs/particle_vector.h>
#include <core/pvs/rigid_ellipsoid_object_vector.h>
#include <core/integrators/rigid_vv.h>

#include <core/rigid_kernels/rigid_motion.h>

void static readXYZ(std::string fname, PyContainer& positions)
{
    enum {X=0, Y=1, Z=2};
    int n;
    float dummy;
    std::string line;

    std::ifstream fin(fname);
    if (!fin.good())
        die("XYZ ellipsoid file %s not found", fname.c_str());
    fin >> n;

    // skip the comment line
    std::getline(fin, line);
    std::getline(fin, line);

    positions.resize(n);
    for (int i=0; i<n; i++)
        fin >> dummy >> positions[i][X] >> positions[i][Y] >> positions[i][Z];
}

RigidIC::RigidIC(ICvector com_q, std::string xyzfname) :
    com_q(com_q)
{
    readXYZ(xyzfname, coords);
}

RigidIC::RigidIC(ICvector com_q, const PyContainer& coords) :
    com_q(com_q), coords(coords)
{}

RigidIC::~RigidIC() = default;


static void copyToPinnedBuffer(const PyContainer& in, PinnedBuffer<float4>& out, hipStream_t stream)
{
    enum {X=0, Y=1, Z=2};
    out.resize_anew(in.size());

    for (int i = 0; i < in.size(); ++i)
        out[i] = make_float4(in[i][X], in[i][Y], in[i][Z], 0);
        
    out.uploadToDevice(stream);    
}

void RigidIC::exec(const MPI_Comm& comm, ParticleVector* pv, DomainInfo domain, hipStream_t stream)
{
    auto ov = dynamic_cast<RigidObjectVector*>(pv);
    if (ov == nullptr)
        die("Can only generate rigid object vector");

    pv->domain = domain;

    copyToPinnedBuffer(coords, ov->initialPositions, stream);

    if (ov->objSize != ov->initialPositions.size())
        die("Object size and XYZ initial conditions don't match in size for '%s': %d vs %d",
                ov->name.c_str(), ov->objSize, ov->initialPositions.size());

    int nObjs=0;
    HostBuffer<RigidMotion> motions;

    for (auto& entry : com_q)
    {
        RigidMotion motion{};
        
        motion.r = {entry[0], entry[1], entry[2]};
        motion.q = make_rigidReal4( make_float4(entry[3], entry[4], entry[5], entry[6]) );
        motion.q = normalize(motion.q);

        if (ov->domain.inSubDomain(motion.r))
        {
            motion.r = make_rigidReal3( ov->domain.global2local(make_float3(motion.r)) );
            motions.resize(nObjs + 1);
            motions[nObjs] = motion;
            nObjs++;
        }
    }

    ov->local()->resize_anew(nObjs * ov->objSize);

    auto ovMotions = ov->local()->extraPerObject.getData<RigidMotion>("motions");
    ovMotions->copy(motions);
    ovMotions->uploadToDevice(stream);

    // Set ids
    // Need to do that, as not all the objects in com_q may be valid
    int totalCount=0; // TODO: int64!
    MPI_Check( MPI_Exscan(&nObjs, &totalCount, 1, MPI_INT, MPI_SUM, comm) );

    auto ids = ov->local()->extraPerObject.getData<int>("ids");
    for (int i=0; i<nObjs; i++)
        (*ids)[i] = totalCount + i;


    for (int i=0; i < ov->local()->size(); i++)
    {
        Particle p(make_float4(0), make_float4(0));
        p.i1 = totalCount*ov->objSize + i;
        ov->local()->coosvels[i] = p;
    }

    ids->uploadToDevice(stream);
    ov->local()->coosvels.uploadToDevice(stream);
    ov->local()->extraPerParticle.getData<Particle>("old_particles")->copy(ov->local()->coosvels, stream);

    info("Read %d %s objects", nObjs, ov->name.c_str());

    // Do the initial rotation
    ov->requireDataPerObject<RigidMotion>("old_motions", false);
    ov->local()->forces.clear(stream);
    IntegratorVVRigid integrator("dummy", 0.0f);
    integrator.stage2(pv, 0, stream);
}

