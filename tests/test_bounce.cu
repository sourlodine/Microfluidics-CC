#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public
#define protected public

#include <core/particle_vector.h>
#include <core/celllist.h>
#include <core/mpi/api.h>
#include <core/logger.h>

#include <core/xml/pugixml.hpp>
#include <core/rigid_object_vector.h>

Logger logger;

Particle addShift(Particle p, float a, float b, float c)
{
	Particle res = p;
	res.r.x += a;
	res.r.y += b;
	res.r.z += c;

	return res;
}

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	MPI_Init(&argc, &argv);
	logger.init(MPI_COMM_WORLD, "redist.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	std::string xml = R"(<node mass="1.0" density="8.0">)";
	pugi::xml_document config;
	config.load_string(xml.c_str());

	float3 length{64,64,64};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.0f;
	ParticleVector dpds("dpd");
	CellList cells(&dpds, rc, length);
	cells.setStream(0);
	cells.makePrimary();

	InitialConditions ic = createIC(config.child("node"));
	ic.exec(MPI_COMM_WORLD, &dpds, {0,0,0}, length);

	const int initialNP = dpds.local()->size();
	HostBuffer<Particle> host(dpds.local()->size());
	const float dt = 0.1;
	for (int i=0; i<dpds.local()->size(); i++)
	{
		dpds.local()->coosvels[i].u.z = 5*(drand48() - 0.5);
		dpds.local()->coosvels[i].u.y = 5*(drand48() - 0.5);
		dpds.local()->coosvels[i].u.z = 5*(drand48() - 0.5);

		dpds.local()->coosvels[i].r += dt * dpds.local()->coosvels[i].u;

		host[i] = dpds.local()->coosvels[i];
	}


	const int nobj = 10;
	PinnedBuffer<RigidObjectVector::RigidMovement> movement(nobj);
	PinnedBuffer<RigidObjectVector::COMandExtent> com_ext(nobj);

	for (int i=0; i<nobj; i++)
	{
		movement[i].omega.x = 2*(drand48() - 0.5);
		movement[i].omega.x = 2*(drand48() - 0.5);
		movement[i].omega.x = 2*(drand48() - 0.5);

		movement[i].vel.x = 2*(drand48() - 0.5);
		movement[i].vel.x = 2*(drand48() - 0.5);
		movement[i].vel.x = 2*(drand48() - 0.5);

		movement[i].force = make_float3(0);
		movement[i].torque = make_float3(0);
}





	for (int i = 0; i<27; i++)
	{
		if (bufs[i].size() != redist.helpers[0]->counts[i])
			printf("%2d-th redist differs in size: %5d, expected %5d\n", i, redist.helpers[0]->counts[i], (int)bufs[i].size());

		std::vector<Particle> got, reference;

		auto cmp = [] (Particle a, Particle b) {
			if (a.i1 < b.i1) return true;
			if (a.i1 > b.i1) return false;

			if (a.r.x > b.r.x + 1e-6) return true;
			if (a.r.y > b.r.y + 1e-6) return true;
			if (a.r.z > b.r.z + 1e-6) return true;

			return false;
		};

		std::sort(bufs[i].begin(), bufs[i].end(), cmp);
		std::sort((Particle*)redist.helpers[0]->sendBufs[i].hostPtr(), ((Particle*)redist.helpers[0]->sendBufs[i].hostPtr()) + redist.helpers[0]->counts[i], cmp);

		std::set_difference(bufs[i].begin(), bufs[i].end(),
				(Particle*)redist.helpers[0]->sendBufs[i].hostPtr(), ((Particle*)redist.helpers[0]->sendBufs[i].hostPtr()) + redist.helpers[0]->counts[i],
				std::inserter(reference, reference.begin()), cmp);

		std::set_difference(
					(Particle*)redist.helpers[0]->sendBufs[i].hostPtr(), ((Particle*)redist.helpers[0]->sendBufs[i].hostPtr()) + redist.helpers[0]->counts[i],
					bufs[i].begin(), bufs[i].end(),
					std::inserter(got, got.begin()), cmp);

		for (int pid = 0; pid < std::max(reference.size(), got.size()); pid++)
		{
			if (pid < got.size())
				printf("redist %2d:  %5d [%12.5e %12.5e %12.5e], ",
					i, got[pid].i1, got[pid].r.x, got[pid].r.y, got[pid].r.z);
			else
				printf("redist none,                                           ");

			printf(" expected ");

			if (pid < reference.size())
				printf("%5d [%12.5e %12.5e %12.5e]\n",
						reference[pid].i1, reference[pid].r.x, reference[pid].r.y, reference[pid].r.z);
			else
				printf("none\n");
		}
	}

	return 0;
}
