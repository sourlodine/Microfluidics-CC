#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public

#include <core/containers.h>
#include <core/celllist.h>
#include <core/halo_exchanger.h>
#include <core/redistributor.h>
#include <core/components.h>
#include <core/logger.h>
#include <core/integrate.h>
#include <core/wall.h>
#include <core/xml/pugixml.hpp>

#include "timer.h"
#include <unistd.h>

Logger logger;

void makeCells(Particle*& __restrict__ coos, Particle*& __restrict__ buffer, int* __restrict__ cellsStart, int* __restrict__ cellsSize,
		int np, CellListInfo cinfo)
{
	for (int i=0; i<cinfo.totcells+1; i++)
		cellsSize[i] = 0;

	for (int i=0; i<np; i++)
		cellsSize[cinfo.getCellId(float3{coos[i].x[0], coos[i].x[1], coos[i].x[2]})]++;

	cellsStart[0] = 0;
	for (int i=1; i<=cinfo.totcells; i++)
		cellsStart[i] = cellsSize[i-1] + cellsStart[i-1];

	for (int i=0; i<np; i++)
	{
		const int cid = cinfo.getCellId(float3{coos[i].x[0], coos[i].x[1], coos[i].x[2]});
		buffer[cellsStart[cid]] = coos[i];
		cellsStart[cid]++;
	}

	for (int i=0; i<cinfo.totcells; i++)
		cellsStart[i] -= cellsSize[i];

	std::swap(coos, buffer);
}

void integrate(Particle* __restrict__ coos, Force* __restrict__ accs, int np, float dt, CellListInfo cinfo)
{
	for (int i=0; i<np; i++)
	{
		coos[i].u[0] += accs[i].f[0]*dt;
		coos[i].u[1] += accs[i].f[1]*dt;
		coos[i].u[2] += accs[i].f[2]*dt;

		coos[i].x[0] += coos[i].u[0]*dt;
		coos[i].x[1] += coos[i].u[1]*dt;
		coos[i].x[2] += coos[i].u[2]*dt;

		if (coos[i].x[0] >   0.5f * cinfo.domainSize.x) coos[i].x[0] -= cinfo.domainSize.x;
		if (coos[i].x[0] <= -0.5f * cinfo.domainSize.x)	coos[i].x[0] += cinfo.domainSize.x;

		if (coos[i].x[1] >   0.5f * cinfo.domainSize.y) coos[i].x[1] -= cinfo.domainSize.y;
		if (coos[i].x[1] <= -0.5f * cinfo.domainSize.x)	coos[i].x[1] += cinfo.domainSize.y;

		if (coos[i].x[2] >   0.5f * cinfo.domainSize.z) coos[i].x[2] -= cinfo.domainSize.z;
		if (coos[i].x[2] <= -0.5f * cinfo.domainSize.x)	coos[i].x[2] += cinfo.domainSize.z;
	}
}


template<typename T>
T minabs(T arg)
{
	return arg;
}

template<typename T, typename... Args>
T minabs(T arg, Args... other)
{
	const T v = minabs(other...	);
	return (std::abs(arg) < std::abs(v)) ? arg : v;
}


void forces(const Particle* __restrict__ coos, Force* __restrict__ accs, const int* __restrict__ cellsStart, const int* __restrict__ cellsSize,
		CellListInfo cinfo)
{

	const float dt = 0.0025;
	const float kBT = 1.0;
	const float gammadpd = 20;
	const float sigma = sqrt(2 * gammadpd * kBT);
	const float sigmaf = sigma / sqrt(dt);
	const float aij = 50;

	auto addForce = [=] (int dstId, int srcId, Force& a)
	{
		float _xr = coos[dstId].x[0] - coos[srcId].x[0];
		float _yr = coos[dstId].x[1] - coos[srcId].x[1];
		float _zr = coos[dstId].x[2] - coos[srcId].x[2];

		_xr = minabs(_xr, _xr - cinfo.domainSize.x, _xr + cinfo.domainSize.x);
		_yr = minabs(_yr, _yr - cinfo.domainSize.y, _yr + cinfo.domainSize.y);
		_zr = minabs(_zr, _zr - cinfo.domainSize.z, _zr + cinfo.domainSize.z);

		const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;

		if (rij2 > 1.0f) return;
		//assert(rij2 < 1);

		const float invrij = 1.0f / sqrt(rij2);
		const float rij = rij2 * invrij;
		const float argwr = 1.0f - rij;
		const float wr = argwr;

		const float xr = _xr * invrij;
		const float yr = _yr * invrij;
		const float zr = _zr * invrij;

		const float rdotv =
				xr * (coos[dstId].u[0] - coos[srcId].u[0]) +
				yr * (coos[dstId].u[1] - coos[srcId].u[1]) +
				zr * (coos[dstId].u[2] - coos[srcId].u[2]);

		const float myrandnr = 0;//Logistic::mean0var1(1, min(srcId, dstId), max(srcId, dstId));

		const float strength = aij * argwr - (gammadpd * wr * rdotv + sigmaf * myrandnr) * wr;

		a.f[0] += strength * xr;
		a.f[1] += strength * yr;
		a.f[2] += strength * zr;
	};

	const int3 ncells = cinfo.ncells;

#pragma omp parallel for collapse(3)
	for (int cx = 0; cx < ncells.x; cx++)
		for (int cy = 0; cy < ncells.y; cy++)
			for (int cz = 0; cz < ncells.z; cz++)
			{
				const int cid = cinfo.encode(cx, cy, cz);

				for (int dstId = cellsStart[cid]; dstId < cellsStart[cid] + cellsSize[cid]; dstId++)
				{
					Force f {0,0,0,0};

					for (int dx = -1; dx <= 1; dx++)
						for (int dy = -1; dy <= 1; dy++)
							for (int dz = -1; dz <= 1; dz++)
							{
								int ncx, ncy, ncz;
								ncx = (cx+dx + ncells.x) % ncells.x;
								ncy = (cy+dy + ncells.y) % ncells.y;
								ncz = (cz+dz + ncells.z) % ncells.z;

								const int srcCid = cinfo.encode(ncx, ncy, ncz);
								if (srcCid >= cinfo.totcells || srcCid < 0) continue;

								for (int srcId = cellsStart[srcCid]; srcId < cellsStart[srcCid] + cellsSize[srcCid]; srcId++)
								{
									if (dstId != srcId)
										addForce(dstId, srcId, f);

									//printf("%d  %f %f %f\n", dstId, a.a[0], a.a[1], a.a[2]);
								}
							}

					accs[dstId].f[0] = f.f[0];
					accs[dstId].f[1] = f.f[1];
					accs[dstId].f[2] = f.f[2];
				}
			}
}


void createSdf(int3 resolution, float3 size, float r0, std::string fname)
{
	const float3 h = size / make_float3(resolution - 1);
	const float3 center = size / 2;
	float *sdf = new float[resolution.x * resolution.y * resolution.z];

	for (int i=0; i<resolution.z; i++)
	{
		for (int j=0; j<resolution.y; j++)
		{
			for (int k=0; k<resolution.x; k++)
			{
				float3 r = h * make_float3(i, j, k); // grid-centered data
				float3 dr = center - r;
				const float val = sqrtf(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z) - r0;
				sdf[ (i*resolution.y + j)*resolution.x + k ] = val;

				//printf("%5.2f  ", val);
			}
			//printf("\n");
		}
		//printf("\n");
	}

	std::ofstream out(fname);
	out << size.x       << " " << size.y       << " " << size.z       << " " << std::endl;
	out << resolution.x << " " << resolution.y << " " << resolution.z << " " << std::endl;
	out.write((char*)sdf, resolution.x * resolution.y * resolution.z * sizeof(float));

	delete[] sdf;
}

void checkFrozenRemaining(Particle* frozen, int nFrozen, Particle* remaining, int nRem, Particle* initial, int n, float3 size, float r)
{
	std::vector<Particle> refFrozen, refRem;

	for (int i=0; i<n; i++)
	{
		const float sdf = sqrt(initial[i].x[0]*initial[i].x[0] + initial[i].x[1]*initial[i].x[1] + initial[i].x[2]*initial[i].x[2]) - r;
		if (sdf < 0.5f) refRem.push_back(initial[i]);
		if (-0.5f < sdf && sdf < 1.5f) refFrozen.push_back(initial[i]);
	}

	auto cmp = [](const Particle& a, const Particle& b) -> bool{
		float d1 = sqrt(a.x[0]*a.x[0] + a.x[1]*a.x[1] + a.x[2]*a.x[2]);
		float d2 = sqrt(b.x[0]*b.x[0] + b.x[1]*b.x[1] + b.x[2]*b.x[2]);

		return d1 < d2 && fabs(d1 - d2) > 1e-6;
	};

	std::sort(refFrozen.begin(), refFrozen.end(), cmp);
	std::sort(refRem.begin(), refRem.end(), cmp);

	std::sort(frozen, frozen + nFrozen, cmp);
	std::sort(remaining, remaining + nRem, cmp);

	std::vector<Particle> res(n);
	auto vecend = std::set_intersection(frozen, frozen + nFrozen, remaining, remaining + nRem, res.begin(), cmp);
	if (vecend - res.begin())
	{
		printf("Whoops, %d  frozen and remaining particles are the same!\n", res.size());
	}

	vecend = std::set_difference(frozen, frozen + nFrozen, refFrozen.begin(), refFrozen.end(), res.begin(), cmp);
	for (auto p=res.begin(); p!=vecend; p++)
	{
		printf("Missing particle in Frozen: [%f %f %f]\n", p->x[0], p->x[1], p->x[2]);
	}

	vecend = std::set_difference(remaining, remaining + nRem, refRem.begin(), refRem.end(), res.begin(), cmp);
	for (auto p=res.begin(); p!=vecend; p++)
	{
		printf("Missing particle in Remaining: [%f %f %f] \n", p->x[0], p->x[1], p->x[2]);
	}

	//======================================

	vecend = std::set_difference(refFrozen.begin(), refFrozen.end(), frozen, frozen + nFrozen, res.begin(), cmp);
	float maxdiff = 0;
	for (auto p=res.begin(); p!=vecend; p++)
	{
		float sdf = sqrt(p->x[0]*p->x[0] + p->x[1]*p->x[1] + p->x[2]*p->x[2]) - r;
		maxdiff = max(maxdiff, min(sdf, 1-sdf));

		//printf("haha: [%f %f %f]  %f\n", p->x[0], p->x[1], p->x[2], sdf);

	}
	printf("Max distance inside frozen layer of missed particles: %f\n", maxdiff);


	vecend = std::set_difference(refRem.begin(), refRem.end(), remaining, remaining + nRem, res.begin(), cmp);
	maxdiff = 0;
	for (auto p=res.begin(); p!=vecend; p++)
	{
		float sdf = sqrt(p->x[0]*p->x[0] + p->x[1]*p->x[1] + p->x[2]*p->x[2]) - r;
		maxdiff = min(maxdiff, sdf);

		//printf("hohoho: [%f %f %f]  %f\n", p->x[0], p->x[1], p->x[2], sdf);

	}
	printf("Min sdf of missed remaining particles: %f\n", maxdiff);
}


float solve2_01(float a, float b, float c)
{
	if (fabs(a) < 1e-6)
	{
		if (fabs(b) < 1e-6) return -1;

		const float t = -c/b;
		if (0 <= t && t <= 1) return t;
		else return -1;
	}

	const float D = b*b - 4*a*c;
	if (D < 0) return -1;

	const float sqrtD = sqrt(D);
	const float t1 = 0.5*(-b + sqrtD) / a;
	const float t2 = 0.5*(-b - sqrtD) / a;

	if ( 0 <= t1 && t1 <= 1 && 0 <= t2 && t2 <= 1 ) return min(t1, t2);
	else if (0 <= t1 && t1 <= 1) return t1;
	else if (0 <= t2 && t2 <= 1) return t2;
	else return -1;
}

float bounce(float r, float3 x0, float3 x1)
{
	const float a = dot(x0 - x1, x0 - x1);
	const float b = 2 * dot(x0, x1-x0);
	const float c = dot(x0, x0) - r*r;

	return solve2_01(a, b, c);
}


void bounceAll(Particle* coosvels, int n, const float r, const float dt)
{
	for (int i=0; i<n; i++)
	{
		const float3 coo = {coosvels[i].x[0], coosvels[i].x[1], coosvels[i].x[2]};
		const float3 vel = {coosvels[i].u[0], coosvels[i].u[1], coosvels[i].u[2]};

		const float3 coo0 = coo - vel*dt;

		const float t = bounce(r, coo0, coo);
		if (t > -0.5f)
		{
			const float3 newcoo = coo0 + t * vel*dt + (1-t) * (vel*(-dt));
			coosvels[i].x[0] = newcoo.x;
			coosvels[i].x[1] = newcoo.y;
			coosvels[i].x[2] = newcoo.z;
		}
	}
}


int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
	    printf("ERROR: The MPI library does not have full thread support\n");
	    MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, "wall.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );


	std::string xml = R"(<node mass="1.0" density="2.0">)";
	pugi::xml_document config;
	config.load_string(xml.c_str());

	float3 length{32, 32, 32};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.0f;
	ParticleVector dpds("dpd");
	CellList cells(&dpds, rc, length);

	InitialConditions ic = createIC(config.child("node"));
	ic.exec(MPI_COMM_WORLD, &dpds, {0,0,0}, length);

	const float radius = 5;
	auto evalSdf = [radius] (float x, float y, float z) {
		return sqrt(x*x + y*y + z*z) - radius;
	};

	createSdf(make_int3(189), make_float3(cells.ncells), radius, "sphere.sdf");

	int c = 0;
	for (int i=0; i<dpds.np; i++)
	{
		dpds.coosvels[i].u[0] = 5*(drand48() - 0.5);
		dpds.coosvels[i].u[1] = 5*(drand48() - 0.5);
		dpds.coosvels[i].u[2] = 5*(drand48() - 0.5);
	}
	dpds.coosvels.uploadToDevice();

	HostBuffer<Particle> initial(dpds.np);
	memcpy(initial.hostPtr(), dpds.coosvels.hostPtr(), dpds.np*sizeof(Particle));

	dpds.forces.clear();

	hipStream_t defStream;
	CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );
	dpds.pushStreamWOhalo(defStream);

	HaloExchanger halo(cartComm, defStream);
	halo.attach(&dpds, &cells);
	Redistributor redist(cartComm);
	redist.attach(&dpds, &cells);

	cells.build(defStream);
	CUDA_Check( hipStreamSynchronize(defStream) );

	const float dt = 0.1;
	const int niters = 50;

	Wall wall("wall", "sphere.sdf", {1/3.0, 1/3.0, 1/3.0}, -1);
	wall.create(cartComm, {0,0,0}, length, length, &dpds, &cells);
	cells.build(defStream);
	wall.attach(&dpds, &cells);

	HostBuffer<Particle> frozen(wall.frozen.size());
	HostBuffer<float> intSdf(wall.sdfRawData.size());

	intSdf.copy(wall.sdfRawData, defStream);
	frozen.copy(wall.frozen.coosvels, defStream);
	dpds.coosvels.downloadFromDevice();

//	printf("============================================================================================\n");
//	for (int i=0; i<wall.resolution.z; i++)
//	{
//		for (int j=0; j<wall.resolution.y; j++)
//		{
//			for (int k=0; k<wall.resolution.x; k++)
//			{
//				printf("%5.2f  ", intSdf[ (i*wall.resolution.y + j)*wall.resolution.x + k ]);
//			}
//			printf("\n");
//		}
//		printf("\n");
//	}

	if (argc > 1)
		checkFrozenRemaining(frozen.hostPtr(), frozen.size(), dpds.coosvels.hostPtr(), dpds.np, initial.hostPtr(), initial.size(), make_float3(cells.ncells), radius);

	integrateNoFlow(&dpds, dt, defStream);

	dpds.coosvels.downloadFromDevice();
	HostBuffer<Particle> particles(dpds.np);
	memcpy(particles.hostPtr(), dpds.coosvels.hostPtr(), dpds.np*sizeof(Particle));

	wall.bounce(dt, defStream);
	dpds.coosvels.downloadFromDevice();

	bounceAll(particles.hostPtr(), particles.size(), radius, dt);

	printf("CPU bounce finished\n");


//	for (int i=0; i<niters; i++)
//	{
//		printf("Iteration %d\n", i);
//		dpds.accs.clear(defStream);
//		computeInternalDPD(dpds, defStream);
//
//		halo.exchangeInit();
//		halo.exchangeFinalize();
//
//		computeHaloDPD(dpds, defStream);
//		CUDA_Check( hipStreamSynchronize(defStream) );
//
//		wall.computeInteractions(defStream);
//		wall.bounce(defStream);
//
//		redist.redistribute(dt);
//
//		buildCellListAndIntegrate(dpds, config, dt, defStream);
//		CUDA_Check( hipStreamSynchronize(defStream) );
//		wall._check();
//	}
//
//	if (argc < 2) return 0;
//
//	int np = particles.size;
//	int totcells = dpds.totcells;
//
//	HostBuffer<Particle> buffer(np);
//	HostBuffer<Force> accs(np);
//	HostBuffer<int>   cellsStart(totcells+1), cellsSize(totcells+1);
//
//	printf("CPU execution\n");
//
//	for (int i=0; i<niters; i++)
//	{
//		printf("%d...", i);
//		fflush(stdout);
//		makeCells(particles.hostPtr(), buffer.hostPtr(), cellsStart.hostPtr(), cellsSize.hostPtr(), np, ncells, totcells, domainStart, 1.0f);
//		forces(particles.hostPtr(), accs.hostPtr(), cellsStart.hostPtr(), cellsSize.hostPtr(), ncells, totcells, domainStart, length);
//		integrate(particles.hostPtr(), accs.hostPtr(), np, dt, domainStart, length);
//	}
//
//	printf("\nDone, checking\n");
//	printf("NP:  %d,  ref  %d\n", dpds.np, np);
//
//
//	dpds.coosvels.synchronize(synchronizeHost);

	int np = dpds.np;
//	std::vector<int> gpuid(np), cpuid(np);
//	for (int i=0; i<np; i++)
//	{
//		gpuid[dpds.coosvels[i].i1] = i;
//		cpuid[particles[i].i1] = i;
//	}

	double l2 = 0, linf = -1;

	for (int i=0; i<np; i++)
	{
		Particle cpuP = particles[i];
		Particle gpuP = dpds.coosvels[i];

		double perr = -1;
		for (int c=0; c<3; c++)
		{
			const double err = fabs(cpuP.x[c] - gpuP.x[c]);// + fabs(cpuP.u[c] - gpuP.u[c]);
			linf = max(linf, err);
			perr = max(perr, err);
			l2 += err * err;
		}

		if (argc > 2 && (perr > 0.01 || evalSdf(cpuP.x[0], cpuP.x[1], cpuP.x[2]) > 0))
		{
			printf("id %8d diff %8e  [%12f %12f %12f  %8d] (%f)\n"
				   "                           ref [%12f %12f %12f  %8d] (%f) \n\n", i, perr,
					gpuP.x[0], gpuP.x[1], gpuP.x[2], gpuP.i1, evalSdf(gpuP.x[0], gpuP.x[1], gpuP.x[2]),
					cpuP.x[0], cpuP.x[1], cpuP.x[2], cpuP.i1, evalSdf(cpuP.x[0], cpuP.x[1], cpuP.x[2]) );
		}
	}

	l2 = sqrt(l2 / dpds.np);
	printf("L2   norm: %f\n", l2);
	printf("Linf norm: %f\n", linf);

	return 0;
}
