#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

// Yo ho ho ho
#define private   public
#define protected public

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <algorithm>

#include <core/particle_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/initial_conditions.h>

Logger logger;

int main(int argc, char **argv)
{
	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
		printf("ERROR: The MPI library does not have full thread support\n");
		MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, "cells->log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );


	std::string xml = R"(<node mass="1.0" density="8.0">)";
	pugi::xml_document config;
	config.load_string(xml.c_str());

	float3 length{66,33,51};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.2f;
	ParticleVector dpds("dpd");
	CellList *cells = new PrimaryCellList(&dpds, rc, length);

	UniformIC ic(config.child("node"));
	ic.exec(MPI_COMM_WORLD, &dpds, {0,0,0}, length, 0);

	const int np = dpds.local()->size();
	HostBuffer<Particle> initial(np);
	auto initPtr = initial.hostPtr();
	for (int i=0; i<np; i++)
		initPtr[i] = dpds.local()->coosvels[i];

	for (int i=0; i<50; i++)
		cells->build(0);

	dpds.local()->coosvels.downloadFromDevice(0, true);

	HostBuffer<uint> hcellsStart(cells->totcells+1);
	HostBuffer<uint8_t> hcellsSize(cells->totcells+1);

	hcellsStart.copy(cells->cellsStartSize, 0);
	hcellsSize. copy(cells->cellsSize, 0);

	HostBuffer<int> cellscount(cells->totcells+1);
	for (int i=0; i<cells->totcells+1; i++)
		cellscount[i] = 0;

	int total = 0;
	for (int pid=0; pid < initial.size(); pid++)
	{
		float3 coo{initial[pid].r.x, initial[pid].r.y, initial[pid].r.z};
		float3 vel{initial[pid].u.x, initial[pid].u.y, initial[pid].u.z};

		//vel += acc * dt;
		//coo += vel * dt;

		int actCid = cells->getCellId(coo);
		if (actCid >= 0)
		{
			cellscount[actCid]++;
			total++;
		}
	}

	printf("np = %d, vs reference  %d\n", dpds.local()->size(), total);
	for (int cid=0; cid < cells->totcells+1; cid++)
		if ( (hcellsStart[cid] >> cells->blendingPower) != cellscount[cid] )
			printf("cid %d:  %d (correct %d),  %d\n", cid, hcellsStart[cid] >> cells->blendingPower, cellscount[cid], hcellsStart[cid] & ((1<<cells->blendingPower) - 1));

	for (int cid=0; cid < cells->totcells; cid++)
	{
		const int start = hcellsStart[cid] & ((1<<cells->blendingPower) - 1);
		const int size = hcellsStart[cid] >> cells->blendingPower;
		for (int pid=start; pid < start + size; pid++)
		{
			const float3 cooDev{dpds.local()->coosvels[pid].r.x, dpds.local()->coosvels[pid].r.y, dpds.local()->coosvels[pid].r.z};
			const float3 velDev{dpds.local()->coosvels[pid].u.x, dpds.local()->coosvels[pid].u.y, dpds.local()->coosvels[pid].u.z};

			const int origId = dpds.local()->coosvels[pid].i1;

			float3 coo{initial[origId].r.x, initial[origId].r.y, initial[origId].r.z};
			float3 vel{initial[origId].u.x, initial[origId].u.y, initial[origId].u.z};

//			vel += acc * dt;
//			coo += vel * dt;

			const float diff = std::max({
				fabs(coo.x - cooDev.x), fabs(coo.y - cooDev.y), fabs(coo.z - cooDev.z),
				fabs(vel.x - velDev.x), fabs(vel.y - velDev.y), fabs(vel.z - velDev.z) });

			int actCid = cells->getCellId<false>(cooDev);

			if (cid != actCid || diff > 1e-5)
				printf("cid  %d,  correct cid  %d  for pid %d:  [%e %e %e  %d]  correct: [%e %e %e  %d]\n",
						cid, actCid, pid, cooDev.x, cooDev.y, cooDev.z, dpds.local()->coosvels[pid].i1,
						coo.x, coo.y, coo.z, initial[origId].i1);
		}
	}

	return 0;
}
