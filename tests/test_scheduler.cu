#include "hip/hip_runtime.h"
#include <core/logger.h>
#include <core/task_scheduler.h>

#include "timer.h"

#define private public

Logger logger;

int main(int argc, char ** argv)
{
	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
		printf("ERROR: The MPI library does not have full thread support\n");
		MPI_Abort(MPI_COMM_WORLD, 1);
	}
	MPI_Errhandler_set(MPI_COMM_WORLD, MPI_ERRORS_RETURN);

	logger.init(MPI_COMM_WORLD, "scheduler.log", 9);

	//  A1,A2 - B -----------
	//              \        \
	//                D1,D2 - E
	//          C - /
	//              \ F
	//                        G

	{
		TaskScheduler scheduler;

		scheduler.addTask("C", [](hipStream_t s){ printf("c\n"); });
		scheduler.addTask("G", [](hipStream_t s){ printf("g\n"); });
		scheduler.addTask("D", [](hipStream_t s){ printf("d2\n"); });
		scheduler.addTask("A", [](hipStream_t s){ printf("a1\n"); });
		scheduler.addTask("E", [](hipStream_t s){ printf("e\n"); });
		scheduler.addTask("A", [](hipStream_t s){ printf("a2\n"); });
		scheduler.addTask("B", [](hipStream_t s){ printf("b\n"); });
		scheduler.addTask("D", [](hipStream_t s){ printf("d1\n"); });
		scheduler.addTask("F", [](hipStream_t s){ printf("f\n"); });

		scheduler.addDependency("B", {}, {"A"});
		scheduler.addDependency("D", {}, {"B", "C"});
		scheduler.addDependency("F", {}, {"C"});
		scheduler.addDependency("E", {}, {"D"});
		scheduler.addDependency("E", {}, {"B"});

		scheduler.compile();
		scheduler.run();
	}

	printf("Benchmarking\n");

	TaskScheduler scheduler;

	float a, b, c, d, e, f, g;
	a=b=c=d=e=f=g = 0;
	scheduler.addTask("C", [&](hipStream_t s){ c++; });
	scheduler.addTask("G", [&](hipStream_t s){ g--; });
	scheduler.addTask("D", [&](hipStream_t s){ d+=2; });
	scheduler.addTask("A", [&](hipStream_t s){ a-=3; });
	scheduler.addTask("E", [&](hipStream_t s){ e*=1.001; });
	scheduler.addTask("A", [&](hipStream_t s){ a*=0.9999; });
	scheduler.addTask("B", [&](hipStream_t s){ b+=5; });
	scheduler.addTask("D", [&](hipStream_t s){ d-=42; });
	scheduler.addTask("F", [&](hipStream_t s){ f*=2; });

	scheduler.addDependency("B", {}, {"A"});
	scheduler.addDependency("D", {}, {"B", "C"});
	scheduler.addDependency("F", {}, {"C"});
	scheduler.addDependency("E", {}, {"D"});
	scheduler.addDependency("E", {}, {"B"});

	scheduler.compile();

	Timer timer;
	timer.start();

	int n = 10000;
	for (int i=0; i<n; i++)
		scheduler.run();

	int64_t tm = timer.elapsed();

	printf("Per run: %f us\n", (double)tm / (1000.0*n));

	return 0;
}
