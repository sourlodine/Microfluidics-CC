#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private   public
#define protected public

#include <core/particle_vector.h>
#include <core/celllist.h>
#include <core/logger.h>
#include <core/xml/pugixml.hpp>
#include <core/containers.h>

#include <core/mpi/api.h>

Logger logger;

Particle addShift(Particle p, float a, float b, float c)
{
	Particle res = p;
	res.r.x += a;
	res.r.y += b;
	res.r.z += c;

	return res;
}

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	MPI_Init(&argc, &argv);
	logger.init(MPI_COMM_WORLD, "halo.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	std::string xml = R"(<node mass="1.0" density="2.0">)";
	pugi::xml_document config;
	config.load_string(xml.c_str());

	float3 length{64,64,64};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.0f;
	ParticleVector dpds("dpd");
	CellList cells(&dpds, rc, length);
	cells.setStream(0);
	cells.makePrimary();

	InitialConditions ic = createIC(config.child("node"));
	ic.exec(MPI_COMM_WORLD, &dpds, {0,0,0}, length);

	cells.build();

	dpds.local()->coosvels.downloadFromDevice(true);

	hipStream_t defStream = 0;

	ParticleHaloExchanger halo(cartComm, 0);
	halo.attach(&dpds, &cells);

	cells.build();
	CUDA_Check( hipStreamSynchronize(defStream) );

	for (int i=0; i<10; i++)
	{
		halo.init();
		halo.finalize();
	}

	std::vector<Particle> bufs[27];
	dpds.local()->coosvels.downloadFromDevice(true);
	dpds.halo()->coosvels.downloadFromDevice(true);

	for (int i=0; i<dpds.local()->size(); i++)
	{
		Particle& p = dpds.local()->coosvels[i];

		int3 code = cells.getCellIdAlongAxis(p.r);
		int cx = code.x,  cy = code.y,  cz = code.z;
		auto ncells = cells.ncells;

		// 6
		if (cx == 0)          bufs[ (1*3 + 1)*3 + 0 ].push_back(addShift(p,  length.x,         0,         0));
		if (cx == ncells.x-1) bufs[ (1*3 + 1)*3 + 2 ].push_back(addShift(p, -length.x,         0,         0));
		if (cy == 0)          bufs[ (1*3 + 0)*3 + 1 ].push_back(addShift(p,         0,  length.y,         0));
		if (cy == ncells.y-1) bufs[ (1*3 + 2)*3 + 1 ].push_back(addShift(p,         0, -length.y,         0));
		if (cz == 0)          bufs[ (0*3 + 1)*3 + 1 ].push_back(addShift(p,         0,         0,  length.z));
		if (cz == ncells.z-1) bufs[ (2*3 + 1)*3 + 1 ].push_back(addShift(p,         0,         0, -length.z));

		// 12
		if (cx == 0          && cy == 0)          bufs[ (1*3 + 0)*3 + 0 ].push_back(addShift(p,  length.x,  length.y,         0));
		if (cx == ncells.x-1 && cy == 0)          bufs[ (1*3 + 0)*3 + 2 ].push_back(addShift(p, -length.x,  length.y,         0));
		if (cx == 0          && cy == ncells.y-1) bufs[ (1*3 + 2)*3 + 0 ].push_back(addShift(p,  length.x, -length.y,         0));
		if (cx == ncells.x-1 && cy == ncells.y-1) bufs[ (1*3 + 2)*3 + 2 ].push_back(addShift(p, -length.x, -length.y,         0));

		if (cy == 0          && cz == 0)          bufs[ (0*3 + 0)*3 + 1 ].push_back(addShift(p,         0,  length.y,  length.z));
		if (cy == ncells.y-1 && cz == 0)          bufs[ (0*3 + 2)*3 + 1 ].push_back(addShift(p,         0, -length.y,  length.z));
		if (cy == 0          && cz == ncells.z-1) bufs[ (2*3 + 0)*3 + 1 ].push_back(addShift(p,         0,  length.y, -length.z));
		if (cy == ncells.y-1 && cz == ncells.z-1) bufs[ (2*3 + 2)*3 + 1 ].push_back(addShift(p,         0, -length.y, -length.z));


		if (cz == 0          && cx == 0)          bufs[ (0*3 + 1)*3 + 0 ].push_back(addShift(p,  length.x,         0,  length.z));
		if (cz == ncells.z-1 && cx == 0)          bufs[ (2*3 + 1)*3 + 0 ].push_back(addShift(p,  length.x,         0, -length.z));
		if (cz == 0          && cx == ncells.x-1) bufs[ (0*3 + 1)*3 + 2 ].push_back(addShift(p, -length.x,         0,  length.z));
		if (cz == ncells.z-1 && cx == ncells.x-1) bufs[ (2*3 + 1)*3 + 2 ].push_back(addShift(p, -length.x,         0, -length.z));

		// 8
		if (cx == 0          && cy == 0          && cz == 0)          bufs[ (0*3 + 0)*3 + 0 ].push_back(addShift(p,  length.x,  length.y,  length.z));
		if (cx == 0          && cy == 0          && cz == ncells.z-1) bufs[ (2*3 + 0)*3 + 0 ].push_back(addShift(p,  length.x,  length.y, -length.z));
		if (cx == 0          && cy == ncells.y-1 && cz == 0)          bufs[ (0*3 + 2)*3 + 0 ].push_back(addShift(p,  length.x, -length.y,  length.z));
		if (cx == 0          && cy == ncells.y-1 && cz == ncells.z-1) bufs[ (2*3 + 2)*3 + 0 ].push_back(addShift(p,  length.x, -length.y, -length.z));
		if (cx == ncells.x-1 && cy == 0          && cz == 0)          bufs[ (0*3 + 0)*3 + 2 ].push_back(addShift(p, -length.x,  length.y,  length.z));
		if (cx == ncells.x-1 && cy == 0          && cz == ncells.z-1) bufs[ (2*3 + 0)*3 + 2 ].push_back(addShift(p, -length.x,  length.y, -length.z));
		if (cx == ncells.x-1 && cy == ncells.y-1 && cz == 0)          bufs[ (0*3 + 2)*3 + 2 ].push_back(addShift(p, -length.x, -length.y,  length.z));
		if (cx == ncells.x-1 && cy == ncells.y-1 && cz == ncells.z-1) bufs[ (2*3 + 2)*3 + 2 ].push_back(addShift(p, -length.x, -length.y, -length.z));
	}

	for (int i = 0; i<27; i++)
	{
		std::sort(bufs[i].begin(), bufs[i].end(), [] (Particle& a, Particle& b) { return a.i1 < b.i1; });

		std::sort((Particle*)halo.helpers[0]->sendBufs[i].hostPtr(), ((Particle*)halo.helpers[0]->sendBufs[i].hostPtr()) + halo.helpers[0]->bufSizes[i],
				[] (Particle& a, Particle& b) { return a.i1 < b.i1; });

		if (bufs[i].size() != halo.helpers[0]->bufSizes[i])
			printf("%2d-th halo differs in size: %5d, expected %5d\n", i, halo.helpers[0]->bufSizes[i], (int)bufs[i].size());
		else
		{
			auto ptr = (Particle*)halo.helpers[0]->sendBufs[i].hostPtr();
			for (int pid = 0; pid < halo.helpers[0]->bufSizes[i]; pid++)
			{
				const float diff = std::max({
					fabs(ptr[pid].r.x - bufs[i][pid].r.x),
					fabs(ptr[pid].r.y - bufs[i][pid].r.y),
					fabs(ptr[pid].r.z - bufs[i][pid].r.z) });

				if (bufs[i][pid].i1 != ptr[pid].i1 || diff > 1e-5)
					printf("Halo %2d:  %5d [%10.3e %10.3e %10.3e], expected %5d [%10.3e %10.3e %10.3e]\n",
							i, ptr[pid].i1, ptr[pid].r.x, ptr[pid].r.y, ptr[pid].r.z,
							bufs[i][pid].i1, bufs[i][pid].r.x, bufs[i][pid].r.y, bufs[i][pid].r.z);
			}
		}
	}

//	for (int i=0; i<dpds.halo()->size(); i++)
//		printf("%d  %f %f %f\n", i, dpds.halo[i].r.x, dpds.halo[i].r.y, dpds.halo[i].r.z);

	return 0;
}
