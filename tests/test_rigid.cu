#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public
#define protected public

#include <core/particle_vector.h>
#include <core/rigid_object_vector.h>
#include <core/celllist.h>
#include <core/mpi/api.h>
#include <core/logger.h>
#include <core/integrate.h>
#include <core/interactions.h>
#include <core/bounce.h>

#include "timer.h"
#include <unistd.h>

Logger logger;

float4 inv_q(float4 q)
{
	return make_float4(q.x, -q.y, -q.z, -q.w);
}

float3 rot(float3 v, float4 q)
{
	//https://en.wikipedia.org/wiki/Rodrigues%27_rotation_formula

	double phi = 2.0*atan2( sqrt( (double)q.y*q.y + (double)q.z*q.z + (double)q.w*q.w),  (double)q.x );
	double sphi_1 = 1.0 / sin(0.5*phi);
	const float3 k = make_float3(q.y * sphi_1, q.z * sphi_1, q.w * sphi_1);

	return v*cos(phi) + cross(k, v) * sin(phi) + k * dot(k, v) * (1-cos(phi));
}

float ellipsoid(LocalRigidObjectVector::RigidMotion motion, float3 invAxes, float3 r)
{
	const float3 v = r - motion.r;
	const float3 vRot = rot(v, inv_q(motion.q));

	return sqr(vRot.x * invAxes.x) + sqr(vRot.y * invAxes.y) + sqr(vRot.z * invAxes.z) - 1.0f;
}

__inline__ __device__ float warpReduceSum(float val)
{
#pragma unroll
	for (int offset = warpSize/2; offset > 0; offset /= 2)
	{
		val += __shfl_down(val, offset);
	}
	return val;
}

__inline__ __device__ float3 warpReduceSum(float3 val)
{
#pragma unroll
	for (int offset = warpSize/2; offset > 0; offset /= 2)
	{
		val.x += __shfl_down(val.x, offset);
		val.y += __shfl_down(val.y, offset);
		val.z += __shfl_down(val.z, offset);
	}
	return val;
}

__global__ void totalMomentumEnergy(const float4* coosvels, const float mass, int n, double* momentum, double* energy)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int wid = tid % warpSize;
	if (tid >= n) return;

	const float3 vel = make_float3(coosvels[2*tid+1]);

	float3 myMomentum = vel*mass;
	float myEnergy = dot(vel, vel) * mass*0.5f;

	myMomentum = warpReduceSum(myMomentum);
	myEnergy   = warpReduceSum(myEnergy);

	if (wid == 0)
	{
		atomicAdd(momentum+0, (double)myMomentum.x);
		atomicAdd(momentum+1, (double)myMomentum.y);
		atomicAdd(momentum+2, (double)myMomentum.z);
		atomicAdd(energy,     (double)myEnergy);
	}
}

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
	    printf("ERROR: The MPI library does not have full thread support\n");
	    MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, "rigid.log", 9);
	srand48(2);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	hipStream_t defStream;
	CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );

	float3 length{32, 32, 32};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.0f;
	const int ndens = 8;

	float3 axes{3, 4, 5};
	float3 invAxes = 1.0 / axes;
	const float maxAxis = std::max({axes.x, axes.y, axes.z});

	int objSize = 4/3.0 * M_PI * axes.x*axes.y*axes.z * ndens + 1;
	RigidObjectVector obj("obj", objSize, 1);

	// Init object
	obj.initialPositions.resize(objSize, defStream);
	memset(&obj.local()->motions[0], 0, sizeof(LocalRigidObjectVector::RigidMotion));
	obj.local()->motions[0].r = make_float3(0);

	obj.local()->motions[0].omega.x = 0;//0*(drand48() - 0.5);
	obj.local()->motions[0].omega.y = 0;//0*(drand48() - 0.5);
	obj.local()->motions[0].omega.z = 10;//0*(drand48() - 0.5);

	obj.local()->motions[0].vel.x = 0*(drand48() - 0.5);
	obj.local()->motions[0].vel.y = 0*(drand48() - 0.5);
	obj.local()->motions[0].vel.z = 0*(drand48() - 0.5);

	obj.local()->motions[0].force  = make_float3(0);
	obj.local()->motions[0].torque = make_float3(0);

	const float phi = M_PI*drand48()*0.001;
	const float sphi = sin(0.5f*phi);
	const float cphi = cos(0.5f*phi);

	float3 v = make_float3(drand48(), drand48(), drand48());
	v = normalize(v);

	float4 q = obj.local()->motions[0].q = make_float4(cphi, sphi*v.x, sphi*v.y, sphi*v.z);

	LocalRigidObjectVector::RigidMotion m;
	m.r = make_float3(0);
	m.q = make_float4(0.999, 0, 0, 0.0447101778);
	m.vel = make_float3(0);

	for (int i=0; i<obj.objSize; i++)
	{
		float4 pos;

		do
		{
			pos.x = 2*maxAxis*(drand48() - 0.5);
			pos.y = 2*maxAxis*(drand48() - 0.5);
			pos.z = 2*maxAxis*(drand48() - 0.5);

		} while ( ellipsoid(m, invAxes, f4tof3(pos)) > 0 );

		obj.initialPositions[i] = pos;
	}

	for (int i=0; i<obj.local()->size(); i++)
	{
		Particle p;
		p.i1 = 0;
		p.s21 = (short)i;

		p.r = rot( f4tof3(obj.initialPositions[i % obj.objSize]), obj.local()->motions[i / objSize].q);
		p.u = make_float3(0);

		obj.local()->coosvels[i] = p;
	}

	obj.initialPositions. uploadToDevice(defStream);
	obj.local()->motions. uploadToDevice(defStream);
	obj.local()->coosvels.uploadToDevice(defStream);
	obj.local()->findExtentAndCOM(0);

	obj.objMass = objSize * 1.0f;
	obj.mass = 1.0f;
	obj.axes = axes;

	HostBuffer<LocalRigidObjectVector::COMandExtent> com_ext;
	com_ext.copy(obj.local()->comAndExtents, 0);

	printf("Object: \n"
			"   phi %f, v [%f %f %f]\n"
			"   ext : [%f %f %f] -- [%f %f %f]\n\n",
			phi, v.x, v.y, v.z,
			com_ext[0].low.x,  com_ext[0].low.y,  com_ext[0].low.z,
			com_ext[0].high.x, com_ext[0].high.y, com_ext[0].high.z);


	ParticleVector dpds("dpd");
	CellList *cells = new PrimaryCellList(&dpds, rc, length);
	CellList *objCells = new CellList(&obj, rc, length);

	dpds.local()->resize(cells->ncells.x*cells->ncells.y*cells->ncells.z * ndens, defStream);

	printf("initializing...\n");

	auto motion = obj.local()->motions[0];
	int c = 0;
	float3 totU = make_float3(0);
	for (int i=0; i<cells->ncells.x; i++)
		for (int j=0; j<cells->ncells.y; j++)
			for (int k=0; k<cells->ncells.z; k++)
				for (int p=0; p<ndens; p++)
				{
					dpds.local()->coosvels[c].r.x = i + drand48() + domainStart.x;
					dpds.local()->coosvels[c].r.y = j + drand48() + domainStart.y;
					dpds.local()->coosvels[c].r.z = k + drand48() + domainStart.z;

					if (ellipsoid(motion, invAxes, dpds.local()->coosvels[c].r) < 0.02)
						continue;

					dpds.local()->coosvels[c].i1 = c;

					dpds.local()->coosvels[c].u.x = 1*(drand48() - 0.5);
					dpds.local()->coosvels[c].u.y = 1*(drand48() - 0.5);
					dpds.local()->coosvels[c].u.z = 1*(drand48() - 0.5);

					totU += dpds.local()->coosvels[c].u;

					c++;
				}

	totU /= c;
	for (int i=0; i<c; i++)
		dpds.local()->coosvels[i].u -= totU;

	printf("generated %d particles\n", c);
	dpds.local()->resize(c, defStream);
	dpds.domainSize = length;
	dpds.mass = 1.0f;
	dpds.local()->coosvels.uploadToDevice(defStream);

	ParticleHaloExchanger halo(cartComm);
	halo.attach(&dpds, cells);
	ParticleRedistributor redist(cartComm);
	redist.attach(&dpds, cells);

	CUDA_Check( hipStreamSynchronize(defStream) );

	const int niters = 4200000;
	const float dt = 0.002;

	std::string xml = R"(<interaction name="dpd" kbt="1.0" gamma="20" a="50" dt="0.002"/>
    <integrate dt="0.002"/>)";
	pugi::xml_document config;
	config.load_string(xml.c_str());

	Interaction *inter = new InteractionDPD(config.child("interaction"));
	Integrator  *noflow = new IntegratorVVNoFlow(config.child("integrate"));
	Integrator  *rigInt = new IntegratorVVRigid(config.child("integrate"));

	printf("GPU execution\n");

	Timer tm;
	tm.start();

	HostBuffer<Force> frcs;

	PinnedBuffer<double> energy(1), momentum(3);

	hipDeviceSynchronize();

	auto prnCoosvels = [defStream] (ParticleVector* pv) {
		pv->local()->coosvels.downloadFromDevice(defStream, true);
		auto ptr = pv->local()->coosvels.hostPtr();
		for (int j=0; j<pv->local()->size(); j++)
		{
			if (ptr[j].s21 == 42)
				printf("??? %4d :  [%f %f %f] [%f %f %f]\n", ptr[j].s21, ptr[j].r.x, ptr[j].r.y, ptr[j].r.z, ptr[j].u.x, ptr[j].u.y, ptr[j].u.z);
		}
	};

	const int nparticles = dpds.local()->size() + obj.local()->size();
	for (int i=0; i<niters; i++)
	{
		energy.clear(defStream);
		momentum.clear(defStream);

		totalMomentumEnergy<<< getNblocks(dpds.local()->size(), 128), 128, 0, defStream >>> (
				(float4*)dpds.local()->coosvels.devPtr(), dpds.mass, dpds.local()->size(), momentum.devPtr(), energy.devPtr());

		totalMomentumEnergy<<< getNblocks(obj.local()->size(), 128), 128, 0, defStream >>> (
				(float4*)obj.local()->coosvels.devPtr(), 1.0f, obj.local()->size(), momentum.devPtr(), energy.devPtr());

		momentum.downloadFromDevice(defStream, false);
		energy.  downloadFromDevice(defStream, true);

		if (i % 100 == 0)
		{
			printf("Iteration %d, temp %f, momentum  %.2e %.2e %.2e\n",
					i, energy[0]/ ( (3/2.0)*nparticles ), momentum[0] / nparticles, momentum[1] / nparticles, momentum[2] / nparticles);

			obj.local()->motions.downloadFromDevice(defStream, true);
			auto motion = obj.local()->motions[0];

			printf("obj  %d  f [%f %f %f],  t [%f %f %f],  r [%f %f %f]   v [%f %f %f] \n"
					"    q [%f %f %f %f]   w [%f %f %f] \n", 0,
					motion.force.x,  motion.force.y,  motion.force.z,
					motion.torque.x, motion.torque.y, motion.torque.z,
					motion.r.x,  motion.r.y,  motion.r.z,
					motion.vel.x,  motion.vel.y,  motion.vel.z,
					motion.q.x,  motion.q.y,  motion.q.z, motion.q.w,
					motion.omega.x,  motion.omega.y,  motion.omega.z);
		}

		cells->build(defStream);
		objCells->build(defStream);

		dpds.local()->forces.clear(defStream);
		obj.local()->forces.clear(defStream);

		cells->forces->clear(defStream);
		objCells->forces->clear(defStream);

		halo.init(defStream);
		inter->regular(&dpds, &dpds, cells, cells,    dt*i, defStream);
		inter->regular(&dpds, &obj,  cells, objCells, dt*i, defStream);
		halo.finalize();

//		CUDA_Check( hipStreamSynchronize(defStream) );

		inter->halo(&dpds, &dpds, cells, cells, dt*i, defStream);

		objCells->addForces(defStream);

		noflow->stage1(&dpds, defStream);
		rigInt->stage1(&obj,  defStream);
		noflow->stage2(&dpds, defStream);
		rigInt->stage2(&obj,  defStream);

		obj.local()->findExtentAndCOM(defStream);
		bounceFromRigidEllipsoid(&dpds, cells, &obj, dt, true, defStream);

		CUDA_Check( hipStreamSynchronize(defStream) );

		redist.redistribute(defStream);
	}

	double elapsed = tm.elapsed() * 1e-9;

	printf("Finished in %f s, 1 step took %f ms\n", elapsed, elapsed / niters * 1000.0);

	return 0;
}
