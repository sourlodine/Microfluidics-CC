#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public

#include <core/particle_vector.h>
#include <core/rigid_object_vector.h>
#include <core/celllist.h>
#include <core/mpi/api.h>
#include <core/logger.h>
#include <core/integrate.h>
#include <core/bounce.h>
#include <core/components.h>

#include "timer.h"
#include <unistd.h>

Logger logger;

float4 inv_q(float4 q)
{
	return make_float4(q.x, -q.y, -q.z, -q.w);
}

float3 rot(float3 v, float4 q)
{
	//https://en.wikipedia.org/wiki/Rodrigues%27_rotation_formula

	double phi = 2.0*atan2( sqrt( (double)q.y*q.y + (double)q.z*q.z + (double)q.w*q.w),  (double)q.x );
	double sphi_1 = 1.0 / sin(0.5*phi);
	const float3 k = make_float3(q.y * sphi_1, q.z * sphi_1, q.w * sphi_1);

	return v*cos(phi) + cross(k, v) * sin(phi) + k * dot(k, v) * (1-cos(phi));
}

float ellipsoid(LocalRigidObjectVector::RigidMotion motion, float3 invAxes, float3 r)
{
	const float3 v = r - motion.r;
	const float3 vRot = rot(v, inv_q(motion.q));

	return sqr(vRot.x * invAxes.x) + sqr(vRot.y * invAxes.y) + sqr(vRot.z * invAxes.z) - 1.0f;
}

__inline__ __device__ float warpReduceSum(float val)
{
#pragma unroll
	for (int offset = warpSize/2; offset > 0; offset /= 2)
	{
		val += __shfl_down(val, offset);
	}
	return val;
}

__inline__ __device__ float3 warpReduceSum(float3 val)
{
#pragma unroll
	for (int offset = warpSize/2; offset > 0; offset /= 2)
	{
		val.x += __shfl_down(val.x, offset);
		val.y += __shfl_down(val.y, offset);
		val.z += __shfl_down(val.z, offset);
	}
	return val;
}

__global__ void totalMomentumEnergy(const float4* coosvels, const float mass, int n, double* momentum, double* energy)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int wid = tid % warpSize;
	if (tid >= n) return;

	const float3 vel = make_float3(coosvels[2*tid+1]);

	float3 myMomentum = vel*mass;
	float myEnergy = dot(vel, vel) * mass*0.5f;

	myMomentum = warpReduceSum(myMomentum);
	myEnergy   = warpReduceSum(myEnergy);

	if (wid == 0)
	{
		atomicAdd(momentum+0, (double)myMomentum.x);
		atomicAdd(momentum+1, (double)myMomentum.y);
		atomicAdd(momentum+2, (double)myMomentum.z);
		atomicAdd(energy,     (double)myEnergy);
	}
}

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	int provided;
	MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
	if (provided < MPI_THREAD_MULTIPLE)
	{
	    printf("ERROR: The MPI library does not have full thread support\n");
	    MPI_Abort(MPI_COMM_WORLD, 1);
	}

	logger.init(MPI_COMM_WORLD, "rigid.log", 9);
	srand48(0);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	hipStream_t defStream;
	CUDA_Check( hipStreamCreateWithPriority(&defStream, hipStreamNonBlocking, 10) );

	float3 length{40, 40, 40};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.0f;
	const int ndens = 8;

	float3 axes{3, 4, 5};
	float3 invAxes = 1.0 / axes;
	const float maxAxis = std::max({axes.x, axes.y, axes.z});

	int objSize = 4/3.0 * M_PI * axes.x*axes.y*axes.z * ndens;
	RigidObjectVector obj("obj", objSize, 1);

	// Init object
	memset(&obj.local()->motions[0], 0, sizeof(LocalRigidObjectVector::RigidMotion));
	obj.local()->motions[0].r = make_float3(0);

	obj.local()->motions[0].omega.x = 0*(drand48() - 0.5);
	obj.local()->motions[0].omega.y = 0*(drand48() - 0.5);
	obj.local()->motions[0].omega.z = 0*(drand48() - 0.5);

	obj.local()->motions[0].vel.x = 0*(drand48() - 0.5);
	obj.local()->motions[0].vel.y = 0*(drand48() - 0.5);
	obj.local()->motions[0].vel.z = 0*(drand48() - 0.5);

	obj.local()->motions[0].force  = make_float3(0);
	obj.local()->motions[0].torque = make_float3(0);

	const float phi = M_PI*drand48();
	const float sphi = sin(0.5f*phi);
	const float cphi = cos(0.5f*phi);

	float3 v = make_float3(drand48(), drand48(), drand48());
	v = normalize(v);

	float4 q = obj.local()->motions[0].q = make_float4(cphi, sphi*v.x, sphi*v.y, sphi*v.z);


	for (int i=0; i<obj.objSize; i++)
	{
		Particle p;
		p.u = make_float3(0);
		p.i1 = 0;
		p.s21 = (short)i;

		do
		{
			p.r.x = 2*maxAxis*(drand48() - 0.5);
			p.r.y = 2*maxAxis*(drand48() - 0.5);
			p.r.z = 2*maxAxis*(drand48() - 0.5);

		} while ( ellipsoid(obj.local()->motions[0], invAxes, p.r) > 0 );

		obj.local()->coosvels[i] = p;
	}

	obj.local()->motions. uploadToDevice();
	obj.local()->coosvels.uploadToDevice();
	obj.local()->findExtentAndCOM(0);

	obj.objMass = objSize * 1.0f;
	obj.mass = 1.0f;
	obj.axes = axes;

	HostBuffer<LocalRigidObjectVector::COMandExtent> com_ext;
	com_ext.copy(obj.local()->comAndExtents, 0);

	printf("Object: \n"
			"   phi %f, v [%f %f %f]\n"
			"   ext : [%f %f %f] -- [%f %f %f]\n\n",
			phi, v.x, v.y, v.z,
			com_ext[0].low.x,  com_ext[0].low.y,  com_ext[0].low.z,
			com_ext[0].high.x, com_ext[0].high.y, com_ext[0].high.z);


	ParticleVector dpds("dpd");
	dpds.local()->pushStream(defStream);
	CellList cells(&dpds, rc, length);
	cells.setStream(defStream);
	cells.makePrimary();

	dpds.local()->resize(cells.ncells.x*cells.ncells.y*cells.ncells.z * ndens);

	printf("initializing...\n");

	auto motion = obj.local()->motions[0];
	int c = 0;
	float3 totU = make_float3(0);
	for (int i=0; i<cells.ncells.x; i++)
		for (int j=0; j<cells.ncells.y; j++)
			for (int k=0; k<cells.ncells.z; k++)
				for (int p=0; p<ndens; p++)
				{
					dpds.local()->coosvels[c].r.x = i + drand48() + domainStart.x;
					dpds.local()->coosvels[c].r.y = j + drand48() + domainStart.y;
					dpds.local()->coosvels[c].r.z = k + drand48() + domainStart.z;

					if (ellipsoid(motion, invAxes, dpds.local()->coosvels[c].r) < 0.5)
						continue;

					dpds.local()->coosvels[c].i1 = c;

					dpds.local()->coosvels[c].u.x = 1*(drand48() - 0.5);
					dpds.local()->coosvels[c].u.y = 1*(drand48() - 0.5);
					dpds.local()->coosvels[c].u.z = 1*(drand48() - 0.5);

					totU += dpds.local()->coosvels[c].u;

					c++;
				}

	totU /= c;
	for (int i=0; i<c; i++)
		dpds.local()->coosvels[i].u -= totU;

	printf("generated %d particles\n", c);
	dpds.local()->resize(c);
	dpds.domainSize = length;
	dpds.mass = 1.0f;
	dpds.local()->coosvels.uploadToDevice();

	ParticleHaloExchanger halo(cartComm, defStream);
	halo.attach(&dpds, &cells);
	ParticleRedistributor redist(cartComm, defStream);
	redist.attach(&dpds, &cells);

	CUDA_Check( hipStreamSynchronize(defStream) );

	const float dt = 0.002;
	const int niters = 300;

	const float kBT = 1.0;
	const float gammadpd = 20;
	const float sigmadpd = sqrt(2 * gammadpd * kBT);
	const float sigma_dt = sigmadpd / sqrt(dt);
	const float adpd = 50;

	auto inter = [=] (InteractionType type, ParticleVector* pv1, ParticleVector* pv2, CellList* cl, const float t, hipStream_t stream) {
		interactionDPD(type, pv1, pv2, cl, t, stream, adpd, gammadpd, sigma_dt, 1, rc);
	};

	printf("GPU execution\n");

	Timer tm;
	tm.start();

	HostBuffer<Force> frcs;

	PinnedBuffer<double> energy(1), momentum(3);
	energy.pushStream(defStream);
	momentum.pushStream(defStream);


	for (int i=0; i<niters; i++)
	{
		energy.clear();
		momentum.clear();

		totalMomentumEnergy<<< getNblocks(dpds.local()->size(), 128), 128, 0, defStream >>> (
				(float4*)dpds.local()->coosvels.devPtr(), dpds.mass, dpds.local()->size(), momentum.devPtr(), energy.devPtr());

		totalMomentumEnergy<<< getNblocks(obj.local()->size(), 128), 128, 0, defStream >>> (
				(float4*)obj.local()->coosvels.devPtr(), 1.0f, obj.local()->size(), momentum.devPtr(), energy.devPtr());

//		momentum.downloadFromDevice();
//		energy.downloadFromDevice(true);
//
//		printf("Iteration %d, energy %f, momentum  %f %f %f\n", i, energy[0], momentum[0], momentum[1], momentum[2]);
		cells.build();
		CUDA_Check( hipStreamSynchronize(defStream) );

		dpds.local()->forces.clear();
		obj.local()->forces.clear();

		halo.init();
		inter(InteractionType::Regular, &dpds, &dpds, &cells, dt*i, defStream);
		inter(InteractionType::Regular, &dpds, &obj,  &cells, dt*i, defStream);
		halo.finalize();

		inter(InteractionType::Halo, &dpds, &dpds, &cells, dt*i, defStream);

//		dpds.local()->coosvels.downloadFromDevice();
//		CUDA_Check( hipStreamSynchronize(defStream) );
//		frcs.copy(dpds.local()->forces, defStream);
//		for (int j=0; j<dpds.local()->size(); j++)
//			if (dpds.local()->coosvels[j].i1 == 42)
//			{
//				printf("%d :  %f %f %f\n", dpds.local()->coosvels[j].i1, frcs[j].f.x, frcs[j].f.y, frcs[j].f.z);
//				break;
//			}

		integrateNoFlow(&dpds, dt, defStream);
		integrateRigid(&obj, dt, defStream, make_float3(0));

		obj.local()->findExtentAndCOM(defStream);
		bounceFromRigidEllipsoid(&dpds, &cells, &obj, dt, true, defStream);

		CUDA_Check( hipStreamSynchronize(defStream) );

		redist.redistribute();
	}

	double elapsed = tm.elapsed() * 1e-9;

	printf("Finished in %f s, 1 step took %f ms\n", elapsed, elapsed / niters * 1000.0);

	return 0;
}
