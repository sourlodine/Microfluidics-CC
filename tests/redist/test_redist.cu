#include "hip/hip_runtime.h"
// Yo ho ho ho
#define private public
#define protected public

#include <core/pvs/particle_vector.h>
#include <core/celllist.h>
#include <core/mpi/api.h>
#include <core/logger.h>

#include <core/containers.h>

#include <core/initial_conditions/uniform_ic.h>

Logger logger;

Particle addShift(Particle p, float a, float b, float c)
{
	Particle res = p;
	res.r.x += a;
	res.r.y += b;
	res.r.z += c;

	return res;
}

int main(int argc, char ** argv)
{
	// Init

	int nranks, rank;
	int ranks[] = {1, 1, 1};
	int periods[] = {1, 1, 1};
	MPI_Comm cartComm;

	MPI_Init(&argc, &argv);
	logger.init(MPI_COMM_WORLD, "redist.log", 9);

	MPI_Check( MPI_Comm_size(MPI_COMM_WORLD, &nranks) );
	MPI_Check( MPI_Comm_rank(MPI_COMM_WORLD, &rank) );
	MPI_Check( MPI_Cart_create(MPI_COMM_WORLD, 3, ranks, periods, 0, &cartComm) );

	float3 length{80,70,55};
	float3 domainStart = -length / 2.0f;
	const float rc = 1.0f;
	ParticleVector dpds("dpd", 1.0f);
	CellList* cells = new PrimaryCellList (&dpds, rc, length);

	InitialConditions* ic = new UniformIC(8.0);
	ic->exec(MPI_COMM_WORLD, &dpds, {0,0,0}, length, 0);

	const int initialNP = dpds.local()->size();
	HostBuffer<Particle> host(dpds.local()->size());
	const float dt = 0.1;
	for (int i=0; i<dpds.local()->size(); i++)
	{
		dpds.local()->coosvels[i].u.x = 10*(drand48() - 0.5);
		dpds.local()->coosvels[i].u.y = 10*(drand48() - 0.5);
		dpds.local()->coosvels[i].u.z = 10*(drand48() - 0.5);

		dpds.local()->coosvels[i].r += dt * dpds.local()->coosvels[i].u;

		host[i] = dpds.local()->coosvels[i];
	}

	dpds.local()->coosvels.uploadToDevice(0);

	ParticleRedistributor redist(cartComm);
	cells->build(0);
	redist.attach(&dpds, cells);

	for (int i=0; i<1; i++)
	{
		redist.init(0);
		hipStreamSynchronize(0);
		redist.finalize(0);

		dpds.redistValid = false;
	}

	std::vector<Particle> bufs[27];

	for (int i=0; i<initialNP; i++)
	{
		Particle& p = host[i];

		int3 code = cells->getCellIdAlongAxes<false>(p.r);
		int cx = code.x,  cy = code.y,  cz = code.z;
		auto ncells = cells->ncells;

		// 8
		if (cx == -1         && cy == -1         && cz == -1)         { bufs[ (0*3 + 0)*3 + 0 ].push_back(addShift(p,  length.x,  length.y,  length.z)); continue; }
		if (cx == -1         && cy == -1         && cz == ncells.z  ) { bufs[ (2*3 + 0)*3 + 0 ].push_back(addShift(p,  length.x,  length.y, -length.z)); continue; }
		if (cx == -1         && cy == ncells.y   && cz == -1)         { bufs[ (0*3 + 2)*3 + 0 ].push_back(addShift(p,  length.x, -length.y,  length.z)); continue; }
		if (cx == -1         && cy == ncells.y   && cz == ncells.z  ) { bufs[ (2*3 + 2)*3 + 0 ].push_back(addShift(p,  length.x, -length.y, -length.z)); continue; }
		if (cx == ncells.x   && cy == -1         && cz == -1)         { bufs[ (0*3 + 0)*3 + 2 ].push_back(addShift(p, -length.x,  length.y,  length.z)); continue; }
		if (cx == ncells.x   && cy == -1         && cz == ncells.z  ) { bufs[ (2*3 + 0)*3 + 2 ].push_back(addShift(p, -length.x,  length.y, -length.z)); continue; }
		if (cx == ncells.x   && cy == ncells.y   && cz == -1)         { bufs[ (0*3 + 2)*3 + 2 ].push_back(addShift(p, -length.x, -length.y,  length.z)); continue; }
		if (cx == ncells.x   && cy == ncells.y   && cz == ncells.z  ) { bufs[ (2*3 + 2)*3 + 2 ].push_back(addShift(p, -length.x, -length.y, -length.z)); continue; }

		// 12
		if (cx == -1         && cy == -1)         { bufs[ (1*3 + 0)*3 + 0 ].push_back(addShift(p,  length.x,  length.y,         0)); continue; }
		if (cx == ncells.x   && cy == -1)         { bufs[ (1*3 + 0)*3 + 2 ].push_back(addShift(p, -length.x,  length.y,         0)); continue; }
		if (cx == -1         && cy == ncells.y  ) { bufs[ (1*3 + 2)*3 + 0 ].push_back(addShift(p,  length.x, -length.y,         0)); continue; }
		if (cx == ncells.x   && cy == ncells.y  ) { bufs[ (1*3 + 2)*3 + 2 ].push_back(addShift(p, -length.x, -length.y,         0)); continue; }

		if (cy == -1         && cz == -1)         { bufs[ (0*3 + 0)*3 + 1 ].push_back(addShift(p,         0,  length.y,  length.z)); continue; }
		if (cy == ncells.y   && cz == -1)         { bufs[ (0*3 + 2)*3 + 1 ].push_back(addShift(p,         0, -length.y,  length.z)); continue; }
		if (cy == -1         && cz == ncells.z  ) { bufs[ (2*3 + 0)*3 + 1 ].push_back(addShift(p,         0,  length.y, -length.z)); continue; }
		if (cy == ncells.y   && cz == ncells.z  ) { bufs[ (2*3 + 2)*3 + 1 ].push_back(addShift(p,         0, -length.y, -length.z)); continue; }


		if (cz == -1         && cx == -1)         { bufs[ (0*3 + 1)*3 + 0 ].push_back(addShift(p,  length.x,         0,  length.z)); continue; }
		if (cz == ncells.z   && cx == -1)         { bufs[ (2*3 + 1)*3 + 0 ].push_back(addShift(p,  length.x,         0, -length.z)); continue; }
		if (cz == -1         && cx == ncells.x  ) { bufs[ (0*3 + 1)*3 + 2 ].push_back(addShift(p, -length.x,         0,  length.z)); continue; }
		if (cz == ncells.z   && cx == ncells.x  ) { bufs[ (2*3 + 1)*3 + 2 ].push_back(addShift(p, -length.x,         0, -length.z)); continue; }

		// 6
		if (cx == -1)         { bufs[ (1*3 + 1)*3 + 0 ].push_back(addShift(p,  length.x,         0,         0)); continue; }
		if (cx == ncells.x  ) { bufs[ (1*3 + 1)*3 + 2 ].push_back(addShift(p, -length.x,         0,         0)); continue; }
		if (cy == -1)         { bufs[ (1*3 + 0)*3 + 1 ].push_back(addShift(p,         0,  length.y,         0)); continue; }
		if (cy == ncells.y  ) { bufs[ (1*3 + 2)*3 + 1 ].push_back(addShift(p,         0, -length.y,         0)); continue; }
		if (cz == -1)         { bufs[ (0*3 + 1)*3 + 1 ].push_back(addShift(p,         0,         0,  length.z)); continue; }
		if (cz == ncells.z  ) { bufs[ (2*3 + 1)*3 + 1 ].push_back(addShift(p,         0,         0, -length.z)); continue; }
	}

	for (int i = 0; i<27; i++)
	{
		//if (bufs[i].size() != redist.helpers[0]->sendSizes[i])
			printf("%2d-th redist differs in size: %5d, expected %5d\n", i, redist.helpers[0]->sendSizes[i], (int)bufs[i].size());

		std::vector<Particle> got, reference;

		auto cmp = [] (Particle a, Particle b) {
			if (a.i1 < b.i1) return true;
			if (a.i1 > b.i1) return false;

			if (a.r.x > b.r.x + 1e-6) return true;
			if (a.r.y > b.r.y + 1e-6) return true;
			if (a.r.z > b.r.z + 1e-6) return true;

			return false;
		};

		auto& helper = redist.helpers[0];
		std::sort(bufs[i].begin(), bufs[i].end(), cmp);
		std::sort((Particle*)helper->sendBuf.hostPtr() + helper->sendOffsets[i],
				  (Particle*)helper->sendBuf.hostPtr() + helper->sendOffsets[i+1], cmp);

		std::set_difference(bufs[i].begin(), bufs[i].end(),
				(Particle*)helper->sendBuf.hostPtr() + helper->sendOffsets[i], (Particle*)helper->sendBuf.hostPtr() + helper->sendOffsets[i+1],
				std::inserter(reference, reference.begin()), cmp);

		std::set_difference(
				(Particle*)helper->sendBuf.hostPtr() + helper->sendOffsets[i], (Particle*)helper->sendBuf.hostPtr() + helper->sendOffsets[i+1],
				bufs[i].begin(), bufs[i].end(),
				std::inserter(got, got.begin()), cmp);

		for (int pid = 0; pid < std::max(reference.size(), got.size()); pid++)
		{
			if (pid < got.size())
				printf("redist %2d:  %5d [%12.5e %12.5e %12.5e], ",
					i, got[pid].i1, got[pid].r.x, got[pid].r.y, got[pid].r.z);
			else
				printf("redist none,                                           ");

			printf(" expected ");

			if (pid < reference.size())
				printf("%5d [%12.5e %12.5e %12.5e]\n",
						reference[pid].i1, reference[pid].r.x, reference[pid].r.y, reference[pid].r.z);
			else
				printf("none\n");
		}
	}

	return 0;
}
